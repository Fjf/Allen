#include "CpuPrefixSum.cuh"

void cpu_prefix_sum_impl(
  uint* host_prefix_sum_buffer,
  const size_t dev_prefix_sum_size,
  uint* host_total_sum_holder)
{
  // Do prefix sum on CPU
  const size_t number_of_elements = (dev_prefix_sum_size >> 2) - 1;
  uint temp = 0;
  uint temp_sum = 0;
  for (int i=0; i<number_of_elements; ++i) {
    temp_sum += host_prefix_sum_buffer[i];
    host_prefix_sum_buffer[i] = temp;
    temp = temp_sum;
  }
  host_prefix_sum_buffer[number_of_elements] = temp;

  if (host_total_sum_holder != nullptr) {
    host_total_sum_holder[0] = host_prefix_sum_buffer[number_of_elements];
  }
}

void cpu_prefix_sum(
  uint* host_prefix_sum_buffer,
  size_t& host_allocated_prefix_sum_space,
  uint* dev_prefix_sum_offset,
  const size_t dev_prefix_sum_size,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event,
  uint* host_total_sum_holder)
{
  // Reallocate if insufficient space on host buffer
  if ((dev_prefix_sum_size >> 2) > host_allocated_prefix_sum_space) {
    host_allocated_prefix_sum_space = (dev_prefix_sum_size >> 2) * 1.2f;
    cudaCheck(hipHostFree(host_prefix_sum_buffer));
    cudaCheck(hipHostMalloc((void**) &host_prefix_sum_buffer, host_allocated_prefix_sum_space * sizeof(uint)));
  }

  cudaCheck(hipMemcpyAsync(
    host_prefix_sum_buffer,
    dev_prefix_sum_offset,
    dev_prefix_sum_size,
    hipMemcpyDeviceToHost,
    hip_stream));

  hipEventRecord(cuda_generic_event, hip_stream);
  hipEventSynchronize(cuda_generic_event);
  
  cpu_prefix_sum_impl(
    host_prefix_sum_buffer,
    dev_prefix_sum_size,
    host_total_sum_holder);
  
  cudaCheck(hipMemcpyAsync(
    dev_prefix_sum_offset,
    host_prefix_sum_buffer,
    dev_prefix_sum_size,
    hipMemcpyHostToDevice,
    hip_stream));
}