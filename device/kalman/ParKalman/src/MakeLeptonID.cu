#include "hip/hip_runtime.h"
/*****************************************************************************\
* (c) Copyright 2022 CERN for the benefit of the LHCb Collaboration           *
*                                                                             *
* This software is distributed under the terms of the Apache License          *
* version 2 (Apache-2.0), copied verbatim in the file "COPYING".              *
*                                                                             *
* In applying this licence, CERN does not waive the privileges and immunities *
* granted to it by virtue of its status as an Intergovernmental Organization  *
* or submit itself to any jurisdiction.                                       *
\*****************************************************************************/
#include "MakeLeptonID.cuh"

INSTANTIATE_ALGORITHM(make_lepton_id::make_lepton_id_t)

void make_lepton_id::make_lepton_id_t::set_arguments_size(
  ArgumentReferences<Parameters> arguments,
  const RuntimeOptions&,
  const Constants&,
  const HostBuffers&) const
{
  auto n_scifi_tracks = first<host_number_of_scifi_tracks_t>(arguments);
  set_size<dev_lepton_id_t>(arguments, n_scifi_tracks);
}

void make_lepton_id::make_lepton_id_t::operator()(
  const ArgumentReferences<Parameters>& arguments,
  const RuntimeOptions& runtime_options,
  const Constants&,
  HostBuffers& host_buffers,
  const Allen::Context& context) const
{
  global_function(make_lepton_id)(dim3(size<dev_event_list_t>(arguments)), property<block_dim_t>(), context)(arguments);
}

__global__ void make_lepton_id::make_lepton_id(make_lepton_id::Parameters parameters)
{
  const unsigned event_number = parameters.dev_event_list[blockIdx.x];
  const auto scifi_tracks = parameters.dev_scifi_tracks_view[event_number];
  const unsigned offset = scifi_tracks.offset();
  const unsigned n_tracks = scifi_tracks.size();
  const uint8_t* event_is_muon = parameters.dev_is_muon + offset;
  const uint8_t* event_is_electron = parameters.dev_is_electron + offset;
  uint8_t* event_lepton_id = parameters.dev_lepton_id + offset;
  for (unsigned i_track = threadIdx.x; i_track < n_tracks; i_track += blockDim.x) {
    event_lepton_id[i_track] = event_is_muon[i_track] | event_is_electron[i_track];
  }
}