#include "hip/hip_runtime.h"
/*****************************************************************************\
* (c) Copyright 2022 CERN for the benefit of the LHCb Collaboration           *
*                                                                             *
* This software is distributed under the terms of the Apache License          *
* version 2 (Apache-2.0), copied verbatim in the file "COPYING".              *
*                                                                             *
* In applying this licence, CERN does not waive the privileges and immunities *
* granted to it by virtue of its status as an Intergovernmental Organization  *
* or submit itself to any jurisdiction.                                       *
\*****************************************************************************/
#include "MakeLeptonID.cuh"

INSTANTIATE_ALGORITHM(make_lepton_id::make_lepton_id_t)

void make_lepton_id::make_lepton_id_t::set_arguments_size(
  ArgumentReferences<Parameters> arguments,
  const RuntimeOptions&,
  const Constants&,
  const HostBuffers&) const
{
  auto n_scifi_tracks = first<host_number_of_scifi_tracks_t>(arguments);
  set_size<dev_lepton_id_t>(arguments, n_scifi_tracks);
}

void make_lepton_id::make_lepton_id_t::operator()(
  const ArgumentReferences<Parameters>& arguments,
  const RuntimeOptions&,
  const Constants&,
  HostBuffers&,
  const Allen::Context& context) const
{
  global_function(make_lepton_id)(dim3(size<dev_event_list_t>(arguments)), property<block_dim_t>(), context)(arguments);
}

__global__ void make_lepton_id::make_lepton_id(make_lepton_id::Parameters parameters)
{
  const unsigned event_number = parameters.dev_event_list[blockIdx.x];
  // Long tracks.
  const auto long_tracks = parameters.dev_long_tracks_view->container(event_number);
  const unsigned n_tracks = long_tracks.size();
  const unsigned offset = long_tracks.offset();
  const auto* event_is_muon = parameters.dev_is_muon + offset;
  const auto* event_is_electron = parameters.dev_is_electron + offset;
  auto* event_lepton_id = parameters.dev_lepton_id + offset;
  for (unsigned i_track = threadIdx.x; i_track < n_tracks; i_track += blockDim.x) {
    event_lepton_id[i_track] = event_is_muon[i_track] | (event_is_electron[i_track] << 1);
  }
}