#include "hip/hip_runtime.h"
/*****************************************************************************\
* (c) Copyright 2022 CERN for the benefit of the LHCb Collaboration           *
*                                                                             *
* This software is distributed under the terms of the Apache License          *
* version 2 (Apache-2.0), copied verbatim in the file "COPYING".              *
*                                                                             *
* In applying this licence, CERN does not waive the privileges and immunities *
* granted to it by virtue of its status as an Intergovernmental Organization  *
* or submit itself to any jurisdiction.                                       *
\*****************************************************************************/
#include "MakeLongTrackParticles.cuh"

INSTANTIATE_ALGORITHM(make_long_track_particles::make_long_track_particles_t)

void make_long_track_particles::make_long_track_particles_t::set_arguments_size(
  ArgumentReferences<Parameters> arguments,
  const RuntimeOptions&,
  const Constants&,
  const HostBuffers&) const
{
  auto n_scifi_tracks = first<host_number_of_reconstructed_scifi_tracks_t>(arguments);
  set_size<dev_long_track_particle_view_t>(arguments, n_scifi_tracks);
  set_size<dev_long_track_particles_view_t>(arguments, first<host_number_of_events_t>(arguments));
  set_size<dev_multi_event_basic_particles_view_t>(arguments, 1);
  set_size<dev_multi_event_container_basic_particles_t>(arguments, 1);
}

void make_long_track_particles::make_long_track_particles_t::operator()(
  const ArgumentReferences<Parameters>& arguments,
  const RuntimeOptions&,
  const Constants&,
  HostBuffers&,
  const Allen::Context& context) const
{
  initialize<dev_long_track_particle_view_t>(arguments, 0, context);

  global_function(make_particles)(dim3(first<host_number_of_events_t>(arguments)), property<block_dim_t>(), context)(
    arguments);
}

void __global__ make_long_track_particles::make_particles(make_long_track_particles::Parameters parameters)
{
  const unsigned number_of_events = parameters.dev_number_of_events[0];
  const unsigned event_number = blockIdx.x;
  const auto* mec =
    static_cast<const Allen::Views::Physics::MultiEventLongTracks*>(parameters.dev_multi_event_long_tracks[0]);
  const auto event_long_tracks = mec->container(event_number);
  const unsigned offset = event_long_tracks.offset();
  const unsigned number_of_tracks = event_long_tracks.size();
  const auto pv_table = parameters.dev_kalman_pv_tables[event_number];

  for (unsigned i = threadIdx.x; i < number_of_tracks; i++) {
    const auto* long_track = &(event_long_tracks.track(i));
    const int i_pv = pv_table.pv(i);
    new (parameters.dev_long_track_particle_view + offset + i) Allen::Views::Physics::BasicParticle {
      long_track,
      parameters.dev_kalman_states_view + event_number,
      i_pv ? parameters.dev_multi_final_vertices + PV::max_number_vertices * event_number + pv_table.pv(i) : nullptr,
      i,
      parameters.dev_lepton_id[offset + i]};
  }

  if (threadIdx.x == 0) {
    new (parameters.dev_long_track_particles_view + event_number) Allen::Views::Physics::BasicParticles {
      parameters.dev_long_track_particle_view, parameters.dev_atomics_scifi, event_number};
  }

  if (blockIdx.x == 0 && threadIdx.x == 0) {
    new (parameters.dev_multi_event_basic_particles_view)
      Allen::Views::Physics::MultiEventBasicParticles {parameters.dev_long_track_particles_view, number_of_events};
    parameters.dev_multi_event_container_basic_particles[0] = parameters.dev_multi_event_basic_particles_view;
  }
}