#include "hip/hip_runtime.h"
/*****************************************************************************\
* (c) Copyright 2021 CERN for the benefit of the LHCb Collaboration           *
\*****************************************************************************/
#include "DecodeRetinaClusters.cuh"
#include "LHCbID.h"

void decode_retinaclusters::cluster_container_checks::operator()(
  const ArgumentReferences<Parameters>& arguments,
  const RuntimeOptions&,
  const Constants&,
  const Allen::Context& context) const
{
  constexpr float velo_cluster_min_x = -100.f;
  constexpr float velo_cluster_max_x = 100.f;
  constexpr float velo_cluster_min_y = -100.f;
  constexpr float velo_cluster_max_y = 100.f;

  const auto velo_cluster_container = make_host_buffer<Parameters::dev_velo_cluster_container_t>(arguments, context);
  const auto offsets_module_pair_cluster =
    make_host_buffer<Parameters::dev_offsets_module_pair_cluster_t>(arguments, context);
  const auto module_cluster_num = make_host_buffer<Parameters::dev_module_cluster_num_t>(arguments, context);

  // Condition to check
  bool x_greater_than_min_value = true;
  bool x_lower_than_max_value = true;
  bool y_greater_than_min_value = true;
  bool y_lower_than_max_value = true;
  bool valid_id_hit = true;
  bool unique_id_hit = true;

  const auto velo_container_view = Velo::ConstClusters {
    velo_cluster_container.data(), first<Parameters::host_total_number_of_velo_clusters_t>(arguments)};
  for (unsigned event_number = 0; event_number < first<Parameters::host_number_of_events_t>(arguments);
       ++event_number) {
    const auto event_number_of_hits =
      offsets_module_pair_cluster[(event_number + 1) * Velo::Constants::n_module_pairs] -
      offsets_module_pair_cluster[event_number * Velo::Constants::n_module_pairs];
    if (event_number_of_hits > 0) {
      for (unsigned i = 0; i < Velo::Constants::n_module_pairs; ++i) {
        const auto module_hit_start = offsets_module_pair_cluster[event_number * Velo::Constants::n_module_pairs + i];
        const auto module_hit_num = module_cluster_num[event_number * Velo::Constants::n_module_pairs + i];

        if (module_hit_num > 0) {
          auto previous_hit_phi = velo_container_view.phi(module_hit_start);
          auto previous_hit_id = velo_container_view.id(module_hit_start);
          auto previous_hit_x = velo_container_view.x(module_hit_start);
          auto previous_hit_y = velo_container_view.y(module_hit_start);
          for (unsigned hit_number = 0; hit_number < module_hit_num; ++hit_number) {
            const auto hit_index = module_hit_start + hit_number;

            valid_id_hit &= lhcb_id::is_velo(velo_container_view.id(hit_index));

            if (hit_number != 0) {
              unique_id_hit &= velo_container_view.id(hit_index) != previous_hit_id;
            }

            if (velo_container_view.id(hit_index) == previous_hit_id and hit_number != 0) {
              std::cout << "Phi " << velo_container_view.phi(hit_index) << " " << previous_hit_phi << std::endl;
              std::cout << "ID " << velo_container_view.id(hit_index) << " " << previous_hit_id << std::endl;
              std::cout << "X " << velo_container_view.x(hit_index) << " " << previous_hit_x << std::endl;
              std::cout << "Y " << velo_container_view.y(hit_index) << " " << previous_hit_y << std::endl;
            }

            x_greater_than_min_value &= velo_container_view.x(hit_index) > velo_cluster_min_x;
            x_lower_than_max_value &= velo_container_view.x(hit_index) < velo_cluster_max_x;
            y_greater_than_min_value &= velo_container_view.y(hit_index) > velo_cluster_min_y;
            y_lower_than_max_value &= velo_container_view.y(hit_index) < velo_cluster_max_y;

            previous_hit_phi = velo_container_view.phi(hit_index);
            previous_hit_id = velo_container_view.id(hit_index);
            previous_hit_x = velo_container_view.x(hit_index);
            previous_hit_y = velo_container_view.y(hit_index);
          }
        }
      }
    }
  }

  require(x_greater_than_min_value, "Require that x be greater than min value");
  require(x_lower_than_max_value, "Require that x be lower than max value");
  require(y_greater_than_min_value, "Require that y be greater than min value");
  require(y_lower_than_max_value, "Require that y be lower than max value");
  require(valid_id_hit, "Require that every hit id is valid");
  require(unique_id_hit, "Require that every hit has unique id");
}
