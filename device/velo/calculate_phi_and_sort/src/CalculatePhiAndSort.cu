#include "hip/hip_runtime.h"
/*****************************************************************************\
* (c) Copyright 2018-2020 CERN for the benefit of the LHCb Collaboration      *
\*****************************************************************************/
#include "VeloDefinitions.cuh"
#include "CalculatePhiAndSort.cuh"
#include "VeloTools.cuh"
#include "Vector.h"
#include <numeric>
#include <algorithm>

using namespace Allen::device;

void velo_calculate_phi_and_sort::velo_calculate_phi_and_sort_t::set_arguments_size(
  ArgumentReferences<Parameters> arguments,
  const RuntimeOptions&,
  const Constants&,
  const HostBuffers&) const
{
  set_size<dev_sorted_velo_cluster_container_t>(arguments, size<dev_velo_cluster_container_t>(arguments));
  set_size<dev_hit_permutation_t>(arguments, first<host_total_number_of_velo_clusters_t>(arguments));
  set_size<dev_hit_phi_t>(arguments, first<host_total_number_of_velo_clusters_t>(arguments));
}

void velo_calculate_phi_and_sort::velo_calculate_phi_and_sort_t::operator()(
  const ArgumentReferences<Parameters>& arguments,
  const RuntimeOptions&,
  const Constants&,
  HostBuffers&,
  hipStream_t& hip_stream,
  hipEvent_t&) const
{
  initialize<dev_hit_permutation_t>(arguments, 0, hip_stream);

  global_function(velo_calculate_phi_and_sort)(
    dim3(first<host_number_of_selected_events_t>(arguments)), property<block_dim_t>(), hip_stream)(arguments);

  // printf("After velo_calculate_phi_and_sort:\n");
  // print_velo_clusters<dev_sorted_velo_cluster_container_t,
  //   dev_offsets_estimated_input_size_t,
  //   dev_module_cluster_num_t,
  //   host_total_number_of_velo_clusters_t>(arguments);
}

/**
 * @brief Track forwarding algorithm based on triplet finding
 */
__global__ void velo_calculate_phi_and_sort::velo_calculate_phi_and_sort(
  velo_calculate_phi_and_sort::Parameters parameters)
{
  __shared__ int16_t shared_hit_phis[Velo::Constants::max_numhits_in_module_pair];

  /* Data initialization */
  // Each event is treated with two blocks, one for each side.
  const unsigned event_number = blockIdx.x;
  const unsigned number_of_events = gridDim.x;

  // Pointers to data within the event
  const unsigned total_estimated_number_of_clusters =
    parameters.dev_offsets_estimated_input_size[Velo::Constants::n_module_pairs * number_of_events];
  const unsigned* module_pair_hit_start =
    parameters.dev_offsets_estimated_input_size + event_number * Velo::Constants::n_module_pairs;
  const unsigned* module_pair_hit_num =
    parameters.dev_module_cluster_num + event_number * Velo::Constants::n_module_pairs;

  const auto velo_cluster_container =
    Velo::ConstClusters {parameters.dev_velo_cluster_container, total_estimated_number_of_clusters};
  auto velo_sorted_cluster_container =
    Velo::Clusters {parameters.dev_sorted_velo_cluster_container, total_estimated_number_of_clusters};

  const unsigned event_hit_start = module_pair_hit_start[0];
  const unsigned event_number_of_hits = module_pair_hit_start[Velo::Constants::n_module_pairs] - event_hit_start;

  // Calculate phi and populate hit_permutations
  dispatch<target::Default, target::CPU>(calculate_phi, calculate_phi_vectorized)(
    shared_hit_phis,
    module_pair_hit_start,
    module_pair_hit_num,
    velo_cluster_container,
    parameters.dev_hit_phi,
    parameters.dev_hit_permutation);

  // Due to phi RAW
  barrier();

  // Sort by phi
  sort_by_phi(
    event_hit_start,
    event_number_of_hits,
    velo_cluster_container,
    velo_sorted_cluster_container,
    parameters.dev_hit_permutation);
}

/**
 * @brief Calculates a phi side
 */
__device__ void velo_calculate_phi_and_sort::calculate_phi(
  int16_t* shared_hit_phis,
  const unsigned* module_pair_hit_start,
  const unsigned* module_pair_hit_num,
  Velo::ConstClusters& velo_cluster_container,
  int16_t* hit_Phis,
  unsigned* hit_permutations)
{
  for (unsigned module_pair = 0; module_pair < Velo::Constants::n_module_pairs; ++module_pair) {
    const auto hit_start = module_pair_hit_start[module_pair];
    const auto hit_num = module_pair_hit_num[module_pair];

    assert(hit_num < Velo::Constants::max_numhits_in_module_pair);

    // Calculate phis
    for (unsigned hit_rel_id = local_id<0>(); hit_rel_id < hit_num; hit_rel_id += local_size<0>()) {
      const auto hit_index = hit_start + hit_rel_id;
      const auto hit_phi_int = hit_phi_16(velo_cluster_container.x(hit_index), velo_cluster_container.y(hit_index));
      shared_hit_phis[hit_rel_id] = hit_phi_int;
    }

    // shared_hit_phis
    barrier();

    // Find the permutations given the phis in shared_hit_phis
    for (unsigned hit_rel_id = local_id<0>(); hit_rel_id < hit_num; hit_rel_id += local_size<0>()) {
      const auto hit_index = hit_start + hit_rel_id;
      const auto phi = shared_hit_phis[hit_rel_id];

      // Find out local position
      unsigned position = 0;
      for (unsigned j = 0; j < hit_num; ++j) {
        const auto other_phi = shared_hit_phis[j];
        // Stable sorting
        position += phi > other_phi || (phi == other_phi && hit_rel_id > j);
      }
      assert(position < Velo::Constants::max_numhits_in_module_pair);

      // Store it in hit permutations and in hit_Phis, already ordered
      const auto global_position = hit_start + position;
      hit_permutations[global_position] = hit_index;
      hit_Phis[global_position] = phi;
    }

    // shared_hit_phis
    barrier();
  }
}

#if defined(TARGET_DEVICE_CPU)
/**
 * @brief Calculates a phi side
 */
__device__ void velo_calculate_phi_and_sort::calculate_phi_vectorized(
  int16_t* shared_hit_phis,
  const unsigned* module_pair_hit_start,
  const unsigned* module_pair_hit_num,
  Velo::ConstClusters& velo_cluster_container,
  int16_t* hit_Phis,
  unsigned* hit_permutations)
{
  for (unsigned module_pair = 0; module_pair < Velo::Constants::n_module_pairs; ++module_pair) {
    const auto hit_start = module_pair_hit_start[module_pair];
    const auto hit_num = module_pair_hit_num[module_pair];

    assert(hit_num < Velo::Constants::max_numhits_in_module_pair);

    // Calculate phis
    for (unsigned hit_rel_id = local_id<0>(); hit_rel_id < hit_num; hit_rel_id += local_size<0>() * vector_length()) {
      if (hit_rel_id + vector_length() <= hit_num) {
        // Do most iterations vectorized
        Vector<float> xs;
        Vector<float> ys;

        for (unsigned i = 0; i < vector_length(); ++i) {
          const auto hit_rel_vector_id = hit_rel_id + i;
          const auto hit_index = hit_start + hit_rel_vector_id;
          xs[i] = velo_cluster_container.x(hit_index);
          ys[i] = velo_cluster_container.y(hit_index);
        }

        const auto atan_value = fast_atan2f(ys, xs);
        const auto float_value = (Velo::Tools::cudart_pi_f_float + atan_value) * Velo::Tools::convert_factor;

        // Cast to uint16
        Vector<uint16_t> uint16_value;
        for (unsigned i = 0; i < vector_length(); ++i) {
          uint16_value[i] = static_cast<uint16_t>(float_value[i]);
        }

        uint16_value.storea(reinterpret_cast<uint16_t*>(shared_hit_phis + hit_rel_id));
      }
      else {
        // Last iterations sequentially
        for (unsigned i = hit_rel_id; i < hit_num; ++i) {
          const auto hit_index = hit_start + i;
          const auto hit_phi_int = hit_phi_16(velo_cluster_container.x(hit_index), velo_cluster_container.y(hit_index));
          shared_hit_phis[i] = hit_phi_int;
        }
      }
    }

    // Sort local_hit_permutations according to shared_hit_phis
    std::array<unsigned, Velo::Constants::max_numhits_in_module_pair> local_hit_permutations;
    std::iota(local_hit_permutations.begin(), local_hit_permutations.begin() + hit_num, 0);
    std::sort(
      local_hit_permutations.begin(),
      local_hit_permutations.begin() + hit_num,
      [&shared_hit_phis](const int a, const int b) {
        const auto phi = shared_hit_phis[a];
        const auto other_phi = shared_hit_phis[b];
        return phi < other_phi || (phi == other_phi && a < b);
      });

    // Populate sorted hits
    for (unsigned i = 0; i < hit_num; ++i) {
      hit_permutations[hit_start + i] = hit_start + local_hit_permutations[i];
      hit_Phis[hit_start + i] = shared_hit_phis[local_hit_permutations[i]];
    }
  }
}
#endif

/**
 * @brief Sorts all VELO decoded data by phi onto another container.
 */
__device__ void velo_calculate_phi_and_sort::sort_by_phi(
  const unsigned event_hit_start,
  const unsigned event_number_of_hits,
  Velo::ConstClusters& velo_cluster_container,
  Velo::Clusters& velo_sorted_cluster_container,
  unsigned* hit_permutations)
{
  for (unsigned i = local_id<0>(); i < event_number_of_hits; i += local_size<0>()) {
    const auto hit_index_global = hit_permutations[event_hit_start + i];
    velo_sorted_cluster_container.set_x(event_hit_start + i, velo_cluster_container.x(hit_index_global));
  }

  for (unsigned i = local_id<0>(); i < event_number_of_hits; i += local_size<0>()) {
    const auto hit_index_global = hit_permutations[event_hit_start + i];
    velo_sorted_cluster_container.set_y(event_hit_start + i, velo_cluster_container.y(hit_index_global));
  }

  for (unsigned i = local_id<0>(); i < event_number_of_hits; i += local_size<0>()) {
    const auto hit_index_global = hit_permutations[event_hit_start + i];
    velo_sorted_cluster_container.set_z(event_hit_start + i, velo_cluster_container.z(hit_index_global));
  }

  for (unsigned i = local_id<0>(); i < event_number_of_hits; i += local_size<0>()) {
    const auto hit_index_global = hit_permutations[event_hit_start + i];
    velo_sorted_cluster_container.set_id(event_hit_start + i, velo_cluster_container.id(hit_index_global));
  }
}