#include "hip/hip_runtime.h"
/*****************************************************************************\
* (c) Copyright 2018-2020 CERN for the benefit of the LHCb Collaboration      *
\*****************************************************************************/
#include "TrackForwarding.cuh"
#include "ClusteringDefinitions.cuh"
#include "SearchByTriplet.cuh"
#include "VeloTools.cuh"
#include <cstdio>

using namespace Velo::Tracking;

void velo_search_by_triplet::velo_search_by_triplet_t::set_arguments_size(
  ArgumentReferences<Parameters> arguments,
  const RuntimeOptions&,
  const Constants&,
  const HostBuffers&) const
{
  set_size<dev_tracks_t>(
    arguments, first<host_number_of_selected_events_t>(arguments) * Velo::Constants::max_tracks);
  set_size<dev_tracklets_t>(
    arguments, first<host_number_of_selected_events_t>(arguments) * Velo::Constants::max_tracks_to_follow);
  set_size<dev_tracks_to_follow_t>(
    arguments, first<host_number_of_selected_events_t>(arguments) * Velo::Constants::max_tracks_to_follow);
  set_size<dev_three_hit_tracks_t>(
    arguments, first<host_number_of_selected_events_t>(arguments) * Velo::Constants::max_three_hit_tracks);
  set_size<dev_hit_used_t>(arguments, first<host_total_number_of_velo_clusters_t>(arguments));
  set_size<dev_atomics_velo_t>(arguments, first<host_number_of_selected_events_t>(arguments) * Velo::num_atomics);
  set_size<dev_number_of_velo_tracks_t>(arguments, first<host_number_of_selected_events_t>(arguments));
  set_size<dev_rel_indices_t>(
    arguments, first<host_number_of_selected_events_t>(arguments) * Velo::Constants::max_numhits_in_module_pair);
}

void velo_search_by_triplet::velo_search_by_triplet_t::operator()(
  const ArgumentReferences<Parameters>& arguments,
  const RuntimeOptions&,
  const Constants& constants,
  HostBuffers&,
  hipStream_t& hip_stream,
  hipEvent_t&) const
{
  initialize<dev_atomics_velo_t>(arguments, 0, hip_stream);
  initialize<dev_hit_used_t>(arguments, 0, hip_stream);
  initialize<dev_number_of_velo_tracks_t>(arguments, 0, hip_stream);

  global_function(velo_search_by_triplet)(
    dim3(first<host_number_of_selected_events_t>(arguments)), dim3(property<block_dim_x_t>().get()), hip_stream)(
    arguments, constants.dev_velo_geometry);
}

/**
 * @brief Track forwarding algorithm based on triplet finding.
 *
 * @detail Search by triplet is a parallel local track forwarding algorithm, whose main building blocks are two steps:
 *         track seeding and track forwarding. These two steps are applied iteratively
 *         throughout the VELO detector. The forward end of the detector is used as the start of the search,
 *         and the detector is traversed in the backwards direction, in groups of two modules at a time:
 *
 *         i-3    i-2   [i-1   i   i+1]
 *                      =============== Track seeding of triplet of module pairs {i-1, i, i+1}
 *
 *         i-3   [i-2] [i-1   i   i+1]
 *               =====                  Track forwarding to module pair i-2
 *
 *         i-3   [i-2   i-1   i]  i+1
 *               ===============        Track seeding of triplet of module pairs {i-2, i-1, i}
 *
 *         [i-3] [i-2   i-1   i   i+1]
 *         =====                        Track forwarding to module pair i-3
 *
 *         [i-3   i-2   i-1]  i   i+1
 *         =================            Track seeding of triplet of module pairs {i-3, i-2, i-1}
 *
 *         * Track seeding: Triplets of hits in consecutive module pairs are sought.
 *         Incoming VELO cluster data is expected to be sorted by phi previously. This fact allows for several
 *         optimizations in the triplet seed search. First, the closest hit in the previous module pair is sought with
 *         a binary search. The closest n candidates in memory are found with a pendulum-like search (more details
 *         in track_seeding). The doublet is extrapolated to the third module, and a triplet is formed. Hits used
 *         to form triplets must be "not used".
 *
 *         * Track forwarding: Triplet track seeds and tracks with more than three hits are
 *         extended to modules by extrapolating the last two hits into the next layer and finding the
 *         best hits. Again, a binary search in phi is used to speed up the search. If hits are found,
 *         the track is extended and all hits found are marked as "used".
 *
 *         Both for track seeding and for track forwarding, a "max_scatter" function is used to determine the best hit.
 *         This function simply minimizes dx^2 + dy^2 in the detector plane.
 *
 *         The "hit used" array imposes a Read-After-Write dependency from every seeding stage to every forwarding
 *         stage, and a Write-After-Read dependency from every forwarding stage to every seeding stage. Hence, execution
 *         of these two stages is separated with control flow barriers.
 *
 *         For more details see:
 *         * https://ieeexplore.ieee.org/document/8778210
 */
__global__ void velo_search_by_triplet::velo_search_by_triplet(
  velo_search_by_triplet::Parameters parameters,
  const VeloGeometry* dev_velo_geometry)
{
  // Shared memory size is a constant, enough to fit information about three module pairs.
  __shared__ Velo::ModulePair module_pair_data[3];

  // Initialize event number and number of events based on kernel invoking parameters
  const unsigned event_number = blockIdx.x;
  const unsigned number_of_events = gridDim.x;

  // Pointers to data within the event
  const unsigned tracks_offset = event_number * Velo::Constants::max_tracks;
  const unsigned total_estimated_number_of_clusters =
    parameters.dev_offsets_estimated_input_size[Velo::Constants::n_module_pairs * number_of_events];
  const unsigned* module_hit_start =
    parameters.dev_offsets_estimated_input_size + event_number * Velo::Constants::n_module_pairs;
  const unsigned* module_hit_num = parameters.dev_module_cluster_num + event_number * Velo::Constants::n_module_pairs;
  const unsigned hit_offset = module_hit_start[0];

  const auto velo_cluster_container =
    Velo::ConstClusters {parameters.dev_sorted_velo_cluster_container, total_estimated_number_of_clusters, hit_offset};

  const auto hit_phi = parameters.dev_hit_phi + hit_offset;

  Velo::TrackHits* tracks = parameters.dev_tracks + tracks_offset;
  bool* hit_used = parameters.dev_hit_used + hit_offset;

  unsigned* tracks_to_follow = parameters.dev_tracks_to_follow + event_number * Velo::Constants::max_tracks_to_follow;
  Velo::TrackletHits* three_hit_tracks =
    parameters.dev_three_hit_tracks + event_number * Velo::Constants::max_three_hit_tracks;
  Velo::TrackletHits* tracklets = parameters.dev_tracklets + event_number * Velo::Constants::max_tracks_to_follow;
  unsigned short* h1_rel_indices = parameters.dev_rel_indices + event_number * Velo::Constants::max_numhits_in_module_pair;

  unsigned* dev_atomics_velo = parameters.dev_atomics_velo + blockIdx.x * Velo::num_atomics;
  const int16_t phi_tolerance = hit_phi_float_to_16(parameters.phi_tolerance);

  unsigned first_module_pair = Velo::Constants::n_module_pairs - 1;

  // Prepare the first seeding iteration
  // Load shared module information
  for (unsigned i = threadIdx.x; i < 3; i += blockDim.x) {
    const auto module_pair_number = first_module_pair - i;
    module_pair_data[i].hit_start = module_hit_start[module_pair_number] - hit_offset;
    module_pair_data[i].hit_num = module_hit_num[module_pair_number];
    module_pair_data[i].z[0] = dev_velo_geometry->module_zs[2 * module_pair_number];
    module_pair_data[i].z[1] = dev_velo_geometry->module_zs[2 * module_pair_number + 1];
  }

  // Due to shared module data initialization
  __syncthreads();

  // Do first track seeding
  track_seeding(
    velo_cluster_container,
    module_pair_data,
    hit_used,
    tracklets,
    tracks_to_follow,
    h1_rel_indices,
    dev_atomics_velo,
    parameters.max_scatter,
    hit_phi,
    phi_tolerance);

  // Prepare forwarding - seeding loop
  // For an explanation on ttf, see below
  unsigned last_ttf = 0;
  --first_module_pair;

  while (first_module_pair > 1) {
    // Due to WAR between track_seeding and population of shared memory.
    __syncthreads();

    // Iterate in modules
    // Load in shared
    for (int i = threadIdx.x; i < 3; i += blockDim.x) {
      const auto module_pair_number = first_module_pair - i;
      module_pair_data[i].hit_start = module_hit_start[module_pair_number] - hit_offset;
      module_pair_data[i].hit_num = module_hit_num[module_pair_number];
      module_pair_data[i].z[0] = dev_velo_geometry->module_zs[2 * module_pair_number];
      module_pair_data[i].z[1] = dev_velo_geometry->module_zs[2 * module_pair_number + 1];
    }

    // ttf stands for "tracks to forward"
    // The tracks to forward are stored in a circular buffer.
    const auto prev_ttf = last_ttf;
    last_ttf = dev_atomics_velo[atomics::tracks_to_follow];
    const auto diff_ttf = last_ttf - prev_ttf;

    // Reset local number of hits
    dev_atomics_velo[atomics::local_number_of_hits] = 0;

    // Due to module data loading
    __syncthreads();

    // Track Forwarding
    track_forwarding(
      velo_cluster_container,
      hit_phi,
      hit_used,
      module_pair_data,
      diff_ttf,
      tracks_to_follow,
      three_hit_tracks,
      prev_ttf,
      tracklets,
      tracks,
      dev_atomics_velo,
      parameters.dev_number_of_velo_tracks,
      phi_tolerance,
      parameters.max_scatter,
      parameters.max_skipped_modules);

    // Due to module data reading
    __syncthreads();

    // Seeding
    track_seeding(
      velo_cluster_container,
      module_pair_data,
      hit_used,
      tracklets,
      tracks_to_follow,
      h1_rel_indices,
      dev_atomics_velo,
      parameters.max_scatter,
      hit_phi,
      phi_tolerance);

    --first_module_pair;
  }

  // Due to last seeding
  __syncthreads();

  const auto prev_ttf = last_ttf;
  last_ttf = dev_atomics_velo[atomics::tracks_to_follow];
  const auto diff_ttf = last_ttf - prev_ttf;

  // Process the last bunch of track_to_follows
  for (unsigned ttf_element = threadIdx.x; ttf_element < diff_ttf; ttf_element += blockDim.x) {
    const auto full_track_number = tracks_to_follow[(prev_ttf + ttf_element) % Velo::Constants::max_tracks_to_follow];
    const bool track_flag = (full_track_number & bits::seed) == bits::seed;

    // Here we are only interested in three-hit tracks,
    // to mark them as "doubtful"
    if (track_flag) {
      const auto track_number = full_track_number & bits::track_number;
      const Velo::TrackHits* t = (Velo::TrackHits*) &(tracklets[track_number]);
      const auto three_hit_tracks_p = atomicAdd(dev_atomics_velo + atomics::number_of_three_hit_tracks, 1);
      three_hit_tracks[three_hit_tracks_p] = Velo::TrackletHits {t->hits[0], t->hits[1], t->hits[2]};
    }
  }
}

/**
 * @brief Search for compatible triplets in
 *        three neighbouring modules on one side
 */
__device__ void track_seeding(
  Velo::ConstClusters& velo_cluster_container,
  const Velo::ModulePair* module_pair_data,
  bool* hit_used,
  Velo::TrackletHits* tracklets,
  unsigned* tracks_to_follow,
  unsigned short* h1_indices,
  unsigned* dev_atomics_velo,
  const float max_scatter,
  const int16_t* hit_phi,
  const int16_t phi_tolerance)
{
  // Add to an array all non-used h1 hits
  for (unsigned h1_rel_index = threadIdx.x; h1_rel_index < module_pair_data[shared::current_module_pair].hit_num;
       h1_rel_index += blockDim.x) {
    const auto h1_index = module_pair_data[shared::current_module_pair].hit_start + h1_rel_index;
    if (!hit_used[h1_index]) {
      const auto current_hit = atomicAdd(dev_atomics_velo + atomics::local_number_of_hits, 1);
      h1_indices[current_hit] = h1_index;
    }
  }

  // Due to h1_indices
  __syncthreads();

  // Assign a h1 to each threadIdx.x
  const auto number_of_hits_h1 = dev_atomics_velo[atomics::local_number_of_hits];
  for (unsigned h1_rel_index = threadIdx.x; h1_rel_index < number_of_hits_h1; h1_rel_index += blockDim.x) {
    // The output we are searching for
    uint16_t best_h0 = 0;
    uint16_t best_h2 = 0;
    float best_fit = max_scatter;

    // Fetch h1
    const auto h1_index_total = h1_indices[h1_rel_index];
    const uint16_t h1_index = h1_index_total & bits::hit_number;
    const Velo::HitBase h1 {
      velo_cluster_container.x(h1_index), velo_cluster_container.y(h1_index), velo_cluster_container.z(h1_index)};
    const auto h1_phi = hit_phi[h1_index];

    // Get candidates on previous module
    unsigned best_h0s[number_of_h0_candidates];

    // Iterate over previous module until the first n candidates are found
    int phi_index = binary_search_leftmost(
      hit_phi + module_pair_data[shared::previous_module_pair].hit_start,
      module_pair_data[shared::previous_module_pair].hit_num,
      h1_phi);

    // Do a "pendulum search" to find the candidates, consisting in iterating in the following manner:
    // phi_index, phi_index + 1, phi_index - 1, phi_index + 2, ...
    int found_h0_candidates = 0;
    for (unsigned i = 0;
         i < module_pair_data[shared::previous_module_pair].hit_num && found_h0_candidates < number_of_h0_candidates;
         ++i) {
      // Note: By setting the sign to the oddity of i, the search behaviour is achieved.
      const auto sign = i & 0x01;
      const int index_diff = sign ? i : -i;
      phi_index += index_diff;

      const auto index_in_bounds =
        (phi_index < 0 ? phi_index + module_pair_data[shared::previous_module_pair].hit_num :
                         (phi_index >= static_cast<int>(module_pair_data[shared::previous_module_pair].hit_num) ?
                            phi_index - static_cast<int>(module_pair_data[shared::previous_module_pair].hit_num) :
                            phi_index));
      const auto h0_index = module_pair_data[shared::previous_module_pair].hit_start + index_in_bounds;

      // Discard the candidate if it is used
      if (!hit_used[h0_index]) {
        best_h0s[found_h0_candidates++] = h0_index;
      }
    }

    // Use the candidates found previously (best_h0s) to find the best triplet
    // Since data is sorted, search using a binary search
    for (int i = 0; i < found_h0_candidates; ++i) {
      const auto h0_index = best_h0s[i];
      const Velo::HitBase h0 {
        velo_cluster_container.x(h0_index), velo_cluster_container.y(h0_index), velo_cluster_container.z(h0_index)};

      const auto td = 1.0f / (h1.z - h0.z);
      const auto txn = (h1.x - h0.x);
      const auto tyn = (h1.y - h0.y);
      const auto tx = txn * td;
      const auto ty = tyn * td;

      // Get candidates by performing a binary search in expected phi
      const auto candidate_h2 = find_forward_candidate(
        module_pair_data[shared::next_module_pair],
        hit_phi,
        h0,
        tx,
        ty,
        module_pair_data[shared::next_module_pair].z[0] - module_pair_data[shared::previous_module_pair].z[0],
        phi_tolerance);

      // First candidate in the next module pair.
      // Since the buffer is circular, finding the container size means finding the first element.
      const auto candidate_h2_index =
        std::get<0>(candidate_h2) < static_cast<int>(module_pair_data[shared::next_module_pair].hit_num) ?
          std::get<0>(candidate_h2) :
          0;
      const auto extrapolated_phi = std::get<1>(candidate_h2);

      for (unsigned i = 0; i < module_pair_data[shared::next_module_pair].hit_num; ++i) {
        const auto index_in_bounds = (candidate_h2_index + i) % module_pair_data[shared::next_module_pair].hit_num;
        const auto h2_index = module_pair_data[shared::next_module_pair].hit_start + index_in_bounds;

        // Check the phi difference is within the tolerance with modulo arithmetic.
        const int16_t phi_diff = hit_phi[h2_index] - extrapolated_phi;
        const int16_t abs_phi_diff = phi_diff < 0 ? -phi_diff : phi_diff;
        if (abs_phi_diff > phi_tolerance) {
          break;
        }

        if (!hit_used[h2_index]) {
          const Velo::HitBase h2 {
            velo_cluster_container.x(h2_index), velo_cluster_container.y(h2_index), velo_cluster_container.z(h2_index)};

          const auto dz = h2.z - h0.z;
          const auto predx = h0.x + tx * dz;
          const auto predy = h0.y + ty * dz;
          const auto dx = predx - h2.x;
          const auto dy = predy - h2.y;

          // Scatter
          const auto scatter = dx * dx + dy * dy;

          // Keep the best one found
          if (scatter < best_fit) {
            best_fit = scatter;
            best_h0 = h0_index;
            best_h2 = h2_index;
          }
        }
      }
    }

    if (best_fit < max_scatter) {
      // Add the track to the container of seeds
      const auto trackP =
        atomicAdd(dev_atomics_velo + atomics::number_of_seeds, 1) % Velo::Constants::max_tracks_to_follow;
      tracklets[trackP] = Velo::TrackletHits {best_h0, h1_index, best_h2};

      // Add the tracks to the bag of tracks to_follow
      // Note: The first bit flag marks this is a tracklet (hitsNum == 3),
      // and hence it is stored in tracklets
      const auto ttfP =
        atomicAdd(dev_atomics_velo + atomics::tracks_to_follow, 1) % Velo::Constants::max_tracks_to_follow;
      tracks_to_follow[ttfP] = bits::seed | trackP;
    }
  }
}

/**
 * @brief Performs the track forwarding of forming tracks
 */
__device__ void track_forwarding(
  Velo::ConstClusters& velo_cluster_container,
  const int16_t* hit_phi,
  bool* hit_used,
  const Velo::ModulePair* module_pair_data,
  const unsigned diff_ttf,
  unsigned* tracks_to_follow,
  Velo::TrackletHits* three_hit_tracks,
  const unsigned prev_ttf,
  Velo::TrackletHits* tracklets,
  Velo::TrackHits* tracks,
  unsigned* dev_atomics_velo,
  unsigned* dev_number_of_velo_tracks,
  const int16_t phi_tolerance,
  const float max_scatter,
  const unsigned max_skipped_modules)
{
  // Assign a track to follow to each thread
  for (unsigned ttf_element = threadIdx.x; ttf_element < diff_ttf; ttf_element += blockDim.x) {
    const auto full_track_number = tracks_to_follow[(prev_ttf + ttf_element) % Velo::Constants::max_tracks_to_follow];
    const bool track_flag = (full_track_number & bits::seed) == bits::seed;
    const auto skipped_modules = (full_track_number & bits::skipped_modules) >> bits::skipped_module_position;
    auto track_number = full_track_number & bits::track_number;

    assert(
      track_flag ? track_number < Velo::Constants::max_tracks_to_follow : track_number < Velo::Constants::max_tracks);

    unsigned number_of_hits;
    Velo::TrackHits* t;

    if (track_flag) {
      t = (Velo::TrackHits*) &(tracklets[track_number]);
      number_of_hits = 3;
    }
    else {
      t = tracks + track_number;
      number_of_hits = t->hitsNum;
    }

    // Load last two hits in h0, h1
    const auto h0_num = t->hits[number_of_hits - 2];
    const auto h1_num = t->hits[number_of_hits - 1];

    const Velo::HitBase h0 {
      velo_cluster_container.x(h0_num), velo_cluster_container.y(h0_num), velo_cluster_container.z(h0_num)};
    const auto h0_module =
      ((velo_cluster_container.id(h0_num) & Allen::VPChannelID::sensorMask) >> Allen::VPChannelID::sensorBits) / 4;

    const Velo::HitBase h1 {
      velo_cluster_container.x(h1_num), velo_cluster_container.y(h1_num), velo_cluster_container.z(h1_num)};

    // Track forwarding over t, for all hits in the next module
    // Line calculations
    const auto td = 1.0f / (h1.z - h0.z);
    const auto txn = (h1.x - h0.x);
    const auto tyn = (h1.y - h0.y);
    const auto tx = txn * td;
    const auto ty = tyn * td;

    // Find the best candidate
    float best_fit = max_scatter;
    int best_h2 = -1;

    // Get candidates by performing a binary search in expected phi
    const auto candidate_h2 = find_forward_candidate(
      module_pair_data[shared::next_module_pair],
      hit_phi,
      h0,
      tx,
      ty,
      module_pair_data[shared::next_module_pair].z[h0_module % 2] - h0.z,
      phi_tolerance);

    // First candidate in the next module pair.
    // Since the buffer is circular, finding the container size means finding the first element.
    const auto candidate_h2_index =
      std::get<0>(candidate_h2) < static_cast<int>(module_pair_data[shared::next_module_pair].hit_num) ?
        std::get<0>(candidate_h2) :
        0;
    const auto extrapolated_phi = std::get<1>(candidate_h2);

    for (unsigned i = 0; i < module_pair_data[shared::next_module_pair].hit_num; ++i) {
      const auto index_in_bounds = (candidate_h2_index + i) % module_pair_data[shared::next_module_pair].hit_num;
      const auto h2_index = module_pair_data[shared::next_module_pair].hit_start + index_in_bounds;

      // Check the phi difference is within the tolerance with modulo arithmetic.
      const int16_t phi_diff = hit_phi[h2_index] - extrapolated_phi;
      const int16_t abs_phi_diff = phi_diff < 0 ? -phi_diff : phi_diff;
      if (abs_phi_diff > phi_tolerance) {
        break;
      }

      const Velo::HitBase h2 {
        velo_cluster_container.x(h2_index), velo_cluster_container.y(h2_index), velo_cluster_container.z(h2_index)};

      const auto dz = h2.z - h0.z;
      const auto predx = h0.x + tx * dz;
      const auto predy = h0.y + ty * dz;
      const auto dx = predx - h2.x;
      const auto dy = predy - h2.y;

      // Scatter
      const auto scatter = dx * dx + dy * dy;

      // We keep the best one found
      if (scatter < best_fit) {
        best_fit = scatter;
        best_h2 = h2_index;
      }
    }

    // Condition for finding a h2
    if (best_h2 != -1) {
      // Mark h2 as used
      hit_used[best_h2] = true;

      // Update the track in the bag
      if (number_of_hits == 3) {
        // Also mark the first three as used
        hit_used[t->hits[0]] = true;
        hit_used[t->hits[1]] = true;
        hit_used[t->hits[2]] = true;

        // If it is a track made out of less than or equal to 4 hits,
        // we have to allocate it in the tracks pointer
        track_number = atomicAdd(dev_number_of_velo_tracks + blockIdx.x, 1);
        tracks[track_number].hits[0] = t->hits[0];
        tracks[track_number].hits[1] = t->hits[1];
        tracks[track_number].hits[2] = t->hits[2];
        tracks[track_number].hits[3] = best_h2;
        tracks[track_number].hitsNum = 4;
      }
      else {
        t->hits[t->hitsNum++] = best_h2;
      }

      if (number_of_hits + 1 < Velo::Constants::max_track_size) {
        // Add the tracks to the bag of tracks to_follow
        const auto ttf_p =
          atomicAdd(dev_atomics_velo + atomics::tracks_to_follow, 1) % Velo::Constants::max_tracks_to_follow;
        tracks_to_follow[ttf_p] = track_number;
      }
    }
    // A track just skipped a module
    // We keep it for another round
    else if (skipped_modules < max_skipped_modules) {
      // Form the new mask
      track_number = ((skipped_modules + 1) << bits::skipped_module_position) |
                     (full_track_number & (bits::seed | bits::track_number));

      // Add the tracks to the bag of tracks to_follow
      const auto ttf_p =
        atomicAdd(dev_atomics_velo + atomics::tracks_to_follow, 1) % Velo::Constants::max_tracks_to_follow;
      tracks_to_follow[ttf_p] = track_number;
    }
    // If there are only three hits in this track,
    // mark it as "doubtful"
    else if (number_of_hits == 3) {
      const auto three_hit_tracks_p = atomicAdd(dev_atomics_velo + atomics::number_of_three_hit_tracks, 1);
      three_hit_tracks[three_hit_tracks_p] = Velo::TrackletHits {t->hits[0], t->hits[1], t->hits[2]};
    }
    // In the "else" case, we couldn't follow up the track,
    // so we won't be track following it anymore.
  }
}
