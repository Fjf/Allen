/*****************************************************************************\
* (c) Copyright 2018-2020 CERN for the benefit of the LHCb Collaboration      *
\*****************************************************************************/
#include "quantum_example.cuh"
#include <complex.h>
#include "Python.h"
#include "numpy/arrayobject.h"

#if defined(TARGET_DEVICE_CUDA)
#include <hip/hip_complex.h>
#include <custatevec.h>
#endif

INSTANTIATE_ALGORITHM(quantum::quantum_t)

int init_np()
{
  import_array();
  return 0;
}

void quantum::quantum_t::operator()(
  const ArgumentReferences<Parameters>& arguments,
  const RuntimeOptions&,
  const Constants&,
  const Allen::Context& context) const
{

  /*
   * Initialize dummy input vector
   */
  std::vector<std::vector<double>> dummy_input = {
    // x, y, z, module
    {-0.013944499509696598, -0.0010376710133435548, 1, 1},
    {0.008785145497894378, 0.011203088748345181, 1, 1},
    {-0.027888999019393197, -0.0020753420266871095, 2, 2},
    {0.017570290995788756, 0.022406177496690362, 2, 2},
    {-0.04183349852908979, -0.003113013040030664, 3, 3},
    {0.026355436493683135, 0.03360926624503554, 3, 3},
  };

  /*
   * Initialize python interpreter and load module
   */
  Py_Initialize();

  int err = init_np();

  PyObject* module_name = PyUnicode_FromString("quantum_circuit");
  PyObject* module = PyImport_Import(module_name);
  // TODO: Fix any errors imporitng not outputting erros here.
  if (!module) {
    std::cout << "quantum_circuit.py couldn't be imported. Ensure this file is in a directory findable by python. "
                 "(e.g., in your PYTHONPATH)"
              << std::endl;
    return;
  }

  PyObject* module_dict = PyModule_GetDict(module);

  /*
   * Load Davides expected input into vector
   */
  PyObject* result = PyList_New(0);
  for (int i = 0; i < dummy_input.size(); i++) {
    PyObject* hit = PyList_New(0);
    for (int j = 0; j < 4; j++) {
      PyList_Append(hit, PyFloat_FromDouble(dummy_input[i][j]));
    }
    PyList_Append(result, hit);
  }

  PyObject* func_args = PyTuple_New(1);
  PyTuple_SetItem(func_args, 0, result);
  PyObject* func = PyDict_GetItemString(module_dict, (char*) "circuit");
  std::cout << PyCallable_Check(func) << std::endl;
  if (PyCallable_Check(func)) {
    printf("Calling python function\n");
    PyObject* ret = PyObject_CallObject(func, func_args);
    PyArrayObject* np_ret = reinterpret_cast<PyArrayObject*>(ret);
    std::cout << np_ret << std::endl;
    npy_intp width = PyArray_DIM(np_ret, 0);
    npy_intp height = PyArray_DIM(np_ret, 1);
    std::cout << "GOt matrix of size" << width << "x" << height << std::endl;
    std::complex<double>* c_out = reinterpret_cast<std::complex<double>*>(PyArray_DATA(np_ret));
    std::cout << c_out[0] << std::endl;
  }
  Py_Finalize();

#if defined(TARGET_DEVICE_CUDA)
  const int nIndexBits = 3;
  const int nSvSize = (1 << nIndexBits);
  const int nTargets = 1;
  const int nControls = 2;
  const int adjoint = 0;

  int targets[] = {2};
  int controls[] = {0, 1};

  hipDoubleComplex h_sv[] = {
    {0.0, 0.0}, {0.0, 0.1}, {0.1, 0.1}, {0.1, 0.2}, {0.2, 0.2}, {0.3, 0.3}, {0.3, 0.4}, {0.4, 0.5}};
  hipDoubleComplex h_sv_result[] = {
    {0.0, 0.0}, {0.0, 0.1}, {0.1, 0.1}, {0.4, 0.5}, {0.2, 0.2}, {0.3, 0.3}, {0.3, 0.4}, {0.1, 0.2}};

  hipDoubleComplex matrix[] = {{0.0, 0.0}, {1.0, 0.0}, {1.0, 0.0}, {0.0, 0.0}};

  hipDoubleComplex* d_sv;
  hipMalloc((void**) &d_sv, nSvSize * sizeof(hipDoubleComplex));

  hipMemcpy(d_sv, h_sv, nSvSize * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

  //--------------------------------------------------------------------------

  // custatevec handle initialization
  custatevecHandle_t handle;

  custatevecCreate(&handle);

  void* extraWorkspace = nullptr;
  size_t extraWorkspaceSizeInBytes = 0;

  // check the size of external workspace
  custatevecApplyMatrixGetWorkspaceSize(
    handle,
    HIP_C_64F,
    nIndexBits,
    matrix,
    HIP_C_64F,
    CUSTATEVEC_MATRIX_LAYOUT_ROW,
    adjoint,
    nTargets,
    nControls,
    CUSTATEVEC_COMPUTE_64F,
    &extraWorkspaceSizeInBytes);

  // allocate external workspace if necessary
  if (extraWorkspaceSizeInBytes > 0) hipMalloc(&extraWorkspace, extraWorkspaceSizeInBytes);

  // apply gate
  custatevecApplyMatrix(
    handle,
    d_sv,
    HIP_C_64F,
    nIndexBits,
    matrix,
    HIP_C_64F,
    CUSTATEVEC_MATRIX_LAYOUT_ROW,
    adjoint,
    targets,
    nTargets,
    controls,
    nullptr,
    nControls,
    CUSTATEVEC_COMPUTE_64F,
    extraWorkspace,
    extraWorkspaceSizeInBytes);

  // destroy handle
  custatevecDestroy(handle);

  //--------------------------------------------------------------------------

  hipMemcpy(h_sv, d_sv, nSvSize * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

  bool correct = true;
  for (int i = 0; i < nSvSize; i++) {
    if ((h_sv[i].x != h_sv_result[i].x) || (h_sv[i].y != h_sv_result[i].y)) {
      correct = false;
      break;
    }
  }

  if (correct)
    printf("example PASSED\n");
  else
    printf("example FAILED: wrong result\n");

  hipFree(d_sv);
  if (extraWorkspaceSizeInBytes) hipFree(extraWorkspace);
#endif
}

///**
// * @brief SAXPY example algorithm
// * @detail Calculates for every event y = a*x + x, where x is the number of velo tracks in one event
// */
//__device__ void quantum::quantum(quantum::Parameters parameters)
//{
//  const auto number_of_events = parameters.dev_number_of_events[0];
//  for (unsigned event_number = threadIdx.x; event_number < number_of_events; event_number += blockDim.x) {
//    Velo::Consolidated::ConstTracks velo_tracks {
//      parameters.dev_atomics_velo, parameters.dev_velo_track_hit_number, event_number, number_of_events};
//    const unsigned number_of_tracks_event = velo_tracks.number_of_tracks(event_number);
//
//    parameters.dev_saxpy_output[event_number] =
//      parameters.saxpy_scale_factor * number_of_tracks_event + number_of_tracks_event;
//  }
//}
