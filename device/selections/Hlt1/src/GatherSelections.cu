#include "GatherSelections.cuh"
#include <numeric>

// Helper traits to traverse dev_input_selections_t
template<typename Arguments, typename Tuple>
struct TupleTraits {
  constexpr static unsigned i = 0;

  constexpr static unsigned get_size(Arguments&) { return 0; }

  template<typename AssignType>
  static void populate_event_offsets(const Arguments& arguments)
  {
    data<AssignType>(arguments)[i] = 0;
  }

  template<typename OffsetsType, typename AssignType, typename Stream>
  static void populate_selections(const Arguments&, Stream&)
  {}

  template<typename AssignType, typename NumberOfEvents, typename Stream>
  static void populate_selection_offsets(const Arguments&, Stream&)
  {}
};

template<typename Arguments, typename T, typename... R>
struct TupleTraits<Arguments, std::tuple<T, R...>> {
  constexpr static unsigned i = TupleTraits<Arguments, std::tuple<R...>>::i + 1;

  constexpr static unsigned get_size(Arguments& arguments)
  {
    return TupleTraits<Arguments, std::tuple<R...>>::get_size(arguments) + size<T>(arguments);
  }

  template<typename AssignType>
  static void populate_event_offsets(const Arguments& arguments)
  {
    TupleTraits<Arguments, std::tuple<R...>>::template populate_event_offsets<AssignType>(arguments);
    data<AssignType>(arguments)[i] = data<AssignType>(arguments)[i - 1] + size<T>(arguments);
  }

  template<typename OffsetsType, typename AssignType, typename Stream>
  static void populate_selections(const Arguments& arguments, Stream& stream)
  {
    TupleTraits<Arguments, std::tuple<R...>>::template populate_selections<OffsetsType, AssignType>(arguments, stream);
    copy<AssignType, T>(arguments, size<T>(arguments), stream, data<OffsetsType>(arguments)[i - 1], 0);
  }

  template<typename AssignType, typename NumberOfEvents, typename Stream>
  static void populate_selection_offsets(const Arguments& arguments, Stream& stream) {
    TupleTraits<Arguments, std::tuple<R...>>::template populate_selection_offsets<AssignType, NumberOfEvents, Stream>(arguments, stream);
    copy<AssignType, T>(arguments, size<T>(arguments), stream, first<NumberOfEvents>(arguments) * (i - 1), 0);
    
    // There should be as many elements as number of events
    assert(first<NumberOfEvents>(arguments) == size<T>(arguments));
  }
};

void gather_selections::gather_selections_t::set_arguments_size(
  ArgumentReferences<Parameters> arguments,
  const RuntimeOptions&,
  const Constants&,
  const HostBuffers&) const
{
  set_size<host_selections_events_offsets_t>(arguments, std::tuple_size<dev_input_selections_t::type>::value + 1);
  set_size<host_selections_offsets_t>(arguments, first<host_number_of_events_t>(arguments) * std::tuple_size<dev_input_selections_t::type>::value + 1);
  set_size<dev_selections_offsets_t>(arguments, first<host_number_of_events_t>(arguments) * std::tuple_size<dev_input_selections_t::type>::value + 1);
  set_size<dev_selections_t>(
    arguments, TupleTraits<ArgumentReferences<Parameters>, dev_input_selections_t::type>::get_size(arguments));

  if (property<verbosity_t>() >= logger::debug) {
    info_cout << "Sizes of gather_selections datatypes: "
      << size<host_selections_offsets_t>(arguments) << ", "
      << size<host_selections_events_offsets_t>(arguments) << ", "
      << size<dev_selections_offsets_t>(arguments) << ", "
      << size<dev_selections_t>(arguments) << "\n";
  }
}

void gather_selections::gather_selections_t::operator()(
  const ArgumentReferences<Parameters>& arguments,
  const RuntimeOptions&,
  const Constants&,
  HostBuffers&,
  hipStream_t& stream,
  hipEvent_t& event) const
{
  // Calculate offsets in host_selections_events_offsets_t
  TupleTraits<ArgumentReferences<Parameters>, TupleReverse<dev_input_selections_t::type>::t>::template populate_event_offsets<
    host_selections_events_offsets_t>(arguments);

  // Populate dev_selections_t
  TupleTraits<ArgumentReferences<Parameters>, TupleReverse<dev_input_selections_t::type>::t>::
    template populate_selections<host_selections_events_offsets_t, dev_selections_t>(arguments, stream);

  // Copy dev_input_selections_offsets_t onto host_selections_events_offsets_t
  TupleTraits<ArgumentReferences<Parameters>, TupleReverse<dev_input_selections_offsets_t::type>::t>::
    template populate_selection_offsets<host_selections_offsets_t, host_number_of_events_t>(arguments, stream);

  // Synchronize
  hipEventRecord(event, stream);
  hipEventSynchronize(event);

  // Do prefix sum
  unsigned temp = 0;
  unsigned temp_sum = 0;
  for (unsigned i = 0; i < size<host_selections_offsets_t>(arguments); ++i) {
    temp_sum += data<host_selections_offsets_t>(arguments)[i];
    data<host_selections_offsets_t>(arguments)[i] = temp;
    temp = temp_sum;
  }

  // Copy host_selections_offsets_t onto dev_selections_offsets_t
  copy<dev_selections_offsets_t, host_selections_offsets_t>(arguments, stream);
}
