#include "hip/hip_runtime.h"
#include "LineInfo.cuh"
#include "RunHlt1.cuh"
#include "DeterministicPostscaler.cuh"
#include "Event/ODIN.h"

__global__ void run_hlt1::run_postscale(
  run_hlt1::Parameters parameters,
  const uint selected_number_of_events,
  const uint event_start)
{
  const auto event_number = blockIdx.x;

  const uint hdr_size(8);
  const unsigned int* odinData = reinterpret_cast<const uint*>(
    parameters.dev_odin_raw_input + parameters.dev_odin_raw_input_offsets[event_number] + hdr_size);

  const uint32_t run_no = odinData[LHCb::ODIN::Data::RunNumber];
  const uint32_t evt_hi = odinData[LHCb::ODIN::Data::L0EventIDHi];
  const uint32_t evt_lo = odinData[LHCb::ODIN::Data::L0EventIDLo];
  const uint32_t gps_hi = odinData[LHCb::ODIN::Data::GPSTimeHi];
  const uint32_t gps_lo = odinData[LHCb::ODIN::Data::GPSTimeLo];

  // Process lines.
  const auto lambda_special_fn = [&](const unsigned long i_line, const float scale_factor) {
    bool* decs = parameters.dev_sel_results + parameters.dev_sel_results_offsets[i_line] + event_number;
    DeterministicPostscaler ps(i_line, scale_factor);
    ps(1, decs, run_no, evt_hi, evt_lo, gps_hi, gps_lo);
  };
  Hlt1::TraverseLinesScaleFactors<configured_lines_t, Hlt1::SpecialLine>::traverse(lambda_special_fn);

  if (blockIdx.x < selected_number_of_events) {
    const uint selected_event_number = blockIdx.x;
    const uint event_number = parameters.dev_event_list[blockIdx.x] - event_start;

    const uint hdr_size(8);
    const unsigned int* odinData = reinterpret_cast<const uint*>(
      parameters.dev_odin_raw_input + parameters.dev_odin_raw_input_offsets[selected_event_number] + hdr_size);

    const uint32_t run_no = odinData[LHCb::ODIN::Data::RunNumber];
    const uint32_t evt_hi = odinData[LHCb::ODIN::Data::L0EventIDHi];
    const uint32_t evt_lo = odinData[LHCb::ODIN::Data::L0EventIDLo];
    const uint32_t gps_hi = odinData[LHCb::ODIN::Data::GPSTimeHi];
    const uint32_t gps_lo = odinData[LHCb::ODIN::Data::GPSTimeLo];

    const auto n_tracks_event = parameters.dev_offsets_forward_tracks[selected_event_number + 1] -
                                parameters.dev_offsets_forward_tracks[selected_event_number];
    const auto n_vertices_event =
      parameters.dev_sv_offsets[selected_event_number + 1] - parameters.dev_sv_offsets[selected_event_number];

    // Process 1-track lines.
    const auto lambda_one_track_fn = [&](const unsigned long i_line, const float scale_factor) {
      bool* decs = parameters.dev_sel_results + parameters.dev_sel_results_offsets[i_line] +
                   parameters.dev_offsets_forward_tracks[selected_event_number];
      DeterministicPostscaler ps(i_line, scale_factor);
      ps(n_tracks_event, decs, run_no, evt_hi, evt_lo, gps_hi, gps_lo);
    };
    Hlt1::TraverseLinesScaleFactors<configured_lines_t, Hlt1::OneTrackLine>::traverse(
      lambda_one_track_fn);

    // Process 2-track lines.
    const auto lambda_two_track_fn = [&](const unsigned long i_line, const float scale_factor) {
      bool* decs = parameters.dev_sel_results + parameters.dev_sel_results_offsets[i_line] +
                   parameters.dev_sv_offsets[event_number];
      DeterministicPostscaler ps(i_line, scale_factor);
      ps(n_vertices_event, decs, run_no, evt_hi, evt_lo, gps_hi, gps_lo);
    };
    Hlt1::TraverseLinesScaleFactors<configured_lines_t, Hlt1::TwoTrackLine>::traverse(
      lambda_two_track_fn);

    // Process Velo lines.
    const auto lambda_velo_fn = [&](const unsigned long i_line, const float scale_factor) {
      bool* decs = parameters.dev_sel_results + parameters.dev_sel_results_offsets[i_line] + selected_event_number;
      DeterministicPostscaler ps(i_line, scale_factor);
      ps(1, decs, run_no, evt_hi, evt_lo, gps_hi, gps_lo);
    };
    Hlt1::TraverseLinesScaleFactors<configured_lines_t, Hlt1::VeloLine>::traverse(lambda_velo_fn);
  }
}
