#include "hip/hip_runtime.h"
/*****************************************************************************\
* (c) Copyright 2018-2020 CERN for the benefit of the LHCb Collaboration      *
\*****************************************************************************/
#include "CountLongTrackHits.cuh"

INSTANTIATE_ALGORITHM(count_long_track_hits::count_long_track_hits_t)

void count_long_track_hits::count_long_track_hits_t::set_arguments_size(
  ArgumentReferences<Parameters> arguments,
  const RuntimeOptions&,
  const Constants&,
  const HostBuffers&) const
{
  set_size<dev_long_track_hit_number_t>(arguments, first<host_number_of_reconstructed_scifi_tracks_t>(arguments));
}

void count_long_track_hits::count_long_track_hits_t::operator()(
  const ArgumentReferences<Parameters>& arguments,
  const RuntimeOptions&,
  const Constants&,
  HostBuffers&,
  const Allen::Context& context) const
{
  global_function(count_hits)(dim3(size<dev_event_list_t>(arguments)), property<block_dim_t>(), context)(arguments);
}

__global__ void count_long_track_hits::count_hits(count_long_track_hits::Parameters parameters)
{
  const unsigned event_number = parameters.dev_event_list[blockIdx.x];
  const unsigned number_of_events = parameters.dev_number_of_events[0];

  // Create velo tracks.
  Velo::Consolidated::Tracks const velo_tracks {
    parameters.dev_atomics_velo, parameters.dev_velo_track_hit_number, event_number, number_of_events};

  // Create UT tracks.
  UT::Consolidated::ConstExtendedTracks ut_tracks {parameters.dev_atomics_ut,
                                                   parameters.dev_ut_track_hit_number,
                                                   parameters.dev_ut_qop,
                                                   parameters.dev_ut_track_velo_indices,
                                                   event_number,
                                                   number_of_events};

  // Create SciFi tracks.
  SciFi::Consolidated::ConstTracks scifi_tracks {parameters.dev_atomics_scifi,
                                                 parameters.dev_scifi_track_hit_number,
                                                 parameters.dev_scifi_qop,
                                                 parameters.dev_scifi_states,
                                                 parameters.dev_scifi_track_ut_indices,
                                                 event_number,
                                                 number_of_events};

  const unsigned n_scifi_tracks = scifi_tracks.number_of_tracks(event_number);
  for (unsigned i_scifi_track = threadIdx.x; i_scifi_track < n_scifi_tracks; i_scifi_track += blockDim.x) {
    const int i_ut_track = scifi_tracks.ut_track(i_scifi_track);
    const int i_velo_track = ut_tracks.velo_track(i_ut_track);
    const unsigned n_velo_hits = velo_tracks.number_of_hits(i_velo_track);
    const unsigned n_ut_hits = ut_tracks.number_of_hits(i_ut_track);
    const unsigned n_scifi_hits = scifi_tracks.number_of_hits(i_scifi_track);
    const unsigned track_idx = scifi_tracks.tracks_offset(event_number) + i_scifi_track;
    parameters.dev_long_track_hit_number[track_idx] = n_velo_hits + n_ut_hits + n_scifi_hits;
  }
}