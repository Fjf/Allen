#include "hip/hip_runtime.h"
/*****************************************************************************\
* (c) Copyright 2020 CERN for the benefit of the LHCb Collaboration           *
\*****************************************************************************/
#include "D2KPiLine.cuh"

INSTANTIATE_LINE(d2kpi_line::d2kpi_line_t, d2kpi_line::Parameters)

__device__ bool d2kpi_line::d2kpi_line_t::select(
  const Parameters& parameters,
  std::tuple<const Allen::Views::Physics::CompositeParticle> input)
{
  const auto particle = std::get<0>(input);
  const bool opposite_sign = particle.charge() == 0;
  if (opposite_sign != parameters.OppositeSign) return false;

  const auto vertex = particle.vertex();
  if (vertex.chi2() < 0) {
    return false;
  }
  const float m1 = particle.m12(Allen::mK, Allen::mPi);
  const float m2 = particle.m12(Allen::mPi, Allen::mK);
  const bool decision = vertex.pt() > parameters.minComboPt && vertex.chi2() < parameters.maxVertexChi2 &&
                        particle.eta() > parameters.minEta && particle.eta() < parameters.maxEta &&
                        particle.doca12() < parameters.maxDOCA && particle.minpt() > parameters.minTrackPt &&
                        particle.minip() > parameters.minTrackIP &&
                        particle.ctau(Allen::mDz) > parameters.ctIPScale * parameters.minTrackIP &&
                        min(fabsf(m1 - Allen::mDz), fabsf(m2 - Allen::mDz)) < parameters.massWindow &&
                        vertex.z() >= parameters.minZ && particle.pv().position.z >= parameters.minZ;

  return decision;
}

__device__ void d2kpi_line::d2kpi_line_t::monitor(
  const Parameters& parameters,
  std::tuple<const Allen::Views::Physics::CompositeParticle> input,
  unsigned index,
  bool sel)
{
  if (sel) {
    const auto& particle = std::get<0>(input);
    // Use the following variables in bandwidth division
    parameters.min_pt[index] = particle.minpt(); // This should range in [800., 2000.]
    parameters.min_ip[index] = particle.minip(); // This should range in [0.06, 0.15]
    parameters.D0_ct[index] = particle.ctau(Allen::mDz);
  }
}
