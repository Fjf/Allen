#include "hip/hip_runtime.h"
#include "TrackMVALine.cuh"

// Explicit instantiation
INSTANTIATE_LINE(track_mva_line::track_mva_line_t, track_mva_line::Parameters)

__device__ bool track_mva_line::track_mva_line_t::select(
  const Parameters& parameters,
  std::tuple<const ParKalmanFilter::FittedTrack&> input) const
{
  const auto& track = std::get<0>(input);
  const auto ptShift = (track.pt() - parameters.alpha) / Gaudi::Units::GeV;
  const bool decision = track.chi2 / track.ndof < parameters.maxChi2Ndof &&
                        ((ptShift > parameters.maxPt && track.ipChi2 > parameters.minIPChi2) ||
                         (ptShift > parameters.minPt && ptShift < parameters.maxPt &&
                          logf(track.ipChi2) > parameters.param1 / (ptShift - parameters.param2) +
                                                 parameters.param3 / (parameters.maxPt * (parameters.maxPt - ptShift)) +
                                                 logf(parameters.minIPChi2)));
  return decision;
}
