#include "hip/hip_runtime.h"
/*****************************************************************************\
* (c) Copyright 2020 CERN for the benefit of the LHCb Collaboration           *
\*****************************************************************************/
#include "D2KKLine.cuh"

INSTANTIATE_LINE(d2kk_line::d2kk_line_t, d2kk_line::Parameters)

__device__ bool d2kk_line::d2kk_line_t::select(
  const Parameters& parameters,
  std::tuple<const VertexFit::TrackMVAVertex&> input)
{
  const auto& vertex = std::get<0>(input);
  if (vertex.chi2 < 0) {
    return false;
  }
  const bool decision = vertex.pt() > parameters.minComboPt && vertex.chi2 < parameters.maxVertexChi2 &&
                        vertex.doca < parameters.maxDOCA && vertex.eta > parameters.minEta &&
                        vertex.eta < parameters.maxEta && vertex.minpt > parameters.minTrackPt &&
                        vertex.minip > parameters.minTrackIP &&
                        fabsf(vertex.m(parameters.mK, parameters.mK) - parameters.mD) < parameters.massWindow;
  return decision;
}
