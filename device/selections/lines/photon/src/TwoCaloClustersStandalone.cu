#include "hip/hip_runtime.h"
/*****************************************************************************\
* (c) Copyright 2020 CERN for the benefit of the LHCb Collaboration           *
\*****************************************************************************/
#include <math.h>
#include "TwoCaloClustersStandalone.cuh"
#include <ROOTHeaders.h>
#include "CaloConstants.cuh"
#include "CaloCluster.cuh"

// Explicit instantiation
INSTANTIATE_LINE(two_calo_clusters_standalone_line::two_calo_clusters_standalone_line_t, two_calo_clusters_standalone_line::Parameters)

void two_calo_clusters_standalone_line::two_calo_clusters_standalone_line_t::set_arguments_size(
  ArgumentReferences<Parameters> arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers) const
{
  // must set_size of all output variables
  static_cast<Line const*>(this)->set_arguments_size(arguments, runtime_options, constants, host_buffers);
  set_size<typename Parameters::dev_histogram_pi0_mass_t>(arguments, 100u);


  set_size<host_ecal_twoclusters_t>(arguments, size<dev_ecal_twoclusters_t>(arguments));

  set_size<host_local_decisions_t>(arguments, get_decisions_size(arguments));

  set_size<dev_local_decisions_t>(arguments, get_decisions_size(arguments));

  set_size<host_ecal_twocluster_offsets_t>(arguments, size<dev_ecal_twocluster_offsets_t>(arguments));
}

__device__ bool two_calo_clusters_standalone_line::two_calo_clusters_standalone_line_t::select(
  const Parameters& parameters,
  std::tuple<const TwoCaloCluster&> input)
{
  const auto& dicluster = std::get<0>(input);
  const unsigned event_number = parameters.dev_event_list[blockIdx.x];

  const float transverse_distance = sqrtf((dicluster.x1-dicluster.x2)*(dicluster.x1-dicluster.x2) + (dicluster.y1-dicluster.y2)*(dicluster.y1-dicluster.y2));

  bool decision = (dicluster.Mass > parameters.minMass) && (dicluster.Mass < parameters.maxMass) &&
                  (dicluster.Et > parameters.minEt) &&
                  (dicluster.et1 > parameters.minEt_clusters && dicluster.et2 > parameters.minEt_clusters) &&
                  (dicluster.et1 + dicluster.et2 > parameters.minSumEt_clusters) &&
                  (dicluster.CaloNeutralE19_1 > parameters.minE19_clusters &&
                   dicluster.CaloNeutralE19_2 > parameters.minE19_clusters) &&
    (transverse_distance > parameters.minTransverseDistance);

  return decision;
}

void two_calo_clusters_standalone_line::two_calo_clusters_standalone_line_t::init_monitor(
  const ArgumentReferences<Parameters>& arguments,
  const Allen::Context& context) const
{
  Allen::memset_async<dev_local_decisions_t>(arguments, false, context);
  Allen::memset_async<dev_histogram_pi0_mass_t>(arguments, 0, context);
}

__device__ void two_calo_clusters_standalone_line::two_calo_clusters_standalone_line_t::monitor(
  const Parameters& parameters,
  std::tuple<const TwoCaloCluster> input,
  unsigned index,
  bool sel)
{
  parameters.dev_local_decisions[index] = sel;
  const auto twocalocluster = std::get<0>(input);
  if (sel) {
    const float m = twocalocluster.Mass;
    if (m > parameters.histogram_pi0_mass_min && m < parameters.histogram_pi0_mass_max) {
      const unsigned int bin = static_cast<unsigned int>(
        (m - parameters.histogram_pi0_mass_min) * parameters.histogram_pi0_mass_nbins /
        (parameters.histogram_pi0_mass_max - parameters.histogram_pi0_mass_min));
      ++parameters.dev_histogram_pi0_mass[bin];
    }
  }
}

//void two_calo_clusters_standalone_line::two_calo_clusters_standalone_line_t::output_monitor(
//  [[maybe_unused]] const ArgumentReferences<Parameters>& arguments,
//  [[maybe_unused]] const RuntimeOptions& runtime_options,
//  [[maybe_unused]] const Allen::Context& context) const
//{
//#ifdef WITH_ROOT
//  auto handler = runtime_options.root_service->handle(name());
//  // Distributions per dicluster
//  auto tree_twoclusters = handler.tree("monitor_tree_twoclusters");
//  if (tree_twoclusters == nullptr) return;
//
//  // Distributions per event
//  auto tree_evts = handler.tree("monitor_tree_evts");
//  if (tree_evts == nullptr) return;
//
//  Allen::copy<host_ecal_twocluster_offsets_t, dev_ecal_twocluster_offsets_t>(arguments, context);
//  Allen::copy<host_ecal_twoclusters_t, dev_ecal_twoclusters_t>(arguments, context);
//  Allen::copy<host_local_decisions_t, dev_local_decisions_t>(arguments, context);
//  Allen::synchronize(context);
//
//  float Mass = 0.f;
//  float Et = 0.f;
//  float Distance = 0.f;
//  float x1 = 0.f;
//  float x2 = 0.f;
//  float y1 = 0.f;
//  float y2 = 0.f;
//  float et1 = 0.f;
//  float et2 = 0.f;
//  float e19_1 = 0.f;
//  float e19_2 = 0.f;
//  unsigned num_twoclusters = 0u;
//  unsigned event_number = 0u;
//
//  handler.branch(tree_twoclusters, "Mass", Mass);
//  handler.branch(tree_twoclusters, "Et", Et);
//  handler.branch(tree_twoclusters, "Distance", Distance);
//  handler.branch(tree_twoclusters, "x1", x1);
//  handler.branch(tree_twoclusters, "x2", x2);
//  handler.branch(tree_twoclusters, "y1", y1);
//  handler.branch(tree_twoclusters, "y2", y2);
//  handler.branch(tree_twoclusters, "et1", et1);
//  handler.branch(tree_twoclusters, "et2", et2);
//  handler.branch(tree_twoclusters, "e19_1", e19_1);
//  handler.branch(tree_twoclusters, "e19_2", e19_2);
//  handler.branch(tree_twoclusters, "num_twoclusters", num_twoclusters);
//  handler.branch(tree_twoclusters, "event_number", event_number);
//
//  handler.branch(tree_evts, "num_twoclusters", num_twoclusters);
//
//  const unsigned n_events = size<host_ecal_twocluster_offsets_t>(arguments) - 1;
//
//  for (unsigned event_index = 0; event_index < n_events; event_index++) {
//    const unsigned& twoclusters_offset = (data<host_ecal_twocluster_offsets_t>(arguments) + event_index)[0];
//    num_twoclusters = (data<host_ecal_twocluster_offsets_t>(arguments) + event_index + 1)[0] - twoclusters_offset;
//    event_number = event_index;
//    tree_evts->Fill();
//
//    for (unsigned twocluster_index = 0; twocluster_index < num_twoclusters; twocluster_index++) {
//      const bool& decision = (data<host_local_decisions_t>(arguments) + twoclusters_offset + twocluster_index)[0];
//      if (decision) {
//        const auto& dicluster = (data<host_ecal_twoclusters_t>(arguments) + twoclusters_offset + twocluster_index)[0];
//
//        Mass = dicluster.Mass;
//        Distance = dicluster.Distance;
//        Et = dicluster.Et;
//        x1 = dicluster.x1;
//        x2 = dicluster.x2;
//        y1 = dicluster.y1;
//        y2 = dicluster.y2;
//        et1 = dicluster.et1;
//        et2 = dicluster.et2;
//        e19_1 = dicluster.CaloNeutralE19_1;
//        e19_2 = dicluster.CaloNeutralE19_2;
//
//        tree_twoclusters->Fill();
//      }
//    }
//  }
//#endif
//}
