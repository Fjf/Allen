#include "hip/hip_runtime.h"
#include "FitSeeds.cuh"

void fit_seeds::pv_fit_seeds_t::set_arguments_size(
  ArgumentReferences<Parameters> arguments,
  const RuntimeOptions&,
  const Constants&,
  const HostBuffers&) const
{
  set_size<dev_vertex_t>(arguments, PatPV::max_number_vertices * first<host_number_of_selected_events_t>(arguments));
  set_size<dev_number_vertex_t>(arguments, first<host_number_of_selected_events_t>(arguments));
}

void fit_seeds::pv_fit_seeds_t::operator()(
  const ArgumentReferences<Parameters>& arguments,
  const RuntimeOptions& runtime_options,
  const Constants&,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t&) const
{
  device_function(fit_seeds)(
    dim3(first<host_number_of_selected_events_t>(arguments)), property<block_dim_t>(), hip_stream)(arguments);

  if (runtime_options.do_check) {
    // Retrieve result
    cudaCheck(hipMemcpyAsync(
      host_buffers.host_reconstructed_pvs,
      data<dev_vertex_t>(arguments),
      size<dev_vertex_t>(arguments),
      hipMemcpyDeviceToHost,
      hip_stream));

    cudaCheck(hipMemcpyAsync(
      host_buffers.host_number_of_vertex,
      data<dev_number_vertex_t>(arguments),
      size<dev_number_vertex_t>(arguments),
      hipMemcpyDeviceToHost,
      hip_stream));
  }
}

__global__ void fit_seeds::fit_seeds(fit_seeds::Parameters parameters)
{
  const uint number_of_events = gridDim.x;
  const uint event_number = blockIdx.x;

  const Velo::Consolidated::Tracks velo_tracks {
    parameters.dev_atomics_velo, parameters.dev_velo_track_hit_number, event_number, number_of_events};
  Velo::Consolidated::ConstKalmanStates velo_states {
    parameters.dev_velo_kalman_beamline_states, velo_tracks.total_number_of_tracks()};
  const uint number_of_tracks_event = velo_tracks.number_of_tracks(event_number);
  const uint event_tracks_offset = velo_tracks.tracks_offset(event_number);

  PV::Vertex vertex;

  int counter_vertex = 0;
  for (uint i_seed = 0; i_seed < parameters.dev_number_seeds[event_number]; i_seed++) {
    bool success = fit_vertex(
      parameters.dev_seeds[event_number * PatPV::max_number_vertices + i_seed],
      velo_states,
      vertex,
      number_of_tracks_event,
      event_tracks_offset);
    if (success) {

      parameters.dev_vertex[PatPV::max_number_vertices * event_number + counter_vertex] = vertex;
      counter_vertex++;
    }
  }

  parameters.dev_number_vertex[event_number] = counter_vertex;
}

__device__ bool fit_vertex(
  const PatPV::XYZPoint& seedPoint,
  Velo::Consolidated::ConstKalmanStates& velo_states,
  PV::Vertex& vtx,
  int number_of_tracks,
  uint tracks_offset)
{

  float tr_state_x[Velo::Constants::max_tracks];
  float tr_state_y[Velo::Constants::max_tracks];
  float tr_state_z[Velo::Constants::max_tracks];

  float tr_state_tx[Velo::Constants::max_tracks];
  float tr_state_ty[Velo::Constants::max_tracks];

  float tr_state_c00[Velo::Constants::max_tracks];
  float tr_state_c11[Velo::Constants::max_tracks];
  float tr_state_c20[Velo::Constants::max_tracks];
  float tr_state_c22[Velo::Constants::max_tracks];
  float tr_state_c31[Velo::Constants::max_tracks];
  float tr_state_c33[Velo::Constants::max_tracks];

  // position at which derivatives are evaluated

  float3 vtxpos {seedPoint.x, seedPoint.y, seedPoint.z};

  // prepare tracks

  uint pvTrack_counter = 0;

  for (int i = 0; i < number_of_tracks; i++) {
    int index = i + tracks_offset;

    KalmanVeloState trk = velo_states.get(index);
    float new_z = vtxpos.z;

    float m_state_x = trk.x;
    float m_state_y = trk.y;
    float m_state_z = trk.z;

    float m_state_tx = trk.tx;
    float m_state_ty = trk.ty;

    float m_state_c00 = trk.c00;
    float m_state_c11 = trk.c11;
    float m_state_c20 = trk.c20;
    float m_state_c22 = trk.c22;
    float m_state_c31 = trk.c31;
    float m_state_c33 = trk.c33;

    const float dz = new_z - m_state_z;
    const float dz2 = dz * dz;

    m_state_x += dz * m_state_tx;
    m_state_y += dz * m_state_ty;
    m_state_z = new_z;
    m_state_c00 += dz2 * m_state_c22 + 2 * dz * m_state_c20;
    m_state_c20 += dz * m_state_c22;
    m_state_c11 += dz2 * m_state_c33 + 2 * dz * m_state_c31;
    m_state_c31 += dz * m_state_c33;

    float2 res {vtxpos.x - m_state_x, vtxpos.y - m_state_y};

    float tr_chi2 = res.x * res.x / m_state_c00 + res.y * res.y / m_state_c11;

    if (tr_chi2 < PatPV::m_maxChi2) {
      // have to use updated values!!
      tr_state_x[pvTrack_counter] = m_state_x;
      tr_state_y[pvTrack_counter] = m_state_y;
      tr_state_z[pvTrack_counter] = m_state_z;

      tr_state_tx[pvTrack_counter] = m_state_tx;
      tr_state_ty[pvTrack_counter] = m_state_ty;

      tr_state_c00[pvTrack_counter] = m_state_c00;
      tr_state_c11[pvTrack_counter] = m_state_c11;
      tr_state_c20[pvTrack_counter] = m_state_c20;
      tr_state_c22[pvTrack_counter] = m_state_c22;
      tr_state_c31[pvTrack_counter] = m_state_c31;
      tr_state_c33[pvTrack_counter] = m_state_c33;

      pvTrack_counter++;
    }
  }

  if (pvTrack_counter < PatPV::m_minTr) {
    return false;
  }

  float vtxcov[6];
  bool converged = false;
  float maxdz = PatPV::m_maxDeltaZ;
  int nbIter = 0;
  int tracks_in_vertex = 0;
  while ((nbIter < PatPV::m_minIter) || (!converged && nbIter < PatPV::m_Iterations)) {
    ++nbIter;

    float halfD2Chi2DX2_00 = 0.f;
    float halfD2Chi2DX2_10 = 0.f;
    float halfD2Chi2DX2_11 = 0.f;
    float halfD2Chi2DX2_20 = 0.f;
    float halfD2Chi2DX2_21 = 0.f;
    float halfD2Chi2DX2_22 = 0.f;
    PatPV::XYZPoint halfDChi2DX(0.f, 0.f, 0.f);

    // add contribution from all tracks
    float chi2(0);
    size_t ntrin(0);
    for (uint index = 0; index < pvTrack_counter; index++) {

      float new_z = vtxpos.z;
      float m_state_x = tr_state_x[index];
      float m_state_y = tr_state_y[index];
      float m_state_z = tr_state_z[index];

      float m_state_tx = tr_state_tx[index];
      float m_state_ty = tr_state_ty[index];

      float m_state_c00 = tr_state_c00[index];
      float m_state_c11 = tr_state_c11[index];
      float m_state_c20 = tr_state_c20[index];
      float m_state_c22 = tr_state_c22[index];
      float m_state_c31 = tr_state_c31[index];
      float m_state_c33 = tr_state_c33[index];

      const float dz = new_z - m_state_z;
      const float dz2 = dz * dz;

      m_state_x += dz * m_state_tx;
      m_state_y += dz * m_state_ty;
      m_state_z = new_z;
      m_state_c00 += dz2 * m_state_c22 + 2 * dz * m_state_c20;
      m_state_c20 += dz * m_state_c22;
      m_state_c11 += dz2 * m_state_c33 + 2 * dz * m_state_c31;
      m_state_c31 += dz * m_state_c33;

      float2 res {vtxpos.x - m_state_x, vtxpos.y - m_state_y};

      float tr_halfD2Chi2DX2_00 = 1.f / m_state_c00;
      float tr_halfD2Chi2DX2_10 = 0.f;
      float tr_halfD2Chi2DX2_11 = 1.f / m_state_c11;
      float tr_halfD2Chi2DX2_20 = -m_state_tx / m_state_c00;
      float tr_halfD2Chi2DX2_21 = -m_state_ty / m_state_c11;
      float tr_halfD2Chi2DX2_22 = m_state_tx * m_state_tx / m_state_c00 + m_state_ty * m_state_ty / m_state_c11;

      float tr_halfDChi2DX_x = res.x / m_state_c00;
      float tr_halfDChi2DX_y = res.y / m_state_c11;
      float tr_halfDChi2DX_z = -m_state_tx * res.x / m_state_c00 - m_state_ty * res.y / m_state_c11;
      float tr_chi2 = res.x * res.x / m_state_c00 + res.y * res.y / m_state_c11;

      float weight = get_tukey_weight(tr_chi2, nbIter);

      // add the track
      if (weight > PatPV::m_minTrackWeight) {
        ++ntrin;
        halfD2Chi2DX2_00 += weight * tr_halfD2Chi2DX2_00;
        halfD2Chi2DX2_10 += weight * tr_halfD2Chi2DX2_10;
        halfD2Chi2DX2_11 += weight * tr_halfD2Chi2DX2_11;
        halfD2Chi2DX2_20 += weight * tr_halfD2Chi2DX2_20;
        halfD2Chi2DX2_21 += weight * tr_halfD2Chi2DX2_21;
        halfD2Chi2DX2_22 += weight * tr_halfD2Chi2DX2_22;

        halfDChi2DX.x += weight * tr_halfDChi2DX_x;
        halfDChi2DX.y += weight * tr_halfDChi2DX_y;
        halfDChi2DX.z += weight * tr_halfDChi2DX_z;

        chi2 += weight * tr_chi2;
      }
    }

    // check nr of tracks that entered the fit
    if (ntrin < PatPV::m_minTr) {

      return false;
    }

    // compute the new vertex covariance

    // replace Cholesky inverter by analytical solution
    float a00 = halfD2Chi2DX2_00;
    float a10 = halfD2Chi2DX2_10;
    float a11 = halfD2Chi2DX2_11;
    float a20 = halfD2Chi2DX2_20;
    float a21 = halfD2Chi2DX2_21;
    float a22 = halfD2Chi2DX2_22;

    float det = a00 * (a22 * a11 - a21 * a21) - a10 * (a22 * a10 - a21 * a20) + a20 * (a21 * a10 - a11 * a20);
    if (det == 0) return false;

    vtxcov[0] = (a22 * a11 - a21 * a21) / det;
    vtxcov[1] = -(a22 * a10 - a20 * a21) / det;
    vtxcov[2] = (a22 * a00 - a20 * a20) / det;
    vtxcov[3] = (a21 * a10 - a20 * a11) / det;
    vtxcov[4] = -(a21 * a00 - a20 * a10) / det;
    vtxcov[5] = (a11 * a00 - a10 * a10) / det;

    // compute the delta
    PatPV::XYZPoint delta {0.f, 0.f, 0.f};
    delta.x = -1.0f * (vtxcov[0] * halfDChi2DX.x + vtxcov[1] * halfDChi2DX.y + vtxcov[3] * halfDChi2DX.z);
    delta.y = -1.0f * (vtxcov[1] * halfDChi2DX.x + vtxcov[2] * halfDChi2DX.y + vtxcov[4] * halfDChi2DX.z);
    delta.z = -1.0f * (vtxcov[3] * halfDChi2DX.x + vtxcov[4] * halfDChi2DX.y + vtxcov[5] * halfDChi2DX.z);

    chi2 += delta.x * halfDChi2DX.x + delta.y * halfDChi2DX.y + delta.z * halfDChi2DX.z;

    // update the position
    vtxpos.x = (vtxpos.x + delta.x);
    vtxpos.y = (vtxpos.y + delta.y);
    vtxpos.z = (vtxpos.z + delta.z);

    vtx.setChi2AndDoF(chi2, 2 * ntrin - 3);

    // loose convergence criteria if close to end of iterations
    if (1.f * nbIter > 0.8f * PatPV::m_Iterations) maxdz = 10.f * PatPV::m_maxDeltaZ;
    converged = fabsf(delta.z) < maxdz;
    tracks_in_vertex = ntrin;
  } // end iteration loop

  if (!converged) return false;

  // set position and covariance
  vtx.setPosition(vtxpos);
  vtx.setCovMatrix(vtxcov);
  // Set tracks. Compute final chi2.

  vtx.nTracks = tracks_in_vertex;

  return true;
}

//=============================================================================
// Get Tukey's weight
//=============================================================================
__device__ float get_tukey_weight(float trchi2, int iter)
{
  if (iter < 1) return 1.f;
  float ctrv = PatPV::m_trackChi * max(PatPV::m_minIter - iter, 1);
  float cT2 = trchi2 / powf(ctrv * PatPV::m_TrackErrorScaleFactor, 2);
  return cT2 < 1.f ? powf(1.f - cT2, 2) : 0.f;
}
