#include "hip/hip_runtime.h"
/*****************************************************************************\
* (c) Copyright 2018-2020 CERN for the benefit of the LHCb Collaboration      *
\*****************************************************************************/
#include "pv_beamline_histo.cuh"

INSTANTIATE_ALGORITHM(pv_beamline_histo::pv_beamline_histo_t)

void pv_beamline_histo::pv_beamline_histo_t::set_arguments_size(
  ArgumentReferences<Parameters> arguments,
  const RuntimeOptions&,
  const Constants&,
  const HostBuffers&) const
{
  set_size<dev_zhisto_t>(
    arguments,
    first<host_number_of_events_t>(arguments) *
      (BeamlinePVConstants::Common::zmax - BeamlinePVConstants::Common::zmin) / BeamlinePVConstants::Common::dz);
}

void pv_beamline_histo::pv_beamline_histo_t::operator()(
  const ArgumentReferences<Parameters>& arguments,
  const RuntimeOptions&,
  const Constants& constants,
  HostBuffers&,
  const Allen::Context& context) const
{
  Allen::memset_async<dev_zhisto_t>(arguments, 0, context);

  global_function(pv_beamline_histo)(dim3(size<dev_event_list_t>(arguments)), property<block_dim_t>(), context)(
    arguments, constants.dev_beamline.data());
}

__device__ float gauss_integral(float x)
{
  const float a = sqrtf(float(2 * BeamlinePVConstants::Histo::order_polynomial + 3));
  const float xi = x / a;
  const float eta = 1.f - xi * xi;
  constexpr float p[] = {0.5f, 0.25f, 0.1875f, 0.15625f};
  // be careful: if you choose here one order more, you also need to choose 'a' differently (a(N)=sqrt(2N+3))
  return 0.5f + xi * (p[0] + eta * (p[1] + eta * p[2]));
}

__global__ void pv_beamline_histo::pv_beamline_histo(pv_beamline_histo::Parameters parameters, float* dev_beamline)
{
  const unsigned event_number = parameters.dev_event_list[blockIdx.x];

  const auto velo_tracks_view = parameters.dev_velo_tracks_view[event_number];
  float* histo_base_pointer = parameters.dev_zhisto + BeamlinePVConstants::Common::Nbins * event_number;

  // printf("beamline x = %f, y = %f \n", dev_beamline[0], dev_beamline[1]);

  for (unsigned index = threadIdx.x; index < velo_tracks_view.size(); index += blockDim.x) {
    PVTrack trk = parameters.dev_pvtracks[velo_tracks_view.offset() + index];
    // apply the z cut here
    if (BeamlinePVConstants::Common::zmin < trk.z && trk.z < BeamlinePVConstants::Common::zmax) {
      const float diffx2 = (trk.x.x - dev_beamline[0]) * (trk.x.x - dev_beamline[0]);
      const float diffy2 = (trk.x.y - dev_beamline[1]) * (trk.x.y - dev_beamline[1]);
      const float blchi2 = diffx2 * trk.W_00 + diffy2 * trk.W_11;
      if (blchi2 >= parameters.max_track_blchi2) continue;

      // bin in which z0 is, in floating point
      const float zbin = (trk.z - BeamlinePVConstants::Common::zmin) / BeamlinePVConstants::Common::dz;

      // to compute the size of the window, we use the track
      // errors. eventually we can just parametrize this as function of
      // track slope.
      const float zweight = trk.tx.x * trk.tx.x * trk.W_00 + trk.tx.y * trk.tx.y * trk.W_11;
      const float zerr = 1.f / sqrtf(zweight);
      // get rid of useless tracks. must be a bit carefull with this.
      const float maxTrackZ0Err = trk.z < BeamlinePVConstants::Common::SMOG2_pp_separation ?
                                    BeamlinePVConstants::Common::SMOG2_maxTrackZ0Err :
                                    BeamlinePVConstants::Common::pp_maxTrackZ0Err;

      if (zerr < maxTrackZ0Err) { // m_nsigma < 10*m_dz ) {
        // find better place to define this
        const float a = sqrtf(float(2 * BeamlinePVConstants::Histo::order_polynomial + 3));
        const float halfwindow = a * zerr / BeamlinePVConstants::Common::dz;
        // this looks a bit funny, but we need the first and last bin of the histogram to remain empty.
        const int minbin = max(int(zbin - halfwindow), 1);
        const int maxbin = min(int(zbin + halfwindow), BeamlinePVConstants::Common::Nbins - 2);
        // we can get rid of this if statement if we make a selection of seeds earlier
        if (maxbin >= minbin) {
          float integral = 0;
          for (auto i = minbin; i < maxbin; ++i) {
            const float relz =
              (BeamlinePVConstants::Common::zmin + (i + 1) * BeamlinePVConstants::Common::dz - trk.z) / zerr;
            const float thisintegral = gauss_integral(relz);
            atomicAdd(histo_base_pointer + i, thisintegral - integral);
            integral = thisintegral;
          }
          // deal with the last bin
          atomicAdd(histo_base_pointer + maxbin, 1.f - integral);
        }
      }
    }
  }
}
