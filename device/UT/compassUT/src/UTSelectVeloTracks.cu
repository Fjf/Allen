#include "hip/hip_runtime.h"
/*****************************************************************************\
* (c) Copyright 2018-2020 CERN for the benefit of the LHCb Collaboration      *
\*****************************************************************************/
#include "UTSelectVeloTracks.cuh"
#include <tuple>

void ut_select_velo_tracks::ut_select_velo_tracks_t::set_arguments_size(
  ArgumentReferences<Parameters> arguments,
  const RuntimeOptions&,
  const Constants&,
  const HostBuffers&) const
{
  set_size<dev_ut_number_of_selected_velo_tracks_t>(arguments, first<host_number_of_events_t>(arguments));
  set_size<dev_ut_selected_velo_tracks_t>(arguments, first<host_number_of_reconstructed_velo_tracks_t>(arguments));
}

void ut_select_velo_tracks::ut_select_velo_tracks_t::operator()(
  const ArgumentReferences<Parameters>& arguments,
  const RuntimeOptions&,
  const Constants&,
  HostBuffers&,
  const Allen::Context& context) const
{
  initialize<dev_ut_number_of_selected_velo_tracks_t>(arguments, 0, context);

  global_function(ut_select_velo_tracks)(dim3(size<dev_event_list_t>(arguments)), property<block_dim_t>(), context)(
    arguments);
}

__global__ void ut_select_velo_tracks::ut_select_velo_tracks(ut_select_velo_tracks::Parameters parameters)
{
  const unsigned event_number = parameters.dev_event_list[blockIdx.x];
  const unsigned number_of_events = parameters.dev_number_of_events[0];

  // Velo consolidated types
  Velo::Consolidated::ConstTracks velo_tracks {
    parameters.dev_atomics_velo, parameters.dev_velo_track_hit_number, event_number, number_of_events};
  Velo::Consolidated::ConstStates velo_beamline_states {parameters.dev_velo_beamline_states, velo_tracks.total_number_of_tracks()};

  const unsigned number_of_tracks_event = velo_tracks.number_of_tracks(event_number);
  const unsigned event_tracks_offset = velo_tracks.tracks_offset(event_number);

  auto ut_number_of_selected_velo_tracks = parameters.dev_ut_number_of_selected_velo_tracks + event_number;
  auto ut_selected_velo_tracks = parameters.dev_ut_selected_velo_tracks + event_tracks_offset;

  for (unsigned i = threadIdx.x; i < number_of_tracks_event; i += blockDim.x) {
    const unsigned current_track_offset = event_tracks_offset + i;
    const auto velo_beamline_state = velo_beamline_states.get(current_track_offset);
    Velo::Consolidated::ConstHits consolidated_hits = velo_tracks.get_hits(parameters.dev_velo_track_hits.get(), i);
    const auto backward = velo_beamline_state.z > consolidated_hits.z(0);
    if (
      !backward && parameters.dev_accepted_velo_tracks[current_track_offset] &&
      velo_track_in_UTA_acceptance(velo_beamline_state)) {
      int current_track = atomicAdd(ut_number_of_selected_velo_tracks, 1);
      ut_selected_velo_tracks[current_track] = i;
    }
  }
}

//=============================================================================
// Reject tracks outside of acceptance or pointing to the beam pipe
//=============================================================================
__device__ bool ut_select_velo_tracks::velo_track_in_UTA_acceptance(const MiniState& state)
{
  const float xMidUT = state.x + state.tx * (UT::Constants::zMidUT - state.z);
  const float yMidUT = state.y + state.ty * (UT::Constants::zMidUT - state.z);

  if (xMidUT * xMidUT + yMidUT * yMidUT < UT::Constants::centralHoleSize * UT::Constants::centralHoleSize) return false;
  if ((fabsf(state.tx) > UT::Constants::maxXSlope) || (fabsf(state.ty) > UT::Constants::maxYSlope)) return false;

  if (
    UT::Constants::passTracks && fabsf(xMidUT) < UT::Constants::passHoleSize &&
    fabsf(yMidUT) < UT::Constants::passHoleSize) {
    return false;
  }

  return true;
}
