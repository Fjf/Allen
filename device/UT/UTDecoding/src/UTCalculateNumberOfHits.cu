#include "hip/hip_runtime.h"
/*****************************************************************************\
* (c) Copyright 2018-2020 CERN for the benefit of the LHCb Collaboration      *
\*****************************************************************************/
#include <MEPTools.h>
#include <UTCalculateNumberOfHits.cuh>

void ut_calculate_number_of_hits::ut_calculate_number_of_hits_t::set_arguments_size(
  ArgumentReferences<Parameters> arguments,
  const RuntimeOptions&,
  const Constants& constants,
  const HostBuffers&) const
{
  set_size<dev_ut_hit_sizes_t>(
    arguments,
    first<host_number_of_events_t>(arguments) * constants.host_unique_x_sector_layer_offsets[UT::Constants::n_layers]);
}

void ut_calculate_number_of_hits::ut_calculate_number_of_hits_t::operator()(
  const ArgumentReferences<Parameters>& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers&,
  const Allen::Context& context) const
{
  initialize<dev_ut_hit_sizes_t>(arguments, 0, context);

  auto const bank_version = first<host_raw_bank_version_t>(arguments);

  if (runtime_options.mep_layout) {
    auto fun = bank_version == 4 ? global_function(ut_calculate_number_of_hits_mep<4>) :
                                   global_function(ut_calculate_number_of_hits_mep<3>);
    fun(dim3(size<dev_event_list_t>(arguments)), property<block_dim_t>(), context)(
      arguments,
      constants.dev_ut_boards.data(),
      constants.dev_ut_region_offsets.data(),
      constants.dev_unique_x_sector_layer_offsets.data(),
      constants.dev_unique_x_sector_offsets.data());
  }
  else {
    auto fun = bank_version == 4 ? global_function(ut_calculate_number_of_hits<4>) :
                                   global_function(ut_calculate_number_of_hits<3>);
    fun(dim3(size<dev_event_list_t>(arguments)), property<block_dim_t>(), context)(
      arguments,
      constants.dev_ut_boards.data(),
      constants.dev_ut_region_offsets.data(),
      constants.dev_unique_x_sector_layer_offsets.data(),
      constants.dev_unique_x_sector_offsets.data());
  }
}

/**
 * @brief Given a UT RawBank, this function calculates the number of hits in a sector_group
  ("virtual" structure for optimized processing; a group of sectors where the start X is of a certain value).
 */
template<int decoding_version>
__device__ void calculate_number_of_hits(
  unsigned const*,
  unsigned const*,
  uint32_t*,
  UTBoards const&,
  UTRawBank<decoding_version> const&)
{}

template<>
__device__ void calculate_number_of_hits<3>(
  unsigned const* dev_ut_region_offsets,
  unsigned const* dev_unique_x_sector_offsets,
  uint32_t* hit_offsets,
  UTBoards const& boards,
  UTRawBank<3> const& raw_bank)
{
  const uint32_t m_nStripsPerHybrid = boards.stripsPerHybrids[raw_bank.sourceID];

  for (unsigned i = threadIdx.y; i < raw_bank.get_n_hits(); i += blockDim.y) {
    const uint32_t channelID = (raw_bank.data[i] & UT::Decoding::v4::chan_mask) >> UT::Decoding::v4::chan_offset;
    const uint32_t index = channelID / m_nStripsPerHybrid;
    const uint32_t fullChanIndex = raw_bank.sourceID * UT::Decoding::ut_number_of_sectors_per_board + index;
    const uint32_t station = boards.stations[fullChanIndex] - 1;
    const uint32_t layer = boards.layers[fullChanIndex] - 1;
    const uint32_t detRegion = boards.detRegions[fullChanIndex] - 1;
    const uint32_t sector = boards.sectors[fullChanIndex] - 1;

    // Calculate the index to get the geometry of the board
    const uint32_t idx = station * UT::Decoding::ut_number_of_sectors_per_board + layer * 3 + detRegion;
    const uint32_t idx_offset = dev_ut_region_offsets[idx] + sector;

    unsigned* hits_sector_group = hit_offsets + dev_unique_x_sector_offsets[idx_offset];
    atomicAdd(hits_sector_group, 1);
  }
}

template<>
__device__ void calculate_number_of_hits<4>(
  unsigned const* dev_ut_region_offsets,
  unsigned const* dev_unique_x_sector_offsets,
  uint32_t* hit_offsets,
  UTBoards const& boards,
  UTRawBank<4> const& raw_bank)
{
  if (raw_bank.get_n_hits() == 0) return;
  for (unsigned lane = threadIdx.y; lane < UT::Decoding::ut_number_of_sectors_per_board; lane += blockDim.y) {
    if (raw_bank.number_of_hits[lane] == 0) continue;
    // find the sector group to which these hits are added
    const uint32_t fullChanIndex = raw_bank.sourceID * UT::Decoding::ut_number_of_sectors_per_board + lane;
    const uint32_t station = boards.stations[fullChanIndex] - 1;
    const uint32_t layer = boards.layers[fullChanIndex] - 1;
    const uint32_t detRegion = boards.detRegions[fullChanIndex] - 1;
    const uint32_t sector = boards.sectors[fullChanIndex] - 1;

    // add the hits to the global counters and offsets
    const uint32_t idx = station * UT::Decoding::ut_number_of_sectors_per_board + layer * 3 + detRegion;
    const uint32_t idx_offset = dev_ut_region_offsets[idx] + sector;
    unsigned* hits_sector_group = hit_offsets + dev_unique_x_sector_offsets[idx_offset];
    atomicAdd(hits_sector_group, raw_bank.number_of_hits[lane]);
  }
}

/**
 * @brief Calculates the number of hits to be decoded for the UT detector.
 */
template<int decoding_version>
__global__ void ut_calculate_number_of_hits::ut_calculate_number_of_hits(
  ut_calculate_number_of_hits::Parameters parameters,
  const char* ut_boards,
  const unsigned* dev_ut_region_offsets,
  const unsigned* dev_unique_x_sector_layer_offsets,
  const unsigned* dev_unique_x_sector_offsets)
{
  const unsigned event_number = parameters.dev_event_list[blockIdx.x];

  const uint32_t event_offset = parameters.dev_ut_raw_input_offsets[event_number];
  const unsigned number_of_unique_x_sectors = dev_unique_x_sector_layer_offsets[UT::Constants::n_layers];
  uint32_t* hit_offsets = parameters.dev_ut_hit_sizes + event_number * number_of_unique_x_sectors;

  const UTRawEvent raw_event(parameters.dev_ut_raw_input + event_offset);
  const UTBoards boards(ut_boards);

  for (unsigned raw_bank_index = threadIdx.x; raw_bank_index < raw_event.number_of_raw_banks;
       raw_bank_index += blockDim.x)
    calculate_number_of_hits(
      dev_ut_region_offsets,
      dev_unique_x_sector_offsets,
      hit_offsets,
      boards,
      raw_event.getUTRawBank<decoding_version>(raw_bank_index));
}

/**
 * @brief Calculates the number of hits to be decoded for the UT detector.
 */
template<int decoding_version>
__global__ void ut_calculate_number_of_hits::ut_calculate_number_of_hits_mep(
  ut_calculate_number_of_hits::Parameters parameters,
  const char* ut_boards,
  const unsigned* dev_ut_region_offsets,
  const unsigned* dev_unique_x_sector_layer_offsets,
  const unsigned* dev_unique_x_sector_offsets)
{
  const unsigned event_number = parameters.dev_event_list[blockIdx.x];

  const unsigned number_of_unique_x_sectors = dev_unique_x_sector_layer_offsets[UT::Constants::n_layers];
  uint32_t* hit_offsets = parameters.dev_ut_hit_sizes + event_number * number_of_unique_x_sectors;

  const UTBoards boards(ut_boards);
  auto const number_of_ut_raw_banks = parameters.dev_ut_raw_input_offsets[0];

  for (unsigned raw_bank_index = threadIdx.x; raw_bank_index < number_of_ut_raw_banks; raw_bank_index += blockDim.x) {

    // Construct UT raw bank from MEP layout
    const auto raw_bank = MEP::raw_bank<UTRawBank<decoding_version>>(
      parameters.dev_ut_raw_input, parameters.dev_ut_raw_input_offsets, event_number, raw_bank_index);

    calculate_number_of_hits(dev_ut_region_offsets, dev_unique_x_sector_offsets, hit_offsets, boards, raw_bank);
  }
}
