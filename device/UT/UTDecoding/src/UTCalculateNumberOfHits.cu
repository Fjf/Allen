#include "hip/hip_runtime.h"
/*****************************************************************************\
* (c) Copyright 2018-2020 CERN for the benefit of the LHCb Collaboration      *
\*****************************************************************************/
#include <MEPTools.h>
#include <UTCalculateNumberOfHits.cuh>
#include <UTRaw.cuh>

INSTANTIATE_ALGORITHM(ut_calculate_number_of_hits::ut_calculate_number_of_hits_t)

void ut_calculate_number_of_hits::ut_calculate_number_of_hits_t::set_arguments_size(
  ArgumentReferences<Parameters> arguments,
  const RuntimeOptions&,
  const Constants& constants,
  const HostBuffers&) const
{
  set_size<dev_ut_hit_sizes_t>(
    arguments,
    first<host_number_of_events_t>(arguments) * constants.host_unique_x_sector_layer_offsets[UT::Constants::n_layers]);
}

void ut_calculate_number_of_hits::ut_calculate_number_of_hits_t::operator()(
  const ArgumentReferences<Parameters>& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers&,
  const Allen::Context& context) const
{
  Allen::memset_async<dev_ut_hit_sizes_t>(arguments, 0, context);

  auto const bank_version = first<host_raw_bank_version_t>(arguments);
  if (bank_version < 0) return; // no UT banks present in data

  auto fun = bank_version == 4 ? (runtime_options.mep_layout ? global_function(ut_calculate_number_of_hits<4, true>) :
                                                               global_function(ut_calculate_number_of_hits<4, false>)) :
                                 (runtime_options.mep_layout ? global_function(ut_calculate_number_of_hits<3, true>) :
                                                               global_function(ut_calculate_number_of_hits<3, false>));
  fun(dim3(size<dev_event_list_t>(arguments)), property<block_dim_t>(), context)(
    arguments,
    constants.dev_ut_boards,
    constants.dev_ut_region_offsets.data(),
    constants.dev_unique_x_sector_layer_offsets.data(),
    constants.dev_unique_x_sector_offsets.data());
}

/**
 * @brief Given a UT RawBank, this function calculates the number of hits in a sector_group
  ("virtual" structure for optimized processing; a group of sectors where the start X is of a certain value).
 */
template<int decoding_version>
__device__ void calculate_number_of_hits(
  unsigned const*,
  unsigned const*,
  uint32_t*,
  UTBoards const&,
  UTRawBank<decoding_version> const&)
{}

template<>
__device__ void calculate_number_of_hits<3>(
  unsigned const* dev_ut_region_offsets,
  unsigned const* dev_unique_x_sector_offsets,
  uint32_t* hit_offsets,
  UTBoards const& boards,
  UTRawBank<3> const& raw_bank)
{
  const uint32_t m_nStripsPerHybrid = boards.stripsPerHybrids[raw_bank.sourceID];

  for (unsigned i = threadIdx.y; i < raw_bank.get_n_hits(); i += blockDim.y) {
    const uint32_t channelID = (raw_bank.data[i] & UT::Decoding::v4::chan_mask) >> UT::Decoding::v4::chan_offset;
    const uint32_t index = channelID / m_nStripsPerHybrid;
    const uint32_t fullChanIndex = raw_bank.sourceID * UT::Decoding::ut_number_of_sectors_per_board + index;
    if (fullChanIndex >= boards.number_of_channels) continue;
    const uint32_t station = boards.stations[fullChanIndex] - 1;
    const uint32_t layer = boards.layers[fullChanIndex] - 1;
    const uint32_t detRegion = boards.detRegions[fullChanIndex] - 1;
    const uint32_t sector = boards.sectors[fullChanIndex] - 1;

    // Calculate the index to get the geometry of the board
    const uint32_t idx = station * UT::Decoding::ut_number_of_sectors_per_board + layer * 3 + detRegion;
    const uint32_t idx_offset = dev_ut_region_offsets[idx] + sector;

    unsigned* hits_sector_group = hit_offsets + dev_unique_x_sector_offsets[idx_offset];
    atomicAdd(hits_sector_group, 1);
  }
}

template<>
__device__ void calculate_number_of_hits<4>(
  unsigned const* dev_ut_region_offsets,
  unsigned const* dev_unique_x_sector_offsets,
  uint32_t* hit_offsets,
  UTBoards const& boards,
  UTRawBank<4> const& raw_bank)
{
  if (raw_bank.get_n_hits() == 0) return;
  for (unsigned lane = threadIdx.y; lane < UT::Decoding::v5::n_lanes; lane += blockDim.y) {
    if (raw_bank.number_of_hits[lane] == 0) continue;
    // find the sector group to which these hits are added
    const uint32_t fullChanIndex = raw_bank.sourceID * UT::Decoding::ut_number_of_sectors_per_board + lane;
    assert(fullChanIndex < boards.number_of_channels);
    const uint32_t s = boards.stations[fullChanIndex];
    if (s == 0) continue;
    // Looking downstream, there are 2 stations UTa with X and U layer and UTb with V and X layer
    const uint32_t station = s - 1;
    const uint32_t layer = boards.layers[fullChanIndex] - 1;
    // The region corresponds to the 3 types of staves that mount the
    // 4 different sensor types (A in the outer, B in the central, C and D in the inner region).
    const uint32_t detRegion = boards.detRegions[fullChanIndex] - 1;
    const uint32_t sector = boards.sectors[fullChanIndex] - 1;

    // add the hits to the global counters and offsets
    const uint32_t idx = station * UT::Decoding::ut_number_of_sectors_per_board + layer * 3 + detRegion;
    assert(idx < UT::Decoding::v5::max_region_index);
    const uint32_t idx_offset = dev_ut_region_offsets[idx] + sector;
    unsigned* hits_sector_group = hit_offsets + dev_unique_x_sector_offsets[idx_offset];
    atomicAdd(hits_sector_group, raw_bank.number_of_hits[lane]);
  }
}

/**
 * @brief Calculates the number of hits to be decoded for the UT detector.
 */
template<int decoding_version, bool mep>
__global__ void ut_calculate_number_of_hits::ut_calculate_number_of_hits(
  ut_calculate_number_of_hits::Parameters parameters,
  const char* ut_boards,
  const unsigned* dev_ut_region_offsets,
  const unsigned* dev_unique_x_sector_layer_offsets,
  const unsigned* dev_unique_x_sector_offsets)
{
  const unsigned event_number = parameters.dev_event_list[blockIdx.x];

  const unsigned number_of_unique_x_sectors = dev_unique_x_sector_layer_offsets[UT::Constants::n_layers];
  uint32_t* hit_offsets = parameters.dev_ut_hit_sizes + event_number * number_of_unique_x_sectors;
  const UTBoards boards {ut_boards};
  const UTRawEvent<mep> raw_event {
    parameters.dev_ut_raw_input, parameters.dev_ut_raw_input_offsets, parameters.dev_ut_raw_input_sizes, event_number};
  for (unsigned raw_bank_index = threadIdx.x; raw_bank_index < raw_event.number_of_raw_banks();
       raw_bank_index += blockDim.x) {
    UTRawBank<decoding_version> bank = raw_event.template raw_bank<decoding_version>(raw_bank_index);
    calculate_number_of_hits(dev_ut_region_offsets, dev_unique_x_sector_offsets, hit_offsets, boards, bank);
  }
}
