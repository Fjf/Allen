#include "hip/hip_runtime.h"
/*****************************************************************************\
* (c) Copyright 2018-2020 CERN for the benefit of the LHCb Collaboration      *
\*****************************************************************************/
#include "PrefixSum.cuh"

/**
 * @brief Up-Sweep
 */
__device__ void up_sweep_2048(unsigned* data_block)
{
  unsigned starting_elem = 1;
  for (unsigned i = 2; i <= 2048; i <<= 1) {
    for (unsigned j = 0; j < (2047 + blockDim.x) / i; ++j) {
      const unsigned element = starting_elem + (j * blockDim.x + threadIdx.x) * i;
      if (element < 2048) {
        data_block[element] += data_block[element - (i >> 1)];
      }
    }
    starting_elem += i;
    __syncthreads();
  }
}

/**
 * @brief Down-sweep
 */
__device__ void down_sweep_2048(unsigned* data_block)
{
  for (unsigned i = 2048; i >= 2; i >>= 1) {
    for (unsigned j = 0; j < (2047 + blockDim.x) / i; ++j) {
      const auto element = 2047 - (j * blockDim.x + threadIdx.x) * i;
      if (element < 2048) {
        const auto other_element = element - (i >> 1);
        const auto value = data_block[other_element];
        data_block[other_element] = data_block[element];
        data_block[element] += value;
      }
    }
    __syncthreads();
  }
}

__device__ void prefix_sum_single_block_implementation(
  unsigned* dev_total_sum,
  unsigned* dev_array,
  const unsigned array_size,
  unsigned* data_block)
{
  // Prefix sum of elements in dev_array
  // Using Blelloch scan https://www.youtube.com/watch?v=mmYv3Haj6uc

  // Let's do it in blocks of 2048 (2^11)
  unsigned prev_last_elem = 0;
  for (unsigned block = 0; block < (array_size >> 11); ++block) {
    const unsigned first_elem = block << 11;

    // Load elements into shared memory, add prev_last_elem
    data_block[2 * threadIdx.x] = dev_array[first_elem + 2 * threadIdx.x];
    data_block[2 * threadIdx.x + 1] = dev_array[first_elem + 2 * threadIdx.x + 1];

    __syncthreads();

    up_sweep_2048((unsigned*) &data_block[0]);

    const unsigned new_last_elem = data_block[2047];

    __syncthreads();
    data_block[2047] = 0;
    __syncthreads();

    down_sweep_2048((unsigned*) &data_block[0]);

    // Store back elements
    dev_array[first_elem + 2 * threadIdx.x] = data_block[2 * threadIdx.x] + prev_last_elem;
    dev_array[first_elem + 2 * threadIdx.x + 1] = data_block[2 * threadIdx.x + 1] + prev_last_elem;
    prev_last_elem += new_last_elem;

    __syncthreads();
  }

  // Last iteration is special because
  // it may contain an unspecified number of elements
  const auto elements_remaining = array_size & 0x7FF; // % 2048
  if (elements_remaining > 0) {
    const auto first_elem = array_size - elements_remaining;

    // Initialize all elements to zero
    data_block[2 * threadIdx.x] = 0;
    data_block[2 * threadIdx.x + 1] = 0;

    // Load elements
    const auto elem_index = first_elem + 2 * threadIdx.x;
    if (elem_index < array_size) {
      data_block[2 * threadIdx.x] = dev_array[elem_index];
    }
    if ((elem_index + 1) < array_size) {
      data_block[2 * threadIdx.x + 1] = dev_array[elem_index + 1];
    }

    __syncthreads();

    up_sweep_2048((unsigned*) &data_block[0]);

    // Store sum of all elements
    if (threadIdx.x == 0) {
      dev_total_sum[0] = prev_last_elem + data_block[2047];
    }

    __syncthreads();
    data_block[2047] = 0;
    __syncthreads();

    down_sweep_2048((unsigned*) &data_block[0]);

    // Store back elements
    if (elem_index < array_size) {
      dev_array[elem_index] = data_block[2 * threadIdx.x] + prev_last_elem;
    }
    if ((elem_index + 1) < array_size) {
      dev_array[elem_index + 1] = data_block[2 * threadIdx.x + 1] + prev_last_elem;
    }
  }
  else {
    // Special case where number of elements is binary
    if (threadIdx.x == 0) {
      dev_total_sum[0] = prev_last_elem;
    }
  }
}

__global__ void prefix_sum_single_block(unsigned* dev_total_sum, unsigned* dev_array, const unsigned array_size)
{
  __shared__ unsigned data_block[2048];

  prefix_sum_single_block_implementation(dev_total_sum, dev_array, array_size, data_block);
}
