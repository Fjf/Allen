#include "hip/hip_runtime.h"
#include <CaloSeedClusters.cuh>

__device__ void seed_clusters(CaloDigit const* digits,
                              CaloSeedCluster* clusters,
                              unsigned* num_clusters,
                              const CaloGeometry& geometry,
                              const uint16_t min_adc) {
  // Loop over all CellIDs.
  for (uint i = threadIdx.x; i < geometry.max_cellid; i += blockDim.x) {
    uint16_t adc = digits[i].adc;
    if (adc == 0) {
      continue;
    }
    uint16_t* neighbors = &(geometry.neighbors[i * Calo::Constants::max_neighbours]);
    bool is_max = true;
    for (uint n = 0; n < Calo::Constants::max_neighbours; n++) {
      is_max = is_max && (adc > digits[neighbors[n]].adc);
    }
    if (is_max && digits[i].adc > min_adc) {
      auto id = atomicAdd(num_clusters, 1);
      clusters[id] = CaloSeedCluster(i, digits[i].adc, geometry.getX(i), geometry.getY(i));
    }
  }
}

__global__ void calo_seed_clusters::calo_seed_clusters(
  calo_seed_clusters::Parameters parameters,
  const char* raw_ecal_geometry,
  const char* raw_hcal_geometry,
  const uint16_t ecal_min_adc,
  const uint16_t hcal_min_adc)
{
  unsigned const event_number = blockIdx.x;

  // Get geometry.
  auto ecal_geometry = CaloGeometry(raw_ecal_geometry, Calo::Constants::ecal_max_cellid);
  auto hcal_geometry = CaloGeometry(raw_hcal_geometry, Calo::Constants::hcal_max_cellid);

  // ECal
  seed_clusters(parameters.dev_ecal_digits + (event_number * ecal_geometry.max_cellid),
                parameters.dev_ecal_seed_clusters + event_number,
                parameters.dev_ecal_num_clusters + event_number,
                ecal_geometry, ecal_min_adc);

  // HCal
  seed_clusters(parameters.dev_hcal_digits + (event_number * hcal_geometry.max_cellid),
                parameters.dev_hcal_seed_clusters + event_number,
                parameters.dev_hcal_num_clusters + event_number,
                hcal_geometry, hcal_min_adc);
}

void calo_seed_clusters::calo_seed_clusters_t::set_arguments_size(
  ArgumentReferences<Parameters> arguments,
  const RuntimeOptions&,
  const Constants&,
  const HostBuffers&) const
{
  auto const n_events = first<host_number_of_selected_events_t>(arguments);
  set_size<dev_ecal_num_clusters_t>(arguments, n_events);
  set_size<dev_hcal_num_clusters_t>(arguments, n_events);

  // TODO: get this from the geometry too
  set_size<dev_ecal_seed_clusters_t>(arguments, Calo::Constants::ecal_max_cells * n_events);
  set_size<dev_hcal_seed_clusters_t>(arguments, Calo::Constants::hcal_max_cells * n_events);
}

void calo_seed_clusters::calo_seed_clusters_t::operator()(
  const ArgumentReferences<Parameters>& arguments,
  const RuntimeOptions&,
  const Constants& constants,
  HostBuffers&,
  hipStream_t& hip_stream,
  hipEvent_t&) const
{
  initialize<dev_ecal_num_clusters_t>(arguments, 0, hip_stream);
  initialize<dev_hcal_num_clusters_t>(arguments, 0, hip_stream);

  // Find local maxima.
  global_function(calo_seed_clusters)(
    dim3(first<host_number_of_selected_events_t>(arguments)), dim3(property<block_dim_x_t>().get()), hip_stream)(
    arguments, constants.dev_ecal_geometry, constants.dev_hcal_geometry,
    property<ecal_min_adc_t>().get(), property<hcal_min_adc_t>().get());
}
