#include "hip/hip_runtime.h"
#include <CaloCluster.cuh>
#include <CaloFindClusters.cuh>
#include <iostream>

__device__ void simple_clusters(CaloDigit const* digits,
                                CaloSeedCluster const* seed_clusters,
                                CaloCluster* clusters,
                                unsigned const num_clusters,
                                const CaloGeometry& calo) {
  for (unsigned c = threadIdx.x; c < num_clusters; c += blockDim.x) {
    auto const& seed_cluster = seed_clusters[c];
    auto& cluster = clusters[c];
    cluster.center_id = seed_cluster.id;
    cluster.e = calo.pedestal + seed_cluster.adc * calo.gain[seed_cluster.id];
    cluster.x = seed_cluster.x;
    cluster.y = seed_cluster.y;

    uint16_t const* neighbors = &(calo.neighbors[seed_cluster.id * Calo::Constants::max_neighbours]);
    for (uint16_t n = 0; n < Calo::Constants::max_neighbours; n++) {
      auto const n_id = neighbors[n];
      int16_t adc = digits[n_id].adc;
      if (n_id != 0 && (adc != SHRT_MAX)) {
        cluster.e += calo.pedestal + adc * calo.gain[n_id];
        cluster.digits[n] = n_id;
      } else {
        cluster.digits[n] = 0;
      }
    }

    // Code to update position, double check
    // for (uint16_t n = 0; n < Calo::Constants::max_neighbours; n++) {
    //   auto const n_id = neighbors[n];
    //   if (n_id != 0 && ((auto const adc = digits[n_id].adc) != SHRT_MAX)) {
    //     float const adc_frac = float(adc) / float(cluster.e);
    //     cluster.x += adc_frac * (geometry.getX(n_id) - seed_cluster.x);
    //     cluster.y += adc_frac * (geometry.getY(n_id) - seed_cluster.y);
    //   }
    // }
  }
}

__global__ void calo_find_clusters::calo_find_clusters(
  calo_find_clusters::Parameters parameters,
  const char* raw_ecal_geometry,
  const char* raw_hcal_geometry,
  const unsigned)
{
  // Get proper geometry.
  auto ecal_geometry = CaloGeometry(raw_ecal_geometry);
  auto hcal_geometry = CaloGeometry(raw_hcal_geometry);

  unsigned const event_number = parameters.dev_event_list[blockIdx.x];

  // Build simple 3x3 clusters from seed clusters
  // Ecal
  unsigned const ecal_offset = parameters.dev_ecal_cluster_offsets[event_number];
  unsigned const ecal_num_clusters = parameters.dev_ecal_cluster_offsets[event_number + 1] - ecal_offset;
  simple_clusters(parameters.dev_ecal_digits + (event_number * ecal_geometry.max_index),
                  parameters.dev_ecal_seed_clusters + ecal_geometry.max_index * event_number,
                  parameters.dev_ecal_clusters + ecal_offset,
                  ecal_num_clusters, ecal_geometry);

  // Hcal
  unsigned const hcal_offset = parameters.dev_hcal_cluster_offsets[event_number];
  unsigned const hcal_num_clusters = parameters.dev_hcal_cluster_offsets[event_number + 1] - hcal_offset;
  simple_clusters(parameters.dev_hcal_digits + (event_number * hcal_geometry.max_index),
                  parameters.dev_hcal_seed_clusters + hcal_geometry.max_index * event_number,
                  parameters.dev_hcal_clusters + ecal_offset,
                  hcal_num_clusters, hcal_geometry);

}

void calo_find_clusters::calo_find_clusters_t::set_arguments_size(
  ArgumentReferences<Parameters> arguments,
  const RuntimeOptions&,
  const Constants&,
  const HostBuffers&) const
{
  auto const n_ecal_clusters = first<host_ecal_number_of_clusters_t>(arguments);
  set_size<dev_ecal_digits_clusters_t>(arguments, n_ecal_clusters);
  set_size<dev_ecal_clusters_t>(arguments, n_ecal_clusters);

  auto const n_hcal_clusters = first<host_hcal_number_of_clusters_t>(arguments);
  set_size<dev_hcal_digits_clusters_t>(arguments, n_hcal_clusters);
  set_size<dev_hcal_clusters_t>(arguments, n_hcal_clusters);
}


__host__ void calo_find_clusters::calo_find_clusters_t::operator()(
  const ArgumentReferences<Parameters>& arguments,
  const RuntimeOptions&,
  const Constants& constants,
  HostBuffers&,
  hipStream_t& hip_stream,
  hipEvent_t&) const
{
  // Find clusters.
  global_function(calo_find_clusters)(
    dim3(size<dev_event_list_t>(arguments)), dim3(property<block_dim_x_t>().get()), hip_stream)(
    arguments,
    constants.dev_ecal_geometry,
    constants.dev_hcal_geometry,
    property<iterations_t>().get());

  if (runtime_options.do_check) {
    safe_assign_to_host_buffer<dev_ecal_cluster_offsets_t>(host_buffers.host_ecal_cluster_offsets, arguments, hip_stream);
    safe_assign_to_host_buffer<dev_hcal_cluster_offsets_t>(host_buffers.host_hcal_cluster_offsets, arguments, hip_stream);
    safe_assign_to_host_buffer<dev_ecal_clusters_t>(host_buffers.host_ecal_clusters, arguments, hip_stream);
    safe_assign_to_host_buffer<dev_hcal_clusters_t>(host_buffers.host_hcal_clusters, arguments, hip_stream);
  }
}
