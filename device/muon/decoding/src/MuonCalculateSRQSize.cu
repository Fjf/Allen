#include "hip/hip_runtime.h"
/*****************************************************************************\
* (c) Copyright 2018-2020 CERN for the benefit of the LHCb Collaboration      *
\*****************************************************************************/
#include <MEPTools.h>
#include <MuonCalculateSRQSize.cuh>

INSTANTIATE_ALGORITHM(muon_calculate_srq_size::muon_calculate_srq_size_t)

__device__ void calculate_srq_size(
  Muon::MuonRawToHits const* muon_raw_to_hits,
  int const batch_index,
  Muon::MuonRawBank const& raw_bank,
  unsigned int* storage_station_region_quarter_sizes)
{
  const auto tell_number = raw_bank.sourceID;

  const uint16_t* p = raw_bank.data;

  // Note: Review this logic
  p += (*p + 3) & 0xFFFE;
  for (int j = 0; j < batch_index; ++j) {
    p += 1 + *p;
  }

  const auto batch_size = *p;
  for (int j = 1; j < batch_size + 1; ++j) {
    const auto pp = *(p + j);
    const auto add = (pp & 0x0FFF);
    const auto tileId = muon_raw_to_hits->muonGeometry->getADDInTell1(tell_number, add);

    if (tileId != 0) {
      const auto tile = Muon::MuonTileID(tileId);

      const auto x1 =
        getLayoutX(muon_raw_to_hits->muonTables, Muon::MuonTables::stripXTableNumber, tile.station(), tile.region());
      const auto y1 =
        getLayoutY(muon_raw_to_hits->muonTables, Muon::MuonTables::stripXTableNumber, tile.station(), tile.region());
      const auto x2 =
        getLayoutX(muon_raw_to_hits->muonTables, Muon::MuonTables::stripYTableNumber, tile.station(), tile.region());
      const auto y2 =
        getLayoutY(muon_raw_to_hits->muonTables, Muon::MuonTables::stripYTableNumber, tile.station(), tile.region());
      const auto layout1 = (x1 > x2 ? Muon::MuonLayout {x1, y1} : Muon::MuonLayout {x2, y2});

      // Store tiles according to their station, region, quarter and layout,
      // to prepare data for easy process in muonaddcoordscrossingmaps.
      const auto storage_srq_layout =
        Muon::Constants::n_layouts * tile.stationRegionQuarter() + (tile.layout() != layout1);
      atomicAdd(storage_station_region_quarter_sizes + storage_srq_layout, 1);
    }
  }
}

template<bool mep_layout>
__global__ void muon_calculate_srq_size_kernel(muon_calculate_srq_size::Parameters parameters)
{
  const unsigned event_number = parameters.dev_event_list[blockIdx.x];
  const auto raw_event =
    Muon::RawEvent<mep_layout> {parameters.dev_muon_raw, parameters.dev_muon_raw_offsets, parameters.dev_muon_raw_sizes, event_number};
  unsigned* storage_station_region_quarter_sizes =
    parameters.dev_storage_station_region_quarter_sizes + event_number * Muon::Constants::n_layouts *
                                                            Muon::Constants::n_stations * Muon::Constants::n_regions *
                                                            Muon::Constants::n_quarters;

  // number_of_raw_banks = 10
  // batches_per_bank = 4
  constexpr uint32_t batches_per_bank_mask = 0x3;
  constexpr uint32_t batches_per_bank_shift = 2;
  for (unsigned i = threadIdx.x; i < raw_event.number_of_raw_banks() * Muon::MuonRawEvent::batches_per_bank;
       i += blockDim.x) {
    const auto bank_index = i >> batches_per_bank_shift;
    const auto batch_index = i & batches_per_bank_mask;
    const auto raw_bank = raw_event.raw_bank(bank_index);

    calculate_srq_size(parameters.dev_muon_raw_to_hits, batch_index, raw_bank, storage_station_region_quarter_sizes);
  }
}

void muon_calculate_srq_size::muon_calculate_srq_size_t::set_arguments_size(
  ArgumentReferences<Parameters> arguments,
  const RuntimeOptions&,
  const Constants&,
  const HostBuffers&) const
{
  set_size<dev_muon_raw_to_hits_t>(arguments, 1);
  set_size<dev_storage_station_region_quarter_sizes_t>(
    arguments,
    first<host_number_of_events_t>(arguments) * Muon::Constants::n_layouts * Muon::Constants::n_stations *
      Muon::Constants::n_regions * Muon::Constants::n_quarters);
}

void muon_calculate_srq_size::muon_calculate_srq_size_t::operator()(
  const ArgumentReferences<Parameters>& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers&,
  const Allen::Context& context) const
{
  // FIXME: this should be done as part of the consumers, but
  // currently it cannot. This is because it is not possible to
  // indicate dependencies between Consumer and/or Producers.
  Muon::MuonRawToHits muonRawToHits {constants.dev_muon_tables, constants.dev_muon_geometry};

  Allen::memcpy_async(
    data<dev_muon_raw_to_hits_t>(arguments), &muonRawToHits, sizeof(muonRawToHits), Allen::memcpyHostToDevice, context);

  initialize<dev_storage_station_region_quarter_sizes_t>(arguments, 0, context);

  global_function(
    runtime_options.mep_layout ? muon_calculate_srq_size_kernel<true> : muon_calculate_srq_size_kernel<false>)(
    size<dev_event_list_t>(arguments),
    // FIXME
    10 * Muon::MuonRawEvent::batches_per_bank,
    context)(arguments);
}
