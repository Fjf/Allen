#include "hip/hip_runtime.h"
/*****************************************************************************\
* (c) Copyright 2018-2020 CERN for the benefit of the LHCb Collaboration      *
\*****************************************************************************/
#include <MEPTools.h>
#include <MuonCalculateSRQSize.cuh>
#include <gsl/gsl>
#include "MuonDefinitions.cuh"


INSTANTIATE_ALGORITHM(muon_calculate_srq_size::muon_calculate_srq_size_t)

__device__ void calculate_srq_size_tell1(
  Muon::MuonRawToHits const* muon_raw_to_hits,
  int const batch_index,
  Muon::MuonRawBank<2> const& raw_bank,
  unsigned int* storage_station_region_quarter_sizes)
{
  const auto tell_number = raw_bank.sourceID;
  const uint16_t* p = raw_bank.data;

  // Note: Review this logic
  p += (*p + 3) & 0xFFFE;
  for (int j = 0; j < batch_index; ++j) {
    p += 1 + *p;
  }

  const auto batch_size = *p;
  for (int j = 1; j < batch_size + 1; ++j) {
    const auto pp = *(p + j);
    const auto add = (pp & 0x0FFF);
    const auto tileId = muon_raw_to_hits->muonGeometry->getADDInTell1(tell_number, add);

    if (tileId != 0) {
      const auto tile = Muon::MuonTileID(tileId);

      const auto x1 =
        getLayoutX(muon_raw_to_hits->muonTables, Muon::MuonTables::stripXTableNumber, tile.station(), tile.region());
      const auto y1 =
        getLayoutY(muon_raw_to_hits->muonTables, Muon::MuonTables::stripXTableNumber, tile.station(), tile.region());
      const auto x2 =
        getLayoutX(muon_raw_to_hits->muonTables, Muon::MuonTables::stripYTableNumber, tile.station(), tile.region());
      const auto y2 =
        getLayoutY(muon_raw_to_hits->muonTables, Muon::MuonTables::stripYTableNumber, tile.station(), tile.region());
      const auto layout1 = (x1 > x2 ? Muon::MuonLayout {x1, y1} : Muon::MuonLayout {x2, y2});

      // Store tiles according to their station, region, quarter and layout,
      // to prepare data for easy process in muonaddcoordscrossingmaps.
      const auto storage_srq_layout =
        Muon::Constants::n_layouts * tile.stationRegionQuarter() + (tile.layout() != layout1);
      atomicAdd(storage_station_region_quarter_sizes + storage_srq_layout, 1);
    }
  }
}


__device__ void calculate_srq_size_tell40(
  Muon::MuonRawToHits const* muon_raw_to_hits,
  int const batch_index,
  Muon::MuonRawBank<3> const& raw_bank,
  unsigned int* storage_station_region_quarter_sizes)
{

  // printf("=======+++==========================================================\n" );
  // for (auto i = 1; i < Muon::Constants::maxTell40Number; i++){
  //   printf("Tell40 number %u \n", i );
  //   for (auto j = 0; j < Muon::Constants::maxTell40PCINumber; j++){
  //     printf("        pci number %u \n", j );
  //     for (auto k = 0; k < Muon::Constants::maxNumberLinks; k++){
  // 	printf( "           link number %u \n", k );
  // 	printf("            QuarterOfLink %u \n" ,  muon_raw_to_hits -> muonGeometry -> QuarterOfLink(i, j, k) ); 
  // 	printf("            RegionOfLink  %u \n" , muon_raw_to_hits -> muonGeometry -> RegionOfLink(i, j, k) ); 
  // 	// for (auto l = 0; l < MuonUpgradeDAQHelper_ODEFrameSize; l++){
  // 	//   printf("              ODE number " << l );
  // 	//   printf("              TileInTell40 map " << m_mapTileInTell40[i][j][k * MuonUpgradeDAQHelper_ODEFrameSize + l] );	  
  // 	// }
  //     }
  //   }    
  // }

  const auto tell_pci = raw_bank.sourceID & 0x00FF;
  const auto tell_number  = tell_pci/2 + 1;
  const auto pci_number   = tell_pci % 2;
  const auto tell_station = muon_raw_to_hits -> muonGeometry -> whichStationIsTell40( tell_number -1 );
  const auto active_links = muon_raw_to_hits -> muonGeometry -> NumberOfActiveLink ( tell_number, pci_number );
 
  printf("SRQSize: sourceID = %u, raw_bank.last - raw_bank.data = %ld, tell_station = %u, active_links = %u \n", raw_bank.sourceID,  raw_bank.last - raw_bank.data, tell_station, active_links);
  
  const gsl::span<const uint8_t> range8 {raw_bank.data, (raw_bank.last - raw_bank.data)/sizeof(uint8_t)};

  // for ( unsigned int i =0; i < range8.size(); i++){
  //   printf("SRQSize: range[%u] = %x \n", i, range8[i]);
  // }

  auto range_data = range8.subspan( 1 );
  unsigned int link_start_pointer = 0;
  unsigned int map_connected_fibers[24] = {};  
  unsigned int synch_evt = ( range8[0] & 0x10 ) >> 4;
  if ( !synch_evt ){
    unsigned int number_of_readout_fibers = muon_raw_to_hits -> muonGeometry -> get_number_of_readout_fibers( range8, active_links, map_connected_fibers);
    printf( "Number of readout fibers is %d \n", number_of_readout_fibers );
 
    unsigned int align_info         = ( range8[0] & 0x20 ) >> 5;
    if (align_info) link_start_pointer += 3;

    for ( unsigned int link = 0; link < number_of_readout_fibers; link++ ) {
      unsigned int reroutered_link = map_connected_fibers[link];
      
      auto regionOfLink  = muon_raw_to_hits -> muonGeometry -> RegionOfLink( tell_number, pci_number, reroutered_link );
      auto quarterOfLink = muon_raw_to_hits -> muonGeometry -> QuarterOfLink( tell_number, pci_number, reroutered_link );
      printf("at link %u, reroutered link %u, tell_number %u, pci_number %u, regionOfLink = %u, quarterOfLink = %u \n", link, reroutered_link, tell_number, pci_number, regionOfLink, quarterOfLink);
      
      uint8_t  curr_byte  = range_data[link_start_pointer];
      unsigned int size_of_link = ( ( curr_byte & 0xF0 ) >> 4 ) + 1;

      printf("size of link is %u, link_start_pointer is %u\n", size_of_link, link_start_pointer);

      if ( size_of_link > 1 ) {
	auto range_link_HitsMap = range_data.subspan( link_start_pointer, 7 );

	bool         first_hitmap_byte  = false;
	bool         last_hitmap_byte   = true;
	unsigned int count_byte         = 0;
	unsigned int pos_in_link        = 0;

	for ( auto r = range_link_HitsMap.rbegin(); r < range_link_HitsMap.rend(); r++ ) {
	  // loop in reverse mode hits map is 47->0
	  count_byte++;
	  if ( count_byte == 7 ) first_hitmap_byte = true;
	  if ( count_byte > 7 ) {
	    printf( "BREAKING BECAUSE OF COUNT BYTE \n" );
	    break; // should never happens
	  }
	  uint8_t data_copy = *r;
	  for ( unsigned int bit_pos_1 = 8; bit_pos_1 > 0; --bit_pos_1 ) {
	    unsigned int bit_pos = bit_pos_1 - 1;
	    
	    if ( first_hitmap_byte && bit_pos < 4 ) 
	      {
		printf( "CONTINUING BECAUSE first_hitmap_byte && bit_pos < 4\n" );
		continue; // should never happens
	      };
	    
	    if ( last_hitmap_byte && bit_pos > 3 ) 
	      {
		printf( "CONTINUING BECAUSE  last_hitmap_byte && bit_pos > 3\n" );
		continue; // should never happens
	      };
	   
	    if ( data_copy & Muon::Constants::single_bit_position[bit_pos] ) {
		auto tileId = muon_raw_to_hits -> muonGeometry ->TileInTell40( tell_number, pci_number, reroutered_link, pos_in_link );
		
		if (tileId != 0) {
		  const auto tile = Muon::MuonTileID(tileId);		  
		  const auto x1 =
		    getLayoutX(muon_raw_to_hits->muonTables, Muon::MuonTables::stripXTableNumber, tile.station(), tile.region());
		  const auto y1 =
		    getLayoutY(muon_raw_to_hits->muonTables, Muon::MuonTables::stripXTableNumber, tile.station(), tile.region());
		  const auto x2 =
		    getLayoutX(muon_raw_to_hits->muonTables, Muon::MuonTables::stripYTableNumber, tile.station(), tile.region());
		  const auto y2 =
		    getLayoutY(muon_raw_to_hits->muonTables, Muon::MuonTables::stripYTableNumber, tile.station(), tile.region());
		  const auto layout1 = (x1 > x2 ? Muon::MuonLayout {x1, y1} : Muon::MuonLayout {x2, y2});
		  
		  printf("x1 %u, y1 %u, x2 %u, y2 %u \n", x1, y1, x2, y2);
		  // Store tiles according to their station, region, quarter and layout,
		  // to prepare data for easy process in muonaddcoordscrossingmaps.		  
		  printf("tell_station * 16 + regionOfLink * 4 + quarterOfLink < 64:  %u \n", tell_station * 16 + regionOfLink * 4 + quarterOfLink < 64);
		  if ( tell_station * 16 + regionOfLink * 4 + quarterOfLink < 64 ) {
		    const auto storage_srq_layout = Muon::Constants::n_layouts * tile.stationRegionQuarter() + (tile.layout() != layout1);
		    atomicAdd(storage_station_region_quarter_sizes + storage_srq_layout, 1);
		    printf("storage_srq_layout is %u \n", storage_srq_layout);
		    printf("storage_station_region_quarter_sizes %u \n", *storage_station_region_quarter_sizes + storage_srq_layout);
		  }
                } 
	    }
	    pos_in_link++;
	  }
	  last_hitmap_byte = false;
        }
      }
      link_start_pointer = link_start_pointer + size_of_link;	
    }
  }

  //printf( "THIS IS THE END OF THE CALCULATESRQ SIZE \n" );
}

template<bool mep_layout>
__global__ void muon_calculate_srq_size_kernel(muon_calculate_srq_size::Parameters parameters, unsigned int muon_bank_version, unsigned int number_of_events)
{
  //const unsigned event_number = parameters.dev_event_list[blockIdx.x];
  //printf("Number of events = %u \n", number_of_events);

  for (unsigned event_index = 0; event_index < number_of_events; ++event_index) {
    const unsigned event_number = parameters.dev_event_list[event_index];
    //printf("at index %u, event_number = %u \n", event_index, event_number);

    // number_of_raw_banks = 10
    // batches_per_bank = 4
    unsigned* storage_station_region_quarter_sizes =
      parameters.dev_storage_station_region_quarter_sizes + event_number * Muon::Constants::n_layouts *
      Muon::Constants::n_stations * Muon::Constants::n_regions *
      Muon::Constants::n_quarters;
    
    if (muon_bank_version == 2){
      const auto raw_event = Muon::RawEvent<mep_layout, 2> {
	parameters.dev_muon_raw, parameters.dev_muon_raw_offsets, parameters.dev_muon_raw_sizes, event_number};
      
      constexpr uint32_t batches_per_bank_mask = 0x3;
      constexpr uint32_t batches_per_bank_shift = 2;
      for (unsigned i = threadIdx.x; i < raw_event.number_of_raw_banks() * Muon::batches_per_bank; i += blockDim.x) {
	const auto bank_index = i >> batches_per_bank_shift;
	const auto batch_index = i & batches_per_bank_mask;
	const auto raw_bank = raw_event.raw_bank(bank_index);    
	calculate_srq_size_tell1(parameters.dev_muon_raw_to_hits, batch_index, raw_bank, storage_station_region_quarter_sizes);
      }
    }
    else if (muon_bank_version == 3){
      const auto raw_event = Muon::RawEvent<mep_layout, 3> {
	parameters.dev_muon_raw, parameters.dev_muon_raw_offsets, parameters.dev_muon_raw_sizes, event_number};

      constexpr uint32_t batches_per_bank_mask = 0x3;
      constexpr uint32_t batches_per_bank_shift = 2;
      //for (unsigned i = threadIdx.x; i < raw_event.number_of_raw_banks() * Muon::batches_per_bank; i += blockDim.x) {
      for (unsigned i = 0; i < raw_event.number_of_raw_banks(); i += blockDim.x) {
     	// const auto bank_index = i >> batches_per_bank_shift;
	// const auto batch_index = i & batches_per_bank_mask;
	const int bank_index = i;
	const int batch_index = 0;
	const auto raw_bank = raw_event.raw_bank(i);    	
	//TODO: remove invalid banks (e.g. 26624 in standalone mode) 
	printf("I will enter in the new calculatingSRQsize part for event %u, bank_index %u, batch_index %u \n ", event_number, bank_index, batch_index);
	
	calculate_srq_size_tell40(parameters.dev_muon_raw_to_hits, batch_index, raw_bank, storage_station_region_quarter_sizes);
      }
    }
    else {
      throw StrException("MuonCalculateSRQSize : unrecognized muon raw bank version \n");
    }
  }
}

void muon_calculate_srq_size::muon_calculate_srq_size_t::set_arguments_size(
  ArgumentReferences<Parameters> arguments,
  const RuntimeOptions&,
  const Constants&,
  const HostBuffers&) const
{
  set_size<dev_muon_raw_to_hits_t>(arguments, 1);
  set_size<dev_storage_station_region_quarter_sizes_t>(
    arguments,
    first<host_number_of_events_t>(arguments) * Muon::Constants::n_layouts * Muon::Constants::n_stations *
      Muon::Constants::n_regions * Muon::Constants::n_quarters);
}

void muon_calculate_srq_size::muon_calculate_srq_size_t::operator()(
  const ArgumentReferences<Parameters>& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers&,
  const Allen::Context& context) const
{
  // FIXME: this should be done as part of the consumers, but
  // currently it cannot. This is because it is not possible to
  // indicate dependencies between Consumer and/or Producers.
  Muon::MuonRawToHits muonRawToHits {constants.dev_muon_tables, constants.dev_muon_geometry};

  Allen::memcpy_async(
    data<dev_muon_raw_to_hits_t>(arguments), &muonRawToHits, sizeof(muonRawToHits), Allen::memcpyHostToDevice, context);

  Allen::memset_async<dev_storage_station_region_quarter_sizes_t>(arguments, 0, context);
  const unsigned int muon_bank_version = first<host_raw_bank_version_t>(arguments);

  global_function( runtime_options.mep_layout ? muon_calculate_srq_size_kernel<true> : muon_calculate_srq_size_kernel<false>)(
    1, //size<dev_event_list_t>(arguments),    
    // FIXME
    1, //10 * Muon::batches_per_bank,
    context)(arguments, muon_bank_version, size<dev_event_list_t>(arguments));

  print<dev_storage_station_region_quarter_sizes_t>(arguments);


}
