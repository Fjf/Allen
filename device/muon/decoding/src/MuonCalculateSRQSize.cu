#include "hip/hip_runtime.h"
/*****************************************************************************\
* (c) Copyright 2018-2020 CERN for the benefit of the LHCb Collaboration      *
\*****************************************************************************/
#include <MEPTools.h>
#include <MuonCalculateSRQSize.cuh>

void muon_calculate_srq_size::muon_calculate_srq_size_t::set_arguments_size(
  ArgumentReferences<Parameters> arguments,
  const RuntimeOptions&,
  const Constants&,
  const HostBuffers&) const
{
  set_size<dev_muon_raw_to_hits_t>(arguments, 1);
  set_size<dev_storage_station_region_quarter_sizes_t>(
    arguments,
    first<host_number_of_events_t>(arguments) * Muon::Constants::n_layouts * Muon::Constants::n_stations *
      Muon::Constants::n_regions * Muon::Constants::n_quarters);
}

void muon_calculate_srq_size::muon_calculate_srq_size_t::operator()(
  const ArgumentReferences<Parameters>& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers&,
  hipStream_t& stream,
  hipEvent_t&) const
{
  // FIXME: this should be done as part of the consumers, but
  // currently it cannot. This is because it is not possible to
  // indicate dependencies between Consumer and/or Producers.
  Muon::MuonRawToHits muonRawToHits {constants.dev_muon_tables, constants.dev_muon_geometry};

  cudaCheck(hipMemcpyAsync(
    data<dev_muon_raw_to_hits_t>(arguments), &muonRawToHits, sizeof(muonRawToHits), hipMemcpyHostToDevice, stream));

  initialize<dev_storage_station_region_quarter_sizes_t>(arguments, 0, stream);

  if (runtime_options.mep_layout) {
    global_function(muon_calculate_srq_size_mep)(
      size<dev_event_list_t>(arguments),
      Muon::MuonRawEvent::number_of_raw_banks * Muon::MuonRawEvent::batches_per_bank,
      stream)(arguments);
  }
  else {
    global_function(muon_calculate_srq_size)(
      size<dev_event_list_t>(arguments),
      Muon::MuonRawEvent::number_of_raw_banks * Muon::MuonRawEvent::batches_per_bank,
      stream)(arguments);
  }
}

__device__ void calculate_srq_size(
  Muon::MuonRawToHits const* muon_raw_to_hits,
  int const batch_index,
  Muon::MuonRawBank const& raw_bank,
  unsigned int* storage_station_region_quarter_sizes)
{
  const auto tell_number = raw_bank.sourceID;

  uint16_t* p = raw_bank.data;

  // Note: Review this logic
  p += (*p + 3) & 0xFFFE;
  for (int j = 0; j < batch_index; ++j) {
    p += 1 + *p;
  }

  const auto batch_size = *p;
  for (int j = 1; j < batch_size + 1; ++j) {
    const auto pp = *(p + j);
    const auto add = (pp & 0x0FFF);
    const auto tileId = muon_raw_to_hits->muonGeometry->getADDInTell1(tell_number, add);

    if (tileId != 0) {
      const auto tile = Muon::MuonTileID(tileId);

      const auto x1 =
        getLayoutX(muon_raw_to_hits->muonTables, Muon::MuonTables::stripXTableNumber, tile.station(), tile.region());
      const auto y1 =
        getLayoutY(muon_raw_to_hits->muonTables, Muon::MuonTables::stripXTableNumber, tile.station(), tile.region());
      const auto x2 =
        getLayoutX(muon_raw_to_hits->muonTables, Muon::MuonTables::stripYTableNumber, tile.station(), tile.region());
      const auto y2 =
        getLayoutY(muon_raw_to_hits->muonTables, Muon::MuonTables::stripYTableNumber, tile.station(), tile.region());
      const auto layout1 = (x1 > x2 ? Muon::MuonLayout {x1, y1} : Muon::MuonLayout {x2, y2});

      // Store tiles according to their station, region, quarter and layout,
      // to prepare data for easy process in muonaddcoordscrossingmaps.
      const auto storage_srq_layout =
        Muon::Constants::n_layouts * tile.stationRegionQuarter() + (tile.layout() != layout1);
      atomicAdd(storage_station_region_quarter_sizes + storage_srq_layout, 1);
    }
  }
}

__global__ void muon_calculate_srq_size::muon_calculate_srq_size(muon_calculate_srq_size::Parameters parameters)
{
  const unsigned event_number = parameters.dev_event_list[blockIdx.x];
  const auto raw_event = Muon::MuonRawEvent(parameters.dev_muon_raw + parameters.dev_muon_raw_offsets[event_number]);
  unsigned* storage_station_region_quarter_sizes =
    parameters.dev_storage_station_region_quarter_sizes + event_number * Muon::Constants::n_layouts *
                                                            Muon::Constants::n_stations * Muon::Constants::n_regions *
                                                            Muon::Constants::n_quarters;

  // number_of_raw_banks = 10
  // batches_per_bank = 4
  constexpr uint32_t batches_per_bank_mask = 0x3;
  constexpr uint32_t batches_per_bank_shift = 2;
  for (unsigned i = threadIdx.x; i < Muon::MuonRawEvent::number_of_raw_banks * Muon::MuonRawEvent::batches_per_bank;
       i += blockDim.x) {
    const auto bank_index = i >> batches_per_bank_shift;
    const auto batch_index = i & batches_per_bank_mask;
    const auto raw_bank = raw_event.getMuonBank(bank_index);

    calculate_srq_size(parameters.dev_muon_raw_to_hits, batch_index, raw_bank, storage_station_region_quarter_sizes);
  }
}

__global__ void muon_calculate_srq_size::muon_calculate_srq_size_mep(muon_calculate_srq_size::Parameters parameters)
{
  const unsigned event_number = parameters.dev_event_list[blockIdx.x];
  unsigned* storage_station_region_quarter_sizes =
    parameters.dev_storage_station_region_quarter_sizes + event_number * Muon::Constants::n_layouts *
                                                            Muon::Constants::n_stations * Muon::Constants::n_regions *
                                                            Muon::Constants::n_quarters;

  // number_of_raw_banks = 10
  // batches_per_bank = 4
  constexpr uint32_t batches_per_bank_mask = 0x3;
  constexpr uint32_t batches_per_bank_shift = 2;
  for (unsigned i = threadIdx.x; i < Muon::MuonRawEvent::number_of_raw_banks * Muon::MuonRawEvent::batches_per_bank;
       i += blockDim.x) {
    const auto bank_index = i >> batches_per_bank_shift;
    const auto batch_index = i & batches_per_bank_mask;
    const auto raw_bank = MEP::raw_bank<Muon::MuonRawBank>(
      parameters.dev_muon_raw, parameters.dev_muon_raw_offsets, event_number, bank_index);

    calculate_srq_size(parameters.dev_muon_raw_to_hits, batch_index, raw_bank, storage_station_region_quarter_sizes);
  }
}
