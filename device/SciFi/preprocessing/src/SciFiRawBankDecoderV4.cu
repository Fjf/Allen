#include "hip/hip_runtime.h"
/*****************************************************************************\
* (c) Copyright 2018-2020 CERN for the benefit of the LHCb Collaboration      *
\*****************************************************************************/
#include <MEPTools.h>
#include <SciFiRawBankDecoderV4.cuh>
#include <assert.h>

void scifi_raw_bank_decoder_v4::scifi_raw_bank_decoder_v4_t::set_arguments_size(
  ArgumentReferences<Parameters> arguments,
  const RuntimeOptions&,
  const Constants&,
  const HostBuffers&) const
{
  set_size<dev_scifi_hits_t>(
    arguments,
    first<host_accumulated_number_of_scifi_hits_t>(arguments) * SciFi::Hits::number_of_arrays * sizeof(uint32_t));
}

void scifi_raw_bank_decoder_v4::scifi_raw_bank_decoder_v4_t::operator()(
  const ArgumentReferences<Parameters>& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers&,
  const Allen::Context& context) const
{
  if (runtime_options.mep_layout) {
    global_function(scifi_raw_bank_decoder_v4_mep)(
      dim3(size<dev_event_list_t>(arguments)), property<raw_bank_decoder_block_dim_t>(), context)(
      arguments, constants.dev_scifi_geometry);
    global_function(scifi_direct_decoder_v4_mep)(
      dim3(size<dev_event_list_t>(arguments)), property<direct_decoder_block_dim_t>(), context)(
      arguments, constants.dev_scifi_geometry);
  }
  else {
    global_function(scifi_raw_bank_decoder_v4)(
      dim3(size<dev_event_list_t>(arguments)), property<raw_bank_decoder_block_dim_t>(), context)(
      arguments, constants.dev_scifi_geometry);
    global_function(scifi_direct_decoder_v4)(
      dim3(size<dev_event_list_t>(arguments)), property<direct_decoder_block_dim_t>(), context)(
      arguments, constants.dev_scifi_geometry);
  }
}

using namespace SciFi;

// Merge of PrStoreFTHit and RawBankDecoder.
__device__ void make_cluster_v4(
  const int hit_index,
  const SciFiGeometry& geom,
  uint32_t chan,
  uint8_t fraction,
  uint8_t pseudoSize,
  SciFi::Hits& hits)
{
  const SciFi::SciFiChannelID id {chan};

  // Offset to save space in geometry structure, see DumpFTGeometry.cpp
  const uint32_t mat = id.uniqueMat() - 512;
  const uint32_t planeCode = id.uniqueLayer() - 4;
  const float dxdy = geom.dxdy[mat];
  const float dzdy = geom.dzdy[mat];
  float uFromChannel = geom.uBegin[mat] + (2 * id.channel() + 1 + fraction) * geom.halfChannelPitch[mat];
  if (id.die()) uFromChannel += geom.dieGap[mat];
  uFromChannel += id.sipm() * geom.sipmPitch[mat];
  const float endPointX = geom.mirrorPointX[mat] + geom.ddxX[mat] * uFromChannel;
  const float endPointY = geom.mirrorPointY[mat] + geom.ddxY[mat] * uFromChannel;
  const float endPointZ = geom.mirrorPointZ[mat] + geom.ddxZ[mat] * uFromChannel;
  const float x0 = endPointX - dxdy * endPointY;
  const float z0 = endPointZ - dzdy * endPointY;

  assert(pseudoSize < 9 && "Pseudosize of cluster is > 8. Out of range.");

  // Apparently the unique* methods are not designed to start at 0, therefore -16
  const uint32_t uniqueZone = ((id.uniqueQuarter() - 16) >> 1);

  const unsigned plane_code = 2 * planeCode + (uniqueZone % 2);
  hits.x0(hit_index) = x0;
  hits.z0(hit_index) = z0;
  hits.channel(hit_index) = chan;
  hits.endPointY(hit_index) = endPointY;
  assert(fraction <= 0x1 && plane_code <= 0x1f && pseudoSize <= 0xf && mat <= 0x7ff);
  hits.assembled_datatype(hit_index) = fraction << 20 | plane_code << 15 | pseudoSize << 11 | mat;
}

__global__ void scifi_raw_bank_decoder_v4::scifi_raw_bank_decoder_v4(
  scifi_raw_bank_decoder_v4::Parameters parameters,
  const char* scifi_geometry)
{
  const unsigned event_number = parameters.dev_event_list[blockIdx.x];
  const unsigned number_of_events = parameters.dev_number_of_events[0];

  const SciFiGeometry geom {scifi_geometry};
  const auto event =
    SciFiRawEvent(parameters.dev_scifi_raw_input + parameters.dev_scifi_raw_input_offsets[event_number]);

  SciFi::Hits hits {parameters.dev_scifi_hits,
                    parameters.dev_scifi_hit_offsets[number_of_events * SciFi::Constants::n_mat_groups_and_mats]};
  SciFi::ConstHitCount hit_count {parameters.dev_scifi_hit_offsets, event_number};
  const unsigned number_of_hits_in_last_zones = hit_count.number_of_hits_in_zones_without_mat_groups();

  for (unsigned i = threadIdx.x; i < number_of_hits_in_last_zones; i += blockDim.x) {
    const uint32_t cluster_reference =
      parameters.dev_cluster_references[hit_count.offset_zones_without_mat_groups() + i];
    const int raw_bank_number = (cluster_reference >> 8) & 0xFF;
    const int it_number = (cluster_reference) &0xFF;
    const auto rawbank = event.getSciFiRawBank(raw_bank_number);
    const uint16_t* it = rawbank.data + 2;
    it += it_number;

    const uint16_t c = *it;
    const uint32_t ch = geom.bank_first_channel[rawbank.sourceID] + channelInBank(c);

    // Call parameters for make_cluster
    uint32_t cluster_chan = ch;
    uint8_t cluster_fraction = fraction(c);
    uint8_t pseudoSize = cSize(c) ? 0 : 4;

    make_cluster_v4(
      hit_count.offset_zones_without_mat_groups() + i, geom, cluster_chan, cluster_fraction, pseudoSize, hits);
  }
}

__global__ void scifi_raw_bank_decoder_v4::scifi_raw_bank_decoder_v4_mep(
  scifi_raw_bank_decoder_v4::Parameters parameters,
  const char* scifi_geometry)
{
  const unsigned event_number = parameters.dev_event_list[blockIdx.x];
  const unsigned number_of_events = parameters.dev_number_of_events[0];

  const SciFiGeometry geom {scifi_geometry};

  SciFi::Hits hits {parameters.dev_scifi_hits,
                    parameters.dev_scifi_hit_offsets[number_of_events * SciFi::Constants::n_mat_groups_and_mats]};
  SciFi::ConstHitCount hit_count {parameters.dev_scifi_hit_offsets, event_number};
  const unsigned number_of_hits_in_last_zones = hit_count.number_of_hits_in_zones_without_mat_groups();

  for (unsigned i = threadIdx.x; i < number_of_hits_in_last_zones; i += blockDim.x) {
    const uint32_t cluster_reference =
      parameters.dev_cluster_references[hit_count.offset_zones_without_mat_groups() + i];

    const int raw_bank_number = (cluster_reference >> 8) & 0xFF;
    const int it_number = (cluster_reference) &0xFF;

    // Create SciFi raw bank from MEP layout
    auto const raw_bank = MEP::raw_bank<SciFiRawBank>(
      parameters.dev_scifi_raw_input, parameters.dev_scifi_raw_input_offsets, event_number, raw_bank_number);

    const uint16_t* it = raw_bank.data + 2;
    it += it_number;

    const uint16_t c = *it;
    const uint32_t ch = geom.bank_first_channel[raw_bank.sourceID] + channelInBank(c);

    // Call parameters for make_cluster
    uint32_t cluster_chan = ch;
    uint8_t cluster_fraction = fraction(c);
    uint8_t pseudoSize = cSize(c) ? 0 : 4;

    make_cluster_v4(
      hit_count.offset_zones_without_mat_groups() + i, geom, cluster_chan, cluster_fraction, pseudoSize, hits);
  }
}
