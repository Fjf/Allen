#include "hip/hip_runtime.h"
#include "LFQualityFilterLength.cuh"

void lf_quality_filter_length::lf_quality_filter_length_t::set_arguments_size(
  ArgumentReferences<Parameters> arguments,
  const RuntimeOptions&,
  const Constants&,
  const HostBuffers&) const
{
  set_size<dev_scifi_lf_length_filtered_tracks_t>(
    arguments,
    first<host_number_of_reconstructed_ut_tracks_t>(arguments) *
      LookingForward::maximum_number_of_candidates_per_ut_track);
  set_size<dev_scifi_lf_length_filtered_atomics_t>(
    arguments, first<host_number_of_selected_events_t>(arguments) * LookingForward::num_atomics);
  set_size<dev_scifi_lf_parametrization_length_filter_t>(
    arguments,
    4 * first<host_number_of_reconstructed_ut_tracks_t>(arguments) *
      LookingForward::maximum_number_of_candidates_per_ut_track);
}

void lf_quality_filter_length::lf_quality_filter_length_t::operator()(
  const ArgumentReferences<Parameters>& arguments,
  const RuntimeOptions&,
  const Constants&,
  HostBuffers&,
  hipStream_t& hip_stream,
  hipEvent_t&) const
{
  initialize<dev_scifi_lf_length_filtered_atomics_t>(arguments, 0, hip_stream);

  device_function(lf_quality_filter_length)(
    dim3(first<host_number_of_selected_events_t>(arguments)), property<block_dim_t>(), hip_stream)(arguments);
}

__global__ void lf_quality_filter_length::lf_quality_filter_length(lf_quality_filter_length::Parameters parameters)
{
  const auto event_number = blockIdx.x;
  const auto number_of_events = gridDim.x;

  // UT consolidated tracks
  UT::Consolidated::ConstTracks ut_tracks {
    parameters.dev_atomics_ut, parameters.dev_ut_track_hit_number, event_number, number_of_events};

  const auto ut_event_tracks_offset = ut_tracks.tracks_offset(event_number);
  const auto ut_total_number_of_tracks = ut_tracks.total_number_of_tracks();
  const auto number_of_tracks = parameters.dev_scifi_lf_atomics[event_number];

  for (uint i = threadIdx.x; i < number_of_tracks; i += blockDim.x) {
    const auto scifi_track_index =
      ut_event_tracks_offset * LookingForward::maximum_number_of_candidates_per_ut_track + i;
    const SciFi::TrackHits& track = parameters.dev_scifi_lf_tracks[scifi_track_index];

    if (track.hitsNum >= LookingForward::track_min_hits) {
      const auto insert_index = atomicAdd(parameters.dev_scifi_lf_length_filtered_atomics + event_number, 1);
      const auto new_scifi_track_index =
        ut_event_tracks_offset * LookingForward::maximum_number_of_candidates_per_ut_track + insert_index;

      parameters.dev_scifi_lf_length_filtered_tracks[new_scifi_track_index] = track;

      // Save track parameters to new container as well
      const auto a1 = parameters.dev_scifi_lf_parametrization[scifi_track_index];
      const auto b1 =
        parameters.dev_scifi_lf_parametrization
          [ut_total_number_of_tracks * LookingForward::maximum_number_of_candidates_per_ut_track + scifi_track_index];
      const auto c1 = parameters.dev_scifi_lf_parametrization
                        [2 * ut_total_number_of_tracks * LookingForward::maximum_number_of_candidates_per_ut_track +
                         scifi_track_index];
      const auto d_ratio =
        parameters.dev_scifi_lf_parametrization
          [3 * ut_total_number_of_tracks * LookingForward::maximum_number_of_candidates_per_ut_track +
           scifi_track_index];

      parameters.dev_scifi_lf_parametrization_length_filter[new_scifi_track_index] = a1;
      parameters.dev_scifi_lf_parametrization_length_filter
        [ut_total_number_of_tracks * LookingForward::maximum_number_of_candidates_per_ut_track +
         new_scifi_track_index] = b1;
      parameters.dev_scifi_lf_parametrization_length_filter
        [2 * ut_total_number_of_tracks * LookingForward::maximum_number_of_candidates_per_ut_track +
         new_scifi_track_index] = c1;
      parameters.dev_scifi_lf_parametrization_length_filter
        [3 * ut_total_number_of_tracks * LookingForward::maximum_number_of_candidates_per_ut_track +
         new_scifi_track_index] = d_ratio;
    }
  }
}
