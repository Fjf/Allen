#include "hip/hip_runtime.h"
/*****************************************************************************\
* (c) Copyright 2022 CERN for the benefit of the LHCb Collaboration           *
*                                                                             *
* This software is distributed under the terms of the Apache License          *
* version 2 (Apache-2.0), copied verbatim in the file "COPYING".              *
*                                                                             *
* In applying this licence, CERN does not waive the privileges and immunities *
* granted to it by virtue of its status as an Intergovernmental Organization  *
* or submit itself to any jurisdiction.                                       *
\*****************************************************************************/
#include "MakeLumiSummary.cuh"

#include "SelectionsEventModel.cuh"
#include "Event/ODIN.h"

INSTANTIATE_ALGORITHM(make_lumi_summary::make_lumi_summary_t)

void make_lumi_summary::make_lumi_summary_t::set_arguments_size(
  ArgumentReferences<Parameters> arguments,
  const RuntimeOptions&,
  const Constants&) const
{
  set_size<host_lumi_summary_offsets_t>(arguments, size<dev_lumi_summary_offsets_t>(arguments));
  set_size<host_lumi_summaries_t>(arguments, first<host_lumi_summaries_size_t>(arguments));
  set_size<dev_lumi_summaries_t>(arguments, first<host_lumi_summaries_size_t>(arguments));
}

void make_lumi_summary::make_lumi_summary_t::init()
{
  std::map<std::string, std::pair<unsigned, unsigned>> schema = property<lumi_counter_schema_t>();
  std::array<unsigned, 2 * Lumi::Constants::n_basic_counters> basic_offsets_and_sizes =
    property<basic_offsets_and_sizes_t>();

  unsigned c_idx(0u);
  for (auto counter_name : Lumi::Constants::basic_counter_names) {
    if (schema.find(counter_name) == schema.end()) {
      std::cout << "LumiSummary schema does not use " << counter_name << std::endl;
    }
    else {
      basic_offsets_and_sizes[2 * c_idx] = schema[counter_name].first;
      basic_offsets_and_sizes[2 * c_idx + 1] = schema[counter_name].second;
    }
    ++c_idx;
  }
  set_property_value<basic_offsets_and_sizes_t>(basic_offsets_and_sizes);
}

void make_lumi_summary::make_lumi_summary_t::operator()(
  const ArgumentReferences<Parameters>& arguments,
  const RuntimeOptions&,
  const Constants&,
  const Allen::Context& context) const
{
  Allen::memset_async<dev_lumi_summaries_t>(arguments, 0xffffffff, context);

  // info aggregating
  std::array<const Lumi::LumiInfo*, Lumi::Constants::n_sub_infos> lumiInfos = {data<dev_velo_info_t>(arguments),
                                                                               data<dev_pv_info_t>(arguments),
                                                                               data<dev_scifi_info_t>(arguments),
                                                                               data<dev_muon_info_t>(arguments),
                                                                               data<dev_calo_info_t>(arguments),
                                                                               data<dev_plume_info_t>(arguments)};
  // set the size to 0 for empty dummy input
  // otherwise set it to the numbers of lumi counters
  std::array<unsigned, Lumi::Constants::n_sub_infos> infoSize = {
    std::min(Lumi::Constants::n_velo_counters, static_cast<unsigned>(size<dev_velo_info_t>(arguments))),
    std::min(Lumi::Constants::n_pv_counters, static_cast<unsigned>(size<dev_pv_info_t>(arguments))),
    std::min(Lumi::Constants::n_scifi_counters, static_cast<unsigned>(size<dev_scifi_info_t>(arguments))),
    std::min(Lumi::Constants::n_muon_counters, static_cast<unsigned>(size<dev_muon_info_t>(arguments))),
    std::min(Lumi::Constants::n_calo_counters, static_cast<unsigned>(size<dev_calo_info_t>(arguments))),
    std::min(Lumi::Constants::n_plume_counters, static_cast<unsigned>(size<dev_plume_info_t>(arguments)))};
  unsigned size_of_aggregate = Lumi::Constants::n_sub_infos;
  for (unsigned i = 1u; i <= size_of_aggregate; ++i) {
    if (infoSize[i - 1] == 0u) {
      // move the items after the empty LumiInfo forward
      // to replace the empty object
      for (unsigned j = i; j < size_of_aggregate; ++j) {
        lumiInfos[j - 1] = lumiInfos[j];
        infoSize[j - 1] = infoSize[j];
      }
      i--;
      size_of_aggregate--;
    }
  }

  global_function(make_lumi_summary)(dim3(4u), property<block_dim_t>(), context)(
    arguments,
    first<host_number_of_events_t>(arguments),
    size<dev_event_list_t>(arguments),
    lumiInfos,
    infoSize,
    size_of_aggregate);

  Allen::copy_async<host_lumi_summaries_t, dev_lumi_summaries_t>(arguments, context);
  Allen::copy_async<host_lumi_summary_offsets_t, dev_lumi_summary_offsets_t>(arguments, context);
}

__device__ void make_lumi_summary::setField(unsigned offset, unsigned size, unsigned* target, unsigned value)
{
  // Check value fits within size bits
  if (size < (8 * sizeof(unsigned)) && value >= (1u << size)) {
    return;
  }

  // Separate offset into a word part and bit part
  unsigned word = offset / (8 * sizeof(unsigned));
  unsigned bitoffset = offset % (8 * sizeof(unsigned));

  // Check size and offset line up with word boundaries
  if (bitoffset + size > (8 * sizeof(unsigned))) {
    return;
  }

  // Apply the value to the matching bits
  unsigned mask = ((1l << size) - 1) << bitoffset;
  target[word] = (target[word] & ~mask) | ((value << bitoffset) & mask);
}

__global__ void make_lumi_summary::make_lumi_summary(
  make_lumi_summary::Parameters parameters,
  const unsigned number_of_events,
  const unsigned number_of_events_passed_gec,
  std::array<const Lumi::LumiInfo*, Lumi::Constants::n_sub_infos> lumiInfos,
  std::array<unsigned, Lumi::Constants::n_sub_infos> infoSize,
  const unsigned size_of_aggregate)
{
  for (unsigned event_number = blockIdx.x * blockDim.x + threadIdx.x; event_number < number_of_events;
       event_number += blockDim.x * gridDim.x) {
    unsigned offset = parameters.dev_lumi_summary_offsets[event_number];

    // skip non-lumi event
    if (offset == parameters.dev_lumi_summary_offsets[event_number + 1]) continue;

    auto* lumi_summary = parameters.dev_lumi_summaries + offset;
    lumi_summary[0] = parameters.key;

    /// ODIN information
    const LHCb::ODIN odin {parameters.dev_odin_data[event_number]};
    uint64_t new_bcid = static_cast<uint32_t>(odin.orbitNumber()) * 3564 + static_cast<uint16_t>(odin.bunchId());
    uint64_t t0 = static_cast<uint64_t>(odin.gpsTime()) - new_bcid * 1000 / 40078;
    // event time
    setField(
      parameters.basic_offsets_and_sizes.get()[0],
      parameters.basic_offsets_and_sizes.get()[1],
      lumi_summary,
      static_cast<unsigned>(t0 & 0xffffffff));
    setField(
      parameters.basic_offsets_and_sizes.get()[2],
      parameters.basic_offsets_and_sizes.get()[3],
      lumi_summary,
      static_cast<unsigned>(t0 >> 32));

    // gps time offset
    setField(
      parameters.basic_offsets_and_sizes.get()[4],
      parameters.basic_offsets_and_sizes.get()[5],
      lumi_summary,
      static_cast<unsigned>(new_bcid & 0xffffffff));
    setField(
      parameters.basic_offsets_and_sizes.get()[6],
      parameters.basic_offsets_and_sizes.get()[7],
      lumi_summary,
      static_cast<unsigned>(new_bcid >> 32));

    // bunch crossing type
    setField(
      parameters.basic_offsets_and_sizes.get()[8],
      parameters.basic_offsets_and_sizes.get()[9],
      lumi_summary,
      static_cast<unsigned>(odin.bunchCrossingType()));

    /// gec counter
    bool passedGEC = false;
    for (unsigned i = 0; i < number_of_events_passed_gec; ++i) {
      if (parameters.dev_event_list[i] == event_number) {
        passedGEC = true;
        break;
      }
    }
    setField(
      parameters.basic_offsets_and_sizes.get()[10],
      parameters.basic_offsets_and_sizes.get()[11],
      lumi_summary,
      passedGEC);

    /// write lumi infos to the summary
    for (unsigned i = 0; i < size_of_aggregate; ++i) {
      if (infoSize[i] == 0 || lumiInfos[i] == nullptr) continue;
      unsigned spanOffset = offset / parameters.lumi_sum_length * infoSize[i];
      for (unsigned j = spanOffset;
           j < parameters.dev_lumi_summary_offsets[event_number + 1] / parameters.lumi_sum_length * infoSize[i];
           ++j) {
        setField(lumiInfos[i][j].offset, lumiInfos[i][j].size, lumi_summary, lumiInfos[i][j].value);
      }
    }
  }
}
