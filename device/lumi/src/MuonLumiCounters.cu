#include "hip/hip_runtime.h"
/*****************************************************************************\
* (c) Copyright 2022 CERN for the benefit of the LHCb Collaboration           *
*                                                                             *
* This software is distributed under the terms of the Apache License          *
* version 2 (Apache-2.0), copied verbatim in the file "COPYING".              *
*                                                                             *
* In applying this licence, CERN does not waive the privileges and immunities *
* granted to it by virtue of its status as an Intergovernmental Organization  *
* or submit itself to any jurisdiction.                                       *
\*****************************************************************************/
#include "MuonLumiCounters.cuh"
#include "LumiCommon.cuh"

INSTANTIATE_ALGORITHM(muon_lumi_counters::muon_lumi_counters_t)

void muon_lumi_counters::muon_lumi_counters_t::set_arguments_size(
  ArgumentReferences<Parameters> arguments,
  const RuntimeOptions&,
  const Constants&) const
{
  // the total size of output info is proportional to the lumi summaries
  set_size<dev_lumi_infos_t>(
    arguments,
    Lumi::Constants::n_muon_counters * first<host_lumi_summaries_size_t>(arguments) / property<lumi_sum_length_t>());
}

void muon_lumi_counters::muon_lumi_counters_t::init()
{
  std::map<std::string, std::pair<unsigned, unsigned>> schema = property<lumi_counter_schema_t>();
  std::array<unsigned, 2 * Lumi::Constants::n_muon_counters> muon_offsets_and_sizes =
    property<muon_offsets_and_sizes_t>();

  unsigned c_idx(0u);
  for (auto counter_name : Lumi::Constants::muon_counter_names) {
    if (schema.find(counter_name) == schema.end()) {
      std::cout << "LumiSummary schema does not use " << counter_name << std::endl;
    }
    else {
      muon_offsets_and_sizes[2 * c_idx] = schema[counter_name].first;
      muon_offsets_and_sizes[2 * c_idx + 1] = schema[counter_name].second;
    }
    ++c_idx;
  }
  set_property_value<muon_offsets_and_sizes_t>(muon_offsets_and_sizes);
}

void muon_lumi_counters::muon_lumi_counters_t::operator()(
  const ArgumentReferences<Parameters>& arguments,
  const RuntimeOptions&,
  const Constants&,
  const Allen::Context& context) const
{
  // do nothing if no lumi event
  if (first<host_lumi_summaries_size_t>(arguments) == 0) return;

  global_function(muon_lumi_counters)(dim3(4u), property<block_dim_t>(), context)(
    arguments, first<host_number_of_events_t>(arguments));
}

__global__ void muon_lumi_counters::muon_lumi_counters(
  muon_lumi_counters::Parameters parameters,
  const unsigned number_of_events)
{
  for (unsigned event_number = blockIdx.x * blockDim.x + threadIdx.x; event_number < number_of_events;
       event_number += blockDim.x * gridDim.x) {
    unsigned lumi_sum_offset = parameters.dev_lumi_summary_offsets[event_number];

    // skip non-lumi event
    if (lumi_sum_offset == parameters.dev_lumi_summary_offsets[event_number + 1]) continue;

    const auto muon_hits_offsets =
      parameters.dev_storage_station_region_quarter_offsets + event_number * Lumi::Constants::MuonBankSize;

    unsigned info_offset = Lumi::Constants::n_muon_counters * lumi_sum_offset / parameters.lumi_sum_length;

    std::array<unsigned, Lumi::Constants::n_muon_station_regions + 1> muon_offsets = {Lumi::Constants::M2R1,
                                                                                      Lumi::Constants::M2R2,
                                                                                      Lumi::Constants::M2R3,
                                                                                      Lumi::Constants::M2R4,
                                                                                      Lumi::Constants::M3R1,
                                                                                      Lumi::Constants::M3R2,
                                                                                      Lumi::Constants::M3R3,
                                                                                      Lumi::Constants::M3R4,
                                                                                      Lumi::Constants::M4R1,
                                                                                      Lumi::Constants::M4R2,
                                                                                      Lumi::Constants::M4R3,
                                                                                      Lumi::Constants::M4R4,
                                                                                      Lumi::Constants::M5R1};

    for (unsigned i = 0; i < Lumi::Constants::n_muon_station_regions; ++i) {
      fillLumiInfo(
        parameters.dev_lumi_infos[info_offset + i],
        parameters.muon_offsets_and_sizes.get()[2 * i],
        parameters.muon_offsets_and_sizes.get()[2 * i + 1],
        muon_hits_offsets[muon_offsets[i + 1]] - muon_hits_offsets[muon_offsets[i]]);
    }

    fillLumiInfo(
      parameters.dev_lumi_infos[info_offset + Lumi::Constants::n_muon_station_regions],
      parameters.muon_offsets_and_sizes.get()[2 * Lumi::Constants::n_muon_station_regions],
      parameters.muon_offsets_and_sizes.get()[2 * Lumi::Constants::n_muon_station_regions + 1],
      parameters.dev_muon_number_of_tracks[event_number]);
  }
}
