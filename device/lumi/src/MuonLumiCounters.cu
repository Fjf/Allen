#include "hip/hip_runtime.h"
/*****************************************************************************\
* (c) Copyright 2022 CERN for the benefit of the LHCb Collaboration           *
*                                                                             *
* This software is distributed under the terms of the Apache License          *
* version 2 (Apache-2.0), copied verbatim in the file "COPYING".              *
*                                                                             *
* In applying this licence, CERN does not waive the privileges and immunities *
* granted to it by virtue of its status as an Intergovernmental Organization  *
* or submit itself to any jurisdiction.                                       *
\*****************************************************************************/
#include "MuonLumiCounters.cuh"
#include "LumiCommon.cuh"

INSTANTIATE_ALGORITHM(muon_lumi_counters::muon_lumi_counters_t)

void muon_lumi_counters::muon_lumi_counters_t::set_arguments_size(
  ArgumentReferences<Parameters> arguments,
  const RuntimeOptions&,
  const Constants&) const
{
  // the total size of output info is proportional to the lumi summaries
  set_size<dev_lumi_infos_t>(
    arguments, Lumi::Constants::n_muon_counters * first<host_lumi_summaries_count_t>(arguments));
}

void muon_lumi_counters::muon_lumi_counters_t::init()
{
  std::map<std::string, std::pair<unsigned, unsigned>> schema = property<lumi_counter_schema_t>();
  std::map<std::string, std::pair<float, float>> shifts_and_scales = property<lumi_counter_shifts_and_scales_t>();
  std::array<unsigned, 2 * Lumi::Constants::n_muon_counters> muon_offsets_and_sizes =
    property<muon_offsets_and_sizes_t>();
  std::array<float, 2 * Lumi::Constants::n_muon_counters> muon_shifts_and_scales = property<muon_shifts_and_scales_t>();

  unsigned c_idx(0u);
  for (auto counter_name : Lumi::Constants::muon_counter_names) {
    if (schema.find(counter_name) == schema.end()) {
      std::cout << "LumiSummary schema does not use " << counter_name << std::endl;
    }
    else {
      muon_offsets_and_sizes[2 * c_idx] = schema[counter_name].first;
      muon_offsets_and_sizes[2 * c_idx + 1] = schema[counter_name].second;
    }
    if (shifts_and_scales.find(counter_name) == shifts_and_scales.end()) {
      muon_shifts_and_scales[2 * c_idx] = 0.f;
      muon_shifts_and_scales[2 * c_idx + 1] = 1.f;
    }
    else {
      muon_shifts_and_scales[2 * c_idx] = shifts_and_scales[counter_name].first;
      muon_shifts_and_scales[2 * c_idx + 1] = shifts_and_scales[counter_name].second;
    }
    ++c_idx;
  }
  set_property_value<muon_offsets_and_sizes_t>(muon_offsets_and_sizes);
  set_property_value<muon_shifts_and_scales_t>(muon_shifts_and_scales);
}

void muon_lumi_counters::muon_lumi_counters_t::operator()(
  const ArgumentReferences<Parameters>& arguments,
  const RuntimeOptions&,
  const Constants&,
  const Allen::Context& context) const
{
  // do nothing if no lumi event
  if (first<host_lumi_summaries_count_t>(arguments) == 0) return;

  global_function(muon_lumi_counters)(dim3(4u), property<block_dim_t>(), context)(
    arguments, first<host_number_of_events_t>(arguments));
}

__global__ void muon_lumi_counters::muon_lumi_counters(
  muon_lumi_counters::Parameters parameters,
  const unsigned number_of_events)
{
  for (unsigned event_number = blockIdx.x * blockDim.x + threadIdx.x; event_number < number_of_events;
       event_number += blockDim.x * gridDim.x) {
    unsigned lumi_evt_index = parameters.dev_lumi_event_indices[event_number];

    // skip non-lumi event
    if (lumi_evt_index == parameters.dev_lumi_event_indices[event_number + 1]) continue;

    const auto muon_hits_offsets =
      parameters.dev_storage_station_region_quarter_offsets + event_number * Lumi::Constants::MuonBankSize;

    unsigned info_offset = Lumi::Constants::n_muon_counters * lumi_evt_index;

    std::array<unsigned, Lumi::Constants::n_muon_station_regions + 1> muon_offsets = {Lumi::Constants::M2R1,
                                                                                      Lumi::Constants::M2R2,
                                                                                      Lumi::Constants::M2R3,
                                                                                      Lumi::Constants::M2R4,
                                                                                      Lumi::Constants::M3R1,
                                                                                      Lumi::Constants::M3R2,
                                                                                      Lumi::Constants::M3R3,
                                                                                      Lumi::Constants::M3R4,
                                                                                      Lumi::Constants::M4R1,
                                                                                      Lumi::Constants::M4R2,
                                                                                      Lumi::Constants::M4R3,
                                                                                      Lumi::Constants::M4R4,
                                                                                      Lumi::Constants::M5R1};

    for (unsigned i = 0; i < Lumi::Constants::n_muon_station_regions; ++i) {
      fillLumiInfo(
        parameters.dev_lumi_infos[info_offset + i],
        parameters.muon_offsets_and_sizes.get()[2 * i],
        parameters.muon_offsets_and_sizes.get()[2 * i + 1],
        muon_hits_offsets[muon_offsets[i + 1]] - muon_hits_offsets[muon_offsets[i]],
        parameters.muon_shifts_and_scales.get()[2 * i],
        parameters.muon_shifts_and_scales.get()[2 * i + 1]);
    }

    fillLumiInfo(
      parameters.dev_lumi_infos[info_offset + Lumi::Constants::n_muon_station_regions],
      parameters.muon_offsets_and_sizes.get()[2 * Lumi::Constants::n_muon_station_regions],
      parameters.muon_offsets_and_sizes.get()[2 * Lumi::Constants::n_muon_station_regions + 1],
      parameters.dev_muon_number_of_tracks[event_number],
      parameters.muon_shifts_and_scales.get()[2 * Lumi::Constants::n_muon_station_regions],
      parameters.muon_shifts_and_scales.get()[2 * Lumi::Constants::n_muon_station_regions + 1]);
  }
}
