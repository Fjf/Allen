#include "hip/hip_runtime.h"
/*****************************************************************************\
* (c) Copyright 2022 CERN for the benefit of the LHCb Collaboration           *
*                                                                             *
* This software is distributed under the terms of the Apache License          *
* version 2 (Apache-2.0), copied verbatim in the file "COPYING".              *
*                                                                             *
* In applying this licence, CERN does not waive the privileges and immunities *
* granted to it by virtue of its status as an Intergovernmental Organization  *
* or submit itself to any jurisdiction.                                       *
\*****************************************************************************/
#include "PlumeLumiCounters.cuh"

INSTANTIATE_ALGORITHM(plume_lumi_counters::plume_lumi_counters_t)

void plume_lumi_counters::plume_lumi_counters_t::set_arguments_size(
  ArgumentReferences<Parameters> arguments,
  const RuntimeOptions&,
  const Constants&) const
{
  // the total size of output info is proportional to the lumi summaries
  set_size<dev_lumi_infos_t>(
    arguments,
    Lumi::Constants::n_plume_counters * first<host_lumi_summaries_size_t>(arguments) / Lumi::Constants::lumi_length);
}

void plume_lumi_counters::plume_lumi_counters_t::operator()(
  const ArgumentReferences<Parameters>& arguments,
  const RuntimeOptions&,
  const Constants&,
  const Allen::Context& context) const
{
  // do nothing if no lumi event
  if (first<host_lumi_summaries_size_t>(arguments) == 0) return;

  global_function(plume_lumi_counters)(dim3(4u), property<block_dim_t>(), context)(
    arguments, first<host_number_of_events_t>(arguments));
}

__global__ void plume_lumi_counters::plume_lumi_counters(
  plume_lumi_counters::Parameters parameters,
  const unsigned number_of_events)
{
  for (unsigned event_number = blockIdx.x * blockDim.x + threadIdx.x; event_number < number_of_events;
       event_number += blockDim.x * gridDim.x) {
    unsigned lumi_sum_offset = parameters.dev_lumi_summary_offsets[event_number];

    // skip non-lumi event
    if (lumi_sum_offset == parameters.dev_lumi_summary_offsets[event_number + 1]) continue;

    // loop over lumi channels
    const Plume_* pl = parameters.dev_plume + event_number;
    std::array<unsigned, 3> plume_counters = {0u, 0u, 0u};
    for (unsigned feb = 0; feb < 2; feb++) {
      unsigned channel_offset = feb * Lumi::Constants::n_plume_channels;
      for (unsigned channel = 0; channel < Lumi::Constants::n_plume_lumi_channels; ++channel) {
        plume_counters[0] += static_cast<unsigned>(pl->ADC_counts[channel_offset + channel].x & 0xffffffff);
        // get the corresonding overthreshold bit
        plume_counters[1 + feb] |= ((pl->ovr_th[feb]) & (1u << (31 - channel)));
      }
    }
    // get average
    plume_counters[0] = plume_counters[0] / 2u / Lumi::Constants::n_plume_lumi_channels;

    std::array<LHCb::LumiSummaryOffsets::V2::counterOffsets, Lumi::Constants::n_plume_counters> counter_offsets = {
      LHCb::LumiSummaryOffsets::V2::PlumeAvgLumiADCOffset,
      LHCb::LumiSummaryOffsets::V2::PlumeLumiOverthrLowOffset,
      LHCb::LumiSummaryOffsets::V2::PlumeLumiOverthrHighOffset};
    std::array<LHCb::LumiSummaryOffsets::V2::counterOffsets, Lumi::Constants::n_plume_counters> counter_sizes = {
      LHCb::LumiSummaryOffsets::V2::PlumeAvgLumiADCSize,
      LHCb::LumiSummaryOffsets::V2::PlumeLumiOverthrLowSize,
      LHCb::LumiSummaryOffsets::V2::PlumeLumiOverthrHighSize};
    auto* lumi_info =
      parameters.dev_lumi_infos + Lumi::Constants::n_plume_counters * lumi_sum_offset / Lumi::Constants::lumi_length;
    for (unsigned info_index = 0u; info_index < Lumi::Constants::n_plume_counters; ++info_index) {
      lumi_info[info_index].offset = counter_offsets[info_index];
      lumi_info[info_index].size = counter_sizes[info_index];
      lumi_info[info_index].value = plume_counters[info_index];
    }
  }
}
