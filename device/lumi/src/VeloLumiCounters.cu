#include "hip/hip_runtime.h"
/*****************************************************************************\
* (c) Copyright 2022 CERN for the benefit of the LHCb Collaboration           *
*                                                                             *
* This software is distributed under the terms of the Apache License          *
* version 2 (Apache-2.0), copied verbatim in the file "COPYING".              *
*                                                                             *
* In applying this licence, CERN does not waive the privileges and immunities *
* granted to it by virtue of its status as an Intergovernmental Organization  *
* or submit itself to any jurisdiction.                                       *
\*****************************************************************************/
#include "VeloLumiCounters.cuh"
#include "LumiCommon.cuh"

INSTANTIATE_ALGORITHM(velo_lumi_counters::velo_lumi_counters_t)

void velo_lumi_counters::velo_lumi_counters_t::set_arguments_size(
  ArgumentReferences<Parameters> arguments,
  const RuntimeOptions&,
  const Constants&) const
{
  // the total size of output info is proportional to the lumi summaries
  set_size<dev_lumi_infos_t>(
    arguments, Lumi::Constants::n_velo_counters * first<host_lumi_summaries_count_t>(arguments));
}

void velo_lumi_counters::velo_lumi_counters_t::init()
{
  std::map<std::string, std::pair<unsigned, unsigned>> schema = property<lumi_counter_schema_t>();
  std::map<std::string, std::pair<float, float>> shifts_and_scales = property<lumi_counter_shifts_and_scales_t>();

  unsigned c_idx(0u);
  for (auto counter_name : Lumi::Constants::velo_counter_names) {
    if (schema.find(counter_name) == schema.end()) {
      std::cout << "LumiSummary schema does not use " << counter_name << std::endl;
    }
    else {
      m_offsets_and_sizes[2 * c_idx] = schema[counter_name].first;
      m_offsets_and_sizes[2 * c_idx + 1] = schema[counter_name].second;
    }
    if (shifts_and_scales.find(counter_name) == shifts_and_scales.end()) {
      m_shifts_and_scales[2 * c_idx] = 0.f;
      m_shifts_and_scales[2 * c_idx + 1] = 1.f;
    }
    else {
      m_shifts_and_scales[2 * c_idx] = shifts_and_scales[counter_name].first;
      m_shifts_and_scales[2 * c_idx + 1] = shifts_and_scales[counter_name].second;
    }
    ++c_idx;
  }
}

void velo_lumi_counters::velo_lumi_counters_t::operator()(
  const ArgumentReferences<Parameters>& arguments,
  const RuntimeOptions&,
  const Constants&,
  const Allen::Context& context) const
{
  // do nothing if no lumi event
  if (first<host_lumi_summaries_count_t>(arguments) == 0) return;

  global_function(velo_lumi_counters)(dim3(4u), property<block_dim_t>(), context)(
    arguments, first<host_number_of_events_t>(arguments), m_offsets_and_sizes, m_shifts_and_scales);
}

__global__ void velo_lumi_counters::velo_lumi_counters(
  velo_lumi_counters::Parameters parameters,
  const unsigned number_of_events,
  const offsets_and_sizes_t offsets_and_sizes,
  const shifts_and_scales_t shifts_and_scales)
{
  for (unsigned event_number = blockIdx.x * blockDim.x + threadIdx.x; event_number < number_of_events;
       event_number += blockDim.x * gridDim.x) {
    unsigned lumi_evt_index = parameters.dev_lumi_event_indices[event_number];

    // skip non-lumi event
    if (lumi_evt_index == parameters.dev_lumi_event_indices[event_number + 1]) continue;

    std::array<unsigned, 10> velo_counters = {0u, 0u, 0u, 0u, 0u, 0u, 0u, 0u, 0u, 0u};

    const auto velo_states = parameters.dev_velo_states_view[event_number];
    // first counter is the total velo tracks
    const unsigned track_offset = parameters.dev_offsets_all_velo_tracks[event_number];
    velo_counters[0] = parameters.dev_offsets_all_velo_tracks[event_number + 1] - track_offset;

    for (unsigned track_index = 0u; track_index < velo_counters[0]; ++track_index) {
      const auto velo_state = velo_states.state(track_index);

      // fiducial cut: doca<3 mm && |poca|<300 mm
      if (velo_state.z() > -300.f && velo_state.z() < 300.f) {
        if (velo_DOCAz(velo_state) < 3.f * Gaudi::Units::mm) {
          ++velo_counters[1];
        }
      }

      // fill eta bins
      float eta = velo_eta(velo_state, parameters.dev_is_backward[track_offset + track_index]);
      if (eta > parameters.tracks_eta_bins.get()[Lumi::Constants::n_velo_eta_bin_edges - 1u] * Gaudi::Units::mm) {
        ++velo_counters[9];
        continue;
      }
      for (unsigned eta_bin = 0; eta_bin < Lumi::Constants::n_velo_eta_bin_edges; ++eta_bin) {
        if (eta < parameters.tracks_eta_bins.get()[eta_bin] * Gaudi::Units::mm) {
          ++velo_counters[2u + eta_bin];
          break;
        }
      }
    }

    unsigned info_offset = Lumi::Constants::n_velo_counters * lumi_evt_index;

    for (unsigned info_index = 0u; info_index < Lumi::Constants::n_velo_counters; ++info_index) {
      fillLumiInfo(
        parameters.dev_lumi_infos[info_offset + info_index],
        offsets_and_sizes[info_index * 2],
        offsets_and_sizes[info_index * 2 + 1],
        velo_counters[info_index],
        shifts_and_scales[2 * info_index],
        shifts_and_scales[2 * info_index + 1]);
    }
  }
}
