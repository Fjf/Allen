#include "../include/Stream.cuh"

hipError_t Stream::operator()(
  const char* host_events_pinned,
  const unsigned int* host_event_offsets_pinned,
  const unsigned int* host_hit_offsets_pinned,
  size_t host_events_pinned_size,
  size_t host_event_offsets_pinned_size,
  size_t host_hit_offsets_pinned_size,
  unsigned int start_event,
  unsigned int number_of_events,
  unsigned int number_of_repetitions
) {
  for (unsigned int repetitions=0; repetitions<number_of_repetitions; ++repetitions) {
    /////////////////////////
    // CalculatePhiAndSort //
    /////////////////////////

    // Optional transmission host to device
    if (transmit_host_to_device) {
      cudaCheck(hipMemcpyAsync(dev_events, host_events_pinned, host_events_pinned_size, hipMemcpyHostToDevice, stream));
      cudaCheck(hipMemcpyAsync(dev_event_offsets, host_event_offsets_pinned, host_event_offsets_pinned_size * sizeof(unsigned int), hipMemcpyHostToDevice, stream));
      cudaCheck(hipMemcpyAsync(dev_hit_offsets, host_hit_offsets_pinned, host_hit_offsets_pinned_size * sizeof(unsigned int), hipMemcpyHostToDevice, stream));
    }

    // Invoke kernel
    calculatePhiAndSort();

    /////////////////////
    // SearchByTriplet //
    /////////////////////

    searchByTriplet();

    ////////////////////////
    // Consolidate tracks //
    ////////////////////////
    
    consolidateTracks();

    // Optional transmission device to host
    if (transmit_device_to_host) {
      cudaCheck(hipMemcpyAsync(host_number_of_tracks_pinned, dev_atomics_storage, number_of_events * sizeof(int), hipMemcpyDeviceToHost, stream));
      hipEventRecord(cuda_generic_event, stream);
      hipEventSynchronize(cuda_generic_event);
      
      int total_number_of_tracks = 0;
      for (int i=0; i<number_of_events; ++i) {
        total_number_of_tracks += host_number_of_tracks_pinned[i];
      }

      cudaCheck(hipMemcpyAsync(host_tracks_pinned, dev_tracklets, total_number_of_tracks * sizeof(Track), hipMemcpyDeviceToHost, stream));
    }
  }

  return hipSuccess;
}

void Stream::print_timing(
  const unsigned int number_of_events,
  const std::vector<std::pair<std::string, float>>& times
) {
  const auto total_time = times[times.size() - 1];
  std::string partial_times = "{\n";
  for (size_t i=0; i<times.size(); ++i) {
    if (i != times.size()-1) {
      partial_times += " " + times[i].first + "\t" + std::to_string(times[i].second) + "\t("
        + std::to_string(100 * (times[i].second / total_time.second)) + " %)\n";
    } else {
      partial_times += " " + times[i].first + "\t" + std::to_string(times[i].second) + "\t("
        + std::to_string(100 * (times[i].second / total_time.second)) + " %)\n}";
    }
  }

  DEBUG << "stream #" << stream_number << ": "
    << number_of_events / total_time.second << " events/s"
    << ", partial timers (s): " << partial_times
    << std::endl;
}
