#include "../include/Stream.cuh"

hipError_t Stream::operator()(
  const char* host_events_pinned,
  const uint* host_event_offsets_pinned,
  const uint* host_hit_offsets_pinned,
  size_t host_events_pinned_size,
  size_t host_event_offsets_pinned_size,
  size_t host_hit_offsets_pinned_size,
  uint start_event,
  uint number_of_events,
  uint number_of_repetitions
) {
  for (uint repetitions=0; repetitions<number_of_repetitions; ++repetitions) {
    ////////////////
    // Clustering //
    ////////////////

    if (transmit_host_to_device) {
      cudaCheck(hipMemcpyAsync(dev_raw_input, host_events_pinned, host_events_pinned_size, hipMemcpyHostToDevice, stream));
      cudaCheck(hipMemcpyAsync(dev_raw_input_offsets, host_event_offsets_pinned, host_event_offsets_pinned_size * sizeof(uint), hipMemcpyHostToDevice, stream));
    }

    // Estimate the input size of each module
    estimateInputSize();

    // Convert the estimated sizes to module hit start format (offsets)
    prefixSum();

    // // Fetch the number of hits we require
    // uint number_of_hits;
    // cudaCheck(hipMemcpyAsync(&number_of_hits, dev_estimated_input_size + number_of_events * 52, sizeof(uint), hipMemcpyDeviceToHost, stream));

    // if (number_of_hits * 6 * sizeof(uint32_t) > velo_cluster_container_size) {
    //   WARNING << "Number of hits: " << number_of_hits << std::endl
    //     << "Size of velo cluster container is larger than previously accomodated." << std::endl
    //     << "Resizing from " << velo_cluster_container_size << " to " << number_of_hits * 6 * sizeof(uint) << " B" << std::endl;

    //   cudaCheck(hipFree(dev_velo_cluster_container));
    //   velo_cluster_container_size = number_of_hits * 6 * sizeof(uint32_t);
    //   cudaCheck(hipMalloc((void**)&dev_velo_cluster_container, velo_cluster_container_size));
    // }

    // Invoke clustering
    maskedVeloClustering();

    /////////////////////////
    // CalculatePhiAndSort //
    /////////////////////////

    // Invoke sorting
    calculatePhiAndSort();

    hipEventRecord(cuda_generic_event, stream);
    hipEventSynchronize(cuda_generic_event);

    // /////////////////////
    // // SearchByTriplet //
    // /////////////////////

    // searchByTriplet();

    // ////////////////////////
    // // Consolidate tracks //
    // ////////////////////////
    
    // consolidateTracks();

    // // Optional transmission device to host
    // if (transmit_device_to_host) {
    //   cudaCheck(hipMemcpyAsync(host_number_of_tracks_pinned, dev_atomics_storage, number_of_events * sizeof(int), hipMemcpyDeviceToHost, stream));
    //   hipEventRecord(cuda_generic_event, stream);
    //   hipEventSynchronize(cuda_generic_event);
      
    //   int total_number_of_tracks = 0;
    //   for (int i=0; i<number_of_events; ++i) {
    //     total_number_of_tracks += host_number_of_tracks_pinned[i];
    //   }

    //   cudaCheck(hipMemcpyAsync(host_tracks_pinned, dev_tracklets, total_number_of_tracks * sizeof(Track), hipMemcpyDeviceToHost, stream));
    // }
  }

  return hipSuccess;
}
