#include "../include/Stream.cuh"

hipError_t Stream::operator()(
  const std::vector<char>& events,
  const std::vector<unsigned int>& event_offsets,
  const std::vector<unsigned int>& hit_offsets,
  unsigned int start_event,
  unsigned int number_of_events,
  unsigned int number_of_repetitions
) {
  cudaCheck(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
  
  for (unsigned int repetitions=0; repetitions<number_of_repetitions; ++repetitions) {
    // Timers
    std::vector<std::pair<std::string, float>> times;
    Timer t_total;

    // Total number of hits
    const auto total_number_of_hits = hit_offsets[hit_offsets.size() - 1];

    /////////////////////////
    // CalculatePhiAndSort //
    /////////////////////////

    Timer t;

    if (dev_events_size < events.size()) {
      // malloc just this datatype
      cudaCheck(hipFree(dev_events));
      dev_events_size = events.size();
      cudaCheck(hipMalloc((void**)&dev_events, dev_events_size));
    }

    if ((total_number_of_hits / number_of_events) > maximum_average_number_of_hits_per_event) {
      std::cerr << "total average number of hits exceeds maximum ("
        << (total_number_of_hits / number_of_events) << " > " << maximum_average_number_of_hits_per_event
        << ")" << std::endl;
    }

    t.stop();
    times.emplace_back("allocate phi buffers", t.get());

    if (transmit_host_to_device) {
      t.restart();
      // Copy required data
      cudaCheck(hipMemcpyAsync(dev_events, events.data(), events.size(), hipMemcpyHostToDevice, stream));
      t.stop();
      times.emplace_back("copy events", t.get());
      
      t.restart();
      cudaCheck(hipMemcpyAsync(dev_event_offsets, event_offsets.data(), event_offsets.size() * sizeof(unsigned int), hipMemcpyHostToDevice, stream));
      cudaCheck(hipMemcpyAsync(dev_hit_offsets, hit_offsets.data(), hit_offsets.size() * sizeof(unsigned int), hipMemcpyHostToDevice, stream));
      t.stop();
      times.emplace_back("copy offsets", t.get());
    }

    // Invoke kernel
    times.emplace_back(
      "calculatePhiAndSort",
      0.001 * Helper::invoke(calculatePhiAndSort)
    );

    cudaCheck(hipPeekAtLastError());

    /////////////////////
    // SearchByTriplet //
    /////////////////////
    
    t.restart();

    // Initialize data
    cudaCheck(hipMemsetAsync(dev_hit_used, false, total_number_of_hits * sizeof(bool), stream));
    cudaCheck(hipMemsetAsync(dev_atomics_storage, 0, number_of_events * atomic_space * sizeof(int), stream));

    t.stop();
    times.emplace_back("initialize sbt data", t.get());

    // Invoke kernel

    times.emplace_back(
      "Search by triplets",
      0.001 * Helper::invoke(searchByTriplet)
    );

    cudaCheck(hipPeekAtLastError());


    ///////////////////////////
    // Calculate VELO states //
    ///////////////////////////

    // Invoke kernel
    times.emplace_back(
      "calculateVeloStates",
      0.001 * Helper::invoke(calculateVeloStates)
    );

    cudaCheck(hipPeekAtLastError());

    // The chain can follow from here on.
    // If the chain follows, we may not need to retrieve the data
    // in the state it is currently, but in a posterior state.
    // In principle, here we need to get back:
    // - dev_hit_permutation: Permutation of hits (reorder)
    // - dev_atomics_storage: Number of tracks
    // - dev_tracks: Tracks
    // - dev_velo_states: VELO filtered states for each track
    
    // Therefore, this is just temporal
    // Fetch required data
    if (transmit_device_to_host) {
      std::vector<int> number_of_tracks (number_of_events);
      std::vector<unsigned short> hit_permutations (total_number_of_hits);
      std::vector<Track> tracks (number_of_events * max_tracks_in_event);
      std::vector<VeloState> velo_states (number_of_events * max_tracks_in_event * STATES_PER_TRACK);

      cudaCheck(hipMemcpyAsync(number_of_tracks.data(), dev_atomics_storage, number_of_events * sizeof(int), hipMemcpyDeviceToHost, stream));
      cudaCheck(hipMemcpyAsync(hit_permutations.data(), dev_hit_permutation, total_number_of_hits * sizeof(unsigned short), hipMemcpyDeviceToHost, stream));
      cudaCheck(hipMemcpyAsync(tracks.data(), dev_tracks, number_of_events * max_tracks_in_event * sizeof(Track), hipMemcpyDeviceToHost, stream));
      cudaCheck(hipMemcpyAsync(velo_states.data(), dev_velo_states, number_of_events * max_tracks_in_event * STATES_PER_TRACK * sizeof(VeloState), hipMemcpyDeviceToHost, stream));
    }


    t_total.stop();
    times.emplace_back("total", t_total.get());

    if (do_print_timing) {
      print_timing(number_of_events, times);
    }
  }

  return hipSuccess;
}

void Stream::print_timing(
  const unsigned int number_of_events,
  const std::vector<std::pair<std::string, float>>& times
) {
  const auto total_time = times[times.size() - 1];
  std::string partial_times = "{\n";
  for (size_t i=0; i<times.size(); ++i) {
    if (i != times.size()-1) {
      partial_times += " " + times[i].first + "\t" + std::to_string(times[i].second) + "\t("
        + std::to_string(100 * (times[i].second / total_time.second)) + " %)\n";
    } else {
      partial_times += " " + times[i].first + "\t" + std::to_string(times[i].second) + "\t("
        + std::to_string(100 * (times[i].second / total_time.second)) + " %)\n}";
    }
  }

  DEBUG << "stream #" << stream_number << ": "
    << number_of_events / total_time.second << " events/s"
    << ", partial timers (s): " << partial_times
    << std::endl;
}
