#include "../include/Stream.cuh"

hipError_t Stream::initialize(
  const std::vector<char>& raw_events,
  const std::vector<uint>& event_offsets,
  const std::vector<char>& geometry,
  const uint number_of_events,
  const size_t param_starting_events_size,
  const bool param_transmit_host_to_device,
  const bool param_transmit_device_to_host,
  const bool param_do_consolidate,
  const uint param_stream_number
) {
  cudaCheck(hipStreamCreate(&stream));
  cudaCheck(hipEventCreate(&cuda_generic_event));
  stream_number = param_stream_number;
  transmit_host_to_device = param_transmit_host_to_device;
  transmit_device_to_host = param_transmit_device_to_host;
  do_consolidate = param_do_consolidate;

  // Blocks and threads for each algorithm
  num_blocks = dim3(number_of_events);
  estimate_input_size_blocks = dim3(number_of_events);
  prefix_sum_blocks = dim3(1);
  masked_velo_clustering_blocks = dim3(number_of_events);
  consolidate_blocks = dim3(number_of_events);

  estimate_input_size_threads = dim3(4, 208);
  prefix_sum_threads = dim3(1024);
  masked_velo_clustering_threads = dim3(256);
  sort_num_threads = dim3(64);
  sbt_num_threads = dim3(NUMTHREADS_X);
  consolidate_num_threads = dim3(32);

  // velo cluster container contains:
  // - cluster_xs
  // - cluster_ys
  // - cluster_zs
  // - cluster_ids
  // - cluster_phis
  // - temporary
  // 
  // The temporary is required to do the sorting in an efficient manner
  velo_cluster_container_size = number_of_events * 2000 * 6;

  // Data preparation
  // Velo module constants
  const std::array<float, 52> velo_module_zs = {-287.5, -275, -262.5, -250, -237.5, -225, -212.5, \
    -200, -137.5, -125, -62.5, -50, -37.5, -25, -12.5, 0, 12.5, 25, 37.5, 50, 62.5, 75, 87.5, 100, \
    112.5, 125, 137.5, 150, 162.5, 175, 187.5, 200, 212.5, 225, 237.5, 250, 262.5, 275, 312.5, 325, \
    387.5, 400, 487.5, 500, 587.5, 600, 637.5, 650, 687.5, 700, 737.5, 750};
  cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(VeloTracking::velo_module_zs), velo_module_zs.data(), velo_module_zs.size() * sizeof(float)));

  // Clustering patterns
  // Fetch patterns and populate in GPU
  cudaCheck(hipMalloc((void**)&dev_sp_patterns, 256));
  cudaCheck(hipMalloc((void**)&dev_sp_sizes, 256));
  cudaCheck(hipMalloc((void**)&dev_sp_fx, 512 * sizeof(float)));
  cudaCheck(hipMalloc((void**)&dev_sp_fy, 512 * sizeof(float)));
  std::vector<unsigned char> sp_patterns (256, 0);
  std::vector<unsigned char> sp_sizes (256, 0);
  std::vector<float> sp_fx (512, 0);
  std::vector<float> sp_fy (512, 0);
  cache_sp_patterns(sp_patterns, sp_sizes, sp_fx, sp_fy);
  cudaCheck(hipMemcpyAsync(dev_sp_patterns, sp_patterns.data(), sp_patterns.size(), hipMemcpyHostToDevice, stream));
  cudaCheck(hipMemcpyAsync(dev_sp_sizes, sp_sizes.data(), sp_sizes.size(), hipMemcpyHostToDevice, stream));
  cudaCheck(hipMemcpyAsync(dev_sp_fx, sp_fx.data(), sp_fx.size() * sizeof(float), hipMemcpyHostToDevice, stream));
  cudaCheck(hipMemcpyAsync(dev_sp_fy, sp_fy.data(), sp_fy.size() * sizeof(float), hipMemcpyHostToDevice, stream));

  // Populate velo geometry
  cudaCheck(hipMalloc((void**)&dev_velo_geometry, geometry.size()));
  cudaCheck(hipMemcpyAsync(dev_velo_geometry, geometry.data(), geometry.size(), hipMemcpyHostToDevice, stream));
  
  // Allocate buffers for algorithms
  // Clustering
  cudaCheck(hipMalloc((void**)&dev_raw_input, param_starting_events_size));
  cudaCheck(hipMalloc((void**)&dev_raw_input_offsets, event_offsets.size() * sizeof(uint)));
  cudaCheck(hipMalloc((void**)&dev_estimated_input_size, (number_of_events * 52 + 2) * sizeof(uint)));
  cudaCheck(hipMalloc((void**)&dev_module_cluster_num, number_of_events * 52 * sizeof(uint)));
  cudaCheck(hipMalloc((void**)&dev_module_candidate_num, number_of_events * sizeof(uint)));
  cudaCheck(hipMalloc((void**)&dev_cluster_candidates, number_of_events * 2000 * sizeof(uint)));
  cudaCheck(hipMalloc((void**)&dev_velo_cluster_container, velo_cluster_container_size * sizeof(uint)));

  // phi and sort
  cudaCheck(hipMalloc((void**)&dev_hit_permutation, average_number_of_hits_per_event * number_of_events * sizeof(uint)));

  // sbt
  // cudaCheck(hipMalloc((void**)&dev_tracks_to_follow, number_of_events * TTF_MODULO * sizeof(uint)));
  dev_tracks_to_follow = dev_cluster_candidates;
  
  cudaCheck(hipMalloc((void**)&dev_tracks, number_of_events * max_tracks_in_event * sizeof(Track)));
  cudaCheck(hipMalloc((void**)&dev_tracklets, average_number_of_hits_per_event * number_of_events * sizeof(Track)));
  cudaCheck(hipMalloc((void**)&dev_weak_tracks, average_number_of_hits_per_event * number_of_events * sizeof(uint)));
  cudaCheck(hipMalloc((void**)&dev_hit_used, average_number_of_hits_per_event * number_of_events * sizeof(bool)));
  cudaCheck(hipMalloc((void**)&dev_atomics_storage, number_of_events * atomic_space * sizeof(int)));
  cudaCheck(hipMalloc((void**)&dev_h0_candidates, 2 * average_number_of_hits_per_event * number_of_events * sizeof(short)));
  cudaCheck(hipMalloc((void**)&dev_h2_candidates, 2 * average_number_of_hits_per_event * number_of_events * sizeof(short)));
  cudaCheck(hipMalloc((void**)&dev_rel_indices, number_of_events * max_numhits_in_module * sizeof(unsigned short)));

  // Memory allocations for host memory (copy back)
  cudaCheck(hipHostMalloc((void**)&host_number_of_tracks_pinned, number_of_events * sizeof(int)));
  cudaCheck(hipHostMalloc((void**)&host_tracks_pinned, number_of_events * max_tracks_in_event * sizeof(Track)));

  // Pre-populate raw_input data, in case the user requested -a 0
  cudaCheck(hipMemcpyAsync(dev_raw_input, raw_events.data(), raw_events.size(), hipMemcpyHostToDevice, stream));
  cudaCheck(hipMemcpyAsync(dev_raw_input_offsets, event_offsets.data(), event_offsets.size() * sizeof(uint), hipMemcpyHostToDevice, stream));

  // Prepare kernels
  estimateInputSize.set(
    estimate_input_size_blocks,
    estimate_input_size_threads,
    stream,
    dev_raw_input,
    dev_raw_input_offsets,
    dev_estimated_input_size,
    dev_module_cluster_num,
    dev_module_candidate_num,
    dev_cluster_candidates
  );

  prefixSum.set(
    prefix_sum_blocks,
    prefix_sum_threads,
    stream,
    dev_estimated_input_size,
    number_of_events * 52
  );

  maskedVeloClustering.set(
    masked_velo_clustering_blocks,
    masked_velo_clustering_threads,
    stream,
    dev_raw_input,
    dev_raw_input_offsets,
    dev_estimated_input_size,
    dev_module_cluster_num,
    dev_module_candidate_num,
    dev_cluster_candidates,
    dev_velo_cluster_container,
    dev_sp_patterns,
    dev_sp_sizes,
    dev_sp_fx,
    dev_sp_fy,
    dev_velo_geometry
  );

  calculatePhiAndSort.set(
    num_blocks,
    sort_num_threads,
    stream,
    dev_estimated_input_size,
    dev_module_cluster_num,
    dev_velo_cluster_container,
    dev_hit_permutation
  );

  searchByTriplet.set(
    num_blocks,
    sbt_num_threads,
    stream,
    dev_velo_cluster_container,
    dev_estimated_input_size,
    dev_module_cluster_num,
    dev_tracks,
    dev_tracklets,
    dev_tracks_to_follow,
    dev_weak_tracks,
    dev_hit_used,
    dev_atomics_storage,
    dev_h0_candidates,
    dev_h2_candidates,
    dev_rel_indices
  );

  consolidateTracks.set(
    consolidate_blocks,
    consolidate_num_threads,
    stream,
    dev_atomics_storage,
    dev_tracks,
    dev_tracklets
  );

  return hipSuccess;
}