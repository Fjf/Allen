#include "../include/Stream.cuh"

hipError_t Stream::initialize(
  const std::vector<char>& raw_events,
  const std::vector<uint>& event_offsets,
  const std::vector<uint>& hit_offsets,
  const std::vector<char>& geometry,
  const uint number_of_events,
  const size_t param_starting_events_size,
  const bool param_transmit_host_to_device,
  const bool param_transmit_device_to_host,
  const uint param_stream_number
) {
  cudaCheck(hipStreamCreate(&stream));
  cudaCheck(hipEventCreate(&cuda_generic_event));
  stream_number = param_stream_number;
  dev_events_size = param_starting_events_size;
  transmit_host_to_device = param_transmit_host_to_device;
  transmit_device_to_host = param_transmit_device_to_host;

  // Blocks and threads for each algorithm
  num_blocks = dim3(number_of_events);
  estimate_input_size_blocks = dim3(number_of_events);
  prefix_sum_blocks = dim3(1);
  masked_velo_clustering_blocks = dim3(number_of_events);
  consolidate_blocks = dim3(number_of_events);

  estimate_input_size_threads = dim3(4, 208);
  prefix_sum_threads = dim3(1024);
  masked_velo_clustering_threads = dim3(256);
  sort_num_threads = dim3(64);
  sbt_num_threads = dim3(NUMTHREADS_X);
  consolidate_num_threads = dim3(32);

  // velo cluster container contains:
  // - cluster_xs
  // - cluster_ys
  // - cluster_zs
  // - cluster_ids
  // - cluster_phis
  // - temporary
  // 
  // The temporary is required to do the sorting in an efficient manner
  velo_cluster_container_size = number_of_events * 2000 * 6;

  // Do memory allocations only once
  // Clustering patterns
  cudaCheck(hipMalloc((void**)&dev_sp_patterns, 256));
  cudaCheck(hipMalloc((void**)&dev_sp_sizes, 256));
  cudaCheck(hipMalloc((void**)&dev_sp_fx, 512 * sizeof(float)));
  cudaCheck(hipMalloc((void**)&dev_sp_fy, 512 * sizeof(float)));

  // Fetch patterns and populate in GPU
  std::vector<unsigned char> sp_patterns (256, 0);
  std::vector<unsigned char> sp_sizes (256, 0);
  std::vector<float> sp_fx (512, 0);
  std::vector<float> sp_fy (512, 0);
  cache_sp_patterns(sp_patterns, sp_sizes, sp_fx, sp_fy);
  cudaCheck(hipMemcpyAsync(dev_sp_patterns, sp_patterns.data(), sp_patterns.size(), hipMemcpyHostToDevice, stream));
  cudaCheck(hipMemcpyAsync(dev_sp_sizes, sp_sizes.data(), sp_sizes.size(), hipMemcpyHostToDevice, stream));
  cudaCheck(hipMemcpyAsync(dev_sp_fx, sp_fx.data(), sp_fx.size() * sizeof(float), hipMemcpyHostToDevice, stream));
  cudaCheck(hipMemcpyAsync(dev_sp_fy, sp_fy.data(), sp_fy.size() * sizeof(float), hipMemcpyHostToDevice, stream));

  // Clustering
  cudaCheck(hipMalloc((void**)&dev_raw_input, dev_events_size));
  cudaCheck(hipMalloc((void**)&dev_raw_input_offsets, event_offsets.size() * sizeof(uint)));
  cudaCheck(hipMalloc((void**)&dev_estimated_input_size, (number_of_events * 52 + 2) * sizeof(uint)));
  cudaCheck(hipMalloc((void**)&dev_module_cluster_num, number_of_events * 52 * sizeof(uint)));
  cudaCheck(hipMalloc((void**)&dev_module_candidate_num, number_of_events * sizeof(uint)));
  cudaCheck(hipMalloc((void**)&dev_cluster_candidates, number_of_events * 2000 * sizeof(uint)));
  cudaCheck(hipMalloc((void**)&dev_velo_geometry, geometry.size()));
  cudaCheck(hipMalloc((void**)&dev_velo_cluster_container, velo_cluster_container_size * sizeof(uint)));
  // Populate velo geometry
  cudaCheck(hipMemcpyAsync(dev_velo_geometry, geometry.data(), geometry.size(), hipMemcpyHostToDevice, stream));

  // phi and sort
  cudaCheck(hipMalloc((void**)&dev_hit_permutation, maximum_average_number_of_hits_per_event * number_of_events * sizeof(unsigned short)));
  // // sbt
  // cudaCheck(hipMalloc((void**)&dev_tracks, number_of_events * max_tracks_in_event * sizeof(Track)));
  // cudaCheck(hipMalloc((void**)&dev_tracks_to_follow, number_of_events * TTF_MODULO * sizeof(uint)));
  // cudaCheck(hipMalloc((void**)&dev_hit_used, maximum_average_number_of_hits_per_event * number_of_events * sizeof(bool)));
  // cudaCheck(hipMalloc((void**)&dev_atomics_storage, number_of_events * atomic_space * sizeof(int)));
  // cudaCheck(hipMalloc((void**)&dev_tracklets, maximum_average_number_of_hits_per_event * number_of_events * sizeof(Track)));
  // cudaCheck(hipMalloc((void**)&dev_weak_tracks, maximum_average_number_of_hits_per_event * number_of_events * sizeof(uint)));
  // cudaCheck(hipMalloc((void**)&dev_h0_candidates, 2 * maximum_average_number_of_hits_per_event * number_of_events * sizeof(short)));
  // cudaCheck(hipMalloc((void**)&dev_h2_candidates, 2 * maximum_average_number_of_hits_per_event * number_of_events * sizeof(short)));
  // cudaCheck(hipMalloc((void**)&dev_rel_indices, number_of_events * max_numhits_in_module * sizeof(unsigned short)));

  // // Memory allocations for host memory (copy back)
  // cudaCheck(hipHostMalloc((void**)&host_number_of_tracks_pinned, number_of_events * sizeof(int)));
  // cudaCheck(hipHostMalloc((void**)&host_tracks_pinned, number_of_events * max_tracks_in_event * sizeof(Track)));

  // // Prepare data (for tests)
  // cudaCheck(hipMemcpyAsync(dev_events, events.data(), events.size(), hipMemcpyHostToDevice, stream));
  // cudaCheck(hipMemcpyAsync(dev_event_offsets, event_offsets.data(), event_offsets.size() * sizeof(uint), hipMemcpyHostToDevice, stream));
  // cudaCheck(hipMemcpyAsync(dev_hit_offsets, hit_offsets.data(), hit_offsets.size() * sizeof(uint), hipMemcpyHostToDevice, stream));

  // Prepare kernels
  estimateInputSize.set(
    estimate_input_size_blocks,
    estimate_input_size_threads,
    stream,
    dev_raw_input,
    dev_raw_input_offsets,
    dev_estimated_input_size,
    dev_module_cluster_num,
    dev_module_candidate_num,
    dev_cluster_candidates
  );

  prefixSum.set(
    prefix_sum_blocks,
    prefix_sum_threads,
    stream,
    dev_estimated_input_size,
    number_of_events * 52
  );

  maskedVeloClustering.set(
    masked_velo_clustering_blocks,
    masked_velo_clustering_threads,
    stream,
    dev_raw_input,
    dev_raw_input_offsets,
    dev_estimated_input_size,
    dev_module_cluster_num,
    dev_module_candidate_num,
    dev_cluster_candidates,
    dev_velo_cluster_container,
    dev_sp_patterns,
    dev_sp_sizes,
    dev_sp_fx,
    dev_sp_fy,
    dev_velo_geometry
  );

  calculatePhiAndSort.set(
    num_blocks,
    sort_num_threads,
    stream,
    dev_estimated_input_size,
    dev_module_cluster_num,
    dev_velo_cluster_container,
    dev_hit_permutation
  );

  searchByTriplet.set(
    num_blocks,
    sbt_num_threads,
    stream,
    dev_tracks,
    dev_events,
    dev_tracks_to_follow,
    dev_hit_used,
    dev_atomics_storage,
    dev_tracklets,
    dev_weak_tracks,
    dev_event_offsets,
    dev_hit_offsets,
    dev_h0_candidates,
    dev_h2_candidates,
    dev_rel_indices,
    dev_hit_phi,
    dev_hit_temp
  );

  consolidateTracks.set(
    consolidate_blocks,
    consolidate_num_threads,
    stream,
    dev_atomics_storage,
    dev_tracks,
    dev_tracklets,
    dev_hit_offsets,
    dev_hit_permutation
  );

  return hipSuccess;
}