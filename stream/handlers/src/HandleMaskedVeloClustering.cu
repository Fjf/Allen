#include "hip/hip_runtime.h"
#include "HandleMaskedVeloClustering.cuh"

void MaskedVeloClustering::operator()() {
  masked_velo_clustering<<<num_blocks, num_threads, 0, *stream>>>(
    dev_raw_input,
    dev_raw_input_offsets,
    dev_module_cluster_start,
    dev_module_cluster_num,
    dev_module_candidate_num,
    dev_cluster_candidates,
    dev_velo_cluster_container,
    dev_velo_geometry
  );
}

void MaskedVeloClustering::print_output(
  const uint number_of_events,
  const int print_max_per_module
) {
  std::vector<uint> module_cluster_start (number_of_events * N_MODULES + 1);
  std::vector<uint> module_cluster_num (number_of_events * N_MODULES);
  cudaCheck(hipMemcpyAsync(module_cluster_start.data(), dev_module_cluster_start, module_cluster_start.size() * sizeof(uint), hipMemcpyDeviceToHost, *stream));
  cudaCheck(hipMemcpyAsync(module_cluster_num.data(), dev_module_cluster_num, module_cluster_num.size() * sizeof(uint), hipMemcpyDeviceToHost, *stream));

  const auto estimated_number_of_clusters = module_cluster_start[module_cluster_start.size() - 1];
  std::vector<uint32_t> velo_cluster_container (6 * estimated_number_of_clusters);
  cudaCheck(hipMemcpyAsync(velo_cluster_container.data(), dev_velo_cluster_container, velo_cluster_container.size() * sizeof(uint32_t), hipMemcpyDeviceToHost, *stream));

  float* cluster_xs = (float*) &velo_cluster_container[0];
  float* cluster_ys = (float*) &velo_cluster_container[estimated_number_of_clusters];
  float* cluster_zs = (float*) &velo_cluster_container[2 * estimated_number_of_clusters];
  uint32_t* cluster_ids = (uint32_t*) &velo_cluster_container[3 * estimated_number_of_clusters];

  // Print number of found clusters per event
  for (uint i=0; i<number_of_events; ++i) {
    uint found_clusters = 0;
    for (uint module=0; module<N_MODULES; ++module) {
      found_clusters += module_cluster_num[N_MODULES*i + module];
    }
    std::cout << "Event " << i << ": " << found_clusters << " clusters" << std::endl;
  }

  // // Print all clusters
  // for (uint i=0; i<number_of_events; ++i) {
  //   std::cout << "Event " << i << std::endl;
  //   for (uint module=0; module<N_MODULES; ++module) {
  //     std::cout << " Module " << module << ":";
  //     const auto mod_start = module_cluster_start[N_MODULES*i + module];
  //     for (uint cluster=0; cluster<module_cluster_num[N_MODULES*i + module]; ++cluster) {
  //       if (print_max_per_module != -1 && cluster >= print_max_per_module) break;

  //       const auto x = cluster_xs[mod_start + cluster];
  //       const auto y = cluster_ys[mod_start + cluster];
  //       const auto z = cluster_zs[mod_start + cluster];
  //       const auto id = cluster_ids[mod_start + cluster];

  //       std::cout << " {" << x << ", " << y << ", " << z << " (#" << id << ")}";
  //     }
  //     std::cout << std::endl;
  //   }
  //   std::cout << std::endl;
  // }

  // // Print checksum for every event
  // for (uint i=0; i<number_of_events; ++i) {
  //   float sum = 0;
  //   for (uint module=0; module<N_MODULES; ++module) {
  //     const auto mod_start = module_cluster_start[N_MODULES*i + module];
  //     for (uint cluster=0; cluster<module_cluster_num[N_MODULES*i + module]; ++cluster) {
  //       if (print_max_per_module != -1 && cluster >= print_max_per_module) break;
  //       const auto x = cluster_xs[mod_start + cluster];
  //       const auto y = cluster_ys[mod_start + cluster];
  //       const auto z = cluster_zs[mod_start + cluster];
  //       sum += x + y + z;
  //     }
  //   }
  //   std::cout << "Event " << i << ": " << sum << std::endl;
  // }
}

void MaskedVeloClustering::check(
  const char* host_events_pinned,
  const uint* host_event_offsets_pinned,
  const size_t host_events_pinned_size,
  const size_t host_event_offsets_pinned_size,
  const std::vector<char>& geometry,
  const uint number_of_events
) {
  std::vector<char> events (host_events_pinned_size);
  std::vector<uint> event_offsets (host_event_offsets_pinned_size);
  std::copy_n(host_events_pinned, host_events_pinned_size, events.data());
  std::copy_n(host_event_offsets_pinned, host_event_offsets_pinned_size, event_offsets.data());

  std::vector<uint> module_cluster_start (number_of_events * N_MODULES + 1);
  std::vector<uint> module_cluster_num (number_of_events * N_MODULES);
  cudaCheck(hipMemcpyAsync(module_cluster_start.data(), dev_module_cluster_start, module_cluster_start.size() * sizeof(uint), hipMemcpyDeviceToHost, *stream));
  cudaCheck(hipMemcpyAsync(module_cluster_num.data(), dev_module_cluster_num, module_cluster_num.size() * sizeof(uint), hipMemcpyDeviceToHost, *stream));

  const auto estimated_number_of_clusters = module_cluster_start[module_cluster_start.size() - 1];
  std::vector<uint32_t> velo_cluster_container (number_of_events * 2000 * 6);
  cudaCheck(hipMemcpyAsync(velo_cluster_container.data(), dev_velo_cluster_container, velo_cluster_container.size() * sizeof(uint32_t), hipMemcpyDeviceToHost, *stream));
  uint32_t* cluster_ids = (uint32_t*) &velo_cluster_container[3 * estimated_number_of_clusters];

  std::vector<std::vector<uint32_t>> found_lhcb_ids;
  for (uint i=0; i<number_of_events; ++i) {
    std::vector<uint32_t> event_found_lhcb_ids;
    for (uint module=0; module<N_MODULES; ++module) {
      const uint mod_num = module_cluster_num[N_MODULES*i + module];
      const uint mod_start = module_cluster_start[N_MODULES*i + module];
      for (uint cluster=0; cluster<mod_num; ++cluster) {
        const auto id = cluster_ids[mod_start + cluster];
        event_found_lhcb_ids.emplace_back(id);
      }
    }
    found_lhcb_ids.emplace_back(event_found_lhcb_ids);
  }

  float reconstruction_efficiency;
  float clone_fraction;
  float ghost_fraction;

  std::cout << "Clustering efficiency across " << number_of_events << " events:" << std::endl;

  checkClustering(
    geometry,
    events,
    event_offsets,
    found_lhcb_ids,
    reconstruction_efficiency,
    clone_fraction,
    ghost_fraction,
    false,
    2.0f
  );

  std::cout << "With LHCb ID comparison max distance 2.0:" << std::endl
    << " Reconstruction efficiency: " << reconstruction_efficiency << std::endl
    << " Clone fraction: " << clone_fraction << std::endl
    << " Ghost fraction: " << ghost_fraction << std::endl << std::endl;
}
