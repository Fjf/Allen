#include "ParKalmanVeloOnly.cuh"
#include "SequenceVisitor.cuh"

template<>
void SequenceVisitor::set_arguments_size<package_kalman_tracks_t>(
  const package_kalman_tracks_t& state,
  package_kalman_tracks_t::arguments_t arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers)
{
  arguments.set_size<dev_kf_tracks>(host_buffers.host_number_of_reconstructed_scifi_tracks[0]);
}

template<>
void SequenceVisitor::visit<package_kalman_tracks_t>(
  package_kalman_tracks_t& state,
  const package_kalman_tracks_t::arguments_t& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  state.set_opts(dim3(host_buffers.host_number_of_selected_events[0]), dim3(256), hip_stream);
  state.set_arguments(
    arguments.offset<dev_atomics_velo>(),
    arguments.offset<dev_velo_track_hit_number>(),
    arguments.offset<dev_atomics_ut>(),
    arguments.offset<dev_ut_track_hit_number>(),
    arguments.offset<dev_ut_qop>(),
    arguments.offset<dev_ut_track_velo_indices>(),
    arguments.offset<dev_atomics_scifi>(),
    arguments.offset<dev_scifi_track_hit_number>(),
    arguments.offset<dev_scifi_qop>(),
    arguments.offset<dev_scifi_states>(),
    arguments.offset<dev_scifi_track_ut_indices>(),
    arguments.offset<dev_velo_kalman_beamline_states>(),
    arguments.offset<dev_is_muon>(),
    arguments.offset<dev_kf_tracks>());
  state.invoke();
}
