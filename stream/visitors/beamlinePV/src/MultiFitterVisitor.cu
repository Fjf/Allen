#include "SequenceVisitor.cuh"
#include "pv_beamline_multi_fitter.cuh"

template<>
void SequenceVisitor::set_arguments_size<pv_beamline_multi_fitter_t>(
  const pv_beamline_multi_fitter_t& state,
  pv_beamline_multi_fitter_t::arguments_t arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers)
{
  // Set arguments size
  arguments.set_size<dev_multi_fit_vertices>(host_buffers.host_number_of_selected_events[0] * PV::max_number_vertices);
  arguments.set_size<dev_number_of_multi_fit_vertices>(host_buffers.host_number_of_selected_events[0]);
  arguments.set_size<dev_pvtracks_denom>(host_buffers.host_number_of_reconstructed_velo_tracks[0]);
}

template<>
void SequenceVisitor::visit<pv_beamline_multi_fitter_t>(
  pv_beamline_multi_fitter_t& state,
  const pv_beamline_multi_fitter_t::arguments_t& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  state.set_opts(dim3(host_buffers.host_number_of_selected_events[0]), hip_stream);
  state.set_arguments(
    arguments.offset<dev_atomics_velo>(),
    arguments.offset<dev_velo_track_hit_number>(),
    arguments.offset<dev_pvtracks>(),
    arguments.offset<dev_pvtracks_denom>(),
    arguments.offset<dev_zpeaks>(),
    arguments.offset<dev_number_of_zpeaks>(),
    arguments.offset<dev_multi_fit_vertices>(),
    arguments.offset<dev_number_of_multi_fit_vertices>(),
    constants.dev_beamline.data());

  cudaCheck(hipMemsetAsync(
    arguments.offset<dev_number_of_multi_fit_vertices>(),
    0,
    arguments.size<dev_number_of_multi_fit_vertices>(),
    hip_stream));

  state.invoke();
}
