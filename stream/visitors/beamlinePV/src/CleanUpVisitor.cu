#include "SequenceVisitor.cuh"
#include "pv_beamline_cleanup.cuh"

template<>
void SequenceVisitor::set_arguments_size<pv_beamline_cleanup_t>(
  pv_beamline_cleanup_t::arguments_t arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers)
{
  // Set arguments size
  arguments.set_size<dev_multi_final_vertices>(
    host_buffers.host_number_of_selected_events[0] * PV::max_number_vertices);
  arguments.set_size<dev_number_of_multi_final_vertices>(host_buffers.host_number_of_selected_events[0]);
}

template<>
void SequenceVisitor::visit<pv_beamline_cleanup_t>(
  pv_beamline_cleanup_t& state,
  const pv_beamline_cleanup_t::arguments_t& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  state.set_opts(dim3(host_buffers.host_number_of_selected_events[0]), dim3(1), hip_stream);
  state.set_arguments(
    arguments.offset<dev_multi_fit_vertices>(),
    arguments.offset<dev_number_of_multi_fit_vertices>(),
    arguments.offset<dev_multi_final_vertices>(),
    arguments.offset<dev_number_of_multi_final_vertices>());

  cudaCheck(hipMemsetAsync(
    arguments.offset<dev_number_of_multi_final_vertices>(),
    0,
    arguments.size<dev_number_of_multi_final_vertices>(),
    hip_stream));

  state.invoke();
  // Wait to receive the result
  hipEventRecord(cuda_generic_event, hip_stream);
  hipEventSynchronize(cuda_generic_event);

  // Retrieve result
  cudaCheck(hipMemcpyAsync(
    host_buffers.host_reconstructed_multi_pvs,
    arguments.offset<dev_multi_final_vertices>(),
    arguments.size<dev_multi_final_vertices>(),
    hipMemcpyDeviceToHost,
    hip_stream));

  cudaCheck(hipMemcpyAsync(
    host_buffers.host_number_of_multivertex,
    arguments.offset<dev_number_of_multi_final_vertices>(),
    arguments.size<dev_number_of_multi_final_vertices>(),
    hipMemcpyDeviceToHost,
    hip_stream));

  // Wait to receive the result
  hipEventRecord(cuda_generic_event, hip_stream);
  hipEventSynchronize(cuda_generic_event);
}
