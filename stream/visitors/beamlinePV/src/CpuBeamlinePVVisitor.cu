#include "SequenceVisitor.cuh"
#include "RunBeamlinePVonCPU.h"
#include "Tools.h"

DEFINE_EMPTY_SET_ARGUMENTS_SIZE(cpu_beamlinePV_t)

template<>
void SequenceVisitor::visit<cpu_beamlinePV_t>(
  cpu_beamlinePV_t& state,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  argument_manager_t& arguments,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  // Synchronize previous CUDA transmissions
  hipEventRecord(cuda_generic_event, hip_stream);
  hipEventSynchronize(cuda_generic_event);

  cudaCheck(hipMemcpyAsync(
    host_buffers.host_kalmanvelo_states,
    arguments.offset<dev_kalmanvelo_states>(),
    arguments.size<dev_kalmanvelo_states>(),
    hipMemcpyDeviceToHost,
    hip_stream));

  state.invoke(
    host_buffers.host_kalmanvelo_states,
    host_buffers.host_atomics_velo,
    host_buffers.host_velo_track_hit_number,
    host_buffers.host_reconstructed_pvs,
    host_buffers.host_number_of_vertex,
    runtime_options.number_of_events);
  
  for ( int i_event = 0; i_event < runtime_options.number_of_events; i_event++ ) {
    debug_cout << "# of PVs found = " << host_buffers.host_number_of_vertex[i_event] << std::endl;
  }

}
