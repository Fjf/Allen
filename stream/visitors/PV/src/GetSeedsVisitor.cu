#include "SequenceVisitor.cuh"
#include "GetSeeds.cuh"

template<>
void SequenceVisitor::set_arguments_size<pv_get_seeds_t>(
  pv_get_seeds_t& state,
  pv_get_seeds_t::arguments_t arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers)
{
  // Set arguments size
  arguments.set_size<dev_seeds>(host_buffers.host_number_of_reconstructed_velo_tracks[0]);
  arguments.set_size<dev_number_seeds>(host_buffers.host_number_of_selected_events[0]);
}

template<>
void SequenceVisitor::visit<pv_get_seeds_t>(
  pv_get_seeds_t& state,
  const pv_get_seeds_t::arguments_t& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{

  state.set_opts(dim3(host_buffers.host_number_of_selected_events[0]), hip_stream);
  state.set_arguments(
    arguments.offset<dev_velo_kalman_beamline_states>(),
    arguments.offset<dev_atomics_velo>(),
    arguments.offset<dev_velo_track_hit_number>(),
    arguments.offset<dev_seeds>(),
    arguments.offset<dev_number_seeds>());

  state.invoke();

  if (runtime_options.do_check) {
    cudaCheck(hipMemcpyAsync(
      host_buffers.host_number_of_seeds,
      arguments.offset<dev_number_seeds>(),
      arguments.size<dev_number_seeds>(),
      hipMemcpyDeviceToHost,
      hip_stream));
  }
}
