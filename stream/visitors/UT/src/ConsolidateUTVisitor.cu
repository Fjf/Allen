#include "SequenceVisitor.cuh"
#include "ConsolidateUT.cuh"

template<>
void SequenceVisitor::set_arguments_size<consolidate_ut_tracks_t>(
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers,
  argument_manager_t& arguments)
{
  arguments.set_size<dev_ut_track_hits>(host_buffers.host_accumulated_number_of_ut_hits[0]*sizeof(UT::Hit));
  arguments.set_size<dev_ut_qop>(host_buffers.host_number_of_reconstructed_ut_tracks[0]);
  arguments.set_size<dev_ut_track_velo_indices>(host_buffers.host_number_of_reconstructed_ut_tracks[0]);
  arguments.set_size<dev_ut_x>(host_buffers.host_number_of_reconstructed_ut_tracks[0]);
  arguments.set_size<dev_ut_z>(host_buffers.host_number_of_reconstructed_ut_tracks[0]);
  arguments.set_size<dev_ut_tx>(host_buffers.host_number_of_reconstructed_ut_tracks[0]);
}

template<>
void SequenceVisitor::visit<consolidate_ut_tracks_t>(
  consolidate_ut_tracks_t& state,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  argument_manager_t& arguments,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  state.set_opts(dim3(host_buffers.host_number_of_selected_events[0]), dim3(256), hip_stream);
  state.set_arguments(
    arguments.offset<dev_ut_hits>(),
    arguments.offset<dev_ut_hit_offsets>(),
    arguments.offset<dev_ut_track_hits>(),
    arguments.offset<dev_atomics_ut>(),
    arguments.offset<dev_ut_track_hit_number>(),
    arguments.offset<dev_ut_qop>(),
    arguments.offset<dev_ut_x>(),
    arguments.offset<dev_ut_tx>(),
    arguments.offset<dev_ut_z>(),
    arguments.offset<dev_ut_track_velo_indices>(),
    arguments.offset<dev_ut_tracks>(),
    constants.dev_unique_x_sector_layer_offsets
  );

  state.invoke();

  // Transmission device to host of UT consolidated tracks
  cudaCheck(hipMemcpyAsync(
    host_buffers.host_atomics_ut,
    arguments.offset<dev_atomics_ut>(),
    (2 * host_buffers.host_number_of_selected_events[0] + 1) * sizeof(uint),
    hipMemcpyDeviceToHost, 
    hip_stream));

  cudaCheck(hipMemcpyAsync(
    host_buffers.host_ut_track_hit_number,
    arguments.offset<dev_ut_track_hit_number>(),
    arguments.size<dev_ut_track_hit_number>(),
    hipMemcpyDeviceToHost,
    hip_stream));
 
  cudaCheck(hipMemcpyAsync(
    host_buffers.host_ut_track_hits,
    arguments.offset<dev_ut_track_hits>(),
    host_buffers.host_accumulated_number_of_hits_in_ut_tracks[0]*sizeof(UT::Hit), 
    hipMemcpyDeviceToHost,
    hip_stream));

  cudaCheck(hipMemcpyAsync(
    host_buffers.host_ut_qop,
    arguments.offset<dev_ut_qop>(),
    arguments.size<dev_ut_qop>(),
    hipMemcpyDeviceToHost,
    hip_stream));

  cudaCheck(hipMemcpyAsync(
    host_buffers.host_ut_x,
    arguments.offset<dev_ut_x>(),
    arguments.size<dev_ut_x>(),
    hipMemcpyDeviceToHost,
    hip_stream));

  cudaCheck(hipMemcpyAsync(
    host_buffers.host_ut_tx,
    arguments.offset<dev_ut_tx>(),
    arguments.size<dev_ut_tx>(),
    hipMemcpyDeviceToHost,
    hip_stream));

  cudaCheck(hipMemcpyAsync(
    host_buffers.host_ut_z,
    arguments.offset<dev_ut_z>(),
    arguments.size<dev_ut_z>(),
    hipMemcpyDeviceToHost,
    hip_stream));
  
  cudaCheck(hipMemcpyAsync(
    host_buffers.host_ut_track_velo_indices,
    arguments.offset<dev_ut_track_velo_indices>(),
    arguments.size<dev_ut_track_velo_indices>(),
    hipMemcpyDeviceToHost,
    hip_stream));
}
