#include "SequenceVisitor.cuh"
#include "UTPreDecode.cuh"

template<>
void SequenceVisitor::set_arguments_size<ut_pre_decode_t>(
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers,
  argument_manager_t& arguments)
{
  arguments.set_size<dev_ut_hits>(UT::Hits::number_of_arrays * host_buffers.host_accumulated_number_of_ut_hits[0]);
  arguments.set_size<dev_ut_hit_count>(host_buffers.host_number_of_selected_events[0] * constants.host_unique_x_sector_layer_offsets[4]);
}

template<>
void SequenceVisitor::visit<ut_pre_decode_t>(
  ut_pre_decode_t& state,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  argument_manager_t& arguments,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  cudaCheck(hipMemsetAsync(arguments.offset<dev_ut_hit_count>(),
    0,
    arguments.size<dev_ut_hit_count>(),
    hip_stream));

  state.set_opts(dim3(host_buffers.host_number_of_selected_events[0]), dim3(64, 4), hip_stream);
  state.set_arguments(
    arguments.offset<dev_ut_raw_input>(),
    arguments.offset<dev_ut_raw_input_offsets>(),
    arguments.offset<dev_event_list>(),
    constants.dev_ut_boards,
    constants.dev_ut_geometry,
    constants.dev_ut_region_offsets,
    constants.dev_unique_x_sector_layer_offsets,
    constants.dev_unique_x_sector_offsets,
    arguments.offset<dev_ut_hit_offsets>(),
    arguments.offset<dev_ut_hits>(),
    arguments.offset<dev_ut_hit_count>()
  );

  state.invoke();
}
