#include "SequenceVisitor.cuh"
#include "VeloUT.cuh"

template<>
void SequenceVisitor::set_arguments_size<veloUT_t>(
  veloUT_t::arguments_t& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers)
{
  arguments.set_size<dev_ut_tracks>(host_buffers.host_number_of_selected_events[0] * UT::Constants::max_num_tracks);
  arguments.set_size<dev_atomics_ut>(host_buffers.host_number_of_selected_events[0] * UT::num_atomics + 1);
}

template<>
void SequenceVisitor::visit<veloUT_t>(
  veloUT_t& state,
  const veloUT_t::arguments_t& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  state.set_opts(dim3(host_buffers.host_number_of_selected_events[0]), dim3(32), hip_stream);
  state.set_arguments(
    arguments.offset<dev_ut_hits>(),
    arguments.offset<dev_ut_hit_offsets>(),
    arguments.offset<dev_atomics_velo>(),
    arguments.offset<dev_velo_track_hit_number>(),
    arguments.offset<dev_velo_track_hits>(),
    arguments.offset<dev_velo_states>(),
    arguments.offset<dev_ut_tracks>(),
    arguments.offset<dev_atomics_ut>(),
    constants.dev_ut_magnet_tool,
    constants.dev_ut_dxDy,
    constants.dev_unique_x_sector_layer_offsets,
    constants.dev_unique_x_sector_offsets,
    constants.dev_unique_sector_xs
  );

  state.invoke();
}
