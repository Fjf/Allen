#include "SequenceVisitor.cuh"
#include "CompassUT.cuh"

template<>
void SequenceVisitor::set_arguments_size<compass_ut_t>(
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers,
  argument_manager_t& arguments)
{
  arguments.set_size<dev_ut_tracks>(host_buffers.host_number_of_selected_events[0] * UT::Constants::max_num_tracks);
  arguments.set_size<dev_atomics_ut>(host_buffers.host_number_of_selected_events[0] * UT::num_atomics + 1);
  // arguments.set_size<dev_active_tracks>(host_buffers.host_number_of_selected_events[0]);
}

template<>
void SequenceVisitor::visit<compass_ut_t>(
  compass_ut_t& state,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  argument_manager_t& arguments,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  state.set_opts(dim3(host_buffers.host_number_of_selected_events[0]), dim3(UT::Constants::num_threads), hip_stream);

  state.set_arguments(
    arguments.offset<dev_ut_hits>(),
    arguments.offset<dev_ut_hit_offsets>(),
    arguments.offset<dev_atomics_velo>(),
    arguments.offset<dev_velo_track_hit_number>(),
    arguments.offset<dev_velo_track_hits>(),
    arguments.offset<dev_velo_states>(),
    constants.dev_ut_magnet_tool,
    constants.dev_ut_dxDy,
    arguments.offset<dev_active_tracks>(),
    constants.dev_unique_x_sector_layer_offsets,
    constants.dev_unique_sector_xs,
    arguments.offset<dev_ut_tracks>(),
    arguments.offset<dev_atomics_ut>(),
    arguments.offset<dev_windows_layers>()    
  );

  cudaCheck(hipMemsetAsync(arguments.offset<dev_active_tracks>(), 0, arguments.size<dev_active_tracks>(), hip_stream));
  cudaCheck(hipMemsetAsync(arguments.offset<dev_atomics_ut>(), 0, arguments.size<dev_atomics_ut>(), hip_stream));

  state.invoke();
}
