#include "SequenceVisitor.cuh"
#include "CompassUT.cuh"

template<>
void SequenceVisitor::set_arguments_size<compass_ut_t>(
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers,
  argument_manager_t& arguments)
{
  arguments.set_size<dev_ut_tracks>(runtime_options.number_of_events * UT::Constants::max_num_tracks);
  arguments.set_size<dev_atomics_ut>(runtime_options.number_of_events * UT::num_atomics + 1);
  arguments.set_size<dev_active_tracks>(runtime_options.number_of_events);
}

template<>
void SequenceVisitor::visit<compass_ut_t>(
  compass_ut_t& state,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  argument_manager_t& arguments,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  state.set_opts(dim3(runtime_options.number_of_events), dim3(CompassUT::num_threads), hip_stream);

  state.set_arguments(
    arguments.offset<dev_ut_hits>(),
    arguments.offset<dev_ut_hit_offsets>(),
    arguments.offset<dev_atomics_velo>(),
    arguments.offset<dev_velo_track_hit_number>(),
    arguments.offset<dev_velo_track_hits>(),
    arguments.offset<dev_velo_states>(),
    constants.dev_ut_magnet_tool,
    constants.dev_ut_dxDy,
    arguments.offset<dev_active_tracks>(),
    constants.dev_unique_x_sector_layer_offsets,
    constants.dev_unique_sector_xs,
    arguments.offset<dev_ut_tracks>(),
    arguments.offset<dev_atomics_ut>(),
    arguments.offset<dev_windows_layers>()    
  );
  state.invoke();
}
