#include "SequenceVisitor.cuh"
#include "CompassUT.cuh"

template<>
void SequenceVisitor::set_arguments_size<compass_ut_t>(
  compass_ut_t::arguments_t& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers)
{
  arguments.set_size<dev_ut_tracks>(host_buffers.host_number_of_selected_events[0] * UT::Constants::max_num_tracks);
  arguments.set_size<dev_atomics_ut>(host_buffers.host_number_of_selected_events[0] * UT::num_atomics + 1);
  arguments.set_size<dev_active_tracks>(host_buffers.host_number_of_selected_events[0]);
}

template<>
void SequenceVisitor::visit<compass_ut_t>(
  compass_ut_t& state,
  const compass_ut_t::arguments_t& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  state.set_opts(dim3(host_buffers.host_number_of_selected_events[0]), dim3(CompassUT::num_threads), hip_stream);

  state.set_arguments(
    arguments.offset<dev_ut_hits>(),
    arguments.offset<dev_ut_hit_offsets>(),
    arguments.offset<dev_atomics_velo>(),
    arguments.offset<dev_velo_track_hit_number>(),
    arguments.offset<dev_velo_track_hits>(),
    arguments.offset<dev_velo_states>(),
    constants.dev_ut_magnet_tool,
    constants.dev_ut_dxDy,
    arguments.offset<dev_active_tracks>(),
    constants.dev_unique_x_sector_layer_offsets,
    constants.dev_unique_sector_xs,
    arguments.offset<dev_ut_tracks>(),
    arguments.offset<dev_atomics_ut>(),
    arguments.offset<dev_windows_layers>()    
  );
  state.invoke();
}
