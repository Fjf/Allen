#include "SequenceVisitor.cuh"
#include "PrefixSum.cuh"
#include "CpuPrefixSum.cuh"

DEFINE_EMPTY_SET_ARGUMENTS_SIZE(copy_and_prefix_sum_single_block_ut_t)

template<>
void SequenceVisitor::visit<copy_and_prefix_sum_single_block_ut_t>(
  copy_and_prefix_sum_single_block_ut_t& state,
  const copy_and_prefix_sum_single_block_ut_t::arguments_t& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  if (runtime_options.cpu_offload) {
    // Copy
    cudaCheck(hipMemcpyAsync(
      (uint*) arguments.offset<dev_atomics_ut>() + host_buffers.host_number_of_selected_events[0],
      (uint*) arguments.offset<dev_atomics_ut>(),
      host_buffers.host_number_of_selected_events[0] * sizeof(uint),
      hipMemcpyDeviceToDevice,
      hip_stream));

    // Prefix sum
    cpu_prefix_sum(
      host_buffers.host_prefix_sum_buffer,
      host_buffers.host_allocated_prefix_sum_space,
      (uint*) arguments.offset<dev_atomics_ut>() + host_buffers.host_number_of_selected_events[0],
      (host_buffers.host_number_of_selected_events[0] + 1) * sizeof(uint),
      hip_stream,
      cuda_generic_event,
      host_buffers.host_number_of_reconstructed_ut_tracks);
  }
  else {
    // Calculate prefix sum of found UT tracks.
    state.set_opts(hip_stream);
    state.set_arguments(
      (uint*) arguments.offset<dev_atomics_ut>() + host_buffers.host_number_of_selected_events[0] * 2,
      (uint*) arguments.offset<dev_atomics_ut>(),
      (uint*) arguments.offset<dev_atomics_ut>() + host_buffers.host_number_of_selected_events[0],
      host_buffers.host_number_of_selected_events[0]);

    state.invoke();

    // Fetch number of reconstructed UT tracks.
    cudaCheck(hipMemcpyAsync(
      host_buffers.host_number_of_reconstructed_ut_tracks,
      arguments.offset<dev_atomics_ut>() + host_buffers.host_number_of_selected_events[0] * 2,
      sizeof(uint),
      hipMemcpyDeviceToHost,
      hip_stream));

    hipEventRecord(cuda_generic_event, hip_stream);
    hipEventSynchronize(cuda_generic_event);
  }
}
