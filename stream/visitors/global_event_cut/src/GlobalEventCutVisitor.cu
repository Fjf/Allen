#include "SequenceVisitor.cuh"
#include "GlobalEventCut.cuh"
#include "CpuGlobalEventCut.cuh"

DEFINE_EMPTY_SET_ARGUMENTS_SIZE(global_event_cut_t)

template<>
void SequenceVisitor::visit<global_event_cut_t>(
  global_event_cut_t& state,
  const global_event_cut_t::arguments_t& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  if (runtime_options.cpu_offload) {
    cpu_global_event_cut(
      std::get<0>(runtime_options.host_ut_events).data(),
      std::get<1>(runtime_options.host_ut_events).data(),
      std::get<0>(runtime_options.host_scifi_events).data(),
      std::get<1>(runtime_options.host_scifi_events).data(),
      host_buffers.host_number_of_selected_events,
      host_buffers.host_event_list,
      runtime_options.number_of_events);

    cudaCheck(hipMemcpyAsync(
      arguments.offset<dev_event_list>(),
      host_buffers.host_event_list,
      runtime_options.number_of_events * sizeof(uint),
      hipMemcpyHostToDevice,
      hip_stream));
  }
  else {
    cudaCheck(hipMemsetAsync(arguments.offset<dev_number_of_selected_events>(), 0, sizeof(uint), hip_stream));

    // Setup opts and arguments for kernel call
    state.set_opts(dim3(runtime_options.number_of_events), hip_stream);
    state.set_arguments(
      arguments.offset<dev_ut_raw_input>(),
      arguments.offset<dev_ut_raw_input_offsets>(),
      arguments.offset<dev_scifi_raw_input>(),
      arguments.offset<dev_scifi_raw_input_offsets>(),
      arguments.offset<dev_number_of_selected_events>(),
      arguments.offset<dev_event_list>());

    state.invoke();

    cudaCheck(hipMemcpyAsync(
      host_buffers.host_number_of_selected_events,
      arguments.offset<dev_number_of_selected_events>(),
      sizeof(uint),
      hipMemcpyDeviceToHost,
      hip_stream));

    cudaCheck(hipMemcpyAsync(
      host_buffers.host_event_list,
      arguments.offset<dev_event_list>(),
      runtime_options.number_of_events * sizeof(uint),
      hipMemcpyHostToDevice,
      hip_stream));

    hipEventRecord(cuda_generic_event, hip_stream);
    hipEventSynchronize(cuda_generic_event);
  }

  if (logger::ll.verbosityLevel >= logger::debug) {
    debug_cout << "Selected " << host_buffers.host_number_of_selected_events[0] << " / "
               << runtime_options.number_of_events << " events with global event cuts" << std::endl;
  }
}
