#include "SequenceVisitor.cuh"
#include "InitEventList.cuh"

template<>
void SequenceVisitor::set_arguments_size<init_event_list_t>(
  const init_event_list_t& state,
  init_event_list_t::arguments_t arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers)
{
  auto event_start = std::get<0>(runtime_options.event_interval);
  auto event_end = std::get<1>(runtime_options.event_interval);
  arguments.set_size<dev_velo_raw_input>(std::get<1>(runtime_options.host_velo_events));
  arguments.set_size<dev_velo_raw_input_offsets>(std::get<2>(runtime_options.host_velo_events).size_bytes());
  arguments.set_size<dev_ut_raw_input>(std::get<1>(runtime_options.host_ut_events));
  arguments.set_size<dev_ut_raw_input_offsets>(std::get<2>(runtime_options.host_ut_events).size_bytes());
  arguments.set_size<dev_scifi_raw_input>(std::get<1>(runtime_options.host_scifi_events));
  arguments.set_size<dev_scifi_raw_input_offsets>(std::get<2>(runtime_options.host_scifi_events).size_bytes());
  arguments.set_size<dev_event_list>(event_end - event_start);
  arguments.set_size<dev_number_of_selected_events>(1);
}

template<>
void SequenceVisitor::visit<init_event_list_t>(
  init_event_list_t& state,
  const init_event_list_t::arguments_t& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  // Fetch required arguments for the global event cuts algorithm and
  // the various decoding algorithms
  // Velo
  data_to_device<dev_velo_raw_input, dev_velo_raw_input_offsets>
    (arguments, runtime_options.host_velo_events, hip_stream);
  // UT
  data_to_device<dev_ut_raw_input, dev_ut_raw_input_offsets>
    (arguments, runtime_options.host_ut_events, hip_stream);
  // SciFi
  data_to_device<dev_scifi_raw_input, dev_scifi_raw_input_offsets>
    (arguments, runtime_options.host_scifi_events, hip_stream);

  // Initialize buffers
  auto event_start = std::get<0>(runtime_options.event_interval);
  auto event_end = std::get<1>(runtime_options.event_interval);
  host_buffers.host_number_of_selected_events[0] = event_end - event_start;
  for (uint i = 0; i < event_end - event_start; ++i) {
    host_buffers.host_event_list[i] = event_start + i;
  }

  cudaCheck(hipMemcpyAsync(
    arguments.offset<dev_event_list>(),
    host_buffers.host_event_list,
    (event_end - event_start) * sizeof(uint),
    hipMemcpyHostToDevice,
    hip_stream));
}
