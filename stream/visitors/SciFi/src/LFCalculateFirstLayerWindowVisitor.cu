#include "LFCalculateFirstLayerWindow.cuh"
#include "SequenceVisitor.cuh"

template<>
void SequenceVisitor::set_arguments_size<lf_calculate_first_layer_window_t>(
  lf_calculate_first_layer_window_t::arguments_t arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers)
{
  const auto number_of_ut_tracks = host_buffers.host_atomics_ut[2 * host_buffers.host_number_of_selected_events[0]];
  arguments.set_size<dev_scifi_lf_first_layer_candidates>(2 * number_of_ut_tracks + 1);
  arguments.set_size<dev_ut_states>(number_of_ut_tracks);
}

template<>
void SequenceVisitor::visit<lf_calculate_first_layer_window_t>(
  lf_calculate_first_layer_window_t& state,
  const lf_calculate_first_layer_window_t::arguments_t& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  cudaCheck(hipMemsetAsync(
    arguments.offset<dev_scifi_lf_first_layer_candidates>(),
    0,
    arguments.size<dev_scifi_lf_first_layer_candidates>(),
    hip_stream
  ));

  // host_buffers.host_number_of_selected_events[0]
  state.set_opts(dim3(host_buffers.host_number_of_selected_events[0]), dim3(64), hip_stream);
  state.set_arguments(
    arguments.offset<dev_scifi_hits>(),
    arguments.offset<dev_scifi_hit_count>(),
    arguments.offset<dev_atomics_velo>(),
    arguments.offset<dev_velo_track_hit_number>(),
    arguments.offset<dev_velo_states>(),
    arguments.offset<dev_atomics_ut>(),
    arguments.offset<dev_ut_track_hits>(),
    arguments.offset<dev_ut_track_hit_number>(),
    arguments.offset<dev_ut_x>(),
    arguments.offset<dev_ut_tx>(),
    arguments.offset<dev_ut_z>(),
    arguments.offset<dev_ut_qop>(),
    arguments.offset<dev_ut_track_velo_indices>(),
    constants.dev_scifi_geometry,
    constants.dev_looking_forward_constants,
    constants.dev_inv_clus_res,
    arguments.offset<dev_scifi_lf_first_layer_candidates>(),
    arguments.offset<dev_ut_states>(),
    LookingForward::seeding_first_layer);

  state.invoke();

}
