#include "LFQualityFilterX.cuh"
#include "SequenceVisitor.cuh"

template<>
void SequenceVisitor::set_arguments_size<lf_quality_filter_x_t>(
  lf_quality_filter_x_t::arguments_t arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers)
{
  arguments.set_size<dev_scifi_lf_filtered_tracks>(host_buffers.host_number_of_selected_events[0] * SciFi::Constants::max_lf_tracks);
  arguments.set_size<dev_scifi_lf_filtered_atomics>(host_buffers.host_number_of_selected_events[0] * LookingForward::num_atomics * 2 + 1);
}

template<>
void SequenceVisitor::visit<lf_quality_filter_x_t>(
  lf_quality_filter_x_t& state,
  const lf_quality_filter_x_t::arguments_t& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  cudaCheck(hipMemsetAsync(
    arguments.offset<dev_scifi_lf_filtered_atomics>(),
    0,
    arguments.size<dev_scifi_lf_filtered_atomics>(),
    hip_stream));

  state.set_opts(dim3(host_buffers.host_number_of_selected_events[0]), dim3(256), hip_stream);
  state.set_arguments(
    arguments.offset<dev_scifi_lf_tracks>(),
    arguments.offset<dev_scifi_lf_atomics>(),
    arguments.offset<dev_scifi_lf_filtered_tracks>(),
    arguments.offset<dev_scifi_lf_filtered_atomics>());
  
  state.invoke();
}
