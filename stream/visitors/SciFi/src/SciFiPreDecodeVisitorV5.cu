#include "SequenceVisitor.cuh"
#include "SciFiPreDecodeV5.cuh"

template<>
void SequenceVisitor::set_arguments_size<scifi_pre_decode_v5_t>(
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers,
  argument_manager_t& arguments)
{
  arguments.set_size<dev_scifi_hits>(host_buffers.scifi_hits_uints());
}

template<>
void SequenceVisitor::visit<scifi_pre_decode_v5_t>(
  scifi_pre_decode_v5_t& state,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  argument_manager_t& arguments,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  state.set_opts(dim3(host_buffers.host_number_of_selected_events[0]), dim3(SciFi::SciFiRawBankParams::NbBanks), hip_stream);
  state.set_arguments(
    arguments.offset<dev_scifi_raw_input>(),
    arguments.offset<dev_scifi_raw_input_offsets>(),
    arguments.offset<dev_event_list>(),
    arguments.offset<dev_scifi_hit_count>(),
    arguments.offset<dev_scifi_hits>(),
    constants.dev_scifi_geometry,
    constants.dev_inv_clus_res
  );

  state.invoke();
}
