#include "PrefixSumHandler.cuh"
#include "SequenceVisitor.cuh"

template<>
void SequenceVisitor::set_arguments_size<lf_prefix_sum_first_layer_window_size_t>(
  lf_prefix_sum_first_layer_window_size_t::arguments_t arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers)
{
  const auto number_of_ut_tracks = host_buffers.host_number_of_reconstructed_ut_tracks[0];
  arguments.set_size<dev_prefix_sum_auxiliary_array_7>(
    lf_prefix_sum_first_layer_window_size_t::aux_array_size(number_of_ut_tracks));
}

template<>
void SequenceVisitor::visit<lf_prefix_sum_first_layer_window_size_t>(
  lf_prefix_sum_first_layer_window_size_t& state,
  const lf_prefix_sum_first_layer_window_size_t::arguments_t& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  const auto number_of_ut_tracks = host_buffers.host_number_of_reconstructed_ut_tracks[0];

  // Set size of the main array to be prefix summed
  state.set_size(number_of_ut_tracks);

  // Set the hip_stream
  state.set_opts(hip_stream);

  // Set arguments: Array to prefix sum and auxiliary array
  state.set_arguments(
    arguments.offset<dev_scifi_lf_first_layer_candidates>() + number_of_ut_tracks,
    arguments.offset<dev_prefix_sum_auxiliary_array_7>());

  // Invoke all steps of prefix sum
  state.invoke();

  // Fetch total number of hits accumulated with all windows
  cudaCheck(hipMemcpyAsync(
    host_buffers.host_lf_total_size_first_window_layer,
    arguments.offset<dev_scifi_lf_first_layer_candidates>() + 2 * number_of_ut_tracks,
    sizeof(int),
    hipMemcpyDeviceToHost,
    hip_stream));

  hipEventRecord(cuda_generic_event, hip_stream);
  hipEventSynchronize(cuda_generic_event);

  // std::vector<uint> lf_first_layer_candidates (arguments.size<dev_scifi_lf_first_layer_candidates>() / sizeof(uint));

  // cudaCheck(hipMemcpyAsync(lf_first_layer_candidates.data(),
  //   arguments.offset<dev_scifi_lf_first_layer_candidates>(),
  //   arguments.size<dev_scifi_lf_first_layer_candidates>(),
  //   hipMemcpyDeviceToHost,
  //   hip_stream));

  // hipEventRecord(cuda_generic_event, hip_stream);
  // hipEventSynchronize(cuda_generic_event);

  // for (uint i=0; i<number_of_ut_tracks; ++i) {
  //   info_cout << "UT track " << i << ", window: (" << lf_first_layer_candidates[i]
  //     << ", " << lf_first_layer_candidates[number_of_ut_tracks + i] << ", "
  //     << lf_first_layer_candidates[number_of_ut_tracks + i + 1] - lf_first_layer_candidates[number_of_ut_tracks + i] << ")" << std::endl;
  // }

  // info_cout << std::endl
  //   << "Total number of candidates in first window: " << host_buffers.host_lf_total_size_first_window_layer[0] << std::endl
  //   << std::endl;
}
