#include "LFCompositeTrackSeeding.cuh"
#include "SequenceVisitor.cuh"

template<>
void SequenceVisitor::set_arguments_size<lf_composite_track_seeding_t>(
  lf_composite_track_seeding_t::arguments_t arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers)
{
  arguments.set_size<dev_scifi_lf_tracks>(host_buffers.host_number_of_reconstructed_ut_tracks[0] * LookingForward::maximum_number_of_candidates_per_ut_track);
  arguments.set_size<dev_scifi_lf_atomics>(host_buffers.host_number_of_reconstructed_ut_tracks[0] * LookingForward::num_atomics * 2 + 1);
}

template<>
void SequenceVisitor::visit<lf_composite_track_seeding_t>(
  lf_composite_track_seeding_t& state,
  const lf_composite_track_seeding_t::arguments_t& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  state.handler_lf_triplet_seeding.set_arguments(
    arguments.offset<dev_scifi_hits>(),
    arguments.offset<dev_scifi_hit_count>(),
    arguments.offset<dev_atomics_ut>(),
    arguments.offset<dev_ut_qop>(),
    arguments.offset<dev_ut_states>(),
    constants.dev_scifi_geometry,
    constants.dev_inv_clus_res,
    arguments.offset<dev_scifi_lf_number_of_candidates>(),
    arguments.offset<dev_scifi_lf_candidates>(),
    constants.dev_looking_forward_constants,
    arguments.offset<dev_scifi_lf_tracks>(),
    arguments.offset<dev_scifi_lf_atomics>());

  state.handler_lf_extend_tracks_x.set_arguments(
    arguments.offset<dev_scifi_hits>(),
    arguments.offset<dev_scifi_hit_count>(),
    arguments.offset<dev_atomics_ut>(),
    arguments.offset<dev_scifi_lf_tracks>(),
    arguments.offset<dev_scifi_lf_atomics>(),
    constants.dev_scifi_geometry,
    constants.dev_looking_forward_constants,
    constants.dev_inv_clus_res,
    arguments.offset<dev_scifi_lf_number_of_candidates>(),
    arguments.offset<dev_scifi_lf_candidates>());

  state.handler_lf_extend_tracks_x.set_opts(dim3(host_buffers.host_number_of_selected_events[0]), dim3(16, 24), hip_stream);
  state.handler_lf_triplet_seeding.set_opts(dim3(host_buffers.host_number_of_selected_events[0], 32), dim3(32), hip_stream); // to do: check gridDim.y = 32

  cudaCheck(hipMemsetAsync(
    arguments.offset<dev_scifi_lf_atomics>(),
    0,
    arguments.size<dev_scifi_lf_atomics>(),
    hip_stream));

  state.handler_lf_triplet_seeding.invoke();

  // Extrapolate to all other layers
  state.handler_lf_extend_tracks_x.invoke();
}
