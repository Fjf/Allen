#include "LFCompositeTrackSeeding.cuh"
#include "SequenceVisitor.cuh"

template<>
void SequenceVisitor::set_arguments_size<lf_composite_track_seeding_t>(
  lf_composite_track_seeding_t::arguments_t arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers)
{
  arguments.set_size<dev_scifi_lf_triplet_best_chi2>(
                                                     host_buffers.host_number_of_reconstructed_ut_tracks[0] * 4 * LookingForward::maximum_number_of_candidates * LookingForward::maximum_number_of_triplets_per_h1);
  arguments.set_size<dev_scifi_lf_triplet_best_h0h2>(
                                                     host_buffers.host_number_of_reconstructed_ut_tracks[0] * 4 * 2 * LookingForward::maximum_number_of_candidates * LookingForward::maximum_number_of_triplets_per_h1);
  arguments.set_size<dev_scifi_lf_tracks>(
    host_buffers.host_number_of_reconstructed_ut_tracks[0] * LookingForward::maximum_number_of_candidates_per_ut_track);
  arguments.set_size<dev_scifi_lf_atomics>(
    host_buffers.host_number_of_reconstructed_ut_tracks[0] * LookingForward::num_atomics * 2 + 1);
}

template<>
void SequenceVisitor::visit<lf_composite_track_seeding_t>(
  lf_composite_track_seeding_t& state,
  const lf_composite_track_seeding_t::arguments_t& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  state.handler_lf_triplet_seeding.set_arguments(
    arguments.offset<dev_scifi_hits>(),
    arguments.offset<dev_scifi_hit_count>(),
    arguments.offset<dev_atomics_ut>(),
    arguments.offset<dev_ut_qop>(),
    arguments.offset<dev_ut_states>(),
    constants.dev_scifi_geometry,
    constants.dev_inv_clus_res,
    arguments.offset<dev_scifi_lf_number_of_candidates>(),
    arguments.offset<dev_scifi_lf_candidates>(),
    constants.dev_looking_forward_constants,
    arguments.offset<dev_scifi_lf_triplet_best_chi2>(),
    arguments.offset<dev_scifi_lf_triplet_best_h0h2>());

  state.handler_lf_triplet_keep_best.set_arguments(
    arguments.offset<dev_scifi_hits>(),
    arguments.offset<dev_scifi_hit_count>(),
    arguments.offset<dev_atomics_ut>(),
    arguments.offset<dev_ut_qop>(),
    arguments.offset<dev_ut_states>(),
    constants.dev_scifi_geometry,
    constants.dev_inv_clus_res,
    arguments.offset<dev_scifi_lf_candidates>(),
    constants.dev_looking_forward_constants,
    arguments.offset<dev_scifi_lf_tracks>(),
    arguments.offset<dev_scifi_lf_atomics>(),
    arguments.offset<dev_scifi_lf_triplet_best_chi2>(),
    arguments.offset<dev_scifi_lf_triplet_best_h0h2>());

  state.handler_lf_extend_tracks_x.set_arguments(
    arguments.offset<dev_scifi_hits>(),
    arguments.offset<dev_scifi_hit_count>(),
    arguments.offset<dev_atomics_ut>(),
    arguments.offset<dev_scifi_lf_tracks>(),
    arguments.offset<dev_scifi_lf_atomics>(),
    constants.dev_scifi_geometry,
    constants.dev_looking_forward_constants,
    constants.dev_inv_clus_res,
    arguments.offset<dev_scifi_lf_number_of_candidates>(),
    arguments.offset<dev_scifi_lf_candidates>());

  state.handler_lf_triplet_seeding.set_opts(
   dim3(host_buffers.host_number_of_selected_events[0]), dim3(LookingForward::n_threads_triplet_seeding), hip_stream);
  state.handler_lf_triplet_keep_best.set_opts(
    dim3(host_buffers.host_number_of_selected_events[0], 4), dim3(32), hip_stream);
  state.handler_lf_extend_tracks_x.set_opts(
    dim3(host_buffers.host_number_of_selected_events[0]), dim3(32, 4), hip_stream);

  cudaCheck(
    hipMemsetAsync(arguments.offset<dev_scifi_lf_atomics>(), 0, arguments.size<dev_scifi_lf_atomics>(), hip_stream));

  // Note: The initialization of dev_scifi_lf_triplet_best_chi2 is the highest positive
  //       number represented as fp32 that can be initialized using hipMemsetAsync,
  //       that is, initializing the bytes individually:
  //       0x7F results in 0x7F7F7F7F, which is 3.3961514e38 in fp32
  cudaCheck(hipMemsetAsync(
    arguments.offset<dev_scifi_lf_triplet_best_chi2>(),
    0x7F,
    arguments.size<dev_scifi_lf_triplet_best_chi2>(),
    hip_stream));

  cudaCheck(hipMemsetAsync(
    arguments.offset<dev_scifi_lf_triplet_best_h0h2>(),
    -1,
    arguments.size<dev_scifi_lf_triplet_best_h0h2>(),
    hip_stream));

  state.handler_lf_triplet_seeding.invoke();
  state.handler_lf_triplet_keep_best.invoke();

  // Extrapolate to all other layers
  state.handler_lf_extend_tracks_x.invoke();
}
