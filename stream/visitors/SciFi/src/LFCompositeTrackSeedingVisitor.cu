#include "LFCompositeTrackSeeding.cuh"
#include "SequenceVisitor.cuh"

DEFINE_EMPTY_SET_ARGUMENTS_SIZE(lf_composite_track_seeding_t)

template<>
void SequenceVisitor::visit<lf_composite_track_seeding_t>(
  lf_composite_track_seeding_t& state,
  const lf_composite_track_seeding_t::arguments_t& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  const auto seeding_set_arguments = [&state, &constants, &arguments] (const uint8_t relative_middle_layer) {
    state.handler_lf_triplet_seeding.set_arguments(
      arguments.offset<dev_scifi_hits>(),
      arguments.offset<dev_scifi_hit_count>(),
      arguments.offset<dev_atomics_ut>(),
      arguments.offset<dev_ut_qop>(),
      arguments.offset<dev_ut_states>(),
      constants.dev_scifi_geometry,
      constants.dev_inv_clus_res,
      arguments.offset<dev_scifi_lf_number_of_candidates>(),
      arguments.offset<dev_scifi_lf_candidates>(),
      constants.dev_looking_forward_constants,
      arguments.offset<dev_scifi_tracks>(),
      arguments.offset<dev_atomics_scifi>(),
      arguments.offset<dev_scifi_lf_candidate_atomics>(),
      arguments.offset<dev_scifi_lf_candidates_flag>(),
      relative_middle_layer);
  };

  const auto forwarding_set_arguments = [&state, &constants, &arguments] (const uint8_t relative_extrapolation_layer) {
    state.handler_lf_extend_tracks_x.set_arguments(
      arguments.offset<dev_scifi_hits>(),
      arguments.offset<dev_scifi_hit_count>(),
      arguments.offset<dev_atomics_ut>(),
      arguments.offset<dev_scifi_tracks>(),
      arguments.offset<dev_atomics_scifi>(),
      constants.dev_scifi_geometry,
      constants.dev_looking_forward_constants,
      constants.dev_inv_clus_res,
      arguments.offset<dev_scifi_lf_number_of_candidates>(),
      arguments.offset<dev_scifi_lf_candidates>(),
      arguments.offset<dev_scifi_lf_candidates_flag>(),
      relative_extrapolation_layer);
  };

  state.handler_lf_triplet_seeding.set_opts(dim3(host_buffers.host_number_of_selected_events[0], 32), dim3(64), hip_stream);
  state.handler_lf_extend_tracks_x.set_opts(dim3(host_buffers.host_number_of_selected_events[0]), dim3(128), hip_stream);

  // We need to:
  // * Forward to layer 4
  // * Seed mid layer 3
  // * Forward to layer 5
  // * Seed mid layer 4
  for (int i=0; i<1; ++i) {
    cudaCheck(hipMemsetAsync(
      arguments.offset<dev_scifi_lf_candidate_atomics>(),
      0,
      arguments.size<dev_scifi_lf_candidate_atomics>(),
      hip_stream));

    forwarding_set_arguments(4 + i);
    seeding_set_arguments(3 + i);

    // state.handler_lf_extend_tracks_x.invoke();
    // state.handler_lf_triplet_seeding.invoke();
  }

  // cudaCheck(hipMemcpyAsync(
  //   host_buffers.host_atomics_scifi,
  //   arguments.offset<dev_atomics_scifi>(),
  //   arguments.size<dev_atomics_scifi>(),
  //   hipMemcpyDeviceToHost,
  //   hip_stream));

  // cudaCheck(hipMemcpyAsync(
  //   host_buffers.host_scifi_tracks,
  //   arguments.offset<dev_scifi_tracks>(),
  //   arguments.size<dev_scifi_tracks>(),
  //   hipMemcpyDeviceToHost,
  //   hip_stream));

  // hipEventRecord(cuda_generic_event, hip_stream);
  // hipEventSynchronize(cuda_generic_event);

  // for (uint i=0; i<host_buffers.host_number_of_selected_events[0]; ++i) {
  //   const auto number_of_tracks = scifi_atomics[i];
  //   int number_of_quadruplets = 0;

  //   for (int j=0; j<number_of_tracks; ++j) {
  //     const auto track = scifi_tracks[i * SciFi::Constants::max_tracks + j];
  //     if (track.hitsNum == 4) {
  //       ++number_of_quadruplets;
  //     }
  //   }

  //   info_cout << "Event " << i << ", number of quadruplet tracks " << number_of_quadruplets << std::endl;

  //   for (int j=0; j<number_of_tracks; ++j) {
  //     const auto track = scifi_tracks[i * SciFi::Constants::max_tracks + j];
  //     if (track.hitsNum >= 4) {
  //       info_cout << "Track ";
  //       for (int k=0; k<track.hitsNum; ++k) {
  //         info_cout << track.hits[k] << ", ";
  //       }
  //       info_cout << track.get_quality() << std::endl;
  //     }
  //   }
  //   info_cout << std::endl;

  //   info_cout << "Event " << i << ", number of tracks " << number_of_tracks << std::endl;

  //   for (int j=0; j<number_of_tracks; ++j) {
  //     const auto track = scifi_tracks[i * SciFi::Constants::max_tracks + j];
  //     info_cout << " Track #" << j << " from ut track " << track.ut_track_index
  //       << ", quality " << track.get_quality()
  //       << ", hits: ";

  //     for (int k=0; k<track.hitsNum; ++k) {
  //       info_cout << track.hits[k] << ", ";
  //     }
  //     info_cout << std::endl;
  //   }
  // }
}
