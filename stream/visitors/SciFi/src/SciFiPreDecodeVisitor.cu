#include "SequenceVisitor.cuh"
#include "SciFiPreDecode.cuh"

template<>
void SequenceVisitor::set_arguments_size<scifi_pre_decode_t>(
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers,
  argument_manager_t& arguments)
{
  arguments.set_size<dev_scifi_hits>(host_buffers.scifi_hits_uints());
}

template<>
void SequenceVisitor::visit<scifi_pre_decode_t>(
  scifi_pre_decode_t& state,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  argument_manager_t& arguments,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  state.set_opts(dim3(runtime_options.number_of_events), dim3(SciFi::SciFiRawBankParams::NbBanks), hip_stream);
  state.set_arguments(
    arguments.offset<dev_scifi_raw_input>(),
    arguments.offset<dev_scifi_raw_input_offsets>(),
    arguments.offset<dev_scifi_hit_count>(),
    arguments.offset<dev_scifi_hits>(),
    constants.dev_scifi_geometry,
    constants.dev_inv_clus_res
  );

  state.invoke();
}
