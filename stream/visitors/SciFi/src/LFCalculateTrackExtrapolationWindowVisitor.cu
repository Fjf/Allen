#include "LFCalculateTrackExtrapolationWindow.cuh"
#include "SequenceVisitor.cuh"

DEFINE_EMPTY_SET_ARGUMENTS_SIZE(lf_calculate_track_extrapolation_window_t)

template<>
void SequenceVisitor::visit<lf_calculate_track_extrapolation_window_t>(
  lf_calculate_track_extrapolation_window_t& state,
  const lf_calculate_track_extrapolation_window_t::arguments_t& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  // TODO: Maybe we don't need this, but for now, keeping it
  cudaCheck(
    hipMemsetAsync(
      arguments.offset<dev_extrapolation_layer_candidates>(),
      0,
      arguments.size<dev_extrapolation_layer_candidates>(),
      hip_stream));

  state.set_opts(dim3(host_buffers.host_number_of_selected_events[0]), dim3(256), hip_stream);
  state.set_arguments(
    arguments.offset<dev_scifi_hits>(),
    arguments.offset<dev_scifi_hit_count>(),
    arguments.offset<dev_atomics_ut>(),
    arguments.offset<dev_scifi_tracks>(),
    arguments.offset<dev_atomics_scifi>(),
    constants.dev_scifi_geometry,
    constants.dev_looking_forward_constants,
    constants.dev_inv_clus_res,
    arguments.offset<dev_ut_states>(),
    6,
    arguments.offset<dev_extrapolation_layer_candidates>());

  state.invoke();

  // std::vector<unsigned short> extrapolation_layer_candidates (arguments.size<dev_extrapolation_layer_candidates>() / sizeof(unsigned short));
  // std::vector<int> scifi_atomics(arguments.size<dev_atomics_scifi>() / sizeof(int));

  // cudaCheck(hipMemcpyAsync(
  //   scifi_atomics.data(),
  //   arguments.offset<dev_atomics_scifi>(),
  //   arguments.size<dev_atomics_scifi>(),
  //   hipMemcpyDeviceToHost,
  //   hip_stream));

  // cudaCheck(hipMemcpyAsync(extrapolation_layer_candidates.data(),
  //   arguments.offset<dev_extrapolation_layer_candidates>(),
  //   arguments.size<dev_extrapolation_layer_candidates>(),
  //   hipMemcpyDeviceToHost,
  //   hip_stream));

  // hipEventRecord(cuda_generic_event, hip_stream);
  // hipEventSynchronize(cuda_generic_event);

  // for (uint i=0; i<host_buffers.host_number_of_selected_events[0]; ++i) {
  //   const auto number_of_candidates = scifi_atomics[host_buffers.host_number_of_selected_events[0] + i];
  //   info_cout << "Event " << i << ":" << std::endl;

  //   for (int j=0; j<number_of_candidates; ++j) {
  //     info_cout << " " << extrapolation_layer_candidates[i * SciFi::Constants::max_tracks + j]
  //       << ", " << extrapolation_layer_candidates[
  //           host_buffers.host_number_of_selected_events[0] * SciFi::Constants::max_tracks + 
  //           i * SciFi::Constants::max_tracks + j] << std::endl;
  //   }
  // }
}
