#include "LFTripletSeeding.cuh"
#include "SequenceVisitor.cuh"

template<>
void SequenceVisitor::set_arguments_size<lf_triplet_seeding_t>(
  lf_triplet_seeding_t::arguments_t arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers)
{
  arguments.set_size<dev_scifi_tracks>(host_buffers.host_number_of_selected_events[0] * SciFi::Constants::max_tracks);
  arguments.set_size<dev_atomics_scifi>(host_buffers.host_number_of_selected_events[0] * SciFi::num_atomics);
  arguments.set_size<dev_scifi_lf_candidates_flag>(host_buffers.host_lf_total_number_of_candidates[0]);
}

template<>
void SequenceVisitor::visit<lf_triplet_seeding_t>(
  lf_triplet_seeding_t& state,
  const lf_triplet_seeding_t::arguments_t& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  cudaCheck(hipMemsetAsync(
    arguments.offset<dev_atomics_scifi>(),
    0,
    arguments.size<dev_atomics_scifi>(),
    hip_stream));

  // No wmma section:
  // 1, 64: 19.53%
  // 1, 256: 23.23%
  // 4, 32: 19.69%
  // 32, 64: 13.04% (sbt: 11.24%)

  // With wmma section (on normal cuda):
  // 1, 64: 21.94%
  // 4, 32: 21.67%
  // 8, 64: 15.23%
  // 16, 64: 14.58%
  // 64, 64: 14.72% (sbt: 11.12%)
  // 128, 64: 15.30% (sbt: 10.73%)
  // 32, 32: 18.80% (sbt: 10.49%)
  // 32, 64: 14.44% (sbt: 11.18%)

  state.set_opts(dim3(host_buffers.host_number_of_selected_events[0], 32), dim3(64), hip_stream);
  state.set_arguments(
    arguments.offset<dev_scifi_hits>(),
    arguments.offset<dev_scifi_hit_count>(),
    arguments.offset<dev_atomics_ut>(),
    arguments.offset<dev_ut_qop>(),
    arguments.offset<dev_ut_states>(),
    constants.dev_scifi_geometry,
    constants.dev_inv_clus_res,
    arguments.offset<dev_scifi_lf_number_of_candidates>(),
    arguments.offset<dev_scifi_lf_candidates>(),
    constants.dev_looking_forward_constants,
    arguments.offset<dev_scifi_tracks>(),
    arguments.offset<dev_atomics_scifi>(),
    arguments.offset<dev_scifi_lf_candidates_flag>(),
    1);

  state.invoke();

  // std::vector<int> scifi_atomics(arguments.size<dev_atomics_scifi>() / sizeof(dev_atomics_scifi::type));
  // std::vector<SciFi::TrackHits> scifi_tracks(arguments.size<dev_scifi_tracks>() / sizeof(dev_scifi_tracks::type));

  // cudaCheck(hipMemcpyAsync(
  //   scifi_atomics.data(),
  //   arguments.offset<dev_atomics_scifi>(),
  //   arguments.size<dev_atomics_scifi>(),
  //   hipMemcpyDeviceToHost,
  //   hip_stream));

  // cudaCheck(hipMemcpyAsync(
  //   scifi_tracks.data(),
  //   arguments.offset<dev_scifi_tracks>(),
  //   arguments.size<dev_scifi_tracks>(),
  //   hipMemcpyDeviceToHost,
  //   hip_stream));

  // hipEventRecord(cuda_generic_event, hip_stream);
  // hipEventSynchronize(cuda_generic_event);

  // for (uint i=0; i<host_buffers.host_number_of_selected_events[0]; ++i) {
  //   info_cout << "Event " << i
  //     << ", number of track candidates " << scifi_atomics[i] << std::endl;
  // }

  // for (uint i=0; i<host_buffers.host_number_of_selected_events[0]; ++i) {
  //   for (int j=0; j<scifi_atomics[i]; ++j) {
  //     const auto track = scifi_tracks[i * SciFi::Constants::max_tracks + j];

  //     info_cout << "Track " << track.hits[0] << ", "
  //       << track.hits[1] << ", "
  //       << track.hits[2] << ", "
  //       << track.quality
  //       << std::endl;
  //   }
  // }
}
