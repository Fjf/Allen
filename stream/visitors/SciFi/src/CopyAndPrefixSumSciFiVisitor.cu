#include "SequenceVisitor.cuh"
#include "PrefixSum.cuh"

DEFINE_EMPTY_SET_ARGUMENTS_SIZE(copy_and_prefix_sum_single_block_scifi_t)

template<>
void SequenceVisitor::visit<copy_and_prefix_sum_single_block_scifi_t>(
  copy_and_prefix_sum_single_block_scifi_t& state,
  const copy_and_prefix_sum_single_block_scifi_t::arguments_t& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  state.set_opts(dim3(1), dim3(1024), hip_stream);
  state.set_arguments(
    (uint*) arguments.offset<dev_atomics_scifi>() + host_buffers.host_number_of_selected_events[0] * 2,
    (uint*) arguments.offset<dev_atomics_scifi>(),
    (uint*) arguments.offset<dev_atomics_scifi>() + host_buffers.host_number_of_selected_events[0],
    host_buffers.host_number_of_selected_events[0]);

  state.invoke();

  // Fetch number of reconstructed SciFi tracks.
  cudaCheck(hipMemcpyAsync(
    host_buffers.host_number_of_reconstructed_scifi_tracks,
    arguments.offset<dev_atomics_scifi>() + host_buffers.host_number_of_selected_events[0] * 2,
    sizeof(uint),
    hipMemcpyDeviceToHost,
    hip_stream));

  hipEventRecord(cuda_generic_event, hip_stream);
  hipEventSynchronize(cuda_generic_event);

  if (logger::ll.verbosityLevel >= logger::debug) {
    debug_cout << "Total # of SciFi tracks = " << host_buffers.host_number_of_reconstructed_scifi_tracks[0] << std::endl;
  }
}
