#include "LFQualityFilterLength.cuh"
#include "SequenceVisitor.cuh"

template<>
void SequenceVisitor::set_arguments_size<lf_quality_filter_length_t>(
  lf_quality_filter_length_t::arguments_t arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers)
{
  arguments.set_size<dev_atomics_scifi>(host_buffers.host_number_of_selected_events[0] * LookingForward::num_atomics * 2 + 1);
  arguments.set_size<dev_scifi_tracks>(host_buffers.host_number_of_selected_events[0] * SciFi::Constants::max_tracks);
  arguments.set_size<dev_scifi_lf_length_filtered_tracks>(host_buffers.host_number_of_reconstructed_ut_tracks[0] * LookingForward::maximum_number_of_candidates_per_ut_track_after_x_filter);
  arguments.set_size<dev_scifi_lf_length_filtered_atomics>(host_buffers.host_number_of_selected_events[0] * LookingForward::num_atomics * 2 + 1); 
}

template<>
void SequenceVisitor::visit<lf_quality_filter_length_t>(
  lf_quality_filter_length_t& state,
  const lf_quality_filter_length_t::arguments_t& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  cudaCheck(hipMemsetAsync(
    arguments.offset<dev_atomics_scifi>(),
    0,
    arguments.size<dev_atomics_scifi>(),
    hip_stream));

  cudaCheck(hipMemsetAsync(
    arguments.offset<dev_scifi_lf_length_filtered_atomics>(),
    0,
    arguments.size<dev_scifi_lf_length_filtered_atomics>(),
    hip_stream));

  // // Code for running this algorithm last, in the SciFi sequence
  // state.set_opts(dim3(host_buffers.host_number_of_selected_events[0]), dim3(256), hip_stream);
  // state.set_arguments(
  //   arguments.offset<dev_atomics_ut>(),
  //   arguments.offset<dev_scifi_lf_x_filtered_tracks>(),
  //   arguments.offset<dev_scifi_lf_x_filtered_atomics>(),
  //   arguments.offset<dev_scifi_tracks>(),
  //   arguments.offset<dev_atomics_scifi>());
  // state.invoke();

  // Code for running the quality filter after this algorithm
  state.set_opts(dim3(host_buffers.host_number_of_selected_events[0]), dim3(256), hip_stream);
  state.set_arguments(
    arguments.offset<dev_atomics_ut>(),
    arguments.offset<dev_scifi_lf_x_filtered_tracks>(),
    arguments.offset<dev_scifi_lf_x_filtered_atomics>(),
    arguments.offset<dev_scifi_lf_length_filtered_tracks>(),
    arguments.offset<dev_scifi_lf_length_filtered_atomics>());
  state.invoke();

  // for (uint i=0; i<host_buffers.host_number_of_selected_events[0]; ++i) {
  //   const auto number_of_tracks = host_buffers.host_atomics_scifi[i];
  //   info_cout << "Event " << i << ", number of tracks " << number_of_tracks << std::endl;

  //   for (int j=0; j<number_of_tracks; ++j) {
  //     const auto track = host_buffers.host_scifi_tracks[i * SciFi::Constants::max_tracks + j];
  //     info_cout << "Track #" << j << ", " << ((int) track.hitsNum) << " hits: ";
  //     for (int k=0; k<track.hitsNum; ++k) {
  //       info_cout << track.hits[k] << ", ";
  //     }
  //     info_cout << " chi2: " << track.get_quality() << std::endl;
  //   }
  //   info_cout << std::endl;
  // }
}
