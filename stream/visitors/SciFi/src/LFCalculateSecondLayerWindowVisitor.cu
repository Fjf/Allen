#include "LFCalculateSecondLayerWindow.cuh"
#include "SequenceVisitor.cuh"

template<>
void SequenceVisitor::set_arguments_size<lf_calculate_second_layer_window_t>(
  lf_calculate_second_layer_window_t::arguments_t arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers)
{
  arguments.set_size<dev_scifi_lf_second_layer_candidates>(8 * host_buffers.host_lf_total_size_first_window_layer[0] + 1);
}

template<>
void SequenceVisitor::visit<lf_calculate_second_layer_window_t>(
  lf_calculate_second_layer_window_t& state,
  const lf_calculate_second_layer_window_t::arguments_t& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  cudaCheck(hipMemsetAsync(
    arguments.offset<dev_scifi_lf_second_layer_candidates>(),
    0,
    arguments.size<dev_scifi_lf_second_layer_candidates>(),
    hip_stream
  ));

  // 1, 32: 19.06%
  // 2, 16: 15.58%
  // 4, 16: 15.83%
  // 8, 16: 16.79%
  state.set_opts(dim3(host_buffers.host_number_of_selected_events[0]), dim3(2, 16), hip_stream);
  state.set_arguments(
    arguments.offset<dev_scifi_hits>(),
    arguments.offset<dev_scifi_hit_count>(),
    arguments.offset<dev_atomics_velo>(),
    arguments.offset<dev_velo_track_hit_number>(),
    arguments.offset<dev_velo_states>(),
    arguments.offset<dev_atomics_ut>(),
    arguments.offset<dev_ut_track_hits>(),
    arguments.offset<dev_ut_track_hit_number>(),
    arguments.offset<dev_ut_x>(),
    arguments.offset<dev_ut_tx>(),
    arguments.offset<dev_ut_z>(),
    arguments.offset<dev_ut_qop>(),
    arguments.offset<dev_ut_track_velo_indices>(),
    constants.dev_scifi_geometry,
    constants.dev_looking_forward_constants,
    constants.dev_inv_clus_res,
    arguments.offset<dev_scifi_lf_first_layer_candidates>(),
    arguments.offset<dev_scifi_lf_second_layer_candidates>(),
    LookingForward::seeding_first_layer,
    LookingForward::seeding_second_layer);

  state.invoke();

  // std::vector<unsigned short> lf_second_layer_candidates (arguments.size<dev_scifi_lf_second_layer_candidates>() / sizeof(unsigned short));

  // cudaCheck(hipMemcpyAsync(lf_second_layer_candidates.data(),
  //   arguments.offset<dev_scifi_lf_second_layer_candidates>(),
  //   arguments.size<dev_scifi_lf_second_layer_candidates>(),
  //   hipMemcpyDeviceToHost,
  //   hip_stream));

  // hipEventRecord(cuda_generic_event, hip_stream);
  // hipEventSynchronize(cuda_generic_event);

  // int total_number_of_candidates = 0;

  // info_cout << std::endl << "Candidates second layer window:" << std::endl;
  // for (size_t i=0; i<host_buffers.host_lf_total_size_first_window_layer[0]; ++i) {
  //   info_cout << "Candidate " << i << ", window: (" << lf_second_layer_candidates[i]
  //     << ", " << lf_second_layer_candidates[host_buffers.host_lf_total_size_first_window_layer[0] + i]
  //     << ", " << lf_second_layer_candidates[2 * host_buffers.host_lf_total_size_first_window_layer[0] + i]
  //     << ", " << lf_second_layer_candidates[3 * host_buffers.host_lf_total_size_first_window_layer[0] + i]
  //     << ", " << lf_second_layer_candidates[4 * host_buffers.host_lf_total_size_first_window_layer[0] + i]
  //     << ", " << lf_second_layer_candidates[5 * host_buffers.host_lf_total_size_first_window_layer[0] + i]
  //     << ", " << lf_second_layer_candidates[6 * host_buffers.host_lf_total_size_first_window_layer[0] + i]
  //     << ", " << lf_second_layer_candidates[7 * host_buffers.host_lf_total_size_first_window_layer[0] + i]
  //     << ")" << std::endl;

  //   total_number_of_candidates += lf_second_layer_candidates[3 * host_buffers.host_lf_total_size_first_window_layer[0] + i];
  // }
  // info_cout << std::endl;

  // info_cout << "Average number of candidates: " << total_number_of_candidates / ((float) host_buffers.host_lf_total_size_first_window_layer[0]) << std::endl;
}
