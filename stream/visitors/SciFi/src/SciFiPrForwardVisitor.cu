#include "PrForward.cuh"
#include "SequenceVisitor.cuh"

template<>
void SequenceVisitor::set_arguments_size<scifi_pr_forward_t>(
  scifi_pr_forward_t::arguments_t arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers)
{
  arguments.set_size<dev_scifi_tracks>(host_buffers.host_number_of_selected_events[0] * SciFi::Constants::max_tracks);
  arguments.set_size<dev_atomics_scifi>(host_buffers.host_number_of_selected_events[0] * SciFi::num_atomics);
}

template<>
void SequenceVisitor::visit<scifi_pr_forward_t>(
  scifi_pr_forward_t& state,
  const scifi_pr_forward_t::arguments_t& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  state.set_opts(dim3(host_buffers.host_number_of_selected_events[0]), dim3(32), hip_stream);
  state.set_arguments(
    arguments.offset<dev_scifi_hits>(),
    arguments.offset<dev_scifi_hit_count>(),
    arguments.offset<dev_atomics_velo>(),
    arguments.offset<dev_velo_track_hit_number>(),
    arguments.offset<dev_velo_states>(),
    arguments.offset<dev_atomics_ut>(),
    arguments.offset<dev_ut_track_hits>(),
    arguments.offset<dev_ut_track_hit_number>(),
    arguments.offset<dev_ut_qop>(),
    arguments.offset<dev_ut_track_velo_indices>(),
    arguments.offset<dev_scifi_tracks>(),
    arguments.offset<dev_atomics_scifi>(),
    constants.dev_scifi_tmva1,
    constants.dev_scifi_tmva2,
    constants.dev_scifi_constArrays,
    constants.dev_magnet_polarity,
    constants.dev_scifi_geometry,
    constants.dev_inv_clus_res);

  state.invoke();

  // cudaCheck(hipMemcpyAsync(host_buffers.host_atomics_scifi,
  //   arguments.offset<dev_atomics_scifi>(),
  //   arguments.size<dev_atomics_scifi>(),
  //   hipMemcpyDeviceToHost,
  //   hip_stream));

  // cudaCheck(hipMemcpyAsync(host_buffers.host_scifi_tracks,
  //   arguments.offset<dev_scifi_tracks>(),
  //   arguments.size<dev_scifi_tracks>(),
  //   hipMemcpyDeviceToHost,
  //   hip_stream));

  // hipEventRecord(cuda_generic_event, hip_stream);
  // hipEventSynchronize(cuda_generic_event);

  // for (uint i=0; i<host_buffers.host_number_of_selected_events[0]; ++i) {
  //   info_cout << "Event " << i
  //     << ", number of tracks " << host_buffers.host_atomics_scifi[i] << std::endl;
  // }
}
