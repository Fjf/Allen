#include "SciFiDirectDecoderV4.cuh"
#include "SequenceVisitor.cuh"

DEFINE_EMPTY_SET_ARGUMENTS_SIZE(scifi_direct_decoder_v4_t)

template<>
void SequenceVisitor::visit<scifi_direct_decoder_v4_t>(
  scifi_direct_decoder_v4_t& state,
  const scifi_direct_decoder_v4_t::arguments_t& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  state.set_opts(dim3(host_buffers.host_number_of_selected_events[0]), hip_stream);
  state.set_arguments(
    arguments.offset<dev_scifi_raw_input>(),
    arguments.offset<dev_scifi_raw_input_offsets>(),
    arguments.offset<dev_scifi_hit_count>(),
    arguments.offset<dev_scifi_hits>(),
    arguments.offset<dev_event_list>(),
    constants.dev_scifi_geometry,
    constants.dev_inv_clus_res);

  state.invoke();

  // SciFi Decoder Debugging
  /*const uint hit_count_uints = 2 * host_buffers.host_number_of_selected_events[0] *
  SciFi::Constants::n_mat_groups_and_mats + 1; uint host_scifi_hit_count[hit_count_uints]; uint* host_scifi_hits = new
  uint[host_buffers.scifi_hits_uints()]; cudaCheck(hipMemcpyAsync(&host_scifi_hit_count,
  arguments.offset<dev_scifi_hit_count>(), hit_count_uints*sizeof(uint), hipMemcpyDeviceToHost, hip_stream));
  cudaCheck(hipMemcpyAsync(host_scifi_hits, arguments.offset<dev_scifi_hits>(), arguments.size<dev_scifi_hits>(),
  hipMemcpyDeviceToHost, hip_stream)); hipEventRecord(cuda_generic_event, hip_stream);
  hipEventSynchronize(cuda_generic_event);
  SciFi::SciFiGeometry host_geom(constants.host_scifi_geometry);
  SciFi::Hits hi(host_scifi_hits, host_scifi_hit_count[host_buffers.host_number_of_selected_events[0] *
  SciFi::Constants::n_mat_groups_and_mats], &host_geom, constants.host_inv_clus_res.data()); std::ofstream
  outfile("dump.v4.txt");

  for(size_t event = 0; event < host_buffers.host_number_of_selected_events[0]; event++) {
    SciFi::HitCount host_scifi_hit_count_struct(host_scifi_hit_count, event);
    //outfile << host_scifi_hit_count_struct.event_number_of_hits() << std::endl;
    for(size_t zone = 0; zone < SciFi::Constants::n_zones; zone++) {
      for(size_t hit = 0; hit < host_scifi_hit_count_struct.zone_number_of_hits(zone); hit++) {
        uint h = host_scifi_hit_count_struct.zone_offset(zone) + hit;
        outfile << std::setprecision(5) << std::fixed
          << zone / 2 << " "
          << zone % 2     << " "
          << hi.LHCbID(h) << " "
          << hi.x0[h]   << " "
          << hi.z0[h]   << " "
          << hi.w(h)    << " "
          << hi.dxdy(h) << " "
          << hi.dzdy(h) << " "
          << hi.yMin(h) << " "
          << hi.yMax(h) << std::endl;
      }
    }
  }*/
}
