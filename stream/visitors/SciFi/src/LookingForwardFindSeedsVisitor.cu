#include "LookingForwardFindSeeds.cuh"
#include "SequenceVisitor.cuh"

template<>
void SequenceVisitor::set_arguments_size<looking_forward_find_seeds_t>(
  looking_forward_find_seeds_t::arguments_t arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers)
{
  arguments.set_size<dev_scifi_track_candidates>(host_buffers.host_number_of_selected_events[0] * SciFi::Constants::max_track_candidates);
  arguments.set_size<dev_atomics_scifi>(host_buffers.host_number_of_selected_events[0] * SciFi::num_atomics);
}

template<>
void SequenceVisitor::visit<looking_forward_find_seeds_t>(
  looking_forward_find_seeds_t& state,
  const looking_forward_find_seeds_t::arguments_t& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  cudaCheck(
    hipMemsetAsync(arguments.offset<dev_atomics_scifi>(), 0, arguments.size<dev_atomics_scifi>(), hip_stream));

  // host_buffers.host_number_of_selected_events[0]
  state.set_opts(dim3(host_buffers.host_number_of_selected_events[0]), dim3(32), hip_stream);
  state.set_arguments(
    arguments.offset<dev_scifi_hits>(),
    arguments.offset<dev_scifi_hit_count>(),
    arguments.offset<dev_atomics_velo>(),
    arguments.offset<dev_velo_track_hit_number>(),
    arguments.offset<dev_velo_states>(),
    arguments.offset<dev_atomics_ut>(),
    arguments.offset<dev_ut_track_hits>(),
    arguments.offset<dev_ut_track_hit_number>(),
    arguments.offset<dev_ut_x>(),
    arguments.offset<dev_ut_tx>(),
    arguments.offset<dev_ut_z>(),
    arguments.offset<dev_ut_qop>(),
    arguments.offset<dev_ut_track_velo_indices>(),
    arguments.offset<dev_scifi_track_candidates>(),
    arguments.offset<dev_atomics_scifi>(),
    constants.dev_scifi_geometry,
    constants.dev_looking_forward_constants,
    constants.dev_inv_clus_res,
    LookingForward::seeding_station);

  state.invoke();

  // std::vector<SciFi::TrackCandidate> scifi_track_candidates (arguments.size<dev_scifi_track_candidates>() / sizeof(SciFi::TrackCandidate));

  // cudaCheck(hipMemcpyAsync(host_buffers.host_atomics_scifi,
  //   arguments.offset<dev_atomics_scifi>(),
  //   arguments.size<dev_atomics_scifi>(),
  //   hipMemcpyDeviceToHost,
  //   hip_stream));

  // cudaCheck(hipMemcpyAsync(scifi_track_candidates.data(),
  //   arguments.offset<dev_scifi_track_candidates>(),
  //   arguments.size<dev_scifi_track_candidates>(),
  //   hipMemcpyDeviceToHost,
  //   hip_stream));

  // hipEventRecord(cuda_generic_event, hip_stream);
  // hipEventSynchronize(cuda_generic_event);

  // for (uint i=0; i<host_buffers.host_number_of_selected_events[0]; ++i) {
  //   info_cout << "Event " << i
  //     << ", number of tracks " << host_buffers.host_atomics_scifi[i] << std::endl;
  // }
}
