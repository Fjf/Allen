#include "SequenceVisitor.cuh"
#include "SciFiRawBankDecoderV4.cuh"

template<>
void SequenceVisitor::set_arguments_size<scifi_raw_bank_decoder_v4_t>(
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers,
  argument_manager_t& arguments)
{
  arguments.set_size<dev_scifi_hits>(host_buffers.scifi_hits_uints());
}


template<>
void SequenceVisitor::visit<scifi_raw_bank_decoder_v4_t>(
  scifi_raw_bank_decoder_v4_t& state,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  argument_manager_t& arguments,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  state.set_opts(dim3(runtime_options.number_of_events), dim3(256), hip_stream);
  state.set_arguments(
    arguments.offset<dev_scifi_raw_input>(),
    arguments.offset<dev_scifi_raw_input_offsets>(),
    arguments.offset<dev_scifi_hit_count>(),
    arguments.offset<dev_scifi_hits>(),
    constants.dev_scifi_geometry,
    constants.dev_inv_clus_res
  );

  state.invoke();

  // // SciFi Decoder Debugging
  // const uint hit_count_uints = 2 * runtime_options.number_of_events * SciFi::Constants::n_mats + 1;
  // uint host_scifi_hit_count[hit_count_uints];
  // uint* host_scifi_hits = new uint[host_buffers.scifi_hits_uints()];
  // cudaCheck(hipMemcpyAsync(&host_scifi_hit_count, arguments.offset<dev_scifi_hit_count>(), hit_count_uints*sizeof(uint), hipMemcpyDeviceToHost, hip_stream));
  // cudaCheck(hipMemcpyAsync(host_scifi_hits, arguments.offset<dev_scifi_hits>(), arguments.size<dev_scifi_hits>(), hipMemcpyDeviceToHost, hip_stream));
  // hipEventRecord(cuda_generic_event, hip_stream);
  // hipEventSynchronize(cuda_generic_event);
  //
  // SciFi::SciFiGeometry host_geom(constants.host_scifi_geometry);
  // SciFi::SciFiHits hi(host_scifi_hits, host_scifi_hit_count[runtime_options.number_of_events * SciFi::Constants::n_mats], &host_geom, constants.host_inv_clus_res.data());
  //
  // std::ofstream outfile("dump.txt");
  // SciFi::SciFiHitCount host_scifi_hit_count_struct;
  // for(size_t event = 0; event < runtime_options.number_of_events; event++) {
  //   host_scifi_hit_count_struct.typecast_after_prefix_sum(host_scifi_hit_count, event, runtime_options.number_of_events);
  //   for(size_t zone = 0; zone < SciFi::Constants::n_zones; zone++) {
  //     for(size_t hit = 0; hit < host_scifi_hit_count_struct.zone_number_of_hits(zone); hit++) {
  //       uint h = host_scifi_hit_count_struct.zone_offset(zone) + hit;
  //       outfile << std::setprecision(8) << std::fixed
  //         << hi.planeCode(h) << " "
  //         << zone % 2     << " "
  //         << hi.LHCbID(h) << " "
  //         << hi.x0[h]   << " "
  //         << hi.z0[h]   << " "
  //         << hi.w(h)    << " "
  //         << hi.dxdy(h) << " "
  //         << hi.dzdy(h) << " "
  //         << hi.yMin(h) << " "
  //         << hi.yMax(h) << std::endl;
  //     }
  //   }
  // }
}
