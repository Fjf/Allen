#include "SequenceVisitor.cuh"
#include "SciFiCalculateClusterCountV4.cuh"

template<>
void SequenceVisitor::set_arguments_size<scifi_calculate_cluster_count_v4_t>(
  scifi_calculate_cluster_count_v4_t::arguments_t arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers)
{
  arguments.set_size<dev_scifi_raw_input>(runtime_options.host_scifi_events_size);
  arguments.set_size<dev_scifi_raw_input_offsets>(runtime_options.host_scifi_event_offsets_size);
  arguments.set_size<dev_scifi_hit_count>(host_buffers.host_number_of_selected_events[0] * SciFi::Constants::n_mat_groups_and_mats + 1);
}

template<>
void SequenceVisitor::visit<scifi_calculate_cluster_count_v4_t>(
  scifi_calculate_cluster_count_v4_t& state,
  const scifi_calculate_cluster_count_v4_t::arguments_t& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  cudaCheck(hipMemcpyAsync(arguments.offset<dev_scifi_raw_input>(),
    runtime_options.host_scifi_events,
    runtime_options.host_scifi_events_size,
    hipMemcpyHostToDevice,
    hip_stream));

  cudaCheck(hipMemcpyAsync(arguments.offset<dev_scifi_raw_input_offsets>(),
    runtime_options.host_scifi_event_offsets,
    runtime_options.host_scifi_event_offsets_size * sizeof(uint),
    hipMemcpyHostToDevice,
    hip_stream));

  cudaCheck(hipMemsetAsync(arguments.offset<dev_scifi_hit_count>(),
    0,
    arguments.size<dev_scifi_hit_count>(),
    hip_stream));

  hipEventRecord(cuda_generic_event, hip_stream);
  hipEventSynchronize(cuda_generic_event);

  state.set_opts(dim3(host_buffers.host_number_of_selected_events[0]), dim3(240), hip_stream);
  state.set_arguments(
    arguments.offset<dev_scifi_raw_input>(),
    arguments.offset<dev_scifi_raw_input_offsets>(),
    arguments.offset<dev_scifi_hit_count>(),
    arguments.offset<dev_event_list>(),
    constants.dev_scifi_geometry
  );

  state.invoke();
}
