#include "SequenceVisitor.cuh"
#include "SciFiPreDecodeV6.cuh"

template<>
void SequenceVisitor::set_arguments_size<scifi_pre_decode_v6_t>(
  scifi_pre_decode_v6_t& state,
  scifi_pre_decode_v6_t::arguments_t arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers)
{
  arguments.set_size<dev_scifi_hits>(host_buffers.scifi_hits_uints());
}

template<>
void SequenceVisitor::visit<scifi_pre_decode_v6_t>(
  scifi_pre_decode_v6_t& state,
  const scifi_pre_decode_v6_t::arguments_t& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  state.set_opts(
    dim3(host_buffers.host_number_of_selected_events[0]), dim3(SciFi::SciFiRawBankParams::NbBanks), hip_stream);
  state.set_arguments(
    arguments.offset<dev_scifi_raw_input>(),
    arguments.offset<dev_scifi_raw_input_offsets>(),
    arguments.offset<dev_event_list>(),
    arguments.offset<dev_scifi_hit_count>(),
    arguments.offset<dev_scifi_hits>(),
    constants.dev_scifi_geometry,
    constants.dev_inv_clus_res);

  state.invoke();

  // Debugging
  /*const uint hit_count_uints = 2 * host_buffers.host_number_of_selected_events[0] * SciFi::Constants::n_mats + 1;
  uint host_scifi_hit_count[hit_count_uints];
  uint* host_scifi_hits = new uint[host_buffers.scifi_hits_uints()];
  cudaCheck(hipMemcpyAsync(&host_scifi_hit_count, arguments.offset<dev_scifi_hit_count>(),
  hit_count_uints*sizeof(uint), hipMemcpyDeviceToHost, hip_stream)); cudaCheck(hipMemcpyAsync(host_scifi_hits,
  arguments.offset<dev_scifi_hits>(), arguments.size<dev_scifi_hits>(), hipMemcpyDeviceToHost, hip_stream));
  hipEventRecord(cuda_generic_event, hip_stream);
  hipEventSynchronize(cuda_generic_event);
  SciFi::SciFiGeometry host_geom(constants.host_scifi_geometry);
  SciFi::Hits hi(host_scifi_hits, host_scifi_hit_count[host_buffers.host_number_of_selected_events[0] *
  SciFi::Constants::n_mat_groups_and_mats], &host_geom, constants.host_inv_clus_res.data()); std::ofstream
  outfile("dump_predecode_v6.txt");

  for(size_t event = 0; event < host_buffers.host_number_of_selected_events[0]; event++) {
    SciFi::HitCount host_scifi_hit_count_struct(host_scifi_hit_count, event);
    for(size_t zone = 0; zone < SciFi::Constants::n_zones; zone++) {
      outfile << "\n\nZone " << zone << ": ";
      for(size_t hit = 0; hit < host_scifi_hit_count_struct.zone_number_of_hits(zone); hit++) {
        uint h = host_scifi_hit_count_struct.zone_offset(zone) + hit;
        outfile << (hi.cluster_reference[h]) << " ";
      }
    }
  }
  outfile << std::endl;*/
}
