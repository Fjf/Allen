#include "PrefixSumHandler.cuh"
#include "SequenceVisitor.cuh"
#include "CpuPrefixSum.cuh"

template<>
void SequenceVisitor::set_arguments_size<prefix_sum_scifi_track_hit_number_t>(
  prefix_sum_scifi_track_hit_number_t::arguments_t arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers)
{
  arguments.set_size<dev_prefix_sum_auxiliary_array_6>(
    prefix_sum_scifi_track_hit_number_t::aux_array_size(host_buffers.host_number_of_reconstructed_scifi_tracks[0]));
}

template<>
void SequenceVisitor::visit<prefix_sum_scifi_track_hit_number_t>(
  prefix_sum_scifi_track_hit_number_t& state,
  const prefix_sum_scifi_track_hit_number_t::arguments_t& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  if (runtime_options.cpu_offload) {
    cpu_prefix_sum(
      host_buffers.host_prefix_sum_buffer,
      host_buffers.host_allocated_prefix_sum_space,
      arguments.offset<dev_scifi_track_hit_number>(),
      arguments.size<dev_scifi_track_hit_number>(),
      hip_stream,
      cuda_generic_event,
      host_buffers.host_accumulated_number_of_hits_in_scifi_tracks);
  } else {
    // Set size of the main array to be prefix summed
    state.set_size(host_buffers.host_number_of_reconstructed_scifi_tracks[0]);

    // Set the hip_stream
    state.set_opts(hip_stream);

    // Set arguments: Array to prefix sum and auxiliary array
    state.set_arguments(
      arguments.offset<dev_scifi_track_hit_number>(), arguments.offset<dev_prefix_sum_auxiliary_array_6>());

    // Invoke all steps of prefix sum
    state.invoke();

    // Fetch total number of hits accumulated with all tracks
    cudaCheck(hipMemcpyAsync(
      host_buffers.host_accumulated_number_of_hits_in_scifi_tracks,
      arguments.offset<dev_scifi_track_hit_number>() + host_buffers.host_number_of_reconstructed_scifi_tracks[0],
      sizeof(uint),
      hipMemcpyDeviceToHost,
      hip_stream));

    hipEventRecord(cuda_generic_event, hip_stream);
    hipEventSynchronize(cuda_generic_event);
  }
}
