#include "LFCollectCandidates.cuh"
#include "SequenceVisitor.cuh"

template<>
void SequenceVisitor::set_arguments_size<lf_collect_candidates_t>(
  lf_collect_candidates_t::arguments_t arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers)
{
  arguments.set_size<dev_scifi_lf_number_of_candidates>(
    host_buffers.host_atomics_ut[host_buffers.host_number_of_selected_events[0] * 2]
    * LookingForward::number_of_x_layers
    + 1);

  arguments.set_size<dev_scifi_lf_candidates>(
    host_buffers.host_atomics_ut[host_buffers.host_number_of_selected_events[0] * 2]
    * SciFi::Tracking::zoneoffsetpar
    * LookingForward::maximum_number_of_candidates);
}

template<>
void SequenceVisitor::visit<lf_collect_candidates_t>(
  lf_collect_candidates_t& state,
  const lf_collect_candidates_t::arguments_t& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  cudaCheck(hipMemsetAsync(
    arguments.offset<dev_scifi_lf_number_of_candidates>(),
    0,
    arguments.size<dev_scifi_lf_number_of_candidates>(),
    hip_stream));

  state.set_opts(dim3(host_buffers.host_number_of_selected_events[0]), dim3(64, 6), hip_stream);
  state.set_arguments(
    arguments.offset<dev_scifi_hits>(),
    arguments.offset<dev_scifi_hit_count>(),
    arguments.offset<dev_atomics_ut>(),
    arguments.offset<dev_ut_track_hit_number>(),
    arguments.offset<dev_ut_qop>(),
    arguments.offset<dev_ut_track_velo_indices>(),
    constants.dev_scifi_geometry,
    constants.dev_inv_clus_res,
    arguments.offset<dev_scifi_lf_initial_windows>(),
    arguments.offset<dev_scifi_lf_number_of_candidates>(),
    arguments.offset<dev_scifi_lf_candidates>());

  state.invoke();

  // std::vector<int> number_of_candidates (arguments.size<dev_scifi_lf_number_of_candidates>() / sizeof(int));

  // cudaCheck(hipMemcpy(
  //   number_of_candidates.data(),
  //   arguments.offset<dev_scifi_lf_number_of_candidates>(),
  //   arguments.size<dev_scifi_lf_number_of_candidates>(),
  //   hipMemcpyDeviceToHost));

  // // const auto ut_total_number_of_tracks = host_buffers.host_atomics_ut[host_buffers.host_number_of_selected_events[0] * 2];
  // for (size_t event_number=0; event_number<host_buffers.host_number_of_selected_events[0]; ++event_number) {
  //   const auto offset = host_buffers.host_atomics_ut[host_buffers.host_number_of_selected_events[0] + event_number];
  //   const auto number_of_ut_tracks = host_buffers.host_atomics_ut[host_buffers.host_number_of_selected_events[0] + event_number+1] - offset;

  //   info_cout << "Event #" << event_number << std::endl;
  //   for (size_t i=0; i<number_of_ut_tracks; ++i) {
  //     info_cout << "Candidates #" << i << ": ";

  //     for (int j=0; j<LookingForward::number_of_x_layers; ++j) {
  //       info_cout << number_of_candidates[(offset + i) * LookingForward::number_of_x_layers + j] << ", ";
  //     }

  //     info_cout << std::endl;
  //   }
  // }
}
