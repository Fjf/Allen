#include "LFPromoteCandidates.cuh"
#include "SequenceVisitor.cuh"

template<>
void SequenceVisitor::set_arguments_size<lf_promote_candidates_t>(
  lf_promote_candidates_t::arguments_t arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers)
{
  arguments.set_size<dev_scifi_track_promoted_candidates>(
    host_buffers.host_number_of_selected_events[0] * SciFi::Constants::max_track_candidates);
}

template<>
void SequenceVisitor::visit<lf_promote_candidates_t>(
  lf_promote_candidates_t& state,
  const lf_promote_candidates_t::arguments_t& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  cudaCheck(hipMemsetAsync(
    arguments.offset<dev_scifi_track_promoted_candidates>(),
    0,
    arguments.size<dev_scifi_track_promoted_candidates>(),
    hip_stream
  ));

  state.set_opts(dim3(host_buffers.host_number_of_selected_events[0]), dim3(256), hip_stream);
  state.set_arguments(
    arguments.offset<dev_scifi_hits>(),
    arguments.offset<dev_scifi_hit_count>(),
    arguments.offset<dev_atomics_velo>(),
    arguments.offset<dev_velo_track_hit_number>(),
    arguments.offset<dev_velo_states>(),
    arguments.offset<dev_atomics_ut>(),
    arguments.offset<dev_ut_track_hits>(),
    arguments.offset<dev_ut_track_hit_number>(),
    arguments.offset<dev_ut_qop>(),
    arguments.offset<dev_ut_track_velo_indices>(),
    arguments.offset<dev_scifi_track_candidates>(),
    arguments.offset<dev_scifi_track_promoted_candidates>(),
    arguments.offset<dev_atomics_scifi>(),
    constants.dev_scifi_geometry,
    constants.dev_looking_forward_constants,
    constants.dev_inv_clus_res,
    arguments.offset<dev_scifi_lf_first_layer_candidates>(),
    arguments.offset<dev_scifi_lf_second_layer_candidates>(),
    arguments.offset<dev_ut_states>());

  state.invoke();
}
