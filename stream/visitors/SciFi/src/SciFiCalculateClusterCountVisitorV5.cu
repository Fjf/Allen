#include "SequenceVisitor.cuh"
#include "SciFiCalculateClusterCountV5.cuh"

template<>
void SequenceVisitor::set_arguments_size<scifi_calculate_cluster_count_v5_t>(
  scifi_calculate_cluster_count_v5_t::arguments_t arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers)
{
  arguments.set_size<dev_scifi_hit_count>(
    2 * host_buffers.host_number_of_selected_events[0] * SciFi::Constants::n_mats + 1);
}

template<>
void SequenceVisitor::visit<scifi_calculate_cluster_count_v5_t>(
  scifi_calculate_cluster_count_v5_t& state,
  const scifi_calculate_cluster_count_v5_t::arguments_t& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  cudaCheck(
    hipMemsetAsync(arguments.offset<dev_scifi_hit_count>(), 0, arguments.size<dev_scifi_hit_count>(), hip_stream));

  hipEventRecord(cuda_generic_event, hip_stream);
  hipEventSynchronize(cuda_generic_event);

  state.set_opts(dim3(host_buffers.host_number_of_selected_events[0]), dim3(SciFi::SciFiRawBankParams::NbBanks), hip_stream);
  state.set_arguments(
    arguments.offset<dev_scifi_raw_input>(),
    arguments.offset<dev_scifi_raw_input_offsets>(),
    arguments.offset<dev_event_list>(),
    arguments.offset<dev_scifi_hit_count>(),
    constants.dev_scifi_geometry);

  state.invoke();

  // Debugging
  /*const uint hit_count_uints = 2 * host_buffers.host_number_of_selected_events[0] * SciFi::Constants::n_mat_groups_and_mats + 1;
  uint host_scifi_hit_count[hit_count_uints];
  cudaCheck(hipMemcpyAsync(&host_scifi_hit_count, arguments.offset<dev_scifi_hit_count>(), hit_count_uints*sizeof(uint), hipMemcpyDeviceToHost, hip_stream));
  hipEventRecord(cuda_generic_event, hip_stream);
  hipEventSynchronize(cuda_generic_event);
  std::ofstream outfile("dump_counts");

  for(size_t event = 0; event < host_buffers.host_number_of_selected_events[0]; event++) {
    for(size_t g = 0; g < SciFi::Constants::n_mat_groups_and_mats; g++) {
      outfile << host_scifi_hit_count[event * SciFi::Constants::n_mat_groups_and_mats + g] << std::endl;
    }
  }*/
}
