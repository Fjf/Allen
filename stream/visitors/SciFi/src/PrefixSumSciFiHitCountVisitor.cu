#include "PrefixSumHandler.cuh"
#include "SequenceVisitor.cuh"

template<>
void SequenceVisitor::set_arguments_size<prefix_sum_scifi_hits_t>(
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers,
  argument_manager_t& arguments)
{
  arguments.set_size<dev_prefix_sum_auxiliary_array_4>(
    prefix_sum_scifi_hits_t::aux_array_size(host_buffers.host_number_of_selected_events[0] * SciFi::Constants::n_mats));
}

template<>
void SequenceVisitor::visit<prefix_sum_scifi_hits_t>(
  prefix_sum_scifi_hits_t& state,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  argument_manager_t& arguments,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  // Set size of the main array to be prefix summed
  state.set_size(host_buffers.host_number_of_selected_events[0] * SciFi::Constants::n_mats);

  // Set the hip_stream
  state.set_opts(hip_stream);

  // Set arguments: Array to prefix sum and auxiliary array
  state.set_arguments(
    arguments.offset<dev_scifi_hit_count>(), 
    arguments.offset<dev_prefix_sum_auxiliary_array_4>());

  // Invoke all steps of prefix sum
  state.invoke();

  // Fetch total number of hits
  cudaCheck(hipMemcpyAsync(
    host_buffers.host_accumulated_number_of_scifi_hits,
    arguments.offset<dev_scifi_hit_count>() + host_buffers.host_number_of_selected_events[0] * SciFi::Constants::n_mats,
    sizeof(uint),
    hipMemcpyDeviceToHost,
    hip_stream));

  hipEventRecord(cuda_generic_event, hip_stream);
  hipEventSynchronize(cuda_generic_event);

  // info_cout << "Total SciFi cluster count: " << *host_buffers.host_accumulated_number_of_scifi_hits << std::endl;
}
