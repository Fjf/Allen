#include "LFCalculateParametrization.cuh"
#include "SequenceVisitor.cuh"

template<>
void SequenceVisitor::set_arguments_size<lf_calculate_parametrization_t>(
  lf_calculate_parametrization_t::arguments_t arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers)
{
  arguments.set_size<dev_scifi_lf_parametrization>(
    4 * host_buffers.host_number_of_reconstructed_ut_tracks[0] * LookingForward::maximum_number_of_candidates_per_ut_track);
}

template<>
void SequenceVisitor::visit<lf_calculate_parametrization_t>(
  lf_calculate_parametrization_t& state,
  const lf_calculate_parametrization_t::arguments_t& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  state.set_arguments(
    arguments.offset<dev_scifi_hits>(),
    arguments.offset<dev_scifi_hit_count>(),
    arguments.offset<dev_atomics_velo>(),
    arguments.offset<dev_velo_track_hit_number>(),
    arguments.offset<dev_velo_states>(),
    arguments.offset<dev_atomics_ut>(),
    arguments.offset<dev_ut_track_hit_number>(),
    arguments.offset<dev_ut_track_velo_indices>(),
    arguments.offset<dev_ut_qop>(),
    arguments.offset<dev_scifi_lf_tracks>(),
    arguments.offset<dev_scifi_lf_atomics>(),
    constants.dev_scifi_geometry,
    constants.dev_looking_forward_constants,
    constants.dev_inv_clus_res,
    arguments.offset<dev_scifi_lf_parametrization>());

  state.set_opts(dim3(host_buffers.host_number_of_selected_events[0]), dim3(64), hip_stream);
  state.invoke();
}
