#include "LFTripletSeeding.cuh"
#include "SequenceVisitor.cuh"

template<>
void SequenceVisitor::set_arguments_size<lf_triplet_seeding_t>(
  lf_triplet_seeding_t::arguments_t arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers)
{
  arguments.set_size<dev_scifi_lf_triplet_best>(
    host_buffers.host_number_of_reconstructed_ut_tracks[0] * LookingForward::n_triplet_seeds *
    LookingForward::maximum_number_of_triplets_per_seed);

  // Momentarily this is here
  arguments.set_size<dev_scifi_lf_tracks>(
    host_buffers.host_number_of_reconstructed_ut_tracks[0] * LookingForward::maximum_number_of_candidates_per_ut_track);
  arguments.set_size<dev_scifi_lf_atomics>(
    host_buffers.host_number_of_reconstructed_ut_tracks[0] * LookingForward::num_atomics * 2 + 1);
}

template<>
void SequenceVisitor::visit<lf_triplet_seeding_t>(
  lf_triplet_seeding_t& state,
  const lf_triplet_seeding_t::arguments_t& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  cudaCheck(
    hipMemsetAsync(arguments.offset<dev_scifi_lf_atomics>(), 0, arguments.size<dev_scifi_lf_atomics>(), hip_stream));

  // Note: The initialization of dev_scifi_lf_triplet_best_chi2 is the highest positive
  //       number represented as fp32 that can be initialized using hipMemsetAsync,
  //       that is, initializing the bytes individually:
  //       0x7F results in 0x7F7F7F7F, which is 3.3961514e38 in fp32
  cudaCheck(hipMemsetAsync(
    arguments.offset<dev_scifi_lf_triplet_best>(), 0x7F, arguments.size<dev_scifi_lf_triplet_best>(), hip_stream));

  state.set_arguments(
    arguments.offset<dev_scifi_hits>(),
    arguments.offset<dev_scifi_hit_count>(),
    arguments.offset<dev_atomics_velo>(),
    arguments.offset<dev_velo_states>(),
    arguments.offset<dev_atomics_ut>(),
    arguments.offset<dev_ut_track_hit_number>(),
    arguments.offset<dev_ut_track_velo_indices>(),
    arguments.offset<dev_ut_qop>(),
    constants.dev_scifi_geometry,
    constants.dev_inv_clus_res,
    arguments.offset<dev_scifi_lf_initial_windows>(),
    constants.dev_looking_forward_constants,
    arguments.offset<dev_ut_states>(),
    arguments.offset<dev_scifi_lf_triplet_best>(),
    arguments.offset<dev_scifi_lf_tracks>(),
    arguments.offset<dev_scifi_lf_atomics>());

  state.set_opts(dim3(host_buffers.host_number_of_selected_events[0]), dim3(32), hip_stream);

  state.invoke();
}
