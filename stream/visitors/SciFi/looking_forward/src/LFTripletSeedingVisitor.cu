#include "LFTripletSeeding.cuh"
#include "SequenceVisitor.cuh"

template<>
void SequenceVisitor::set_arguments_size<lf_triplet_seeding_t>(
  lf_triplet_seeding_t::arguments_t arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers)
{
  arguments.set_size<dev_scifi_lf_triplet_best>(
    host_buffers.host_number_of_reconstructed_ut_tracks[0] * 4 * LookingForward::maximum_number_of_candidates *
    LookingForward::maximum_number_of_triplets_per_h1);

  // Momentarily this is here
  arguments.set_size<dev_scifi_lf_tracks>(
    host_buffers.host_number_of_reconstructed_ut_tracks[0] * LookingForward::maximum_number_of_candidates_per_ut_track);
  arguments.set_size<dev_scifi_lf_atomics>(
    host_buffers.host_number_of_reconstructed_ut_tracks[0] * LookingForward::num_atomics * 2 + 1);
}

template<>
void SequenceVisitor::visit<lf_triplet_seeding_t>(
  lf_triplet_seeding_t& state,
  const lf_triplet_seeding_t::arguments_t& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{  
  cudaCheck(
    hipMemsetAsync(arguments.offset<dev_scifi_lf_atomics>(), 0, arguments.size<dev_scifi_lf_atomics>(), hip_stream));

  state.set_arguments(
    arguments.offset<dev_scifi_hits>(),
    arguments.offset<dev_scifi_hit_count>(),
    arguments.offset<dev_atomics_velo>(),
    arguments.offset<dev_velo_states>(),
    arguments.offset<dev_atomics_ut>(),
    arguments.offset<dev_ut_track_hit_number>(),
    arguments.offset<dev_ut_track_velo_indices>(),
    arguments.offset<dev_ut_qop>(),
    constants.dev_scifi_geometry,
    constants.dev_inv_clus_res,
    arguments.offset<dev_scifi_lf_initial_windows>(),
    constants.dev_looking_forward_constants,
    arguments.offset<dev_ut_states>(),
    arguments.offset<dev_scifi_lf_triplet_best>(),
    arguments.offset<dev_scifi_lf_tracks>(),
    arguments.offset<dev_scifi_lf_atomics>());
    
  state.set_opts(
    dim3(host_buffers.host_number_of_selected_events[0]), dim3(32), hip_stream);

  state.invoke();
}
