#include "LFQualityFilterLength.cuh"
#include "SequenceVisitor.cuh"

template<>
void SequenceVisitor::set_arguments_size<lf_quality_filter_length_t>(
  lf_quality_filter_length_t::arguments_t arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers)
{
  arguments.set_size<dev_scifi_lf_length_filtered_tracks>(
    host_buffers.host_number_of_reconstructed_ut_tracks[0] *
    LookingForward::maximum_number_of_candidates_per_ut_track_after_x_filter);
  arguments.set_size<dev_scifi_lf_length_filtered_atomics>(
    host_buffers.host_number_of_selected_events[0] * LookingForward::num_atomics * 2 + 1);
  arguments.set_size<dev_scifi_lf_parametrization_length_filter>(
    4 * host_buffers.host_number_of_reconstructed_ut_tracks[0] *
    LookingForward::maximum_number_of_candidates_per_ut_track_after_x_filter);
}

template<>
void SequenceVisitor::visit<lf_quality_filter_length_t>(
  lf_quality_filter_length_t& state,
  const lf_quality_filter_length_t::arguments_t& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  cudaCheck(hipMemsetAsync(
    arguments.offset<dev_scifi_lf_length_filtered_atomics>(),
    0,
    arguments.size<dev_scifi_lf_length_filtered_atomics>(),
    hip_stream));

  // Code for running the quality filter after this algorithm
  state.set_opts(dim3(host_buffers.host_number_of_selected_events[0]), dim3(256), hip_stream);
  state.set_arguments(
    arguments.offset<dev_atomics_ut>(),
    arguments.offset<dev_scifi_lf_x_filtered_tracks>(),
    arguments.offset<dev_scifi_lf_x_filtered_atomics>(),
    arguments.offset<dev_scifi_lf_length_filtered_tracks>(),
    arguments.offset<dev_scifi_lf_length_filtered_atomics>(),
    arguments.offset<dev_scifi_lf_parametrization_x_filter>(),
    arguments.offset<dev_scifi_lf_parametrization_length_filter>());
  state.invoke();
}
