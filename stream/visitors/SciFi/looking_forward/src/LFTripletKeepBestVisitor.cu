#include "LFTripletKeepBest.cuh"
#include "SequenceVisitor.cuh"

template<>
void SequenceVisitor::set_arguments_size<lf_triplet_keep_best_t>(
  lf_triplet_keep_best_t::arguments_t arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers)
{
  arguments.set_size<dev_scifi_lf_tracks>(
    host_buffers.host_number_of_reconstructed_ut_tracks[0] * LookingForward::maximum_number_of_candidates_per_ut_track);
  arguments.set_size<dev_scifi_lf_atomics>(
    host_buffers.host_number_of_reconstructed_ut_tracks[0] * LookingForward::num_atomics * 2 + 1);
}

template<>
void SequenceVisitor::visit<lf_triplet_keep_best_t>(
  lf_triplet_keep_best_t& state,
  const lf_triplet_keep_best_t::arguments_t& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  cudaCheck(
    hipMemsetAsync(arguments.offset<dev_scifi_lf_atomics>(), 0, arguments.size<dev_scifi_lf_atomics>(), hip_stream));

  state.set_opts(
    dim3(host_buffers.host_number_of_selected_events[0], 4), dim3(32), hip_stream);
  state.set_arguments(
    arguments.offset<dev_scifi_hits>(),
    arguments.offset<dev_scifi_hit_count>(),
    arguments.offset<dev_atomics_velo>(),
    arguments.offset<dev_velo_states>(),
    arguments.offset<dev_atomics_ut>(),
    arguments.offset<dev_ut_track_velo_indices>(),
    constants.dev_scifi_geometry,
    constants.dev_inv_clus_res,
    constants.dev_looking_forward_constants,
    arguments.offset<dev_scifi_lf_tracks>(),
    arguments.offset<dev_scifi_lf_atomics>(),
    arguments.offset<dev_scifi_lf_triplet_best>(),
    arguments.offset<dev_scifi_lf_initial_windows>());

  state.invoke();
}
