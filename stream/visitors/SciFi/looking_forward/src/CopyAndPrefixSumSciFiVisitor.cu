#include "SequenceVisitor.cuh"
#include "PrefixSum.cuh"
#include "CpuPrefixSum.cuh"

DEFINE_EMPTY_SET_ARGUMENTS_SIZE(copy_and_prefix_sum_single_block_scifi_t)

template<>
void SequenceVisitor::visit<copy_and_prefix_sum_single_block_scifi_t>(
  copy_and_prefix_sum_single_block_scifi_t& state,
  const copy_and_prefix_sum_single_block_scifi_t::arguments_t& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  if (runtime_options.cpu_offload) {
    // Copy
    cudaCheck(hipMemcpyAsync(
      (uint*) arguments.offset<dev_atomics_scifi>() + host_buffers.host_number_of_selected_events[0],
      (uint*) arguments.offset<dev_atomics_scifi>(),
      host_buffers.host_number_of_selected_events[0] * sizeof(uint),
      hipMemcpyDeviceToDevice,
      hip_stream));

    // Prefix sum
    cpu_prefix_sum(
      host_buffers.host_prefix_sum_buffer,
      host_buffers.host_allocated_prefix_sum_space,
      (uint*) arguments.offset<dev_atomics_scifi>() + host_buffers.host_number_of_selected_events[0],
      (host_buffers.host_number_of_selected_events[0] + 1) * sizeof(uint),
      hip_stream,
      cuda_generic_event,
      host_buffers.host_number_of_reconstructed_scifi_tracks);
  }
  else {
    state.set_opts(dim3(1), dim3(1024), hip_stream);
    state.set_arguments(
      (uint*) arguments.offset<dev_atomics_scifi>() + host_buffers.host_number_of_selected_events[0] * 2,
      (uint*) arguments.offset<dev_atomics_scifi>(),
      (uint*) arguments.offset<dev_atomics_scifi>() + host_buffers.host_number_of_selected_events[0],
      host_buffers.host_number_of_selected_events[0]);

    state.invoke();

    // Fetch number of reconstructed SciFi tracks.
    cudaCheck(hipMemcpyAsync(
      host_buffers.host_number_of_reconstructed_scifi_tracks,
      arguments.offset<dev_atomics_scifi>() + host_buffers.host_number_of_selected_events[0] * 2,
      sizeof(uint),
      hipMemcpyDeviceToHost,
      hip_stream));

    hipEventRecord(cuda_generic_event, hip_stream);
    hipEventSynchronize(cuda_generic_event);
  }

  if (logger::ll.verbosityLevel >= logger::debug) {
    debug_cout << "Total # of SciFi tracks = " << host_buffers.host_number_of_reconstructed_scifi_tracks[0]
               << std::endl;
  }
}
