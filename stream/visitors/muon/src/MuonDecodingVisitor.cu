#include "SequenceVisitor.cuh"
#include "MuonDecoding.cuh"
#include "MuonRawToHits.cuh"
#include "MuonTables.cuh"
#include "MuonGeometry.cuh"

template<>
void SequenceVisitor::set_arguments_size<muon_decoding_t>(
  muon_decoding_t& state,
  muon_decoding_t::arguments_t arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers)
{
  arguments.set_size<dev_muon_raw>(std::get<0>(runtime_options.host_muon_events).size_bytes());
  arguments.set_size<dev_muon_raw_offsets>(std::get<1>(runtime_options.host_muon_events).size_bytes());
  arguments.set_size<dev_muon_raw_to_hits>(1);
  arguments.set_size<dev_muon_hits>(host_buffers.host_number_of_selected_events[0]);
}

template<>
void SequenceVisitor::visit<muon_decoding_t>(
  muon_decoding_t& state,
  const muon_decoding_t::arguments_t& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{

  // FIXME: this should be done as part of the consumers, but
  // currently it cannot. This is because it is not possible to
  // indicate dependencies between Consumer and/or Producers.
  Muon::MuonRawToHits muonRawToHits {constants.dev_muon_tables, constants.dev_muon_geometry};
  cudaCheck(hipMemcpyAsync(
    arguments.offset<dev_muon_raw_to_hits>(),
    &muonRawToHits,
    sizeof(muonRawToHits),
    hipMemcpyHostToDevice,
    hip_stream));
  cudaCheck(hipMemcpyAsync(
    arguments.offset<dev_muon_raw>(),
    std::get<0>(runtime_options.host_muon_events).begin(),
    std::get<0>(runtime_options.host_muon_events).size_bytes(),
    hipMemcpyHostToDevice,
    hip_stream));
  cudaCheck(hipMemcpyAsync(
    arguments.offset<dev_muon_raw_offsets>(),
    std::get<1>(runtime_options.host_muon_events).begin(),
    std::get<1>(runtime_options.host_muon_events).size_bytes(),
    hipMemcpyHostToDevice,
    hip_stream));
  state.set_opts(
    host_buffers.host_number_of_selected_events[0],
    Muon::Constants::n_stations * Muon::Constants::n_regions * Muon::Constants::n_quarters,
    hip_stream);
  state.set_arguments(
    arguments.offset<dev_event_list>(),
    arguments.offset<dev_muon_raw>(),
    arguments.offset<dev_muon_raw_offsets>(),
    arguments.offset<dev_muon_raw_to_hits>(),
    arguments.offset<dev_muon_hits>());
  state.invoke();
}
