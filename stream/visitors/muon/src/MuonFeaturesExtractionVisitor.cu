#include "SequenceVisitor.cuh"
#include "MuonFeaturesExtraction.cuh"

template<>
void SequenceVisitor::set_arguments_size<muon_catboost_features_extraction_t>(
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers,
  argument_manager_t& arguments)
{ 
  std::cerr << host_buffers.host_number_of_reconstructed_scifi_tracks[0] << std::endl;
  //arguments.set_size<dev_scifi_states>(host_buffers.host_number_of_reconstructed_scifi_tracks[0]);
  arguments.set_size<dev_muon_hits>(runtime_options.number_of_events);
  //arguments.set_size<dev_muon_catboost_features>(host_buffers.host_number_of_reconstructed_scifi_tracks[0] * constants.muon_catboost_n_features);
  std::cerr << constants.muon_catboost_n_features << "lol"  << std::endl;;
  //arguments.set_size<dev_muon_catboost_features>(host_buffers.host_number_of_reconstructed_scifi_tracks[0] * constants.muon_catboost_n_features);
  arguments.set_size<dev_muon_catboost_features>(constants.muon_catboost_n_features * host_buffers.host_number_of_reconstructed_scifi_tracks[0]);
  std::cerr<< arguments.size<dev_muon_catboost_features>() << "kek ";
}

template<>
void SequenceVisitor::visit<muon_catboost_features_extraction_t>(
  muon_catboost_features_extraction_t& state,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  argument_manager_t& arguments,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  // Copy memory from host to device
  cudaCheck(hipMemcpyAsync(
    arguments.offset<dev_muon_hits>(),
    runtime_options.host_muon_hits_events.data(),
    runtime_options.number_of_events * sizeof(Muon::HitsSoA),
    hipMemcpyHostToDevice,
    hip_stream
  ));

  // Setup opts for kernel call
//  state.set_opts(dim3(host_buffers.host_number_of_reconstructed_scifi_tracks[0], Muon::Constants::n_stations), dim3(1), hip_stream);
  state.set_opts(dim3(runtime_options.number_of_events, Muon::Constants::n_stations), dim3(1), hip_stream);

  // Setup arguments for kernel call
  state.set_arguments(
    arguments.offset<dev_atomics_scifi>(),
    arguments.offset<dev_scifi_track_hit_number>(),
    arguments.offset<dev_scifi_qop>(),
    arguments.offset<dev_scifi_states>(),
    arguments.offset<dev_scifi_track_ut_indices>(),
    arguments.offset<dev_muon_hits>(),
    arguments.offset<dev_muon_catboost_features>()
  );

  // Kernel call
  state.invoke();

  std::cerr<< arguments.size<dev_muon_catboost_features>() << "kek ";
  // Retrieve result
  std::vector<float> features(constants.muon_catboost_n_features * host_buffers.host_number_of_reconstructed_scifi_tracks[0]);
  cudaCheck(hipMemcpyAsync(
//    host_buffers.host_muon_catboost_features,
    features.data(),
    arguments.offset<dev_muon_catboost_features>(),
    arguments.size<dev_muon_catboost_features>(),
    //80,
    hipMemcpyDeviceToHost,
    hip_stream
  ));

  // Wait to receive the result
  hipEventRecord(cuda_generic_event, hip_stream);
  hipEventSynchronize(cuda_generic_event);

  // Check the output
  debug_cout << "MUON FEATURES: " << std::endl;
  for (int i = 0; i < constants.muon_catboost_n_features * host_buffers.host_number_of_reconstructed_scifi_tracks[0]/*constants.muon_catboost_n_features*/ ; i++) {
    debug_cout << i % 20 << " " << features[i] << "\n";
    //debug_cout <<  host_buffers.host_muon_catboost_features[i] << " ";
  }
  debug_cout << std::endl << std::endl;
}
