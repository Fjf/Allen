#include "SequenceVisitor.cuh"
#include "IsMuon.cuh"

template<>
void SequenceVisitor::set_arguments_size<is_muon_t>(
  is_muon_t::arguments_t arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers)
{
  arguments.set_size<dev_muon_track_occupancies>(
    Muon::Constants::n_stations * host_buffers.host_number_of_reconstructed_scifi_tracks[0]
  );
  arguments.set_size<dev_is_muon>(host_buffers.host_number_of_reconstructed_scifi_tracks[0]);
}

template<>
void SequenceVisitor::visit<is_muon_t>(
  is_muon_t& state,
  const is_muon_t::arguments_t& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  // Setup opts for kernel call
  state.set_opts(dim3(host_buffers.host_number_of_selected_events[0]), dim3(32, Muon::Constants::n_stations), hip_stream);

  // Setup arguments for kernel call
  state.set_arguments(
    arguments.offset<dev_atomics_scifi>(),
    arguments.offset<dev_scifi_track_hit_number>(),
    arguments.offset<dev_scifi_qop>(),
    arguments.offset<dev_scifi_states>(),
    arguments.offset<dev_scifi_track_ut_indices>(),
    arguments.offset<dev_muon_hits>(),
    arguments.offset<dev_muon_track_occupancies>(),
    arguments.offset<dev_is_muon>(),
    arguments.offset<dev_event_list>(),
    constants.dev_muon_foi,
    constants.dev_muon_momentum_cuts
  );

  // Kernel call
  state.invoke();

  if (runtime_options.do_check) {
    cudaCheck(hipMemcpyAsync(
      host_buffers.host_is_muon,
      arguments.offset<dev_is_muon>(),
      arguments.size<dev_is_muon>(),
      hipMemcpyDeviceToHost,
      hip_stream
    ));
  }
}
