#include "SequenceVisitor.cuh"
#include "VeloKalmanFilter.cuh"

template<>
void SequenceVisitor::set_arguments_size<velo_kalman_fit_t>(
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers,
  argument_manager_t& arguments)
{
  // Set arguments size
  arguments.set_size<dev_kalmanvelo_states>(
    host_buffers.host_number_of_reconstructed_velo_tracks[0] * sizeof(VeloState));
}

template<>
void SequenceVisitor::visit<velo_kalman_fit_t>(
  velo_kalman_fit_t& state,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  argument_manager_t& arguments,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  state.set_opts(dim3(runtime_options.number_of_events), dim3(256), hip_stream);

  state.set_arguments(
    arguments.offset<dev_atomics_velo>(),
    arguments.offset<dev_velo_track_hit_number>(),
    arguments.offset<dev_velo_track_hits>(),
    arguments.offset<dev_velo_states>(),
    arguments.offset<dev_kalmanvelo_states>());

  state.invoke();
}
