#include "SequenceVisitor.cuh"
#include "FillCandidates.cuh"

template<>
void SequenceVisitor::set_arguments_size<velo_fill_candidates_t>(
  velo_fill_candidates_t& state,
  velo_fill_candidates_t::arguments_t arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers)
{
  arguments.set_size<dev_h0_candidates>(2 * host_buffers.host_total_number_of_velo_clusters[0]);
  arguments.set_size<dev_h2_candidates>(2 * host_buffers.host_total_number_of_velo_clusters[0]);
}

template<>
void SequenceVisitor::visit<velo_fill_candidates_t>(
  velo_fill_candidates_t& state,
  const velo_fill_candidates_t::arguments_t& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  // Setup opts and arguments
  state.set_opts(dim3(host_buffers.host_number_of_selected_events[0], 48), hip_stream);
  state.set_arguments(
    arguments.offset<dev_velo_cluster_container>(),
    arguments.offset<dev_estimated_input_size>(),
    arguments.offset<dev_module_cluster_num>(),
    arguments.offset<dev_h0_candidates>(),
    arguments.offset<dev_h2_candidates>());
  state.invoke();
}
