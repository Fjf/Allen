#include "SequenceVisitor.cuh"
#include "WeakTracksAdder.cuh"

DEFINE_EMPTY_SET_ARGUMENTS_SIZE(velo_weak_tracks_adder_t)

template<>
void SequenceVisitor::visit<velo_weak_tracks_adder_t>(
  velo_weak_tracks_adder_t& state,
  const velo_weak_tracks_adder_t::arguments_t& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  // Setup opts and arguments
  state.set_opts(dim3(host_buffers.host_number_of_selected_events[0]), hip_stream);
  state.set_arguments(
    arguments.offset<dev_velo_cluster_container>(),
    arguments.offset<dev_estimated_input_size>(),
    arguments.offset<dev_tracks>(),
    arguments.offset<dev_weak_tracks>(),
    arguments.offset<dev_hit_used>(),
    arguments.offset<dev_atomics_velo>());

  state.invoke();
}
