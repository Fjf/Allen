#include "SequenceVisitor.cuh"
#include "WeakTracksAdder.cuh"

DEFINE_EMPTY_SET_ARGUMENTS_SIZE(velo_weak_tracks_adder_t)

template<>
void SequenceVisitor::visit<velo_weak_tracks_adder_t>(
  velo_weak_tracks_adder_t& state,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  argument_manager_t& arguments,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  // Setup opts and arguments
  state.set_opts(dim3(host_buffers.host_number_of_selected_events[0]), dim3(256), hip_stream);
  state.set_arguments(
    arguments.offset<dev_velo_cluster_container>(),
    arguments.offset<dev_estimated_input_size>(),
    arguments.offset<dev_tracks>(),
    arguments.offset<dev_weak_tracks>(),
    arguments.offset<dev_hit_used>(),
    arguments.offset<dev_atomics_velo>()
  );

  state.invoke();
}
