#include "SequenceVisitor.cuh"
#include "PrefixSum.cuh"

template<>
void SequenceVisitor::set_arguments_size<copy_velo_track_hit_number_t>(
  copy_velo_track_hit_number_t::arguments_t arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers)
{
  arguments.set_size<dev_velo_track_hit_number>(host_buffers.velo_track_hit_number_size());
}

template<>
void SequenceVisitor::visit<copy_velo_track_hit_number_t>(
  copy_velo_track_hit_number_t& state,
  const copy_velo_track_hit_number_t::arguments_t& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  state.set_opts(dim3(host_buffers.host_number_of_selected_events[0]), dim3(512), hip_stream);
  state.set_arguments(
    arguments.offset<dev_tracks>(),
    arguments.offset<dev_atomics_velo>(),
    arguments.offset<dev_velo_track_hit_number>());

  state.invoke();
}
