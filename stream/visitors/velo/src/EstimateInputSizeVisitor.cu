#include "SequenceVisitor.cuh" 
#include "EstimateInputSize.cuh"

template<>
void SequenceVisitor::set_arguments_size<velo_estimate_input_size_t>(
  velo_estimate_input_size_t::arguments_t arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers)
{
  if (logger::ll.verbosityLevel >= logger::debug) {
    debug_cout << "# of events = " << host_buffers.host_number_of_selected_events[0] << std::endl;
  }
  
  arguments.set_size<dev_velo_raw_input>(runtime_options.host_velopix_events_size);
  arguments.set_size<dev_velo_raw_input_offsets>(runtime_options.host_velopix_event_offsets_size);
  arguments.set_size<dev_estimated_input_size>(host_buffers.host_number_of_selected_events[0] * Velo::Constants::n_modules + 1);
  arguments.set_size<dev_module_cluster_num>(host_buffers.host_number_of_selected_events[0] * Velo::Constants::n_modules);
  arguments.set_size<dev_module_candidate_num>(host_buffers.host_number_of_selected_events[0]);
  arguments.set_size<dev_cluster_candidates>(host_buffers.host_number_of_selected_events[0] * VeloClustering::max_candidates_event);
  arguments.set_size<dev_event_order>(host_buffers.host_number_of_selected_events[0]);
} 

template<>
void SequenceVisitor::visit<velo_estimate_input_size_t>(
  velo_estimate_input_size_t& state,
  const velo_estimate_input_size_t::arguments_t& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  // Setup opts and arguments for kernel call
  state.set_opts(dim3(host_buffers.host_number_of_selected_events[0]), dim3(32, 26), hip_stream);
  state.set_arguments(
    arguments.offset<dev_velo_raw_input>(),
    arguments.offset<dev_velo_raw_input_offsets>(),
    arguments.offset<dev_estimated_input_size>(),
    arguments.offset<dev_module_cluster_num>(),
    arguments.offset<dev_module_candidate_num>(),
    arguments.offset<dev_cluster_candidates>(),
    arguments.offset<dev_event_list>(),         
    arguments.offset<dev_event_order>(), 
    constants.dev_velo_candidate_ks
  );

  // Kernel call
  state.invoke();
}
