#include "SequenceVisitor.cuh"
#include "PrefixSum.cuh"
#include "CpuPrefixSum.cuh"

DEFINE_EMPTY_SET_ARGUMENTS_SIZE(copy_and_prefix_sum_single_block_velo_t)

template<>
void SequenceVisitor::visit<copy_and_prefix_sum_single_block_velo_t>(
  copy_and_prefix_sum_single_block_velo_t& state,
  const copy_and_prefix_sum_single_block_velo_t::arguments_t& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  if (runtime_options.cpu_offload) {
    // Copy
    cudaCheck(hipMemcpyAsync(
      (uint*) arguments.offset<dev_atomics_velo>() + host_buffers.host_number_of_selected_events[0],
      (uint*) arguments.offset<dev_atomics_velo>(),
      host_buffers.host_number_of_selected_events[0] * sizeof(uint),
      hipMemcpyDeviceToDevice,
      hip_stream));

    // Prefix sum
    cpu_prefix_sum(
      host_buffers.host_prefix_sum_buffer,
      (uint*) arguments.offset<dev_atomics_velo>() + host_buffers.host_number_of_selected_events[0],
      (host_buffers.host_number_of_selected_events[0] + 1) * sizeof(uint),
      hip_stream,
      cuda_generic_event,
      host_buffers.host_number_of_reconstructed_velo_tracks);
  } else {
    state.set_opts(dim3(1), dim3(1024), hip_stream);
    state.set_arguments(
      (uint*) arguments.offset<dev_atomics_velo>() + host_buffers.host_number_of_selected_events[0] * 2,
      (uint*) arguments.offset<dev_atomics_velo>(),
      (uint*) arguments.offset<dev_atomics_velo>() + host_buffers.host_number_of_selected_events[0],
      host_buffers.host_number_of_selected_events[0]);

    state.invoke();

    // Fetch number of reconstructed tracks
    cudaCheck(hipMemcpyAsync(
      host_buffers.host_number_of_reconstructed_velo_tracks,
      arguments.offset<dev_atomics_velo>() + host_buffers.host_number_of_selected_events[0] * 2,
      sizeof(uint),
      hipMemcpyDeviceToHost,
      hip_stream));

    hipEventRecord(cuda_generic_event, hip_stream);
    hipEventSynchronize(cuda_generic_event);
  }

  if (logger::ll.verbosityLevel >= logger::debug) {
    debug_cout << "number of velo tracks = " << host_buffers.host_number_of_reconstructed_velo_tracks[0] << std::endl;
  }
}
