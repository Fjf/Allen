#include "SequenceVisitor.cuh"
#include "SciFiCalculateClusterCount.cuh"

template<>
void SequenceVisitor::visit<scifi_calculate_cluster_count_t>(
  scifi_calculate_cluster_count_t& state,
  const int sequence_step,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  ArgumentManager<argument_tuple_t>& arguments,
  DynamicScheduler<sequence_t, argument_tuple_t>& scheduler,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  arguments.set_size<arg::dev_scifi_raw_input>(runtime_options.host_scifi_events_size);
  arguments.set_size<arg::dev_scifi_raw_input_offsets>(runtime_options.host_scifi_event_offsets_size);
  arguments.set_size<arg::dev_scifi_hit_count>(2 * runtime_options.number_of_events * SciFi::number_of_mats + 1);
  scheduler.setup_next(arguments, sequence_step);

  cudaCheck(hipMemcpyAsync(arguments.offset<arg::dev_scifi_raw_input>(),
    runtime_options.host_scifi_events,
    runtime_options.host_scifi_events_size,
    hipMemcpyHostToDevice,
    hip_stream));

  cudaCheck(hipMemcpyAsync(arguments.offset<arg::dev_scifi_raw_input_offsets>(),
    runtime_options.host_scifi_event_offsets,
    runtime_options.host_scifi_event_offsets_size * sizeof(uint),
    hipMemcpyHostToDevice,
    hip_stream));

  cudaCheck(hipMemsetAsync(arguments.offset<arg::dev_scifi_hit_count>(),
    0,
    arguments.size<arg::dev_scifi_hit_count>(),
    hip_stream));

  hipEventRecord(cuda_generic_event, hip_stream);
  hipEventSynchronize(cuda_generic_event);

  state.set_opts(dim3(runtime_options.number_of_events), dim3(240), hip_stream);
  state.set_arguments(
    arguments.offset<arg::dev_scifi_raw_input>(),
    arguments.offset<arg::dev_scifi_raw_input_offsets>(),
    arguments.offset<arg::dev_scifi_hit_count>(),
    constants.dev_scifi_geometry
  );

  state.invoke();
}
