#include "SequenceVisitor.cuh"
#include "EstimateClusterCount.cuh"

template<>
void SequenceVisitor::set_arguments_size<estimate_cluster_count_t>(
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers,
  argument_manager_t& arguments)
{
  arguments.set_size<dev_scifi_raw_input>(runtime_options.host_scifi_events_size);
  arguments.set_size<dev_scifi_raw_input_offsets>(runtime_options.host_scifi_event_offsets_size);
  arguments.set_size<dev_scifi_hit_count>(2 * runtime_options.number_of_events * SciFi::Constants::n_zones + 1);
}

template<>
void SequenceVisitor::visit<estimate_cluster_count_t>(
  estimate_cluster_count_t& state,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  argument_manager_t& arguments,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  cudaCheck(hipMemcpyAsync(arguments.offset<dev_scifi_raw_input>(),
    runtime_options.host_scifi_events,
    runtime_options.host_scifi_events_size,
    hipMemcpyHostToDevice,
    hip_stream));

  cudaCheck(hipMemcpyAsync(arguments.offset<dev_scifi_raw_input_offsets>(),
    runtime_options.host_scifi_event_offsets,
    runtime_options.host_scifi_event_offsets_size * sizeof(uint),
    hipMemcpyHostToDevice,
    hip_stream));

  cudaCheck(hipMemsetAsync(arguments.offset<dev_scifi_hit_count>(),
    0,
    arguments.size<dev_scifi_hit_count>(),
    hip_stream));

  hipEventRecord(cuda_generic_event, hip_stream);
  hipEventSynchronize(cuda_generic_event);

  state.set_opts(dim3(runtime_options.number_of_events), dim3(240), hip_stream);
  state.set_arguments(
    arguments.offset<dev_scifi_raw_input>(),
    arguments.offset<dev_scifi_raw_input_offsets>(),
    arguments.offset<dev_scifi_hit_count>(),
    constants.dev_scifi_geometry
  );
  
  state.invoke();
}
