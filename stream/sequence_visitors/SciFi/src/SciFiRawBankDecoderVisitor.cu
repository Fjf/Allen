#include "SequenceVisitor.cuh"
#include "SciFiRawBankDecoder.cuh"

template<>
void SequenceVisitor::visit<scifi_raw_bank_decoder_t>(
  scifi_raw_bank_decoder_t& state,
  const int sequence_step,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  ArgumentManager<argument_tuple_t>& arguments,
  DynamicScheduler<sequence_t, argument_tuple_t>& scheduler,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  arguments.set_size<arg::dev_scifi_hits>(host_buffers.scifi_hits_uints());
  scheduler.setup_next(arguments, sequence_step);

  state.set_opts(dim3(runtime_options.number_of_events), dim3(256), hip_stream);
  state.set_arguments(
    arguments.offset<arg::dev_scifi_raw_input>(),
    arguments.offset<arg::dev_scifi_raw_input_offsets>(),
    arguments.offset<arg::dev_scifi_hit_count>(),
    arguments.offset<arg::dev_scifi_hits>(),
    constants.dev_scifi_geometry
  );

  state.invoke();

  // SciFi Decoder Debugging
  const uint hit_count_uints = 2 * runtime_options.number_of_events * SciFi::number_of_mats + 1;
  uint host_scifi_hit_count[hit_count_uints];
  uint* host_scifi_hits = new uint[host_buffers.scifi_hits_uints()];
  cudaCheck(hipMemcpyAsync(&host_scifi_hit_count, arguments.offset<arg::dev_scifi_hit_count>(), hit_count_uints*sizeof(uint), hipMemcpyDeviceToHost, hip_stream));
  cudaCheck(hipMemcpyAsync(host_scifi_hits, arguments.offset<arg::dev_scifi_hits>(), arguments.size<arg::dev_scifi_hits>(), hipMemcpyDeviceToHost, hip_stream));
  hipEventRecord(cuda_generic_event, hip_stream);
  hipEventSynchronize(cuda_generic_event);

  SciFi::SciFiGeometry host_geom(constants.host_scifi_geometry);
  SciFi::SciFiHits hi(host_scifi_hits, host_scifi_hit_count[runtime_options.number_of_events * SciFi::number_of_mats], &host_geom);

  std::ofstream outfile("dump.txt");
  SciFi::SciFiHitCount host_scifi_hit_count_struct;
  for(size_t event = 0; event < runtime_options.number_of_events; event++) {
    host_scifi_hit_count_struct.typecast_after_prefix_sum(host_scifi_hit_count, event, runtime_options.number_of_events);
    for(size_t zone = 0; zone < SciFi::number_of_zones; zone++) {
      for(size_t hit = 0; hit < host_scifi_hit_count_struct.zone_number_of_hits(zone); hit++) {
        uint h = host_scifi_hit_count_struct.zone_offset(zone) + hit;
        outfile << std::setprecision(8) << std::fixed
          << hi.planeCode(h) << " "
          << zone % 2     << " "
          << hi.LHCbID(h) << " "
          << hi.x0[h]   << " "
          << hi.z0[h]   << " "
          << hi.w(h)    << " "
          << hi.dxdy(h) << " "
          << hi.dzdy(h) << " "
          << hi.yMin(h) << " "
          << hi.yMax(h) << std::endl;
      }
    }
  }
}
