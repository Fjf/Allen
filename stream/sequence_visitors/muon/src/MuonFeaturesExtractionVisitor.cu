#include "SequenceVisitor.cuh"
#include "MuonFeaturesExtraction.cuh"

template<>
void SequenceVisitor::set_arguments_size<muon_catboost_features_extraction_t>(
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers,
  argument_manager_t& arguments)
{ 
  // Set arguments size
  arguments.set_size<dev_muon_track>(1);
  arguments.set_size<dev_muon_hits>(1);
  arguments.set_size<dev_muon_catboost_features>(20);
}

template<>
void SequenceVisitor::visit<muon_catboost_features_extraction_t>(
  muon_catboost_features_extraction_t& state,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  argument_manager_t& arguments,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  // Copy memory from host to device
  cudaCheck(hipMemcpyAsync(
    arguments.offset<dev_muon_track>(),
    host_buffers.host_muon_track,
    1 * sizeof(Muon::State),
    hipMemcpyHostToDevice,
    hip_stream
  ));
  cudaCheck(hipMemcpyAsync(
    arguments.offset<dev_muon_hits>(),
    &runtime_options.host_muon_hits_events[0],
    1 * sizeof(Muon::HitsSoA),
    hipMemcpyHostToDevice,
    hip_stream
  ));

  // Setup opts for kernel call
  state.set_opts(dim3(1), dim3(1), hip_stream);

  // Setup arguments for kernel call
  state.set_arguments(
    arguments.offset<dev_muon_track>(),
    arguments.offset<dev_muon_hits>(),
    arguments.offset<dev_muon_catboost_features>()
  );

  // Kernel call
  state.invoke();

  // Retrieve result
  cudaCheck(hipMemcpyAsync(
    host_buffers.host_muon_catboost_features,
    arguments.offset<dev_muon_catboost_features>(),
    arguments.size<dev_muon_catboost_features>(),
    hipMemcpyDeviceToHost,
    hip_stream
  ));

  // Wait to receive the result
  hipEventRecord(cuda_generic_event, hip_stream);
  hipEventSynchronize(cuda_generic_event);

  // Check the output
  info_cout << "MUON FEATURES: " << std::endl;
  for (int i = 0; i < 20; i++) {
    info_cout << host_buffers.host_muon_catboost_features[i] << " ";
  }
  info_cout << std::endl << std::endl;
}


