#include "SequenceVisitor.cuh"
#include "MuonFeaturesExtraction.cuh"

template<>
void SequenceVisitor::set_arguments_size<muon_catboost_evaluator_t>(
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers,
  argument_manager_t& arguments)
{ 
}

template<>
void SequenceVisitor::visit<muon_catboost_evaluator_t>(
  muon_catboost_evaluator_t& state,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  argument_manager_t& arguments,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
}


