#include "SequenceVisitor.cuh"
#include "MuonCatboostEvaluator.cuh"
#include<vector>

template<>
void SequenceVisitor::set_arguments_size<muon_catboost_evaluator_t>(
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers,
  argument_manager_t& arguments)
{ 
  int event_N = 12;
  arguments.set_size<dev_muon_catboost_output>(event_N);
}

template<>
void SequenceVisitor::visit<muon_catboost_evaluator_t>(
  muon_catboost_evaluator_t& state,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  argument_manager_t& arguments,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  int event_N = 12;
  state.set_opts(dim3((event_N+255)/256), dim3(256), hip_stream);
  state.set_arguments(
    arguments.offset<dev_muon_catboost_output>(),
    event_N
  );
  state.invoke();
  std::vector<float> output(event_N);
  


  cudaCheck(hipMemcpyAsync(
    output.data(),
    arguments.offset<dev_muon_catboost_output>(),
    arguments.size<dev_muon_catboost_output>(),
    hipMemcpyDeviceToHost,
    hip_stream
  ));

  // Wait to receive the result
  hipEventRecord(cuda_generic_event, hip_stream);
  hipEventSynchronize(cuda_generic_event);
  for(int i = 0; i < event_N; ++i) {
    info_cout << output[i] << std::endl;
  }
}
