#include "SequenceVisitor.cuh"
#include "RunBeamlinePVonCPU.h"
#include "Tools.h"

DEFINE_EMPTY_SET_ARGUMENTS_SIZE(cpu_beamlinePV_t)

template<>
void SequenceVisitor::visit<cpu_beamlinePV_t>(
  cpu_beamlinePV_t& state,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  argument_manager_t& arguments,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  // Synchronize previous CUDA transmissions
  hipEventRecord(cuda_generic_event, hip_stream);
  hipEventSynchronize(cuda_generic_event);

  cudaCheck(hipMemcpyAsync(
    host_buffers.host_kalmanvelo_states,
    arguments.offset<dev_kalmanvelo_states>(),
    arguments.size<dev_kalmanvelo_states>(),
    hipMemcpyDeviceToHost,
    hip_stream));

  state.invoke(
    host_buffers.host_kalmanvelo_states,
    host_buffers.host_velo_tracks_atomics,
    host_buffers.host_velo_track_hit_number,
    runtime_options.number_of_events);
}
