#include "SequenceVisitor.cuh"
#include "UTFindPermutation.cuh"

template<>
void SequenceVisitor::set_arguments_size<ut_find_permutation_t>(
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers,
  argument_manager_t& arguments)
{
  arguments.set_size<dev_ut_hit_permutations>(host_buffers.host_accumulated_number_of_ut_hits[0]);
}

template<>
void SequenceVisitor::visit<ut_find_permutation_t>(
  ut_find_permutation_t& state,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  argument_manager_t& arguments,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event)
{
  state.set_opts(dim3(runtime_options.number_of_events, constants.host_unique_x_sector_layer_offsets[4]), dim3(16), hip_stream);
  state.set_arguments(
    arguments.offset<dev_ut_hits>(),
    arguments.offset<dev_ut_hit_offsets>(),
    arguments.offset<dev_ut_hit_permutations>(),
    constants.dev_unique_x_sector_layer_offsets,
    constants.dev_unique_x_sector_offsets,
    constants.dev_unique_sector_xs
  );

  state.invoke();
}
