#include "Stream.cuh"

// Include the sequence checker specializations
#include "VeloSequenceCheckers_impl.cuh"
#include "UTSequenceCheckers_impl.cuh"
#include "SciFiSequenceCheckers_impl.cuh"
#include "PVSequenceCheckers_impl.cuh"
#include "KalmanSequenceCheckers_impl.cuh"
#include "RateCheckers_impl.cuh"

/**
 * @brief Sets up the chain that will be executed later.
 */
hipError_t Stream::initialize(
  const uint max_number_of_events,
  const bool param_do_print_memory_manager,
  const uint param_start_event_offset,
  const size_t reserve_mb,
  const uint param_stream_number,
  const Constants& param_constants,
  const bool do_check)
{
  // Set stream and events
  cudaCheck(hipStreamCreate(&hip_stream));
  cudaCheck(hipEventCreateWithFlags(&cuda_generic_event, hipEventBlockingSync));

  // Set stream options
  stream_number = param_stream_number;
  do_print_memory_manager = param_do_print_memory_manager;
  start_event_offset = param_start_event_offset;
  constants = param_constants;

  // Reserve host buffers
  host_buffers.reserve(max_number_of_events, do_check);

  // Malloc a configurable reserved memory
  cudaCheck(hipMalloc((void**) &dev_base_pointer, reserve_mb * 1024 * 1024));

  // Prepare scheduler
  scheduler.initialize(do_print_memory_manager, reserve_mb * 1024 * 1024, dev_base_pointer);

  return hipSuccess;
}

hipError_t Stream::run_sequence(const RuntimeOptions& runtime_options)
{
  // The sequence is only run if there are events to run on
  number_of_input_events = runtime_options.number_of_events;
  if (runtime_options.number_of_events > 0) {
    for (uint repetition = 0; repetition < runtime_options.number_of_repetitions; ++repetition) {
      // Initialize selected_number_of_events with requested_number_of_events
      host_buffers.host_number_of_selected_events[0] = runtime_options.number_of_events;

      // Reset scheduler
      scheduler.reset();

      // Visit all algorithms in configured sequence
      Sch::RunSequenceTuple<
        scheduler_t,
        SequenceVisitor,
        configured_sequence_t,
        std::tuple<const RuntimeOptions&, const Constants&, const HostBuffers&>,
        std::tuple<const RuntimeOptions&, const Constants&, HostBuffers&, hipStream_t&, hipEvent_t&>>::
        run(
          scheduler,
          sequence_visitor,
          scheduler.sequence_tuple,
          // Arguments to set_arguments_size
          runtime_options,
          constants,
          host_buffers,
          // Arguments to visit
          runtime_options,
          constants,
          host_buffers,
          hip_stream,
          cuda_generic_event);

      // Synchronize CUDA device
      hipEventRecord(cuda_generic_event, hip_stream);
      hipEventSynchronize(cuda_generic_event);
    }
  }

  return hipSuccess;
}

std::vector<bool> Stream::reconstructed_events() const
{
  std::vector<bool> mask(number_of_input_events, false);
  for (uint i = 0; i < host_buffers.host_number_of_selected_events[0]; ++i) {
    mask[host_buffers.host_event_list[i]] = true;
  }
  return mask;
}

void Stream::run_monte_carlo_test(
  CheckerInvoker& invoker,
  MCEvents const& mc_events,
  std::vector<Checker::Tracks> const& forward_tracks)
{

  // FIXME: Filter mc_events using host buffers

  Sch::RunChecker<
    SequenceVisitor,
    configured_sequence_t,
    std::tuple<HostBuffers&, const Constants&, const CheckerInvoker&, const MCEvents&>>::
    check(sequence_visitor, host_buffers, constants, invoker, mc_events);

  if (forward_tracks.size() > 0) {
    info_cout << "Running test on imported tracks" << std::endl;
    std::vector<std::vector<float>> p_events_scifi;
    auto& checker = invoker.checker<TrackCheckerForward>("PrCheckerPlots.root");
    checker.accumulate<TrackCheckerForward>(mc_events, forward_tracks, p_events_scifi);
  }
}
