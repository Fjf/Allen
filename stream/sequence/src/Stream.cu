#include "Stream.cuh"

// Include the sequence checker specializations
#include "VeloSequenceCheckers_impl.cuh"
#include "UTSequenceCheckers_impl.cuh"
#include "SciFiSequenceCheckers_impl.cuh"
#include "PVSequenceCheckers_impl.cuh"
#include "KalmanSequenceCheckers_impl.cuh"
#include "RateCheckers_impl.cuh"

/**
 * @brief Sets up the chain that will be executed later.
 */
hipError_t Stream::initialize(
  const bool param_do_print_memory_manager,
  const uint param_start_event_offset,
  const size_t reserve_mb,
  const uint param_stream_number,
  const Constants& param_constants,
  HostBuffersManager const* buffers_manager)
{
  // Set stream and events
  cudaCheck(hipStreamCreate(&hip_stream));
  cudaCheck(hipEventCreateWithFlags(&cuda_generic_event, hipEventBlockingSync));

  // Set stream options
  stream_number = param_stream_number;
  do_print_memory_manager = param_do_print_memory_manager;
  start_event_offset = param_start_event_offset;
  constants = param_constants;
  
  // Reserve host buffers
  host_buffers_manager = buffers_manager;

  // Malloc a configurable reserved memory
  cudaCheck(hipMalloc((void**) &dev_base_pointer, reserve_mb * 1024 * 1024));

  // Prepare scheduler
  scheduler.initialize(do_print_memory_manager, reserve_mb * 1024 * 1024, dev_base_pointer);

  return hipSuccess;
}

hipError_t Stream::run_sequence(const uint buf_idx, const RuntimeOptions& runtime_options)
{
  host_buffers = host_buffers_manager->getBuffers(buf_idx);
  // The sequence is only run if there are events to run on
  number_of_input_events = runtime_options.number_of_events;
  if (runtime_options.number_of_events > 0) {
    for (uint repetition = 0; repetition < runtime_options.number_of_repetitions; ++repetition) {
      // Initialize selected_number_of_events with requested_number_of_events
      host_buffers->host_number_of_selected_events[0] = runtime_options.number_of_events;

      // Reset scheduler
      scheduler.reset();

      try {
        // Visit all algorithms in configured sequence
        Sch::RunSequenceTuple<
          scheduler_t,
          SequenceVisitor,
          configured_sequence_t,
          std::tuple<const RuntimeOptions&, const Constants&, const HostBuffers&>,
          std::tuple<const RuntimeOptions&, const Constants&, HostBuffers&, hipStream_t&, hipEvent_t&>>::
          run(
            scheduler,
            sequence_visitor,
            scheduler.sequence_tuple,
            // Arguments to set_arguments_size
            runtime_options,
            constants,
            *host_buffers,
            // Arguments to visit
            runtime_options,
            constants,
            *host_buffers,
            hip_stream,
            cuda_generic_event);

        // Synchronize CUDA device
        hipEventRecord(cuda_generic_event, hip_stream);
        hipEventSynchronize(cuda_generic_event);

      } catch (const MemoryException& e) {
        warning_cout << "Insufficient memory to process slice - will sub-divide and retry." << std::endl;
        return hipErrorOutOfMemory;
      }
    }
  }

  return hipSuccess;
}

std::vector<bool> Stream::reconstructed_events() const
{
  std::vector<bool> mask(number_of_input_events, false);
  for (uint i = 0; i < host_buffers->host_number_of_selected_events[0]; ++i) {
    mask[host_buffers->host_event_list[i]] = true;
  }
  return mask;
}

void Stream::run_monte_carlo_test(
  CheckerInvoker& invoker,
  MCEvents const& mc_events,
  std::vector<Checker::Tracks> const& forward_tracks)
{
  Sch::RunChecker<
    SequenceVisitor,
    configured_sequence_t,
    std::tuple<HostBuffers&, const Constants&, const CheckerInvoker&, const MCEvents&>>::
    check(sequence_visitor, *host_buffers, constants, invoker, mc_events);

  if (forward_tracks.size() > 0) {
    info_cout << "Running test on imported tracks" << std::endl;
    auto& checker = invoker.checker<TrackCheckerForward>("PrCheckerPlots.root");
    checker.accumulate<TrackCheckerForward>(mc_events, forward_tracks);
  }
}

hipError_t Stream::free(const bool do_check)
{
  if (host_buffers)
    host_buffers->free(do_check);
  cudaCheck(hipFree(dev_base_pointer));
  
  return hipSuccess;
}
