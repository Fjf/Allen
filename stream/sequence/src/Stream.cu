#include "../include/Stream.cuh"

hipError_t Stream::operator()(
  const char* host_events,
  const uint* host_event_offsets,
  size_t host_events_size,
  size_t host_event_offsets_size,
  uint number_of_events,
  uint number_of_repetitions
) {
  const bool do_print_memory_manager = false;

  // Generate object for populating arguments
  DynamicArgumentGenerator<decltype(arguments)> argen {arguments, dev_base_pointer};

  for (uint repetition=0; repetition<number_of_repetitions; ++repetition) {
    std::vector<std::pair<std::string, float>> times;
    Timer t_total;
    std::map<uint, uint> offsets;
    uint sequence_step = 0;

    // Reset scheduler
    scheduler.reset();

    ////////////////
    // Clustering //
    ////////////////

    // Reserve memory for this step datatypes
    scheduler.setup_next(
      std::map<uint, size_t>{
        argen.size_pair<arg::dev_raw_input>(host_events_size),
        argen.size_pair<arg::dev_raw_input_offsets>(host_event_offsets_size),
        argen.size_pair<arg::dev_estimated_input_size>(number_of_events * VeloTracking::n_modules + 1),
        argen.size_pair<arg::dev_module_cluster_num>(number_of_events * VeloTracking::n_modules),
        argen.size_pair<arg::dev_module_candidate_num>(number_of_events),
        argen.size_pair<arg::dev_cluster_candidates>(number_of_events * VeloClustering::max_candidates_event)
      },
      offsets,
      sequence_step++,
      do_print_memory_manager
    );

    // Setup arguments for kernel call
    sequence.item<seq::estimate_input_size>().set_arguments(
      argen.generate<arg::dev_raw_input>(offsets),
      argen.generate<arg::dev_raw_input_offsets>(offsets),
      argen.generate<arg::dev_estimated_input_size>(offsets),
      argen.generate<arg::dev_module_cluster_num>(offsets),
      argen.generate<arg::dev_module_candidate_num>(offsets),
      argen.generate<arg::dev_cluster_candidates>(offsets)
    );

    if (transmit_host_to_device) {
      cudaCheck(hipMemcpyAsync(argen.generate<arg::dev_raw_input>(offsets), host_events, host_events_size, hipMemcpyHostToDevice, stream));
      cudaCheck(hipMemcpyAsync(argen.generate<arg::dev_raw_input_offsets>(offsets), host_event_offsets, host_event_offsets_size * sizeof(uint), hipMemcpyHostToDevice, stream));
      hipEventRecord(cuda_generic_event, stream);
      hipEventSynchronize(cuda_generic_event);
    }

    // Estimate the input size of each module
    Helper::invoke(
      sequence.item<seq::estimate_input_size>(),
      "Estimate input size",
      times,
      cuda_event_start,
      cuda_event_stop,
      print_individual_rates
    );

    // Reserve memory
    scheduler.setup_next(
      {
        argen.size_pair<arg::dev_cluster_offset>(number_of_events)
      },
      offsets,
      sequence_step++,
      do_print_memory_manager
    );

    // Setup sequence step
    sequence.item<seq::prefix_sum_reduce>().set_arguments(
      argen.generate<arg::dev_estimated_input_size>(offsets),
      argen.generate<arg::dev_cluster_offset>(offsets),
      VeloTracking::n_modules * number_of_events
    );

    // Convert the estimated sizes to module hit start format (offsets)
    Helper::invoke(
      sequence.item<seq::prefix_sum_reduce>(),
      "Prefix sum reduce",
      times,
      cuda_event_start,
      cuda_event_stop,
      print_individual_rates
    );

    // Reserve memory
    scheduler.setup_next(
      {},
      offsets,
      sequence_step++,
      do_print_memory_manager
    );

    // Setup sequence step
    sequence.item<seq::prefix_sum_single_block>().set_arguments(
      argen.generate<arg::dev_estimated_input_size>(offsets) + VeloTracking::n_modules * number_of_events,
      argen.generate<arg::dev_cluster_offset>(offsets),
      prefixSumBlocks
    );

    Helper::invoke(
      sequence.item<seq::prefix_sum_single_block>(),
      "Prefix sum single block",
      times,
      cuda_event_start,
      cuda_event_stop,
      print_individual_rates
    );

    // Reserve memory
    scheduler.setup_next(
      {},
      offsets,
      sequence_step++,
      do_print_memory_manager
    );

    // Setup sequence step
    sequence.item<seq::prefix_sum_scan>().set_arguments(
      argen.generate<arg::dev_estimated_input_size>(offsets),
      argen.generate<arg::dev_cluster_offset>(offsets),
      VeloTracking::n_modules * number_of_events
    );

    Helper::invoke(
      sequence.item<seq::prefix_sum_scan>(),
      "Prefix sum scan",
      times,
      cuda_event_start,
      cuda_event_stop,
      print_individual_rates
    );

    // Fetch the number of hits we require
    cudaCheck(hipMemcpyAsync(host_total_number_of_velo_clusters, argen.generate<arg::dev_estimated_input_size>(offsets) + number_of_events * VeloTracking::n_modules, sizeof(uint), hipMemcpyDeviceToHost, stream));
    hipEventRecord(cuda_generic_event, stream);
    hipEventSynchronize(cuda_generic_event);

    // Reserve memory
    scheduler.setup_next(
      {
        argen.size_pair<arg::dev_velo_cluster_container>(6 * host_total_number_of_velo_clusters[0])
      },
      offsets,
      sequence_step++,
      do_print_memory_manager
    );

    // Setup sequence step
    sequence.item<seq::masked_velo_clustering>().set_arguments(
      argen.generate<arg::dev_raw_input>(offsets),
      argen.generate<arg::dev_raw_input_offsets>(offsets),
      argen.generate<arg::dev_estimated_input_size>(offsets),
      argen.generate<arg::dev_module_cluster_num>(offsets),
      argen.generate<arg::dev_module_candidate_num>(offsets),
      argen.generate<arg::dev_cluster_candidates>(offsets),
      argen.generate<arg::dev_velo_cluster_container>(offsets),
      dev_velo_geometry
    );

    // Invoke clustering
    Helper::invoke(
      sequence.item<seq::masked_velo_clustering>(),
      "Masked velo clustering",
      times,
      cuda_event_start,
      cuda_event_stop,
      print_individual_rates
    );

    /////////////////////////
    // CalculatePhiAndSort //
    /////////////////////////

    // Reserve memory
    scheduler.setup_next(
      {
        argen.size_pair<arg::dev_hit_permutation>(host_total_number_of_velo_clusters[0])
      },
      offsets,
      sequence_step++,
      do_print_memory_manager
    );

    // Setup sequence step
    sequence.item<seq::calculate_phi_and_sort>().set_arguments(
      argen.generate<arg::dev_estimated_input_size>(offsets),
      argen.generate<arg::dev_module_cluster_num>(offsets),
      argen.generate<arg::dev_velo_cluster_container>(offsets),
      argen.generate<arg::dev_hit_permutation>(offsets)
    );

    Helper::invoke(
      sequence.item<seq::calculate_phi_and_sort>(),
      "Calculate phi and sort",
      times,
      cuda_event_start,
      cuda_event_stop,
      print_individual_rates
    );

    /////////////////////
    // SearchByTriplet //
    /////////////////////

    // Reserve memory
    scheduler.setup_next(
      {
        argen.size_pair<arg::dev_tracks>(number_of_events * VeloTracking::max_tracks),
        argen.size_pair<arg::dev_tracklets>(number_of_events * VeloTracking::ttf_modulo),
        argen.size_pair<arg::dev_tracks_to_follow>(number_of_events * VeloTracking::ttf_modulo),
        argen.size_pair<arg::dev_weak_tracks>(number_of_events * VeloTracking::ttf_modulo),
        argen.size_pair<arg::dev_hit_used>(host_total_number_of_velo_clusters[0]),
        argen.size_pair<arg::dev_atomics_storage>(number_of_events * VeloTracking::num_atomics),
        argen.size_pair<arg::dev_h0_candidates>(2 * host_total_number_of_velo_clusters[0]),
        argen.size_pair<arg::dev_h2_candidates>(2 * host_total_number_of_velo_clusters[0]),
        argen.size_pair<arg::dev_rel_indices>(number_of_events * VeloTracking::max_numhits_in_module)
      },
      offsets,
      sequence_step++,
      do_print_memory_manager
    );

    // Setup sequence step
    sequence.item<seq::search_by_triplet>().set_arguments(
      argen.generate<arg::dev_velo_cluster_container>(offsets),
      argen.generate<arg::dev_estimated_input_size>(offsets),
      argen.generate<arg::dev_module_cluster_num>(offsets),
      argen.generate<arg::dev_tracks>(offsets),
      argen.generate<arg::dev_tracklets>(offsets),
      argen.generate<arg::dev_tracks_to_follow>(offsets),
      argen.generate<arg::dev_weak_tracks>(offsets),
      argen.generate<arg::dev_hit_used>(offsets),
      argen.generate<arg::dev_atomics_storage>(offsets),
      argen.generate<arg::dev_h0_candidates>(offsets),
      argen.generate<arg::dev_h2_candidates>(offsets),
      argen.generate<arg::dev_rel_indices>(offsets)
    );

    Helper::invoke(
      sequence.item<seq::search_by_triplet>(),
      "Search by triplet",
      times,
      cuda_event_start,
      cuda_event_stop,
      print_individual_rates
    );
    
    ////////////////////////
    // Consolidate tracks //
    ////////////////////////

    // Calculate accumulated tracks

    // Reserve memory
    scheduler.setup_next(
      {},
      offsets,
      sequence_step++,
      do_print_memory_manager
    );

    // Setup sequence step
    sequence.item<seq::copy_and_prefix_sum_single_block>().set_arguments(
      (uint*) argen.generate<arg::dev_atomics_storage>(offsets) + number_of_events*2,
      (uint*) argen.generate<arg::dev_atomics_storage>(offsets),
      (uint*) argen.generate<arg::dev_atomics_storage>(offsets) + number_of_events,
      number_of_events
    );
    
    Helper::invoke(
      sequence.item<seq::copy_and_prefix_sum_single_block>(),
      "Calculate accumulated tracks",
      times,
      cuda_event_start,
      cuda_event_stop,
      print_individual_rates
    );

    // Fetch number of reconstructed tracks
    cudaCheck(hipMemcpyAsync(host_number_of_reconstructed_velo_tracks, argen.generate<arg::dev_atomics_storage>(offsets) + number_of_events * 2, sizeof(uint), hipMemcpyDeviceToHost, stream));
    hipEventRecord(cuda_generic_event, stream);
    hipEventSynchronize(cuda_generic_event);
    size_t velo_track_hit_number_size = host_number_of_reconstructed_velo_tracks[0] + 1;

    // Prefix sum of accumulated tracks
    // 1. Copy velo track hit number to a consecutive container
    // 2. Reduce
    // 3. Single block
    // 4. Scan

    // Reserve memory
    scheduler.setup_next(
      {
        argen.size_pair<arg::dev_velo_track_hit_number>(velo_track_hit_number_size)
      },
      offsets,
      sequence_step++,
      do_print_memory_manager
    );

    // Setup sequence step
    sequence.item<seq::copy_velo_track_hit_number>().set_arguments(
      argen.generate<arg::dev_tracks>(offsets),
      argen.generate<arg::dev_atomics_storage>(offsets),
      argen.generate<arg::dev_velo_track_hit_number>(offsets)
    );

    Helper::invoke(
      sequence.item<seq::copy_velo_track_hit_number>(),
      "Copy Velo track hit number",
      times,
      cuda_event_start,
      cuda_event_stop,
      print_individual_rates
    );

    // Prefix sum in three kernels
    const size_t prefix_sum_auxiliary_array_2_size = (host_number_of_reconstructed_velo_tracks[0] + 511) / 512;
    scheduler.setup_next(
      {
        argen.size_pair<arg::dev_prefix_sum_auxiliary_array_2>(prefix_sum_auxiliary_array_2_size)
      },
      offsets,
      sequence_step++,
      do_print_memory_manager
    );

    // Setup sequence step
    sequence.item<seq::prefix_sum_reduce_velo_track_hit_number>().set_arguments(
      argen.generate<arg::dev_velo_track_hit_number>(offsets),
      argen.generate<arg::dev_prefix_sum_auxiliary_array_2>(offsets),
      host_number_of_reconstructed_velo_tracks[0]
    );

    // Setup sequence opts
    sequence.item<seq::prefix_sum_reduce_velo_track_hit_number>().set_opts(dim3(prefix_sum_auxiliary_array_2_size), dim3(256), stream);

    // Convert the estimated sizes to module hit start format (offsets)
    Helper::invoke(
      sequence.item<seq::prefix_sum_reduce_velo_track_hit_number>(),
      "Prefix sum reduce: Velo track hit number",
      times,
      cuda_event_start,
      cuda_event_stop,
      print_individual_rates
    );

    // Reserve memory
    scheduler.setup_next({}, offsets, sequence_step++, do_print_memory_manager);

    // Setup sequence step
    sequence.item<seq::prefix_sum_single_block_velo_track_hit_number>().set_arguments(
      argen.generate<arg::dev_velo_track_hit_number>(offsets) + host_number_of_reconstructed_velo_tracks[0],
      argen.generate<arg::dev_prefix_sum_auxiliary_array_2>(offsets),
      prefix_sum_auxiliary_array_2_size
    );

    Helper::invoke(
      sequence.item<seq::prefix_sum_single_block_velo_track_hit_number>(),
      "Prefix sum single block: Velo track hit number",
      times,
      cuda_event_start,
      cuda_event_stop,
      print_individual_rates
    );

    // Reserve memory
    scheduler.setup_next({}, offsets, sequence_step++, do_print_memory_manager);

    // Setup sequence step
    sequence.item<seq::prefix_sum_scan_velo_track_hit_number>().set_arguments(
      argen.generate<arg::dev_velo_track_hit_number>(offsets),
      argen.generate<arg::dev_prefix_sum_auxiliary_array_2>(offsets),
      host_number_of_reconstructed_velo_tracks[0]
    );

    // Setup sequence opts
    const uint pss_velo_track_hit_number_opts =
      prefix_sum_auxiliary_array_2_size==1 ? 1 : (prefix_sum_auxiliary_array_2_size-1);
    sequence.item<seq::prefix_sum_scan_velo_track_hit_number>().set_opts(dim3(pss_velo_track_hit_number_opts), dim3(512), stream);

    Helper::invoke(
      sequence.item<seq::prefix_sum_scan_velo_track_hit_number>(),
      "Prefix sum scan",
      times,
      cuda_event_start,
      cuda_event_stop,
      print_individual_rates
    );

    // Fetch total number of hits accumulated
    // with all tracks
    cudaCheck(hipMemcpyAsync(host_accumulated_number_of_hits_in_velo_tracks,
      argen.generate<arg::dev_velo_track_hit_number>(offsets) + host_number_of_reconstructed_velo_tracks[0],
      sizeof(uint), hipMemcpyDeviceToHost, stream));
    hipEventRecord(cuda_generic_event, stream);
    hipEventSynchronize(cuda_generic_event);

    // Reserve memory
    scheduler.setup_next(
      {
        argen.size_pair<arg::dev_velo_track_hits>(host_accumulated_number_of_hits_in_velo_tracks[0]),
        argen.size_pair<arg::dev_velo_states>(host_number_of_reconstructed_velo_tracks[0])
      },
      offsets,
      sequence_step++,
      do_print_memory_manager
    );

    // Setup sequence step
    sequence.item<seq::consolidate_tracks>().set_arguments(
      argen.generate<arg::dev_atomics_storage>(offsets),
      argen.generate<arg::dev_tracks>(offsets),
      argen.generate<arg::dev_velo_track_hit_number>(offsets),
      argen.generate<arg::dev_velo_cluster_container>(offsets),
      argen.generate<arg::dev_estimated_input_size>(offsets),
      argen.generate<arg::dev_module_cluster_num>(offsets),
      argen.generate<arg::dev_velo_track_hits>(offsets),
      argen.generate<arg::dev_velo_states>(offsets)
    );    

    Helper::invoke(
      sequence.item<seq::consolidate_tracks>(),
      "Consolidate tracks",
      times,
      cuda_event_start,
      cuda_event_stop,
      print_individual_rates
    );

    ////////////////////////////////////////
    // Optional: Simplified Kalman filter //
    ////////////////////////////////////////

    // if (do_simplified_kalman_filter) {
    //   Helper::invoke(
    //     simplifiedKalmanFilter,
    //     "Simplified Kalman filter",
    //     times,
    //     cuda_event_start,
    //     cuda_event_stop,
    //     print_individual_rates
    //   );
    // }
    
    // Transmission device to host
    if (transmit_device_to_host) {
      cudaCheck(hipMemcpyAsync(host_number_of_tracks, argen.generate<arg::dev_atomics_storage>(offsets), number_of_events * sizeof(int), hipMemcpyDeviceToHost, stream));
      cudaCheck(hipMemcpyAsync(host_accumulated_tracks, argen.generate<arg::dev_atomics_storage>(offsets) + number_of_events, number_of_events * sizeof(int), hipMemcpyDeviceToHost, stream));
      cudaCheck(hipMemcpyAsync(host_velo_track_hit_number, argen.generate<arg::dev_velo_track_hit_number>(offsets), argen.size<arg::dev_velo_track_hit_number>(velo_track_hit_number_size), hipMemcpyDeviceToHost, stream));
      cudaCheck(hipMemcpyAsync(host_velo_track_hits, argen.generate<arg::dev_velo_track_hits>(offsets), argen.size<arg::dev_velo_track_hits>(host_accumulated_number_of_hits_in_velo_tracks[0]), hipMemcpyDeviceToHost, stream));
    }

    hipEventRecord(cuda_generic_event, stream);
    hipEventSynchronize(cuda_generic_event);

    if (print_individual_rates) {
      t_total.stop();
      times.emplace_back("total", t_total.get());
      print_timing(number_of_events, times);
    }

    ///////////////////////
    // Monte Carlo Check //
    ///////////////////////

    if (mc_check_enabled) {
      if (repetition == 0 && do_check) { // only check efficiencies once
        // Fetch data
        cudaCheck(hipMemcpyAsync(host_number_of_tracks, argen.generate<arg::dev_atomics_storage>(offsets), number_of_events * sizeof(int), hipMemcpyDeviceToHost, stream));
        cudaCheck(hipMemcpyAsync(host_accumulated_tracks, argen.generate<arg::dev_atomics_storage>(offsets) + number_of_events, number_of_events * sizeof(int), hipMemcpyDeviceToHost, stream));
        cudaCheck(hipMemcpyAsync(host_velo_track_hit_number, argen.generate<arg::dev_velo_track_hit_number>(offsets), argen.size<arg::dev_velo_track_hit_number>(velo_track_hit_number_size), hipMemcpyDeviceToHost, stream));
        cudaCheck(hipMemcpyAsync(host_velo_track_hits, argen.generate<arg::dev_velo_track_hits>(offsets), argen.size<arg::dev_velo_track_hits>(host_accumulated_number_of_hits_in_velo_tracks[0]), hipMemcpyDeviceToHost, stream));
        hipEventRecord(cuda_generic_event, stream);
        hipEventSynchronize(cuda_generic_event);

        checkTracks(
          host_number_of_tracks,
          host_accumulated_tracks,
          host_velo_track_hit_number,
          reinterpret_cast<Hit<true>*>(host_velo_track_hits),
          number_of_events,
          folder_name_MC
        );
      }
    }
  }
  return hipSuccess;
}

void Stream::print_timing(
  const unsigned int number_of_events,
  const std::vector<std::pair<std::string, float>>& times
) {
  const auto total_time = times[times.size() - 1];
  std::string partial_times = "{\n";
  for (size_t i=0; i<times.size(); ++i) {
    if (i != times.size()-1) {
      partial_times += " " + times[i].first + "\t" + std::to_string(times[i].second) + "\t("
        + std::to_string(100 * (times[i].second / total_time.second)) + " %)\n";
    } else {
      partial_times += " " + times[i].first + "\t" + std::to_string(times[i].second) + "\t("
        + std::to_string(100 * (times[i].second / total_time.second)) + " %)\n}";
    }
  }

  info_cout << "stream #" << stream_number << ": "
    << number_of_events / total_time.second << " events/s"
    << ", partial timers (s): " << partial_times
    << std::endl;
}
