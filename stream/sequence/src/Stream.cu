#include "Stream.cuh"

/**
 * @brief Sets up the chain that will be executed later.
 */
hipError_t Stream::initialize(
  const std::vector<char>& velopix_geometry,
  const PrUTMagnetTool* host_ut_magnet_tool,
  const uint max_number_of_events,
  const bool param_transmit_device_to_host,
  const bool param_do_check,
  const bool param_do_simplified_kalman_filter,
  const bool param_do_print_memory_manager,
  const std::string& param_folder_name_MC,
  const size_t reserve_mb,
  const uint param_stream_number
) {
  // Set stream and events
  cudaCheck(hipStreamCreate(&stream));
  cudaCheck(hipEventCreate(&cuda_generic_event));
  cudaCheck(hipEventCreate(&cuda_event_start));
  cudaCheck(hipEventCreate(&cuda_event_stop));

  // Set stream options
  stream_number = param_stream_number;
  transmit_device_to_host = param_transmit_device_to_host;
  do_check = param_do_check;
  do_simplified_kalman_filter = param_do_simplified_kalman_filter;
  do_print_memory_manager = param_do_print_memory_manager;
  folder_name_MC = param_folder_name_MC;

  // Special case
  // Populate velo geometry
  cudaCheck(hipMalloc((void**)&dev_velo_geometry, velopix_geometry.size()));
  cudaCheck(hipMemcpyAsync(dev_velo_geometry, velopix_geometry.data(), velopix_geometry.size(), hipMemcpyHostToDevice, stream));

  // Populate UT magnet tool values
  cudaCheck(hipMalloc((void**)&dev_ut_magnet_tool, sizeof(PrUTMagnetTool)));
  cudaCheck(hipMemcpyAsync(dev_ut_magnet_tool, host_ut_magnet_tool, sizeof(PrUTMagnetTool), hipMemcpyHostToDevice, stream));
  // copy to constant memory variables, defined in PrVeloUTDefinitions.cuh 
  cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(VeloUTTracking::dev_dxDyTable), VeloUTTracking::dxDyTable, VeloUTTracking::n_layers * sizeof(float)));
  float(hipMemcpyToSymbol(HIP_SYMBOL(PrVeloUTConst::dev_minValsBdl), PrVeloUTConst::minValsBdl, 3 * sizeof(float)));
  cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(PrVeloUTConst::dev_maxValsBdl), PrVeloUTConst::maxValsBdl, 3 * sizeof(float)));
  cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(PrVeloUTConst::dev_deltaBdl), PrVeloUTConst::deltaBdl, 3 * sizeof(float)));
  cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(PrVeloUTConst::dev_dxDyHelper), PrVeloUTConst::dxDyHelper, 4 * sizeof(float)));
  
  // Memory allocations for host memory (copy back)
  cudaCheck(hipHostMalloc((void**)&host_number_of_tracks, max_number_of_events * sizeof(int)));
  cudaCheck(hipHostMalloc((void**)&host_accumulated_tracks, max_number_of_events * sizeof(int)));
  cudaCheck(hipHostMalloc((void**)&host_velo_track_hit_number, max_number_of_events * VeloTracking::max_tracks * sizeof(uint)));
  cudaCheck(hipHostMalloc((void**)&host_velo_track_hits, max_number_of_events * VeloTracking::max_tracks * 20 * sizeof(VeloTracking::Hit<mc_check_enabled>)));
  cudaCheck(hipHostMalloc((void**)&host_total_number_of_velo_clusters, sizeof(uint)));
  cudaCheck(hipHostMalloc((void**)&host_number_of_reconstructed_velo_tracks, sizeof(uint)));
  cudaCheck(hipHostMalloc((void**)&host_accumulated_number_of_hits_in_velo_tracks, sizeof(uint)));
  cudaCheck(hipHostMalloc((void**)&host_velo_states, max_number_of_events * VeloTracking::max_tracks * sizeof(VeloState)));
  cudaCheck(hipHostMalloc((void**)&host_veloUT_tracks, max_number_of_events * VeloUTTracking::max_num_tracks * sizeof(VeloUTTracking::TrackUT)));

  // Define sequence of algorithms to execute
  sequence.set(sequence_algorithms());

  // Get sequence and argument names
  sequence_names = get_sequence_names();
  argument_names = get_argument_names();

  // Set options for each algorithm
  // (number of blocks, number of threads, stream, dynamic shared memory space)
  // Setup sequence items opts that are static and will not change
  // regardless of events on flight
  sequence.item<seq::prefix_sum_single_block>().set_opts(                      dim3(1), dim3(1024), stream);
  sequence.item<seq::copy_and_prefix_sum_single_block>().set_opts(             dim3(1), dim3(1024), stream);
  sequence.item<seq::prefix_sum_single_block_velo_track_hit_number>().set_opts(dim3(1), dim3(1024), stream);

  // Get dependencies for each algorithm
  std::vector<std::vector<int>> sequence_dependencies = get_sequence_dependencies();

  // Get output arguments from the sequence
  std::vector<int> sequence_output_arguments = get_sequence_output_arguments();

  // Prepare dynamic scheduler
  scheduler = BaseDynamicScheduler{sequence_names, argument_names,
    sequence_dependencies, sequence_output_arguments,
    reserve_mb * 1024 * 1024, do_print_memory_manager};

  // Malloc a configurable reserved memory
  cudaCheck(hipMalloc((void**)&dev_base_pointer, reserve_mb * 1024 * 1024));

  return hipSuccess;
}

void Stream::print_timing(
  const unsigned int number_of_events,
  const std::vector<std::pair<std::string, float>>& times
) {
  const auto total_time = times[times.size() - 1];
  std::string partial_times = "{\n";
  for (size_t i=0; i<times.size(); ++i) {
    if (i != times.size()-1) {
      partial_times += " " + times[i].first + "\t" + std::to_string(times[i].second) + "\t("
        + std::to_string(100 * (times[i].second / total_time.second)) + " %)\n";
    } else {
      partial_times += " " + times[i].first + "\t" + std::to_string(times[i].second) + "\t("
        + std::to_string(100 * (times[i].second / total_time.second)) + " %)\n}";
    }
  }

  info_cout << "stream #" << stream_number << ": "
    << number_of_events / total_time.second << " events/s"
    << ", partial timers (s): " << partial_times
    << std::endl;
}
