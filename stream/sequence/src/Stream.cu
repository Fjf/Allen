#include "../include/Stream.cuh"
#include "../../../main/include/Common.h"

#include "../../../PrVeloUT/src/PrVeloUT.h"

hipError_t Stream::operator()(
  const char* host_events_pinned,
  const uint* host_event_offsets_pinned,
  size_t host_events_pinned_size,
  size_t host_event_offsets_pinned_size,
  const VeloUTTracking::HitsSoA hits_layers_events[],
  const uint32_t n_hits_layers_events[][VeloUTTracking::n_layers],
  uint number_of_events,
  uint number_of_repetitions
) {
  for (uint repetition=0; repetition<number_of_repetitions; ++repetition) {
    std::vector<std::pair<std::string, float>> times;
    Timer t_total;

    ////////////////
    // Clustering //
    ////////////////

    if (transmit_host_to_device) {
      cudaCheck(hipMemcpyAsync(estimateInputSize.dev_raw_input, host_events_pinned, host_events_pinned_size, hipMemcpyHostToDevice, stream));
      cudaCheck(hipMemcpyAsync(estimateInputSize.dev_raw_input_offsets, host_event_offsets_pinned, host_event_offsets_pinned_size * sizeof(uint), hipMemcpyHostToDevice, stream));
    }

    // Estimate the input size of each module
    Helper::invoke(
      estimateInputSize,
      "Estimate input size",
      times,
      cuda_event_start,
      cuda_event_stop,
      print_individual_rates
    );

    // Convert the estimated sizes to module hit start format (offsets)
    Helper::invoke(
      prefixSumReduce,
      "Prefix sum reduce",
      times,
      cuda_event_start,
      cuda_event_stop,
      print_individual_rates
    );

    Helper::invoke(
      prefixSumSingleBlock,
      "Prefix sum single block",
      times,
      cuda_event_start,
      cuda_event_stop,
      print_individual_rates
    );

    Helper::invoke(
      prefixSumScan,
      "Prefix sum scan",
      times,
      cuda_event_start,
      cuda_event_stop,
      print_individual_rates
    );

    // // Fetch the number of hits we require
    // uint number_of_hits;
    // cudaCheck(hipMemcpyAsync(&number_of_hits, estimateInputSize.dev_estimated_input_size + number_of_events * VeloTracking::n_modules, sizeof(uint), hipMemcpyDeviceToHost, stream));
    // const auto required_size = number_of_hits * 6;

    // if (required_size > velo_cluster_container_size) {
    //   warning_cout << "Number of hits: " << number_of_hits << std::endl
    //     << "Size of velo cluster container is larger than previously accomodated." << std::endl
    //     << "Resizing from " << velo_cluster_container_size * sizeof(uint) << " to " << required_size * sizeof(uint) << " B" << std::endl;

    //   cudaCheck(hipFree(maskedVeloClustering.dev_velo_cluster_container));
    //   cudaCheck(hipMalloc((void**)&maskedVeloClustering.dev_velo_cluster_container, required_size * sizeof(uint)));
    // }

    // Invoke clustering
    Helper::invoke(
      maskedVeloClustering,
      "Masked velo clustering",
      times,
      cuda_event_start,
      cuda_event_stop,
      print_individual_rates
    );

    // Print output
    // maskedVeloClustering.print_output(number_of_events, 3);

    if (do_check) {
      // Check results
      maskedVeloClustering.check(
        host_events_pinned,
        host_event_offsets_pinned,
        host_events_pinned_size,
        host_event_offsets_pinned_size,
        geometry,
        number_of_events
      );
    }

    /////////////////////////
    // CalculatePhiAndSort //
    /////////////////////////

    Helper::invoke(
      calculatePhiAndSort,
      "Calculate phi and sort",
      times,
      cuda_event_start,
      cuda_event_stop,
      print_individual_rates
    );

    // Print output
    // calculatePhiAndSort.print_output(number_of_events);

    /////////////////////
    // SearchByTriplet //
    /////////////////////

    Helper::invoke(
      searchByTriplet,
      "Search by triplet",
      times,
      cuda_event_start,
      cuda_event_stop,
      print_individual_rates
     );

    // Print output
    // searchByTriplet.print_output(number_of_events);

    //////////////////////////////
    // Simplified Kalman filter //
    //////////////////////////////

    Helper::invoke(
      simplifiedKalmanFilter,
      "Simplified Kalman filter",
      times,
      cuda_event_start,
      cuda_event_stop,
      print_individual_rates
     );
  
    ////////////////////////
    // Consolidate tracks //
    ////////////////////////
    
    Helper::invoke(
      consolidateTracks,
      "Consolidate tracks",
      times,
      cuda_event_start,
      cuda_event_stop,
      print_individual_rates
    );
    
    // Transmission device to host
    if (transmit_device_to_host) {
      cudaCheck(hipMemcpyAsync(host_number_of_tracks_pinned, searchByTriplet.dev_atomics_storage, number_of_events * sizeof(int), hipMemcpyDeviceToHost, stream));
      cudaCheck(hipMemcpyAsync(host_tracks_pinned, consolidateTracks.dev_output_tracks, number_of_events * max_tracks_in_event * sizeof(VeloTracking::Track<do_mc_check>), hipMemcpyDeviceToHost, stream));
      cudaCheck(hipMemcpyAsync(host_accumulated_tracks, (void*)(searchByTriplet.dev_atomics_storage + number_of_events), number_of_events * sizeof(int), hipMemcpyDeviceToHost, stream));
      cudaCheck(hipMemcpyAsync(host_velo_states, consolidateTracks.dev_velo_states_out, number_of_events * max_tracks_in_event * VeloTracking::states_per_track * sizeof(VeloState), hipMemcpyDeviceToHost, stream));
    }

    hipEventRecord(cuda_generic_event, stream);
    hipEventSynchronize(cuda_generic_event);

    if (print_individual_rates) {
      t_total.stop();
      times.emplace_back("total", t_total.get());
      print_timing(number_of_events, times);
    }

    ///////////////////////
    // Monte Carlo Check //
    ///////////////////////


    if (do_mc_check) {
      if (repetition == 0) { // only check efficiencies once
        // Fetch data
        cudaCheck(hipMemcpyAsync(host_number_of_tracks_pinned, searchByTriplet.dev_atomics_storage, number_of_events * sizeof(int), hipMemcpyDeviceToHost, stream));
        cudaCheck(hipMemcpyAsync(host_accumulated_tracks, (void*)(searchByTriplet.dev_atomics_storage + number_of_events), number_of_events * sizeof(int), hipMemcpyDeviceToHost, stream));
        cudaCheck(hipMemcpyAsync(host_tracks_pinned, consolidateTracks.dev_output_tracks, number_of_events * max_tracks_in_event * sizeof(VeloTracking::Track<do_mc_check>), hipMemcpyDeviceToHost, stream));
        hipEventRecord(cuda_generic_event, stream);
        hipEventSynchronize(cuda_generic_event);

        const std::vector< trackChecker::Tracks > tracks_events = prepareTracks(
          host_tracks_pinned,
      	  host_accumulated_tracks,
      	  host_number_of_tracks_pinned,
      	  number_of_events);
      
        const bool fromNtuple = true;
        const std::string trackType = "Velo";
        callPrChecker(
	  tracks_events,
      	  folder_name_MC,
	  fromNtuple,
	  trackType);
      }
    }

    /* Plugin VeloUT CPU code here 
       ATTENTION: assumes we run with 1 stream only
     */
    // Fill Velo states into vector
    PrVeloUT velout;
    if ( velout.initialize() ) {
      for ( int i_event = 0; i_event < number_of_events; ++i_event ) {
	VeloState* velo_states_event = host_velo_states + host_accumulated_tracks[i_event];
	std::vector<VeloUTTracking::TrackVelo> tracks;
	for ( uint i_track = 0; i_track < host_number_of_tracks_pinned[i_event]; i_track++ ) {
	  VeloUTTracking::TrackVelo states;
	  states.push_back( velo_states_event[i_track] );
	  tracks.push_back( states );
	}
	velout(tracks); 
      }
    }

       
    
  }
  return hipSuccess;
}

void Stream::print_timing(
  const unsigned int number_of_events,
  const std::vector<std::pair<std::string, float>>& times
) {
  const auto total_time = times[times.size() - 1];
  std::string partial_times = "{\n";
  for (size_t i=0; i<times.size(); ++i) {
    if (i != times.size()-1) {
      partial_times += " " + times[i].first + "\t" + std::to_string(times[i].second) + "\t("
        + std::to_string(100 * (times[i].second / total_time.second)) + " %)\n";
    } else {
      partial_times += " " + times[i].first + "\t" + std::to_string(times[i].second) + "\t("
        + std::to_string(100 * (times[i].second / total_time.second)) + " %)\n}";
    }
  }

  info_cout << "stream #" << stream_number << ": "
    << number_of_events / total_time.second << " events/s"
    << ", partial timers (s): " << partial_times
    << std::endl;
}
