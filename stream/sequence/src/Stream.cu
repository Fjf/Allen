#include "Stream.cuh"
#include "StreamWrapper.cuh"

// Include the sequence checker specializations
#include "VeloSequenceCheckers_impl.cuh"
#include "UTSequenceCheckers_impl.cuh"
#include "SciFiSequenceCheckers_impl.cuh"
#include "PVSequenceCheckers_impl.cuh"
#include "KalmanSequenceCheckers_impl.cuh"
//#include "RateCheckers_impl.cuh"

StreamWrapper::StreamWrapper() {
  number_of_hlt1_lines = std::tuple_size<configured_lines_t>::value;
  uint passthrough_line_index = 0;
  const auto lambda_fn = [&] (const unsigned long i) {
    passthrough_line_index = i;
  };
  Hlt1::TraverseLines<configured_lines_t, Hlt1::SpecialLine, decltype(lambda_fn)>::traverse(lambda_fn);
  passthrough_line = passthrough_line_index;
}

void StreamWrapper::initialize_streams(
  const uint n,
  const bool print_memory_usage,
  const uint start_event_offset,
  const size_t reserve_mb,
  const Constants& constants,
  const std::map<std::string, std::map<std::string, std::string>>& config)
{
  for (uint i = 0; i < n; ++i) {
    streams.push_back(new Stream());
    streams.back()->configure_algorithms(config);
  }

  for (size_t i = 0; i < streams.size(); ++i) {
    streams[i]->initialize(
      print_memory_usage, start_event_offset, reserve_mb, constants);
  }
}

void StreamWrapper::initialize_streams_host_buffers_manager(
  HostBuffersManager* buffers_manager) {
  for (size_t i = 0; i < streams.size(); ++i) {
    streams[i]->set_host_buffer_manager(buffers_manager);
  }
}

hipError_t StreamWrapper::run_stream(const uint i, const uint buf_idx, const RuntimeOptions& runtime_options)
{
  return streams[i]->run_sequence(buf_idx, runtime_options);
}

std::vector<bool> StreamWrapper::reconstructed_events(const uint i) const { return streams[i]->reconstructed_events(); }

void StreamWrapper::run_monte_carlo_test(
  uint const i,
  CheckerInvoker& invoker,
  MCEvents const& mc_events,
  std::vector<Checker::Tracks> const& forward_tracks)
{
  streams[i]->run_monte_carlo_test(invoker, mc_events, forward_tracks);
}

std::map<std::string, std::map<std::string, std::string>> StreamWrapper::get_algorithm_configuration()
{
  return streams.front()->get_algorithm_configuration();
}

StreamWrapper::~StreamWrapper()
{
  for (auto& stream : streams) {
    delete stream;
  }
}

void print_configured_sequence()
{
  info_cout << "\nConfigured sequence of algorithms:\n";
  Sch::PrintAlgorithmSequence<configured_sequence_t>::print();
  info_cout << std::endl;
}

/**
 * @brief Sets up the chain that will be executed later.
 */
hipError_t Stream::initialize(
  const bool param_do_print_memory_manager,
  const uint param_start_event_offset,
  const size_t reserve_mb,
  const Constants& param_constants)
{
  // Set stream and events
  cudaCheck(hipStreamCreate(&hip_stream));
  cudaCheck(hipEventCreateWithFlags(&cuda_generic_event, hipEventBlockingSync));

  // Set stream options
  do_print_memory_manager = param_do_print_memory_manager;
  start_event_offset = param_start_event_offset;
  constants = param_constants;

  // Malloc a configurable reserved memory on the host
  // TODO: Make configurable
  cudaCheck(hipHostMalloc((void**) &host_base_pointer, 10 * 1024 * 1024));

  // Malloc a configurable reserved memory on the device
  cudaCheck(hipMalloc((void**) &dev_base_pointer, reserve_mb * 1024 * 1024));

  // Prepare scheduler
  scheduler.initialize(do_print_memory_manager, reserve_mb * 1024 * 1024, dev_base_pointer, 10 * 1024 * 1024, host_base_pointer);

  return hipSuccess;
}

void Stream::set_host_buffer_manager(HostBuffersManager* buffers_manager) {
  // Set host buffers manager
  host_buffers_manager = buffers_manager;
}

hipError_t Stream::run_sequence(const uint buf_idx, const RuntimeOptions& runtime_options)
{
  host_buffers = host_buffers_manager->getBuffers(buf_idx);
  // The sequence is only run if there are events to run on
  auto event_start = std::get<0>(runtime_options.event_interval);
  auto event_end = std::get<1>(runtime_options.event_interval);

  number_of_input_events = event_end - event_start;
  if (event_end > event_start) {
    for (uint repetition = 0; repetition < runtime_options.number_of_repetitions; ++repetition) {
      // Initialize selected_number_of_events with requested_number_of_events
      host_buffers->host_number_of_selected_events[0] = event_end - event_start;

      // Reset scheduler
      scheduler.reset();

      try {
        // Visit all algorithms in configured sequence
        Sch::RunSequenceTuple<
          scheduler_t,
          configured_sequence_t,
          std::tuple<const RuntimeOptions&, const Constants&, const HostBuffers&>,
          std::tuple<const RuntimeOptions&, const Constants&, HostBuffers&, hipStream_t&, hipEvent_t&>>::
          run(
            scheduler,
            scheduler.sequence_tuple,
            // Arguments to set_arguments_size
            runtime_options,
            constants,
            *host_buffers,
            // Arguments to visit
            runtime_options,
            constants,
            *host_buffers,
            hip_stream,
            cuda_generic_event);

        // Synchronize CUDA device
        hipEventRecord(cuda_generic_event, hip_stream);
        hipEventSynchronize(cuda_generic_event);
      } catch (const MemoryException& e) {
        warning_cout << "Insufficient memory to process slice - will sub-divide and retry." << std::endl;
        return hipErrorOutOfMemory;
      }
    }
  }

  return hipSuccess;
}

std::vector<bool> Stream::reconstructed_events() const
{
  std::vector<bool> mask(number_of_input_events, false);
  for (uint i = 0; i < host_buffers->host_number_of_selected_events[0]; ++i) {
    mask[host_buffers->host_event_list[i]] = true;
  }
  return mask;
}

void Stream::run_monte_carlo_test(
  CheckerInvoker& invoker,
  MCEvents const& mc_events,
  std::vector<Checker::Tracks> const& forward_tracks)
{
  Sch::RunChecker<
    configured_sequence_t,
    std::tuple<HostBuffers&, const Constants&, const CheckerInvoker&, const MCEvents&>>::
    check(*host_buffers, constants, invoker, mc_events);

  if (forward_tracks.size() > 0) {
    info_cout << "Running test on imported tracks" << std::endl;
    auto& checker = invoker.checker<TrackCheckerForward>("PrCheckerPlots.root");
    checker.accumulate<TrackCheckerForward>(mc_events, forward_tracks);
  }
}
