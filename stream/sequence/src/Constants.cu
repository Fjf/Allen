#include "Constants.cuh"

void Constants::reserve_constants()
{
  cudaCheck(hipMalloc((void**) &dev_velo_module_zs, Velo::Constants::n_modules * sizeof(float)));
  cudaCheck(hipMalloc((void**) &dev_velo_candidate_ks, 9 * sizeof(uint8_t)));
  cudaCheck(hipMalloc((void**) &dev_velo_sp_patterns, 256 * sizeof(uint8_t)));
  cudaCheck(hipMalloc((void**) &dev_velo_sp_fx, 512 * sizeof(float)));
  cudaCheck(hipMalloc((void**) &dev_velo_sp_fy, 512 * sizeof(float)));
  cudaCheck(hipMalloc((void**) &dev_scifi_tmva1, sizeof(SciFi::Tracking::TMVA)));
  cudaCheck(hipMalloc((void**) &dev_scifi_tmva2, sizeof(SciFi::Tracking::TMVA)));
  cudaCheck(hipMalloc((void**) &dev_scifi_constArrays, sizeof(SciFi::Tracking::Arrays)));
  cudaCheck(hipMalloc((void**) &dev_inv_clus_res, host_inv_clus_res.size() * sizeof(float)));
  cudaCheck(hipMalloc((void**) &dev_kalman_params, sizeof(ParKalmanFilter::KalmanParametrizations)));
  cudaCheck(hipMalloc((void**) &dev_looking_forward_constants, sizeof(LookingForward::Constants)));
  cudaCheck(hipMalloc((void**) &dev_muon_foi, sizeof(Muon::Constants::FieldOfInterest)));
  cudaCheck(hipMalloc((void**) &dev_muon_momentum_cuts, 3 * sizeof(float)));
  cudaCheck(hipMalloc((void**) &dev_magnet_polarity, sizeof(float)));
  cudaCheck(hipMalloc((void**) &dev_beamline, 2 * sizeof(float)));
}

void Constants::initialize_constants(
  const std::vector<float>& muon_field_of_interest_params,
  const std::string& folder_params_kalman
) {
  // Magnet polarity
  const float host_magnet_polarity = -1.f;
  cudaCheck(hipMemcpy(
    dev_magnet_polarity, &host_magnet_polarity, sizeof(float), hipMemcpyHostToDevice));

  // PV constants
  const float host_beamline[2] = {0.0f, 0.0f};
  cudaCheck(hipMemcpy(
    dev_beamline, &host_beamline, 2 * sizeof(float), hipMemcpyHostToDevice));

  // Velo module constants
  const std::array<float, Velo::Constants::n_modules> velo_module_zs = {
    -287.5, -275,  -262.5, -250,  -237.5, -225,  -212.5, -200,  -137.5, -125,  -62.5, -50,   -37.5,
    -25,    -12.5, 0,      12.5,  25,     37.5,  50,     62.5,  75,     87.5,  100,   112.5, 125,
    137.5,  150,   162.5,  175,   187.5,  200,   212.5,  225,   237.5,  250,   262.5, 275,   312.5,
    325,    387.5, 400,    487.5, 500,    587.5, 600,    637.5, 650,    687.5, 700,   737.5, 750};
  cudaCheck(hipMemcpy(
    dev_velo_module_zs, velo_module_zs.data(), velo_module_zs.size() * sizeof(float), hipMemcpyHostToDevice));

  // Velo clustering candidate ks
  host_candidate_ks = {0, 0, 1, 4, 4, 5, 5, 5, 5};
  cudaCheck(hipMemcpy(
    dev_velo_candidate_ks,
    host_candidate_ks.data(),
    host_candidate_ks.size() * sizeof(uint8_t),
    hipMemcpyHostToDevice));

  // Velo clustering patterns
  // Fetch patterns and populate in GPU
  std::vector<uint8_t> sp_patterns(256, 0);
  std::vector<uint8_t> sp_sizes(256, 0);
  std::vector<float> sp_fx(512, 0);
  std::vector<float> sp_fy(512, 0);
  cache_sp_patterns(sp_patterns, sp_sizes, sp_fx, sp_fy);

  cudaCheck(hipMemcpy(dev_velo_sp_patterns, sp_patterns.data(), sp_patterns.size(), hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(dev_velo_sp_fx, sp_fx.data(), sp_fx.size() * sizeof(float), hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(dev_velo_sp_fy, sp_fy.data(), sp_fy.size() * sizeof(float), hipMemcpyHostToDevice));

  // SciFi constants
  SciFi::Tracking::TMVA host_tmva1;
  SciFi::Tracking::TMVA host_tmva2;
  SciFi::Tracking::TMVA1_Init(host_tmva1);
  SciFi::Tracking::TMVA2_Init(host_tmva2);
  SciFi::Tracking::Arrays host_constArrays;

  cudaCheck(hipMemcpy(dev_scifi_tmva1, &host_tmva1, sizeof(SciFi::Tracking::TMVA), hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(dev_scifi_tmva2, &host_tmva2, sizeof(SciFi::Tracking::TMVA), hipMemcpyHostToDevice));
  cudaCheck(
    hipMemcpy(dev_scifi_constArrays, &host_constArrays, sizeof(SciFi::Tracking::Arrays), hipMemcpyHostToDevice));
  host_inv_clus_res = {1 / 0.05, 1 / 0.08, 1 / 0.11, 1 / 0.14, 1 / 0.17, 1 / 0.20, 1 / 0.23, 1 / 0.26, 1 / 0.29};
  cudaCheck(hipMemcpy(dev_inv_clus_res, &host_inv_clus_res, host_inv_clus_res.size() * sizeof(float), hipMemcpyHostToDevice));

  // Kalman filter constants.
  ParKalmanFilter::KalmanParametrizations host_kalman_params;
  host_kalman_params.SetParameters(folder_params_kalman, ParKalmanFilter::Polarity::Down);
  cudaCheck(hipMemcpy(
    dev_kalman_params, &host_kalman_params, sizeof(ParKalmanFilter::KalmanParametrizations), hipMemcpyHostToDevice));

  cudaCheck(hipMemcpy(
    dev_looking_forward_constants, &host_looking_forward_constants, sizeof(LookingForward::Constants), hipMemcpyHostToDevice))

  // Muon constants
  Muon::Constants::FieldOfInterest host_muon_foi;
  const float* foi_iterator = muon_field_of_interest_params.data();
  for (int i_station = 0; i_station < Muon::Constants::n_stations; i_station++) {
    std::copy_n(foi_iterator, Muon::Constants::n_regions, host_muon_foi.param_a_x[i_station]);
    foi_iterator += Muon::Constants::n_regions;// * sizeof(float);
    std::copy_n(foi_iterator, Muon::Constants::n_regions, host_muon_foi.param_a_y[i_station]);
    foi_iterator += Muon::Constants::n_regions;// * sizeof(float);
    std::copy_n(foi_iterator, Muon::Constants::n_regions, host_muon_foi.param_b_x[i_station]);
    foi_iterator += Muon::Constants::n_regions;// * sizeof(float);
    std::copy_n(foi_iterator, Muon::Constants::n_regions, host_muon_foi.param_b_y[i_station]);
    foi_iterator += Muon::Constants::n_regions;// * sizeof(float);
    std::copy_n(foi_iterator, Muon::Constants::n_regions, host_muon_foi.param_c_x[i_station]);
    foi_iterator += Muon::Constants::n_regions;// * sizeof(float);
    std::copy_n(foi_iterator, Muon::Constants::n_regions, host_muon_foi.param_c_y[i_station]);
    foi_iterator += Muon::Constants::n_regions;
  }
  cudaCheck(hipMemcpy(dev_muon_momentum_cuts, &Muon::Constants::momentum_cuts, 3 * sizeof(float), hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(dev_muon_foi, &host_muon_foi, sizeof(Muon::Constants::FieldOfInterest), hipMemcpyHostToDevice));
}

void Constants::initialize_muon_catboost_model_constants(
  const int n_trees,
  const std::vector<int>& tree_depths,
  const std::vector<int>& tree_offsets,
  const std::vector<float>& leaf_values,
  const std::vector<int>& leaf_offsets,
  const std::vector<float>& split_borders,
  const std::vector<int>& split_features
) {
  muon_catboost_n_trees = n_trees;
  cudaCheck(hipMalloc((void**) &dev_muon_catboost_split_features, split_features.size() * sizeof(int)));
  cudaCheck(hipMalloc((void**) &dev_muon_catboost_split_borders, split_borders.size() * sizeof(float)));
  cudaCheck(hipMalloc((void**) &dev_muon_catboost_leaf_values, leaf_values.size() * sizeof(float)));
  cudaCheck(hipMalloc((void**) &dev_muon_catboost_tree_depths, tree_depths.size() * sizeof(int)));
  cudaCheck(hipMalloc((void**) &dev_muon_catboost_tree_offsets, tree_offsets.size() * sizeof(int)));
  cudaCheck(hipMalloc((void**) &dev_muon_catboost_leaf_offsets, leaf_offsets.size() * sizeof(int)));

  cudaCheck(hipMemcpy(
    dev_muon_catboost_split_features,
    split_features.data(),
    split_features.size() * sizeof(int),
    hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(
    dev_muon_catboost_split_borders,
    split_borders.data(),
    split_borders.size() * sizeof(float),
    hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(
    dev_muon_catboost_leaf_values, leaf_values.data(), leaf_values.size() * sizeof(float), hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(
    dev_muon_catboost_tree_depths, tree_depths.data(), tree_depths.size() * sizeof(int), hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(
    dev_muon_catboost_tree_offsets, tree_offsets.data(), tree_offsets.size() * sizeof(int), hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(
    dev_muon_catboost_leaf_offsets, leaf_offsets.data(), leaf_offsets.size() * sizeof(int), hipMemcpyHostToDevice));
}
