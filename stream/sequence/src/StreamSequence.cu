#include "hip/hip_runtime.h"
#include "Stream.cuh"

#include <iostream>
#include <fstream>

hipError_t Stream::run_sequence(
  const uint i_stream,
  const char* host_velopix_events,
  const uint* host_velopix_event_offsets,
  const size_t host_velopix_events_size,
  const size_t host_velopix_event_offsets_size,
  const char* host_ut_events,
  const uint* host_ut_event_offsets,
  const size_t host_ut_events_size,
  const size_t host_ut_event_offsets_size,
  VeloUTTracking::HitsSoA *host_ut_hits_events,
  const PrUTMagnetTool* host_ut_magnet_tool,
  const uint number_of_events,
  const uint number_of_repetitions
) {
  // Generate object for populating arguments
  DynamicArgumentGenerator<argument_tuple_t> argen {arguments, dev_base_pointer};

  // Sizes and offsets of arguments
  std::array<size_t, std::tuple_size<argument_tuple_t>::value> argument_sizes;
  std::array<uint, std::tuple_size<argument_tuple_t>::value> argument_offsets;

  for (uint repetition=0; repetition<number_of_repetitions; ++repetition) {
    uint sequence_step = 0;

    // Reset scheduler
    scheduler.reset();

    // Estimate input size
    // Set arguments and reserve memory
    argument_sizes[arg::dev_raw_input] = argen.size<arg::dev_raw_input>(host_velopix_events_size);
    argument_sizes[arg::dev_raw_input_offsets] = argen.size<arg::dev_raw_input_offsets>(host_velopix_event_offsets_size);
    argument_sizes[arg::dev_estimated_input_size] = argen.size<arg::dev_estimated_input_size>(number_of_events * VeloTracking::n_modules + 1);
    argument_sizes[arg::dev_module_cluster_num] = argen.size<arg::dev_module_cluster_num>(number_of_events * VeloTracking::n_modules);
    argument_sizes[arg::dev_module_candidate_num] = argen.size<arg::dev_raw_input_offsets>(number_of_events);
    argument_sizes[arg::dev_cluster_candidates] = argen.size<arg::dev_cluster_candidates>(number_of_events * VeloClustering::max_candidates_event);
    scheduler.setup_next(argument_sizes, argument_offsets, sequence_step++);
    // Setup opts and arguments for kernel call
    sequence.item<seq::estimate_input_size>().set_opts(dim3(number_of_events), dim3(32, 26), stream);
    sequence.item<seq::estimate_input_size>().set_arguments(
      argen.generate<arg::dev_raw_input>(argument_offsets),
      argen.generate<arg::dev_raw_input_offsets>(argument_offsets),
      argen.generate<arg::dev_estimated_input_size>(argument_offsets),
      argen.generate<arg::dev_module_cluster_num>(argument_offsets),
      argen.generate<arg::dev_module_candidate_num>(argument_offsets),
      argen.generate<arg::dev_cluster_candidates>(argument_offsets),
      gpu_constants.dev_velo_candidate_ks
    );
    cudaCheck(hipMemcpyAsync(argen.generate<arg::dev_raw_input>(argument_offsets), host_velopix_events, host_velopix_events_size, hipMemcpyHostToDevice, stream));
    cudaCheck(hipMemcpyAsync(argen.generate<arg::dev_raw_input_offsets>(argument_offsets), host_velopix_event_offsets, host_velopix_event_offsets_size * sizeof(uint), hipMemcpyHostToDevice, stream));
    hipEventRecord(cuda_generic_event, stream);
    hipEventSynchronize(cuda_generic_event);

    // Kernel call
    sequence.item<seq::estimate_input_size>().invoke();

    // Convert the estimated sizes to module hit start format (argument_offsets)
    // Set arguments and reserve memory
    argument_sizes[arg::dev_cluster_offset] = argen.size<arg::dev_cluster_offset>(number_of_events);
    scheduler.setup_next(argument_sizes, argument_offsets, sequence_step++);
    // Setup sequence step
    const auto prefix_sum_blocks = (VeloTracking::n_modules * number_of_events + 511) / 512;
    sequence.item<seq::prefix_sum_reduce>().set_opts(dim3(prefix_sum_blocks), dim3(256), stream);
    sequence.item<seq::prefix_sum_reduce>().set_arguments(
      argen.generate<arg::dev_estimated_input_size>(argument_offsets),
      argen.generate<arg::dev_cluster_offset>(argument_offsets),
      VeloTracking::n_modules * number_of_events
    );
    // Kernel call
    sequence.item<seq::prefix_sum_reduce>().invoke();

    // Prefix Sum Single Block
    scheduler.setup_next(argument_sizes, argument_offsets, sequence_step++);
    sequence.item<seq::prefix_sum_single_block>().set_arguments(
      argen.generate<arg::dev_estimated_input_size>(argument_offsets) + VeloTracking::n_modules * number_of_events,
      argen.generate<arg::dev_cluster_offset>(argument_offsets),
      prefix_sum_blocks
    );
    sequence.item<seq::prefix_sum_single_block>().invoke();

    // Prefix sum scan
    scheduler.setup_next(argument_sizes, argument_offsets, sequence_step++);
    const auto prefix_sum_scan_blocks = prefix_sum_blocks==1 ? 1 : (prefix_sum_blocks-1);
    sequence.item<seq::prefix_sum_scan>().set_opts(dim3(prefix_sum_scan_blocks), dim3(512), stream);
    sequence.item<seq::prefix_sum_scan>().set_arguments(
      argen.generate<arg::dev_estimated_input_size>(argument_offsets),
      argen.generate<arg::dev_cluster_offset>(argument_offsets),
      VeloTracking::n_modules * number_of_events
    );
    sequence.item<seq::prefix_sum_scan>().invoke();

    // Fetch the number of hits we require
    cudaCheck(hipMemcpyAsync(host_total_number_of_velo_clusters, argen.generate<arg::dev_estimated_input_size>(argument_offsets) + number_of_events * VeloTracking::n_modules, sizeof(uint), hipMemcpyDeviceToHost, stream));
    hipEventRecord(cuda_generic_event, stream);
    hipEventSynchronize(cuda_generic_event);

    // Masked Velo clustering
    argument_sizes[arg::dev_velo_cluster_container] = argen.size<arg::dev_velo_cluster_container>(6 * host_total_number_of_velo_clusters[0]);
    scheduler.setup_next(argument_sizes, argument_offsets, sequence_step++);
    sequence.item<seq::masked_velo_clustering>().set_opts(dim3(number_of_events), dim3(256), stream);
    sequence.item<seq::masked_velo_clustering>().set_arguments(
      argen.generate<arg::dev_raw_input>(argument_offsets),
      argen.generate<arg::dev_raw_input_offsets>(argument_offsets),
      argen.generate<arg::dev_estimated_input_size>(argument_offsets),
      argen.generate<arg::dev_module_cluster_num>(argument_offsets),
      argen.generate<arg::dev_module_candidate_num>(argument_offsets),
      argen.generate<arg::dev_cluster_candidates>(argument_offsets),
      argen.generate<arg::dev_velo_cluster_container>(argument_offsets),
      dev_velo_geometry,
      gpu_constants.dev_velo_sp_patterns,
      gpu_constants.dev_velo_sp_fx,
      gpu_constants.dev_velo_sp_fy
    );
    sequence.item<seq::masked_velo_clustering>().invoke();

    // Calculate phi and sort
    argument_sizes[arg::dev_hit_permutation] = argen.size<arg::dev_hit_permutation>(host_total_number_of_velo_clusters[0]);
    scheduler.setup_next(argument_sizes, argument_offsets, sequence_step++);
    sequence.item<seq::calculate_phi_and_sort>().set_opts(dim3(number_of_events), dim3(64), stream);
    sequence.item<seq::calculate_phi_and_sort>().set_arguments(
      argen.generate<arg::dev_estimated_input_size>(argument_offsets),
      argen.generate<arg::dev_module_cluster_num>(argument_offsets),
      argen.generate<arg::dev_velo_cluster_container>(argument_offsets),
      argen.generate<arg::dev_hit_permutation>(argument_offsets)
    );
    sequence.item<seq::calculate_phi_and_sort>().invoke();

    // Fill candidates
    argument_sizes[arg::dev_h0_candidates] = argen.size<arg::dev_h0_candidates>(2 * host_total_number_of_velo_clusters[0]);
    argument_sizes[arg::dev_h2_candidates] = argen.size<arg::dev_h2_candidates>(2 * host_total_number_of_velo_clusters[0]);
    scheduler.setup_next(argument_sizes, argument_offsets, sequence_step++);
    // Setup opts and arguments
    sequence.item<seq::fill_candidates>().set_opts(dim3(number_of_events, 48), dim3(128), stream);
    sequence.item<seq::fill_candidates>().set_arguments(
      argen.generate<arg::dev_velo_cluster_container>(argument_offsets),
      argen.generate<arg::dev_estimated_input_size>(argument_offsets),
      argen.generate<arg::dev_module_cluster_num>(argument_offsets),
      argen.generate<arg::dev_h0_candidates>(argument_offsets),
      argen.generate<arg::dev_h2_candidates>(argument_offsets)
    );
    sequence.item<seq::fill_candidates>().invoke();

    // Search by triplet
    argument_sizes[arg::dev_tracks] = argen.size<arg::dev_tracks>(number_of_events * VeloTracking::max_tracks);
    argument_sizes[arg::dev_tracklets] = argen.size<arg::dev_tracklets>(number_of_events * VeloTracking::ttf_modulo);
    argument_sizes[arg::dev_tracks_to_follow] = argen.size<arg::dev_tracks_to_follow>(number_of_events * VeloTracking::ttf_modulo);
    argument_sizes[arg::dev_weak_tracks] = argen.size<arg::dev_weak_tracks>(number_of_events * VeloTracking::max_weak_tracks);
    argument_sizes[arg::dev_hit_used] = argen.size<arg::dev_hit_used>(host_total_number_of_velo_clusters[0]);
    argument_sizes[arg::dev_atomics_storage] = argen.size<arg::dev_atomics_storage>(number_of_events * VeloTracking::num_atomics);
    argument_sizes[arg::dev_rel_indices] = argen.size<arg::dev_rel_indices>(number_of_events * 2 * VeloTracking::max_numhits_in_module);
    scheduler.setup_next(argument_sizes, argument_offsets, sequence_step++);
    // Setup opts and arguments
    sequence.item<seq::search_by_triplet>().set_opts(dim3(number_of_events), dim3(32), stream, 32 * sizeof(float));
    sequence.item<seq::search_by_triplet>().set_arguments(
      argen.generate<arg::dev_velo_cluster_container>(argument_offsets),
      argen.generate<arg::dev_estimated_input_size>(argument_offsets),
      argen.generate<arg::dev_module_cluster_num>(argument_offsets),
      argen.generate<arg::dev_tracks>(argument_offsets),
      argen.generate<arg::dev_tracklets>(argument_offsets),
      argen.generate<arg::dev_tracks_to_follow>(argument_offsets),
      argen.generate<arg::dev_weak_tracks>(argument_offsets),
      argen.generate<arg::dev_hit_used>(argument_offsets),
      argen.generate<arg::dev_atomics_storage>(argument_offsets),
      argen.generate<arg::dev_h0_candidates>(argument_offsets),
      argen.generate<arg::dev_h2_candidates>(argument_offsets),
      argen.generate<arg::dev_rel_indices>(argument_offsets)
    );
    sequence.item<seq::search_by_triplet>().invoke();

    // Weak tracks adder
    scheduler.setup_next(argument_sizes, argument_offsets, sequence_step++);
    // Setup opts and arguments
    sequence.item<seq::weak_tracks_adder>().set_opts(dim3(number_of_events), dim3(32), stream);
    sequence.item<seq::weak_tracks_adder>().set_arguments(
      argen.generate<arg::dev_velo_cluster_container>(argument_offsets),
      argen.generate<arg::dev_estimated_input_size>(argument_offsets),
      argen.generate<arg::dev_tracks>(argument_offsets),
      argen.generate<arg::dev_weak_tracks>(argument_offsets),
      argen.generate<arg::dev_hit_used>(argument_offsets),
      argen.generate<arg::dev_atomics_storage>(argument_offsets)
    );
    sequence.item<seq::weak_tracks_adder>().invoke();
    
    // Calculate prefix sum of found tracks
    scheduler.setup_next(argument_sizes, argument_offsets, sequence_step++);
    sequence.item<seq::copy_and_prefix_sum_single_block>().set_arguments(
      (uint*) argen.generate<arg::dev_atomics_storage>(argument_offsets) + number_of_events*2,
      (uint*) argen.generate<arg::dev_atomics_storage>(argument_offsets),
      (uint*) argen.generate<arg::dev_atomics_storage>(argument_offsets) + number_of_events,
      number_of_events
    );
    sequence.item<seq::copy_and_prefix_sum_single_block>().invoke();

    // Fetch number of reconstructed tracks
    cudaCheck(hipMemcpyAsync(host_number_of_reconstructed_velo_tracks, argen.generate<arg::dev_atomics_storage>(argument_offsets) + number_of_events * 2, sizeof(uint), hipMemcpyDeviceToHost, stream));
    hipEventRecord(cuda_generic_event, stream);
    hipEventSynchronize(cuda_generic_event);
    size_t velo_track_hit_number_size = host_number_of_reconstructed_velo_tracks[0] + 1;

    // Prefix sum of tracks hits
    // 1. Copy velo track hit number to a consecutive container
    // 2. Reduce
    // 3. Single block
    // 4. Scan

    // Copy Velo track hit number
    argument_sizes[arg::dev_velo_track_hit_number] = argen.size<arg::dev_velo_track_hit_number>(velo_track_hit_number_size);
    scheduler.setup_next(argument_sizes, argument_offsets, sequence_step++);
    sequence.item<seq::copy_velo_track_hit_number>().set_opts(dim3(number_of_events), dim3(512), stream);
    sequence.item<seq::copy_velo_track_hit_number>().set_arguments(
      argen.generate<arg::dev_tracks>(argument_offsets),
      argen.generate<arg::dev_atomics_storage>(argument_offsets),
      argen.generate<arg::dev_velo_track_hit_number>(argument_offsets)
    );
    sequence.item<seq::copy_velo_track_hit_number>().invoke();

    // Prefix sum: Reduce
    const size_t prefix_sum_auxiliary_array_2_size = (host_number_of_reconstructed_velo_tracks[0] + 511) / 512;
    argument_sizes[arg::dev_prefix_sum_auxiliary_array_2] = argen.size<arg::dev_prefix_sum_auxiliary_array_2>(prefix_sum_auxiliary_array_2_size);
    scheduler.setup_next(argument_sizes, argument_offsets, sequence_step++);
    sequence.item<seq::prefix_sum_reduce_velo_track_hit_number>().set_opts(dim3(prefix_sum_auxiliary_array_2_size), dim3(256), stream);
    sequence.item<seq::prefix_sum_reduce_velo_track_hit_number>().set_arguments(
      argen.generate<arg::dev_velo_track_hit_number>(argument_offsets),
      argen.generate<arg::dev_prefix_sum_auxiliary_array_2>(argument_offsets),
      host_number_of_reconstructed_velo_tracks[0]
    );
    sequence.item<seq::prefix_sum_reduce_velo_track_hit_number>().invoke();

    // Prefix sum: Single block
    scheduler.setup_next(argument_sizes, argument_offsets, sequence_step++);
    sequence.item<seq::prefix_sum_single_block_velo_track_hit_number>().set_arguments(
      argen.generate<arg::dev_velo_track_hit_number>(argument_offsets) + host_number_of_reconstructed_velo_tracks[0],
      argen.generate<arg::dev_prefix_sum_auxiliary_array_2>(argument_offsets),
      prefix_sum_auxiliary_array_2_size
    );
    sequence.item<seq::prefix_sum_single_block_velo_track_hit_number>().invoke();

    // Prefix sum: Scan
    scheduler.setup_next(argument_sizes, argument_offsets, sequence_step++);
    const uint pss_velo_track_hit_number_opts =
      prefix_sum_auxiliary_array_2_size==1 ? 1 : (prefix_sum_auxiliary_array_2_size-1);
    sequence.item<seq::prefix_sum_scan_velo_track_hit_number>().set_opts(dim3(pss_velo_track_hit_number_opts), dim3(512), stream);
    sequence.item<seq::prefix_sum_scan_velo_track_hit_number>().set_arguments(
      argen.generate<arg::dev_velo_track_hit_number>(argument_offsets),
      argen.generate<arg::dev_prefix_sum_auxiliary_array_2>(argument_offsets),
      host_number_of_reconstructed_velo_tracks[0]
    );
    sequence.item<seq::prefix_sum_scan_velo_track_hit_number>().invoke();

    // Fetch total number of hits accumulated with all tracks
    cudaCheck(hipMemcpyAsync(host_accumulated_number_of_hits_in_velo_tracks,
      argen.generate<arg::dev_velo_track_hit_number>(argument_offsets) + host_number_of_reconstructed_velo_tracks[0],
      sizeof(uint), hipMemcpyDeviceToHost, stream));
    hipEventRecord(cuda_generic_event, stream);
    hipEventSynchronize(cuda_generic_event);

    // Consolidate tracks
    argument_sizes[arg::dev_velo_track_hits] = argen.size<arg::dev_velo_track_hits>(host_accumulated_number_of_hits_in_velo_tracks[0]);
    argument_sizes[arg::dev_velo_states] = argen.size<arg::dev_velo_states>(host_number_of_reconstructed_velo_tracks[0]);
    scheduler.setup_next(argument_sizes, argument_offsets, sequence_step++);
    sequence.item<seq::consolidate_tracks>().set_opts(dim3(number_of_events), dim3(32), stream);
    sequence.item<seq::consolidate_tracks>().set_arguments(
      argen.generate<arg::dev_atomics_storage>(argument_offsets),
      argen.generate<arg::dev_tracks>(argument_offsets),
      argen.generate<arg::dev_velo_track_hit_number>(argument_offsets),
      argen.generate<arg::dev_velo_cluster_container>(argument_offsets),
      argen.generate<arg::dev_estimated_input_size>(argument_offsets),
      argen.generate<arg::dev_module_cluster_num>(argument_offsets),
      argen.generate<arg::dev_velo_track_hits>(argument_offsets),
      argen.generate<arg::dev_velo_states>(argument_offsets)
    );
    sequence.item<seq::consolidate_tracks>().invoke();

    // Estimate number of UT hits
    // Set arguments and reserve memory
    argument_sizes[arg::dev_ut_raw_input] = argen.size<arg::dev_ut_raw_input>(host_ut_events_size);
    argument_sizes[arg::dev_ut_raw_input_offsets] = argen.size<arg::dev_ut_raw_input_offsets>(host_ut_event_offsets_size);
    argument_sizes[arg::dev_ut_hit_count] = argen.size<arg::dev_ut_hit_count>(2 * number_of_events * VeloUTTracking::n_layers + 1);
    scheduler.setup_next(argument_sizes, argument_offsets, sequence_step++);
    // Setup opts and arguments for kernel call
    cudaCheck(hipMemcpyAsync(argen.generate<arg::dev_ut_raw_input>(argument_offsets), host_ut_events, host_ut_events_size, hipMemcpyHostToDevice, stream));
    cudaCheck(hipMemcpyAsync(argen.generate<arg::dev_ut_raw_input_offsets>(argument_offsets), host_ut_event_offsets, host_ut_event_offsets_size * sizeof(uint32_t), hipMemcpyHostToDevice, stream));
    hipEventRecord(cuda_generic_event, stream);
    hipEventSynchronize(cuda_generic_event);
    sequence.item<seq::ut_estimate_number_of_hits>().set_opts(dim3(number_of_events), dim3(192), stream);
    sequence.item<seq::ut_estimate_number_of_hits>().set_arguments(
      argen.generate<arg::dev_ut_raw_input>(argument_offsets),
      argen.generate<arg::dev_ut_raw_input_offsets>(argument_offsets),
      dev_ut_boards,
      argen.generate<arg::dev_ut_hit_count>(argument_offsets)
    );
    // Invoke kernel
    sequence.item<seq::ut_estimate_number_of_hits>().invoke();

    // Fetch UT hit count
    cudaCheck(hipMemcpyAsync(host_ut_hit_count, argen.generate<arg::dev_ut_hit_count>(argument_offsets), argen.size<arg::dev_ut_hit_count>(2 * number_of_events * VeloUTTracking::n_layers + 1), hipMemcpyDeviceToHost, stream));
    hipEventRecord(cuda_generic_event, stream);
    hipEventSynchronize(cuda_generic_event);

    for (int e=0; e<number_of_events; ++e) {
      info_cout << "Event " << e << " (#hit)" << std::endl;
      uint32_t* count = host_ut_hit_count + e * VeloUTTracking::n_layers;
      for (uint32_t i = 0; i < 4; ++i) {
        info_cout << " layer " << i << ": " << count[i] << std::endl;
      }
      info_cout << std::endl;
    }

    // Reserve hit buffer
    

    // sequence.item<seq::decode_raw_banks>().set_opts(dim3(number_of_events), dim3(192), stream);
    // sequence.item<seq::decode_raw_banks>().set_arguments(
    //   argen.generate<arg::dev_ut_raw_input>(argument_offsets),
    //   argen.generate<arg::dev_ut_raw_input_offsets>(argument_offsets),
    //   dev_ut_boards,
    //   dev_ut_geometry,
    //   argen.generate<arg::dev_ut_hits_decoded>(argument_offsets),
    //   argen.generate<arg::dev_ut_hit_count>(argument_offsets)
    // );
    // sequence.item<seq::decode_raw_banks>().invoke();

    // cudaCheck(hipMemcpyAsync(
    //   host_ut_hits_decoded,
    //   argen.generate<arg::dev_ut_hits_decoded>(argument_offsets),
    //   argen.size<arg::dev_ut_hits_decoded>(number_of_events),
    //   hipMemcpyDeviceToHost,
    //   stream
    // ));

    // // Wait to receive the result
    // hipEventRecord(cuda_generic_event, stream);
    // hipEventSynchronize(cuda_generic_event);

    // for (uint32_t ut_event_number = 0; ut_event_number < number_of_events; ++ut_event_number) {
    //   std::cout << "UT event number " << ut_event_number << std::endl;
      
    //   std::vector<UTHit> hits_vector;

    //   for (uint32_t hit_layer = 0; hit_layer < ut_number_of_layers; ++hit_layer) {
    //     const UTHits & hits_event = host_ut_hits_decoded[ut_event_number];
    //     for (uint32_t hit_number = 0; hit_number < hits_event.n_hits_layers[hit_layer]; ++hit_number) {
    //       UTHit hit = hits_event.getHit(hit_number, hit_layer);

    //       if (hit.LHCbID == 19733777) {
    //         info_cout << "LHCb ID, hit number, hit layer: "
    //           << hit.LHCbID << ", " << hit_number << ", " << hit_layer << std::endl;
    //       }

    //       hits_vector.push_back(hit);
    //     }
    //   }

    //   // sort(hits_vector.begin(), hits_vector.end(), [](const UTHit & a, const UTHit & b) -> bool {
    //   //     return a.LHCbID > b.LHCbID; 
    //   // });



    //   std::vector<UTHit> hits_compare;
    //   const std::string fileName = "../input/minbias/ut_hits_compare/" + std::to_string(ut_event_number) + ".bin";
    //   std::ifstream in_hits(fileName.c_str(), std::ios::in | std::ios::binary);

    //   if (!in_hits) {
    //     std::cout << "Error while loading file: " << fileName << std::endl;
    //     continue;
    //   }

    //   uint32_t number_of_hits_compare = 0;
    //   in_hits.read((char *) &(number_of_hits_compare), sizeof(float));

    //   for (uint32_t i = 0; i < number_of_hits_compare; ++i) {

    //     UTHit hit;
    //     float ut_dxDy; // Unused
    //     in_hits.read((char *) &(hit.cos           ), sizeof(float));
    //     in_hits.read((char *) &(hit.yBegin        ), sizeof(float));
    //     in_hits.read((char *) &(hit.yEnd          ), sizeof(float));
    //     in_hits.read((char *) &(ut_dxDy           ), sizeof(float));
    //     in_hits.read((char *) &(hit.zAtYEq0       ), sizeof(float));
    //     in_hits.read((char *) &(hit.xAtYEq0       ), sizeof(float));
    //     in_hits.read((char *) &(hit.weight        ), sizeof(float));
    //     in_hits.read((char *) &(hit.highThreshold ), sizeof(float));
    //     in_hits.read((char *) &(hit.LHCbID        ), sizeof(float));

    //     hits_compare.push_back(hit);
    //   }

    //   in_hits.close();

    //   // sort(hits_compare.begin(), hits_compare.end(), [](const UTHit & a, const UTHit & b) -> bool {
    //   //     return a.LHCbID > b.LHCbID; 
    //   // });

    //   info_cout << " Expected " << hits_compare.size() << " hits" << std::endl
    //     << " Found " << hits_vector.size() << " hits" << std::endl;

    //   for (auto hit : hits_compare) {
    //     if (std::find(hits_vector.begin(), hits_vector.end(), hit) == std::end(hits_vector)) {
    //       error_cout << "hit " << hit << " only in hits_compare" << std::endl;
    //     }

    //     const auto count_instances = std::count(hits_compare.begin(), hits_compare.end(), hit);
    //     if (count_instances > 1) {
    //       info_cout << "Hit " << hit << " found " << count_instances << " times in hits_compare" << std::endl;
    //     }
    //   }

    //   for (auto hit : hits_vector) {
    //     if (std::find(hits_compare.begin(), hits_compare.end(), hit) == std::end(hits_compare)) {
    //       error_cout << "hit " << hit << " only in hits_vector" << std::endl;
    //     }

    //     const auto count_instances = std::count(hits_vector.begin(), hits_vector.end(), hit);
    //     if (count_instances > 1) {
    //       info_cout << "Hit " << hit << " found " << count_instances << " times in hits_vector" << std::endl;
    //     }
    //   }
    // }
    // // Check the output
    // info_cout << "decode_raw_banks finished" << std::endl << std::endl;
    
    // // UT hit sorting by x
    // argument_sizes[arg::dev_ut_hits] = argen.size<arg::dev_ut_hits>(number_of_events);
    // argument_sizes[arg::dev_ut_hits_sorted] = argen.size<arg::dev_ut_hits_sorted>(number_of_events);
    // argument_sizes[arg::dev_ut_hit_permutations] = argen.size<arg::dev_ut_hit_permutations>(number_of_events * VeloUTTracking::max_numhits_per_event);
    // scheduler.setup_next(argument_sizes, argument_offsets, sequence_step++);
    // cudaCheck(hipMemcpyAsync(argen.generate<arg::dev_ut_hits>(argument_offsets), host_ut_hits_events, argen.size<arg::dev_ut_hits>(number_of_events), hipMemcpyHostToDevice, stream ));
    // sequence.item<seq::sort_by_x>().set_opts(dim3(number_of_events), dim3(32), stream);
    // sequence.item<seq::sort_by_x>().set_arguments(
    //   argen.generate<arg::dev_ut_hits>(argument_offsets),
    //   argen.generate<arg::dev_ut_hits_sorted>(argument_offsets),
    //   argen.generate<arg::dev_ut_hit_permutations>(argument_offsets) );
    // sequence.item<seq::sort_by_x>().invoke();
    
    // // VeloUT tracking
    // argument_sizes[arg::dev_veloUT_tracks] = argen.size<arg::dev_veloUT_tracks>(number_of_events*VeloUTTracking::max_num_tracks);
    // argument_sizes[arg::dev_atomics_veloUT] = argen.size<arg::dev_atomics_veloUT>(VeloUTTracking::num_atomics*number_of_events);
    // scheduler.setup_next(argument_sizes, argument_offsets, sequence_step++);
    // sequence.item<seq::veloUT>().set_opts(dim3(number_of_events), dim3(32), stream);
    // sequence.item<seq::veloUT>().set_arguments(
    //   argen.generate<arg::dev_ut_hits_sorted>(argument_offsets),
    //   argen.generate<arg::dev_atomics_storage>(argument_offsets),
    //   argen.generate<arg::dev_velo_track_hit_number>(argument_offsets),
    //   argen.generate<arg::dev_velo_track_hits>(argument_offsets),
    //   argen.generate<arg::dev_velo_states>(argument_offsets),
    //   argen.generate<arg::dev_veloUT_tracks>(argument_offsets),
    //   argen.generate<arg::dev_atomics_veloUT>(argument_offsets),
    //   dev_ut_magnet_tool );
    // sequence.item<seq::veloUT>().invoke();

    // // Transmission device to host
    // if (transmit_device_to_host) {
    //   // Velo tracks
    //   cudaCheck(hipMemcpyAsync(host_number_of_tracks, argen.generate<arg::dev_atomics_storage>(argument_offsets), number_of_events * sizeof(int), hipMemcpyDeviceToHost, stream));
    //   cudaCheck(hipMemcpyAsync(host_accumulated_tracks, argen.generate<arg::dev_atomics_storage>(argument_offsets) + number_of_events, number_of_events * sizeof(int), hipMemcpyDeviceToHost, stream));
    //   cudaCheck(hipMemcpyAsync(host_velo_track_hit_number, argen.generate<arg::dev_velo_track_hit_number>(argument_offsets), argen.size<arg::dev_velo_track_hit_number>(velo_track_hit_number_size), hipMemcpyDeviceToHost, stream));
    //   cudaCheck(hipMemcpyAsync(host_velo_track_hits, argen.generate<arg::dev_velo_track_hits>(argument_offsets), argen.size<arg::dev_velo_track_hits>(host_accumulated_number_of_hits_in_velo_tracks[0]), hipMemcpyDeviceToHost, stream));
    //   cudaCheck(hipMemcpyAsync(host_velo_states, argen.generate<arg::dev_velo_states>(argument_offsets), argen.size<arg::dev_velo_states>(host_number_of_reconstructed_velo_tracks[0]), hipMemcpyDeviceToHost, stream)); 

    //   // VeloUT tracks
    //   cudaCheck(hipMemcpyAsync(host_atomics_veloUT, argen.generate<arg::dev_atomics_veloUT>(argument_offsets), argen.size<arg::dev_atomics_veloUT>(VeloUTTracking::num_atomics*number_of_events), hipMemcpyDeviceToHost, stream));
    //   cudaCheck(hipMemcpyAsync(host_veloUT_tracks, argen.generate<arg::dev_veloUT_tracks>(argument_offsets), argen.size<arg::dev_veloUT_tracks>(number_of_events*VeloUTTracking::max_num_tracks), hipMemcpyDeviceToHost, stream));
    // }

    // hipEventRecord(cuda_generic_event, stream);
    // hipEventSynchronize(cuda_generic_event);

    ///////////////////////
    // Monte Carlo Check //
    ///////////////////////
    
    if (do_check && i_stream == 0) {
      if (repetition == 0) { // only check efficiencies once

        /* CHECKING Velo TRACKS */
        if ( !transmit_device_to_host ) { // Fetch data
          cudaCheck(hipMemcpyAsync(host_number_of_tracks, argen.generate<arg::dev_atomics_storage>(argument_offsets), number_of_events * sizeof(int), hipMemcpyDeviceToHost, stream));
          cudaCheck(hipMemcpyAsync(host_accumulated_tracks, argen.generate<arg::dev_atomics_storage>(argument_offsets) + number_of_events, number_of_events * sizeof(int), hipMemcpyDeviceToHost, stream));
          cudaCheck(hipMemcpyAsync(host_velo_track_hit_number, argen.generate<arg::dev_velo_track_hit_number>(argument_offsets), argen.size<arg::dev_velo_track_hit_number>(velo_track_hit_number_size), hipMemcpyDeviceToHost, stream));
          cudaCheck(hipMemcpyAsync(host_velo_track_hits, argen.generate<arg::dev_velo_track_hits>(argument_offsets), argen.size<arg::dev_velo_track_hits>(host_accumulated_number_of_hits_in_velo_tracks[0]), hipMemcpyDeviceToHost, stream));
          cudaCheck(hipMemcpyAsync(host_velo_states, argen.generate<arg::dev_velo_states>(argument_offsets), argen.size<arg::dev_velo_states>(host_number_of_reconstructed_velo_tracks[0]), hipMemcpyDeviceToHost, stream)); 
          hipEventRecord(cuda_generic_event, stream);
          hipEventSynchronize(cuda_generic_event);
        }

  std::cout << "CHECKING VELO TRACKS " << std::endl; 
  
        const std::vector< trackChecker::Tracks > tracks_events = prepareTracks(
          host_velo_track_hit_number,
          reinterpret_cast<VeloTracking::Hit<true>*>(host_velo_track_hits),
          host_accumulated_tracks,
          host_number_of_tracks,
          number_of_events);
      
        std::string trackType = "Velo";
        call_pr_checker(
          tracks_events,
          folder_name_MC,
          start_event_offset,
          trackType
        );
      
        /* CHECKING VeloUT TRACKS */
        if ( !transmit_device_to_host ) { // Fetch data
          cudaCheck(hipMemcpyAsync(host_atomics_veloUT, argen.generate<arg::dev_atomics_veloUT>(argument_offsets), argen.size<arg::dev_atomics_veloUT>(VeloUTTracking::num_atomics*number_of_events), hipMemcpyDeviceToHost, stream));
          cudaCheck(hipMemcpyAsync(host_veloUT_tracks, argen.generate<arg::dev_veloUT_tracks>(argument_offsets), argen.size<arg::dev_veloUT_tracks>(number_of_events*VeloUTTracking::max_num_tracks), hipMemcpyDeviceToHost, stream));
        }
      
        const std::vector< trackChecker::Tracks > veloUT_tracks = prepareVeloUTTracks(
          host_veloUT_tracks,
          host_atomics_veloUT,
          number_of_events
        );  
      
        std::cout << "CHECKING VeloUT TRACKS from GPU" << std::endl;
        trackType = "VeloUT";
        call_pr_checker (
          veloUT_tracks,
          folder_name_MC,
          start_event_offset,
          trackType
        );
      
        /* Run VeloUT on x86 architecture */
        if ( run_on_x86 ) {
          std::vector<trackChecker::Tracks> ut_tracks_events;
        
          int rv = run_veloUT_on_CPU(
                     ut_tracks_events,
                     host_ut_hits_events,
                     host_ut_magnet_tool,
                     host_velo_states,
                     host_accumulated_tracks,
                     host_velo_track_hit_number,
                     host_velo_track_hits,
                     host_number_of_tracks,
                     number_of_events
                    );

          if ( rv != 0 )
            continue;
          
          std::cout << "CHECKING VeloUT TRACKS from x86" << std::endl;
          trackType = "VeloUT";
          call_pr_checker (
            ut_tracks_events,
            folder_name_MC,
            start_event_offset,
            trackType);
        }
      } // only in first repitition
    } // mc_check_enabled
  } // repetitions

  return hipSuccess;
}
