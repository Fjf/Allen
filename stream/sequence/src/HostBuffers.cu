#include "HostBuffers.cuh"
#include "SciFiDefinitions.cuh"
#include "BeamlinePVConstants.cuh"

void HostBuffers::reserve(const uint max_number_of_events, const bool do_check)
{
  // Datatypes needed to run, regardless of checking
  // Note: These datatypes must be pinned to allow for asynchronicity
  cudaCheck(hipHostMalloc((void**) &host_number_of_selected_events, sizeof(uint)));
  cudaCheck(hipHostMalloc((void**) &host_total_number_of_velo_clusters, sizeof(uint)));
  cudaCheck(hipHostMalloc((void**) &host_number_of_reconstructed_velo_tracks, sizeof(uint)));
  cudaCheck(hipHostMalloc((void**) &host_accumulated_number_of_hits_in_velo_tracks, sizeof(uint)));
  cudaCheck(hipHostMalloc((void**) &host_accumulated_number_of_ut_hits, sizeof(uint)));
  cudaCheck(hipHostMalloc((void**) &host_number_of_reconstructed_ut_tracks, sizeof(uint)));
  cudaCheck(hipHostMalloc((void**) &host_accumulated_number_of_hits_in_ut_tracks, sizeof(uint)));
  cudaCheck(hipHostMalloc((void**) &host_accumulated_number_of_scifi_hits, sizeof(uint)));
  cudaCheck(hipHostMalloc((void**) &host_number_of_reconstructed_scifi_tracks, sizeof(uint)));
  cudaCheck(hipHostMalloc((void**) &host_accumulated_number_of_hits_in_scifi_tracks, sizeof(uint)));
  cudaCheck(hipHostMalloc((void**) &host_lf_total_number_of_candidates, sizeof(uint)));
  cudaCheck(hipHostMalloc((void**) &host_lf_total_size_first_window_layer, sizeof(uint)));
  cudaCheck(hipHostMalloc((void**) &host_number_of_svs, sizeof(uint)));

  // Buffer for performing GEC on CPU
  cudaCheck(hipHostMalloc((void**) &host_event_list, max_number_of_events * sizeof(uint)));

  // Buffer for performing prefix sum
  // Note: If it is of insufficient space, it will get reallocated
  host_allocated_prefix_sum_space = 10000000;
  cudaCheck(hipHostMalloc((void**) &host_prefix_sum_buffer, host_allocated_prefix_sum_space * sizeof(uint)));

  if (do_check) {
    // Datatypes to be reserved only if checking is on
    // Note: These datatypes in principle do not require to be pinned
    cudaCheck(hipHostMalloc((void**) &host_atomics_velo, (2 * max_number_of_events + 1) * sizeof(int)));
    cudaCheck(hipHostMalloc(
      (void**) &host_velo_track_hit_number, max_number_of_events * Velo::Constants::max_tracks * sizeof(uint)));
    cudaCheck(hipHostMalloc(
      (void**) &host_velo_track_hits,
      max_number_of_events * Velo::Constants::max_tracks * Velo::Constants::max_track_size * sizeof(Velo::Hit)));
    cudaCheck(hipHostMalloc(
      (void**) &host_kalmanvelo_states, max_number_of_events * Velo::Constants::max_tracks * sizeof(VeloState)));

    cudaCheck(hipHostMalloc((void**) &host_atomics_ut, UT::num_atomics * max_number_of_events * sizeof(int)));
    cudaCheck(hipHostMalloc(
      (void**) &host_ut_tracks, max_number_of_events * UT::Constants::max_num_tracks * sizeof(UT::TrackHits)));

    cudaCheck(hipHostMalloc(
      (void**) &host_ut_track_hit_number, max_number_of_events * UT::Constants::max_num_tracks * sizeof(uint)));
    cudaCheck(hipHostMalloc(
      (void**) &host_ut_track_hits,
      max_number_of_events * UT::Constants::max_num_tracks * UT::Constants::max_track_size * sizeof(UT::Hit)));
    cudaCheck(
      hipHostMalloc((void**) &host_ut_qop, max_number_of_events * UT::Constants::max_num_tracks * sizeof(float)));
    cudaCheck(hipHostMalloc((void**) &host_ut_x, max_number_of_events * UT::Constants::max_num_tracks * sizeof(float)));
    cudaCheck(hipHostMalloc((void**) &host_ut_tx, max_number_of_events * UT::Constants::max_num_tracks * sizeof(float)));
    cudaCheck(hipHostMalloc((void**) &host_ut_z, max_number_of_events * UT::Constants::max_num_tracks * sizeof(float)));
    cudaCheck(hipHostMalloc(
      (void**) &host_ut_track_velo_indices, max_number_of_events * UT::Constants::max_num_tracks * sizeof(int)));

    cudaCheck(hipHostMalloc(
      (void**) &host_scifi_tracks, max_number_of_events * UT::Constants::max_num_tracks * SciFi::Constants::max_SciFi_tracks_per_UT_track * sizeof(SciFi::TrackHits)));
    cudaCheck(hipHostMalloc((void**) &host_atomics_scifi, max_number_of_events * SciFi::num_atomics * sizeof(int)));

    cudaCheck(hipHostMalloc(
      (void**) &host_scifi_track_hit_number, max_number_of_events * SciFi::Constants::max_tracks * sizeof(uint)));
    cudaCheck(hipHostMalloc(
      (void**) &host_scifi_track_hits,
      max_number_of_events * SciFi::Constants::max_tracks * SciFi::Constants::max_track_size * sizeof(SciFi::Hit)));
    cudaCheck(
      hipHostMalloc((void**) &host_scifi_qop, max_number_of_events * SciFi::Constants::max_tracks * sizeof(float)));
    cudaCheck(hipHostMalloc(
      (void**) &host_scifi_states, max_number_of_events * SciFi::Constants::max_tracks * sizeof(MiniState)));
    cudaCheck(hipHostMalloc(
      (void**) &host_scifi_track_ut_indices, max_number_of_events * SciFi::Constants::max_tracks * sizeof(uint)));

    cudaCheck(hipHostMalloc(
      (void**) &host_reconstructed_pvs, max_number_of_events * PV::max_number_vertices * sizeof(PV::Vertex)));
    cudaCheck(hipHostMalloc((void**) &host_number_of_vertex, max_number_of_events * sizeof(int)));
    cudaCheck(hipHostMalloc((void**) &host_number_of_seeds, max_number_of_events * sizeof(int)));
    cudaCheck(hipHostMalloc((void**) &host_zhisto, max_number_of_events * sizeof(float) * (zmax - zmin) / dz));

    cudaCheck(hipHostMalloc((void**) &host_peaks, max_number_of_events * sizeof(float) * PV::max_number_vertices));
    cudaCheck(hipHostMalloc((void**) &host_number_of_peaks, max_number_of_events * sizeof(uint)));

    cudaCheck(hipHostMalloc(
      (void**) &host_reconstructed_multi_pvs, max_number_of_events * PV::max_number_vertices * sizeof(PV::Vertex)));
    cudaCheck(hipHostMalloc((void**) &host_number_of_multivertex, max_number_of_events * sizeof(int)));

    cudaCheck(hipHostMalloc(
      (void**) &host_kf_tracks,
      max_number_of_events * SciFi::Constants::max_tracks * sizeof(ParKalmanFilter::FittedTrack)));
    cudaCheck(hipHostMalloc((void**)&host_muon_catboost_output, max_number_of_events * SciFi::Constants::max_tracks * sizeof(float)));
    cudaCheck(hipHostMalloc((void**)&host_is_muon, max_number_of_events * SciFi::Constants::max_tracks * sizeof(bool)));

    cudaCheck(hipHostMalloc(
      (void**) &host_one_track_decisions,
      max_number_of_events * SciFi::Constants::max_tracks * sizeof(bool)));
    // For some reason this is being exceeded? This seems like a bug.
    //int n_max_svs = SciFi::Constants::max_tracks * (SciFi::Constants::max_tracks - 1) / 2;
    int n_max_svs = SciFi::Constants::max_tracks * SciFi::Constants::max_tracks;
    cudaCheck(hipHostMalloc(
      (void**) &host_sv_offsets,
      (max_number_of_events + 1) * sizeof(uint)));
    cudaCheck(hipHostMalloc(
      (void**) &host_two_track_decisions,
      max_number_of_events * n_max_svs * sizeof(bool)));
    cudaCheck(hipHostMalloc(
      (void**) & host_secondary_vertices,
      n_max_svs * sizeof(VertexFit::Vertex)));
  }
}

size_t HostBuffers::velo_track_hit_number_size() const { return host_number_of_reconstructed_velo_tracks[0] + 1; }

size_t HostBuffers::ut_track_hit_number_size() const { return host_number_of_reconstructed_ut_tracks[0] + 1; }

size_t HostBuffers::scifi_track_hit_number_size() const { return host_number_of_reconstructed_scifi_tracks[0] + 1; }

uint32_t HostBuffers::scifi_hits_uints() const
{
  return (sizeof(SciFi::Hit) / sizeof(uint32_t) + 1) * host_accumulated_number_of_scifi_hits[0];
}
