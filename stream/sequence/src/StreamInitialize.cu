#include "Stream.cuh"

hipError_t Stream::initialize(
  const std::vector<char>& raw_events,
  const std::vector<uint>& event_offsets,
  const std::vector<char>& param_geometry,
  const uint number_of_events,
  const bool param_transmit_host_to_device,
  const bool param_transmit_device_to_host,
  const bool param_do_check,
  const bool param_do_simplified_kalman_filter,
  const bool param_print_individual_rates,
  const std::string param_folder_name_MC,
  const uint param_stream_number
) {
  cudaCheck(hipStreamCreate(&stream));
  cudaCheck(hipEventCreate(&cuda_generic_event));
  cudaCheck(hipEventCreate(&cuda_event_start));
  cudaCheck(hipEventCreate(&cuda_event_stop));
  stream_number = param_stream_number;
  transmit_host_to_device = param_transmit_host_to_device;
  transmit_device_to_host = param_transmit_device_to_host;
  do_check = param_do_check;
  do_simplified_kalman_filter = param_do_simplified_kalman_filter;
  print_individual_rates = param_print_individual_rates;
  geometry = param_geometry;
  folder_name_MC = param_folder_name_MC;
  
  // Blocks and threads for each algorithm
  const uint prefixSumBlocks = (VeloTracking::n_modules * number_of_events + 511) / 512;
  const uint prefixSumScanBlocks = prefixSumBlocks==1 ? 1 : (prefixSumBlocks-1);

  estimateInputSize.set(     dim3(number_of_events),    dim3(32, 26), stream);
  prefixSumReduce.set(       dim3(prefixSumBlocks),     dim3(256),    stream);
  prefixSumSingleBlock.set(  dim3(1),                   dim3(1024),   stream);
  prefixSumScan.set(         dim3(prefixSumScanBlocks), dim3(512),    stream);
  maskedVeloClustering.set(  dim3(number_of_events),    dim3(256),    stream);
  calculatePhiAndSort.set(   dim3(number_of_events),    dim3(64),     stream);
  searchByTriplet.set(       dim3(number_of_events),    dim3(32),     stream);
  consolidateTracks.set(     dim3(number_of_events),    dim3(32),     stream);
  simplifiedKalmanFilter.set(dim3(number_of_events),    dim3(1024),   stream);

  // Datatypes for definitions below
  // Note: The malloc'ing could be eventually moved to each handler, together
  //       with these datatype definitions.
  //       Keeping it like this is for now is flexible and allows easy testing.
  // 
  // Clustering input
  char* dev_raw_input;
  uint* dev_raw_input_offsets;
  uint* dev_estimated_input_size;
  uint* dev_module_cluster_num;
  uint* dev_module_candidate_num;
  uint* dev_cluster_offset;
  uint* dev_cluster_candidates;
  uint32_t* dev_velo_cluster_container;
  char* dev_velo_geometry;
  // Velo tracking
  TrackHits* dev_tracks;
  uint* dev_tracks_to_follow;
  bool* dev_hit_used;
  int* dev_atomics_storage;
  TrackHits* dev_tracklets;
  uint* dev_weak_tracks;
  Track<do_mc_check>* dev_output_tracks;
  short* dev_h0_candidates;
  short* dev_h2_candidates;
  unsigned short* dev_rel_indices;
  uint* dev_hit_permutation;
  // Velo states
  VeloState* dev_velo_states;

  // velo cluster container contains:
  // - cluster_xs
  // - cluster_ys
  // - cluster_zs
  // - cluster_ids
  // - cluster_phis
  // - temporary
  // 
  // The temporary is required to do the sortinge in an efficient manner
  velo_cluster_container_size = number_of_events * VeloClustering::max_candidates_event * 2 * 6;

  // Data preparation
  // Populate velo geometry
  cudaCheck(hipMalloc((void**)&dev_velo_geometry, geometry.size()));
  cudaCheck(hipMemcpyAsync(dev_velo_geometry, geometry.data(), geometry.size(), hipMemcpyHostToDevice, stream));
  
  // Allocate buffers for algorithms
  // Clustering
  cudaCheck(hipMalloc((void**)&dev_raw_input, raw_events.size()));
  cudaCheck(hipMalloc((void**)&dev_raw_input_offsets, event_offsets.size() * sizeof(uint)));
  // DvB: why +2?
  cudaCheck(hipMalloc((void**)&dev_estimated_input_size, (number_of_events * VeloTracking::n_modules + 2) * sizeof(uint)));
  cudaCheck(hipMalloc((void**)&dev_cluster_offset, number_of_events * sizeof(uint)));
  cudaCheck(hipMalloc((void**)&dev_module_cluster_num, number_of_events * VeloTracking::n_modules * sizeof(uint)));
  cudaCheck(hipMalloc((void**)&dev_module_candidate_num, number_of_events * sizeof(uint)));
  cudaCheck(hipMalloc((void**)&dev_cluster_candidates, number_of_events * VeloClustering::max_candidates_event * sizeof(uint)));
  cudaCheck(hipMalloc((void**)&dev_velo_cluster_container, velo_cluster_container_size * sizeof(uint)));

  // phi and sort
  cudaCheck(hipMalloc((void**)&dev_hit_permutation, VeloTracking::max_number_of_hits_per_event * number_of_events * sizeof(uint)));

  // sbt
  cudaCheck(hipMalloc((void**)&dev_tracks_to_follow, number_of_events * VeloTracking::ttf_modulo * sizeof(uint)));
  // Note: Don't reuse buffers unless we are on a "performance" branch
  // dev_tracks_to_follow = dev_cluster_candidates;
  
  cudaCheck(hipMalloc((void**)&dev_tracks, number_of_events * max_tracks_in_event * sizeof(TrackHits)));
  cudaCheck(hipMalloc((void**)&dev_weak_tracks, VeloTracking::max_number_of_hits_per_event * number_of_events * sizeof(uint)));
  
  cudaCheck(hipMalloc((void**)&dev_tracklets, VeloTracking::max_number_of_hits_per_event * number_of_events * sizeof(TrackHits)));
  cudaCheck(hipMalloc((void**)&dev_output_tracks, max_tracks_in_event * number_of_events * sizeof(Track<do_mc_check>)));

  // Note: This is buffer reuse, as the above
  // std::cout << VeloTracking::max_number_of_hits_per_event << " " << number_of_events << " " << sizeof(TrackHits)
  //   << " = " << VeloTracking::max_number_of_hits_per_event * number_of_events * sizeof(TrackHits) << std::endl
  //   << (max_tracks_in_event / 3) << " " << number_of_events << " " << sizeof(Track<do_mc_check>) << " = "
  //   << (max_tracks_in_event / 3) * number_of_events * sizeof(Track<do_mc_check>) << std::endl;

  // const auto tracklets_size = VeloTracking::max_number_of_hits_per_event * number_of_events * sizeof(TrackHits);
  // const auto output_tracks_size = ((tracklets_size / sizeof(Track<do_mc_check>)) + 1) * sizeof(Track<do_mc_check>);
  // std::cout << output_tracks_size << std::endl;

  // cudaCheck(hipMalloc((void**)&dev_output_tracks, output_tracks_size));
  // dev_tracklets = (TrackHits*) dev_output_tracks;

  cudaCheck(hipMalloc((void**)&dev_hit_used, VeloTracking::max_number_of_hits_per_event * number_of_events * sizeof(bool)));
  cudaCheck(hipMalloc((void**)&dev_atomics_storage, number_of_events * atomic_space * sizeof(int)));
  cudaCheck(hipMalloc((void**)&dev_h0_candidates, 2 * VeloTracking::max_number_of_hits_per_event * number_of_events * sizeof(short)));
  cudaCheck(hipMalloc((void**)&dev_h2_candidates, 2 * VeloTracking::max_number_of_hits_per_event * number_of_events * sizeof(short)));
  cudaCheck(hipMalloc((void**)&dev_rel_indices, number_of_events * max_numhits_in_module * sizeof(unsigned short)));

  if (do_simplified_kalman_filter) {
    // simplified kalman filter
    cudaCheck(hipMalloc((void**)&dev_velo_states, number_of_events * max_tracks_in_event * VeloTracking::states_per_track * sizeof(VeloState)));
  }

  // Memory allocations for host memory (copy back)
  cudaCheck(hipHostMalloc((void**)&host_number_of_tracks_pinned, number_of_events * sizeof(int)));
  cudaCheck(hipHostMalloc((void**)&host_accumulated_tracks, number_of_events * sizeof(int)));
  cudaCheck(hipHostMalloc((void**)&host_tracks_pinned, number_of_events * max_tracks_in_event * sizeof(Track<do_mc_check>)));
  // Pre-populate raw_input data, in case the user requested -a 0
  cudaCheck(hipMemcpyAsync(dev_raw_input, raw_events.data(), raw_events.size(), hipMemcpyHostToDevice, stream));
  cudaCheck(hipMemcpyAsync(dev_raw_input_offsets, event_offsets.data(), event_offsets.size() * sizeof(uint), hipMemcpyHostToDevice, stream));

  // Prepare kernels
  estimateInputSize.setParameters(
    dev_raw_input,
    dev_raw_input_offsets,
    dev_estimated_input_size,
    dev_module_cluster_num,
    dev_module_candidate_num,
    dev_cluster_candidates
  );

  prefixSumReduce.setParameters(
    dev_estimated_input_size,
    dev_cluster_offset,
    VeloTracking::n_modules * number_of_events
  );

  prefixSumSingleBlock.setParameters(
    dev_estimated_input_size + VeloTracking::n_modules * number_of_events,
    dev_cluster_offset,
    prefixSumBlocks
  );

  prefixSumScan.setParameters(
    dev_estimated_input_size,
    dev_cluster_offset,
    VeloTracking::n_modules * number_of_events
  );

  maskedVeloClustering.setParameters(
    dev_raw_input,
    dev_raw_input_offsets,
    dev_estimated_input_size,
    dev_module_cluster_num,
    dev_module_candidate_num,
    dev_cluster_candidates,
    dev_velo_cluster_container,
    dev_velo_geometry
  );

  calculatePhiAndSort.setParameters(
    dev_estimated_input_size,
    dev_module_cluster_num,
    dev_velo_cluster_container,
    dev_hit_permutation
  );

  searchByTriplet.setParameters(
    dev_velo_cluster_container,
    dev_estimated_input_size,
    dev_module_cluster_num,
    dev_tracks,
    dev_tracklets,
    dev_tracks_to_follow,
    dev_weak_tracks,
    dev_hit_used,
    dev_atomics_storage,
    dev_h0_candidates,
    dev_h2_candidates,
    dev_rel_indices
  );
  
  simplifiedKalmanFilter.setParameters(
    dev_velo_cluster_container,
    dev_estimated_input_size,
    dev_atomics_storage,
    dev_tracks,
    dev_velo_states
  );

  consolidateTracks.setParameters(
    dev_atomics_storage,
    dev_tracks,
    dev_output_tracks,
    dev_velo_cluster_container,
    dev_estimated_input_size,
    dev_module_cluster_num
  );

  return hipSuccess;
}
