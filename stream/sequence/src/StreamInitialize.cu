#include "Stream.cuh"

/**
 * @brief Sets up statically the chain that will be
 *        executed later.
 *        
 * @details 
 */
hipError_t Stream::initialize(
  const std::vector<char>& raw_events,
  const std::vector<uint>& event_offsets,
  const std::vector<char>& geometry,
  const uint max_number_of_events,
  const bool param_transmit_host_to_device,
  const bool param_transmit_device_to_host,
  const bool param_do_check,
  const bool param_do_simplified_kalman_filter,
  const bool param_print_individual_rates,
  const std::string& param_folder_name_MC,
  const size_t reserve_mb,
  const uint param_stream_number
) {
  // Set stream and events
  cudaCheck(hipStreamCreate(&stream));
  cudaCheck(hipEventCreate(&cuda_generic_event));
  cudaCheck(hipEventCreate(&cuda_event_start));
  cudaCheck(hipEventCreate(&cuda_event_stop));

  // Set stream options
  stream_number = param_stream_number;
  transmit_host_to_device = param_transmit_host_to_device;
  transmit_device_to_host = param_transmit_device_to_host;
  do_check = param_do_check;
  do_simplified_kalman_filter = param_do_simplified_kalman_filter;
  print_individual_rates = param_print_individual_rates;
  folder_name_MC = param_folder_name_MC;

  // Special case
  // Populate velo geometry
  cudaCheck(hipMalloc((void**)&dev_velo_geometry, geometry.size()));
  cudaCheck(hipMemcpyAsync(dev_velo_geometry, geometry.data(), geometry.size(), hipMemcpyHostToDevice, stream));

  // Memory allocations for host memory (copy back)
  cudaCheck(hipHostMalloc((void**)&host_number_of_tracks, max_number_of_events * sizeof(int)));
  cudaCheck(hipHostMalloc((void**)&host_accumulated_tracks, max_number_of_events * sizeof(int)));
  cudaCheck(hipHostMalloc((void**)&host_velo_track_hit_number, max_number_of_events * VeloTracking::max_tracks * sizeof(uint)));
  cudaCheck(hipHostMalloc((void**)&host_velo_track_hits, max_number_of_events * VeloTracking::max_tracks * 20 * sizeof(Hit<mc_check_enabled>)));
  cudaCheck(hipHostMalloc((void**)&host_total_number_of_velo_clusters, sizeof(uint)));
  cudaCheck(hipHostMalloc((void**)&host_number_of_reconstructed_velo_tracks, sizeof(uint)));
  cudaCheck(hipHostMalloc((void**)&host_accumulated_number_of_hits_in_velo_tracks, sizeof(uint)));

  // Define sequence of algorithms to execute
  sequence.set(
    estimate_input_size,
    prefix_sum_reduce,
    prefix_sum_single_block,
    prefix_sum_scan,
    masked_velo_clustering,
    calculatePhiAndSort,
    searchByTriplet,
    copy_and_prefix_sum_single_block,
    copy_velo_track_hit_number,
    prefix_sum_reduce,
    prefix_sum_single_block,
    prefix_sum_scan,
    consolidate_tracks
  );

  // Get sequence and argument names
  sequence_names = get_sequence_names();
  argument_names = get_argument_names();

  // Set options for each algorithm
  // (number of blocks, number of threads, stream, dynamic shared memory space)
  // Setup sequence items opts that are static and will not change
  // regardless of events on flight
  sequence.item<seq::prefix_sum_single_block>().set_opts(                      dim3(1), dim3(1024), stream);
  sequence.item<seq::copy_and_prefix_sum_single_block>().set_opts(             dim3(1), dim3(1024), stream);
  sequence.item<seq::prefix_sum_single_block_velo_track_hit_number>().set_opts(dim3(1), dim3(1024), stream);

  // Set dependencies for each algorithm
  std::vector<std::vector<uint>> sequence_dependencies (std::tuple_size<argument_tuple_t>::value);

  sequence_dependencies[seq::estimate_input_size] = {
    arg::dev_raw_input,
    arg::dev_raw_input_offsets,
    arg::dev_estimated_input_size,
    arg::dev_module_cluster_num,
    arg::dev_module_candidate_num,
    arg::dev_cluster_candidates
  };
  sequence_dependencies[seq::prefix_sum_reduce] = {
    arg::dev_estimated_input_size,
    arg::dev_cluster_offset
  };
  sequence_dependencies[seq::prefix_sum_single_block] = {
    arg::dev_estimated_input_size,
    arg::dev_cluster_offset
  };
  sequence_dependencies[seq::prefix_sum_scan] = {
    arg::dev_estimated_input_size,
    arg::dev_cluster_offset
  };
  sequence_dependencies[seq::masked_velo_clustering] = {
    arg::dev_raw_input,
    arg::dev_raw_input_offsets,
    arg::dev_estimated_input_size,
    arg::dev_module_cluster_num,
    arg::dev_module_candidate_num,
    arg::dev_cluster_candidates,
    arg::dev_velo_cluster_container
  };
  sequence_dependencies[seq::calculate_phi_and_sort] = {
    arg::dev_estimated_input_size,
    arg::dev_module_cluster_num,
    arg::dev_velo_cluster_container,
    arg::dev_hit_permutation
  };
  sequence_dependencies[seq::search_by_triplet] = {
    arg::dev_velo_cluster_container,
    arg::dev_estimated_input_size,
    arg::dev_module_cluster_num,
    arg::dev_tracks,
    arg::dev_tracklets,
    arg::dev_tracks_to_follow,
    arg::dev_weak_tracks,
    arg::dev_hit_used,
    arg::dev_atomics_storage,
    arg::dev_h0_candidates,
    arg::dev_h2_candidates,
    arg::dev_rel_indices
  };
  sequence_dependencies[seq::copy_and_prefix_sum_single_block] = {
    arg::dev_atomics_storage
  };
  sequence_dependencies[seq::copy_velo_track_hit_number] = {
    arg::dev_tracks,
    arg::dev_atomics_storage,
    arg::dev_velo_track_hit_number
  };
  sequence_dependencies[seq::prefix_sum_reduce_velo_track_hit_number] = {
    arg::dev_velo_track_hit_number,
    arg::dev_prefix_sum_auxiliary_array_2
  };
  sequence_dependencies[seq::prefix_sum_single_block_velo_track_hit_number] = {
    arg::dev_velo_track_hit_number,
    arg::dev_prefix_sum_auxiliary_array_2
  };
  sequence_dependencies[seq::prefix_sum_scan_velo_track_hit_number] = {
    arg::dev_velo_track_hit_number,
    arg::dev_prefix_sum_auxiliary_array_2
  };
  sequence_dependencies[seq::consolidate_tracks] = {
    arg::dev_atomics_storage,
    arg::dev_tracks,
    arg::dev_velo_track_hit_number,
    arg::dev_velo_cluster_container,
    arg::dev_estimated_input_size,
    arg::dev_module_cluster_num,
    arg::dev_velo_track_hits,
    arg::dev_velo_states
  };

  // Prepare dynamic scheduler
  scheduler = BaseDynamicScheduler{sequence_names, argument_names,
    sequence_dependencies, reserve_mb * 1024 * 1024};

  // Malloc a configurable reserved memory
  cudaCheck(hipMalloc((void**)&dev_base_pointer, reserve_mb * 1024 * 1024));

  return hipSuccess;
}
