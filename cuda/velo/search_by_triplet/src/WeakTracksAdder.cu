#include "hip/hip_runtime.h"
#include "SearchByTriplet.cuh"
#include "WeakTracksAdder.cuh"

using namespace velo_weak_tracks_adder;

/**
 * @brief Calculates the parameters according to a root means square fit
 *        and returns the chi2.
 */
__device__ float means_square_fit_chi2(
  const Velo::Clusters<const uint>& velo_cluster_container,
  const Velo::TrackletHits& track) {
  VeloState state;

  // Fit parameters
  float s0, sx, sz, sxz, sz2;
  float u0, uy, uz, uyz, uz2;
  s0 = sx = sz = sxz = sz2 = 0.0f;
  u0 = uy = uz = uyz = uz2 = 0.0f;

  // Iterate over hits
  for (unsigned short h = 0; h < 3; ++h) {
    const auto hitno = track.hits[h];
    const auto x = velo_cluster_container.x(hitno);
    const auto y = velo_cluster_container.y(hitno);
    const auto z = velo_cluster_container.z(hitno);

    const auto wx = Velo::Tracking::param_w;
    const auto wx_t_x = wx * x;
    const auto wx_t_z = wx * z;
    s0 += wx;
    sx += wx_t_x;
    sz += wx_t_z;
    sxz += wx_t_x * z;
    sz2 += wx_t_z * z;

    const auto wy = Velo::Tracking::param_w;
    const auto wy_t_y = wy * y;
    const auto wy_t_z = wy * z;
    u0 += wy;
    uy += wy_t_y;
    uz += wy_t_z;
    uyz += wy_t_y * z;
    uz2 += wy_t_z * z;
  }

  {
    // Calculate tx, ty and backward
    const auto dens = 1.0f / (sz2 * s0 - sz * sz);
    state.tx = (sxz * s0 - sx * sz) * dens;
    state.x = (sx * sz2 - sxz * sz) * dens;

    const auto denu = 1.0f / (uz2 * u0 - uz * uz);
    state.ty = (uyz * u0 - uy * uz) * denu;
    state.y = (uy * uz2 - uyz * uz) * denu;
  }

  float chi2;
  {
    //=========================================================================
    // Chi2 / degrees-of-freedom of straight-line fit
    //=========================================================================
    float ch = 0.0f;
    int nDoF = -4;
    for (uint h = 0; h < 3; ++h) {
      const auto hitno = track.hits[h];

      const auto z = velo_cluster_container.z(hitno);
      const auto x = state.x + state.tx * z;
      const auto y = state.y + state.ty * z;

      const auto dx = x - velo_cluster_container.x(hitno);
      const auto dy = y - velo_cluster_container.y(hitno);

      ch += dx * dx * Velo::Tracking::param_w + dy * dy * Velo::Tracking::param_w;

      // Nice :)
      // TODO: We can get rid of the X and Y read here
      // float sum_w_xzi_2 = CL_Velo::Tracking::param_w * x; // for each hit
      // float sum_w_xi_2 = CL_Velo::Tracking::param_w * velo_cluster_container.x(hitno]; // for each hit
      // ch = (sum_w_xzi_2 - sum_w_xi_2) + (sum_w_yzi_2 - sum_w_yi_2);

      nDoF += 2;
    }
    chi2 = ch / nDoF;
  }

  return chi2;
}

/**
 * @brief Calculates the scatter of the three hits.
 *        Unused, but it can be a replacement of the above if needed.
 */
__device__ float scatter(const Velo::Clusters<const uint>& velo_cluster_container, const Velo::TrackletHits& track) {
  const Velo::HitBase h0 {velo_cluster_container.x(track.hits[0]),
                          velo_cluster_container.y(track.hits[0]),
                          velo_cluster_container.z(track.hits[0])};
  const Velo::HitBase h1 {velo_cluster_container.x(track.hits[1]),
                          velo_cluster_container.y(track.hits[1]),
                          velo_cluster_container.z(track.hits[1])};
  const Velo::HitBase h2 {velo_cluster_container.x(track.hits[2]),
                          velo_cluster_container.y(track.hits[2]),
                          velo_cluster_container.z(track.hits[2])};

  // Calculate prediction
  const auto z2_tz = (h2.z - h0.z) / (h1.z - h0.z);
  const auto x = h0.x + (h1.x - h0.x) * z2_tz;
  const auto y = h0.y + (h1.y - h0.y) * z2_tz;
  const auto dx = x - h2.x;
  const auto dy = y - h2.y;

  // Calculate scatter
  return (dx * dx) + (dy * dy);
}

__device__ void weak_tracks_adder_impl(
  uint* weaktracks_insert_pointer,
  uint* tracks_insert_pointer,
  Velo::TrackletHits* weak_tracks,
  Velo::TrackHits* tracks,
  bool* hit_used,
  const Velo::Clusters<const uint>& velo_cluster_container) {
  // Compute the weak tracks
  const auto weaktracks_total = weaktracks_insert_pointer[0];
  for (uint weaktrack_no = threadIdx.x; weaktrack_no < weaktracks_total; weaktrack_no += blockDim.x) {
    const Velo::TrackletHits& t = weak_tracks[weaktrack_no];
    const bool any_used = hit_used[t.hits[0]] || hit_used[t.hits[1]] || hit_used[t.hits[2]];
    const float chi2 = means_square_fit_chi2(velo_cluster_container, t);

    // Store them in the tracks bag
    if (!any_used && chi2 < Configuration::velo_search_by_triplet::max_chi2) {
      const uint trackno = atomicAdd(tracks_insert_pointer, 1);
      assert(trackno < Velo::Constants::max_tracks);
      tracks[trackno] = Velo::TrackHits {t};
    }
  }
}

__global__ void velo_weak_tracks_adder::velo_weak_tracks_adder(
  dev_sorted_velo_cluster_container_t dev_sorted_velo_cluster_container,
  dev_offsets_estimated_input_size_t dev_offsets_estimated_input_size,
  dev_tracks_t dev_tracks,
  dev_weak_tracks_t dev_weak_tracks,
  dev_hit_used_t dev_hit_used,
  dev_atomics_velo_t dev_atomics_velo,
  dev_number_of_velo_tracks_t dev_number_of_velo_tracks) {
  /* Data initialization */
  // Each event is treated with two blocks, one for each side.
  const uint event_number = blockIdx.x;
  const uint number_of_events = gridDim.x;
  const uint tracks_offset = event_number * Velo::Constants::max_tracks;

  // Pointers to data within the event
  const uint total_estimated_number_of_clusters =
    dev_offsets_estimated_input_size[Velo::Constants::n_modules * number_of_events];
  const uint* module_hitStarts = dev_offsets_estimated_input_size + event_number * Velo::Constants::n_modules;
  const uint hit_offset = module_hitStarts[0];
  assert((module_hitStarts[52] - module_hitStarts[0]) < Velo::Constants::max_number_of_hits_per_event);

  // TODO: Offseted container
  const auto velo_cluster_container =
    Velo::Clusters<const uint>{dev_sorted_velo_cluster_container.get() + hit_offset, total_estimated_number_of_clusters};

  // Per event datatypes
  Velo::TrackHits* tracks = dev_tracks + tracks_offset;
  uint* tracks_insert_pointer = (uint*) dev_number_of_velo_tracks + event_number;

  // Per side datatypes
  bool* hit_used = dev_hit_used + hit_offset;
  Velo::TrackletHits* weak_tracks =
    dev_weak_tracks + event_number * Configuration::velo_search_by_triplet::max_weak_tracks;

  // Initialize variables according to event number and module side
  // Insert pointers (atomics)
  uint* weaktracks_insert_pointer = (uint*) dev_atomics_velo + event_number * Velo::num_atomics;

  weak_tracks_adder_impl(
    weaktracks_insert_pointer, tracks_insert_pointer, weak_tracks, tracks, hit_used, velo_cluster_container);
}
