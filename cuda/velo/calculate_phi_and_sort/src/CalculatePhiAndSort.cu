#include "hip/hip_runtime.h"
#include "CalculatePhiAndSort.cuh"

using namespace velo_calculate_phi_and_sort;

/**
 * @brief Track forwarding algorithm based on triplet finding
 */
__global__ void velo_calculate_phi_and_sort::velo_calculate_phi_and_sort(Arguments arguments) {
  __shared__ float shared_hit_phis[Velo::Constants::max_numhits_in_module];

  /* Data initialization */
  // Each event is treated with two blocks, one for each side.
  const uint event_number = blockIdx.x;
  const uint number_of_events = gridDim.x;

  // Pointers to data within the event
  const uint total_estimated_number_of_clusters =
    arguments.dev_offsets_estimated_input_size[Velo::Constants::n_modules * number_of_events];
  const uint* module_hitStarts = arguments.dev_offsets_estimated_input_size + event_number * Velo::Constants::n_modules;
  const uint* module_hitNums = arguments.dev_module_cluster_num + event_number * Velo::Constants::n_modules;

  const auto velo_cluster_container =
    Velo::Clusters<const uint> {arguments.dev_velo_cluster_container.get(), total_estimated_number_of_clusters};
  auto velo_sorted_cluster_container =
    Velo::Clusters<uint> {arguments.dev_sorted_velo_cluster_container.get(), total_estimated_number_of_clusters};

  const uint event_hit_start = module_hitStarts[0];
  const uint event_number_of_hits = module_hitStarts[Velo::Constants::n_modules] - event_hit_start;

  // Calculate phi and populate hit_permutations
  calculate_phi(
    module_hitStarts,
    module_hitNums,
    velo_cluster_container,
    arguments.dev_hit_phi,
    arguments.dev_hit_permutation,
    shared_hit_phis);

  // Due to phi RAW
  __syncthreads();

  // Sort by phi
  sort_by_phi(
    event_hit_start,
    event_number_of_hits,
    velo_cluster_container,
    velo_sorted_cluster_container,
    arguments.dev_hit_permutation);
}
