#include "hip/hip_runtime.h"
#include "CalculatePhiAndSort.cuh"

using namespace velo_calculate_phi_and_sort;

/**
 * @brief Track forwarding algorithm based on triplet finding
 */
__global__ void velo_calculate_phi_and_sort::velo_calculate_phi_and_sort(
  dev_offsets_estimated_input_size_t dev_offsets_estimated_input_size,
  dev_module_cluster_num_t dev_module_cluster_num,
  dev_velo_cluster_container_t dev_velo_cluster_container,
  dev_sorted_velo_cluster_container_t dev_sorted_velo_cluster_container,
  dev_hit_permutation_t dev_hit_permutations,
  dev_hit_phi_t dev_hit_phi) {
  __shared__ float shared_hit_phis[Velo::Constants::max_numhits_in_module];

  /* Data initialization */
  // Each event is treated with two blocks, one for each side.
  const uint event_number = blockIdx.x;
  const uint number_of_events = gridDim.x;

  // Pointers to data within the event
  const uint total_estimated_number_of_clusters =
    dev_offsets_estimated_input_size[Velo::Constants::n_modules * number_of_events];
  const uint* module_hitStarts = dev_offsets_estimated_input_size + event_number * Velo::Constants::n_modules;
  const uint* module_hitNums = dev_module_cluster_num + event_number * Velo::Constants::n_modules;

  const auto velo_cluster_container = Velo::Clusters<const uint>{dev_velo_cluster_container.get(), total_estimated_number_of_clusters};
  auto velo_sorted_cluster_container = Velo::Clusters<uint>{dev_sorted_velo_cluster_container.get(), total_estimated_number_of_clusters};

  const uint event_hit_start = module_hitStarts[0];
  const uint event_number_of_hits = module_hitStarts[Velo::Constants::n_modules] - event_hit_start;

  // Calculate phi and populate hit_permutations
  calculate_phi(
    module_hitStarts, module_hitNums, velo_cluster_container, dev_hit_phi, dev_hit_permutations, shared_hit_phis);

  // Due to phi RAW
  __syncthreads();

  // Sort by phi
  sort_by_phi(
    event_hit_start, event_number_of_hits, velo_cluster_container, velo_sorted_cluster_container, dev_hit_permutations);
}
