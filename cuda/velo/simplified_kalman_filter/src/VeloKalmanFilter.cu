#include "hip/hip_runtime.h"
#include "../include/VeloKalmanFilter.cuh"

/**
 * @brief Helper function to filter one hit
 */
__device__ float velo_kalman_filter_step(
  const float z,
  const float zhit,
  const float xhit,
  const float whit,
  float& x,
  float& tx,
  float& covXX,
  float& covXTx,
  float& covTxTx)
{
  // compute the prediction
  const float dz = zhit - z;
  const float predx = x + dz * tx;

  const float dz_t_covTxTx = dz * covTxTx;
  const float predcovXTx = covXTx + dz_t_covTxTx;
  const float dx_t_covXTx = dz * covXTx;

  const float predcovXX = covXX + 2 * dx_t_covXTx + dz * dz_t_covTxTx;
  const float predcovTxTx = covTxTx;
  // compute the gain matrix
  const float R = 1.0f / ((1.0f / whit) + predcovXX);
  const float Kx = predcovXX * R;
  const float KTx = predcovXTx * R;
  // update the state vector
  const float r = xhit - predx;
  x = predx + Kx * r;
  tx = tx + KTx * r;
  // update the covariance matrix. we can write it in many ways ...
  covXX /*= predcovXX  - Kx * predcovXX */ = (1 - Kx) * predcovXX;
  covXTx /*= predcovXTx - predcovXX * predcovXTx / R */ = (1 - Kx) * predcovXTx;
  covTxTx = predcovTxTx - KTx * predcovXTx;
  // return the chi2
  return r * r * R;
}

__global__ void velo_kalman_fit(
  int* dev_atomics_velo,
  uint* dev_velo_track_hit_number,
  char* dev_velo_track_hits,
  char* dev_velo_states,
  char* dev_velo_kalman_beamline_states)
{
  const uint number_of_events = gridDim.x;
  const uint event_number = blockIdx.x;

  // Consolidated datatypes
  const Velo::Consolidated::Tracks velo_tracks {
    (uint*) dev_atomics_velo, dev_velo_track_hit_number, event_number, number_of_events};
  Velo::Consolidated::States velo_states {dev_velo_states, velo_tracks.total_number_of_tracks};
  Velo::Consolidated::States kalmanvelo_states {dev_velo_kalman_beamline_states, velo_tracks.total_number_of_tracks};

  const uint number_of_tracks_event = velo_tracks.number_of_tracks(event_number);
  const uint event_tracks_offset = velo_tracks.tracks_offset(event_number);

  for (uint i = threadIdx.x; i < number_of_tracks_event; i += blockDim.x) {

    Velo::Consolidated::Hits consolidated_hits = velo_tracks.get_hits(dev_velo_track_hits, i);
    const uint n_hits = velo_tracks.number_of_hits(i);

    VeloState stateAtBeamline = velo_states.get(event_tracks_offset + i);

    VeloState kalmanbeam_state = simplified_fit<true>(consolidated_hits, stateAtBeamline, n_hits);

    kalmanvelo_states.set(event_tracks_offset + i, kalmanbeam_state);
  }
}
