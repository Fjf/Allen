#include "hip/hip_runtime.h"
#include "VeloConsolidateTracks.cuh"

/**
 * @brief Calculates the parameters according to a root means square fit
 */
__device__ VeloState means_square_fit(const Velo::Consolidated::Hits& consolidated_hits, const uint number_of_hits)
{
  VeloState state;

  // Fit parameters
  float s0, sx, sz, sxz, sz2;
  float u0, uy, uz, uyz, uz2;
  s0 = sx = sz = sxz = sz2 = 0.0f;
  u0 = uy = uz = uyz = uz2 = 0.0f;

  // Iterate over hits
  for (unsigned short h = 0; h < number_of_hits; ++h) {
    const auto x = consolidated_hits.x(h);
    const auto y = consolidated_hits.y(h);
    const auto z = consolidated_hits.z(h);

    const auto wx = Velo::Tracking::param_w;
    const auto wx_t_x = wx * x;
    const auto wx_t_z = wx * z;
    s0 += wx;
    sx += wx_t_x;
    sz += wx_t_z;
    sxz += wx_t_x * z;
    sz2 += wx_t_z * z;

    const auto wy = Velo::Tracking::param_w;
    const auto wy_t_y = wy * y;
    const auto wy_t_z = wy * z;
    u0 += wy;
    uy += wy_t_y;
    uz += wy_t_z;
    uyz += wy_t_y * z;
    uz2 += wy_t_z * z;
  }

  // Calculate tx, ty and backward
  const auto dens = 1.0f / (sz2 * s0 - sz * sz);
  state.tx = (sxz * s0 - sx * sz) * dens;
  state.x = (sx * sz2 - sxz * sz) * dens;

  const auto denu = 1.0f / (uz2 * u0 - uz * uz);
  state.ty = (uyz * u0 - uy * uz) * denu;
  state.y = (uy * uz2 - uyz * uz) * denu;

  state.z = -(state.x * state.tx + state.y * state.ty) / (state.tx * state.tx + state.ty * state.ty);
  state.backward = state.z > consolidated_hits.z(0);

  state.x = state.x + state.tx * state.z;
  state.y = state.y + state.ty * state.z;

  return state;
}

template<typename F>
__device__ void populate(const Velo::TrackHits* track, const uint number_of_hits, const F& assign)
{
  for (uint i = 0; i < number_of_hits; ++i) {
    const auto hit_index = track->hits[i];
    assign(i, hit_index);
  }
}

__global__ void velo_consolidate_tracks::velo_consolidate_tracks(velo_consolidate_tracks::Parameters parameters)
{
  const uint number_of_events = gridDim.x;
  const uint event_number = blockIdx.x;

  const Velo::TrackHits* event_tracks = parameters.dev_tracks + event_number * Velo::Constants::max_tracks;
  const Velo::TrackletHits* three_hit_tracks =
    parameters.dev_three_hit_tracks_output + event_number * Velo::Constants::max_tracks;

  // Consolidated datatypes
  const Velo::Consolidated::Tracks velo_tracks {parameters.dev_offsets_all_velo_tracks,
                                                parameters.dev_offsets_velo_track_hit_number,
                                                event_number,
                                                number_of_events};
  Velo::Consolidated::States velo_states {parameters.dev_velo_states, velo_tracks.total_number_of_tracks()};

  const uint event_number_of_tracks = velo_tracks.number_of_tracks(event_number);
  const uint event_tracks_offset = velo_tracks.tracks_offset(event_number);

  const auto event_number_of_three_hit_tracks_filtered =
    parameters.dev_offsets_number_of_three_hit_tracks_filtered[event_number + 1] -
    parameters.dev_offsets_number_of_three_hit_tracks_filtered[event_number];
  const auto event_number_of_tracks_in_main_track_container =
    event_number_of_tracks - event_number_of_three_hit_tracks_filtered;

  // Pointers to data within event
  const uint total_estimated_number_of_clusters =
    parameters.dev_offsets_estimated_input_size[Velo::Constants::n_module_pairs * number_of_events];
  const uint* module_hitStarts =
    parameters.dev_offsets_estimated_input_size + event_number * Velo::Constants::n_module_pairs;
  const uint hit_offset = module_hitStarts[0];

  // Offset'ed container
  const auto velo_cluster_container =
    Velo::ConstClusters {parameters.dev_sorted_velo_cluster_container, total_estimated_number_of_clusters, hit_offset};

  for (uint i = threadIdx.x; i < event_number_of_tracks; i += blockDim.x) {
    Velo::Consolidated::Hits consolidated_hits = velo_tracks.get_hits(parameters.dev_velo_track_hits, i);

    Velo::TrackHits* track;
    uint number_of_hits;

    if (i < event_number_of_tracks_in_main_track_container) {
      track = const_cast<Velo::TrackHits*>(event_tracks) + i;
      number_of_hits = track->hitsNum;
    }
    else {
      track = const_cast<Velo::TrackHits*>(reinterpret_cast<const Velo::TrackHits*>(
        three_hit_tracks + i - event_number_of_tracks_in_main_track_container));
      number_of_hits = 3;
    }

    // Populate hits in a coalesced manner, taking into account
    // the underlying container.
    populate(
      track, number_of_hits, [&velo_cluster_container, &consolidated_hits](const uint i, const uint hit_index) {
        consolidated_hits.set_x(i, velo_cluster_container.x(hit_index));
        consolidated_hits.set_y(i, velo_cluster_container.y(hit_index));
        consolidated_hits.set_z(i, velo_cluster_container.z(hit_index));
      });

    populate(
      track, number_of_hits, [&velo_cluster_container, &consolidated_hits](const uint i, const uint hit_index) {
        consolidated_hits.set_id(i, velo_cluster_container.id(hit_index));
      });

    // Calculate and store fit in consolidated container
    const VeloState beam_state = means_square_fit(consolidated_hits, number_of_hits);
    velo_states.set(event_tracks_offset + i, beam_state);
  }
}
