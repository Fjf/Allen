#include "hip/hip_runtime.h"
#include "ConsolidateTracks.cuh"

__global__ void consolidate_tracks(
  int* dev_atomics_storage,
  const Track* dev_tracks,
  Track* dev_output_tracks
) {
  const unsigned int number_of_events = gridDim.x;
  const unsigned int event_number = blockIdx.x;

  unsigned int accumulated_tracks = 0;
  const Track* event_tracks = dev_tracks + event_number * MAX_TRACKS;

  // Obtain accumulated tracks
  for (unsigned int i=0; i<event_number; ++i) {
    const unsigned int number_of_tracks = dev_atomics_storage[i];
    accumulated_tracks += number_of_tracks;
  }

  // Store accumulated tracks after the number of tracks
  int* accumulated_tracks_base_pointer = dev_atomics_storage + number_of_events;
  accumulated_tracks_base_pointer[event_number] = accumulated_tracks;

  // Consolidate tracks in dev_output_tracks
  const unsigned int number_of_tracks = dev_atomics_storage[event_number];
  Track* destination_tracks = dev_output_tracks + accumulated_tracks;
  for (unsigned int j=0; j<(number_of_tracks + blockDim.x - 1) / blockDim.x; ++j) {
    const unsigned int element = j * blockDim.x + threadIdx.x;
    if (element < number_of_tracks) {
      const Track t = event_tracks[element];
      destination_tracks[element] = t;
    }
  }
}
