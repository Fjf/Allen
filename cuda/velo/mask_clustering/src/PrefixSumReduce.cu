#include "hip/hip_runtime.h"
#include "PrefixSum.cuh"

/**
 * @brief Up-Sweep
 */
__device__ void up_sweep_512(
  uint* data_block
) {
  uint starting_elem = 1;
  for (uint i=2; i<=512; i<<=1) {
    for (uint j=0; j<(511 + blockDim.x) / i; ++j) {
      const uint element = starting_elem + (j*blockDim.x + threadIdx.x) * i;
      if (element < 512) {
        data_block[element] += data_block[element - (i>>1)];
      }
    }
    starting_elem += i;
    __syncthreads();
  }
}

/**
 * @brief Down-sweep
 */
__device__ void down_sweep_512(
  uint* data_block
) {
  for (uint i=512; i>=2; i>>=1) {
    for (uint j=0; j<(511 + blockDim.x) / i; ++j) {
      const auto element = 511 - (j*blockDim.x + threadIdx.x) * i;
      if (element < 512) {
        const auto other_element = element - (i>>1);
        const auto value = data_block[other_element];
        data_block[other_element] = data_block[element];
        data_block[element] += value;
      }
    }
    __syncthreads();
  }
}

__global__ void prefix_sum_reduce(
  uint* dev_estimated_input_size,
  uint* dev_cluster_offset,
  const uint array_size // VeloTracking::n_modules * number_of_events
) {
  
  // Prefix sum of elements in dev_estimated_input_size
  // Using Blelloch scan https://www.youtube.com/watch?v=mmYv3Haj6uc
  __shared__ uint data_block [512];

  // Let's do it in blocks of 512 (2^9)
  const uint last_block = array_size >> 9;
  if (blockIdx.x < last_block) {
    const uint first_elem = blockIdx.x << 9;

    // Load elements into shared memory, add prev_last_elem
    data_block[threadIdx.x] = dev_estimated_input_size[first_elem + threadIdx.x];
    data_block[threadIdx.x + blockDim.x] = dev_estimated_input_size[first_elem + threadIdx.x + blockDim.x];

    __syncthreads();

    up_sweep_512((uint*) &data_block[0]);

    if (threadIdx.x == 0) {
      dev_cluster_offset[blockIdx.x] = data_block[511];
      data_block[511] = 0;
    }

    __syncthreads();

    down_sweep_512((uint*) &data_block[0]);

    // Store back elements
    //assert( first_elem + threadIdx.x + blockDim.x < number_of_events * VeloTracking::n_modules + 2);
    dev_estimated_input_size[first_elem + threadIdx.x] = data_block[threadIdx.x];
    dev_estimated_input_size[first_elem + threadIdx.x + blockDim.x] = data_block[threadIdx.x + blockDim.x];

    __syncthreads();
  }

  // Last block is special because
  // it may contain an unspecified number of elements
  else {
    const auto elements_remaining = array_size & 0x1FF; // % 512
    if (elements_remaining > 0) {
      const auto first_elem = array_size - elements_remaining;

      // Initialize all elements to zero
      data_block[threadIdx.x] = 0;
      data_block[threadIdx.x + blockDim.x] = 0;

      // Load elements
      const auto elem_index = first_elem + threadIdx.x;
      if (elem_index < array_size) {
        data_block[threadIdx.x] = dev_estimated_input_size[elem_index];
      }
      if ((elem_index+blockDim.x) < array_size) {
        data_block[threadIdx.x + blockDim.x] = dev_estimated_input_size[elem_index + blockDim.x];
      }

      __syncthreads();

      up_sweep_512((uint*) &data_block[0]);

      // Store sum of all elements
      if (threadIdx.x == 0) {
        dev_cluster_offset[blockIdx.x] = data_block[511];
        data_block[511] = 0;
      }
      
      __syncthreads();

      down_sweep_512((uint*) &data_block[0]);

      // Store back elements
      if (elem_index < array_size) {
        dev_estimated_input_size[elem_index] = data_block[threadIdx.x];
      }
      if ((elem_index+blockDim.x) < array_size) {
        dev_estimated_input_size[elem_index + blockDim.x] = data_block[threadIdx.x + blockDim.x];
      }
    }
  }
}
