#include "hip/hip_runtime.h"
#include "Tools.cuh"

std::map<std::string, float> calcResults(std::vector<float>& times){
    // sqrt ( E( (X - m)2) )
    std::map<std::string, float> results;
    float deviation = 0.0f, variance = 0.0f, mean = 0.0f, min = float_max(), max = 0.0f;

    for(auto it = times.begin(); it != times.end(); it++){
        const float seconds = (*it);
        mean += seconds;
        variance += seconds * seconds;

        if (seconds < min) min = seconds;
        if (seconds > max) max = seconds;
    }

    mean /= times.size();
    variance = (variance / times.size()) - (mean * mean);
    deviation = std::sqrt(variance);

    results["variance"] = variance;
    results["deviation"] = deviation;
    results["mean"] = mean;
    results["min"] = min;
    results["max"] = max;

    return results;
}

float float_max() {
    const int value = 0x7f800000;
    const float* const fvalue = (const float*) &value;
    return *(float*)& fvalue[0];
}
