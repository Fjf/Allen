#include "hip/hip_runtime.h"
#include "SortByX.cuh"

__global__ void sort_by_x(
  uint32_t* dev_ut_hits,
  uint32_t* dev_ut_hit_count,
  uint* dev_hit_permutations
) {
  const uint number_of_events = gridDim.x;
  const uint event_number = blockIdx.x;

  // Two UTHits objects are created: one typecasts the base_pointer assuming
  // the data is unsorted, the other assuming the data is sorted.
  // This makes sorting more readable
  UTHits unsorted_ut_hits, sorted_ut_hits;
  unsorted_ut_hits.typecast_unsorted(dev_ut_hits, dev_ut_hit_count[number_of_events * VeloUTTracking::n_layers]);
  sorted_ut_hits.typecast_sorted(dev_ut_hits, dev_ut_hit_count[number_of_events * VeloUTTracking::n_layers]);

  VeloUTTracking::HitsSoA* hits_layers        = dev_ut_hits + event_number;
  VeloUTTracking::HitsSoA* hits_layers_sorted = dev_ut_hits_sorted + event_number;

  uint* hit_permutations = dev_hit_permutations + event_number * VeloUTTracking::max_numhits_per_event;
  for ( int i_layer = 0; i_layer < VeloUTTracking::n_layers; ++i_layer ) {
    const uint n_hits = hits_layers->n_hits_layers[i_layer];
    const uint layer_offset = hits_layers->layer_offset[i_layer];
    for (unsigned int i=0; i<(n_hits + blockDim.x - 1) / blockDim.x; ++i) {
      const auto index = i*blockDim.x + threadIdx.x;
      if (index < n_hits) {
        hit_permutations[layer_offset + index] = 0;
      }
    }
    
    __syncthreads();
    
   
    if ( threadIdx.x == 0 ) {
      hits_layers_sorted->n_hits_layers[i_layer] = n_hits;
      hits_layers_sorted->layer_offset[i_layer] = layer_offset;
    }
    
    find_permutation<float>( 
        hits_layers->m_xAtYEq0,
        layer_offset,
      	hit_permutations,
      	n_hits
      );

    __syncthreads();

    apply_permutation<float>( hit_permutations, layer_offset, n_hits, hits_layers->m_cos, hits_layers_sorted->m_cos );
    apply_permutation<float>( hit_permutations, layer_offset, n_hits, hits_layers->m_weight, hits_layers_sorted->m_weight );
    apply_permutation<float>( hit_permutations, layer_offset, n_hits, hits_layers->m_xAtYEq0, hits_layers_sorted->m_xAtYEq0 );
    apply_permutation<float>( hit_permutations, layer_offset, n_hits, hits_layers->m_yBegin, hits_layers_sorted->m_yBegin );
    apply_permutation<float>( hit_permutations, layer_offset, n_hits, hits_layers->m_yEnd, hits_layers_sorted->m_yEnd );
    apply_permutation<float>( hit_permutations, layer_offset, n_hits, hits_layers->m_zAtYEq0, hits_layers_sorted->m_zAtYEq0 );
    apply_permutation<unsigned int>( hit_permutations, layer_offset, n_hits, hits_layers->m_LHCbID, hits_layers_sorted->m_LHCbID );
    apply_permutation<int>( hit_permutations, layer_offset, n_hits, hits_layers->m_planeCode, hits_layers_sorted->m_planeCode );
    apply_permutation<int>( hit_permutations, layer_offset, n_hits, hits_layers->m_highThreshold, hits_layers_sorted->m_highThreshold );
  }
  
}
