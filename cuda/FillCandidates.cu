#include "hip/hip_runtime.h"
#include "SearchByTriplet.cuh"

/**
 * @brief Fills dev_hit_candidates.
 * 
 * @param hit_candidates    
 * @param hit_h2_candidates 
 * @param number_of_sensors 
 * @param sensor_hitStarts  
 * @param sensor_hitNums    
 * @param hit_Xs            
 * @param hit_Ys            
 * @param hit_Zs            
 * @param sensor_Zs         
 */
__device__ void fillCandidates(
  int* const hit_candidates,
  int* const hit_h2_candidates,
  const int number_of_sensors,
  const int* const sensor_hitStarts,
  const int* const sensor_hitNums,
  const float* const hit_Xs,
  const float* const hit_Ys,
  const float* const hit_Zs,
  const int* sensor_Zs
) {
  const int blockDim_product = blockDim.x * blockDim.y;
  int first_sensor = number_of_sensors - 1;
  while (first_sensor >= 2) {
    const int second_sensor = first_sensor - 2;

    const bool process_h1_candidates = first_sensor >= 4;
    const bool process_h2_candidates = first_sensor <= number_of_sensors - 3;

    // Sensor dependent calculations
    const int z_s0 = process_h2_candidates ? sensor_Zs[first_sensor + 2] : 0;
    const int z_s2 = process_h2_candidates ? sensor_Zs[second_sensor] : 0;

    // Iterate in all hits in z0
    for (int i=0; i<(sensor_hitNums[first_sensor] + blockDim_product - 1) / blockDim_product; ++i) {
      const int h0_element = blockDim_product * i + threadIdx.y * blockDim.x + threadIdx.x;
      bool inside_bounds = h0_element < sensor_hitNums[first_sensor];

      if (inside_bounds) {
        bool first_h1_found = false, last_h1_found = false;
        bool first_h2_found = false, last_h2_found = false;
        const int h0_index = sensor_hitStarts[first_sensor] + h0_element;
        int h1_index;
        const Hit h0 {hit_Xs[h0_index], 0.f, hit_Zs[h0_index]};
        const int hitstarts_s2 = sensor_hitStarts[second_sensor];
        const int hitnums_s2 = sensor_hitNums[second_sensor];

        float xmin_h2, xmax_h2;
        if (process_h2_candidates) {
          // Note: Here, we take h0 as if it were h1, the rest
          // of the notation is fine.
          
          // Min and max possible x0s
          const float h_dist = fabs(h0.z - z_s0);
          const float dxmax = PARAM_MAXXSLOPE_CANDIDATES * h_dist;
          const float x0_min = h0.x - dxmax;
          const float x0_max = h0.x + dxmax;

          // Min and max possible h1s for that h0
          float z2_tz = (((float) z_s2 - z_s0)) / (h0.z - z_s0);
          float x = x0_max + (h0.x - x0_max) * z2_tz;
          xmin_h2 = x - PARAM_TOLERANCE_CANDIDATES;

          x = x0_min + (h0.x - x0_min) * z2_tz;
          xmax_h2 = x + PARAM_TOLERANCE_CANDIDATES;
        }
        
        if (first_sensor >= 4) {
          // Iterate in all hits in z1
          for (int h1_element=0; h1_element<hitnums_s2; ++h1_element) {
            inside_bounds = h1_element < hitnums_s2;

            if (inside_bounds) {
              h1_index = hitstarts_s2 + h1_element;
              const Hit h1 {hit_Xs[h1_index], 0.f, hit_Zs[h1_index]};

              if (process_h1_candidates && !last_h1_found) {
                // Check if h0 and h1 are compatible
                const float h_dist = fabs(h1.z - h0.z);
                const float dxmax = PARAM_MAXXSLOPE_CANDIDATES * h_dist;
                const bool tol_condition = fabs(h1.x - h0.x) < dxmax;
                
                // Find the first one
                if (!first_h1_found && tol_condition) {
                  ASSERT(2 * h0_index < 2 * (sensor_hitStarts[number_of_sensors-1] + sensor_hitNums[number_of_sensors-1]))

                  hit_candidates[2 * h0_index] = h1_index;
                  first_h1_found = true;
                }
                // The last one, only if the first one has already been found
                else if (first_h1_found && !tol_condition) {
                  ASSERT(2 * h0_index + 1 < 2 * (sensor_hitStarts[number_of_sensors-1] + sensor_hitNums[number_of_sensors-1]))

                  hit_candidates[2 * h0_index + 1] = h1_index;
                  last_h1_found = true;
                }
              }

              if (process_h2_candidates && !last_h2_found) {
                if (!first_h2_found && h1.x > xmin_h2) {
                  ASSERT(2 * h0_index < 2 * (sensor_hitStarts[number_of_sensors-1] + sensor_hitNums[number_of_sensors-1]))

                  hit_h2_candidates[2 * h0_index] = h1_index;
                  first_h2_found = true;
                }
                else if (first_h2_found && h1.x > xmax_h2) {
                  ASSERT(2 * h0_index + 1 < 2 * (sensor_hitStarts[number_of_sensors-1] + sensor_hitNums[number_of_sensors-1]))

                  hit_h2_candidates[2 * h0_index + 1] = h1_index;
                  last_h2_found = true;
                }
              }

              if ((!process_h1_candidates || last_h1_found) &&
                  (!process_h2_candidates || last_h2_found)) {
                break;
              }
            }
          }

          // Note: If first is not found, then both should be -1
          // and there wouldn't be any iteration
          if (process_h1_candidates && first_h1_found && !last_h1_found) {
            ASSERT(2 * h0_index + 1 < 2 * (sensor_hitStarts[number_of_sensors-1] + sensor_hitNums[number_of_sensors-1]))

            hit_candidates[2 * h0_index + 1] = hitstarts_s2 + hitnums_s2;
          }

          if (process_h2_candidates && first_h2_found && !last_h2_found) {
            ASSERT(2 * h0_index + 1 < 2 * (sensor_hitStarts[number_of_sensors-1] + sensor_hitNums[number_of_sensors-1]))

            hit_h2_candidates[2 * h0_index + 1] = hitstarts_s2 + hitnums_s2;
          }
        }
      }
    }

    --first_sensor;
  }
}
