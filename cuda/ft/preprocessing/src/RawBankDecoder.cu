#include "hip/hip_runtime.h"
#include "RawBankDecoder.cuh"
#include <stdio.h>
#include "assert.h"

__device__ uint32_t channelInBank(uint32_t c) {
  return (c >> FTRawBankParams::cellShift);
}

__device__ uint16_t getLinkInBank(uint16_t c){
  return (c >> FTRawBankParams::linkShift);
}

__device__ int cell(uint16_t c) {
  return (c >> FTRawBankParams::cellShift     ) & FTRawBankParams::cellMaximum;
}

__device__ int fraction(uint16_t c) {
  return (c >> FTRawBankParams::fractionShift ) & FTRawBankParams::fractionMaximum;
}

__device__ bool cSize(uint16_t c) {
  return (c >> FTRawBankParams::sizeShift     ) & FTRawBankParams::sizeMaximum;
}


__global__ void raw_bank_decoder(uint *ft_event_offsets,  uint *ft_cluster_offsets, char *ft_events, FTLiteCluster *ft_clusters, uint* ft_cluster_nums) {
  // TODO: Optimize parallelization (as in estimate_cluster_count).
  const uint event_id = blockIdx.x;
  //if first thread...
  *(ft_cluster_nums + event_id) = 0;
  __syncthreads();

  const uint32_t temporary_hardcoded_readoutmap[] = {262144u, 264192u, 266240u, 268288u, 270336u, 278528u, 280576u, 282624u, 284672u, 286720u, 294912u, 296960u, 299008u, 301056u, 303104u, 311296u, 313344u, 315392u, 317440u, 319488u, 327680u, 329728u, 331776u, 333824u, 335872u, 344064u, 346112u, 348160u, 350208u, 352256u, 360448u, 362496u, 364544u, 366592u, 368640u, 376832u, 378880u, 380928u, 382976u, 385024u, 393216u, 395264u, 397312u, 399360u, 401408u, 409600u, 411648u, 413696u, 415744u, 417792u, 425984u, 428032u, 430080u, 432128u, 434176u, 442368u, 444416u, 446464u, 448512u, 450560u, 458752u, 460800u, 462848u, 464896u, 466944u, 475136u, 477184u, 479232u, 481280u, 483328u, 491520u, 493568u, 495616u, 497664u, 499712u, 507904u, 509952u, 512000u, 514048u, 516096u, 524288u, 526336u, 528384u, 530432u, 532480u, 540672u, 542720u, 544768u, 546816u, 548864u, 557056u, 559104u, 561152u, 563200u, 565248u, 573440u, 575488u, 577536u, 579584u, 581632u, 589824u, 591872u, 593920u, 595968u, 598016u, 606208u, 608256u, 610304u, 612352u, 614400u, 622592u, 624640u, 626688u, 628736u, 630784u, 638976u, 641024u, 643072u, 645120u, 647168u, 655360u, 657408u, 659456u, 661504u, 663552u, 671744u, 673792u, 675840u, 677888u, 679936u, 688128u, 690176u, 692224u, 694272u, 696320u, 704512u, 706560u, 708608u, 710656u, 712704u, 720896u, 722944u, 724992u, 727040u, 729088u, 737280u, 739328u, 741376u, 743424u, 745472u, 753664u, 755712u, 757760u, 759808u, 761856u, 770048u, 772096u, 774144u, 776192u, 778240u, 786432u, 788480u, 791040u, 793600u, 796160u, 802816u, 804864u, 807424u, 809984u, 812544u, 819200u, 821248u, 823808u, 826368u, 828928u, 835584u, 837632u, 840192u, 842752u, 845312u, 851968u, 854016u, 856576u, 859136u, 861696u, 868352u, 870400u, 872960u, 875520u, 878080u, 884736u, 886784u, 889344u, 891904u, 894464u, 901120u, 903168u, 905728u, 908288u, 910848u, 917504u, 919552u, 922112u, 924672u, 927232u, 933888u, 935936u, 938496u, 941056u, 943616u, 950272u, 952320u, 954880u, 957440u, 960000u, 966656u, 968704u, 971264u, 973824u, 976384u, 983040u, 985088u, 987648u, 990208u, 992768u, 999424u, 1001472u, 1004032u, 1006592u, 1009152u, 1015808u, 1017856u, 1020416u, 1022976u, 1025536u, 1032192u, 1034240u, 1036800u, 1039360u, 1041920u};

  const auto event = FTRawEvent(ft_events + ft_event_offsets[event_id]);
  const uint rawbank_chunk = (event.number_of_raw_banks + blockDim.x - 1) / blockDim.x; // ceiling int division
  assert(event.version == 5u);

  auto make_cluster = [&](uint32_t chan, uint8_t fraction, uint8_t pseudoSize) {
    uint clusterIndex = atomicAdd(ft_cluster_nums + event_id, 1u) + (event_id == 0? 0 : ft_cluster_offsets[event_id-1]);
    ft_clusters[clusterIndex] = {chan, fraction, pseudoSize};
    //printf("making cluster %u: chan %u \n", clusterIndex, ft_clusters[clusterIndex].channelID.channelID);
  };

  //copied straight from FTRawBankDecoder.cpp
  auto make_clusters = [&](uint32_t firstChannel, uint16_t c, uint16_t c2) {
    unsigned int delta = (cell(c2) - cell(c));
    uint m_clusterMaxWidth = 4u; //TODO: evaluate whether hardcoding is fine

    // fragmented clusters, size > 2*max size
    // only edges were saved, add middles now
    if ( delta  > m_clusterMaxWidth ) {
      //add the first edge cluster, and then the middle clusters
      for(unsigned int  i = m_clusterMaxWidth; i < delta ; i+= m_clusterMaxWidth){
        // all middle clusters will have same size as the first cluster,
        // so re-use the fraction
        make_cluster( firstChannel+i, fraction(c), 0 );
      }
      //add the last edge
      make_cluster  ( firstChannel+delta, fraction(c2), 0 );
    } else { //big cluster size upto size 8
      unsigned int widthClus  =  2 * delta - 1 + fraction(c2);
      make_cluster( firstChannel+(widthClus-1)/2 - int( (m_clusterMaxWidth-1)/2 ),
                    (widthClus-1)%2, widthClus );
    }//end if adjacent clusters
  };//End lambda make_clusters

  for(uint i = threadIdx.x; i < event.number_of_raw_banks; i+=event.number_of_raw_banks/rawbank_chunk)
  {
    uint start = (i == 0? 0 : event.raw_bank_offset[i-1]);
    FTRawBank rawbank(event.payload + start, event.payload + event.raw_bank_offset[i]);

    uint16_t* it = rawbank.data + 2;
    uint16_t* last = rawbank.last;
    //printf("start: %u\n", start);
    if (*(last-1) == 0) --last;//Remove padding at the end
    for( ;  it < last; ++it ){ // loop over the clusters
      uint16_t c = *it;
      uint32_t ch = temporary_hardcoded_readoutmap[rawbank.sourceID] + channelInBank(c);
      //printf("byte %x, cib %u, ch %u, station %u, layer %u, quarter %u, module %u, mat %u, sipm %u, channel %u\n", c, channelInBank(c), ch.channelID, ch.station(), ch.layer(), ch.quarter(), ch.module(), ch.mat(), ch.sipm(), ch.channel());

      if( !cSize(c) || it+1 == last ) { //No size flag or last cluster
        make_cluster(ch, fraction(c), 4);
      } else {//Flagged or not the last one.
        unsigned c2 = *(it+1);
        if( cSize(c2) && getLinkInBank(c) == getLinkInBank(c2) ) {
          make_clusters(ch,c,c2);
          ++it;
        } else {
          make_cluster(ch, fraction(c), 4);
        }
      }
    }
    //printf("global offset: %x \n", event.payload + start - ft_events);
    //printf("sourceID: %u \n", rawbank.sourceID);
  }
}
