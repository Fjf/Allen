#include "hip/hip_runtime.h"
#include "RunHlt1.cuh"
#include "TrackMVALines.cuh"
#include "MuonLines.cuh"
#include "LineHandler.cuh"

__global__ void run_hlt1::run_hlt1(run_hlt1::Parameters parameters)
{
  const uint number_of_events = gridDim.x;
  const uint event_number = blockIdx.x;

  // Tracks.
  const auto* event_tracks_offsets = parameters.dev_atomics_scifi + number_of_events;
  const auto* event_svs_offsets = parameters.dev_sv_atomics + number_of_events;
  const ParKalmanFilter::FittedTrack* event_tracks = parameters.dev_kf_tracks + event_tracks_offsets[event_number];
  bool* event_one_track_results = parameters.dev_one_track_results + event_tracks_offsets[event_number];
  bool* event_single_muon_results = parameters.dev_single_muon_results + event_tracks_offsets[event_number];
  const auto n_tracks_event = parameters.dev_atomics_scifi[event_number];

  // Vertices.
  const VertexFit::TrackMVAVertex* event_vertices = parameters.dev_consolidated_svs + event_svs_offsets[event_number];
  bool* event_two_track_results = parameters.dev_two_track_results + event_svs_offsets[event_number];
  bool* event_disp_dimuon_results = parameters.dev_disp_dimuon_results + event_svs_offsets[event_number];
  bool* event_high_mass_dimuon_results = parameters.dev_high_mass_dimuon_results + event_svs_offsets[event_number];
  bool* event_dimuon_soft_results = parameters.dev_dimuon_soft_results + event_svs_offsets[event_number];

  const auto n_vertices_event = parameters.dev_sv_atomics[event_number];

  LineHandler<ParKalmanFilter::FittedTrack> oneTrackHandler {TrackMVALines::OneTrackMVA};
  LineHandler<VertexFit::TrackMVAVertex> twoTrackHandler {TrackMVALines::TwoTrackMVA};
  LineHandler<ParKalmanFilter::FittedTrack> singleMuonHandler {MuonLines::SingleMuon};
  LineHandler<VertexFit::TrackMVAVertex> dispDiMuonHandler {MuonLines::DisplacedDiMuon};
  LineHandler<VertexFit::TrackMVAVertex> highMassDiMuonHandler {MuonLines::HighMassDiMuon};
  LineHandler<VertexFit::TrackMVAVertex> diMuonSoftHandler {MuonLines::DiMuonSoft};

  // One track lines.
  oneTrackHandler(event_tracks, n_tracks_event, event_one_track_results);

  singleMuonHandler(event_tracks, n_tracks_event, event_single_muon_results);

  // Two track lines.
  twoTrackHandler(event_vertices, n_vertices_event, event_two_track_results);
  
  dispDiMuonHandler(event_vertices, n_vertices_event, event_disp_dimuon_results);

  highMassDiMuonHandler(event_vertices, n_vertices_event, event_high_mass_dimuon_results);
  diMuonSoftHandler(event_vertices, n_vertices_event, event_dimuon_soft_results);
}
