#include "hip/hip_runtime.h"
#include "RunHlt1.cuh"
#include "RawBanksDefinitions.cuh"
#include "TrackMVALines.cuh"
#include "MuonLines.cuh"

#include "Handler.cuh"
#include "ArgumentsSciFi.cuh"
#include "ArgumentsKalmanFilter.cuh"
#include "ArgumentsPV.cuh"
#include "ArgumentsSelections.cuh"
#include "ArgumentsVertex.cuh"

__global__ void run_hlt1(
  const ParKalmanFilter::FittedTrack* dev_kf_tracks,
  const VertexFit::TrackMVAVertex* dev_consolidated_svs,
  const uint* dev_atomics_scifi,
  const uint* dev_sv_atomics,
  bool* dev_sel_results,
  uint* dev_sel_results_atomics)
{
  const uint number_of_events = gridDim.x;
  const uint event_number = blockIdx.x;

  const uint total_tracks = dev_atomics_scifi[2 * number_of_events];
  const uint total_svs = dev_sv_atomics[2 * number_of_events];
  
  const uint* dev_sel_result_offsets = dev_sel_results_atomics + Hlt1::Hlt1Lines::End;
  
  // Tracks.
  const auto* event_tracks_offsets = dev_atomics_scifi + number_of_events;
  const auto* event_svs_offsets = dev_sv_atomics + number_of_events;
  const ParKalmanFilter::FittedTrack* event_tracks = dev_kf_tracks + event_tracks_offsets[event_number];
  bool* event_one_track_results = dev_sel_results +
    dev_sel_result_offsets[Hlt1::Hlt1Lines::OneTrackMVA] + event_tracks_offsets[event_number];
  bool* event_single_muon_results = dev_sel_results +
    dev_sel_result_offsets[Hlt1::Hlt1Lines::SingleMuon] + event_tracks_offsets[event_number];
  const auto n_tracks_event = dev_atomics_scifi[event_number];

  // Vertices.
  const VertexFit::TrackMVAVertex* event_vertices = dev_consolidated_svs + event_svs_offsets[event_number];
  bool* event_two_track_results = dev_sel_results + 
    dev_sel_result_offsets[Hlt1::Hlt1Lines::TwoTrackMVA] + event_svs_offsets[event_number];
  bool* event_disp_dimuon_results = dev_sel_results + 
    dev_sel_result_offsets[Hlt1::Hlt1Lines::DisplacedDiMuon] + event_svs_offsets[event_number];
  bool* event_high_mass_dimuon_results = dev_sel_results + 
    dev_sel_result_offsets[Hlt1::Hlt1Lines::HighMassDiMuon] + event_svs_offsets[event_number];
  bool* event_dimuon_soft_results = dev_sel_results + 
    dev_sel_result_offsets[Hlt1::Hlt1Lines::SoftDiMuon] + event_svs_offsets[event_number];
  const auto n_vertices_event = dev_sv_atomics[event_number];
  
  LineHandler<ParKalmanFilter::FittedTrack> oneTrackHandler {TrackMVALines::OneTrackMVA};
  LineHandler<ParKalmanFilter::FittedTrack> singleMuonHandler {MuonLines::SingleMuon};
  LineHandler<VertexFit::TrackMVAVertex> twoTrackHandler {TrackMVALines::TwoTrackMVA};
  LineHandler<VertexFit::TrackMVAVertex> dispDiMuonHandler {MuonLines::DisplacedDiMuon};
  LineHandler<VertexFit::TrackMVAVertex> highMassDiMuonHandler {MuonLines::HighMassDiMuon};
  LineHandler<VertexFit::TrackMVAVertex> diMuonSoftHandler {MuonLines::DiMuonSoft};

  // One track lines.
  oneTrackHandler(event_tracks, n_tracks_event, event_one_track_results);
  singleMuonHandler(event_tracks, n_tracks_event, event_single_muon_results);

  // Two track lines.
  twoTrackHandler(event_vertices, n_vertices_event, event_two_track_results);  
  dispDiMuonHandler(event_vertices, n_vertices_event, event_disp_dimuon_results);
  highMassDiMuonHandler(event_vertices, n_vertices_event, event_high_mass_dimuon_results);
  diMuonSoftHandler(event_vertices, n_vertices_event, event_dimuon_soft_results);
}
