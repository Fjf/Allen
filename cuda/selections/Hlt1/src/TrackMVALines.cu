#include "hip/hip_runtime.h"
#include "TrackMVALines.cuh"

namespace TrackMVALines {

  __device__ bool OneTrackMVA(const ParKalmanFilter::FittedTrack& track)
  {
    float ptShift = track.pt() - alpha;
    bool decision =  track.chi2/track.ndof < maxChi2Ndof;
    decision &= ((ptShift > maxPt && track.ipChi2 > minIPChi2) ||
               (ptShift > minPt && ptShift < maxPt &&
                std::log(track.ipChi2) > param1 / (ptShift/1000. - param2) / (ptShift/1000. - param2)
                + param3 / maxPt * (maxPt - ptShift) + std::log(minIPChi2)));
    return decision;
  }

  __device__ bool TwoTrackMVA(const VertexFit::TrackMVAVertex& vertex)
  {
    if (vertex.chi2 < 0) {
      return false;
    }
    bool decision = vertex.pt() > minComboPt;
    decision &= vertex.chi2 < maxVertexChi2;
    decision &= vertex.mcor > minMCor;
    decision &= vertex.eta > minEta && vertex.eta < maxEta;
    decision &= vertex.ntrksassoc <= maxNTrksAssoc;
    decision &= vertex.fdchi2 > minFDChi2;
    return decision;
  }
  
}
