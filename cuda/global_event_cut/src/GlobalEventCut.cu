#include "hip/hip_runtime.h"
#include <MEPTools.cuh>
#include <GlobalEventCut.cuh>

__global__ void global_event_cut(
  char* ut_raw_input,
  uint* ut_raw_input_offsets,
  char* scifi_raw_input,
  uint* scifi_raw_input_offsets,
  uint* number_of_selected_events,
  uint* event_list)
{
  const uint event_number = blockIdx.x;

  // Check SciFi clusters
  const SciFi::SciFiRawEvent scifi_event(scifi_raw_input + scifi_raw_input_offsets[event_number]);
  __shared__ uint n_SciFi_clusters;
  if (threadIdx.x == 0) n_SciFi_clusters = 0;
  __syncthreads();
  for (uint i = threadIdx.x; i < scifi_event.number_of_raw_banks; i += blockDim.x) {
    // get bank size in bytes, subtract four bytes for header word
    uint bank_size = scifi_event.raw_bank_offset[i + 1] - scifi_event.raw_bank_offset[i] - 4;
    atomicAdd(&n_SciFi_clusters, bank_size);
  }
  __syncthreads();
  // Bank size is given in bytes. There are 2 bytes per cluster.
  // 4 bytes are removed for the header.
  // Note that this overestimates slightly the number of clusters
  // due to bank padding in 32b. For v5, it further overestimates the
  // number of clusters due to the merging of clusters.
  if (threadIdx.x == 0) n_SciFi_clusters = n_SciFi_clusters / 2 - 2;
  __syncthreads();

  // if (n_SciFi_clusters >= max_scifi_ut_clusters || n_SciFi_clusters < min_scifi_ut_clusters) return;

  // Check UT clusters
  const uint32_t ut_event_offset = ut_raw_input_offsets[event_number];
  const UTRawEvent ut_event(ut_raw_input + ut_event_offset);
  __shared__ uint n_UT_clusters;
  if (threadIdx.x == 0) n_UT_clusters = 0;
  __syncthreads();
  for (uint i = threadIdx.x; i < ut_event.number_of_raw_banks; i += blockDim.x) {
    const UTRawBank ut_bank = ut_event.getUTRawBank(i);
    const int n_UT_clusters_before = atomicAdd(&n_UT_clusters, ut_bank.number_of_hits);
    // if (n_UT_clusters_before + ut_bank.number_of_hits >= max_scifi_ut_clusters) return;
  }
  __syncthreads();

  const auto num_combined_clusters = n_UT_clusters + n_SciFi_clusters;

  if (num_combined_clusters >= max_scifi_ut_clusters) return;
  // if (num_combined_clusters < min_scifi_ut_clusters) return;

  // passed cut
  if (threadIdx.x == 0) {
    const int selected_event = atomicAdd(number_of_selected_events, 1);
    event_list[selected_event] = event_number;
  }
}

__global__ void global_event_cut_mep(
  char* ut_raw_input,
  uint* ut_raw_input_offsets,
  char*,
  uint* scifi_raw_input_offsets,
  uint* number_of_selected_events,
  uint* event_list)
{
  const uint event_number = blockIdx.x;

  // Check SciFi clusters
  __shared__ uint n_SciFi_clusters;
  if (threadIdx.x == 0) n_SciFi_clusters = 0;
  __syncthreads();

  auto const number_of_scifi_raw_banks = scifi_raw_input_offsets[0];

  for (uint i = threadIdx.x; i < number_of_scifi_raw_banks; i += blockDim.x) {
    // get bank size in bytes, subtract four bytes for header word
    uint const offset_index = 2 + number_of_scifi_raw_banks * (1 + event_number);
    uint bank_size = scifi_raw_input_offsets[offset_index + i + 1] - scifi_raw_input_offsets[offset_index + i];
    atomicAdd(&n_SciFi_clusters, bank_size);
  }
  __syncthreads();
  // Bank size is given in bytes. There are 2 bytes per cluster.
  // 4 bytes are removed for the header.
  // Note that this overestimates slightly the number of clusters
  // due to bank padding in 32b. For v5, it further overestimates the
  // number of clusters due to the merging of clusters.
  if (threadIdx.x == 0) n_SciFi_clusters = n_SciFi_clusters / 2 - 2;
  __syncthreads();

  // if (n_SciFi_clusters >= max_scifi_ut_clusters || n_SciFi_clusters < min_scifi_ut_clusters) return;

  // Check UT clusters
  auto const number_of_ut_raw_banks = ut_raw_input_offsets[0];

  __shared__ uint n_UT_clusters;
  if (threadIdx.x == 0) n_UT_clusters = 0;
  __syncthreads();
  for (uint i = threadIdx.x; i < number_of_ut_raw_banks; i += blockDim.x) {
    // Build UT raw bank from MEP layout
    const auto ut_bank = MEP::raw_bank<UTRawBank>(ut_raw_input, ut_raw_input_offsets,
                                                  event_number, i);
    const int n_UT_clusters_before = atomicAdd(&n_UT_clusters, ut_bank.number_of_hits);
    // if (n_UT_clusters_before + ut_bank.number_of_hits >= max_scifi_ut_clusters) return;
  }
  __syncthreads();

  const auto num_combined_clusters = n_UT_clusters + n_SciFi_clusters;

  if (num_combined_clusters >= max_scifi_ut_clusters) return;
  // if (num_combined_clusters < min_scifi_ut_clusters) return;

  // passed cut
  if (threadIdx.x == 0) {
    const int selected_event = atomicAdd(number_of_selected_events, 1);
    event_list[selected_event] = event_number;
  }
}
