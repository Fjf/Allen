#include "hip/hip_runtime.h"
#include "pv_beamline_multi_fitter.cuh"

__global__ void pv_beamline_multi_fitter(
  uint* dev_atomics_storage,
  uint* dev_velo_track_hit_number,
  PVTrack* dev_pvtracks,
  float* dev_pvtracks_denom,
  float* dev_zpeaks,
  uint* dev_number_of_zpeaks,
  PV::Vertex* dev_multi_fit_vertices,
  uint* dev_number_of_multi_fit_vertices,
  float* dev_beamline,
  const float* dev_pvtrack_z)
{
  const uint number_of_events = gridDim.x;
  const uint event_number = blockIdx.x;
  uint* number_of_multi_fit_vertices = dev_number_of_multi_fit_vertices + event_number;

  const Velo::Consolidated::Tracks velo_tracks {
    (uint*) dev_atomics_storage, dev_velo_track_hit_number, event_number, number_of_events};

  const uint number_of_tracks = velo_tracks.number_of_tracks(event_number);
  const uint event_tracks_offset = velo_tracks.tracks_offset(event_number);

  const float* zseeds = dev_zpeaks + event_number * PV::max_number_vertices;
  const uint number_of_seeds = dev_number_of_zpeaks[event_number];

  const PVTrack* tracks = dev_pvtracks + event_tracks_offset;

  PV::Vertex* vertices = dev_multi_fit_vertices + event_number * PV::max_number_vertices;
  PV::Vertex vertex;
  float* pvtracks_denom = dev_pvtracks_denom + event_tracks_offset;

  const float2 seed_pos_xy {dev_beamline[0], dev_beamline[1]};

  // Find out the tracks we have to process
  // Exploit the fact tracks are sorted by z
  int first_track_in_range = -1;
  uint number_of_tracks_in_range = 0;
  for (uint i = 0; i < number_of_tracks; i++) {
    const auto z = dev_pvtrack_z[event_tracks_offset + i];
    if (zmin < z && z < zmax) {
      if (first_track_in_range == -1) {
        first_track_in_range = i;
      }
      ++number_of_tracks_in_range;
    }
  }

  


  // make sure that we have one thread per seed
  for (uint i_thisseed = threadIdx.x; i_thisseed < number_of_seeds; i_thisseed += blockDim.x) {
    float exp_chi2_0[1200];

    bool converged = false;
    bool accept = true;
    float vtxcov[6] = {0.f, 0.f, 0.f, 0.f, 0.f, 0.f};

    // initial vertex posisiton, use x,y of the beamline and z of the seed
    float2 vtxpos_xy = seed_pos_xy;
    const float seed_pos_z = zseeds[i_thisseed];
    auto vtxpos_z = seed_pos_z;
    float chi2tot = 0.f;
    float sum_weights = 0.f;

    for (uint iter = 0; (iter < maxFitIter && !converged) || iter < minFitIter; ++iter) {
      auto halfD2Chi2DX2_00 = 0.f;
      auto halfD2Chi2DX2_11 = 0.f;
      auto halfD2Chi2DX2_20 = 0.f;
      auto halfD2Chi2DX2_21 = 0.f;
      auto halfD2Chi2DX2_22 = 0.f;
      float3 halfDChi2DX {0.f, 0.f, 0.f};
      sum_weights = 0.f;

      uint nselectedtracks = 0;
      chi2tot = 0.f;
      float local_chi2tot = 0.f;
      float local_sum_weights = 0.f;

      for (uint i = threadIdx.y; i < number_of_tracks_in_range; i += blockDim.y) {
        // compute the chi2
        const PVTrackInVertex& trk = tracks[first_track_in_range + i];

        const auto dz = vtxpos_z - trk.z;
        const float2 res = vtxpos_xy - (trk.x + trk.tx * dz);
        const auto chi2 = res.x * res.x * trk.W_00 + res.y * res.y * trk.W_11;
        if(iter == 0) exp_chi2_0[i] = expf(chi2 * (-0.5f));


        // compute the weight.
        if (chi2 < maxChi2) {
          
          // for more information on the weighted fitting, see e.g.
          // Adaptive Multi-vertex fitting, R. Frühwirth, W. Waltenberger
          // https://cds.cern.ch/record/803519/files/p280.pdf
                    // use seed position for chi2 calculation of nominator
          const float dz_seed = seed_pos_z - trk.z;
          const float2 res_seed = seed_pos_xy - (trk.x + trk.tx * dz_seed);
          const float chi2_seed = res_seed.x * res_seed.x * trk.W_00 + res_seed.y * res_seed.y * trk.W_11;
          const auto nom = expf(chi2_seed * (-0.5f));
          

          const auto denom = chi2CutExp + nom;
          //substract this term to avoid double counting

          const auto track_weight = nom / (denom + pvtracks_denom[first_track_in_range + i] - exp_chi2_0[i]);



          // unfortunately branchy, but reduces fake rate
           if (trk.weight > 0.5) {
              ++nselectedtracks;

            const float3 HWr {
              res.x * trk.W_00, res.y * trk.W_11, -trk.tx.x * res.x * trk.W_00 - trk.tx.y * res.y * trk.W_11};

            halfDChi2DX += HWr * track_weight;
            halfD2Chi2DX2_00 += track_weight * trk.HWH_00;
            halfD2Chi2DX2_11 += track_weight * trk.HWH_11;
            halfD2Chi2DX2_20 += track_weight * trk.HWH_20;
            halfD2Chi2DX2_21 += track_weight * trk.HWH_21;
            halfD2Chi2DX2_22 += track_weight * trk.HWH_22;

            local_chi2tot += track_weight * chi2;
            local_sum_weights += track_weight;
          }
        }
      }

      // __syncthreads();

      // for (int i = 16; i > 0; i = i / 2) {
      //   halfD2Chi2DX2_00 += __shfl_down_sync(0xFFFFFFFF, halfD2Chi2DX2_00, i);
      //   halfD2Chi2DX2_11 += __shfl_down_sync(0xFFFFFFFF, halfD2Chi2DX2_11, i);
      //   halfD2Chi2DX2_20 += __shfl_down_sync(0xFFFFFFFF, halfD2Chi2DX2_20, i);
      //   halfD2Chi2DX2_21 += __shfl_down_sync(0xFFFFFFFF, halfD2Chi2DX2_21, i);
      //   halfD2Chi2DX2_22 += __shfl_down_sync(0xFFFFFFFF, halfD2Chi2DX2_22, i);
      //   halfDChi2DX.x += __shfl_down_sync(0xFFFFFFFF, halfDChi2DX.x, i);
      //   halfDChi2DX.y += __shfl_down_sync(0xFFFFFFFF, halfDChi2DX.y, i);
      //   halfDChi2DX.z += __shfl_down_sync(0xFFFFFFFF, halfDChi2DX.z, i);
      //   local_chi2tot += __shfl_down_sync(0xFFFFFFFF, chi2tot, i);
      //   local_sum_weights += __shfl_down_sync(0xFFFFFFFF, sum_weights, i);
      //   nselectedtracks += __shfl_down_sync(0xFFFFFFFF, nselectedtracks, i);
      // }

      // __syncthreads();

      if (threadIdx.y == 0) {
        chi2tot += local_chi2tot;
        sum_weights += local_sum_weights;

        if (nselectedtracks >= 2) {
          // compute the new vertex covariance using analytical inversion
          const auto a00 = halfD2Chi2DX2_00;
          const auto a11 = halfD2Chi2DX2_11;
          const auto a20 = halfD2Chi2DX2_20;
          const auto a21 = halfD2Chi2DX2_21;
          const auto a22 = halfD2Chi2DX2_22;

          const auto det = a00 * (a22 * a11 - a21 * a21) + a20 * (-a11 * a20);
          const auto inv_det = 1.f / det;

          // maybe we should catch the case when det = 0
          // if (det == 0) return false;

          vtxcov[0] = (a22 * a11 - a21 * a21) * inv_det;
          vtxcov[1] = -(-a20 * a21) * inv_det;
          vtxcov[2] = (a22 * a00 - a20 * a20) * inv_det;
          vtxcov[3] = (-a20 * a11) * inv_det;
          vtxcov[4] = -(a21 * a00) * inv_det;
          vtxcov[5] = (a11 * a00) * inv_det;

          const float2 delta_xy {
            -1.f * (vtxcov[0] * halfDChi2DX.x + vtxcov[1] * halfDChi2DX.y + vtxcov[3] * halfDChi2DX.z),
            -1.f * (vtxcov[1] * halfDChi2DX.x + vtxcov[2] * halfDChi2DX.y + vtxcov[4] * halfDChi2DX.z)};

          const auto delta_z =
            -1.f * (vtxcov[3] * halfDChi2DX.x + vtxcov[4] * halfDChi2DX.y + vtxcov[5] * halfDChi2DX.z);
          chi2tot += delta_xy.x * halfDChi2DX.x + delta_xy.y * halfDChi2DX.y + delta_z * halfDChi2DX.z;

          // update the position
          vtxpos_xy = vtxpos_xy + delta_xy;
          vtxpos_z = vtxpos_z + delta_z;
          converged = fabsf(delta_z) < maxDeltaZConverged;
        }
        else {
          // Finish loop and do not accept vertex
          converged = true;
          accept = false;
        }
      }

      // converged = __any_sync(0xFFFFFFFF, converged);
    } // end iteration loop

    if (accept && threadIdx.y == 0) {
      vertex.chi2 = chi2tot;
      vertex.setPosition(vtxpos_xy, vtxpos_z);
      vertex.setCovMatrix(vtxcov);
      vertex.nTracks = sum_weights;

      // TODO integrate beamline position
      const auto beamlinedx = vertex.position.x - dev_beamline[0];
      const auto beamlinedy = vertex.position.y - dev_beamline[1];
      const auto beamlinerho2 = beamlinedx * beamlinedx + beamlinedy * beamlinedy;
      if (vertex.nTracks >= 2.f && beamlinerho2 < maxVertexRho2) {
        uint vertex_index = atomicAdd(number_of_multi_fit_vertices, 1);
        vertices[vertex_index] = vertex;
      }
    }
  }
}
