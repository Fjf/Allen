#include "hip/hip_runtime.h"
#include "blpv_extrapolate.cuh"


__global__ void blpv_extrapolate(uint* dev_kalmanvelo_states,
  int * dev_atomics_storage,
  uint* dev_velo_track_hit_number) {



  const uint number_of_events = gridDim.x;
  const uint event_number = blockIdx.x;

  const Velo::Consolidated::Tracks velo_tracks {(uint*) dev_atomics_storage, dev_velo_track_hit_number, event_number, number_of_events};
  const Velo::Consolidated::States velo_states {dev_kalmanvelo_states, velo_tracks.total_number_of_tracks};
  const uint number_of_tracks_event = velo_tracks.number_of_tracks(event_number);
  const uint event_tracks_offset = velo_tracks.tracks_offset(event_number);


}