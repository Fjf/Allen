#include "hip/hip_runtime.h"
#include "TrackBeamLineVertexFinder.cuh"

__host__ __device__ PVTrack::PVTrack(const KalmanVeloState& state, float dz) :
  z {float(state.z + dz)}, x {float(state.x + dz * state.tx), float(state.y + dz * state.ty)}, tx {float(state.tx),
                                                                                                   float(state.ty)}
{

  float state_tmp_c00 = state.c00;
  float state_tmp_c11 = state.c11;

  float dz2 = dz * dz;

  // TODO: check if fabsf is needed here
  state_tmp_c00 += dz2 * state.c22 + 2.f * fabsf(dz * state.c20);
  state_tmp_c11 += dz2 * state.c33 + 2.f * fabsf(dz * state.c31);
  W_00 = 1.f / state_tmp_c00;
  W_11 = 1.f / state_tmp_c11;
}
