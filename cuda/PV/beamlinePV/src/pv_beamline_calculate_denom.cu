#include "hip/hip_runtime.h"
#include "pv_beamline_calculate_denom.cuh"
#include "Invoke.cuh"

void pv_beamline_calculate_denom_t::invoke() {
  invoke_helper(handler);
}

__global__ void pv_beamline_calculate_denom(
  uint* dev_atomics_storage,
  uint* dev_velo_track_hit_number,
  PVTrack* dev_pvtracks,
  float* dev_pvtracks_denom,
  float* dev_zpeaks,
  uint* dev_number_of_zpeaks)
{
  const uint number_of_events = gridDim.x;
  const uint event_number = blockIdx.x;

  const Velo::Consolidated::Tracks velo_tracks {
    (uint*) dev_atomics_storage, dev_velo_track_hit_number, event_number, number_of_events};

  const uint number_of_tracks = velo_tracks.number_of_tracks(event_number);
  const uint event_tracks_offset = velo_tracks.tracks_offset(event_number);

  const float* zseeds = dev_zpeaks + event_number * PV::max_number_vertices;
  const uint number_of_seeds = dev_number_of_zpeaks[event_number];

  const PVTrack* tracks = dev_pvtracks + event_tracks_offset;
  float* pvtracks_denom = dev_pvtracks_denom + event_tracks_offset;

  // Precalculate all track denoms
  for (uint i = threadIdx.x; i < number_of_tracks; i += blockDim.x) {
    auto track_denom = 0.f;
    const auto track = tracks[i];

    for (uint j = 0; j < number_of_seeds; ++j) {
      const auto dz = zseeds[j] - track.z;
      const float2 res = track.x + track.tx * dz;
      const auto chi2 = res.x * res.x * track.W_00 + res.y * res.y * track.W_11;
      track_denom += expf(chi2 * (-0.5f));
    }

    pvtracks_denom[i] = track_denom;
  }
}
