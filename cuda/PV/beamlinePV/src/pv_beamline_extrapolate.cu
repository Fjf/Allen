#include "hip/hip_runtime.h"
#include "pv_beamline_extrapolate.cuh"

__global__ void pv_beamline_extrapolate(
  char* dev_velo_kalman_beamline_states,
  uint* dev_atomics_storage,
  uint* dev_velo_track_hit_number,
  PVTrack* dev_pvtracks)
{
  const uint number_of_events = gridDim.x;
  const uint event_number = blockIdx.x;

  const Velo::Consolidated::Tracks velo_tracks {
    (uint*) dev_atomics_storage, dev_velo_track_hit_number, event_number, number_of_events};
  const Velo::Consolidated::KalmanStates velo_states =
    Velo::Consolidated::KalmanStates(dev_velo_kalman_beamline_states, velo_tracks.total_number_of_tracks);
  const uint number_of_tracks_event = velo_tracks.number_of_tracks(event_number);
  const uint event_tracks_offset = velo_tracks.tracks_offset(event_number);

  for (auto index = threadIdx.x; index < number_of_tracks_event; index += blockDim.x) {
    KalmanVeloState s = velo_states.get(event_tracks_offset + index);
    PatPV::XYZPoint beamline {0.f, 0.f, 0.f};
    const auto tx = s.tx;
    const auto ty = s.ty;
    float dz = (tx * (beamline.x - s.x) + ty * (beamline.y - s.y)) / (tx * tx + ty * ty);

    if (dz * s.c20 < 0.f || dz * s.c31 < 0.f) dz = -9999.f;
    PVTrack pvtrack = PVTrack {s, dz};
    dev_pvtracks[event_tracks_offset + index] = pvtrack;
  }
}
