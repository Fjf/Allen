#include "hip/hip_runtime.h"
#include "pv_beamline_extrapolate.cuh"

__global__ void pv_beamline_extrapolate(
  char* dev_velo_kalman_beamline_states,
  uint* dev_atomics_storage,
  uint* dev_velo_track_hit_number,
  PVTrack* dev_pvtracks,
  float* dev_pvtrack_z)
{
  const uint number_of_events = gridDim.x;
  const uint event_number = blockIdx.x;

  const Velo::Consolidated::Tracks velo_tracks {
    (uint*) dev_atomics_storage, dev_velo_track_hit_number, event_number, number_of_events};
  const Velo::Consolidated::KalmanStates velo_states =
    Velo::Consolidated::KalmanStates(dev_velo_kalman_beamline_states, velo_tracks.total_number_of_tracks);
  const uint number_of_tracks_event = velo_tracks.number_of_tracks(event_number);
  const uint event_tracks_offset = velo_tracks.tracks_offset(event_number);
  const uint total_number_of_tracks = velo_tracks.total_number_of_tracks;

  for (uint index = threadIdx.x; index < number_of_tracks_event; index += blockDim.x) {
    const KalmanVeloState s = velo_states.get(event_tracks_offset + index);
    PatPV::XYZPoint beamline {0.f, 0.f, 0.f};
    const float dz = (s.tx * (beamline.x - s.x) + s.ty * (beamline.y - s.y)) / (s.tx * s.tx + s.ty * s.ty);
    
    float z = -9999.f;
    if (dz * s.c20 >= 0.f && dz * s.c31 >= 0.f) {
      z = s.z + dz;
    }

    dev_pvtrack_z[total_number_of_tracks + event_tracks_offset + index] = z;
  }

  __syncthreads();

  // Insert in order
  for (uint index = threadIdx.x; index < number_of_tracks_event; index += blockDim.x) {
    const auto z = dev_pvtrack_z[total_number_of_tracks + event_tracks_offset + index];
    uint insert_position = 0;

    for (uint other = 0; other < number_of_tracks_event; ++other) {
      const auto other_z = dev_pvtrack_z[total_number_of_tracks + event_tracks_offset + other];
      insert_position += z > other_z || (z == other_z && index > other);
    }

    const KalmanVeloState s = velo_states.get(event_tracks_offset + index);
    PVTrack pvtrack = PVTrack {s, z - s.z};
    dev_pvtracks[event_tracks_offset + insert_position] = pvtrack;
    dev_pvtrack_z[event_tracks_offset + index] = z;
  }
}
