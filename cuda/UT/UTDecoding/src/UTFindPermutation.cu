#include "hip/hip_runtime.h"
#include "UTFindPermutation.cuh"
#include <cstdio>

__global__ void ut_find_permutation(
  uint32_t* dev_ut_hits,
  uint32_t* dev_ut_hit_offsets,
  uint* dev_hit_permutations,
  const uint* dev_unique_x_sector_layer_offsets,
  const uint* dev_unique_x_sector_offsets,
  const float* dev_unique_sector_xs)
{
  const uint number_of_events = gridDim.x;
  const uint event_number = blockIdx.x;
  const uint sector_group_number = blockIdx.y;
  const uint number_of_unique_x_sectors = dev_unique_x_sector_layer_offsets[4];
  
  const UTHitOffsets ut_hit_offsets {dev_ut_hit_offsets, event_number, number_of_unique_x_sectors, dev_unique_x_sector_layer_offsets};
  const UTHits ut_hits {dev_ut_hits, dev_ut_hit_offsets[number_of_events * number_of_unique_x_sectors]};

  // // Prints out all hits
  // if (blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0) {
  //   printf(" Sector group %i, x %f:\n", sector_group_number, dev_unique_sector_xs[sector_group_number]);
  //   uint group_offset = ut_hit_offsets.sector_group_offset(sector_group_number);
  //   uint n_hits_group = ut_hit_offsets.sector_group_number_of_hits(sector_group_number);
  //   for (int j=0; j<n_hits_group; ++j) {
  //     const auto hit_index = group_offset + j;
  //     // printf("  yBegin = %f, yEnd = %f, zAtYEq0 = %f, xAtYEq0 = %f, weight = %f, highThreshold = %u \n",
  //     //  ut_hits.yBegin[hit_index],
  //     //  ut_hits.yEnd[hit_index],
  //     //  ut_hits.zAtYEq0[hit_index],
  //     //  ut_hits.xAtYEq0[hit_index],
  //     //  ut_hits.weight[hit_index],
  //     //  ut_hits.highThreshold[hit_index]);
  //     printf("  yBegin = %f\n", ut_hits.yBegin[hit_index]);
  //   }
  // }

  const uint sector_group_offset = ut_hit_offsets.sector_group_offset(sector_group_number);
  const uint sector_group_number_of_hits = ut_hit_offsets.sector_group_number_of_hits(sector_group_number);

  if (sector_group_number_of_hits > 0) {
    // Load yBegin into a shared memory container
    // TODO: Find a proper maximum and cover corner cases
    __shared__ float s_y_begin [256];
    assert(sector_group_number_of_hits < 256);

    for (int i=threadIdx.x; i<sector_group_number_of_hits; i+=blockDim.x) {
      s_y_begin[i] = ut_hits.yBegin[sector_group_offset + i];
    }

    __syncthreads();

    // Note: This could be a specialization of find_permutation,
    //       but we would need to parameterize the hit_start in sector_group.
    //       At this stage, it would make sense perhaps to have one permutation
    //       specialization for shared memory cases, and one for global cases.

    // Sort according to the natural order in s_y_begin
    // Store the permutation found into dev_hit_permutations
    const auto sort_function = [] (const int a, const int b) -> int {
      return (s_y_begin[a] > s_y_begin[b]) - (s_y_begin[a] < s_y_begin[b]);
    };

    for (uint i=threadIdx.x; i<sector_group_number_of_hits; i+=blockDim.x) {
      const int hit_index = i;
      uint position = 0;
      for (uint j = 0; j < sector_group_number_of_hits; ++j) {
        const int other_hit_index = j;
        const int sort_result = sort_function(hit_index, other_hit_index);
        position += sort_result>0 || (sort_result==0 && i>j);
      }
      assert(position < sector_group_number_of_hits);
      dev_hit_permutations[sector_group_offset + position] = sector_group_offset + hit_index; 
    }
  }
}
