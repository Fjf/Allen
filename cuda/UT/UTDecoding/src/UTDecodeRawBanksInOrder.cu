#include "hip/hip_runtime.h"
#include "UTDecodeRawBanksInOrder.cuh"

__global__ void ut_decode_raw_banks_in_order::ut_decode_raw_banks_in_order(
  ut_decode_raw_banks_in_order::Parameters parameters,
  const char* ut_boards,
  const char* ut_geometry,
  const uint* dev_ut_region_offsets,
  const uint* dev_unique_x_sector_layer_offsets)
{
  const uint32_t number_of_events = gridDim.x;
  const uint32_t event_number = blockIdx.x;
  const uint selected_event_number = parameters.dev_event_list[event_number];

  const uint layer_number = blockIdx.y;
  const uint32_t event_offset = parameters.dev_ut_raw_input_offsets[selected_event_number];

  const uint number_of_unique_x_sectors = dev_unique_x_sector_layer_offsets[UT::Constants::n_layers];

  const UT::HitOffsets ut_hit_offsets {
    parameters.dev_ut_hit_offsets, event_number, number_of_unique_x_sectors, dev_unique_x_sector_layer_offsets};
  UT::Hits ut_hits {parameters.dev_ut_hits, parameters.dev_ut_hit_offsets[number_of_events * number_of_unique_x_sectors]};

  const UTRawEvent raw_event(parameters.dev_ut_raw_input + event_offset);
  const UTBoards boards(ut_boards);
  const UTGeometry geometry(ut_geometry);

  // if (threadIdx.x==0) {
  //   printf("%i, %i\n", event_hit_starting_offset, ut_hit_offsets.event_number_of_hits());
  // }

  const uint layer_offset = ut_hit_offsets.layer_offset(layer_number);
  const uint layer_number_of_hits = ut_hit_offsets.layer_number_of_hits(layer_number);

  for (uint i = threadIdx.x; i < layer_number_of_hits; i += blockDim.x) {
    const uint hit_index = layer_offset + i;
    const uint32_t raw_bank_hit_index = ut_hits.raw_bank_index(parameters.dev_ut_hit_permutations[hit_index]);
    const uint raw_bank_index = raw_bank_hit_index >> 24;
    const uint hit_index_inside_raw_bank = raw_bank_hit_index & 0xFFFFFF;

    const UTRawBank raw_bank = raw_event.getUTRawBank(raw_bank_index);
    const uint16_t value = raw_bank.data[hit_index_inside_raw_bank];
    const uint32_t nStripsPerHybrid = boards.stripsPerHybrids[raw_bank.sourceID];

    // Extract values from raw_data
    const uint32_t fracStrip = (value & UT::Decoding::frac_mask) >> UT::Decoding::frac_offset;
    const uint32_t channelID = (value & UT::Decoding::chan_mask) >> UT::Decoding::chan_offset;
    // const uint32_t threshold = (value & UT::Decoding::thre_mask) >> UT::Decoding::thre_offset;

    // Calculate the relative index of the corresponding board
    const uint32_t index = channelID / nStripsPerHybrid;
    const uint32_t strip = channelID - (index * nStripsPerHybrid) + 1;

    const uint32_t fullChanIndex = raw_bank.sourceID * UT::Decoding::ut_number_of_sectors_per_board + index;
    const uint32_t station = boards.stations[fullChanIndex] - 1;
    const uint32_t layer = boards.layers[fullChanIndex] - 1;
    const uint32_t detRegion = boards.detRegions[fullChanIndex] - 1;
    const uint32_t sector = boards.sectors[fullChanIndex] - 1;
    const uint32_t chanID = boards.chanIDs[fullChanIndex];

    // Calculate the index to get the geometry of the board
    const uint32_t idx = station * UT::Decoding::ut_number_of_sectors_per_board + layer * 3 + detRegion;
    const uint32_t idx_offset = dev_ut_region_offsets[idx] + sector;

    const uint32_t firstStrip = geometry.firstStrip[idx_offset];
    const float pitch = geometry.pitch[idx_offset];
    const float dy = geometry.dy[idx_offset];
    const float dp0diX = geometry.dp0diX[idx_offset];
    const float dp0diY = geometry.dp0diY[idx_offset];
    const float dp0diZ = geometry.dp0diZ[idx_offset];
    const float p0X = geometry.p0X[idx_offset];
    const float p0Y = geometry.p0Y[idx_offset];
    const float p0Z = geometry.p0Z[idx_offset];

    const float numstrips = 0.25f * fracStrip + strip - firstStrip;

    // Calculate values of the hit
    const float yBegin = p0Y + numstrips * dp0diY;
    const float yEnd = dy + yBegin;
    const float zAtYEq0 = p0Z + numstrips * dp0diZ;
    const float xAtYEq0 = p0X + numstrips * dp0diX;
    const float weight = 12.f / (pitch * pitch);

    // const uint32_t highThreshold = threshold;
    const uint32_t channelStripID = chanID + strip;
    const uint32_t LHCbID = (((uint32_t) 0xB) << 28) | channelStripID;
    // const uint32_t planeCode = 2 * station + (layer & 1);

    ut_hits.yBegin(hit_index) = yBegin;
    ut_hits.yEnd(hit_index) = yEnd;
    ut_hits.zAtYEq0(hit_index) = zAtYEq0;
    ut_hits.xAtYEq0(hit_index) = xAtYEq0;
    ut_hits.weight(hit_index) = weight;
    ut_hits.id(hit_index) = LHCbID;
  }
}
