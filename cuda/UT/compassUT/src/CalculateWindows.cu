#include "hip/hip_runtime.h"
#include "BinarySearch.cuh"
#include "VeloTools.cuh"
#include "CalculateWindows.cuh"
#include "BinarySearchFirstCandidate.cuh"

//=============================================================================
// Reject tracks outside of acceptance or pointing to the beam pipe
//=============================================================================
__device__ bool velo_track_in_UTA_acceptance(const MiniState& state)
{
  const float xMidUT = state.x + state.tx * (PrVeloUTConst::zMidUT - state.z);
  const float yMidUT = state.y + state.ty * (PrVeloUTConst::zMidUT - state.z);

  if (xMidUT * xMidUT + yMidUT * yMidUT < PrVeloUTConst::centralHoleSize * PrVeloUTConst::centralHoleSize) return false;
  if ((std::abs(state.tx) > PrVeloUTConst::maxXSlope) || (std::abs(state.ty) > PrVeloUTConst::maxYSlope)) return false;

  if (
    PrVeloUTConst::passTracks && std::abs(xMidUT) < PrVeloUTConst::passHoleSize &&
    std::abs(yMidUT) < PrVeloUTConst::passHoleSize) {
    return false;
  }

  return true;
}

//=========================================================================
// Check if hit is inside tolerance and refine by Y
//=========================================================================
__host__ __device__ __inline__ bool is_valid_tol_refine (
  const int hit_index,
  const UTHits& ut_hits,
  const MiniState& velo_state,
  const float normFactNum,
  const float xTol,
  const float dxDy)
{
  const float xTolNormFact = xTol * (1.0f / normFactNum);

  const float zInit = ut_hits.zAtYEq0[hit_index];
  const float yApprox = velo_state.y + velo_state.ty * (zInit - velo_state.z);
  const float xOnTrackProto = velo_state.x + velo_state.tx * (zInit - velo_state.z);

  const float xx = ut_hits.xAt(hit_index, yApprox, dxDy);
  const float dx = xx - xOnTrackProto;

  if (dx < -xTolNormFact || dx > xTolNormFact) return false;

  // Now refine the tolerance in Y
  if (ut_hits.isNotYCompatible(
        hit_index, yApprox, PrVeloUTConst::yTol + PrVeloUTConst::yTolSlope * std::abs(dx * (1.0f / normFactNum))))
    return false;

  return true;
}

//=============================================================================
// Get the windows
//=============================================================================
__device__ std::tuple<int, int, int, int, int, int> calculate_windows(
  const int i_track,
  const int layer,
  const MiniState& velo_state,
  const float* fudge_factors,
  const UTHits& ut_hits,
  const UTHitOffsets& ut_hit_offsets,
  const float* ut_dxDy,
  const float* dev_unique_sector_xs,
  const uint* dev_unique_x_sector_layer_offsets,
  const Velo::Consolidated::Tracks& velo_tracks)
{
  // -- This is hardcoded, so faster
  // -- If you ever change the Table in the magnet tool, this will be wrong
  const float absSlopeY = std::abs(velo_state.ty);
  const int index       = (int) (absSlopeY * 100 + 0.5f);
  assert(3 + 4 * index < PrUTMagnetTool::N_dxLay_vals);
  const float normFact[4]{
    fudge_factors[4 * index], fudge_factors[1 + 4 * index], fudge_factors[2 + 4 * index], fudge_factors[3 + 4 * index]};

  // -- this 500 seems a little odd...
  // to do: change back!
  const float invTheta = std::min(500.0f, 1.0f / std::sqrt(velo_state.tx * velo_state.tx + velo_state.ty * velo_state.ty));
  const float minMom   = std::max(PrVeloUTConst::minPT * invTheta, 1.5f * Gaudi::Units::GeV);
  const float xTol     = std::abs(1.0f / (PrVeloUTConst::distToMomentum * minMom));
  // const float yTol     = PrVeloUTConst::yTol + PrVeloUTConst::yTolSlope * xTol;

  int layer_offset = ut_hit_offsets.layer_offset(layer);

  const float dx_dy      = ut_dxDy[layer];
  const float z_at_layer = ut_hits.zAtYEq0[layer_offset];
  const float y_track     = velo_state.y + velo_state.ty * (z_at_layer - velo_state.z);
  const float x_track     = velo_state.x + velo_state.tx * (z_at_layer - velo_state.z);
  const float invNormFact = 1.0f / normFact[layer];

  // Second sector group search
  // const float tolerance_in_x = xTol * invNormFact;

  // Find sector group for lowerBoundX and upperBoundX
  const int first_sector_group_in_layer = dev_unique_x_sector_layer_offsets[layer];
  const int last_sector_group_in_layer  = dev_unique_x_sector_layer_offsets[layer + 1];
  const int sector_group_size           = last_sector_group_in_layer - first_sector_group_in_layer;

  const int local_sector_group =
    binary_search_leftmost(dev_unique_sector_xs + first_sector_group_in_layer, sector_group_size, x_track);
  int sector_group = first_sector_group_in_layer + local_sector_group;

  int first_candidate = -1, last_candidate = -1;
  int left_group_first_candidate = -1, left_group_last_candidate = -1;
  int right_group_first_candidate = -1, right_group_last_candidate = -1;
  if (sector_group != 0) {
    // The sector we are interested on is sector_group - 1
    sector_group -= 1;
    const auto sector_candidates = find_candidates_in_sector_group(
      ut_hits,
      ut_hit_offsets,
      velo_state,
      dev_unique_sector_xs,
      x_track,
      y_track,
      dx_dy,
      normFact[layer],
      invNormFact,
      xTol,
      sector_group
    );

    first_candidate = std::get<0>(sector_candidates);
    last_candidate = std::get<1>(sector_candidates);

    // Left group
    const int left_group = sector_group - 1;
    if (left_group >= first_sector_group_in_layer) {
      // We found a sector group with potentially compatible hits
      // Look for them
      const auto left_group_candidates = find_candidates_in_sector_group(
        ut_hits,
        ut_hit_offsets,
        velo_state,
        dev_unique_sector_xs,
        x_track,
        y_track,
        dx_dy,
        normFact[layer],
        invNormFact,
        xTol,
        left_group
      );

      left_group_first_candidate = std::get<0>(left_group_candidates);
      left_group_last_candidate = std::get<1>(left_group_candidates);
    }

    // Right group
    const int right_group = sector_group + 1;
    if (right_group < last_sector_group_in_layer) {
      // We found a sector group with potentially compatible hits
      // Look for them
      const auto right_group_candidates = find_candidates_in_sector_group(
        ut_hits,
        ut_hit_offsets,
        velo_state,
        dev_unique_sector_xs,
        x_track,
        y_track,
        dx_dy,
        normFact[layer],
        invNormFact,
        xTol,
        right_group
      );

      right_group_first_candidate = std::get<0>(right_group_candidates);
      right_group_last_candidate = std::get<1>(right_group_candidates);
    }
  }

  return {first_candidate, last_candidate,
    left_group_first_candidate, left_group_last_candidate,
    right_group_first_candidate, right_group_last_candidate
  };
} 

__device__ std::tuple<int, int> find_candidates_in_sector_group(
  const UTHits& ut_hits,
  const UTHitOffsets& ut_hit_offsets,
  const MiniState& velo_state,
  const float* dev_unique_sector_xs,
  const float x_track,
  const float y_track,
  const float dx_dy,
  const float normFact,
  const float invNormFact,
  const float xTol,
  const int sector_group)
{
  const float x_at_left_sector  = dev_unique_sector_xs[sector_group];
  const float x_at_right_sector = dev_unique_sector_xs[sector_group + 1];
  const float xx_at_left_sector  = x_at_left_sector + y_track * dx_dy;
  const float xx_at_right_sector = x_at_right_sector + y_track * dx_dy;
  const float dx_max = std::max(xx_at_left_sector - x_track, xx_at_right_sector - x_track);

  const float tol = PrVeloUTConst::yTol + PrVeloUTConst::yTolSlope * std::abs(dx_max * invNormFact);
  const uint sector_group_offset = ut_hit_offsets.sector_group_offset(sector_group);

  int first_candidate = -1, last_candidate = -1;
  first_candidate = binary_search_first_candidate(
    ut_hits.yEnd + sector_group_offset,
    ut_hit_offsets.sector_group_number_of_hits(sector_group),
    y_track,
    tol,
    [&] (const auto value, const auto array_element, const int index, const float margin) {
      return (value + margin > ut_hits.yBegin[sector_group_offset + index] && value - margin < array_element);
    });

  if (first_candidate != -1) {
    last_candidate = binary_search_second_candidate(
      ut_hits.yBegin + sector_group_offset + first_candidate,
      ut_hit_offsets.sector_group_number_of_hits(sector_group) - first_candidate,
      y_track,
      tol);
    first_candidate += sector_group_offset;
    last_candidate = last_candidate == 0 ? first_candidate + 1 : first_candidate + last_candidate;

    bool found = false;

    // refine first candidate
    for (int i=first_candidate; i<last_candidate; ++i) {
      if (is_valid_tol_refine(i, ut_hits, velo_state, normFact, xTol, dx_dy)) {
        first_candidate = i;
        found = true;
        break;
      }
    }

    if (!found) {
      first_candidate = -1;
      last_candidate = -1;      
    } 
    else {
      bool last_found = false;

      for (int i=last_candidate; i>first_candidate; --i) {
        if (is_valid_tol_refine(i, ut_hits, velo_state, normFact, xTol,dx_dy)) {
          last_candidate = i;
          last_found = true;
          break;
        } 
      }

      if (!last_found) {
        last_candidate = first_candidate + 1;
      }      
    }
  }

  return {first_candidate, last_candidate};
}
