#include "hip/hip_runtime.h"
#include "FindBestHits.cuh"

//=========================================================================
// Get the best 3 or 4 hits, 1 per layer, for a given VELO track
// When iterating over a panel, 3 windows are given, we set the index
// to be only in the windows
//=========================================================================
__device__ std::tuple<int,int,int,int,BestParams> find_best_hits(
  const int* win_size_shared,
  const UTHits& ut_hits,
  const UTHitOffsets& ut_hit_count,
  const MiniState& velo_state,
  const float* ut_dxDy)
{
  const float yyProto = velo_state.y - velo_state.ty * velo_state.z;
  WindowIndicator win_ranges(win_size_shared); 
  const auto* ranges = win_ranges.get_track_candidates(threadIdx.x);

  int best_hits [4] = { -1, -1, -1, -1};

  bool found = false;
  bool forward = false;

  // float xhitLayer0, xhitLayer2;
  // float zhitLayer0, zhitLayer2;
  // float tx;

  int best_number_of_hits = 3;
  int best_fit = PrVeloUTConst::maxPseudoChi2;
  BestParams best_params;

  // Find compatible doublet forward
  const int total_hits_2layers_0 = ranges->layer[0].size0 + ranges->layer[0].size1 + ranges->layer[0].size2 +
                                   ranges->layer[3].size0 + ranges->layer[3].size1 + ranges->layer[3].size2;

  for (int i=0; !found && i<total_hits_2layers_0; ++i) {
    const int i_hit0 = set_index(i, ranges->layer[0], ranges->layer[3]);

    // Get the hit to check with next layer
    const float yy0 = yyProto + (velo_state.ty * ut_hits.zAtYEq0[i_hit0]);
    const auto xhitLayer0 = ut_hits.xAt(i_hit0, yy0, ut_dxDy[0]);
    const auto zhitLayer0 = ut_hits.zAtYEq0[i_hit0];

    // set range for nested layer if forward or backward
    LayerCandidates layer_2;
    if (i < ranges->layer[0].size0 + ranges->layer[0].size1 + ranges->layer[0].size2) {
      forward = true;
      layer_2 = ranges->layer[2];
    } else {
      forward = false;
      layer_2 = ranges->layer[1];
    }

    // loop over layer 2
    for (int j=0; !found && j<layer_2.size0 + layer_2.size1 + layer_2.size2 ; ++j) {
      int i_hit2 = set_index(j, layer_2);

      // Get the hit to check with next layer
      const float yy2 = yyProto + (velo_state.ty * ut_hits.zAtYEq0[i_hit2]);
      const auto xhitLayer2 = ut_hits.xAt(i_hit2, yy2, ut_dxDy[2]);
      const auto zhitLayer2 = ut_hits.zAtYEq0[i_hit2];

      const auto tx = (xhitLayer2 - xhitLayer0) / (zhitLayer2 - zhitLayer0);
      if (std::abs(tx - velo_state.tx) <= PrVeloUTConst::deltaTx2) {

        int temp_best_hits [4] = { i_hit0,-1 ,i_hit2, -1};

        const int layers [2] = {
          forward ? 1 : 2,
          forward ? 3 : 0
        };

        float hitTol = PrVeloUTConst::hitTol2;

        // search for triplet in layer1
        for (int i1=0; i1<ranges->layer[layers[0]].size0 + ranges->layer[layers[0]].size1 + ranges->layer[layers[0]].size2; ++i1) {

          int i_hit1 = set_index(i1, ranges->layer[layers[0]]);

          // Get the hit to check with next layer
          const float yy1 = yyProto + (velo_state.ty * ut_hits.zAtYEq0[i_hit1]);
          const float xhitLayer1 = ut_hits.xAt(i_hit1, yy1, ut_dxDy[layers[0]]);
          const float zhitLayer1 = ut_hits.zAtYEq0[i_hit1];
          const float xextrapLayer1 = xhitLayer0 + tx * (zhitLayer1 - zhitLayer0);

          if (std::abs(xhitLayer1 - xextrapLayer1) < hitTol) {
            hitTol = std::abs(xhitLayer1 - xextrapLayer1);
            temp_best_hits[1] = i_hit1;
          }
        }

        // search for quadruplet in layer3
        hitTol = PrVeloUTConst::hitTol2;
        for (int i3=0; i3<ranges->layer[layers[1]].size0 + ranges->layer[layers[1]].size1 + ranges->layer[layers[1]].size2; ++i3) {

          int i_hit3 = set_index(i3, ranges->layer[layers[1]]);

          // Get the hit to check
          const float yy3 = yyProto + (velo_state.ty * ut_hits.zAtYEq0[i_hit3]);
          const float xhitLayer3 = ut_hits.xAt(i_hit3, yy3, ut_dxDy[layers[1]]);
          const float zhitLayer3 = ut_hits.zAtYEq0[i_hit3];
          const float xextrapLayer3 = xhitLayer2 + tx * (zhitLayer3 - zhitLayer2);
          if (std::abs(xhitLayer3 - xextrapLayer3) < hitTol) {
            hitTol = std::abs(xhitLayer3 - xextrapLayer3);
            temp_best_hits[3] = i_hit3;
          }          
        }



        // Fit the hits to get q/p, chi2
        const auto number_of_hits = 2 + (temp_best_hits[1] != -1) + (temp_best_hits[3] != -1);
        if (number_of_hits >= best_number_of_hits) {
          best_number_of_hits = number_of_hits;
          const auto params = pkick_fit(temp_best_hits, ut_hits, velo_state, ut_dxDy, yyProto, forward);

          if (params.chi2UT < best_fit) {
            found = true;
            best_hits[0] = temp_best_hits[0];
            best_hits[1] = temp_best_hits[1];
            best_hits[2] = temp_best_hits[2];
            best_hits[3] = temp_best_hits[3];
            best_params = params;
          }
        }
      }
    }
  }

  // if (found) {
  //   const int layers [2] = {
  //     forward ? 1 : 2,
  //     forward ? 3 : 0
  //   };

  //   float hitTol = PrVeloUTConst::hitTol2;

  //   // search for triplet in layer1
  //   for (int i1=0; i1<ranges->layer[layers[0]].size0 + ranges->layer[layers[0]].size1 + ranges->layer[layers[0]].size2; ++i1) {

  //     int i_hit1 = set_index(i1, ranges->layer[layers[0]]);

  //     // Get the hit to check with next layer
  //     const float yy1 = yyProto + (velo_state.ty * ut_hits.zAtYEq0[i_hit1]);
  //     const float xhitLayer1 = ut_hits.xAt(i_hit1, yy1, ut_dxDy[layers[0]]);
  //     const float zhitLayer1 = ut_hits.zAtYEq0[i_hit1];
  //     const float xextrapLayer1 = xhitLayer0 + tx * (zhitLayer1 - zhitLayer0);

  //     if (std::abs(xhitLayer1 - xextrapLayer1) < hitTol) {
  //       hitTol = std::abs(xhitLayer1 - xextrapLayer1);
  //       best_hits[1] = i_hit1;
  //     }
  //   }

  //   // search for quadruplet in layer3
  //   hitTol = PrVeloUTConst::hitTol2;
  //   for (int i3=0; i3<ranges->layer[layers[1]].size0 + ranges->layer[layers[1]].size1 + ranges->layer[layers[1]].size2; ++i3) {

  //     int i_hit3 = set_index(i3, ranges->layer[layers[1]]);

  //     // Get the hit to check
  //     const float yy3 = yyProto + (velo_state.ty * ut_hits.zAtYEq0[i_hit3]);
  //     const float xhitLayer3 = ut_hits.xAt(i_hit3, yy3, ut_dxDy[layers[1]]);
  //     const float zhitLayer3 = ut_hits.zAtYEq0[i_hit3];
  //     const float xextrapLayer3 = xhitLayer2 + tx * (zhitLayer3 - zhitLayer2);
  //     if (std::abs(xhitLayer3 - xextrapLayer3) < hitTol) {
  //       hitTol = std::abs(xhitLayer3 - xextrapLayer3);
  //       best_hits[3] = i_hit3;
  //     }          
  //   }

  //   // Fit the hits to get q/p, chi2
  //   const auto number_of_hits = 2 + (best_hits[1] != -1) + (best_hits[3] != -1);
  //   if (number_of_hits >= best_number_of_hits) {
  //     best_number_of_hits = number_of_hits;
  //     const auto params = pkick_fit(best_hits, ut_hits, velo_state, ut_dxDy, yyProto, forward);

  //     if (params.chi2UT < best_fit) {
  //       return {best_hits[0], best_hits[1], best_hits[2], best_hits[3], params};
  //     }
  //   }
  // }
  return {best_hits[0], best_hits[1], best_hits[2], best_hits[3], best_params};
  // return {-1, -1, -1, -1, {}};
}

//=========================================================================
// apply the p-kick method to the triplet/quadruplet
// TODO return the chi2?
// TODO precalculate zDiff (its always the same)
//=========================================================================
__device__ BestParams pkick_fit(
  const int best_hits[N_LAYERS],
  const UTHits& ut_hits,
  const MiniState& velo_state,
  const float* ut_dxDy,
  const float yyProto,
  const bool forward)
{
  BestParams best_params;

  // Helper stuff from velo state
  const float xMidField = velo_state.x + velo_state.tx * (PrVeloUTConst::zKink - velo_state.z);
  const float a = PrVeloUTConst::sigmaVeloSlope * (PrVeloUTConst::zKink - velo_state.z);
  const float wb = 1.0f / (a * a);

  float mat[3] = {wb, wb * PrVeloUTConst::zDiff, wb * PrVeloUTConst::zDiff * PrVeloUTConst::zDiff};
  float rhs[2] = {wb * xMidField, wb * xMidField * PrVeloUTConst::zDiff};

  // add hits
  #pragma unroll
  for (int i = 0; i < N_LAYERS; ++i) {
    int hit_index = best_hits[i];
    if (hit_index >= 0) {
      const float wi = ut_hits.weight[hit_index];
      const int plane_code = forward ? i : N_LAYERS - 1 - i;
      const float dxDy = ut_dxDy[plane_code];
      const float ci = ut_hits.cosT(hit_index, dxDy);
      const float dz = 0.001f * (ut_hits.zAtYEq0[hit_index] - PrVeloUTConst::zMidUT);
      // x_pos_layer
      const float yy = yyProto + (velo_state.ty * ut_hits.zAtYEq0[hit_index]);
      const float ui = ut_hits.xAt(hit_index, yy, dxDy);

      mat[0] += wi * ci;
      mat[1] += wi * ci * dz;
      mat[2] += wi * ci * dz * dz;
      rhs[0] += wi * ui;
      rhs[1] += wi * ui * dz;
    }
  }

  const float denom = 1.0f / (mat[0] * mat[2] - mat[1] * mat[1]);
  const float xSlopeUTFit = 0.001f * (mat[0] * rhs[1] - mat[1] * rhs[0]) * denom;
  const float xUTFit = (mat[2] * rhs[0] - mat[1] * rhs[1]) * denom;

  // new VELO slope x
  const float xb = xUTFit + xSlopeUTFit * (PrVeloUTConst::zKink - PrVeloUTConst::zMidUT);
  const float invKinkVeloDist = 1 / (PrVeloUTConst::zKink - velo_state.z);
  const float xSlopeVeloFit = (xb - velo_state.x) * invKinkVeloDist;
  const float chi2VeloSlope = (velo_state.tx - xSlopeVeloFit) * PrVeloUTConst::invSigmaVeloSlope;

  // chi2 takes chi2 from velo fit + chi2 from UT fit
  float chi2UT = chi2VeloSlope * chi2VeloSlope;
  // add chi2
  int total_num_hits = 0;
  #pragma unroll
  for (int i = 0; i < N_LAYERS; ++i) {
    int hit_index = best_hits[i];
    if (hit_index >= 0) {
      const float zd = ut_hits.zAtYEq0[hit_index];
      const float xd = xUTFit + xSlopeUTFit * (zd - PrVeloUTConst::zMidUT);
      // x_pos_layer
      const int plane_code = forward ? i : N_LAYERS - 1 - i;
      const float dxDy = ut_dxDy[plane_code];
      const float yy = yyProto + (velo_state.ty * ut_hits.zAtYEq0[hit_index]);
      const float x = ut_hits.xAt(hit_index, yy, dxDy);

      const float du = xd - x;
      chi2UT += (du * du) * ut_hits.weight[hit_index];

      // count the number of processed htis
      total_num_hits++;
    }
  }

  chi2UT /= (total_num_hits - 1);

  // Save the best parameters if chi2 is good
  if (chi2UT < PrVeloUTConst::maxPseudoChi2) {
    // calculate q/p
    const float sinInX = xSlopeVeloFit * std::sqrt(1.0f + xSlopeVeloFit * xSlopeVeloFit);
    const float sinOutX = xSlopeUTFit * std::sqrt(1.0f + xSlopeUTFit * xSlopeUTFit);

    best_params.qp = sinInX - sinOutX;
    best_params.chi2UT = chi2UT;
    best_params.n_hits = total_num_hits;
  }

  return best_params;
}

__device__ __inline__ int set_index(
  const int i, 
  const LayerCandidates& layer_cand0,
  const LayerCandidates& layer_cand2)
{
  int hit = -1;
  if (i < layer_cand0.size0) {
    hit = layer_cand0.from0 + i;
  } else if (i < layer_cand0.size0 + layer_cand0.size1) {
    hit = layer_cand0.from1 + i - layer_cand0.size0;
  } else if (i < layer_cand0.size0 + layer_cand0.size1 + layer_cand0.size2) {
    hit = layer_cand0.from2 + i - layer_cand0.size0 - layer_cand0.size1;
  } else if (i < layer_cand0.size0 + layer_cand0.size1 + layer_cand0.size2 + layer_cand2.size0) {
    hit = layer_cand2.from0 + i - (layer_cand0.size0 + layer_cand0.size1 + layer_cand0.size2) ;
  } else if (i < layer_cand0.size0 + layer_cand0.size1 + layer_cand0.size2 + layer_cand2.size0 + layer_cand2.size1) {
    hit = layer_cand2.from1 + i - layer_cand2.size0 - (layer_cand0.size0 + layer_cand0.size1 + layer_cand0.size2);
  } else if (i < layer_cand0.size0 + layer_cand0.size1 + layer_cand0.size2 + layer_cand2.size0 + layer_cand2.size1 + layer_cand2.size2) {
    hit = layer_cand2.from2 + i - layer_cand2.size0 - layer_cand2.size1 - (layer_cand0.size0 + layer_cand0.size1 + layer_cand0.size2);
  }

  return hit;
}

__device__ __inline__ int set_index(
  const int i, 
  const LayerCandidates& layer_cand)
{
  int hit = -1;
  if (i < layer_cand.size0) {
    hit = layer_cand.from0 + i;
  } else if (i < layer_cand.size0 + layer_cand.size1) {
    hit = layer_cand.from1 + i - layer_cand.size0;
  } else if (i < layer_cand.size0 + layer_cand.size1 + layer_cand.size2) {
    hit = layer_cand.from2 + i - layer_cand.size0 - layer_cand.size1;
  } 

  return hit;
}

//=========================================================================
// Check if hit is inside tolerance and refine by Y
//=========================================================================
__device__ __inline__ bool check_tol_refine(
  const int hit_index,
  const UTHits& ut_hits,
  const MiniState& velo_state,
  const float normFactNum,
  const float xTol,
  const float dxDy)
{
  const float xTolNormFact = xTol * (1.0f / normFactNum);

  const float zInit = ut_hits.zAtYEq0[hit_index];
  const float yApprox = velo_state.y + velo_state.ty * (zInit - velo_state.z);
  const float xOnTrackProto = velo_state.x + velo_state.tx * (zInit - velo_state.z);

  const float xx = ut_hits.xAt(hit_index, yApprox, dxDy);
  const float dx = xx - xOnTrackProto;

  if (dx < -xTolNormFact || dx > xTolNormFact) return false;

  // Now refine the tolerance in Y
  if (ut_hits.isNotYCompatible(
        hit_index, yApprox, PrVeloUTConst::yTol + PrVeloUTConst::yTolSlope * std::abs(dx * (1.0f / normFactNum))))
    return false;

  return true;
}