#include "hip/hip_runtime.h"
#include "FindBestHits.cuh"

//=========================================================================
// Get the best 3 or 4 hits, 1 per layer, for a given VELO track
// When iterating over a panel, 3 windows are given, we set the index
// to be only in the windows
//=========================================================================
__device__ void find_best_hits(
  const int* win_size_shared,
  const UTHits& ut_hits,
  const UTHitOffsets& ut_hit_count,
  const MiniState& velo_state,
  const float* ut_dxDy,
  const bool forward,
  int* best_hits,
  BestParams& best_params)
{
  // handle forward / backward cluster search
  int layers[N_LAYERS];
  #pragma unroll
  for (int i_layer = 0; i_layer < N_LAYERS; ++i_layer) {
    if (forward)
      layers[i_layer] = i_layer;
    else
      layers[i_layer] = N_LAYERS - 1 - i_layer;
  }

  const float yyProto = velo_state.y - velo_state.ty * velo_state.z;

  // // Get windows of all layers
  WindowIndicator win_ranges(win_size_shared); 
  const auto* ranges = win_ranges.get_track_candidates(threadIdx.x);

  int best_number_of_hits = 3;
  int temp_best_hits[N_LAYERS] = {-1, -1, -1, -1};
  bool found = false;
  int considered = 0;

  // loop over the 3 windows, putting the index in the windows
  // loop over layer 0
  float best_fit = PrVeloUTConst::maxPseudoChi2;
  for (int i0=0; (!found || considered < CompassUT::max_considered_before_found) &&
       i0<ranges->layer[layers[0]].size0 + ranges->layer[layers[0]].size1 + ranges->layer[layers[0]].size2; ++i0) {

    int i_hit0 = set_index(i0, ranges->layer[layers[0]]);

    // Get the hit to check with next layer
    const float yy0 = yyProto + (velo_state.ty * ut_hits.zAtYEq0[i_hit0]);
    const float xhitLayer0 = ut_hits.xAt(i_hit0, yy0, ut_dxDy[layers[0]]);
    const float zhitLayer0 = ut_hits.zAtYEq0[i_hit0];
    temp_best_hits[0] = i_hit0;

    // loop over layer 2
    for (int i2=0; (!found || considered < CompassUT::max_considered_before_found) &&
         i2<ranges->layer[layers[2]].size0 + ranges->layer[layers[2]].size1 + ranges->layer[layers[2]].size2; ++i2) {

      int i_hit2 = set_index(i2, ranges->layer[layers[2]]);

      // Get the hit to check with next layer
      const float yy2 = yyProto + (velo_state.ty * ut_hits.zAtYEq0[i_hit2]);
      const float xhitLayer2 = ut_hits.xAt(i_hit2, yy2, ut_dxDy[layers[2]]);
      const float zhitLayer2 = ut_hits.zAtYEq0[i_hit2];
      temp_best_hits[2] = i_hit2;

      const float tx = (xhitLayer2 - xhitLayer0) / (zhitLayer2 - zhitLayer0);
      if (std::abs(tx - velo_state.tx) > PrVeloUTConst::deltaTx2) continue;

      float hitTol = PrVeloUTConst::hitTol2;
      temp_best_hits[1] = -1;

      // search for triplet in layer1
      for (int i1=0; i1<ranges->layer[layers[1]].size0 + ranges->layer[layers[1]].size1 + ranges->layer[layers[1]].size2; ++i1) {

        int i_hit1 = set_index(i1, ranges->layer[layers[1]]);

        // Get the hit to check with next layer
        const float yy1 = yyProto + (velo_state.ty * ut_hits.zAtYEq0[i_hit1]);
        const float xhitLayer1 = ut_hits.xAt(i_hit1, yy1, ut_dxDy[layers[1]]);
        const float zhitLayer1 = ut_hits.zAtYEq0[i_hit1];
        const float xextrapLayer1 = xhitLayer0 + tx * (zhitLayer1 - zhitLayer0);

        if (std::abs(xhitLayer1 - xextrapLayer1) < hitTol) {
          hitTol = std::abs(xhitLayer1 - xextrapLayer1);
          temp_best_hits[1] = i_hit1;
        }
      }

      // search for quadruplet in layer3
      temp_best_hits[3] = -1;
      hitTol = PrVeloUTConst::hitTol2;
      for (int i3=0; i3<ranges->layer[layers[3]].size0 + ranges->layer[layers[3]].size1 + ranges->layer[layers[3]].size2; ++i3) {

        int i_hit3 = set_index(i3, ranges->layer[layers[3]]);

        // Get the hit to check
        const float yy3 = yyProto + (velo_state.ty * ut_hits.zAtYEq0[i_hit3]);
        const float xhitLayer3 = ut_hits.xAt(i_hit3, yy3, ut_dxDy[layers[3]]);
        const float zhitLayer3 = ut_hits.zAtYEq0[i_hit3];
        const float xextrapLayer3 = xhitLayer2 + tx * (zhitLayer3 - zhitLayer2);
        if (std::abs(xhitLayer3 - xextrapLayer3) < hitTol) {
          hitTol = std::abs(xhitLayer3 - xextrapLayer3);
          temp_best_hits[3] = i_hit3;
        }          
      }

      // Fit the hits to get q/p, chi2
      const auto temp_number_of_hits = 2 + (temp_best_hits[1] != -1) + (temp_best_hits[3] != -1);
      const auto params = pkick_fit(temp_best_hits, ut_hits, velo_state, ut_dxDy, yyProto, forward);
      ++considered;

      if (params.chi2UT < best_fit && temp_number_of_hits >= best_number_of_hits) {
        best_hits[0] = temp_best_hits[0];
        best_hits[1] = temp_best_hits[1];
        best_hits[2] = temp_best_hits[2];
        best_hits[3] = temp_best_hits[3];
        best_number_of_hits = temp_number_of_hits;
        best_params = params;
        best_fit = params.chi2UT;

        found = true;
      }
    }
  }
}

//=========================================================================
// apply the p-kick method to the triplet/quadruplet
// TODO return the chi2?
// TODO precalculate zDiff (its always the same)
//=========================================================================
__device__ BestParams pkick_fit(
  const int best_hits[N_LAYERS],
  const UTHits& ut_hits,
  const MiniState& velo_state,
  const float* ut_dxDy,
  const float yyProto,
  const bool forward)
{
  BestParams best_params;

  // Helper stuff from velo state
  const float xMidField = velo_state.x + velo_state.tx * (PrVeloUTConst::zKink - velo_state.z);
  const float a = PrVeloUTConst::sigmaVeloSlope * (PrVeloUTConst::zKink - velo_state.z);
  const float wb = 1.0f / (a * a);

  float mat[3] = {wb, wb * PrVeloUTConst::zDiff, wb * PrVeloUTConst::zDiff * PrVeloUTConst::zDiff};
  float rhs[2] = {wb * xMidField, wb * xMidField * PrVeloUTConst::zDiff};

  // add hits
  #pragma unroll
  for (int i = 0; i < N_LAYERS; ++i) {
    int hit_index = best_hits[i];
    if (hit_index >= 0) {
      const float wi = ut_hits.weight[hit_index];
      const int plane_code = forward ? i : N_LAYERS - 1 - i;
      const float dxDy = ut_dxDy[plane_code];
      const float ci = ut_hits.cosT(hit_index, dxDy);
      const float dz = 0.001f * (ut_hits.zAtYEq0[hit_index] - PrVeloUTConst::zMidUT);
      // x_pos_layer
      const float yy = yyProto + (velo_state.ty * ut_hits.zAtYEq0[hit_index]);
      const float ui = ut_hits.xAt(hit_index, yy, dxDy);

      mat[0] += wi * ci;
      mat[1] += wi * ci * dz;
      mat[2] += wi * ci * dz * dz;
      rhs[0] += wi * ui;
      rhs[1] += wi * ui * dz;
    }
  }

  const float denom = 1.0f / (mat[0] * mat[2] - mat[1] * mat[1]);
  const float xSlopeUTFit = 0.001f * (mat[0] * rhs[1] - mat[1] * rhs[0]) * denom;
  const float xUTFit = (mat[2] * rhs[0] - mat[1] * rhs[1]) * denom;

  // new VELO slope x
  const float xb = xUTFit + xSlopeUTFit * (PrVeloUTConst::zKink - PrVeloUTConst::zMidUT);
  const float invKinkVeloDist = 1 / (PrVeloUTConst::zKink - velo_state.z);
  const float xSlopeVeloFit = (xb - velo_state.x) * invKinkVeloDist;
  const float chi2VeloSlope = (velo_state.tx - xSlopeVeloFit) * PrVeloUTConst::invSigmaVeloSlope;

  // chi2 takes chi2 from velo fit + chi2 from UT fit
  float chi2UT = chi2VeloSlope * chi2VeloSlope;
  // add chi2
  int total_num_hits = 0;
  #pragma unroll
  for (int i = 0; i < N_LAYERS; ++i) {
    int hit_index = best_hits[i];
    if (hit_index >= 0) {
      const float zd = ut_hits.zAtYEq0[hit_index];
      const float xd = xUTFit + xSlopeUTFit * (zd - PrVeloUTConst::zMidUT);
      // x_pos_layer
      const int plane_code = forward ? i : N_LAYERS - 1 - i;
      const float dxDy = ut_dxDy[plane_code];
      const float yy = yyProto + (velo_state.ty * ut_hits.zAtYEq0[hit_index]);
      const float x = ut_hits.xAt(hit_index, yy, dxDy);

      const float du = xd - x;
      chi2UT += (du * du) * ut_hits.weight[hit_index];

      // count the number of processed htis
      total_num_hits++;
    }
  }

  chi2UT /= (total_num_hits - 1);

  // Save the best parameters if chi2 is good
  if (chi2UT < PrVeloUTConst::maxPseudoChi2) {
    // calculate q/p
    const float sinInX = xSlopeVeloFit * std::sqrt(1.0f + xSlopeVeloFit * xSlopeVeloFit);
    const float sinOutX = xSlopeUTFit * std::sqrt(1.0f + xSlopeUTFit * xSlopeUTFit);

    best_params.qp = sinInX - sinOutX;
    best_params.chi2UT = chi2UT;
    best_params.n_hits = total_num_hits;
  }

  return best_params;
}

__device__ __inline__ int set_index(
  const int i, 
  const LayerCandidates& layer_cand)
{
  int hit = 0;
  if (i < layer_cand.size0) {
    hit = layer_cand.from0 + i;
  } else if (i < layer_cand.size0 + layer_cand.size1) {
    hit = layer_cand.from1 + i - layer_cand.size0;
  } else {
    hit = layer_cand.from2 + i - layer_cand.size0- layer_cand.size1;
  }
  return hit;
}

//=========================================================================
// Check if hit is inside tolerance and refine by Y
//=========================================================================
__host__ __device__ __inline__ bool check_tol_refine(
  const int hit_index,
  const UTHits& ut_hits,
  const MiniState& velo_state,
  const float normFactNum,
  const float xTol,
  const float dxDy)
{
  const float xTolNormFact = xTol * (1.0f / normFactNum);

  const float zInit = ut_hits.zAtYEq0[hit_index];
  const float yApprox = velo_state.y + velo_state.ty * (zInit - velo_state.z);
  const float xOnTrackProto = velo_state.x + velo_state.tx * (zInit - velo_state.z);

  const float xx = ut_hits.xAt(hit_index, yApprox, dxDy);
  const float dx = xx - xOnTrackProto;

  if (dx < -xTolNormFact || dx > xTolNormFact) return false;

  // Now refine the tolerance in Y
  if (ut_hits.isNotYCompatible(
        hit_index, yApprox, PrVeloUTConst::yTol + PrVeloUTConst::yTolSlope * std::abs(dx * (1.0f / normFactNum))))
    return false;

  return true;
}