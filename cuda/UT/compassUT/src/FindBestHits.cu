#include "hip/hip_runtime.h"
#include "FindBestHits.cuh"

//=========================================================================
// Get the best 3 or 4 hits, 1 per layer, for a given VELO track
// When iterating over a panel, 3 windows are given, we set the index
// to be only in the windows
//=========================================================================
__device__ std::tuple<int,int,int,int,BestParams> find_best_hits(
  const short* win_size_shared,
  const uint number_of_tracks_event,
  const int i_track,
  const UTHits& ut_hits,
  const UTHitOffsets& ut_hit_count,
  const MiniState& velo_state,
  const float* ut_dxDy)
{
  const float yyProto = velo_state.y - velo_state.ty * velo_state.z;

  TrackCandidates ranges (win_size_shared, number_of_tracks_event, i_track);
  // WindowIndicator win_ranges(win_size_shared); 
  // const auto* ranges = win_ranges.get_track_candidates(threadIdx.x);

  int best_hits [4] = {-1, -1, -1, -1};

  bool found = false;
  bool forward = false;
  int considered = 0;

  int best_number_of_hits = 3;
  int best_fit = VeloUTConst::maxPseudoChi2;
  BestParams best_params;

  // Get total number of hits for forward + backward in first layer (0 for fwd, 3 for bwd)
  const int total_hits_2layers_0 = sum_layer_hits(ranges.layers[0], ranges.layers[3]);
  for (int i=0; (!found || considered < CompassUT::max_considered_before_found) && i<total_hits_2layers_0; ++i) {
    // const int i_hit0 = calc_index(i, ranges->layer[0], ranges->layer[3]);
    const int i_hit0 = calc_index(i, ranges.layers[0], ranges.layers[3]);

    // set range for next layer if forward or backward
    int layer_2;
    int dxdy_layer = -1;
    if (i < sum_layer_hits(ranges.layers[0])) {
      forward = true;
      layer_2 = 2;
      dxdy_layer = 0;
    } else {
      forward = false;
      layer_2 = 1;
      dxdy_layer = 3;
    }

    // Get info to calculate slope
    const float yy0 = yyProto + (velo_state.ty * ut_hits.zAtYEq0[i_hit0]);
    const auto xhitLayer0 = ut_hits.xAt(i_hit0, yy0, ut_dxDy[dxdy_layer]);
    const auto zhitLayer0 = ut_hits.zAtYEq0[i_hit0];

    // 2nd layer
    const int total_hits_2layers_2 = sum_layer_hits(ranges.layers[layer_2]);
    for (int j=0; (!found || considered < CompassUT::max_considered_before_found) && j<total_hits_2layers_2; ++j) {
      int i_hit2 = calc_index(j, ranges.layers[layer_2]);

      // Get info to calculate slope
      const int dxdy_layer_2 = forward ? 2 : 1;
      const float yy2 = yyProto + (velo_state.ty * ut_hits.zAtYEq0[i_hit2]);
      const auto xhitLayer2 = ut_hits.xAt(i_hit2, yy2, ut_dxDy[dxdy_layer_2]);
      const auto zhitLayer2 = ut_hits.zAtYEq0[i_hit2];

      // if slope is out of delta range, don't look for triplet/quadruplet
      const auto tx = (xhitLayer2 - xhitLayer0) / (zhitLayer2 - zhitLayer0);
      if (std::abs(tx - velo_state.tx) <= VeloUTConst::deltaTx2) {

        int temp_best_hits [4] = {i_hit0, -1, i_hit2, -1};

        const int layers [2] = {
          forward ? 1 : 2,
          forward ? 3 : 0
        };

        float hitTol = VeloUTConst::hitTol2;

        // search for a triplet in 3rd layer
        const int total_hits_2layers_1 = sum_layer_hits(ranges.layers[layers[0]]);
        for (int i1=0; i1<total_hits_2layers_1; ++i1) {

          int i_hit1 = calc_index(i1, ranges.layers[layers[0]]);

          // Get info to check tolerance
          const float yy1 = yyProto + (velo_state.ty * ut_hits.zAtYEq0[i_hit1]);
          const float xhitLayer1 = ut_hits.xAt(i_hit1, yy1, ut_dxDy[layers[0]]);
          const float zhitLayer1 = ut_hits.zAtYEq0[i_hit1];
          const float xextrapLayer1 = xhitLayer0 + tx * (zhitLayer1 - zhitLayer0);

          if (std::abs(xhitLayer1 - xextrapLayer1) < hitTol) {
            hitTol = std::abs(xhitLayer1 - xextrapLayer1);
            temp_best_hits[1] = i_hit1;
          }
        }

        // search for triplet/quadruplet in 4th layer
        hitTol = VeloUTConst::hitTol2;
        const int total_hits_2layers_3 = sum_layer_hits(ranges.layers[layers[1]]);
        for (int i3=0; i3<total_hits_2layers_3; ++i3) {

          int i_hit3 = calc_index(i3, ranges.layers[layers[1]]);

          // Get info to check tolerance
          const float yy3 = yyProto + (velo_state.ty * ut_hits.zAtYEq0[i_hit3]);
          const float xhitLayer3 = ut_hits.xAt(i_hit3, yy3, ut_dxDy[layers[1]]);
          const float zhitLayer3 = ut_hits.zAtYEq0[i_hit3];
          const float xextrapLayer3 = xhitLayer2 + tx * (zhitLayer3 - zhitLayer2);
          if (std::abs(xhitLayer3 - xextrapLayer3) < hitTol) {
            hitTol = std::abs(xhitLayer3 - xextrapLayer3);
            temp_best_hits[3] = i_hit3;
          }          
        }

        // Fit the hits to get q/p, chi2
        const auto temp_number_of_hits = 2 + (temp_best_hits[1] != -1) + (temp_best_hits[3] != -1);
        const auto params = pkick_fit(temp_best_hits, ut_hits, velo_state, ut_dxDy, yyProto, forward);
        ++considered;

        // Save the best chi2 and number of hits triplet/quadruplet
        if (params.chi2UT < best_fit && temp_number_of_hits >= best_number_of_hits) {
          if (forward) {
            best_hits[0] = temp_best_hits[0];
            best_hits[1] = temp_best_hits[1];
            best_hits[2] = temp_best_hits[2];
            best_hits[3] = temp_best_hits[3];
          } else {
            best_hits[0] = temp_best_hits[3];
            best_hits[1] = temp_best_hits[2];
            best_hits[2] = temp_best_hits[1];
            best_hits[3] = temp_best_hits[0];
          }
          best_number_of_hits = temp_number_of_hits;
          best_params = params;
          best_fit = params.chi2UT;

          found = true;
        }
      }
    }
  }

  return {best_hits[0], best_hits[1], best_hits[2], best_hits[3], best_params};
}

//=========================================================================
// Apply the p-kick method to the triplet/quadruplet
//=========================================================================
__device__ BestParams pkick_fit(
  const int best_hits[N_LAYERS],
  const UTHits& ut_hits,
  const MiniState& velo_state,
  const float* ut_dxDy,
  const float yyProto,
  const bool forward)
{
  BestParams best_params;

  // Helper stuff from velo state
  const float xMidField = velo_state.x + velo_state.tx * (VeloUTConst::zKink - velo_state.z);
  const float a = VeloUTConst::sigmaVeloSlope * (VeloUTConst::zKink - velo_state.z);
  const float wb = 1.0f / (a * a);

  float mat[3] = {wb, wb * VeloUTConst::zDiff, wb * VeloUTConst::zDiff * VeloUTConst::zDiff};
  float rhs[2] = {wb * xMidField, wb * xMidField * VeloUTConst::zDiff};

  // add hits
  #pragma unroll
  for (int i = 0; i < N_LAYERS; ++i) {
    int hit_index = best_hits[i];
    if (hit_index >= 0) {
      const float wi = ut_hits.weight[hit_index];
      const int plane_code = forward ? i : N_LAYERS - 1 - i;
      const float dxDy = ut_dxDy[plane_code];
      const float ci = ut_hits.cosT(hit_index, dxDy);
      const float dz = 0.001f * (ut_hits.zAtYEq0[hit_index] - VeloUTConst::zMidUT);
      // x_pos_layer
      const float yy = yyProto + (velo_state.ty * ut_hits.zAtYEq0[hit_index]);
      const float ui = ut_hits.xAt(hit_index, yy, dxDy);

      mat[0] += wi * ci;
      mat[1] += wi * ci * dz;
      mat[2] += wi * ci * dz * dz;
      rhs[0] += wi * ui;
      rhs[1] += wi * ui * dz;
    }
  }

  const float denom = 1.0f / (mat[0] * mat[2] - mat[1] * mat[1]);
  const float xSlopeUTFit = 0.001f * (mat[0] * rhs[1] - mat[1] * rhs[0]) * denom;
  const float xUTFit = (mat[2] * rhs[0] - mat[1] * rhs[1]) * denom;

  // new VELO slope x
  const float xb = xUTFit + xSlopeUTFit * (VeloUTConst::zKink - VeloUTConst::zMidUT);
  const float invKinkVeloDist = 1 / (VeloUTConst::zKink - velo_state.z);
  const float xSlopeVeloFit = (xb - velo_state.x) * invKinkVeloDist;
  const float chi2VeloSlope = (velo_state.tx - xSlopeVeloFit) * VeloUTConst::invSigmaVeloSlope;

  // chi2 takes chi2 from velo fit + chi2 from UT fit
  float chi2UT = chi2VeloSlope * chi2VeloSlope;
  // add chi2
  int total_num_hits = 0;
  #pragma unroll
  for (int i = 0; i < N_LAYERS; ++i) {
    int hit_index = best_hits[i];
    if (hit_index >= 0) {
      const float zd = ut_hits.zAtYEq0[hit_index];
      const float xd = xUTFit + xSlopeUTFit * (zd - VeloUTConst::zMidUT);
      // x_pos_layer
      const int plane_code = forward ? i : N_LAYERS - 1 - i;
      const float dxDy = ut_dxDy[plane_code];
      const float yy = yyProto + (velo_state.ty * ut_hits.zAtYEq0[hit_index]);
      const float x = ut_hits.xAt(hit_index, yy, dxDy);

      const float du = xd - x;
      chi2UT += (du * du) * ut_hits.weight[hit_index];

      // count the number of processed htis
      total_num_hits++;
    }
  }

  chi2UT /= (total_num_hits - 1);

  // Save the best parameters if chi2 is good
  if (chi2UT < VeloUTConst::maxPseudoChi2) {
    // calculate q/p
    const float sinInX = xSlopeVeloFit * std::sqrt(1.0f + xSlopeVeloFit * xSlopeVeloFit);
    const float sinOutX = xSlopeUTFit * std::sqrt(1.0f + xSlopeUTFit * xSlopeUTFit);

    best_params.qp = sinInX - sinOutX;
    best_params.chi2UT = chi2UT;
    best_params.n_hits = total_num_hits;
  }

  return best_params;
}

//=========================================================================
// Give total number of hits for N windows in 2 layers
//=========================================================================
__device__ __inline__ int sum_layer_hits(
  const LayerCandidates& first_candidate,
  const LayerCandidates& second_candidate)
{
  return  sum_layer_hits(first_candidate) +
          sum_layer_hits(second_candidate);
}

//=========================================================================
// Give total number of hits for N windows in a layer
//=========================================================================
__device__ __inline__ int sum_layer_hits(
  const LayerCandidates& layer_candidate)
{
  return  layer_candidate.size0 + 
          layer_candidate.size1 + 
          layer_candidate.size2 + 
          layer_candidate.size3 + 
          layer_candidate.size4; 
}

//=========================================================================
// Given a panel, 
// return the index in the correct place depending on the iteration.
// Put the index first in the central window, then left, then right
//=========================================================================
__device__ __inline__ int calc_index(
  const int i, 
  const LayerCandidates& layer_cand)
{
  int hit = -1;
  if (i < layer_cand.size0) {
    hit = layer_cand.from0 + i;
  } else if (i < layer_cand.size0 + layer_cand.size1) {
    hit = layer_cand.from1 + i - layer_cand.size0;
  } else if (i < layer_cand.size0 + layer_cand.size1 + layer_cand.size2) {
    hit = layer_cand.from2 + i - (layer_cand.size0 + layer_cand.size1);
  } else if (i < layer_cand.size0 + layer_cand.size1 + layer_cand.size2 + layer_cand.size3) {
    hit = layer_cand.from3 + i - (layer_cand.size0 + layer_cand.size1 + layer_cand.size2);
  } else if (i < layer_cand.size0 + layer_cand.size1 + layer_cand.size2 + layer_cand.size3 + layer_cand.size4) {
    hit = layer_cand.from4 + i - (layer_cand.size0 + layer_cand.size1 + layer_cand.size2 + layer_cand.size3);
  }

  return hit;
}

//=========================================================================
// Given 2 panels (forward backward case),
// return the index in the correct place depending on the iteration.
// Put the index first in the central window, then left, then right
//=========================================================================
__device__ __inline__ int calc_index(
  const int i, 
  const LayerCandidates& layer_cand0,
  const LayerCandidates& layer_cand2)
{
  int hit = -1;
  int cand0size = layer_cand0.size0 + layer_cand0.size1 + layer_cand0.size2 + layer_cand0.size3 + layer_cand0.size4;
  if (i < layer_cand0.size0) {
    hit = layer_cand0.from0 + i;
  } else if (i < layer_cand0.size0 + layer_cand0.size1) {
    hit = layer_cand0.from1 + i - layer_cand0.size0;
  } else if (i < layer_cand0.size0 + layer_cand0.size1 + layer_cand0.size2) {
    hit = layer_cand0.from2 + i - (layer_cand0.size0 + layer_cand0.size1);
  } else if (i < layer_cand0.size0 + layer_cand0.size1 + layer_cand0.size2 + layer_cand0.size3) {
    hit = layer_cand0.from2 + i - (layer_cand0.size0 + layer_cand0.size1 + layer_cand0.size2);
  } else if (i < layer_cand0.size0 + layer_cand0.size1 + layer_cand0.size2 + layer_cand0.size3 + layer_cand0.size4) {
    hit = layer_cand0.from2 + i - (layer_cand0.size0 + layer_cand0.size1 + layer_cand0.size2 + layer_cand0.size3);
  }
  // layer_cand2
  else if (i < cand0size + layer_cand2.size0) {
    hit = layer_cand2.from0 + i - cand0size ;
  } else if (i < cand0size + layer_cand2.size0 + layer_cand2.size1) {
    hit = layer_cand2.from1 + i - layer_cand2.size0 - (cand0size);
  } else if (i < cand0size + layer_cand2.size0 + layer_cand2.size1 + layer_cand2.size2) {
    hit = layer_cand2.from2 + i - (layer_cand2.size0 + layer_cand2.size1) - (cand0size);
  } else if (i < cand0size + layer_cand2.size0 + layer_cand2.size1 + layer_cand2.size2 + layer_cand2.size3) {
    hit = layer_cand2.from3 + i - (layer_cand2.size0 + layer_cand2.size1 + layer_cand2.size2) - (cand0size);
  } else if (i < cand0size + layer_cand2.size0 + layer_cand2.size1 + layer_cand2.size2 + layer_cand2.size3 + layer_cand2.size4) {
    hit = layer_cand2.from3 + i - (layer_cand2.size0 + layer_cand2.size1 + layer_cand2.size2 + layer_cand2.size3) - (cand0size);
  }

  return hit;
}

//=========================================================================
// Check if hit is inside tolerance and refine by Y
//=========================================================================
__device__ __inline__ bool check_tol_refine(
  const int hit_index,
  const UTHits& ut_hits,
  const MiniState& velo_state,
  const float normFactNum,
  const float xTol,
  const float dxDy)
{
  const float xTolNormFact = xTol * (1.0f / normFactNum);

  const float zInit = ut_hits.zAtYEq0[hit_index];
  const float yApprox = velo_state.y + velo_state.ty * (zInit - velo_state.z);
  const float xOnTrackProto = velo_state.x + velo_state.tx * (zInit - velo_state.z);

  const float xx = ut_hits.xAt(hit_index, yApprox, dxDy);
  const float dx = xx - xOnTrackProto;

  if (dx < -xTolNormFact || dx > xTolNormFact) return false;

  // Now refine the tolerance in Y
  if (ut_hits.isNotYCompatible(
        hit_index, yApprox, VeloUTConst::yTol + VeloUTConst::yTolSlope * std::abs(dx * (1.0f / normFactNum))))
    return false;

  return true;
}