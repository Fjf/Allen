#include "hip/hip_runtime.h"
#include "compassUT.cuh"

#include <float.h>

__global__ void compassUT(
  uint* dev_ut_hits, // actual hit content
  const uint* dev_ut_hit_offsets,
  int* dev_atomics_storage, // semi_prefixsum, offset to tracks
  uint* dev_velo_track_hit_number,
  uint* dev_velo_track_hits,
  uint* dev_velo_states,
  PrUTMagnetTool* dev_ut_magnet_tool,
  const float* dev_ut_dxDy,
  int* dev_active_tracks,
  const uint* dev_unique_x_sector_layer_offsets, // prefixsum to point to the x hit of the sector, per layer
  const uint* dev_unique_x_sector_offsets, // TODO remove this, only needed for decoding
  const float* dev_unique_sector_xs, // list of xs that define the groups
  VeloUTTracking::TrackUT* dev_compassUT_tracks,
  int* dev_atomics_compassUT, // size of number of events
  int* dev_windows_layers)
{
  const uint number_of_events = gridDim.x;
  const uint event_number = blockIdx.x;

  const uint number_of_unique_x_sectors = dev_unique_x_sector_layer_offsets[4];
  const uint total_number_of_hits = dev_ut_hit_offsets[number_of_events * number_of_unique_x_sectors];

  // Velo consolidated types
  const Velo::Consolidated::Tracks velo_tracks{
    (uint*) dev_atomics_storage, dev_velo_track_hit_number, event_number, number_of_events};
  const Velo::Consolidated::States velo_states{dev_velo_states, velo_tracks.total_number_of_tracks};
  const uint number_of_tracks_event = velo_tracks.number_of_tracks(event_number);
  const uint event_tracks_offset = velo_tracks.tracks_offset(event_number);

  UTHitOffsets ut_hit_offsets{
    dev_ut_hit_offsets, event_number, number_of_unique_x_sectors, dev_unique_x_sector_layer_offsets};

  UTHits ut_hits;
  ut_hits.typecast_sorted(dev_ut_hits, total_number_of_hits);

  // active track pointer
  // int* active_tracks = dev_active_tracks + event_number;

  // dev_atomics_compassUT contains in an SoA:
  //   1. # of veloUT tracks
  //   2. # velo tracks in UT acceptance
  // This is to write the final track
  int* n_veloUT_tracks_event = dev_atomics_compassUT + event_number;
  VeloUTTracking::TrackUT* veloUT_tracks_event = dev_compassUT_tracks + event_number * VeloUTTracking::max_num_tracks;

  // initialize atomic veloUT tracks counter && active track
  if (threadIdx.x == 0) {
    *n_veloUT_tracks_event = 0;
    // *active_tracks         = 0;
  }

  // int shared_active_tracks[2 * VeloUTTracking::num_threads - 1];

  // __syncthreads();

  // const float* fudgeFactors = &(dev_ut_magnet_tool->dxLayTable[0]);
  // const float* bdlTable     = &(dev_ut_magnet_tool->bdlTable[0]);

  // array to store indices of selected hits in layers
  // -> can then access the hit information in the HitsSoA
  // int hitCandidatesInLayers[VeloUTTracking::n_layers][VeloUTTracking::max_hit_candidates_per_layer];
  // int n_hitCandidatesInLayers[VeloUTTracking::n_layers];

  for (int i_track = threadIdx.x; i_track < number_of_tracks_event; i_track += blockDim.x) {

    // __syncthreads();

    // TODO the non active tracks should be -1
    // const int i_track = shared_active_tracks[threadIdx.x];

    const uint velo_states_index = event_tracks_offset + i_track;
    const MiniState velo_state{velo_states, velo_states_index};

    //   __syncthreads();

    // TrackHelper helper{velo_state};

    // indices within hitCandidatesInLayers for selected hits belonging to best track
    // int hitCandidateIndices[N_LAYERS];

    // TODO remove the x_hit_layer (not needed)
    float x_hit_layer[N_LAYERS];
    int best_hits[N_LAYERS] = {-1, -1, -1, -1};
    BestParams best_params;

    // TODO search backwards if we didn't found 4 hits
    find_best_hits(
      i_track,
      dev_windows_layers,
      ut_hits,
      ut_hit_offsets,
      velo_state,
      dev_ut_dxDy,
      true,
      x_hit_layer,
      best_hits,
      best_params);

    // Count best hits
    int total_num_hits = 0;
    for (int i = 0; i < N_LAYERS; ++i) {
      if (best_hits[i] >= 0) total_num_hits++;
    }

    const float* bdl_table = &(dev_ut_magnet_tool->bdlTable[0]);

    // write the final track
    if (total_num_hits > 0) {
      save_track(
        i_track,
        bdl_table,
        velo_state,
        best_params,
        dev_velo_track_hits,
        velo_tracks,
        total_num_hits,
        best_hits,
        ut_hits,
        dev_ut_dxDy,
        n_veloUT_tracks_event,
        veloUT_tracks_event);
    }

    //     if ( helper.n_hits > 0 ) {
    //       const uint velo_track_hit_number = velo_tracks.number_of_hits(i_track);
    //       const Velo::Consolidated::Hits velo_track_hits = velo_tracks.get_hits(dev_velo_track_hits, i_track);

    //       prepareOutputTrack(
    //         i_track,
    //         velo_track_hits,
    //         velo_track_hit_number,
    //         helper,
    //         velo_state,
    //         (int*) &windows_layers[0],
    //         ut_hits,
    //         ut_hit_count,
    //         (float*) &x_hit_layer[0],
    //         (int*) &hitCandidateIndices[0],
    //         bdlTable,
    //         veloUT_tracks_event,
    //         n_veloUT_tracks_event);
    //     }

    //     const int j = blockDim.x + threadIdx.x;
    //     if (j < *active_tracks) {
    //       shared_active_tracks[threadIdx.x] = shared_active_tracks[j];
    //     }

    //     __syncthreads();

    //     if (threadIdx.x == 0) {
    //       *active_tracks -= blockDim.x;
    //     }
  }

  // // remaining tracks
  // if (threadIdx.x < *active_tracks) {

  //   // store a window(2 positions) for each layer, for each thrack
  //   __shared__ int windows_layers[VeloUTTracking::num_threads * VeloUTTracking::n_layers * 2];

  //   const int i_track = shared_active_tracks[threadIdx.x];

  //   // MiniState aux_velo_state {velo_states, velo_states_index};
  //   const uint velo_states_index = event_tracks_offset + i_track;
  //   const MiniState velo_state {velo_states, velo_states_index};

  //   get_windows(
  //     i_track,
  //     velo_state,
  //     fudgeFactors,
  //     ut_hits,
  //     ut_hit_count,
  //     dev_ut_dxDy,
  //     (int*) &windows_layers[0]);

  //   __syncthreads();

  //   // for storing calculated x position of hits for this track
  //   // float x_pos_layers[VeloUTTracking::n_layers][VeloUTTracking::max_hit_candidates_per_layer];

  // }
}

//=========================================================================
// hits_to_track
//=========================================================================
__host__ __device__ void find_best_hits(
  const int i_track,
  const int* dev_windows_layers,
  const UTHits& ut_hits,
  const UTHitOffsets& ut_hit_count,
  const MiniState& velo_state,
  const float* ut_dxDy,
  const bool forward,
  float* x_hit_layer,
  int* best_hits,
  BestParams& best_params)
{
  // handle forward / backward cluster search
  int layers[N_LAYERS];
  for (int i_layer = 0; i_layer < N_LAYERS; ++i_layer) {
    if (forward)
      layers[i_layer] = i_layer;
    else
      layers[i_layer] = N_LAYERS - 1 - i_layer;
  }

  // Get windows of all layers
  WindowIndicator win_ranges(dev_windows_layers); 
  const auto* ranges = win_ranges.get_track_candidates(i_track);
  const int from0 = ranges->layer[0].first;
  const int to0 = ranges->layer[0].last;
  const int from2 = ranges->layer[2].first;
  const int to2 = ranges->layer[2].last;
  const int from1 = ranges->layer[1].first;
  const int to1 = ranges->layer[1].last;
  const int from3 = ranges->layer[3].first;
  const int to3 = ranges->layer[3].last;
  
  // printf("from0: %i, to0: %i, from1: %i, to1: %i, from2: %i, to2: %i, from3: %i, to3: %i\n", from0, to0, from1, to1, from2, to2, from3, to3);

  const float yyProto = velo_state.y - velo_state.ty * velo_state.z;

  // auto is_valid = [](float dx, int layer, float y){
  //   if (dx < )
  // }

  // const float normFactNum = normFact[layer];
  // const float invNormFact = 1.0/normFactNum;
  // xTol*invNormFact
  // const auto zInit = ut_hits.zAtYEq0[layer_offset + posBeg];
  // const auto xOnTrackProto = myState.x + myState.tx*(zInit - myState.z);
  // const auto yApprox = myState.y + myState.ty * (zInit - myState.z);
  // const auto xx = ut_hits.xAt(layer_offset + i, yApprox, dxDy); 
  // const auto dx = xx - xOnTrackProto;
  
  // if( dx < -xTolNormFact ) continue;
  // if( dx >  xTolNormFact ) continue; 
  
  // // -- Now refine the tolerance in Y
  // if ( ut_hits.isNotYCompatible( layer_offset + i, yApprox, PrVeloUTConst::yTol + PrVeloUTConst::yTolSlope * std::abs(dx*invNormFact)) ) continue;

  // BestParams best_params;

  for (int i_hit0 = from0; i_hit0 < to0; ++i_hit0) {

    const float yy0 = yyProto + (velo_state.ty * ut_hits.zAtYEq0[i_hit0]);
    x_hit_layer[0] = ut_hits.xAt(i_hit0, yy0, ut_dxDy[layers[0]]);
    const float zhitLayer0 = ut_hits.zAtYEq0[i_hit0];
    best_hits[0] = i_hit0;

    for (int i_hit2 = from2; i_hit2 < to2; ++i_hit2) {

      const float yy2 = yyProto + (velo_state.ty * ut_hits.zAtYEq0[i_hit2]);
      x_hit_layer[2] = ut_hits.xAt(i_hit2, yy2, ut_dxDy[layers[2]]);
      const float zhitLayer2 = ut_hits.zAtYEq0[i_hit2];
      best_hits[2] = i_hit2;

      // same bool check for the hit
      const float tx = (x_hit_layer[2] - x_hit_layer[0]) / (zhitLayer2 - zhitLayer0);
      if (std::abs(tx - velo_state.tx) <= PrVeloUTConst::deltaTx2) {
        float hitTol = PrVeloUTConst::hitTol2;

        // Search for triplet
        for (int i_hit1 = from1; i_hit1 < to1; ++i_hit1) {
          const float yy1 = yyProto + (velo_state.ty * ut_hits.zAtYEq0[i_hit1]);
          x_hit_layer[1] = ut_hits.xAt(i_hit1, yy1, ut_dxDy[layers[1]]);
          const float zhitLayer1 = ut_hits.zAtYEq0[i_hit1];
          const float xextrapLayer1 = x_hit_layer[0] + tx * (zhitLayer1 - zhitLayer0);
          if (std::abs(x_hit_layer[1] - xextrapLayer1) < hitTol) {
            hitTol = std::abs(x_hit_layer[1] - xextrapLayer1);
            // index_best_hit_1 = i_hit1;
            best_hits[1] = i_hit1;
          }
        }

        // Search for cuadruplet
        hitTol = PrVeloUTConst::hitTol2;
        for (int i_hit3 = from3; i_hit3 < to3; ++i_hit3) {
          const float yy3 = yyProto + (velo_state.ty * ut_hits.zAtYEq0[i_hit3]);
          x_hit_layer[3] = ut_hits.xAt(i_hit3, yy3, ut_dxDy[layers[3]]);
          const float zhitLayer3 = ut_hits.zAtYEq0[i_hit3];
          const float xextrapLayer3 = x_hit_layer[2] + tx * (zhitLayer3 - zhitLayer2);
          if (std::abs(x_hit_layer[3] - xextrapLayer3) < hitTol) {
            hitTol = std::abs(x_hit_layer[3] - xextrapLayer3);
            // index_best_hit_3 = i_hit3;
            best_hits[3] = i_hit3;
          }
        }

        // Fit the hits to get q/p, chi2
        best_params = pkick_fit(best_hits, ut_hits, velo_state, ut_dxDy, yyProto);
      }
    }
  }

  if (best_hits[0] != -1 && best_hits[1] != -1 && best_hits[2] != -1 && best_hits[3] != -1) {
    printf("hit0: %i, hit1: %i, hit2: %i, hit3: %i, q/p: %f, chi2: %f\n", best_hits[0], best_hits[1], best_hits[2], best_hits[3], best_params.qp, best_params.chi2UT);
  }
}

//=========================================================================
// apply the p-kick method to the triplet/quadruplet
// TODO return the chi2?
// TODO precalculate zDiff (its always the same)
//=========================================================================
__host__ __device__ BestParams pkick_fit(
  const int best_hits[N_LAYERS],
  const UTHits& ut_hits,
  const MiniState& velo_state,
  const float* ut_dxDy,
  const float yyProto)
{
  BestParams best_params;

  // accumulate the high threshold
  int n_high_thres = 0;
  for (int i = 0; i < N_LAYERS; ++i) {
    if (best_hits[i] >= 0) { n_high_thres += ut_hits.highThreshold[best_hits[i]]; }
  }

  // Veto hit combinations with no high threshold hit
  // = likely spillover
  if (n_high_thres < PrVeloUTConst::minHighThres) return best_params;

  // Scale the z-component, to not run into numerical problems with floats
  // first add to sum values from hit at xMidField, zMidField hit
  const float zDiff = 0.001 * (PrVeloUTConst::zKink - PrVeloUTConst::zMidUT);

  // Helper stuff from velo state
  const float xMidField = velo_state.x + velo_state.tx * (PrVeloUTConst::zKink - velo_state.z);
  const float a = PrVeloUTConst::sigmaVeloSlope * (PrVeloUTConst::zKink - velo_state.z);
  const float wb = 1. / (a * a);

  float mat[3] = {wb, wb * zDiff, wb * zDiff * zDiff};
  float rhs[2] = {wb * xMidField, wb * xMidField * zDiff};

  // add hits
  for (int i = 0; i < N_LAYERS; ++i) {
    int hit_index = best_hits[i];
    if (hit_index >= 0) {

      const float wi = ut_hits.weight[hit_index];
      const float dxDy = ut_dxDy[ut_hits.planeCode[hit_index]];
      const float ci = ut_hits.cosT(hit_index, dxDy);
      const float dz = 0.001 * (ut_hits.zAtYEq0[hit_index] - PrVeloUTConst::zMidUT);
      // x_pos_layer
      const float yy = yyProto + (velo_state.ty * ut_hits.zAtYEq0[hit_index]);
      const float ui = ut_hits.xAt(hit_index, yy, dxDy);

      mat[0] += wi * ci;
      mat[1] += wi * ci * dz;
      mat[2] += wi * ci * dz * dz;
      rhs[0] += wi * ui;
      rhs[1] += wi * ui * dz;
    }
  }

  const float denom = 1. / (mat[0] * mat[2] - mat[1] * mat[1]);
  const float xSlopeUTFit = 0.001 * (mat[0] * rhs[1] - mat[1] * rhs[0]) * denom;
  const float xUTFit = (mat[2] * rhs[0] - mat[1] * rhs[1]) * denom;

  // new VELO slope x
  const float xb = xUTFit + xSlopeUTFit * (PrVeloUTConst::zKink - PrVeloUTConst::zMidUT);
  const float invKinkVeloDist = 1 / (PrVeloUTConst::zKink - velo_state.z);
  const float xSlopeVeloFit = (xb - velo_state.x) * invKinkVeloDist;
  const float chi2VeloSlope = (velo_state.tx - xSlopeVeloFit) * PrVeloUTConst::invSigmaVeloSlope;

  // chi2 takes chi2 from velo fit + chi2 from UT fit
  float chi2UT = chi2VeloSlope * chi2VeloSlope;
  // add chi2
  int total_num_hits = 0;
  for (int i = 0; i < N_LAYERS; ++i) {
    int hit_index = best_hits[i];
    if (hit_index >= 0) {
      const float zd = ut_hits.zAtYEq0[hit_index];
      const float xd = xUTFit + xSlopeUTFit * (zd - PrVeloUTConst::zMidUT);
      // x_pos_layer
      const float dxDy = ut_dxDy[ut_hits.planeCode[hit_index]];
      const float yy = yyProto + (velo_state.ty * ut_hits.zAtYEq0[hit_index]);
      const float x = ut_hits.xAt(hit_index, yy, dxDy);

      const float du = xd - x;
      chi2UT += (du * du) * ut_hits.weight[hit_index];

      // count the number of processed htis
      total_num_hits++;
    }
  }

  chi2UT /= (total_num_hits - 1);

  // Save the best parameters if chi2 is good
  if (chi2UT < PrVeloUTConst::maxPseudoChi2) {
    // calculate q/p
    const float sinInX = xSlopeVeloFit * std::sqrt(1. + xSlopeVeloFit * xSlopeVeloFit);
    const float sinOutX = xSlopeUTFit * std::sqrt(1. + xSlopeUTFit * xSlopeUTFit);

    best_params.qp = sinInX - sinOutX;
    best_params.chi2UT = chi2UT;
    best_params.xUTFit = xUTFit;
    best_params.xSlopeUTFit = xSlopeUTFit;
  }

  return best_params;
}

// These things are all hardcopied from the PrTableForFunction and PrUTMagnetTool
// If the granularity or whatever changes, this will give wrong results
__host__ __device__ int master_index(const int index1, const int index2, const int index3){
  return (index3*11 + index2)*31 + index1;
}

//=========================================================================
// prepare the final track
//=========================================================================
__device__ void save_track(
  const int i_track,
  const float* bdl_table,
  const MiniState& velo_state,
  const BestParams& best_params,
  uint* dev_velo_track_hits,
  const Velo::Consolidated::Tracks& velo_tracks,
  const int num_best_hits,
  const int* best_hits,
  const UTHits& ut_hits,
  const float* ut_dxDy,
  int* n_veloUT_tracks, // increment number of tracks
  VeloUTTracking::TrackUT VeloUT_tracks[VeloUTTracking::max_num_tracks]) // write the track
{
  //== Handle states. copy Velo one, add UT.
  const float zOrigin = (std::fabs(velo_state.ty) > 0.001) ? velo_state.z - velo_state.y / velo_state.ty
                                                           : velo_state.z - velo_state.x / velo_state.tx;

  // -- These are calculations, copied and simplified from PrTableForFunction
  const float var[3] = {velo_state.ty, zOrigin, velo_state.z};

  const int index1 = std::max(0, std::min(30, int((var[0] + 0.3) / 0.6 * 30)));
  const int index2 = std::max(0, std::min(10, int((var[1] + 250) / 500 * 10)));
  const int index3 = std::max(0, std::min(10, int(var[2] / 800 * 10)));

  assert(master_index(index1, index2, index3) < PrUTMagnetTool::N_bdl_vals);
  float bdl = bdl_table[master_index(index1, index2, index3)];

  const int num_idx = 3;
  const float bdls[num_idx] = {bdl_table[master_index(index1 + 1, index2, index3)],
                               bdl_table[master_index(index1, index2 + 1, index3)],
                               bdl_table[master_index(index1, index2, index3 + 1)]};
  const float deltaBdl[num_idx] = {0.02, 50.0, 80.0};
  const float boundaries[num_idx] = {
    -0.3f + float(index1) * deltaBdl[0], -250.0f + float(index2) * deltaBdl[1], 0.0f + float(index3) * deltaBdl[2]};

  // This is an interpolation, to get a bit more precision
  float addBdlVal = 0.0;
  const float minValsBdl[num_idx] = {-0.3, -250.0, 0.0};
  const float maxValsBdl[num_idx] = {0.3, 250.0, 800.0};
  for (int i = 0; i < num_idx; ++i) {
    if (var[i] < minValsBdl[i] || var[i] > maxValsBdl[i]) continue;
    const float dTab_dVar = (bdls[i] - bdl) / deltaBdl[i];
    const float dVar = (var[i] - boundaries[i]);
    addBdlVal += dTab_dVar * dVar;
  }
  bdl += addBdlVal;

  const float qpxz2p = -1 * std::sqrt(1. + velo_state.ty * velo_state.ty) / bdl * 3.3356 / Gaudi::Units::GeV;
  const float qop = (std::abs(bdl) < 1.e-8) ? 0.0 : best_params.qp * qpxz2p;

  // -- Don't make tracks that have grossly too low momentum
  // -- Beware of the momentum resolution!
  const float p = 1.3 * std::abs(1 / qop);
  const float pt = p * std::sqrt(velo_state.tx * velo_state.tx + velo_state.ty * velo_state.ty);

  if (p < PrVeloUTConst::minMomentum || pt < PrVeloUTConst::minPT) return;

  // the track will be added
  uint n_tracks = atomicAdd(n_veloUT_tracks, 1);

  // const float txUT = best_params.xSlopeUTFit;

  // TODO change this to use the pointer to the hits
  // TODO dev_velo_tracks_hits should be const
  const uint velo_track_hit_number = velo_tracks.number_of_hits(i_track);
  const Velo::Consolidated::Hits velo_track_hits = velo_tracks.get_hits(dev_velo_track_hits, i_track);

  // TODO Maybe have a look and optimize this if possible
  // add VELO hits to VeloUT track
  VeloUTTracking::TrackUT track;
  track.hitsNum = 0;
  for (int i=0; i<velo_track_hit_number; ++i) {
    track.addLHCbID(velo_track_hits.LHCbID[i]);
    assert( track.hitsNum < VeloUTTracking::max_track_size);
  }
  track.set_qop( qop );

  // const float yyProto = velo_state.y - velo_state.ty * velo_state.z;

  // Adding overlap hits
  for ( int i = 0; i < num_best_hits; ++i ) {
    int hit_index = best_hits[i];
    if (hit_index >= 0) {
      track.addLHCbID( ut_hits.LHCbID[hit_index] );
      assert( track.hitsNum < VeloUTTracking::max_track_size);

      // TODO add one overlap hit?
    }
  }
  assert( n_tracks < VeloUTTracking::max_num_tracks );
  VeloUT_tracks[n_tracks] = track;  
}