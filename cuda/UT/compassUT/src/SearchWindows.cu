#include "hip/hip_runtime.h"
#include "CalculateWindows.cuh"
#include "SearchWindows.cuh"
#include "Handler.cuh"
#include <tuple>
#include "Invoke.cuh"

void ut_search_windows_t::invoke() {
  invoke_helper(handler);
}

__constant__ float Configuration::ut_search_windows_t::min_momentum;
__constant__ float Configuration::ut_search_windows_t::min_pt;
__constant__ float Configuration::ut_search_windows_t::y_tol;
__constant__ float Configuration::ut_search_windows_t::y_tol_slope;

__global__ void ut_search_windows(
  uint* dev_ut_hits, // actual hit content
  const uint* dev_ut_hit_offsets,
  uint* dev_atomics_storage, // semi_prefixsum, offset to tracks
  uint* dev_velo_track_hit_number,
  char* dev_velo_states,
  UTMagnetTool* dev_ut_magnet_tool,
  const float* dev_ut_dxDy,
  const uint* dev_unique_x_sector_layer_offsets, // prefixsum to point to the x hit of the sector, per layer
  const float* dev_unique_sector_xs,             // list of xs that define the groups
  short* dev_windows_layers,
  uint* dev_active_tracks,
  bool* dev_accepted_velo_tracks)
{
  const uint number_of_events = gridDim.x;
  const uint event_number = blockIdx.x;
  const uint number_of_unique_x_sectors = dev_unique_x_sector_layer_offsets[UT::Constants::n_layers];
  const uint total_number_of_hits = dev_ut_hit_offsets[number_of_events * number_of_unique_x_sectors];

  // Velo consolidated types
  const Velo::Consolidated::Tracks velo_tracks {
    (uint*) dev_atomics_storage, dev_velo_track_hit_number, event_number, number_of_events};
  const Velo::Consolidated::States velo_states {dev_velo_states, velo_tracks.total_number_of_tracks};
  const uint number_of_tracks_event = velo_tracks.number_of_tracks(event_number);
  const uint event_tracks_offset = velo_tracks.tracks_offset(event_number);

  UT::HitOffsets ut_hit_offsets {
    dev_ut_hit_offsets, event_number, number_of_unique_x_sectors, dev_unique_x_sector_layer_offsets};
  UT::Hits ut_hits {dev_ut_hits, total_number_of_hits};

  const float* fudge_factors = &(dev_ut_magnet_tool->dxLayTable[0]);
  uint* active_tracks = dev_active_tracks + event_number;

  // Store only the valid tracks into shared memory.
  // Fill the array until with find enough valid tracks = block size
  __shared__ int shared_active_tracks[2 * UT::Constants::num_thr_searchwin - 1];

  for (uint layer = threadIdx.x; layer < UT::Constants::n_layers; layer += blockDim.x) {
    const uint layer_offset = ut_hit_offsets.layer_offset(layer);

    for (uint i = 0; i < ((number_of_tracks_event + blockDim.y - 1) / blockDim.y) + 1; i += 1) {
      const auto i_track = i * blockDim.y + threadIdx.y;

      __syncthreads();

      // filter the tracks that won't be valid
      if (threadIdx.x == 0) {
        const uint current_track_offset = event_tracks_offset + i_track;
        const MiniState velo_state = velo_states.get(current_track_offset);
        if (i_track < number_of_tracks_event) {
          if (
            !velo_states.backward[current_track_offset] && dev_accepted_velo_tracks[current_track_offset] &&
            velo_track_in_UTA_acceptance(velo_state)) {
            int current_track = atomicAdd(active_tracks, 1);
            shared_active_tracks[current_track] = i_track;
          }
        }
      }

      __syncthreads();

      // process only the active tracks
      if (*active_tracks >= blockDim.y) {

        const uint current_track_offset = event_tracks_offset + shared_active_tracks[threadIdx.y];
        const MiniState velo_state = velo_states.getMiniState(current_track_offset);

        const auto candidates = calculate_windows(
          layer,
          velo_state,
          fudge_factors,
          ut_hits,
          ut_hit_offsets,
          dev_ut_dxDy,
          dev_unique_sector_xs,
          dev_unique_x_sector_layer_offsets);

        // Write the windows in SoA style
        short* windows_layers =
          dev_windows_layers + event_tracks_offset * CompassUT::num_elems * UT::Constants::n_layers;

        const int track_pos = UT::Constants::n_layers * number_of_tracks_event;
        const int layer_pos = layer * number_of_tracks_event + shared_active_tracks[threadIdx.y];

        windows_layers[0 * track_pos + layer_pos] = std::get<0>(candidates) - layer_offset; // first_candidate
        windows_layers[1 * track_pos + layer_pos] = std::get<2>(candidates) - layer_offset; // left_group_first
        windows_layers[2 * track_pos + layer_pos] = std::get<4>(candidates) - layer_offset; // right_group_first
        windows_layers[3 * track_pos + layer_pos] = std::get<6>(candidates) - layer_offset; // left2_group_first
        windows_layers[4 * track_pos + layer_pos] = std::get<8>(candidates) - layer_offset; // right2_group_first
        windows_layers[5 * track_pos + layer_pos] = std::get<1>(candidates) - std::get<0>(candidates); // last_size
        windows_layers[6 * track_pos + layer_pos] = std::get<3>(candidates) - std::get<2>(candidates); // left_size_last
        windows_layers[7 * track_pos + layer_pos] =
          std::get<5>(candidates) - std::get<4>(candidates); // right_size_first
        windows_layers[8 * track_pos + layer_pos] =
          std::get<7>(candidates) - std::get<6>(candidates); // left2_size_last
        windows_layers[9 * track_pos + layer_pos] =
          std::get<9>(candidates) - std::get<8>(candidates); // right2_size_first

        __syncthreads();

        if (threadIdx.x == 0) {
          const int j = blockDim.y + threadIdx.y;
          if (j < *active_tracks) {
            shared_active_tracks[threadIdx.y] = shared_active_tracks[j];
          }
        }

        __syncthreads();

        if (threadIdx.x == 0 && threadIdx.y == 0) {
          *active_tracks -= blockDim.y;
        }
      }
    }

    __syncthreads();

    // remaining tracks
    if (threadIdx.y < *active_tracks) {

      const int i_track = shared_active_tracks[threadIdx.y];
      const uint current_track_offset = event_tracks_offset + i_track;

      const auto velo_state = velo_states.get(current_track_offset);

      const auto candidates = calculate_windows(
        layer,
        velo_state,
        fudge_factors,
        ut_hits,
        ut_hit_offsets,
        dev_ut_dxDy,
        dev_unique_sector_xs,
        dev_unique_x_sector_layer_offsets);

      // Write the windows in SoA style
      short* windows_layers = dev_windows_layers + event_tracks_offset * CompassUT::num_elems * UT::Constants::n_layers;

      const int track_pos = UT::Constants::n_layers * number_of_tracks_event;
      const int layer_pos = layer * number_of_tracks_event + shared_active_tracks[threadIdx.y];
      windows_layers[0 * track_pos + layer_pos] = std::get<0>(candidates) - layer_offset; // first_candidate
      windows_layers[1 * track_pos + layer_pos] = std::get<2>(candidates) - layer_offset; // left_group_first
      windows_layers[2 * track_pos + layer_pos] = std::get<4>(candidates) - layer_offset; // right_group_first
      windows_layers[3 * track_pos + layer_pos] = std::get<6>(candidates) - layer_offset; // left2_group_first
      windows_layers[4 * track_pos + layer_pos] = std::get<8>(candidates) - layer_offset; // right2_group_first
      windows_layers[5 * track_pos + layer_pos] = std::get<1>(candidates) - std::get<0>(candidates); // last_size
      windows_layers[6 * track_pos + layer_pos] = std::get<3>(candidates) - std::get<2>(candidates); // left_size_last
      windows_layers[7 * track_pos + layer_pos] = std::get<5>(candidates) - std::get<4>(candidates); // right_size_first
      windows_layers[8 * track_pos + layer_pos] = std::get<7>(candidates) - std::get<6>(candidates); // left2_size_last
      windows_layers[9 * track_pos + layer_pos] =
        std::get<9>(candidates) - std::get<8>(candidates); // right2_size_first
    }
  }
}
