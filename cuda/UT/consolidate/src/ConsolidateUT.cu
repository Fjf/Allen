#include "hip/hip_runtime.h"
#include "ConsolidateUT.cuh"

template<typename F>
__device__ void populate(const UT::TrackHits& track, const F& assign)
{
  int hit_number = 0;
  for (uint i = 0; i < UT::Constants::n_layers; ++i) {
    const auto hit_index = track.hits[i];
    if (hit_index != -1) {
      assign(hit_number++, hit_index);
    }
  }
}

template<typename F>
__device__ void populate_plane_code(const UT::TrackHits& track, const F& assign)
{
  int hit_number = 0;
  for (uint i = 0; i < UT::Constants::n_layers; ++i) {
    const auto hit_index = track.hits[i];
    if (hit_index != -1) {
      assign(hit_number++, i);
    }
  }
}

__global__ void ut_consolidate_tracks::ut_consolidate_tracks(
  ut_consolidate_tracks::Parameters parameters,
  const uint* dev_unique_x_sector_layer_offsets)
{
  const uint number_of_events = gridDim.x;
  const uint event_number = blockIdx.x;
  const uint number_of_unique_x_sectors = dev_unique_x_sector_layer_offsets[4];
  const uint total_number_of_hits = parameters.dev_ut_hit_offsets[number_of_events * number_of_unique_x_sectors];
  const UT::TrackHits* event_veloUT_tracks = parameters.dev_ut_tracks + event_number * UT::Constants::max_num_tracks;

  const UT::HitOffsets ut_hit_offsets {
    parameters.dev_ut_hit_offsets, event_number, number_of_unique_x_sectors, dev_unique_x_sector_layer_offsets};
  const auto event_offset = ut_hit_offsets.event_offset();

  UT::ConstHits ut_hits {parameters.dev_ut_hits, total_number_of_hits};

  // Create consolidated SoAs.
  UT::Consolidated::Tracks ut_tracks {parameters.dev_atomics_ut,
                                      parameters.dev_ut_track_hit_number,
                                      parameters.dev_ut_qop,
                                      parameters.dev_ut_track_velo_indices,
                                      event_number,
                                      number_of_events};

  const uint number_of_tracks_event = ut_tracks.number_of_tracks(event_number);
  const uint event_tracks_offset = ut_tracks.tracks_offset(event_number);

  // Loop over tracks.
  for (uint i = threadIdx.x; i < number_of_tracks_event; i += blockDim.x) {
    const UT::TrackHits& track = event_veloUT_tracks[i];

    ut_tracks.velo_track(i) = track.velo_track_index;
    ut_tracks.qop(i) = track.qop;

    const int track_index = event_tracks_offset + i;
    parameters.dev_ut_x[track_index] = track.x;
    parameters.dev_ut_z[track_index] = track.z;
    parameters.dev_ut_tx[track_index] = track.tx;

    UT::Consolidated::Hits consolidated_hits = ut_tracks.get_hits(parameters.dev_ut_track_hits, i);

    // Populate the consolidated hits.
    populate(track, [&consolidated_hits, &ut_hits, &event_offset](const uint hit_number, const uint j) {
      consolidated_hits.yBegin(hit_number) = ut_hits.yBegin(j + event_offset);
    });

    populate(track, [&consolidated_hits, &ut_hits, &event_offset](const uint hit_number, const uint j) {
      consolidated_hits.yEnd(hit_number) = ut_hits.yEnd(j + event_offset);
    });

    populate(track, [&consolidated_hits, &ut_hits, &event_offset](const uint hit_number, const uint j) {
      consolidated_hits.zAtYEq0(hit_number) = ut_hits.zAtYEq0(j + event_offset);
    });

    populate(track, [&consolidated_hits, &ut_hits, &event_offset](const uint hit_number, const uint j) {
      consolidated_hits.xAtYEq0(hit_number) = ut_hits.xAtYEq0(j + event_offset);
    });

    populate(track, [&consolidated_hits, &ut_hits, &event_offset](const uint hit_number, const uint j) {
      consolidated_hits.id(hit_number) = ut_hits.id(j + event_offset);
    });

    populate(track, [&consolidated_hits, &ut_hits, &event_offset](const uint hit_number, const uint j) {
      consolidated_hits.weight(hit_number) = ut_hits.weight(j + event_offset);
    });

    populate_plane_code(track, [&consolidated_hits, &event_offset](const uint hit_number, const uint j) {
      consolidated_hits.plane_code(hit_number) = static_cast<uint8_t>(j + event_offset);
    });
  }
}
