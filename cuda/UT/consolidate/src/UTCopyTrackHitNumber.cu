#include "hip/hip_runtime.h"
#include "UTCopyTrackHitNumber.cuh"

void ut_copy_track_hit_number_t::set_arguments_size(
  ArgumentRefManager<Arguments> arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers) const
{
  arguments.set_size<dev_ut_track_hit_number>(host_buffers.ut_track_hit_number_size());
}

void ut_copy_track_hit_number_t::operator()(
  const ArgumentRefManager<Arguments>& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event) const
{
  function.invoke(dim3(host_buffers.host_number_of_selected_events[0]), block_dimension(), hip_stream)(
    arguments.offset<dev_ut_tracks>(),
    arguments.offset<dev_atomics_ut>(),
    arguments.offset<dev_ut_track_hit_number>());
}

/**
 * @brief Copies UT track hit numbers on a consecutive container
 */
__global__ void ut_copy_track_hit_number(
  const UT::TrackHits* dev_veloUT_tracks,
  uint* dev_atomics_veloUT,
  uint* dev_ut_track_hit_number)
{
  const auto number_of_events = gridDim.x;
  const auto event_number = blockIdx.x;
  const auto* event_tracks = dev_veloUT_tracks + event_number * UT::Constants::max_num_tracks;
  const auto accumulated_tracks = dev_atomics_veloUT[number_of_events + event_number];
  const auto number_of_tracks = dev_atomics_veloUT[event_number];

  // Pointer to ut_track_hit_number of current event.
  uint* ut_track_hit_number = dev_ut_track_hit_number + accumulated_tracks;

  // Loop over tracks.
  for (uint element = threadIdx.x; element < number_of_tracks; ++element) {
    ut_track_hit_number[element] = event_tracks[element].hits_num;
  }
}
