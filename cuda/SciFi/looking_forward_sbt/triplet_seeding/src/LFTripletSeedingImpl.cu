#include "hip/hip_runtime.h"
#include "LFTripletSeedingImpl.cuh"
#include "BinarySearchTools.cuh"

__device__ void lf_triplet_seeding_impl(
  const SciFi::Hits& scifi_hits,
  const uint h0_candidate_offset,
  const uint h1_candidate_offset,
  const uint h2_candidate_offset,
  const uint8_t h0_candidate_size,
  const uint8_t h1_candidate_size,
  const uint8_t h2_candidate_size,
  const uint8_t relative_middle_layer,
  const short* dev_scifi_lf_candidates,
  const float max_chi2,
  float* best_chi2,
  int8_t* best_h0_h2,
  const uint event_offset)
{
  // Tile size is 16
  constexpr int tile_size = 16;
  __shared__ float shared_chi2[tile_size * tile_size];
  __shared__ uint8_t shared_index[tile_size * tile_size];

  // Required constants for the chi2 calculation below
  const auto dz1 = (z1 - z0);
  const auto dz2 = (z2 - z0);
  const auto zdiff_inv = 1.f / dz1;
  auto extrap1 = SciFi::LookingForward::forward_param * qop * dz1 * dz1;
  extrap1 *= extrap1;
  const auto extrap2 = SciFi::LookingForward::forward_param * qop * dz2 * dz2;

  // Search best triplet
  for (int8_t i = 0; i < (h0_candidate_size + tile_size + 1) / tile_size; ++i) {
    // No Tensor core version
    // Note: BlockDim.x should be tile_size
    const int8_t h0_rel = i * tile_size + threadIdx.x;
    for (int8_t j = 0; j < (h2_candidate_size + tile_size + 1) / tile_size; ++j) {

      // Note: wmma can be allocated here

      // Iterate over all h1s
      for (int8_t h1_rel = 0; h1_rel < h1_candidate_size; ++h1_rel) {
        const auto h1_rel_index = dev_scifi_lf_candidates[h1_candidate_offset + h1_rel];
        const auto x1 = scifi_hits.x0[event_offset + h1_rel_index];

        // Note: wmma addition can be allocated here

        // Initialize shared_chi2
        __syncthreads();
        for (uint8_t k = threadIdx.x; k < tile_size * tile_size; k += tile_size) {
          shared_chi2[k] = max_chi2;
          shared_index[k] = k;
        }
        __syncthreads();

        // Note: This block of code is doable by wmma
        for (int8_t k = 0; k < tile_size && ++k) {
          const int8_t h2_rel = j * tile_size + k;
          if (h0_rel < h0_candidate_size && h2_rel < h2_candidate_size) {
            const auto h0_rel_index = dev_scifi_lf_candidates[h0_candidate_offset + h0_rel];
            const auto x0 = scifi_hits.x0[event_offset + h0_rel_index];
            
            const auto h2_rel_index = dev_scifi_lf_candidates[h2_candidate_offset + h2_rel];
            const auto x2 = scifi_hits.x0[event_offset + h2_rel_index];
            
            const auto tx = x1 * zdiff_inv - x0 * zdiff_inv;
            const auto expected_x2 = x0 + tx * dz2 + extrap2;
            const auto chi2 = extrap1 + (x2 - expected_x2) * (x2 - expected_x2);
            shared_chi2[i * tile_size + j] = chi2;
          }
        }

        __syncthreads();

        // shared_chi2 contains all calculated chi2s
        // Use a reduction to fetch the best one
        // uint16_t to be on the safe side
        for (uint16_t s = 1; s < blockDim.x; s *= 2) {
          if (threadIdx.x % (2 * s) == 0 && shared_chi2[threadIdx.x + s] < shared_chi2[threadIdx.x]) {
            shared_chi2[threadIdx.x] = shared_chi2[threadIdx.x + s];
            shared_index[threadIdx.x] = shared_index[threadIdx.x + s];
          }
          __syncthreads();
        }

        // Note: We could keep in a buffer the best shared_chi2 found (shared_chi2[0]) and
        //       the best h0h2 found (shared_index[0]), and do the following check simultaneously
        //       for all h1s
        //
        // Now shared_chi2 and shared_index contain the best chi2 and best index, respectively
        // Compare that to the currently stored and save it if's better
        if (threadIdx.x == 0 && shared_chi2[0] < best_chi2[h1_rel]) {
          best_chi2[h1_rel] = shared_chi2[0];
          best_h0_h2[h1_rel] = i * tile_size + (shared_index[0] >> 4);
          best_h0_h2[64 + h1_rel] = j * tile_size + (shared_index[0] % 16);
        }

        __syncthreads();
      }
    }
  }
}
