#include "hip/hip_runtime.h"
#include "LFTripletSeedingImpl.cuh"
#include "BinarySearchTools.cuh"
#include <mma.h>

__device__ void lf_triplet_seeding_choose_best_triplets_for_h1(
  const float* scifi_hits_x0,
  const short* scifi_lf_candidates,
  const uint8_t relative_l0,
  const float zdiff,
  const float* shared_partial_chi2,
  const float extrap1,
  const int h1_candidate_size,
  const int8_t h0_tile_index,
  const int8_t h2_tile_index,
  const float max_chi2,
  float* best_chi2,
  int8_t* best_h0_h2)
{

  // Iterate over all h1s
  // Find best chi2, h0 and h2 using the partial chi2 from before
  for (int16_t h1_rel = threadIdx.x; h1_rel < h1_candidate_size; h1_rel += blockDim.x) {
    const float x1_zdiff =
      scifi_hits_x0
      [scifi_lf_candidates[(relative_l0 + 1) * LookingForward::maximum_number_of_candidates + h1_rel]] *
      zdiff;

    // populate chi2 by which we sort
    float chi2_tile[LookingForward::tile_size*LookingForward::tile_size];
    for (int16_t k = 0; k < LookingForward::tile_size * LookingForward::tile_size; ++k) {
      float chi2 = shared_partial_chi2[k] - x1_zdiff;
      chi2 = extrap1 + chi2 * chi2;
      chi2_tile[k] = chi2;
    }

    // Sort chi2 of this tile
    int16_t best_chi2_tile[LookingForward::maximum_number_of_triplets_per_h1];
    for (int16_t k = 0; k <  LookingForward::maximum_number_of_triplets_per_h1; ++k) {
      best_chi2_tile[k] = -1;
    }
    for (int16_t k = 0; k < LookingForward::tile_size * LookingForward::tile_size; ++k) {
      const float chi2 = chi2_tile[k];
      if (chi2 < max_chi2) {
        int16_t insert_position = 0;
        for (int16_t l = 0; l < LookingForward::tile_size * LookingForward::tile_size; ++l) {
          const float other_chi2 = chi2_tile[l];
          if (chi2 > other_chi2 || (chi2 == other_chi2 && k < l)) {
            ++insert_position;
          }
        }
        if (insert_position < LookingForward::maximum_number_of_triplets_per_h1) {
          best_chi2_tile[insert_position] = k;
        }
      }
    }

    // Insert the best candidates in the array of all triplets for this h1 (not only the tile)
    // if the chi2 is better than previous candidates
    int16_t pos = 0;
    for (int16_t k = 0; k < LookingForward::maximum_number_of_triplets_per_h1; ++k) {
      if (best_chi2_tile[pos] < 0) break;
      if ( chi2_tile[best_chi2_tile[pos]] < best_chi2[h1_rel * LookingForward::maximum_number_of_triplets_per_h1 + k] ) {
        best_chi2[h1_rel * LookingForward::maximum_number_of_triplets_per_h1 + k] = chi2_tile[best_chi2_tile[pos]];
        best_h0_h2[h1_rel * LookingForward::maximum_number_of_triplets_per_h1 + k] = h0_tile_index * LookingForward::tile_size + (best_chi2_tile[pos] >> LookingForward::tile_size_shift_div);
        best_h0_h2[LookingForward::maximum_number_of_candidates * LookingForward::maximum_number_of_triplets_per_h1 + h1_rel * LookingForward::maximum_number_of_triplets_per_h1 + k] =
          h2_tile_index * LookingForward::tile_size + (best_chi2_tile[pos] & LookingForward::tile_size_mask);
        pos++;
      }
    }
  }
}

__device__ void lf_triplet_seeding_impl(
  const float* scifi_hits_x0,
  const uint8_t h0_candidate_size,
  const uint8_t h1_candidate_size,
  const uint8_t h2_candidate_size,
  const uint8_t relative_l0,
  const float max_chi2,
  float* best_chi2,
  int8_t* best_h0_h2,
  const short* scifi_lf_candidates,
  const float dz1,
  const float dz2,
  const float qop)
{
  __shared__ float shared_partial_chi2[LookingForward::tile_size * LookingForward::tile_size];

  // Required constants for the chi2 calculation below
  float extrap1 = LookingForward::forward_param * qop * dz1 * dz1;
  extrap1 *= extrap1;
  const float zdiff = dz2 / dz1;
  const float extrap2 = LookingForward::forward_param * qop * dz2 * dz2;

// Tensor core specialization
#if __CUDA_ARCH__ >= 700

  const half zdiff_half = ((half) zdiff);
  const half big_max_chi2 = 1000.f * max_chi2;

  // Tensor core magic
  half* shared_wmma_a = (half*) shared_partial_chi2;
  half* shared_wmma_b = (half*) (shared_partial_chi2 + ((LookingForward::tile_size * LookingForward::tile_size) >> 1));

  // __shared__ half shared_wmma_a[LookingForward::tile_size * LookingForward::tile_size];
  // __shared__ half shared_wmma_b[LookingForward::tile_size * LookingForward::tile_size];
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, LookingForward::tile_size, LookingForward::tile_size, LookingForward::tile_size, half, nvcuda::wmma::col_major> a_frag;
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, LookingForward::tile_size, LookingForward::tile_size, LookingForward::tile_size, half, nvcuda::wmma::row_major> b_frag;
  nvcuda::wmma::fragment<nvcuda::wmma::accumulator, LookingForward::tile_size, LookingForward::tile_size, LookingForward::tile_size, float> c_frag;
  nvcuda::wmma::fragment<nvcuda::wmma::accumulator, LookingForward::tile_size, LookingForward::tile_size, LookingForward::tile_size, float> d_frag;
  nvcuda::wmma::fill_fragment(c_frag, -extrap2);

  // Search best triplets per h1

  // Tiled processing of h0 and h2
  for (int8_t i = 0; i<(h0_candidate_size + LookingForward::tile_size - 1)>> LookingForward::tile_size_shift_div; ++i) {
    for (int8_t j = 0; j<(h2_candidate_size + LookingForward::tile_size - 1)>> LookingForward::tile_size_shift_div; ++j) {
      // Initialize wmma shared memory arrays
      for (int16_t k = threadIdx.x; k < LookingForward::tile_size * LookingForward::tile_size; k += blockDim.x) {
        shared_partial_chi2[k] = 0;
      }
      for (int16_t k = threadIdx.x; k < LookingForward::tile_size; k += blockDim.x) {
        shared_wmma_a[k] = 1;
        shared_wmma_b[LookingForward::tile_size + k] = 1;
        shared_wmma_b[2 * LookingForward::tile_size + k] = zdiff_half;
      }

      for (int16_t k = threadIdx.x; k < LookingForward::tile_size; k += blockDim.x) {
        const int8_t h0_rel = i * LookingForward::tile_size + k;
        if (h0_rel < h0_candidate_size) {
          const half x0 =
            scifi_hits_x0[scifi_lf_candidates[relative_l0 * LookingForward::maximum_number_of_candidates + h0_rel]];
          shared_wmma_a[LookingForward::tile_size + k] = -x0;
          shared_wmma_a[2 * LookingForward::tile_size + k] = x0;
        }
        else {
          shared_wmma_a[LookingForward::tile_size + k] = big_max_chi2;
          shared_wmma_a[2 * LookingForward::tile_size + k] = big_max_chi2;
        }
      }
      // TODO: Needed?
      __syncthreads();
      nvcuda::wmma::load_matrix_sync(a_frag, shared_wmma_a, LookingForward::tile_size);

      for (int16_t k = threadIdx.x; k < LookingForward::tile_size; k += blockDim.x) {
        const int8_t h2_rel = j * LookingForward::tile_size + k;
        if (h2_rel < h2_candidate_size) {
          shared_wmma_b[k] = scifi_hits_x0
            [scifi_lf_candidates[(relative_l0 + 2) * LookingForward::maximum_number_of_candidates + h2_rel]];
        }
        else {
          shared_wmma_b[k] = big_max_chi2;
        }
      }
      // TODO: Needed?
      __syncthreads();
      nvcuda::wmma::load_matrix_sync(b_frag, shared_wmma_b, LookingForward::tile_size);

      // Magic :)
      nvcuda::wmma::mma_sync(d_frag, a_frag, b_frag, c_frag);
      nvcuda::wmma::store_matrix_sync(shared_partial_chi2, d_frag, LookingForward::tile_size, nvcuda::wmma::mem_row_major);

      lf_triplet_seeding_choose_best_triplets_for_h1(
        scifi_hits_x0,
        scifi_lf_candidates,
        relative_l0,
        zdiff,
        shared_partial_chi2,
        extrap1,
        h1_candidate_size,
        i,
        j,
        max_chi2,
        best_chi2,
        best_h0_h2);
    }
  }

#else

  // Search best triplets per h1

  // Tiled processing of h0 and h2
  for (int8_t i = 0; i<(h0_candidate_size + LookingForward::tile_size - 1)>> LookingForward::tile_size_shift_div; ++i) {
    for (int8_t j = 0; j<(h2_candidate_size + LookingForward::tile_size - 1)>> LookingForward::tile_size_shift_div; ++j) {

      __syncthreads();

      for (int16_t k = threadIdx.x; k < LookingForward::tile_size * LookingForward::tile_size; k += blockDim.x) {
        const int8_t h0_rel = i * LookingForward::tile_size + (k & LookingForward::tile_size_mask);
        const int8_t h2_rel = j * LookingForward::tile_size + (k >> LookingForward::tile_size_shift_div);

        float partial_chi2 = 1000.f * max_chi2;
        if (h0_rel < h0_candidate_size && h2_rel < h2_candidate_size) {
          const auto x0 =
            scifi_hits_x0[scifi_lf_candidates[relative_l0 * LookingForward::maximum_number_of_candidates + h0_rel]];
          const auto x2 = scifi_hits_x0
            [scifi_lf_candidates[(relative_l0 + 2) * LookingForward::maximum_number_of_candidates + h2_rel]];
          partial_chi2 = x2 - x0 + x0 * zdiff - extrap2;
          // Note: To get the chi2 from the partial_chi2:
          // extrap1 + (partial_chi2 - x1 * zdiff) * (partial_chi2 - x1 * zdiff)
        }
        shared_partial_chi2[k] = partial_chi2;
      }

      __syncthreads();

       lf_triplet_seeding_choose_best_triplets_for_h1(
        scifi_hits_x0,
        scifi_lf_candidates,
        relative_l0,
        zdiff,
        shared_partial_chi2,
        extrap1,
        h1_candidate_size,
        i,
        j,
        max_chi2,
        best_chi2,
        best_h0_h2);
    }
  }
#endif
}
