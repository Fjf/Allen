#include "hip/hip_runtime.h"
#include "LFQualityFilter.cuh"

__global__ void lf_quality_filter(
  const uint32_t* dev_scifi_hits,
  const uint32_t* dev_scifi_hit_count,
  const int* dev_atomics_velo,
  const uint* dev_velo_track_hit_number,
  const char* dev_velo_states,
  const int* dev_atomics_ut,
  const char* dev_ut_track_hits,
  const uint* dev_ut_track_hit_number,
  const float* dev_ut_qop,
  const uint* dev_ut_track_velo_indices,
  SciFi::TrackHits* dev_scifi_lf_tracks,
  const int* dev_scifi_lf_atomics,
  float* dev_scifi_lf_track_params,
  const char* dev_scifi_geometry,
  const float* dev_inv_clus_res,
  const SciFi::Tracking::TMVA* dev_tmva1,
  const SciFi::Tracking::TMVA* dev_tmva2,
  const SciFi::Tracking::Arrays* constArrays,
  const LookingForward::Constants* dev_looking_forward_constants,
  const float* dev_magnet_polarity,
  int* dev_atomics_scifi,
  SciFi::TrackHits* dev_scifi_tracks)
{
  const auto number_of_events = gridDim.x;
  const auto event_number = blockIdx.x;

  // Velo consolidated types
  const Velo::Consolidated::Tracks velo_tracks {
    (uint*) dev_atomics_velo, (uint*) dev_velo_track_hit_number, event_number, number_of_events};
  const Velo::Consolidated::States velo_states {(char*) dev_velo_states, velo_tracks.total_number_of_tracks};
  const uint velo_tracks_offset_event = velo_tracks.tracks_offset(event_number);

  // UT consolidated tracks
  const UT::Consolidated::Tracks ut_tracks {(uint*) dev_atomics_ut,
                                            (uint*) dev_ut_track_hit_number,
                                            (float*) dev_ut_qop,
                                            (uint*) dev_ut_track_velo_indices,
                                            event_number,
                                            number_of_events};
  const int ut_event_tracks_offset = ut_tracks.tracks_offset(event_number);
  const int ut_event_number_of_tracks = ut_tracks.number_of_tracks(event_number);

  // SciFi hits
  const uint total_number_of_hits = dev_scifi_hit_count[number_of_events * SciFi::Constants::n_mat_groups_and_mats];
  const SciFi::HitCount scifi_hit_count {(uint32_t*) dev_scifi_hit_count, event_number};
  const SciFi::SciFiGeometry scifi_geometry {dev_scifi_geometry};
  const SciFi::Hits scifi_hits {
    const_cast<uint32_t*>(dev_scifi_hits), total_number_of_hits, &scifi_geometry, dev_inv_clus_res};
  const auto event_offset = scifi_hit_count.event_offset();

  const auto number_of_tracks = dev_scifi_lf_atomics[event_number];

  for (int i = threadIdx.x; i < number_of_tracks; i += blockDim.x) {
    SciFi::TrackHits& track = dev_scifi_lf_tracks[ut_event_tracks_offset * LookingForward::maximum_number_of_candidates_per_ut_track_after_x_filter + i];
    const auto current_ut_track_index = ut_event_tracks_offset + track.ut_track_index;
    const auto velo_states_index = velo_tracks_offset_event + ut_tracks.velo_track[track.ut_track_index];
    float* trackParams = dev_scifi_lf_track_params + ut_event_tracks_offset * LookingForward::maximum_number_of_candidates_per_ut_track_after_x_filter * SciFi::Tracking::nTrackParams + i * SciFi::Tracking::nTrackParams;

    const MiniState velo_state = velo_states.getMiniState(velo_states_index);

    // float trackParams[SciFi::Tracking::nTrackParams];
    // lf_fit_impl(
    //   track,
    //   event_offset,
    //   scifi_hits,
    //   dev_looking_forward_constants,
    //   constArrays,
    //   velo_state,
    //   trackParams);

    track.quality = lf_track_quality(track, velo_state, dev_ut_qop[current_ut_track_index], trackParams, constArrays, dev_magnet_polarity[0], dev_tmva1, dev_tmva2);
  }

  __syncthreads();

  for (int i = threadIdx.x; i < ut_event_number_of_tracks; i += blockDim.x) {
    float best_quality = LookingForward::track_min_quality;
    short best_track_index = -1;

    for (int j = 0; j < number_of_tracks; j++) {
      const SciFi::TrackHits& track = dev_scifi_lf_tracks[ut_event_tracks_offset * LookingForward::maximum_number_of_candidates_per_ut_track_after_x_filter + j];
      if (track.ut_track_index == i && track.quality > best_quality) {
        best_quality = track.quality;
        best_track_index = j;
      }
    }

    if (best_track_index != -1) {
      const auto insert_index = atomicAdd(dev_atomics_scifi + event_number, 1);
      assert(insert_index < ut_event_number_of_tracks); // only one candidate per UT track, to do: check efficiency when allowing for more
      const auto& track = dev_scifi_lf_tracks[ut_event_tracks_offset * LookingForward::maximum_number_of_candidates_per_ut_track_after_x_filter + best_track_index];
      dev_scifi_tracks[event_number * SciFi::Constants::max_tracks + insert_index] = track;
    }
  }

}
