#include "hip/hip_runtime.h"
#include "LFQualityFilter.cuh"

__global__ void lf_quality_filter(
  const int* dev_atomics_velo,
  const uint* dev_velo_track_hit_number,
  const char* dev_velo_states,
  const int* dev_atomics_ut,
  const char* dev_ut_track_hits,
  const uint* dev_ut_track_hit_number,
  const float* dev_ut_qop,
  const uint* dev_ut_track_velo_indices,
  SciFi::TrackHits* dev_scifi_lf_tracks,
  const int* dev_scifi_lf_atomics,
  const float* dev_scifi_lf_track_params,
  const SciFi::Tracking::TMVA* dev_tmva1,
  const SciFi::Tracking::TMVA* dev_tmva2,
  const SciFi::Tracking::Arrays* constArrays,
  const float* dev_magnet_polarity,
  int* dev_atomics_scifi,
  SciFi::TrackHits* dev_scifi_tracks)
{
  const auto number_of_events = gridDim.x;
  const auto event_number = blockIdx.x;

  // Velo consolidated types
  const Velo::Consolidated::Tracks velo_tracks {
    (uint*) dev_atomics_velo, (uint*) dev_velo_track_hit_number, event_number, number_of_events};
  const Velo::Consolidated::States velo_states {(char*) dev_velo_states, velo_tracks.total_number_of_tracks};
  const uint velo_tracks_offset_event = velo_tracks.tracks_offset(event_number);

  // UT consolidated tracks
  const UT::Consolidated::Tracks ut_tracks {(uint*) dev_atomics_ut,
                                            (uint*) dev_ut_track_hit_number,
                                            (float*) dev_ut_qop,
                                            (uint*) dev_ut_track_velo_indices,
                                            event_number,
                                            number_of_events};
  const int ut_event_tracks_offset = ut_tracks.tracks_offset(event_number);
  const int ut_event_number_of_tracks = ut_tracks.number_of_tracks(event_number);

  Consolidated::TracksDescription ut_tracks_counter {(uint*) dev_atomics_ut, number_of_events};
  const int ut_event_tracks_offset_ = ut_tracks_counter.tracks_offset(event_number);
  const int ut_event_number_of_tracks_ = ut_tracks_counter.number_of_tracks(event_number);

  const auto number_of_tracks = dev_scifi_lf_atomics[event_number];

  for (int i = threadIdx.x; i < number_of_tracks; i += blockDim.x) {
    SciFi::TrackHits& track = dev_scifi_lf_tracks[ut_event_tracks_offset * LookingForward::maximum_number_of_candidates_per_ut_track_after_x_filter + i];
    const auto current_ut_track_index = ut_event_tracks_offset + track.ut_track_index;
    const auto velo_states_index = velo_tracks_offset_event + ut_tracks.velo_track[track.ut_track_index];
    const float* trackParams = dev_scifi_lf_track_params + ut_event_tracks_offset * LookingForward::maximum_number_of_candidates_per_ut_track_after_x_filter * SciFi::Tracking::nTrackParams + i * SciFi::Tracking::nTrackParams;
    const MiniState velo_state = velo_states.getMiniState(velo_states_index);
    track.quality = lf_track_quality(track, velo_state, dev_ut_qop[current_ut_track_index], trackParams, constArrays, dev_magnet_polarity[0], dev_tmva1, dev_tmva2);
  }

  __syncthreads();

  for (int i = threadIdx.x; i < ut_event_number_of_tracks; i += blockDim.x) {
    float best_quality = LookingForward::track_min_quality;
    short best_track_index = -1;

    for (int j = 0; j < number_of_tracks; j++) {
      const SciFi::TrackHits& track = dev_scifi_lf_tracks[ut_event_tracks_offset * LookingForward::maximum_number_of_candidates_per_ut_track_after_x_filter + j];
      if (track.ut_track_index == i && track.quality > best_quality) {
        best_quality = track.quality;
        best_track_index = j;
      }
    }

    if (best_track_index != -1) {
      const auto insert_index = atomicAdd(dev_atomics_scifi + event_number, 1);
      assert(insert_index < ut_event_number_of_tracks); // only one candidate per UT track, to do: check efficiency when allowing for more
      const auto& track = dev_scifi_lf_tracks[ut_event_tracks_offset * LookingForward::maximum_number_of_candidates_per_ut_track_after_x_filter + best_track_index];
      dev_scifi_tracks[event_number * SciFi::Constants::max_tracks + insert_index] = track;
    }
  }

}
