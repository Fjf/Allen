#include "hip/hip_runtime.h"
#include "LFQualityFilterLength.cuh"

__global__ void lf_quality_filter_length(
  const int* dev_atomics_ut,
  const SciFi::TrackHits* dev_scifi_lf_tracks,
  const int* dev_scifi_lf_atomics,
  const float* dev_scifi_lf_xAtRef,
  float* dev_scifi_lf_xAtRef_after_length_filter,
  SciFi::TrackHits* dev_scifi_lf_filtered_tracks,
  int* dev_scifi_lf_filtered_atomics)
{
  const auto event_number = blockIdx.x;
  const auto number_of_events = gridDim.x;

  const int ut_event_tracks_offset = dev_atomics_ut[number_of_events + event_number];
  const auto number_of_tracks = dev_scifi_lf_atomics[event_number];

  for (int i = threadIdx.x; i < number_of_tracks; i += blockDim.x) {
    const SciFi::TrackHits& track =
      dev_scifi_lf_tracks[ut_event_tracks_offset * LookingForward::maximum_number_of_candidates_per_ut_track_after_x_filter + i];
    if (track.hitsNum >= 9) {
      const auto insert_index = atomicAdd(dev_scifi_lf_filtered_atomics + event_number, 1);
      dev_scifi_lf_filtered_tracks
        [ut_event_tracks_offset * LookingForward::maximum_number_of_candidates_per_ut_track_after_x_filter + insert_index] = track;
      dev_scifi_lf_xAtRef_after_length_filter[ut_event_tracks_offset * LookingForward::maximum_number_of_candidates_per_ut_track_after_x_filter + insert_index] = dev_scifi_lf_xAtRef[ut_event_tracks_offset * LookingForward::maximum_number_of_candidates_per_ut_track_after_x_filter + i];
    }
  }
}
