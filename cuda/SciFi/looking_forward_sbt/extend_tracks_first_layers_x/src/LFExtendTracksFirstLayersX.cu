#include "hip/hip_runtime.h"
#include "LFExtendTracksFirstLayersX.cuh"

__global__ void lf_extend_tracks_first_layers_x(
  const uint32_t* dev_scifi_hits,
  const uint32_t* dev_scifi_hit_count,
  const int* dev_atomics_ut,
  SciFi::TrackHits* dev_scifi_tracks,
  int* dev_atomics_scifi,
  const char* dev_scifi_geometry,
  const LookingForward::Constants* dev_looking_forward_constants,
  const float* dev_inv_clus_res,
  const uint* dev_scifi_lf_number_of_candidates,
  const short* dev_scifi_lf_candidates,
  const uint8_t relative_extrapolation_layer)
{
  const auto number_of_events = gridDim.x;
  const auto event_number = blockIdx.x;

  // UT consolidated tracks
  const int ut_event_tracks_offset = dev_atomics_ut[number_of_events + event_number];

  // SciFi hits
  const uint total_number_of_hits = dev_scifi_hit_count[number_of_events * SciFi::Constants::n_mat_groups_and_mats];
  const SciFi::HitCount scifi_hit_count {(uint32_t*) dev_scifi_hit_count, event_number};
  const SciFi::SciFiGeometry scifi_geometry {dev_scifi_geometry};
  const SciFi::Hits scifi_hits {
    const_cast<uint32_t*>(dev_scifi_hits), total_number_of_hits, &scifi_geometry, dev_inv_clus_res};
  const auto event_offset = scifi_hit_count.event_offset();

  // TODO: Maybe move this somewhere else
  if (threadIdx.x == 0) {
    const int temp_number_of_tracks = dev_atomics_scifi[event_number];
    if (temp_number_of_tracks > SciFi::Constants::max_lf_tracks) {
      dev_atomics_scifi[event_number] = SciFi::Constants::max_lf_tracks;
    }
  }

  __syncthreads();

  // SciFi un-consolidated track types
  const int number_of_tracks = dev_atomics_scifi[event_number];

  for (int i = threadIdx.x; i < number_of_tracks; i += blockDim.x) {
    SciFi::TrackHits& track = dev_scifi_tracks[event_number * SciFi::Constants::max_lf_tracks + i];
    const auto current_ut_track_index = ut_event_tracks_offset + track.ut_track_index;

    // Candidates pointer for current UT track
    const auto scifi_lf_candidates = dev_scifi_lf_candidates + current_ut_track_index *
                                                                 LookingForward::number_of_x_layers *
                                                                 LookingForward::maximum_number_of_candidates;

    const int8_t number_of_candidates =
      dev_scifi_lf_number_of_candidates
        [current_ut_track_index * LookingForward::number_of_x_layers + relative_extrapolation_layer + 1] -
      dev_scifi_lf_number_of_candidates
        [current_ut_track_index * LookingForward::number_of_x_layers + relative_extrapolation_layer];

    // TODO: Use here first hits in track
    const auto h0 = event_offset + track.hits[0];
    const auto h1 = event_offset + track.hits[1];

    const auto layer0 = scifi_hits.planeCode(h0) >> 1;
    const auto layer1 = scifi_hits.planeCode(h1) >> 1;

    if (relative_extrapolation_layer != dev_looking_forward_constants->convert_layer[layer0]
      && relative_extrapolation_layer != dev_looking_forward_constants->convert_layer[layer1]) {

      const auto x0 = scifi_hits.x0[h0];
      const auto x1 = scifi_hits.x0[h1];

      const auto z0 = dev_looking_forward_constants->Zone_zPos[layer0];
      const auto z1 = dev_looking_forward_constants->Zone_zPos[layer1];
      const auto z2 = dev_looking_forward_constants->Zone_zPos_xlayers[relative_extrapolation_layer];

      lf_extend_tracks_first_layers_x_impl(
        scifi_hits.x0 + event_offset,
        scifi_lf_candidates + relative_extrapolation_layer * LookingForward::maximum_number_of_candidates,
        number_of_candidates,
        track,
        x0,
        x1,
        z0,
        z1,
        z2,
        dev_looking_forward_constants->chi2_mean_extrapolation_to_x_layers[0] +
          2.5f * dev_looking_forward_constants->chi2_stddev_extrapolation_to_x_layers[0]);
    }
  }
}
