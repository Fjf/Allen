#include "hip/hip_runtime.h"
#include "LFSearchInitialWindowsImpl.cuh"

__host__ __device__ inline float evalCubicParameterization(
  const float value_at_ref,
  const float t,
  const float z)
{
  float dz = z - SciFi::Tracking::zReference;
  return value_at_ref + t * dz;
}

//=========================================================================
// From LHCb Forward tracking description
//
// Collect all X hits, within a window defined by the minimum Pt.
// Better restrictions possible, if we use the momentum of the input track.
// Ask for the presence of a stereo hit in the same biLayer compatible.
// This reduces the efficiency. X-alone hits to be re-added later in the processing
//
// side = 1  -> upper y half
// side = -1 -> lower y half
//=========================================================================
__host__ __device__ void lf_search_initial_windows_impl(
  const SciFi::Hits& scifi_hits,
  const SciFi::HitCount& scifi_hit_count,
  const float xAtRef,
  const float yAtRef,
  const MiniState& velo_state,
  const SciFi::Tracking::Arrays* constArrays,
  const float qOverP,
  const int side,
  int* forward_windows)
{
  // Find size of search window on reference plane, using Velo slopes and min pT as input
  float dxRef = 0.9f * calcDxRef(SciFi::Tracking::minPt, velo_state);
  // find position within magnet where bending happens
  float zMag = zMagnet(velo_state, constArrays);

  const float q = qOverP > 0.f ? 1.f : -1.f;
  const float dir = q * SciFi::Tracking::magscalefactor * (-1.f);

  float slope2 = velo_state.tx * velo_state.tx + velo_state.ty * velo_state.ty;
  const float pt = sqrtf(fabsf(1.f / (qOverP * qOverP))) * (slope2) / (1.f + slope2);
  const bool wSignTreatment = SciFi::Tracking::useWrongSignWindow && pt > SciFi::Tracking::wrongSignPT;

  float dxRefWS = 0.f;
  if (wSignTreatment) {
    // DvB: what happens if we use the actual momentum from VeloUT here instead of a constant?
    dxRefWS = 0.9f * calcDxRef(
                       SciFi::Tracking::wrongSignPT,
                       velo_state); // make windows a bit too small - FIXME check effect of this, seems wrong
  }

  int iZoneStartingPoint = side > 0 ? constArrays->zoneoffsetpar : 0;

  for (int iZone = iZoneStartingPoint; iZone < iZoneStartingPoint + constArrays->zoneoffsetpar; iZone++) {
    // Initialize windows
    const auto relative_iZone = iZone - iZoneStartingPoint;
    forward_windows[relative_iZone * 4 + 0] = -1;
    forward_windows[relative_iZone * 4 + 1] = -1;
    forward_windows[relative_iZone * 4 + 2] = -1;
    forward_windows[relative_iZone * 4 + 3] = -1;

    assert(iZone - iZoneStartingPoint < SciFi::Constants::n_zones);
    assert(iZone - iZoneStartingPoint < 12);
    const float zZone = constArrays->xZone_zPos[iZone - iZoneStartingPoint];
    const float xInZone = evalCubicParameterization(xAtRef, velo_state.tx, zZone);
    const float yInZone = evalCubicParameterization(yAtRef, velo_state.ty, zZone);

    // Now the code checks if the x and y are in the zone limits. I am really not sure
    // why this is done here, surely could just check if within limits for the last zone
    // in T3 and go from there? Need to think more about this.
    //
    // Here for now I assume the same min/max x and y for all stations, this again needs to
    // be read from some file blablabla although actually I suspect having some general tolerances
    // here is anyway good enough since we are doing a straight line extrapolation in the first place
    // check (roughly) whether the extrapolated velo track is within the current zone
    if (side > 0) {
      if (
        !isInside(xInZone, SciFi::Tracking::xLim_Min, SciFi::Tracking::xLim_Max) ||
        !isInside(yInZone, SciFi::Tracking::yLim_Min, SciFi::Tracking::yLim_Max))
        continue;
    }
    else {
      if (
        !isInside(xInZone, SciFi::Tracking::xLim_Min, SciFi::Tracking::xLim_Max) ||
        !isInside(yInZone, side * SciFi::Tracking::yLim_Max, side * SciFi::Tracking::yLim_Min))
        continue;
    }

    // extrapolate dxRef (x window on reference plane) to plane of current zone
    const float xTol = (zZone < SciFi::Tracking::zReference) ?
                         dxRef * zZone / SciFi::Tracking::zReference :
                         dxRef * (zZone - zMag) / (SciFi::Tracking::zReference - zMag);
    float xMin = xInZone - xTol;
    float xMax = xInZone + xTol;

    if (SciFi::Tracking::useMomentumEstimate) { // For VeloUT tracks, suppress check if track actually has qOverP set,
                                                // get the option right!
      float xTolWS = 0.0;
      if (wSignTreatment) {
        xTolWS = (zZone < SciFi::Tracking::zReference) ?
                   dxRefWS * zZone / SciFi::Tracking::zReference :
                   dxRefWS * (zZone - zMag) / (SciFi::Tracking::zReference - zMag);
      }
      if (dir > 0) {
        xMin = xInZone - xTolWS;
      }
      else {
        xMax = xInZone + xTolWS;
      }
    }

    // Get the hits within the bounds
    assert(iZone < SciFi::Constants::n_layers);
    assert(constArrays->xZones[iZone] < SciFi::Constants::n_zones);
    int x_zone_offset_begin = scifi_hit_count.zone_offset(constArrays->xZones[iZone]);
    int x_zone_offset_end = x_zone_offset_begin + scifi_hit_count.zone_number_of_hits(constArrays->xZones[iZone]);
    const int itH = getLowerBound(scifi_hits.x0, xMin, x_zone_offset_begin, x_zone_offset_end);
    const int itEnd = getLowerBound(scifi_hits.x0, xMax, x_zone_offset_begin, x_zone_offset_end);

    // Initialize windows
    forward_windows[relative_iZone * 4 + 0] = itH;
    forward_windows[relative_iZone * 4 + 1] = itEnd;
    
    assert(itH >= x_zone_offset_begin && itH <= x_zone_offset_end);
    assert(itEnd >= x_zone_offset_begin && itEnd <= x_zone_offset_end);

    // Skip making range but continue if the end is before or equal to the start
    if (!(itEnd > itH)) continue;

    // Now match the stereo hits
    const float this_uv_z = constArrays->uvZone_zPos[iZone - iZoneStartingPoint];
    const float xInUv = evalCubicParameterization(xAtRef, velo_state.tx, this_uv_z);
    const float zRatio = (this_uv_z - zMag) / (zZone - zMag);
    const float dx = yInZone * constArrays->uvZone_dxdy[iZone - iZoneStartingPoint];
    const float xCentral = xInZone + dx;
    const float xPredUv = xInUv + (scifi_hits.x0[itH] - xInZone) * zRatio - dx;
    const float maxDx = SciFi::Tracking::tolYCollectX +
                        (fabsf(scifi_hits.x0[itH] - xCentral) + fabsf(yInZone)) * SciFi::Tracking::tolYSlopeCollectX;
    const float xMinUV = xPredUv - maxDx;

    // Get bounds in UV layers
    // do one search on the same side as the x module
    // if we are close to y = 0, also look within a region on the other side module ("triangle search")
    assert(constArrays->uvZones[iZone] < SciFi::Constants::n_zones);
    const int uv_zone_offset_begin = scifi_hit_count.zone_offset(constArrays->uvZones[iZone]);
    const int uv_zone_offset_end =
      uv_zone_offset_begin + scifi_hit_count.zone_number_of_hits(constArrays->uvZones[iZone]);
    const int triangleOffset = side > 0 ? -1 : 1;
    assert(constArrays->uvZones[iZone + constArrays->zoneoffsetpar * triangleOffset] < SciFi::Constants::n_zones);
    const int triangle_zone_offset_begin =
      scifi_hit_count.zone_offset(constArrays->uvZones[iZone + constArrays->zoneoffsetpar * triangleOffset]);
    assert(constArrays->uvZones[iZone + constArrays->zoneoffsetpar * triangleOffset] < SciFi::Constants::n_zones);
    const int triangle_zone_offset_end =
      triangle_zone_offset_begin +
      scifi_hit_count.zone_number_of_hits(constArrays->uvZones[iZone + constArrays->zoneoffsetpar * triangleOffset]);
    int itUV1 = getLowerBound(scifi_hits.x0, xMinUV, uv_zone_offset_begin, uv_zone_offset_end);
    int itUV2 = getLowerBound(scifi_hits.x0, xMinUV, triangle_zone_offset_begin, triangle_zone_offset_end);

    forward_windows[relative_iZone * 4 + 2] = itUV1;
    forward_windows[relative_iZone * 4 + 3] = itUV2;
  }
}
