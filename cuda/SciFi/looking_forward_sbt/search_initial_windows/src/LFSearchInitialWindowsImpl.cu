#include "hip/hip_runtime.h"
#include "LFSearchInitialWindowsImpl.cuh"
#include "LookingForwardConstants.cuh"
#include "LookingForwardTools.cuh"
#include "BinarySearch.cuh"

__device__ inline float linear_parameterization(const float value_at_ref, const float t, const float z)
{
  float dz = z - SciFi::Tracking::zReference;
  return value_at_ref + t * dz;
}

__device__ void lf_search_initial_windows_p_impl(
  const SciFi::Hits& scifi_hits,
  const SciFi::HitCount& scifi_hit_count,
  const MiniState& velo_state,
  const MiniState& UT_state,
  const SciFi::Tracking::Arrays* constArrays,
  const float magnet_polarity,
  const LookingForward::Constants* looking_forward_constants,
  const float qop,
  const int side,
  int* initial_windows,
  const int number_of_tracks)
{
  // find position within magnet where bending happens
  float zMag = zMagnet(UT_state, constArrays);

  int iZoneStartingPoint = (side > 0) ? constArrays->zoneoffsetpar : 0;

  for (int i = threadIdx.y; i < LookingForward::number_of_x_layers; i += blockDim.y) {
    const auto iZone = iZoneStartingPoint + i;
    const float zZone = constArrays->xZone_zPos[i];

    // TODO this could be done in a more optimized way
    const auto stateInZone = LookingForward::propagate_state_from_velo_multi_par(
      UT_state, qop, looking_forward_constants->x_layers[i], looking_forward_constants);

    const float xInZone = stateInZone.x;
    //const float yInZone = stateInZone.y;

    const float xMag = LookingForward::state_at_z(UT_state, LookingForward::z_magnet).x;

    const float xTol = 1.5f * LookingForward::dx_calc(UT_state.tx, qop);
    float xMin = xInZone - xTol;
    float xMax = xInZone + xTol;

    // Get the hits within the bounds
    const int x_zone_offset_begin = scifi_hit_count.zone_offset(constArrays->xZones[iZone]);
    const int x_zone_size = scifi_hit_count.zone_number_of_hits(constArrays->xZones[iZone]);
    int hits_within_bounds_start = binary_search_leftmost(scifi_hits.x0 + x_zone_offset_begin, x_zone_size, xMin);
    int hits_within_bounds_size = binary_search_leftmost(
      scifi_hits.x0 + x_zone_offset_begin + hits_within_bounds_start, x_zone_size - hits_within_bounds_start, xMax);
    hits_within_bounds_start += x_zone_offset_begin;

    // Initialize windows
    initial_windows[i * 8 * number_of_tracks] = hits_within_bounds_start;
    initial_windows[(i * 8 + 1) * number_of_tracks] = hits_within_bounds_size;

    // Skip making range but continue if the size is zero
    if (hits_within_bounds_size > 0) {
      // Now match the stereo hits
      const float this_uv_z = constArrays->uvZone_zPos[i];
      const float dz = this_uv_z - zZone;
      const float xInUv = LookingForward::linear_propagation(xInZone, stateInZone.tx, dz);
      const float UvCorr =
        LookingForward::y_at_z(stateInZone, this_uv_z) * constArrays->uvZone_dxdy[i];
      const float xInUvCorr = xInUv - UvCorr;
      const float xMinUV = xInUvCorr - 800.f;
      const float dz_ratio = (this_uv_z - zZone) / (LookingForward::z_magnet - zZone);

      // Get bounds in UV layers
      // do one search on the same side as the x module
      // if we are close to y = 0, also look within a region on the other side module ("triangle search")
      const int uv_zone_offset_begin = scifi_hit_count.zone_offset(constArrays->uvZones[iZone]);
      const int uv_zone_size = scifi_hit_count.zone_number_of_hits(constArrays->uvZones[iZone]);
      const int hits_within_uv_bounds =
        binary_search_leftmost(scifi_hits.x0 + uv_zone_offset_begin, uv_zone_size, xMinUV);

      initial_windows[(i * 8 + 2) * number_of_tracks] = hits_within_uv_bounds + uv_zone_offset_begin;
      initial_windows[(i * 8 + 3) * number_of_tracks] = uv_zone_size - hits_within_uv_bounds;

      float* initial_windows_f = (float*) &initial_windows[0];
      initial_windows_f[(i * 8 + 4) * number_of_tracks] = xMag;
      initial_windows_f[(i * 8 + 5) * number_of_tracks] = UvCorr;
      // TODO this should be read from the constants
      initial_windows_f[(i * 8 + 6) * number_of_tracks] = constArrays->uv_dx[i];
      initial_windows_f[(i * 8 + 7) * number_of_tracks] = dz_ratio;
    }
  }
}

__device__ void lf_search_initial_windows_impl(
  const SciFi::Hits& scifi_hits,
  const SciFi::HitCount& scifi_hit_count,
  const float xAtRef,
  const float yAtRef,
  const MiniState& UT_state,
  const SciFi::Tracking::Arrays* constArrays,
  const float magnet_polarity,
  const float qop,
  const int side,
  int* initial_windows,
  const int number_of_tracks)
{
  // Find size of search window on reference plane, using Velo slopes and min pT as input
  const float tx2 = UT_state.tx * UT_state.tx;
  const float ty2 = UT_state.ty * UT_state.ty;
  const float slope2 = tx2 + ty2;
  const float dxRef =
    3973000.f * sqrtf(slope2) / SciFi::Tracking::minPt - 2200.f * ty2 - 1000.f * tx2; // tune this window
  // dxRef *= 0.9f;

  // find position within magnet where bending happens
  float zMag = zMagnet(UT_state, constArrays);

  const float q = qop > 0.f ? 1.f : -1.f;
  const float dir = q * magnet_polarity * (-1.f);

  const float pt = std::sqrt(slope2 / (1.f + slope2)) / std::abs(qop);
  const bool wSignTreatment = SciFi::Tracking::useWrongSignWindow && pt > SciFi::Tracking::wrongSignPT;

  float dxRefWS = 0.f;
  if (wSignTreatment) {
    // DvB: what happens if we use the actual momentum from VeloUT here instead of a constant?
    dxRefWS = 0.9f * calcDxRef(
                       SciFi::Tracking::wrongSignPT,
                       UT_state); // make windows a bit too small - FIXME check effect of this, seems wrong
  }

  int iZoneStartingPoint = side > 0 ? constArrays->zoneoffsetpar : 0;

  for (int i = threadIdx.y; i < LookingForward::number_of_x_layers; i += blockDim.y) {
    const auto iZone = iZoneStartingPoint + i;
    const float zZone = constArrays->xZone_zPos[i];
    const float xInZone = linear_parameterization(xAtRef, UT_state.tx, zZone);
    const float yInZone = linear_parameterization(yAtRef, UT_state.ty, zZone);

    // extrapolate dxRef (x window on reference plane) to plane of current zone
    const float xTol = (zZone < SciFi::Tracking::zReference) ?
                         dxRef * zZone / SciFi::Tracking::zReference :
                         dxRef * (zZone - zMag) / (SciFi::Tracking::zReference - zMag);
    float xMin = xInZone - xTol;
    float xMax = xInZone + xTol;

    if (SciFi::Tracking::useMomentumEstimate) {
      float xTolWS = 0.0;
      if (wSignTreatment) {
        xTolWS = (zZone < SciFi::Tracking::zReference) ?
                   dxRefWS * zZone / SciFi::Tracking::zReference :
                   dxRefWS * (zZone - zMag) / (SciFi::Tracking::zReference - zMag);
      }
      if (dir > 0) {
        xMin = xInZone - xTolWS;
      }
      else {
        xMax = xInZone + xTolWS;
      }
    }

    // Get the hits within the bounds
    const int x_zone_offset_begin = scifi_hit_count.zone_offset(constArrays->xZones[iZone]);
    const int x_zone_size = scifi_hit_count.zone_number_of_hits(constArrays->xZones[iZone]);
    int hits_within_bounds_start = binary_search_leftmost(scifi_hits.x0 + x_zone_offset_begin, x_zone_size, xMin);
    int hits_within_bounds_size = binary_search_leftmost(
      scifi_hits.x0 + x_zone_offset_begin + hits_within_bounds_start, x_zone_size - hits_within_bounds_start, xMax);
    hits_within_bounds_start += x_zone_offset_begin;

    // Initialize windows
    initial_windows[i * 8 * number_of_tracks] = hits_within_bounds_start;
    initial_windows[(i * 8 + 1) * number_of_tracks] = hits_within_bounds_size;

    // Skip making range but continue if the size is zero
    if (hits_within_bounds_size > 0) {
      // Now match the stereo hits
      const float this_uv_z = constArrays->uvZone_zPos[i];
      const float xInUv = linear_parameterization(xAtRef, UT_state.tx, this_uv_z);
      const float zRatio = (this_uv_z - zMag) / (zZone - zMag);
      const float dx = yInZone * constArrays->uvZone_dxdy[iZone - iZoneStartingPoint];
      const float xCentral = xInZone + dx;
      const float xPredUv = xInUv + (scifi_hits.x0[hits_within_bounds_start] - xInZone) * zRatio - dx;
      const float maxDx = SciFi::Tracking::tolYCollectX +
                          (std::abs(scifi_hits.x0[hits_within_bounds_start] - xCentral) + std::abs(yInZone)) *
                            SciFi::Tracking::tolYSlopeCollectX;
      const float xMinUV = xPredUv - maxDx;
      const float xPredUVProto = xInUv - xInZone * zRatio - dx;
      const float maxDxProto = SciFi::Tracking::tolYCollectX + std::abs(yInZone) * SciFi::Tracking::tolYSlopeCollectX;

      // Get bounds in UV layers
      // do one search on the same side as the x module
      // if we are close to y = 0, also look within a region on the other side module ("triangle search")
      const int uv_zone_offset_begin = scifi_hit_count.zone_offset(constArrays->uvZones[iZone]);
      const int uv_zone_size = scifi_hit_count.zone_number_of_hits(constArrays->uvZones[iZone]);
      const int hits_within_uv_bounds =
        binary_search_leftmost(scifi_hits.x0 + uv_zone_offset_begin, uv_zone_size, xMinUV);

      initial_windows[(i * 8 + 2) * number_of_tracks] = hits_within_uv_bounds + uv_zone_offset_begin;
      initial_windows[(i * 8 + 3) * number_of_tracks] = uv_zone_size - hits_within_uv_bounds;

      float* initial_windows_f = (float*) &initial_windows[0];
      initial_windows_f[(i * 8 + 4) * number_of_tracks] = xPredUVProto;
      initial_windows_f[(i * 8 + 5) * number_of_tracks] = zRatio;
      initial_windows_f[(i * 8 + 6) * number_of_tracks] = maxDxProto;
      initial_windows_f[(i * 8 + 7) * number_of_tracks] = xCentral;
    }
  }
}
