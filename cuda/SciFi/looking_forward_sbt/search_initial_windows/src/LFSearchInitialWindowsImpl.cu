#include "hip/hip_runtime.h"
#include "LFSearchInitialWindowsImpl.cuh"
#include "LookingForwardConstants.cuh"
#include "LookingForwardTools.cuh"
#include "BinarySearch.cuh"

__device__ inline float linear_parameterization(const float value_at_ref, const float t, const float z)
{
  float dz = z - SciFi::Tracking::zReference;
  return value_at_ref + t * dz;
}

__device__ void lf_search_initial_windows_p_impl(
  const SciFi::Hits& scifi_hits,
  const SciFi::HitCount& scifi_hit_count,
  const MiniState& velo_state,
  const MiniState& UT_state,
  const SciFi::Tracking::Arrays* constArrays,
  const float magnet_polarity,
  const LookingForward::Constants* looking_forward_constants,
  const float qop,
  const int side,
  int* initial_windows,
  const int number_of_tracks)
{
  const auto xAtRef = xFromVelo(SciFi::Tracking::zReference, UT_state);

  // Find size of search window on reference plane, using Velo slopes and min pT as input
  const auto x_at_ref = LookingForward::propagate_x_from_velo(velo_state, qop, 5, looking_forward_constants);
  // find position within magnet where bending happens
  float zMag = zMagnet(velo_state, constArrays);

  const float slope2 = velo_state.tx * velo_state.tx + velo_state.ty * velo_state.ty;
  const float pt = std::sqrt(slope2 / (1.f + slope2)) / std::abs(qop);
  int iZoneStartingPoint = (side > 0) ? constArrays->zoneoffsetpar : 0;

  for (int i = threadIdx.y; i < LookingForward::number_of_x_layers; i += blockDim.y) {
    const auto iZone = iZoneStartingPoint + i;
    const float zZone = constArrays->xZone_zPos[iZone - iZoneStartingPoint];
    float xInZone_original = linear_parameterization(xAtRef, velo_state.tx, zZone);

    // const float xInZone = LookingForward::scifi_propagation(x_at_ref, velo_state.tx, qop, dz_x);
    // const float xInZone = LookingForward::propagate_x_from_velo(velo_state, qop,
    // looking_forward_constants->x_layers[iZone - iZoneStartingPoint], looking_forward_constants);
    const float xInZone = LookingForward::propagate_x_from_velo(
      velo_state, qop, looking_forward_constants->x_layers[iZone - iZoneStartingPoint], looking_forward_constants);

    // xInZone_original = xInZone;
    // const float xInZone = linear_parameterization(xAtRef, velo_state.tx, zZone);

    // const auto xInZone = LookingForward::propagate_x_from_velo(velo_state, qop,
    // looking_forward_constants->x_layers[iZone - iZoneStartingPoint], looking_forward_constants);

    const float yInZone =
      linear_parameterization(yFromVelo(SciFi::Tracking::zReference, UT_state), velo_state.ty, zZone);

    // if (side > 0) {
    //   if (
    //     !isInside(xInZone, SciFi::Tracking::xLim_Min, SciFi::Tracking::xLim_Max) ||
    //     !isInside(yInZone, SciFi::Tracking::yLim_Min, SciFi::Tracking::yLim_Max))
    //     continue;
    // }
    // else {
    //   if (
    //     !isInside(xInZone, SciFi::Tracking::xLim_Min, SciFi::Tracking::xLim_Max) ||
    //     !isInside(yInZone, side * SciFi::Tracking::yLim_Max, side * SciFi::Tracking::yLim_Min))
    //     continue;
    // }

    // extrapolate dxRef (x window on reference plane) to plane of current zone
    /*const float xTol = (zZone < SciFi::Tracking::zReference) ?
                         dxRef * zZone / SciFi::Tracking::zReference :
                         dxRef * (zZone - zMag) / (SciFi::Tracking::zReference - zMag);
*/
    const float xTol = LookingForward::dx_calc(velo_state.tx, qop);
    float xMin = xInZone - xTol;
    float xMax = xInZone + xTol;

    // Get the hits within the bounds
    const int x_zone_offset_begin = scifi_hit_count.zone_offset(constArrays->xZones[iZone]);
    const int x_zone_size = scifi_hit_count.zone_number_of_hits(constArrays->xZones[iZone]);
    int hits_within_bounds_start = binary_search_leftmost(scifi_hits.x0 + x_zone_offset_begin, x_zone_size, xMin);
    int hits_within_bounds_size = binary_search_leftmost(
      scifi_hits.x0 + x_zone_offset_begin + hits_within_bounds_start, x_zone_size - hits_within_bounds_start, xMax);

    const auto max_candidates_first_round = 96;
    const auto max_candidates_second_round = 64;

    while (hits_within_bounds_size > max_candidates_first_round) {
      auto x_diff_min = xInZone - xMin;
      auto x_diff_max = xMax - xInZone;

      // Reduce distance by a percentage
      xMin = xInZone - 0.7f * x_diff_min;
      xMax = xInZone + 0.7f * x_diff_max;

      hits_within_bounds_start = binary_search_leftmost(scifi_hits.x0 + x_zone_offset_begin, x_zone_size, xMin);
      hits_within_bounds_size = binary_search_leftmost(
        scifi_hits.x0 + x_zone_offset_begin + hits_within_bounds_start, x_zone_size - hits_within_bounds_start, xMax);
    }

    // Try to cut it not too much when it gets close to the window size
    while (hits_within_bounds_size > max_candidates_second_round) {
      auto x_diff_min = xInZone - xMin;
      auto x_diff_max = xMax - xInZone;

      // Reduce distance by a percentage
      xMin = xInZone - 0.9f * x_diff_min;
      xMax = xInZone + 0.9f * x_diff_max;

      hits_within_bounds_start = binary_search_leftmost(scifi_hits.x0 + x_zone_offset_begin, x_zone_size, xMin);
      hits_within_bounds_size = binary_search_leftmost(
        scifi_hits.x0 + x_zone_offset_begin + hits_within_bounds_start, x_zone_size - hits_within_bounds_start, xMax);
    }

    hits_within_bounds_start += x_zone_offset_begin;

    // Initialize windows
    initial_windows[i * 8 * number_of_tracks] = hits_within_bounds_start;
    initial_windows[(i * 8 + 1) * number_of_tracks] = hits_within_bounds_size;

    //     if (std::abs(1./qop - (-43687.8891)) < 1e-3){
    //     // printf("layer %i, wSign %i, p %f, x  %f, xTol %lf\n", looking_forward_constants->x_layers[iZone -
    //     iZoneStartingPoint], wSignTreatment, 1.f/qop, xInZone, xTol);
    //       printf("layer %i, wSign %i, p %f, x_min %lf, x_max %lf firt in win  %f window size %d\n",
    //              looking_forward_constants->x_layers[iZone - iZoneStartingPoint], wSignTreatment, 1.f/qop, xMin,
    //              xMax, scifi_hits.x0[hits_within_bounds_start], hits_within_bounds_size);
    // // printf("layer %i, wSign %i, p %f, x original: %f, x crap: %f, x diff: %f\n",
    // looking_forward_constants->x_layers[iZone - iZoneStartingPoint], wSignTreatment, 1.f/qop, xInZone_original,
    // xInZone, xInZone_original - xInZone);
    //     }

    // Skip making range but continue if the size is zero
    if (hits_within_bounds_size > 0) {
      // Now match the stereo hits
      const float this_uv_z = constArrays->uvZone_zPos[iZone - iZoneStartingPoint];
      const float xInUv = linear_parameterization(xAtRef, velo_state.tx, this_uv_z);
      // const float xInUv = LookingForward::propagate_x_from_velo(velo_state, qop, layeruv[iZone - iZoneStartingPoint],
      // looking_forward_constants);
      const float zRatio = (this_uv_z - zMag) / (zZone - zMag);
      const float dx = yInZone * constArrays->uvZone_dxdy[iZone - iZoneStartingPoint];
      const float xCentral = xInZone_original + dx;
      const float xPredUv = xInUv + (scifi_hits.x0[hits_within_bounds_start] - xInZone_original) * zRatio - dx;
      // const float xPredUv = xInUv - dx;
      const float maxDx = SciFi::Tracking::tolYCollectX +
                          (std::abs(scifi_hits.x0[hits_within_bounds_start] - xCentral) + std::abs(yInZone)) *
                            SciFi::Tracking::tolYSlopeCollectX;
      const float xMinUV = xPredUv - maxDx;
      const float xPredUVProto = xInUv - xInZone_original * zRatio - dx;
      const float maxDxProto = SciFi::Tracking::tolYCollectX + std::abs(yInZone) * SciFi::Tracking::tolYSlopeCollectX;

      // Get bounds in UV layers
      // do one search on the same side as the x module
      // if we are close to y = 0, also look within a region on the other side module ("triangle search")
      const int uv_zone_offset_begin = scifi_hit_count.zone_offset(constArrays->uvZones[iZone]);
      const int uv_zone_size = scifi_hit_count.zone_number_of_hits(constArrays->uvZones[iZone]);
      const int hits_within_uv_bounds =
        binary_search_leftmost(scifi_hits.x0 + uv_zone_offset_begin, uv_zone_size, xMinUV);

      initial_windows[(i * 8 + 2) * number_of_tracks] = hits_within_uv_bounds + uv_zone_offset_begin;
      initial_windows[(i * 8 + 3) * number_of_tracks] = uv_zone_size - hits_within_uv_bounds;

      float* initial_windows_f = (float*) &initial_windows[0];
      initial_windows_f[(i * 8 + 4) * number_of_tracks] = xPredUVProto;
      initial_windows_f[(i * 8 + 5) * number_of_tracks] = zRatio;
      initial_windows_f[(i * 8 + 6) * number_of_tracks] = maxDxProto;
      initial_windows_f[(i * 8 + 7) * number_of_tracks] = xCentral;
    }
  }
}

__device__ void lf_search_initial_windows_impl(
  const SciFi::Hits& scifi_hits,
  const SciFi::HitCount& scifi_hit_count,
  const float xAtRef,
  const float yAtRef,
  const MiniState& UT_state,
  const SciFi::Tracking::Arrays* constArrays,
  const float magnet_polarity,
  const float qop,
  const int side,
  int* initial_windows,
  const int number_of_tracks)
{
  // Find size of search window on reference plane, using Velo slopes and min pT as input
  const float tx2 = UT_state.tx * UT_state.tx;
  const float ty2 = UT_state.ty * UT_state.ty;
  const float slope2 = tx2 + ty2;
  const float dxRef =
    3973000.f * sqrtf(slope2) / SciFi::Tracking::minPt - 2200.f * ty2 - 1000.f * tx2; // tune this window
  // dxRef *= 0.9f;

  // find position within magnet where bending happens
  float zMag = zMagnet(UT_state, constArrays);

  const float q = qop > 0.f ? 1.f : -1.f;
  const float dir = q * magnet_polarity * (-1.f);

  const float pt = std::sqrt(slope2 / (1.f + slope2)) / std::abs(qop);
  const bool wSignTreatment = SciFi::Tracking::useWrongSignWindow && pt > SciFi::Tracking::wrongSignPT;

  float dxRefWS = 0.f;
  if (wSignTreatment) {
    // DvB: what happens if we use the actual momentum from VeloUT here instead of a constant?
    dxRefWS = 0.9f * calcDxRef(
                       SciFi::Tracking::wrongSignPT,
                       UT_state); // make windows a bit too small - FIXME check effect of this, seems wrong
  }

  int iZoneStartingPoint = side > 0 ? constArrays->zoneoffsetpar : 0;

  for (int i = threadIdx.y; i < LookingForward::number_of_x_layers; i += blockDim.y) {
    const auto iZone = iZoneStartingPoint + i;
    const float zZone = constArrays->xZone_zPos[iZone - iZoneStartingPoint];
    const float xInZone = linear_parameterization(xAtRef, UT_state.tx, zZone);
    const float yInZone = linear_parameterization(yAtRef, UT_state.ty, zZone);

    // extrapolate dxRef (x window on reference plane) to plane of current zone
    const float xTol = (zZone < SciFi::Tracking::zReference) ?
                         dxRef * zZone / SciFi::Tracking::zReference :
                         dxRef * (zZone - zMag) / (SciFi::Tracking::zReference - zMag);
    float xMin = xInZone - xTol;
    float xMax = xInZone + xTol;

    if (SciFi::Tracking::useMomentumEstimate) {
      float xTolWS = 0.0;
      if (wSignTreatment) {
        xTolWS = (zZone < SciFi::Tracking::zReference) ?
                   dxRefWS * zZone / SciFi::Tracking::zReference :
                   dxRefWS * (zZone - zMag) / (SciFi::Tracking::zReference - zMag);
      }
      if (dir > 0) {
        xMin = xInZone - xTolWS;
      }
      else {
        xMax = xInZone + xTolWS;
      }
    }

    // Get the hits within the bounds
    const int x_zone_offset_begin = scifi_hit_count.zone_offset(constArrays->xZones[iZone]);
    const int x_zone_size = scifi_hit_count.zone_number_of_hits(constArrays->xZones[iZone]);
    int hits_within_bounds_start = binary_search_leftmost(scifi_hits.x0 + x_zone_offset_begin, x_zone_size, xMin);
    int hits_within_bounds_size = binary_search_leftmost(
      scifi_hits.x0 + x_zone_offset_begin + hits_within_bounds_start, x_zone_size - hits_within_bounds_start, xMax);

    // If the number of hits within bounds is too high, reduce the window
    const auto max_candidates_first_round = 96;
    const auto max_candidates_second_round = 64;

    while (hits_within_bounds_size > max_candidates_first_round) {
      auto x_diff_min = xInZone - xMin;
      auto x_diff_max = xMax - xInZone;

      // Reduce distance by a percentage
      xMin = xInZone - 0.7f * x_diff_min;
      xMax = xInZone + 0.7f * x_diff_max;

      hits_within_bounds_start = binary_search_leftmost(scifi_hits.x0 + x_zone_offset_begin, x_zone_size, xMin);
      hits_within_bounds_size = binary_search_leftmost(
        scifi_hits.x0 + x_zone_offset_begin + hits_within_bounds_start, x_zone_size - hits_within_bounds_start, xMax);
    }

    // Try to cut it not too much when it gets close to the window size
    while (hits_within_bounds_size > max_candidates_second_round) {
      auto x_diff_min = xInZone - xMin;
      auto x_diff_max = xMax - xInZone;

      // Reduce distance by a percentage
      xMin = xInZone - 0.9f * x_diff_min;
      xMax = xInZone + 0.9f * x_diff_max;

      hits_within_bounds_start = binary_search_leftmost(scifi_hits.x0 + x_zone_offset_begin, x_zone_size, xMin);
      hits_within_bounds_size = binary_search_leftmost(
        scifi_hits.x0 + x_zone_offset_begin + hits_within_bounds_start, x_zone_size - hits_within_bounds_start, xMax);
    }

    hits_within_bounds_start += x_zone_offset_begin;

    // Initialize windows
    initial_windows[i * 8 * number_of_tracks] = hits_within_bounds_start;
    initial_windows[(i * 8 + 1) * number_of_tracks] = hits_within_bounds_size;

    // Skip making range but continue if the size is zero
    if (hits_within_bounds_size > 0) {
      // Now match the stereo hits
      const float this_uv_z = constArrays->uvZone_zPos[iZone - iZoneStartingPoint];
      const float xInUv = linear_parameterization(xAtRef, UT_state.tx, this_uv_z);
      const float zRatio = (this_uv_z - zMag) / (zZone - zMag);
      const float dx = yInZone * constArrays->uvZone_dxdy[iZone - iZoneStartingPoint];
      const float xCentral = xInZone + dx;
      const float xPredUv = xInUv + (scifi_hits.x0[hits_within_bounds_start] - xInZone) * zRatio - dx;
      const float maxDx = SciFi::Tracking::tolYCollectX +
                          (std::abs(scifi_hits.x0[hits_within_bounds_start] - xCentral) + std::abs(yInZone)) *
                            SciFi::Tracking::tolYSlopeCollectX;
      const float xMinUV = xPredUv - maxDx;
      const float xPredUVProto = xInUv - xInZone * zRatio - dx;
      const float maxDxProto = SciFi::Tracking::tolYCollectX + std::abs(yInZone) * SciFi::Tracking::tolYSlopeCollectX;

      // Get bounds in UV layers
      // do one search on the same side as the x module
      // if we are close to y = 0, also look within a region on the other side module ("triangle search")
      const int uv_zone_offset_begin = scifi_hit_count.zone_offset(constArrays->uvZones[iZone]);
      const int uv_zone_size = scifi_hit_count.zone_number_of_hits(constArrays->uvZones[iZone]);
      const int hits_within_uv_bounds =
        binary_search_leftmost(scifi_hits.x0 + uv_zone_offset_begin, uv_zone_size, xMinUV);

      initial_windows[(i * 8 + 2) * number_of_tracks] = hits_within_uv_bounds + uv_zone_offset_begin;
      initial_windows[(i * 8 + 3) * number_of_tracks] = uv_zone_size - hits_within_uv_bounds;

      float* initial_windows_f = (float*) &initial_windows[0];
      initial_windows_f[(i * 8 + 4) * number_of_tracks] = xPredUVProto;
      initial_windows_f[(i * 8 + 5) * number_of_tracks] = zRatio;
      initial_windows_f[(i * 8 + 6) * number_of_tracks] = maxDxProto;
      initial_windows_f[(i * 8 + 7) * number_of_tracks] = xCentral;
    }
  }
}
