#include "hip/hip_runtime.h"
#include "SciFiCopyTrackHitNumber.cuh"

void scifi_copy_track_hit_number_t::set_arguments_size(
  ArgumentRefManager<Arguments> arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers) const
{
  arguments.set_size<dev_scifi_track_hit_number>(host_buffers.scifi_track_hit_number_size());
}

void scifi_copy_track_hit_number_t::operator()(
  const ArgumentRefManager<Arguments>& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event) const
{
  function(dim3(host_buffers.host_number_of_selected_events[0]), block_dimension(), hip_stream)(
    offset<dev_atomics_ut_t>(arguments),
    offset<dev_scifi_tracks_t>(arguments),
    offset<dev_atomics_scifi_t>(arguments),
    offset<dev_scifi_track_hit_number_t>(arguments));
}

/**
 * @brief Copies UT track hit numbers on a consecutive container
 */
__global__ void scifi_copy_track_hit_number(
  const uint* dev_atomics_ut,
  const SciFi::TrackHits* dev_scifi_tracks,
  uint* dev_n_scifi_tracks,
  uint* dev_scifi_track_hit_number)
{
  const auto number_of_events = gridDim.x;
  const auto event_number = blockIdx.x;
  const auto ut_event_tracks_offset = dev_atomics_ut[number_of_events + event_number];

  const auto* event_tracks =
    dev_scifi_tracks + ut_event_tracks_offset * SciFi::Constants::max_SciFi_tracks_per_UT_track;
  // const SciFi::TrackHits* event_tracks =
  //   dev_scifi_tracks + ut_event_tracks_offset *
  //   LookingForward::maximum_number_of_candidates_per_ut_track_after_x_filter;
  const auto accumulated_tracks = dev_n_scifi_tracks[number_of_events + event_number];
  const auto number_of_tracks = dev_n_scifi_tracks[event_number];

  // Pointer to scifi_track_hit_number of current event.
  uint* scifi_track_hit_number = dev_scifi_track_hit_number + accumulated_tracks;

  // Loop over tracks.
  for (uint element = threadIdx.x; element < number_of_tracks; ++element) {
    scifi_track_hit_number[element] = event_tracks[element].hitsNum;
  }
}
