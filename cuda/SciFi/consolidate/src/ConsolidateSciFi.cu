#include "hip/hip_runtime.h"
#include "ConsolidateSciFi.cuh"

template<typename F>
__device__ void populate(const SciFi::TrackHits& track, const F& assign)
{
  for (int i = 0; i < track.hitsNum; i++) {
    const auto hit_index = track.hits[i];
    assign(i, hit_index);
  }
};

__global__ void scifi_consolidate_tracks::scifi_consolidate_tracks(scifi_consolidate_tracks::Parameters parameters)
{
  const uint number_of_events = gridDim.x;
  const uint event_number = blockIdx.x;

  const uint ut_event_tracks_offset = parameters.dev_atomics_ut[number_of_events + event_number];
  const auto ut_total_number_of_tracks = parameters.dev_atomics_ut[2 * number_of_events];

  // const SciFi::TrackHits* event_scifi_tracks =
  //   parameters.dev_scifi_tracks + ut_event_tracks_offset *
  //   LookingForward::maximum_number_of_candidates_per_ut_track_after_x_filter;
  const SciFi::TrackHits* event_scifi_tracks =
    parameters.dev_scifi_tracks + ut_event_tracks_offset * SciFi::Constants::max_SciFi_tracks_per_UT_track;

  const uint total_number_of_scifi_hits =
    parameters.dev_scifi_hit_count[number_of_events * SciFi::Constants::n_mat_groups_and_mats];

  SciFi::ConstHits scifi_hits {parameters.dev_scifi_hits, total_number_of_scifi_hits};
  SciFi::ConstHitCount scifi_hit_count {parameters.dev_scifi_hit_count, event_number};

  // Create consolidated SoAs.
  SciFi::Consolidated::Tracks scifi_tracks {parameters.dev_atomics_scifi,
                                            parameters.dev_scifi_track_hit_number,
                                            parameters.dev_scifi_qop,
                                            parameters.dev_scifi_states,
                                            parameters.dev_scifi_track_ut_indices,
                                            event_number,
                                            number_of_events};
  const uint number_of_tracks_event = scifi_tracks.number_of_tracks(event_number);
  const uint event_offset = scifi_hit_count.event_offset();

  // Loop over tracks.
  for (uint i = threadIdx.x; i < number_of_tracks_event; i += blockDim.x) {
    scifi_tracks.ut_track(i) = event_scifi_tracks[i].ut_track_index;
    scifi_tracks.qop(i) = event_scifi_tracks[i].qop;
    const auto scifi_track_index = ut_event_tracks_offset * SciFi::Constants::max_SciFi_tracks_per_UT_track + i;

    const auto curvature = parameters.dev_scifi_lf_parametrization_consolidate[scifi_track_index];
    const auto tx = parameters.dev_scifi_lf_parametrization_consolidate
                      [ut_total_number_of_tracks * SciFi::Constants::max_SciFi_tracks_per_UT_track + scifi_track_index];
    const auto x0 =
      parameters.dev_scifi_lf_parametrization_consolidate
        [2 * ut_total_number_of_tracks * SciFi::Constants::max_SciFi_tracks_per_UT_track + scifi_track_index];
    const auto d_ratio =
      parameters.dev_scifi_lf_parametrization_consolidate
        [3 * ut_total_number_of_tracks * SciFi::Constants::max_SciFi_tracks_per_UT_track + scifi_track_index];
    const auto y0 =
      parameters.dev_scifi_lf_parametrization_consolidate
        [4 * ut_total_number_of_tracks * SciFi::Constants::max_SciFi_tracks_per_UT_track + scifi_track_index];
    const auto ty =
      parameters.dev_scifi_lf_parametrization_consolidate
        [5 * ut_total_number_of_tracks * SciFi::Constants::max_SciFi_tracks_per_UT_track + scifi_track_index];

    const auto dz = SciFi::Constants::ZEndT - LookingForward::z_mid_t;
    const MiniState scifi_state {x0 + tx * dz + curvature * dz * dz * (1.f + d_ratio * dz),
                                 y0 + ty * SciFi::Constants::ZEndT,
                                 SciFi::Constants::ZEndT,
                                 tx + 2.f * dz * curvature + 3.f * dz * dz * curvature * d_ratio,
                                 ty};

    scifi_tracks.states(i) = scifi_state;

    auto consolidated_hits = scifi_tracks.get_hits(parameters.dev_scifi_track_hits, i);
    const SciFi::TrackHits& track = event_scifi_tracks[i];

    // Populate arrays
    populate(track, [&consolidated_hits, &scifi_hits, &event_offset](const uint i, const uint hit_index) {
      consolidated_hits.x0(i) = scifi_hits.x0(event_offset + hit_index);
    });

    populate(track, [&consolidated_hits, &scifi_hits, &event_offset](const uint i, const uint hit_index) {
      consolidated_hits.z0(i) = scifi_hits.z0(event_offset + hit_index);
    });

    populate(track, [&consolidated_hits, &scifi_hits, &event_offset](const uint i, const uint hit_index) {
      consolidated_hits.endPointY(i) = scifi_hits.endPointY(event_offset + hit_index);
    });

    populate(track, [&consolidated_hits, &scifi_hits, &event_offset](const uint i, const uint hit_index) {
      consolidated_hits.channel(i) = scifi_hits.channel(event_offset + hit_index);
    });

    populate(track, [&consolidated_hits, &scifi_hits, &event_offset](const uint i, const uint hit_index) {
      consolidated_hits.assembled_datatype(i) = scifi_hits.assembled_datatype(event_offset + hit_index);
    });
  }
}
