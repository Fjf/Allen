#include "hip/hip_runtime.h"
#include "SciFiDirectDecoderV4.cuh"
#include "assert.h"

using namespace SciFi;

__global__ void scifi_direct_decoder_v4(
  char *scifi_events,
  uint *scifi_event_offsets,
  uint *scifi_hit_count,
  uint *scifi_hits,
  char* scifi_geometry,
  const float* dev_inv_clus_res)
{
  const uint number_of_events = gridDim.x;
  const uint event_number = blockIdx.x;
  const SciFiGeometry geom(scifi_geometry);
  const auto event = SciFiRawEvent(scifi_events + scifi_event_offsets[event_number]);

  SciFi::Hits hits {scifi_hits, scifi_hit_count[number_of_events * SciFi::Constants::n_mats], &geom, dev_inv_clus_res};
  SciFi::HitCount hit_count;
  hit_count.typecast_after_prefix_sum(scifi_hit_count, event_number, number_of_events);

  for (uint i_raw_bank = threadIdx.x; i_raw_bank < SciFi::Constants::n_consecutive_raw_banks; i_raw_bank += blockDim.x) {
    const uint raw_bank_offset = hit_count.mat_group_offset(i_raw_bank);

    const uint j = (i_raw_bank / 10) % 4;
    const bool reverse_cluster_order = j == 1 | j == 2;

    const uint k = i_raw_bank % 10;
    const bool reverse_raw_bank_order = k < 5;
    const uint current_raw_bank = reverse_raw_bank_order ?
      5 * (i_raw_bank / 5) + (4 - i_raw_bank % 5) :
      i_raw_bank;

    const auto rawbank = event.getSciFiRawBank(current_raw_bank);
    uint16_t* it = rawbank.data + 2;
    uint16_t* last = rawbank.last;

    if (*(last - 1) == 0) --last; // Remove padding at the end
    const uint number_of_clusters = last - it;

    for (int i_cluster = threadIdx.y; i_cluster < number_of_clusters; i_cluster += blockDim.y) {
      const uint16_t current_cluster = reverse_cluster_order ? (number_of_clusters - 1 - i_cluster) : i_cluster;

      uint16_t c = *(it + current_cluster);
      uint8_t cluster_fraction = fraction(c);
      uint32_t ch = geom.bank_first_channel[rawbank.sourceID] + channelInBank(c);
      const SciFi::SciFiChannelID id {ch};

      // Offset to save space in geometry structure, see DumpFTGeometry.cpp
      const uint32_t mat = id.uniqueMat() - 512;
      const uint32_t planeCode = id.uniqueLayer() - 4;
      const float dxdy = geom.dxdy[mat];
      const float dzdy = geom.dzdy[mat];
      float uFromChannel = geom.uBegin[mat] + (2 * id.channel() + 1 + cluster_fraction) * geom.halfChannelPitch[mat];
      if (id.die()) uFromChannel += geom.dieGap[mat];
      uFromChannel += id.sipm() * geom.sipmPitch[mat];
      const float endPointX = geom.mirrorPointX[mat] + geom.ddxX[mat] * uFromChannel;
      const float endPointY = geom.mirrorPointY[mat] + geom.ddxY[mat] * uFromChannel;
      const float endPointZ = geom.mirrorPointZ[mat] + geom.ddxZ[mat] * uFromChannel;
      const float x0 = endPointX - dxdy * endPointY;
      const float z0 = endPointZ - dzdy * endPointY;

      assert(pseudoSize < 9 && "Pseudosize of cluster is > 8. Out of range.");

      // Apparently the unique* methods are not designed to start at 0, therefore -16
      const uint32_t uniqueZone = ((id.uniqueQuarter() - 16) >> 1);
      const uint plane_code = 2 * planeCode + (uniqueZone % 2);
      const uint hit_index = raw_bank_offset + current_cluster;
      const uint8_t pseudoSize = cSize(c) ? 0 : 4;

      hits.x0[hit_index] = x0;
      hits.z0[hit_index] = z0;
      hits.channel[hit_index] = ch;
      hits.m_endPointY[hit_index] = endPointY;
      assert(fraction <= 0x1 && plane_code <= 0x1f && pseudoSize <= 0xf && mat <= 0x7ff);
      hits.assembled_datatype[hit_index] = cluster_fraction << 20 | plane_code << 15 | pseudoSize << 11 | mat;
    }
  }
}
