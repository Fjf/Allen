#include "hip/hip_runtime.h"
#include <MEPTools.h>
#include <SciFiRawBankDecoderV4.cuh>
#include <assert.h>

using namespace SciFi;

// Merge of PrStoreFTHit and RawBankDecoder.
__device__ void make_cluster_v4(
  const int hit_index,
  const SciFiGeometry& geom,
  uint32_t chan,
  uint8_t fraction,
  uint8_t pseudoSize,
  SciFi::Hits& hits)
{
  const SciFi::SciFiChannelID id {chan};

  // Offset to save space in geometry structure, see DumpFTGeometry.cpp
  const uint32_t mat = id.uniqueMat() - 512;
  const uint32_t planeCode = id.uniqueLayer() - 4;
  const float dxdy = geom.dxdy[mat];
  const float dzdy = geom.dzdy[mat];
  float uFromChannel = geom.uBegin[mat] + (2 * id.channel() + 1 + fraction) * geom.halfChannelPitch[mat];
  if (id.die()) uFromChannel += geom.dieGap[mat];
  uFromChannel += id.sipm() * geom.sipmPitch[mat];
  const float endPointX = geom.mirrorPointX[mat] + geom.ddxX[mat] * uFromChannel;
  const float endPointY = geom.mirrorPointY[mat] + geom.ddxY[mat] * uFromChannel;
  const float endPointZ = geom.mirrorPointZ[mat] + geom.ddxZ[mat] * uFromChannel;
  const float x0 = endPointX - dxdy * endPointY;
  const float z0 = endPointZ - dzdy * endPointY;

  assert(pseudoSize < 9 && "Pseudosize of cluster is > 8. Out of range.");

  // Apparently the unique* methods are not designed to start at 0, therefore -16
  const uint32_t uniqueZone = ((id.uniqueQuarter() - 16) >> 1);

  const uint plane_code = 2 * planeCode + (uniqueZone % 2);
  hits.x0(hit_index) = x0;
  hits.z0(hit_index) = z0;
  hits.channel(hit_index) = chan;
  hits.endPointY(hit_index) = endPointY;
  assert(fraction <= 0x1 && plane_code <= 0x1f && pseudoSize <= 0xf && mat <= 0x7ff);
  hits.assembled_datatype(hit_index) = fraction << 20 | plane_code << 15 | pseudoSize << 11 | mat;
}

__global__ void scifi_raw_bank_decoder_v4::scifi_raw_bank_decoder_v4(
  scifi_raw_bank_decoder_v4::Parameters parameters,
  const char* scifi_geometry)
{
  const uint number_of_events = gridDim.x;
  const uint event_number = blockIdx.x;
  const uint selected_event_number = parameters.dev_event_list[event_number];

  const SciFiGeometry geom {scifi_geometry};
  const auto event =
    SciFiRawEvent(parameters.dev_scifi_raw_input + parameters.dev_scifi_raw_input_offsets[selected_event_number]);

  SciFi::Hits hits {parameters.dev_scifi_hits,
                    parameters.dev_scifi_hit_count[number_of_events * SciFi::Constants::n_mat_groups_and_mats]};
  SciFi::ConstHitCount hit_count {parameters.dev_scifi_hit_count, event_number};
  const uint number_of_hits_in_last_zones = hit_count.number_of_hits_in_zones_without_mat_groups();

  for (uint i = threadIdx.x; i < number_of_hits_in_last_zones; i += blockDim.x) {
    const uint32_t cluster_reference = hits.cluster_reference(hit_count.offset_zones_without_mat_groups() + i);
    const int raw_bank_number = (cluster_reference >> 8) & 0xFF;
    const int it_number = (cluster_reference) &0xFF;
    const auto rawbank = event.getSciFiRawBank(raw_bank_number);
    const uint16_t* it = rawbank.data + 2;
    it += it_number;

    const uint16_t c = *it;
    const uint32_t ch = geom.bank_first_channel[rawbank.sourceID] + channelInBank(c);

    // Call parameters for make_cluster
    uint32_t cluster_chan = ch;
    uint8_t cluster_fraction = fraction(c);
    uint8_t pseudoSize = cSize(c) ? 0 : 4;

    make_cluster_v4(
      hit_count.offset_zones_without_mat_groups() + i, geom, cluster_chan, cluster_fraction, pseudoSize, hits);
  }
}

__global__ void scifi_raw_bank_decoder_v4::scifi_raw_bank_decoder_v4_mep(
  scifi_raw_bank_decoder_v4::Parameters parameters,
  const char* scifi_geometry)
{
  const uint number_of_events = gridDim.x;
  const uint event_number = blockIdx.x;
  const uint selected_event_number = parameters.dev_event_list[event_number];

  const SciFiGeometry geom {scifi_geometry};

  SciFi::Hits hits {parameters.dev_scifi_hits,
                    parameters.dev_scifi_hit_count[number_of_events * SciFi::Constants::n_mat_groups_and_mats]};
  SciFi::ConstHitCount hit_count {parameters.dev_scifi_hit_count, event_number};
  const uint number_of_hits_in_last_zones = hit_count.number_of_hits_in_zones_without_mat_groups();

  for (uint i = threadIdx.x; i < number_of_hits_in_last_zones; i += blockDim.x) {
    const uint32_t cluster_reference = hits.cluster_reference(hit_count.offset_zones_without_mat_groups() + i);

    const int raw_bank_number = (cluster_reference >> 8) & 0xFF;
    const int it_number = (cluster_reference) &0xFF;

    // Create SciFi raw bank from MEP layout
    auto const raw_bank = MEP::raw_bank<SciFiRawBank>(
      parameters.dev_scifi_raw_input, parameters.dev_scifi_raw_input_offsets, selected_event_number, raw_bank_number);

    const uint16_t* it = raw_bank.data + 2;
    it += it_number;

    const uint16_t c = *it;
    const uint32_t ch = geom.bank_first_channel[raw_bank.sourceID] + channelInBank(c);

    // Call parameters for make_cluster
    uint32_t cluster_chan = ch;
    uint8_t cluster_fraction = fraction(c);
    uint8_t pseudoSize = cSize(c) ? 0 : 4;

    make_cluster_v4(
      hit_count.offset_zones_without_mat_groups() + i, geom, cluster_chan, cluster_fraction, pseudoSize, hits);
  }
}
