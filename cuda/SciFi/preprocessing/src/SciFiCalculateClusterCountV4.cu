#include "hip/hip_runtime.h"
#include "SciFiCalculateClusterCountV4.cuh"
#include "Invoke.cuh"

void scifi_calculate_cluster_count_v4_t::invoke() {
  invoke_helper(handler);
}

using namespace SciFi;

/**
 * @brief This function calculates the amount of clusters in all mats.
 * @details More details about the SciFi format:
 *          https://cds.cern.ch/record/2630154/files/LHCb-INT-2018-024.pdf
 */
__global__ void scifi_calculate_cluster_count_v4(
  char* scifi_raw_input,
  uint* scifi_raw_input_offsets,
  uint* scifi_hit_count,
  const uint* event_list,
  char* scifi_geometry)
{
  const uint event_number = blockIdx.x;
  const uint selected_event_number = event_list[event_number];

  const SciFiRawEvent event(scifi_raw_input + scifi_raw_input_offsets[selected_event_number]);
  const SciFiGeometry geom(scifi_geometry);
  SciFi::HitCount hit_count {scifi_hit_count, event_number};

  for (uint i = threadIdx.x; i < SciFi::Constants::n_consecutive_raw_banks; i += blockDim.x) {
    const uint current_raw_bank = getRawBankIndexOrderedByX(i);
    const auto rawbank = event.getSciFiRawBank(current_raw_bank);
    uint16_t* it = rawbank.data + 2;
    uint16_t* last = rawbank.last;

    if (*(last - 1) == 0) --last; // Remove padding at the end
    const uint number_of_clusters = last - it;

    if (last > it) {
      hit_count.mat_offsets[i] = number_of_clusters;
    }
  }

  const uint mats_difference = 3 * SciFi::Constants::n_consecutive_raw_banks;
  for (uint i = SciFi::Constants::n_consecutive_raw_banks + threadIdx.x; i < event.number_of_raw_banks;
       i += blockDim.x) {
    uint32_t* hits_mat;
    const auto rawbank = event.getSciFiRawBank(i);
    uint16_t* it = rawbank.data + 2;
    uint16_t* last = rawbank.last;

    // For details see RawBankDecoder
    if (*(last - 1) == 0) --last; // Remove phadding at the end
    for (; it < last; ++it) {     // loop over the clusters
      uint16_t c = *it;
      uint32_t ch = geom.bank_first_channel[rawbank.sourceID] + channelInBank(c);
      hits_mat = hit_count.mat_offsets + SciFiChannelID(ch).correctedUniqueMat() - mats_difference;
      atomicAdd(hits_mat, 1);
    }
  }
}
