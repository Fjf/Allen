#include "hip/hip_runtime.h"
#include "LFSearchInitialWindows.cuh"
#include "LFSearchInitialWindowsImpl.cuh"

void lf_search_initial_windows_t::set_arguments_size(
  ArgumentRefManager<Arguments> arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  const HostBuffers& host_buffers) const
{
  arguments.set_size<dev_scifi_lf_initial_windows>(
    LookingForward::number_of_elements_initial_window * host_buffers.host_number_of_reconstructed_ut_tracks[0] *
    LookingForward::number_of_x_layers);
  arguments.set_size<dev_ut_states>(host_buffers.host_number_of_reconstructed_ut_tracks[0]);
  arguments.set_size<dev_scifi_lf_process_track>(host_buffers.host_number_of_reconstructed_ut_tracks[0]);
}

void lf_search_initial_windows_t::operator()(
  const ArgumentRefManager<Arguments>& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event) const
{
  cudaCheck(hipMemsetAsync(
    arguments.offset<dev_scifi_lf_initial_windows>(), 0, arguments.size<dev_scifi_lf_initial_windows>(), hip_stream));

  function.invoke(dim3(host_buffers.host_number_of_selected_events[0]), block_dimension(), hip_stream)(
    arguments.offset<dev_scifi_hits>(),
    arguments.offset<dev_scifi_hit_count>(),
    arguments.offset<dev_atomics_velo>(),
    arguments.offset<dev_velo_track_hit_number>(),
    arguments.offset<dev_velo_states>(),
    arguments.offset<dev_atomics_ut>(),
    arguments.offset<dev_ut_track_hit_number>(),
    arguments.offset<dev_ut_x>(),
    arguments.offset<dev_ut_tx>(),
    arguments.offset<dev_ut_z>(),
    arguments.offset<dev_ut_qop>(),
    arguments.offset<dev_ut_track_velo_indices>(),
    constants.dev_scifi_geometry,
    constants.dev_inv_clus_res,
    constants.dev_looking_forward_constants,
    arguments.offset<dev_scifi_lf_initial_windows>(),
    arguments.offset<dev_ut_states>(),
    arguments.offset<dev_scifi_lf_process_track>());
}

__global__ void lf_search_initial_windows(
  uint32_t* dev_scifi_hits,
  const uint32_t* dev_scifi_hit_count,
  const uint* dev_atomics_velo,
  const uint* dev_velo_track_hit_number,
  const char* dev_velo_states,
  const uint* dev_atomics_ut,
  const uint* dev_ut_track_hit_number,
  const float* dev_ut_x,
  const float* dev_ut_tx,
  const float* dev_ut_z,
  const float* dev_ut_qop,
  const uint* dev_ut_track_velo_indices,
  const char* dev_scifi_geometry,
  const float* dev_inv_clus_res,
  const LookingForward::Constants* dev_looking_forward_constants,
  int* dev_initial_windows,
  MiniState* dev_ut_states,
  bool* dev_scifi_lf_process_track)
{
  const uint number_of_events = gridDim.x;
  const uint event_number = blockIdx.x;

  // Velo consolidated types
  const Velo::Consolidated::Tracks velo_tracks {
    (uint*) dev_atomics_velo, (uint*) dev_velo_track_hit_number, event_number, number_of_events};
  const Velo::Consolidated::States velo_states {(char*) dev_velo_states, velo_tracks.total_number_of_tracks};
  const uint velo_event_tracks_offset = velo_tracks.tracks_offset(event_number);

  // UT consolidated tracks
  UT::Consolidated::Tracks ut_tracks {(uint*) dev_atomics_ut,
                                      (uint*) dev_ut_track_hit_number,
                                      (float*) dev_ut_qop,
                                      (uint*) dev_ut_track_velo_indices,
                                      event_number,
                                      number_of_events};

  const int ut_event_number_of_tracks = ut_tracks.number_of_tracks(event_number);
  const int ut_event_tracks_offset = ut_tracks.tracks_offset(event_number);

  // SciFi hits
  const uint total_number_of_hits = dev_scifi_hit_count[number_of_events * SciFi::Constants::n_mat_groups_and_mats];
  const SciFi::HitCount scifi_hit_count {(uint32_t*) dev_scifi_hit_count, event_number};
  const SciFi::SciFiGeometry scifi_geometry {dev_scifi_geometry};
  const SciFi::Hits scifi_hits(dev_scifi_hits, total_number_of_hits, &scifi_geometry, dev_inv_clus_res);
  const auto event_offset = scifi_hit_count.event_offset();

  MiniState* ut_states = dev_ut_states + ut_event_tracks_offset;

  for (int i = threadIdx.x; i < ut_event_number_of_tracks; i += blockDim.x) {
    const int velo_track_index = ut_tracks.velo_track[i];
    const int ut_track_index = ut_event_tracks_offset + i;
    const float ut_qop = ut_tracks.qop[i];

    // Note: These data should be accessed like
    //       the previous ut_tracks.qop[i] in the future
    const float ut_x = dev_ut_x[ut_track_index];
    const float ut_tx = dev_ut_tx[ut_track_index];
    const float ut_z = dev_ut_z[ut_track_index];

    const uint velo_states_index = velo_event_tracks_offset + velo_track_index;
    const MiniState velo_state = velo_states.getMiniState(velo_states_index);

    // extrapolate velo y & ty to z of UT x and tx
    // use ty from Velo state
    const MiniState ut_state {ut_x, LookingForward::y_at_z(velo_state, ut_z), ut_z, ut_tx, velo_state.ty};
    const MiniState state_at_z_last_ut_plane = LookingForward::state_at_z(ut_state, LookingForward::z_last_UT_plane);

    // Store state for access in other algorithms
    ut_states[i] = state_at_z_last_ut_plane;

    // Parameters for the calculation of the windows
    const float y_projection =
      LookingForward::y_at_z_dzdy_corrected(state_at_z_last_ut_plane, dev_looking_forward_constants->Zone_zPos_xlayers[0]);

    lf_search_initial_windows_impl(
      scifi_hits,
      scifi_hit_count,
      state_at_z_last_ut_plane,
      dev_looking_forward_constants,
      ut_qop,
      y_projection >= 0.f,
      dev_initial_windows + ut_event_tracks_offset + i,
      ut_tracks.total_number_of_tracks,
      event_offset,
      dev_scifi_lf_process_track,
      ut_track_index);
  }
}
