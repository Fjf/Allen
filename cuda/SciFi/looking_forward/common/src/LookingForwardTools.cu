#include "hip/hip_runtime.h"
#include "LookingForwardTools.cuh"
#include "BinarySearch.cuh"

// straight line extrapolation of MiniState to other z position
__device__ MiniState LookingForward::state_at_z(const MiniState& state, const float z)
{
  return {state.x + (z - state.z) * state.tx, state.y + (z - state.z) * state.ty, z, state.tx, state.ty};
}

__device__ float LookingForward::x_at_z(const MiniState& state, const float z)
{
  float xf = state.x + (z - state.z) * state.tx;
  return xf;
}

// straight line extrapolation of y to other z position
__device__ float LookingForward::y_at_z(const MiniState& state, const float z)
{
  return state.y + (z - state.z) * state.ty;
}

__device__ float LookingForward::linear_propagation(float x_0, float tx, float dz) { return x_0 + tx * dz; }

__device__ MiniState LookingForward::propagate_state_from_velo(
  const MiniState& UT_state,
  float qop,
  int layer,
  const LookingForward::Constants* dev_looking_forward_constants)
{
  // center of the magnet
  const MiniState magnet_state = state_at_z(UT_state, dev_looking_forward_constants->zMagnetParams[0]);

  MiniState final_state = magnet_state;
  final_state.tx = dev_looking_forward_constants->ds_p_param[layer] * qop + UT_state.tx;

  const auto dp_x_mag =
    (qop > 0) ? dev_looking_forward_constants->dp_x_mag_plus : dev_looking_forward_constants->dp_x_mag_minus;
  const auto dp_y_mag =
    (qop > 0) ? dev_looking_forward_constants->dp_y_mag_plus : dev_looking_forward_constants->dp_y_mag_minus;

  const float x_mag_correction = dp_x_mag[layer][0] + magnet_state.x * dp_x_mag[layer][1] +
                                 magnet_state.x * magnet_state.x * dp_x_mag[layer][2] +
                                 magnet_state.x * magnet_state.x * magnet_state.x * dp_x_mag[layer][3] +
                                 magnet_state.x * magnet_state.x * magnet_state.x * magnet_state.x * dp_x_mag[layer][4];

  const float y_mag_correction =
    dp_y_mag[layer][0] + magnet_state.y * dp_y_mag[layer][1] + magnet_state.y * magnet_state.y * dp_y_mag[layer][2];

  final_state = state_at_z(final_state, dev_looking_forward_constants->Zone_zPos[layer]);
  final_state.x += -y_mag_correction - x_mag_correction;

  return final_state;
}

__device__ float LookingForward::dx_calc(const float state_tx, float qop)
{
  float ret_val;
  float qop_window = std::abs(LookingForward::dx_slope * qop + LookingForward::dx_min);
  float tx_window = std::abs(LookingForward::tx_slope * state_tx + LookingForward::tx_min);
  ret_val = LookingForward::tx_weight * tx_window + LookingForward::dx_weight * qop_window;
  if (ret_val > LookingForward::max_window_layer0) {
    ret_val = LookingForward::max_window_layer0;
  }
  return ret_val;
}

__device__ std::tuple<int, int> LookingForward::find_x_in_window(
  const SciFi::Hits& hits,
  const int zone_offset,
  const int num_hits,
  const float x_min,
  const float x_max)
{
  int first_candidate = binary_search_leftmost(hits.x0 + zone_offset, num_hits, x_min);
  int last_candidate = -1;

  if (first_candidate != -1) {
    last_candidate = binary_search_leftmost(hits.x0 + zone_offset + first_candidate, num_hits - first_candidate, x_max);

    first_candidate = zone_offset + first_candidate;
    last_candidate = last_candidate != -1 ? first_candidate + last_candidate + 1 : -1;
  }

  return {first_candidate, last_candidate};
}

__device__ std::tuple<int, int> LookingForward::get_offset_and_n_hits_for_layer(
  const int first_zone,
  const SciFi::HitCount& scifi_hit_count,
  const float y)
{
  assert(first_zone < SciFi::Constants::n_zones - 1);
  const auto offset = (y < 0) ? 0 : 1;

  return {scifi_hit_count.zone_offset(first_zone + offset), scifi_hit_count.zone_number_of_hits(first_zone + offset)};
}

__device__ std::tuple<int, float> LookingForward::get_best_hit(
  const SciFi::Hits& hits,
  const SciFi::HitCount& hit_count,
  const float m,
  const std::tuple<int, int>& layer_candidates,
  const std::tuple<float, float>& hit_layer_0_z_x,
  const std::tuple<float, float>& hit_layer_3_z_x,
  const float layer_projected_state_z,
  const float layer_projected_state_y,
  const int layer,
  const LookingForward::Constants* dev_looking_forward_constants)
{
  const auto q = std::get<1>(hit_layer_0_z_x) - std::get<0>(hit_layer_0_z_x) * m;
  const auto x_adjustment = layer_projected_state_y * dev_looking_forward_constants->Zone_dxdy[layer];

  int best_index = -1;
  float min_chi2 = LookingForward::chi2_cut;
  for (int i = 0; i < std::get<1>(layer_candidates); i++) {
    const auto hit_index = hit_count.event_offset() + std::get<0>(layer_candidates) + i;
    const auto chi_2 = chi2(
      m,
      q,
      hit_layer_0_z_x,
      std::make_tuple(layer_projected_state_z, hits.x0[hit_index] + x_adjustment),
      hit_layer_3_z_x);

    if (chi_2 < min_chi2) {
      best_index = hit_index;
      min_chi2 = chi_2;
    }
  }

  return {best_index, min_chi2};
}
