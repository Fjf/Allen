#include "hip/hip_runtime.h"
#include "LFCalculateFirstLayerWindow.cuh"

__global__ void lf_calculate_first_layer_window(
  uint32_t* dev_scifi_hits,
  const uint32_t* dev_scifi_hit_count,
  const int* dev_atomics_velo,
  const uint* dev_velo_track_hit_number,
  const char* dev_velo_states,
  const int* dev_atomics_ut,
  const char* dev_ut_track_hits,
  const uint* dev_ut_track_hit_number,
  const float* dev_ut_x,
  const float* dev_ut_tx,
  const float* dev_ut_z,
  const float* dev_ut_qop,
  const uint* dev_ut_track_velo_indices,
  const char* dev_scifi_geometry,
  const LookingForward::Constants* dev_looking_forward_constants,
  const float* dev_inv_clus_res,
  short* dev_first_layer_candidates,
  const int seeding_first_layer)
{
  const auto number_of_events = gridDim.x;
  const auto event_number = blockIdx.x;

  // Velo consolidated types
  const Velo::Consolidated::Tracks velo_tracks {
    (uint*) dev_atomics_velo, (uint*) dev_velo_track_hit_number, event_number, number_of_events};
  const Velo::Consolidated::States velo_states {(char*) dev_velo_states, velo_tracks.total_number_of_tracks};
  const uint velo_tracks_offset_event = velo_tracks.tracks_offset(event_number);

  // UT consolidated tracks
  UT::Consolidated::Tracks ut_tracks {(uint*) dev_atomics_ut,
                                      (uint*) dev_ut_track_hit_number,
                                      (float*) dev_ut_qop,
                                      (uint*) dev_ut_track_velo_indices,
                                      event_number,
                                      number_of_events};
  const int ut_event_number_of_tracks = ut_tracks.number_of_tracks(event_number);
  const int ut_event_tracks_offset = ut_tracks.tracks_offset(event_number);

  // SciFi hits
  const uint total_number_of_hits = dev_scifi_hit_count[number_of_events * SciFi::Constants::n_mat_groups_and_mats];
  const SciFi::HitCount scifi_hit_count {(uint32_t*) dev_scifi_hit_count, event_number};
  const SciFi::SciFiGeometry scifi_geometry {dev_scifi_geometry};
  const SciFi::Hits scifi_hits {dev_scifi_hits, total_number_of_hits, &scifi_geometry, dev_inv_clus_res};

  // SciFi un-consolidated track types
  short* first_candidates = dev_first_layer_candidates + ut_event_tracks_offset;
  short* last_candidates  = dev_first_layer_candidates + ut_event_tracks_offset + ut_event_number_of_tracks;

  // Loop over the veloUT input tracks
  for (int i=threadIdx.x; i<ut_event_number_of_tracks; i+=blockDim.x) {
    const int velo_track_index = ut_tracks.velo_track[i];
    const int ut_track_index = ut_event_tracks_offset + i;
    
    const float ut_qop = ut_tracks.qop[i];

    // Note: These data should be accessed like
    //       the previous ut_tracks.qop[i] in the future
    const float ut_x = dev_ut_x[ut_track_index];
    const float ut_tx = dev_ut_tx[ut_track_index];
    const float ut_z = dev_ut_z[ut_track_index];

    const uint velo_states_index = velo_tracks_offset_event + velo_track_index;
    const MiniState velo_state {velo_states, velo_states_index};

    // extrapolate velo y & ty to z of UT x and tx
    // use ty from Velo state
    const MiniState ut_state {ut_x, LookingForward::y_at_z(velo_state, ut_z), ut_z, ut_tx, velo_state.ty};
    const MiniState state_at_z_last_ut_plane = LookingForward::state_at_z(ut_state, LookingForward::z_last_UT_plane);

    lf_calculate_first_layer_window_impl(
      state_at_z_last_ut_plane,
      ut_qop,
      scifi_hits,
      scifi_hit_count,
      seeding_first_layer,
      dev_looking_forward_constants,
      first_candidates,
      last_candidates);
  }
}
