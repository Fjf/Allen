#include "hip/hip_runtime.h"
#include "LFQualityFilter.cuh"

__global__ void lf_quality_filter(
  const uint32_t* dev_scifi_hits,
  const uint32_t* dev_scifi_hit_count,
  const uint* dev_atomics_velo,
  const uint* dev_velo_track_hit_number,
  const char* dev_velo_states,
  const uint* dev_atomics_ut,
  const uint* dev_ut_track_hit_number,
  const float* dev_ut_qop,
  const uint* dev_ut_track_velo_indices,
  SciFi::TrackHits* dev_scifi_lf_tracks,
  const uint* dev_scifi_lf_atomics,
  float* dev_scifi_lf_track_params,
  const char* dev_scifi_geometry,
  const float* dev_inv_clus_res,
  const SciFi::Tracking::TMVA* dev_tmva1,
  const SciFi::Tracking::Arrays* constArrays,
  const float* dev_magnet_polarity,
  uint* dev_atomics_scifi,
  uint* dev_scifi_selected_track_indices,
  SciFi::TrackHits* dev_scifi_tracks)
{
  if (Configuration::verbosity_level >= logger::debug) {
    if (blockIdx.y == 0) {
      printf("\n\n------------- Quality filter ---------------\n");
    }
  }

  const auto number_of_events = gridDim.x;
  const auto event_number = blockIdx.x;

  // Velo consolidated types
  const Velo::Consolidated::Tracks velo_tracks {
    (uint*) dev_atomics_velo, (uint*) dev_velo_track_hit_number, event_number, number_of_events};
  const Velo::Consolidated::States velo_states {(char*) dev_velo_states, velo_tracks.total_number_of_tracks};
  const uint velo_tracks_offset_event = velo_tracks.tracks_offset(event_number);

  // UT consolidated tracks
  const UT::Consolidated::Tracks ut_tracks {(uint*) dev_atomics_ut,
                                            (uint*) dev_ut_track_hit_number,
                                            (float*) dev_ut_qop,
                                            (uint*) dev_ut_track_velo_indices,
                                            event_number,
                                            number_of_events};
  const auto ut_event_tracks_offset = ut_tracks.tracks_offset(event_number);
  const auto ut_event_number_of_tracks = ut_tracks.number_of_tracks(event_number);

  // SciFi hits
  const uint total_number_of_hits = dev_scifi_hit_count[number_of_events * SciFi::Constants::n_mat_groups_and_mats];
  const SciFi::HitCount scifi_hit_count {(uint32_t*) dev_scifi_hit_count, event_number};
  const SciFi::SciFiGeometry scifi_geometry {dev_scifi_geometry};
  const SciFi::Hits scifi_hits {
    const_cast<uint32_t*>(dev_scifi_hits), total_number_of_hits, &scifi_geometry, dev_inv_clus_res};

  const auto number_of_tracks = dev_scifi_lf_atomics[event_number];

  for (uint i = threadIdx.x; i < number_of_tracks; i += blockDim.x) {
    SciFi::TrackHits& track = dev_scifi_lf_tracks
      [ut_event_tracks_offset * LookingForward::maximum_number_of_candidates_per_ut_track_after_x_filter + i];
    const auto current_ut_track_index = ut_event_tracks_offset + track.ut_track_index;
    const auto velo_states_index = velo_tracks_offset_event + ut_tracks.velo_track[track.ut_track_index];
    float* trackParams = dev_scifi_lf_track_params +
                         ut_event_tracks_offset *
                           LookingForward::maximum_number_of_candidates_per_ut_track_after_x_filter *
                           SciFi::Tracking::nTrackParams +
                         i * SciFi::Tracking::nTrackParams;

    const MiniState velo_state = velo_states.getMiniState(velo_states_index);

    track.quality = lf_track_quality(
      track,
      velo_state,
      dev_ut_qop[current_ut_track_index],
      trackParams,
      constArrays,
      dev_magnet_polarity[0],
      dev_tmva1);

    // Save all tracks for efficiency study
    // if (track.quality > 0.01f) {
    // const auto insert_index = atomicAdd(dev_atomics_scifi + event_number, 1);
    // dev_scifi_tracks[ut_event_tracks_offset * SciFi::Constants::max_SciFi_tracks_per_UT_track + insert_index] =
    // track; dev_scifi_selected_track_indices
    //  [ut_event_tracks_offset * SciFi::Constants::max_SciFi_tracks_per_UT_track + insert_index] = i;
    //}
  }

  __syncthreads();

  for (uint i = threadIdx.x; i < ut_event_number_of_tracks; i += blockDim.x) {
    float best_quality = LookingForward::track_min_quality;
    short best_track_index = -1;

    for (uint j = 0; j < number_of_tracks; j++) {
      const SciFi::TrackHits& track = dev_scifi_lf_tracks
        [ut_event_tracks_offset * LookingForward::maximum_number_of_candidates_per_ut_track_after_x_filter + j];
      if (track.ut_track_index == i && track.quality > best_quality) {
        best_quality = track.quality;
        best_track_index = j;
      }
    }

    if (best_track_index != -1) {
      const auto insert_index = atomicAdd(dev_atomics_scifi + event_number, 1);
      assert(insert_index < ut_event_number_of_tracks * SciFi::Constants::max_SciFi_tracks_per_UT_track);
      const auto& track = dev_scifi_lf_tracks
        [ut_event_tracks_offset * LookingForward::maximum_number_of_candidates_per_ut_track_after_x_filter +
         best_track_index];

      if (Configuration::verbosity_level >= logger::debug) {
        track.print(event_number);
      }

      dev_scifi_tracks[ut_event_tracks_offset * SciFi::Constants::max_SciFi_tracks_per_UT_track + insert_index] = track;
      dev_scifi_selected_track_indices
        [ut_event_tracks_offset * SciFi::Constants::max_SciFi_tracks_per_UT_track + insert_index] = best_track_index;
    }
  }
}
