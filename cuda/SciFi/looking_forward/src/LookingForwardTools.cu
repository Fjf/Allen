#include "hip/hip_runtime.h"
#include "LookingForwardTools.cuh"

// straight line extrapolation of MiniState to other z position
__device__ MiniState LookingForward::state_at_z(const MiniState& state, const float z)
{
  return {state.x + (z - state.z) * state.tx, state.y + (z - state.z) * state.ty, z, state.tx, state.ty};
}

// straight line extrapolation of y to other z position
__device__ float LookingForward::y_at_z(const MiniState& state, const float z)
{
  return state.y + (z - state.z) * state.ty;
}

__device__ MiniState LookingForward::propagate_state_from_velo(const MiniState& UT_state, float qop, int layer)
{
  MiniState final_state;
  MiniState magnet_state;

  float x_mag_correction;
  float y_mag_correction;

  // center of the magnet
  magnet_state = state_at_z(UT_state, SciFi::LookingForward::zMagnetParams[0]);

  final_state = magnet_state;

  final_state.tx = SciFi::LookingForward::ds_p_param[layer] * qop + UT_state.tx;

  // TODO this could be done withoud branching
  if (qop > 0) {
    y_mag_correction = SciFi::LookingForward::dp_y_mag_plus[layer][0] +
                       magnet_state.y * SciFi::LookingForward::dp_y_mag_plus[layer][1] +
                       magnet_state.y * magnet_state.y * SciFi::LookingForward::dp_y_mag_plus[layer][2];
    // SciFi::LookingForward::dp_plus_offset[layer];

    x_mag_correction =
      SciFi::LookingForward::dp_x_mag_plus[layer][0] + magnet_state.x * SciFi::LookingForward::dp_x_mag_plus[layer][1] +
      magnet_state.x * magnet_state.x * SciFi::LookingForward::dp_x_mag_plus[layer][2] +
      magnet_state.x * magnet_state.x * magnet_state.x * SciFi::LookingForward::dp_x_mag_plus[layer][3] +
      magnet_state.x * magnet_state.x * magnet_state.x * magnet_state.x *
        SciFi::LookingForward::dp_x_mag_plus[layer][4];
  }
  else {
    y_mag_correction = SciFi::LookingForward::dp_y_mag_minus[layer][0] +
                       magnet_state.y * SciFi::LookingForward::dp_y_mag_minus[layer][1] +
                       magnet_state.y * magnet_state.y * SciFi::LookingForward::dp_y_mag_minus[layer][2]; //+
    // SciFi::LookingForward::dp_minus_offset[layer];

    x_mag_correction =
      SciFi::LookingForward::dp_x_mag_minus[layer][0] +
      magnet_state.x * SciFi::LookingForward::dp_x_mag_minus[layer][1] +
      magnet_state.x * magnet_state.x * SciFi::LookingForward::dp_x_mag_minus[layer][2] +
      magnet_state.x * magnet_state.x * magnet_state.x * SciFi::LookingForward::dp_x_mag_minus[layer][3] +
      magnet_state.x * magnet_state.x * magnet_state.x * magnet_state.x *
        SciFi::LookingForward::dp_x_mag_minus[layer][4];
  }
  final_state = state_at_z(final_state, SciFi::LookingForward::Zone_zPos[layer]);
  final_state.x += -y_mag_correction - x_mag_correction;

  return final_state;
}

__device__ float LookingForward::dx_calc(const float qop)
{
  float ret_val = std::abs(window_params.dx_slope * qop + LookingForward::);
  if (ret_val > window_params.max_window_layer0) {
    ret_val = window_params.max_window_layer0;
  }
  return ret_val;
}

__device__ void LookingForward::linear_regression(
  const std::vector<float>& x,
  const std::vector<float>& y,
  float& m,
  float& q,
  float& chi_2)
{
  float x_avg = 0;
  float x_var = 0;
  float x_y_covar = 0;
  float y_avg = 0;
  m = 0;
  q = 0;
  chi_2 = 0;
  for (int k = 0; k < x.size(); k++) {
    x_avg += x[k];
    y_avg += y[k];
  }
  x_avg /= x.size();
  y_avg /= x.size();

  for (int k = 0; k < x.size(); k++) {
    x_y_covar += (x[k] - x_avg) * (y[k] - y_avg);
    x_var += (x[k] - x_avg) * (x[k] - x_avg);
  }

  m = x_y_covar / x_var;

  q = y_avg - m * x_avg;
  chi_2 = get_chi_2(x, y, [&m, &q](double x) { return m * x + q; });
}

__device__ float LookingForward::get_chi_2(
  const std::vector<float>& x,
  const std::vector<float>& y,
  std::function<float(float)> expected_function)
{
  float chi_2 = 0;
  for (int k = 0; k < x.size(); k++) {
    const float expected_y = expected_function(x[k]);
    chi_2 += (y[k] - expected_y) * (y[k] - expected_y);
  }

  return chi_2;
}

__device__ std::tuple<int, int> LookingForward::find_x_in_window(
  const SciFi::Hits& hits,
  const int zone_offset,
  const int num_hits,
  const float x_min,
  const float x_max)
{
  int first_candidate = binary_search_leftmost(hits.x0 + zone_offset, num_hits, x_min);
  int last_candidate = -1;

  if (first_candidate != -1) {
    last_candidate = binary_search_leftmost(hits.x0 + zone_offset + first_candidate, num_hits - first_candidate, x_max);

    first_candidate = zone_offset + first_candidate;
    last_candidate = last_candidate != -1 ? first_candidate + last_candidate : -1;
  }

  return {first_candidate, last_candidate};
}
