#include "hip/hip_runtime.h"
#include "LFSearchUVWindows.cuh"

__global__ void lf_search_uv_windows(
  const uint32_t* dev_scifi_hits,
  const uint32_t* dev_scifi_hit_count,
  const uint* dev_atomics_ut,
  const SciFi::TrackHits* dev_scifi_tracks,
  const uint* dev_atomics_scifi,
  const char* dev_scifi_geometry,
  const LookingForward::Constants* dev_looking_forward_constants,
  const float* dev_inv_clus_res,
  const MiniState* dev_ut_states,
  short* dev_scifi_lf_uv_windows,
  const int* dev_scifi_lf_initial_windows,
  const float* dev_scifi_lf_parametrization_x_filter)
{
  __shared__ SciFi::SciFiGeometry scifi_geometry;
  __shared__ SciFi::HitCount scifi_hit_count;
  __shared__ SciFi::Hits scifi_hits;

  const auto number_of_events = gridDim.x;
  const auto event_number = blockIdx.x;

  // UT consolidated tracks
  const int ut_event_tracks_offset = dev_atomics_ut[number_of_events + event_number];
  const int ut_event_number_of_tracks = dev_atomics_ut[number_of_events + event_number + 1] - ut_event_tracks_offset;
  const int ut_total_number_of_tracks = dev_atomics_ut[2 * number_of_events];

  // SciFi hits
  const uint total_number_of_hits = dev_scifi_hit_count[number_of_events * SciFi::Constants::n_mat_groups_and_mats];

  if (threadIdx.x == 0) {
    scifi_geometry = SciFi::SciFiGeometry {dev_scifi_geometry};
    scifi_hit_count = SciFi::HitCount {(uint32_t*) dev_scifi_hit_count, event_number};
    scifi_hits =
      SciFi::Hits {const_cast<uint32_t*>(dev_scifi_hits), total_number_of_hits, &scifi_geometry, dev_inv_clus_res};
  }

  __syncthreads();

  const auto event_offset = scifi_hit_count.event_offset();
  const auto number_of_tracks = dev_atomics_scifi[event_number];

  for (uint i = threadIdx.x; i < number_of_tracks; i += blockDim.x) {
    const auto scifi_track_index =
      ut_event_tracks_offset * LookingForward::maximum_number_of_candidates_per_ut_track_after_x_filter + i;
    const SciFi::TrackHits& track = dev_scifi_tracks[scifi_track_index];
    const auto current_ut_track_index = ut_event_tracks_offset + track.ut_track_index;

    // Load parametrization
    const auto a1 = dev_scifi_lf_parametrization_x_filter[scifi_track_index];
    const auto b1 = dev_scifi_lf_parametrization_x_filter
      [ut_total_number_of_tracks * LookingForward::maximum_number_of_candidates_per_ut_track_after_x_filter +
       scifi_track_index];
    const auto c1 = dev_scifi_lf_parametrization_x_filter
      [2 * ut_total_number_of_tracks * LookingForward::maximum_number_of_candidates_per_ut_track_after_x_filter +
       scifi_track_index];
    const auto d_ratio = dev_scifi_lf_parametrization_x_filter
      [3 * ut_total_number_of_tracks * LookingForward::maximum_number_of_candidates_per_ut_track_after_x_filter +
       scifi_track_index];

    for (int relative_uv_layer = 0; relative_uv_layer < 6; ++relative_uv_layer) {
      const auto layer4 = dev_looking_forward_constants->extrapolation_uv_layers[relative_uv_layer];
      const auto z4 = dev_looking_forward_constants->Zone_zPos[layer4];
      const auto projection_y = LookingForward::y_at_z_dzdy_corrected(dev_ut_states[current_ut_track_index], z4);

      const auto projection_x =
        c1 + b1 * (z4 - LookingForward::z_mid_t) +
        a1 * (z4 - LookingForward::z_mid_t) * (z4 - LookingForward::z_mid_t) *
          (1.f + d_ratio * (z4 - LookingForward::z_mid_t)) -
        dev_looking_forward_constants->Zone_dxdy_uvlayers[relative_uv_layer & 0x1] * projection_y;

      const auto uv_search_window_start = dev_scifi_lf_initial_windows
        [ut_event_tracks_offset + track.ut_track_index + (relative_uv_layer * 8 + 2) * ut_total_number_of_tracks];
      const auto uv_search_window_size = dev_scifi_lf_initial_windows
        [ut_event_tracks_offset + track.ut_track_index + (relative_uv_layer * 8 + 3) * ut_total_number_of_tracks];

      const auto layer_candidates = LookingForward::find_x_in_window(
        scifi_hits.x0 + event_offset,
        uv_search_window_start,
        uv_search_window_size,
        projection_x,
        LookingForward::chi2_max_extrapolation_to_uv_layers_single);

      dev_scifi_lf_uv_windows
        [6 * ut_event_tracks_offset * LookingForward::maximum_number_of_candidates_per_ut_track_after_x_filter +
         relative_uv_layer * ut_event_number_of_tracks *
           LookingForward::maximum_number_of_candidates_per_ut_track_after_x_filter +
         i] = std::get<0>(layer_candidates);

      dev_scifi_lf_uv_windows
        [6 * ut_total_number_of_tracks * LookingForward::maximum_number_of_candidates_per_ut_track_after_x_filter +
         6 * ut_event_tracks_offset * LookingForward::maximum_number_of_candidates_per_ut_track_after_x_filter +
         relative_uv_layer * ut_event_number_of_tracks *
           LookingForward::maximum_number_of_candidates_per_ut_track_after_x_filter +
         i] = std::get<1>(layer_candidates);

      // printf("Candidates: %i, %i\n", std::get<0>(layer_candidates), std::get<1>(layer_candidates));
    }
  }
}
