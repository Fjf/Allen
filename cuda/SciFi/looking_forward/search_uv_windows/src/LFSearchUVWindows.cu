#include "hip/hip_runtime.h"
#include "LFSearchUVWindows.cuh"

__global__ void lf_search_uv_windows(
  const uint32_t* dev_scifi_hits,
  const uint32_t* dev_scifi_hit_count,
  const uint* dev_atomics_ut,
  const SciFi::TrackHits* dev_scifi_tracks,
  const uint* dev_atomics_scifi,
  const char* dev_scifi_geometry,
  const LookingForward::Constants* dev_looking_forward_constants,
  const float* dev_inv_clus_res,
  const MiniState* dev_ut_states,
  short* dev_scifi_lf_uv_windows,
  const int* dev_scifi_lf_initial_windows)
{
  __shared__ SciFi::SciFiGeometry scifi_geometry;
  __shared__ SciFi::HitCount scifi_hit_count;
  __shared__ SciFi::Hits scifi_hits;

  const auto number_of_events = gridDim.x;
  const auto event_number = blockIdx.x;

  // UT consolidated tracks
  const int ut_event_tracks_offset = dev_atomics_ut[number_of_events + event_number];
  const int ut_event_number_of_tracks = dev_atomics_ut[number_of_events + event_number + 1] - ut_event_tracks_offset;
  const int total_number_of_ut_tracks = dev_atomics_ut[2 * number_of_events];

  // SciFi hits
  const uint total_number_of_hits = dev_scifi_hit_count[number_of_events * SciFi::Constants::n_mat_groups_and_mats];

  if (threadIdx.x == 0) {
    scifi_geometry = SciFi::SciFiGeometry {dev_scifi_geometry};
    scifi_hit_count = SciFi::HitCount {(uint32_t*) dev_scifi_hit_count, event_number};
    scifi_hits =
      SciFi::Hits {const_cast<uint32_t*>(dev_scifi_hits), total_number_of_hits, &scifi_geometry, dev_inv_clus_res};
  }

  __syncthreads();

  const auto event_offset = scifi_hit_count.event_offset();
  const auto number_of_tracks = dev_atomics_scifi[event_number];

  for (uint i = threadIdx.x; i < number_of_tracks; i += blockDim.x) {
    const SciFi::TrackHits& track = dev_scifi_tracks
      [ut_event_tracks_offset * LookingForward::maximum_number_of_candidates_per_ut_track_after_x_filter + i];
    const auto current_ut_track_index = ut_event_tracks_offset + track.ut_track_index;

    // Note: The notation 1, 2, 3 is used here (instead of h0, h1, h2)
    //       to avoid mistakes, as the code is similar to that of Hybrid Seeding
    const auto h1 = event_offset + track.hits[0];
    const auto h2 = event_offset + track.hits[1];
    const auto h3 = event_offset + track.hits[2];
    const auto x1 = scifi_hits.x0[h1];
    const auto x2 = scifi_hits.x0[h2];
    const auto x3 = scifi_hits.x0[h3];
    const auto layer1 = scifi_hits.planeCode(h1) / 2;
    const auto layer2 = scifi_hits.planeCode(h2) / 2;
    const auto layer3 = scifi_hits.planeCode(h3) / 2;
    const auto z1_noref = dev_looking_forward_constants->Zone_zPos[layer1];
    const auto z2_noref = dev_looking_forward_constants->Zone_zPos[layer2];
    const auto z3_noref = dev_looking_forward_constants->Zone_zPos[layer3];

    // From hybrid seeding
    constexpr float z_mid_t = 8520.f * Gaudi::Units::mm;
    constexpr float d_ratio = -0.0000262f;

    const auto z1 = z1_noref - z_mid_t;
    const auto z2 = z2_noref - z_mid_t;
    const auto z3 = z3_noref - z_mid_t;
    const auto corrZ1 = 1.f + d_ratio * z1;
    const auto corrZ2 = 1.f + d_ratio * z2;
    const auto corrZ3 = 1.f + d_ratio * z3;

    const auto det = z1 * z1 * corrZ1 * z2 + z1 * z3 * z3 * corrZ3 + z2 * z2 * corrZ2 * z3 - z2 * z3 * z3 * corrZ3 -
                     z1 * z2 * z2 * corrZ2 - z3 * z1 * z1 * corrZ1;
    const auto det1 = x1 * z2 + z1 * x3 + x2 * z3 - z2 * x3 - z1 * x2 - z3 * x1;
    const auto det2 = z1 * z1 * corrZ1 * x2 + x1 * z3 * z3 * corrZ3 + z2 * z2 * corrZ2 * x3 - x2 * z3 * z3 * corrZ3 -
                      x1 * z2 * z2 * corrZ2 - x3 * z1 * z1 * corrZ1;
    const auto det3 = z1 * z1 * corrZ1 * z2 * x3 + z1 * z3 * z3 * corrZ3 * x2 + z2 * z2 * corrZ2 * z3 * x1 -
                      z2 * z3 * z3 * corrZ3 * x1 - z1 * z2 * z2 * corrZ2 * x3 - z3 * z1 * z1 * corrZ1 * x2;

    const auto recdet = 1.f / det;
    const auto a1 = recdet * det1;
    const auto b1 = recdet * det2;
    const auto c1 = recdet * det3;

    for (int relative_uv_layer = 0; relative_uv_layer < 6; ++relative_uv_layer) {
      const auto layer4 = dev_looking_forward_constants->extrapolation_uv_layers[relative_uv_layer];
      const auto z4 = dev_looking_forward_constants->Zone_zPos[layer4];
      const auto projection_y = LookingForward::y_at_z_dzdy_corrected(dev_ut_states[current_ut_track_index], z4);

      const auto projection_x =
        c1 + b1 * (z4 - z_mid_t) + a1 * (z4 - z_mid_t) * (z4 - z_mid_t) * (1.f + d_ratio * (z4 - z_mid_t)) -
        dev_looking_forward_constants->Zone_dxdy_uvlayers[relative_uv_layer & 0x1] * projection_y;

      const auto uv_search_window_start = dev_scifi_lf_initial_windows
        [ut_event_tracks_offset + track.ut_track_index + (relative_uv_layer * 8 + 2) * total_number_of_ut_tracks];
      const auto uv_search_window_size = dev_scifi_lf_initial_windows
        [ut_event_tracks_offset + track.ut_track_index + (relative_uv_layer * 8 + 3) * total_number_of_ut_tracks];

      const auto layer_candidates = LookingForward::find_x_in_window(
        scifi_hits.x0 + event_offset,
        uv_search_window_start,
        uv_search_window_size,
        projection_x,
        LookingForward::chi2_max_extrapolation_to_uv_layers_single);

      dev_scifi_lf_uv_windows
        [6 * ut_event_tracks_offset * LookingForward::maximum_number_of_candidates_per_ut_track_after_x_filter +
         relative_uv_layer * ut_event_number_of_tracks *
           LookingForward::maximum_number_of_candidates_per_ut_track_after_x_filter +
         i] = std::get<0>(layer_candidates);

      dev_scifi_lf_uv_windows
        [6 * total_number_of_ut_tracks * LookingForward::maximum_number_of_candidates_per_ut_track_after_x_filter +
         6 * ut_event_tracks_offset * LookingForward::maximum_number_of_candidates_per_ut_track_after_x_filter +
         relative_uv_layer * ut_event_number_of_tracks *
           LookingForward::maximum_number_of_candidates_per_ut_track_after_x_filter +
         i] = std::get<1>(layer_candidates);

      // printf("Candidates: %i, %i\n", std::get<0>(layer_candidates), std::get<1>(layer_candidates));
    }
  }
}
