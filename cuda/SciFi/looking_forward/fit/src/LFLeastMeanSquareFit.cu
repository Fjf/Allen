#include "hip/hip_runtime.h"
#include "LFLeastMeanSquareFit.cuh"

void lf_least_mean_square_fit_t::operator()(
  const ArgumentRefManager<Arguments>& arguments,
  const RuntimeOptions& runtime_options,
  const Constants& constants,
  HostBuffers& host_buffers,
  hipStream_t& hip_stream,
  hipEvent_t& cuda_generic_event) const
{
  function.invoke(dim3(host_buffers.host_number_of_selected_events[0]), block_dimension(), hip_stream)(
    arguments.offset<dev_scifi_hits>(),
    arguments.offset<dev_scifi_hit_count>(),
    arguments.offset<dev_atomics_ut>(),
    arguments.offset<dev_scifi_lf_x_filtered_tracks>(),
    arguments.offset<dev_scifi_lf_x_filtered_atomics>(),
    constants.dev_scifi_geometry,
    constants.dev_looking_forward_constants,
    constants.dev_inv_clus_res,
    arguments.offset<dev_scifi_lf_parametrization_x_filter>());
}

__global__ void lf_least_mean_square_fit(
  const uint32_t* dev_scifi_hits,
  const uint32_t* dev_scifi_hit_count,
  const uint* dev_atomics_ut,
  SciFi::TrackHits* dev_scifi_tracks,
  const uint* dev_atomics_scifi,
  const char* dev_scifi_geometry,
  const LookingForward::Constants* dev_looking_forward_constants,
  const float* dev_inv_clus_res,
  float* dev_scifi_lf_parametrization_x_filter)
{
  const uint number_of_events = gridDim.x;
  const uint event_number = blockIdx.x;

  const auto ut_event_tracks_offset = dev_atomics_ut[number_of_events + event_number];
  const auto ut_total_number_of_tracks = dev_atomics_ut[2 * number_of_events];

  // SciFi hits
  const uint total_number_of_hits = dev_scifi_hit_count[number_of_events * SciFi::Constants::n_mat_groups_and_mats];
  const SciFi::HitCount scifi_hit_count {(uint32_t*) dev_scifi_hit_count, event_number};
  const SciFi::SciFiGeometry scifi_geometry {dev_scifi_geometry};
  const SciFi::Hits scifi_hits {
    const_cast<uint32_t*>(dev_scifi_hits), total_number_of_hits, &scifi_geometry, dev_inv_clus_res};
  const auto event_offset = scifi_hit_count.event_offset();
  const auto number_of_tracks = dev_atomics_scifi[event_number];

  float s00 = 0.f;
  float s01 = 0.f;
  float s02 = 0.f;
  float s11 = 0.f;
  float s12 = 0.f;
  float s22 = 0.f;
  float b0 = 0.f;
  float b1 = 0.f;
  float b2 = 0.f;

  for (uint i = threadIdx.x; i < number_of_tracks; i += blockDim.x) {
    const auto scifi_track_index =
      ut_event_tracks_offset * LookingForward::maximum_number_of_candidates_per_ut_track + i;
    SciFi::TrackHits& track = dev_scifi_tracks[scifi_track_index];

    // Load parametrization
    const auto prev_curvature = dev_scifi_lf_parametrization_x_filter[scifi_track_index];
    const auto prev_tx = dev_scifi_lf_parametrization_x_filter
      [ut_total_number_of_tracks * LookingForward::maximum_number_of_candidates_per_ut_track +
       scifi_track_index];
    const auto prev_offset = dev_scifi_lf_parametrization_x_filter
      [2 * ut_total_number_of_tracks * LookingForward::maximum_number_of_candidates_per_ut_track +
       scifi_track_index];
    const auto d_ratio = dev_scifi_lf_parametrization_x_filter
      [3 * ut_total_number_of_tracks * LookingForward::maximum_number_of_candidates_per_ut_track +
       scifi_track_index];

    for (uint i_hit = 0; i_hit < track.hitsNum; ++i_hit) {
      const auto hit_index = event_offset + track.hits[i_hit];
      const auto layer_index = scifi_hits.planeCode(hit_index) / 2;
      const auto x = scifi_hits.x0[hit_index];
      const auto z = dev_looking_forward_constants->Zone_zPos[layer_index];

      const auto dz = z - LookingForward::z_mid_t;
      const auto predicted_x = prev_offset + prev_tx * dz + prev_curvature * dz * dz * (1.f + d_ratio * dz);

      const auto dz2 = dz * dz;
      const auto deta = dz2 * (1.f + d_ratio * dz);
      const auto dzeta = dz * deta;
      const auto deta2 = deta * deta;

      s01 += dz;
      s02 += deta;
      s11 += dz2;
      s12 += dzeta;
      s22 += deta2;

      const auto dx = x - predicted_x;
      const auto dzdx = dz * dx;
      const auto detadx = deta * dx;

      b0 += dx;
      b1 += dzdx;
      b2 += detadx;
    }

    s00 = track.hitsNum;

    const auto d = s00 * (s11 * s22 - s12 * s12) - s01 * (s01 * s22 - s12 * s02) + s02 * (s01 * s12 - s11 * s02);
    const auto d_a = b0 * (s11 * s22 - s12 * s12) - b1 * (s01 * s22 - s12 * s02) + b2 * (s01 * s12 - s11 * s02);
    const auto d_b = -b0 * (s01 * s22 - s12 * s02) + b1 * (s00 * s22 - s02 * s02) - b2 * (s00 * s12 - s02 * s01);
    const auto d_c = b0 * (s01 * s12 - s11 * s02) - b1 * (s00 * s12 - s01 * s02) + b2 * (s00 * s11 - s01 * s01);

    const auto d_inv = 1.f / d;
    const auto offset = prev_offset + d_a * d_inv;
    const auto tx = prev_tx + d_b * d_inv;
    const auto curvature = prev_curvature + d_c * d_inv;

    // Update parametrization
    dev_scifi_lf_parametrization_x_filter[scifi_track_index] = curvature;
    dev_scifi_lf_parametrization_x_filter
      [ut_total_number_of_tracks * LookingForward::maximum_number_of_candidates_per_ut_track +
       scifi_track_index] = tx;
    dev_scifi_lf_parametrization_x_filter
      [2 * ut_total_number_of_tracks * LookingForward::maximum_number_of_candidates_per_ut_track +
       scifi_track_index] = offset;

    // Update track quality
    track.quality = 0.f;
    for (uint i_hit = 0; i_hit < track.hitsNum; ++i_hit) {
      const auto hit_index = event_offset + track.hits[i_hit];
      const auto layer_index = scifi_hits.planeCode(hit_index) / 2;
      const auto x = scifi_hits.x0[hit_index];
      const auto z = dev_looking_forward_constants->Zone_zPos[layer_index];

      const auto dz = z - LookingForward::z_mid_t;
      const auto predicted_x = offset + tx * dz + curvature * dz * dz * (1.f + d_ratio * dz);

      track.quality += (x - predicted_x) * (x - predicted_x);
    }
  }
}
