#include "hip/hip_runtime.h"
#include "LFTripletSeedingImpl.cuh"
#include "BinarySearchTools.cuh"
#include "LookingForwardTools.cuh"

struct CombinedTripletValue {
  float chi2 = 10000.f;
  int16_t h0 = -1;
  int16_t h1 = -1;
  int16_t h2 = -1;
};

__device__ void lf_triplet_seeding_impl(
  const float* scifi_hits_x0,
  const uint layer_0,
  const uint layer_1,
  const uint layer_2,
  const int l0_start,
  const int l1_start,
  const int l2_start,
  const int l0_extrapolated,
  const int l1_extrapolated,
  const int l2_extrapolated,
  const int l0_size,
  const int l1_size,
  const int l2_size,
  const float z0,
  const float z1,
  const float z2,
  const float qop,
  const MiniState* ut_state,
  float* shared_partial_chi2,
  SciFi::TrackHits* scifi_tracks,
  uint* atomics_scifi,
  const LookingForward::Constants* dev_looking_forward_constants,
  const uint number_of_ut_track,
  const uint number_of_seeds)
{
  std::vector<CombinedTripletValue> best_combined;

  if (Configuration::verbosity_level >= logger::debug) {
    printf("---- Seeding of event %i with x layers {%i, %i, %i} ----\n", blockIdx.x,
      layer_0, layer_1, layer_2);
  }

  const auto tx = ut_state->tx;

  constexpr float p0 = -2.1156e-07f;  //   +/-   3.87224e-07
  constexpr float p1 = 0.000829677f;  //   +/-   4.70098e-06
  constexpr float p2 = -0.000174757f; //   +/-   1.00272e-05

  // // Required constants for the chi2 calculation below
  // const float zdiff = (z2 - z0) / (z1 - z0);
  // float extrap1 = LookingForward::get_extrap(qop, z1 - z0);
  // extrap1 *= extrap1;
  // const float extrap2 = LookingForward::get_extrap(qop, (z2 - z0));

  // Dumb search of best triplet
  for (int i = 0; i < l0_size; ++i) {
    const auto x0 = scifi_hits_x0[l0_start + i];

    for (int j = 0; j < l2_size; ++j) {
      const auto x2 = scifi_hits_x0[l2_start + j];
      // const auto partial_chi2 = x2 - x0 + x0 * zdiff - extrap2;

      const float slope_t1_t3 = (x0 - x2) / (z0 - z2);
      const float delta_slope = fabsf(tx - slope_t1_t3);
      const auto updated_qop = 1.f / (1.f / (p0 + p1 * delta_slope - p2 * delta_slope * delta_slope) + 5.08211e+02f);
      const auto expected_x1 = x0 + slope_t1_t3 * (z1 - z0) + 0.02528f + 13624.f * updated_qop;

      for (int k = 0; k < l1_size; ++k) {
        const auto x1 = scifi_hits_x0[l1_start + k];
        const auto chi2 = fabsf(expected_x1 - x1);

        if (chi2 < LookingForward::chi2_max_triplet_single) {
          best_combined.push_back(CombinedTripletValue {chi2, (int16_t) i, (int16_t) k, (int16_t) j});
        }
      }
    }
  }

  std::sort(
    best_combined.begin(), best_combined.end(), [](const CombinedTripletValue& a, const CombinedTripletValue& b) {
      return a.chi2 < b.chi2;
    });

  // Note: LookingForward::maximum_number_of_candidates_per_ut_track / number of seeds is the maximum that can be stored
  for (int i = 0;
       i < (LookingForward::maximum_number_of_candidates_per_ut_track / number_of_seeds) && i < best_combined.size();
       ++i) {
    const auto best_combo = best_combined[i];

    if (best_combo.h0 != -1) {
      const auto h0 = l0_start + best_combo.h0;
      const auto h1 = l1_start + best_combo.h1;
      const auto h2 = l2_start + best_combo.h2;

      const int insert_index = atomicAdd(atomics_scifi, 1);

      const auto l1_station = layer_1 / 2;
      const auto track = SciFi::TrackHits {
        h0,
        h1,
        h2,
        (uint16_t) layer_0,
        (uint16_t) layer_1,
        (uint16_t) layer_2,
        best_combo.chi2,
        LookingForward::qop_update_multi_par(
          *ut_state, scifi_hits_x0[h0], z0, scifi_hits_x0[h1], z1, l1_station, dev_looking_forward_constants),
        number_of_ut_track};
      scifi_tracks[insert_index] = track;

      if (Configuration::verbosity_level >= logger::debug) {
        track.print(blockIdx.x);
      }
    }
  }
}
