#include "hip/hip_runtime.h"
#include "LFPromoteCandidates.cuh"

__global__ void lf_promote_candidates(
  uint32_t* dev_scifi_hits,
  const uint32_t* dev_scifi_hit_count,
  const int* dev_atomics_ut,
  const SciFi::TrackCandidate* dev_scifi_track_candidates,
  const unsigned short* dev_extrapolation_layer_candidates,
  SciFi::TrackHits* dev_scifi_tracks,
  bool* dev_scifi_track_promoted_candidates,
  int* dev_atomics_scifi,
  const char* dev_scifi_geometry,
  const LookingForward::Constants* dev_looking_forward_constants,
  const float* dev_inv_clus_res,
  const MiniState* dev_ut_states,
  const uint8_t layer)
{
  const auto number_of_events = gridDim.x;
  const auto event_number = blockIdx.x;

  // UT consolidated tracks
  const int ut_event_tracks_offset = dev_atomics_ut[number_of_events + event_number];

  // SciFi hits
  const uint total_number_of_hits = dev_scifi_hit_count[number_of_events * SciFi::Constants::n_mat_groups_and_mats];
  const SciFi::HitCount scifi_hit_count {(uint32_t*) dev_scifi_hit_count, event_number};
  const SciFi::SciFiGeometry scifi_geometry {dev_scifi_geometry};
  SciFi::Hits scifi_hits(dev_scifi_hits, total_number_of_hits, &scifi_geometry, dev_inv_clus_res);

  // SciFi un-consolidated track types
  int number_of_candidates = dev_atomics_scifi[event_number];
  int* track_insert_atomic = dev_atomics_scifi + number_of_events + event_number;

  // Only proceed if we have candidates in the first window
  for (int i=threadIdx.x; i<number_of_candidates; i+=blockDim.x) {
    if (dev_scifi_track_promoted_candidates[event_number * SciFi::Constants::max_track_candidates + i] == false) {
      const unsigned short* extrapolation_layer_candidates = dev_extrapolation_layer_candidates + event_number * SciFi::Constants::max_track_candidates + i;
      const SciFi::TrackCandidate track_candidate = dev_scifi_track_candidates[event_number * SciFi::Constants::max_track_candidates + i];
      const MiniState state_at_z_last_ut_plane = dev_ut_states[ut_event_tracks_offset + track_candidate.ut_track_index];
      const float projection_y = LookingForward::y_at_z(state_at_z_last_ut_plane, dev_looking_forward_constants->Zone_zPos[layer]);

      const auto best_index_quality = lf_promote_candidates_impl(
        projection_y,
        extrapolation_layer_candidates[0],
        extrapolation_layer_candidates[number_of_events * SciFi::Constants::max_track_candidates],
        scifi_hits,
        scifi_hit_count,
        dev_looking_forward_constants,
        track_candidate,
        layer);

      if (std::get<0>(best_index_quality) != -1) {
        dev_scifi_track_promoted_candidates[event_number * SciFi::Constants::max_track_candidates + i] = true;
        SciFi::TrackHits track {track_candidate};
        track.add_hit_with_quality((uint16_t) std::get<0>(best_index_quality), std::get<1>(best_index_quality));

        const int current_insert_index = atomicAdd(track_insert_atomic, 1);
        if (current_insert_index < SciFi::Constants::max_tracks) {
          dev_scifi_tracks[event_number * SciFi::Constants::max_tracks + current_insert_index] = track;
        }
      }
    }
  }
}
