#include "hip/hip_runtime.h"
#include "LFQualityFilterLength.cuh"

__global__ void lf_quality_filter_length::lf_quality_filter_length(lf_quality_filter_length::Parameters parameters)
{
  const auto event_number = blockIdx.x;
  const auto number_of_events = gridDim.x;

  const auto ut_event_tracks_offset = parameters.dev_atomics_ut[number_of_events + event_number];
  const auto number_of_tracks = parameters.dev_scifi_lf_atomics[event_number];
  const auto ut_total_number_of_tracks = parameters.dev_atomics_ut[2 * number_of_events];

  for (uint i = threadIdx.x; i < number_of_tracks; i += blockDim.x) {
    const auto scifi_track_index =
      ut_event_tracks_offset * LookingForward::maximum_number_of_candidates_per_ut_track + i;
    const SciFi::TrackHits& track = parameters.dev_scifi_lf_tracks[scifi_track_index];

    if (track.hitsNum >= LookingForward::track_min_hits) {
      const auto insert_index = atomicAdd(parameters.dev_scifi_lf_length_filtered_atomics + event_number, 1);
      const auto new_scifi_track_index =
        ut_event_tracks_offset * LookingForward::maximum_number_of_candidates_per_ut_track + insert_index;

      parameters.dev_scifi_lf_length_filtered_tracks[new_scifi_track_index] = track;

      // Save track parameters to new container as well
      const auto a1 = parameters.dev_scifi_lf_parametrization[scifi_track_index];
      const auto b1 =
        parameters.dev_scifi_lf_parametrization
          [ut_total_number_of_tracks * LookingForward::maximum_number_of_candidates_per_ut_track + scifi_track_index];
      const auto c1 = parameters.dev_scifi_lf_parametrization
                        [2 * ut_total_number_of_tracks * LookingForward::maximum_number_of_candidates_per_ut_track +
                         scifi_track_index];
      const auto d_ratio =
        parameters.dev_scifi_lf_parametrization
          [3 * ut_total_number_of_tracks * LookingForward::maximum_number_of_candidates_per_ut_track +
           scifi_track_index];

      parameters.dev_scifi_lf_parametrization_length_filter[new_scifi_track_index] = a1;
      parameters.dev_scifi_lf_parametrization_length_filter
        [ut_total_number_of_tracks * LookingForward::maximum_number_of_candidates_per_ut_track +
         new_scifi_track_index] = b1;
      parameters.dev_scifi_lf_parametrization_length_filter
        [2 * ut_total_number_of_tracks * LookingForward::maximum_number_of_candidates_per_ut_track +
         new_scifi_track_index] = c1;
      parameters.dev_scifi_lf_parametrization_length_filter
        [3 * ut_total_number_of_tracks * LookingForward::maximum_number_of_candidates_per_ut_track +
         new_scifi_track_index] = d_ratio;
    }
  }
}
