#include "hip/hip_runtime.h"
// *********************************************************************************
// ************************ Introduction to Forward Tracking **********************
// *********************************************************************************
//
//  A detailed introduction in Forward tracking (with real pictures!) can be
//  found here:
//  (2002) http://cds.cern.ch/record/684710/files/lhcb-2002-008.pdf
//  (2007) http://cds.cern.ch/record/1033584/files/lhcb-2007-015.pdf
//  (2014) http://cds.cern.ch/record/1641927/files/LHCb-PUB-2014-001.pdf
//
// *** Short Introduction in geometry:
//
// The SciFi Tracker Detector, or simple Fibre Tracker (FT) consits out of 3 stations.
// Each station consists out of 4 planes/layers. Thus there are in total 12 layers,
// in which a particle can leave a hit. The reasonable maximum number of hits a track
// can have is thus also 12 (sometimes 2 hits per layer are picked up).
//
// Each layer consists out of several Fibre mats. A fibre has a diameter of below a mm.(FIXME)
// Several fibres are glued alongside each other to form a mat.
// A Scintilating Fibre produces light, if a particle traverses. This light is then
// detected on the outside of the Fibre mat.
//
// Looking from the collision point, one (X-)layer looks like the following:
//
//                    y       6m
//                    ^  ||||||||||||| Upper side
//                    |  ||||||||||||| 2.5m
//                    |  |||||||||||||
//                   -|--||||||o||||||----> -x
//                       |||||||||||||
//                       ||||||||||||| Lower side
//                       ||||||||||||| 2.5m
//
// All fibres are aranged parallel to the y-axis. There are three different
// kinds of layers, denoted by X,U,V. The U/V layers are rotated with respect to
// the X-layers by +/- 5 degrees, to also get a handle of the y position of the
// particle. As due to the magnetic field particles are only deflected in
// x-direction, this configuration offers the best resolution.
// The layer structure in the FT is XUVX-XUVX-XUVX.
//
// The detector is divided into an upeer and a lower side (>/< y=0). As particles
// are only deflected in x direction there are only very(!) few particles that go
// from the lower to the upper side, or vice versa. The reconstruction algorithm
// can therefore be split into two independent steps: First track reconstruction
// for tracks in the upper side, and afterwards for tracks in the lower side.
//
// Due to construction issues this is NOT true for U/V layers. In these layers the
// complete(!) fibre modules are rotated, producing a zic-zac pattern at y=0, also
// called  "the triangles". Therefore for U/V layers it must be explicetly also
// searched for these hit on the "other side", if the track is close to y=0.
// Sketch (rotation exagerated!):
//                                          _.*
//     y ^   _.*                         _.*
//       | .*._      Upper side       _.*._
//       |     *._                 _.*     *._
//       |--------*._           _.*           *._----------------> x
//       |           *._     _.*                 *._     _.*
//                      *._.*       Lower side      *._.*
//
//
//
//
//
//       Zone ordering defined on PrKernel/PrFTInfo.h
//
//     y ^
//       |    1  3  5  7     9 11 13 15    17 19 21 23
//       |    |  |  |  |     |  |  |  |     |  |  |  |
//       |    x  u  v  x     x  u  v  x     x  u  v  x   <-- type of layer
//       |    |  |  |  |     |  |  |  |     |  |  |  |
//       |------------------------------------------------> z
//       |    |  |  |  |     |  |  |  |     |  |  |  |
//       |    |  |  |  |     |  |  |  |     |  |  |  |
//       |    0  2  4  6     8 10 12 14    16 18 20 22
//
//
// *** Short introduction in the Forward Tracking algorithm
//
// The track reconstruction is seperated into several steps:
//
// 1) Using only X-hits
//    1.1) Preselection: collectAllXHits()
//    1.2) Hough Transformation: xAtRef_SamePlaneHits()
//    1.3) Cluster search: selectXCandidates()
//    1.4) Linear and than Cubic Fit of X-Projection
// 2) Introducing U/V hits or also called stereo hits
//    2.1) Preselection: collectStereoHits
//    2.2) Cluster search: selectStereoHits
//    2.3) Fit Y-Projection
// 3) Using all (U+V+X) hits
//    3.1) Fitting X-Projection
//    3.2) calculating track quality with a Neural Net
//    3.3) final clone+ghost killing
//
// *****************************************************************

#include "PrForward.cuh"

//-----------------------------------------------------------------------------
// Implementation file for class : PrForward
//
// Based on code written by :
// 2012-03-20 : Olivier Callot
// 2013-03-15 : Thomas Nikodem
// 2015-02-13 : Sevda Esen [additional search in the triangles by Marian Stahl]
// 2016-03-09 : Thomas Nikodem [complete restructuring]
// 2018-08    : Vava Gligorov [extract code from Rec, make compile within GPU framework
// 2018-09    : Dorothea vom Bruch [convert to CUDA, runs on GPU]
//-----------------------------------------------------------------------------

//=============================================================================

// Kernel to call Forward tracking on GPU
// Loop over veloUT input tracks using threadIdx.x
__global__ void scifi_pr_forward(
  uint32_t* dev_scifi_hits,
  const uint32_t* dev_scifi_hit_count,
  const int* dev_atomics_velo,
  const uint* dev_velo_track_hit_number,
  const char* dev_velo_states,
  const int* dev_atomics_ut,
  const char* dev_ut_track_hits,
  const uint* dev_ut_track_hit_number,
  const float* dev_ut_qop,
  const uint* dev_ut_track_velo_indices,
  SciFi::TrackHits* dev_scifi_tracks,
  int* dev_atomics_scifi,
  const SciFi::Tracking::TMVA* dev_tmva1,
  const SciFi::Tracking::TMVA* dev_tmva2,
  const SciFi::Tracking::Arrays* dev_constArrays,
  const char* dev_scifi_geometry,
  const float* dev_inv_clus_res)
{
  const uint number_of_events = gridDim.x;
  const uint event_number = blockIdx.x;

  // Velo consolidated types
  const Velo::Consolidated::Tracks velo_tracks {
    (uint*) dev_atomics_velo, (uint*) dev_velo_track_hit_number, event_number, number_of_events};
  const Velo::Consolidated::States velo_states {(char*) dev_velo_states, velo_tracks.total_number_of_tracks};
  const uint velo_tracks_offset_event = velo_tracks.tracks_offset(event_number);

  // UT consolidated tracks
  UT::Consolidated::Tracks ut_tracks {(uint*) dev_atomics_ut,
                                      (uint*) dev_ut_track_hit_number,
                                      (float*) dev_ut_qop,
                                      (uint*) dev_ut_track_velo_indices,
                                      event_number,
                                      number_of_events};
  const int n_veloUT_tracks_event = ut_tracks.number_of_tracks(event_number);

  // SciFi un-consolidated track types
  SciFi::TrackHits* scifi_tracks_event = dev_scifi_tracks + event_number * SciFi::Constants::max_tracks;
  int* atomics_scifi_event = dev_atomics_scifi + event_number;

  // SciFi hits
  const uint total_number_of_hits = dev_scifi_hit_count[number_of_events * SciFi::Constants::n_mat_groups_and_mats];
  const SciFi::HitCount scifi_hit_count {(uint32_t*) dev_scifi_hit_count, event_number};
  const SciFi::SciFiGeometry scifi_geometry {dev_scifi_geometry};
  SciFi::Hits scifi_hits(dev_scifi_hits, total_number_of_hits, &scifi_geometry, dev_inv_clus_res);

  // initialize atomic SciFi tracks counter
  if (threadIdx.x == 0) {
    *atomics_scifi_event = 0;
  }
  __syncthreads();

  // Loop over the veloUT input tracks
  for (int i = 0; i < (n_veloUT_tracks_event + blockDim.x - 1) / blockDim.x; ++i) {
    const int i_veloUT_track = i * blockDim.x + threadIdx.x;
    if (i_veloUT_track < n_veloUT_tracks_event) {
      const float qop_ut = ut_tracks.qop[i_veloUT_track];

      const int i_velo_track = ut_tracks.velo_track[i_veloUT_track];
      const uint velo_states_index = velo_tracks_offset_event + i_velo_track;
      const MiniState velo_state {velo_states, velo_states_index};

      find_forward_tracks(
        scifi_hits,
        scifi_hit_count,
        qop_ut,
        i_veloUT_track,
        scifi_tracks_event,
        (uint*) atomics_scifi_event,
        dev_tmva1,
        dev_tmva2,
        dev_constArrays,
        velo_state);
    }
  }
}
