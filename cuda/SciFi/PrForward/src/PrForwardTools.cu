#include "hip/hip_runtime.h"
#include "PrForwardTools.cuh"

/* Look first in x layers, then in stereo layers for hits
   do 1D Hough transform for x- and stereo hits
   do global 1D Hough transform
   use TMVAs to obtain track quality */
__host__ __device__ void find_forward_tracks(
  const SciFi::Hits& scifi_hits,
  const SciFi::HitCount& scifi_hit_count,
  const float qop_ut,
  const int i_veloUT_track,
  SciFi::TrackHits* outputTracks,
  uint* n_forward_tracks,
  const SciFi::Tracking::TMVA* tmva1,
  const SciFi::Tracking::TMVA* tmva2,
  const SciFi::Tracking::Arrays* constArrays,
  const MiniState& velo_state)
{

  // The LHCb framework code had a PT preselection for the VeloUT tracks
  // here, which I am removing because this should be done explicitly through
  // track selectors if we do it at all, not hacked inside the tracking code

  const float zRef_track = SciFi::Tracking::zReference;
  const float xAtRef = xFromVelo(zRef_track, velo_state);
  const float xParams_seed[4] = {xAtRef, velo_state.tx, 0.f, 0.f};
  const float yAtRef = yFromVelo(zRef_track, velo_state);
  const float yParams_seed[4] = {yAtRef, velo_state.ty, 0.f, 0.f};

  // First loop Hough cluster search, set initial search windows
  SciFi::Tracking::HitSearchCuts pars_first {SciFi::Tracking::minXHits,
                                             SciFi::Tracking::maxXWindow,
                                             SciFi::Tracking::maxXWindowSlope,
                                             SciFi::Tracking::maxXGap,
                                             4u};
  SciFi::Tracking::HitSearchCuts pars_second {SciFi::Tracking::minXHits_2nd,
                                              SciFi::Tracking::maxXWindow_2nd,
                                              SciFi::Tracking::maxXWindowSlope_2nd,
                                              SciFi::Tracking::maxXGap_2nd,
                                              4u};

  int allXHits[2][SciFi::Tracking::max_x_hits];
  int n_x_hits[2] = {0};
  float coordX[2][SciFi::Tracking::max_x_hits];

  if (yAtRef > -5.f)
    collectAllXHits(
      scifi_hits,
      scifi_hit_count,
      allXHits[1],
      n_x_hits[1],
      coordX[1],
      xParams_seed,
      yParams_seed,
      constArrays,
      velo_state,
      qop_ut,
      1);
  if (yAtRef < 5.f)
    collectAllXHits(
      scifi_hits,
      scifi_hit_count,
      allXHits[0],
      n_x_hits[0],
      coordX[0],
      xParams_seed,
      yParams_seed,
      constArrays,
      velo_state,
      qop_ut,
      -1);

  SciFi::Tracking::Track candidate_tracks[SciFi::Tracking::max_candidate_tracks];
  int n_candidate_tracks = 0;
  bool usedHits[2][SciFi::Tracking::max_x_hits] = {false};

  if (yAtRef > -5.f)
    selectXCandidates(
      scifi_hits,
      scifi_hit_count,
      allXHits[1],
      n_x_hits[1],
      usedHits[1],
      coordX[1],
      candidate_tracks,
      n_candidate_tracks,
      zRef_track,
      xParams_seed,
      yParams_seed,
      velo_state,
      pars_first,
      constArrays,
      1,
      false);
  if (yAtRef < 5.f)
    selectXCandidates(
      scifi_hits,
      scifi_hit_count,
      allXHits[0],
      n_x_hits[0],
      usedHits[0],
      coordX[0],
      candidate_tracks,
      n_candidate_tracks,
      zRef_track,
      xParams_seed,
      yParams_seed,
      velo_state,
      pars_first,
      constArrays,
      -1,
      false);

  SciFi::Tracking::Track selected_tracks[SciFi::Tracking::max_selected_tracks];
  int n_selected_tracks = 0;

  selectFullCandidates(
    scifi_hits,
    scifi_hit_count,
    candidate_tracks,
    n_candidate_tracks,
    selected_tracks,
    n_selected_tracks,
    xParams_seed,
    yParams_seed,
    velo_state,
    qop_ut,
    pars_first,
    tmva1,
    tmva2,
    constArrays,
    false);

  bool ok = false;
  for (int i_track = 0; i_track < n_selected_tracks; ++i_track) {
    if (selected_tracks[i_track].hitsNum > 10) ok = true;
  }
  assert(n_selected_tracks < SciFi::Tracking::max_selected_tracks);

  SciFi::Tracking::Track candidate_tracks2[SciFi::Tracking::max_tracks_second_loop];
  int n_candidate_tracks2 = 0;

  if (!ok && SciFi::Tracking::secondLoop) { // If you found nothing begin the 2nd loop
    if (yAtRef > -5.f)
      selectXCandidates(
        scifi_hits,
        scifi_hit_count,
        allXHits[1],
        n_x_hits[1],
        usedHits[1],
        coordX[1],
        candidate_tracks2,
        n_candidate_tracks2,
        zRef_track,
        xParams_seed,
        yParams_seed,
        velo_state,
        pars_second,
        constArrays,
        1,
        true);
    if (yAtRef < 5.f)
      selectXCandidates(
        scifi_hits,
        scifi_hit_count,
        allXHits[0],
        n_x_hits[0],
        usedHits[0],
        coordX[0],
        candidate_tracks2,
        n_candidate_tracks2,
        zRef_track,
        xParams_seed,
        yParams_seed,
        velo_state,
        pars_second,
        constArrays,
        -1,
        true);

    SciFi::Tracking::Track selected_tracks2[SciFi::Tracking::max_tracks_second_loop];
    int n_selected_tracks2 = 0;

    selectFullCandidates(
      scifi_hits,
      scifi_hit_count,
      candidate_tracks2,
      n_candidate_tracks2,
      selected_tracks2,
      n_selected_tracks2,
      xParams_seed,
      yParams_seed,
      velo_state,
      qop_ut,
      pars_second,
      tmva1,
      tmva2,
      constArrays,
      true);

    for (int i_track = 0; i_track < n_selected_tracks2; ++i_track) {
      assert(n_selected_tracks < SciFi::Tracking::max_selected_tracks);
      selected_tracks[n_selected_tracks++] = selected_tracks2[i_track];
    }

    ok = (n_selected_tracks > 0);
  }

  if (ok || !SciFi::Tracking::secondLoop) {

    if (n_selected_tracks > 1) {
      // not using thrust::sort due to temporary_buffer::allocate:: get_temporary_buffer failed" error
      // thrust::sort( thrust::seq, selected_tracks, selected_tracks + n_selected_tracks, lowerByQuality);
      sort_tracks(selected_tracks, n_selected_tracks, [](SciFi::Tracking::Track t1, SciFi::Tracking::Track t2) {
        if (t1.quality < t2.quality) return -1;
        if (t1.quality == t2.quality) return 0;
        return 1;
      });
    }

    const uint event_hit_offset = scifi_hit_count.event_offset();
    float minQuality = SciFi::Tracking::maxQuality;
    for (int i_track = 0; i_track < n_selected_tracks; ++i_track) {
      SciFi::Tracking::Track& track = selected_tracks[i_track];
      if (track.quality + SciFi::Tracking::deltaQuality < minQuality)
        minQuality = track.quality + SciFi::Tracking::deltaQuality;
      if (!(track.quality > minQuality)) {

        SciFi::TrackHits tr = makeTrack(track);
        tr.UTTrackIndex = i_veloUT_track;

        // add LHCbIDs from SciFi part of the track
        for (int i_hit = 0; i_hit < track.hitsNum; ++i_hit) {
          // save local hit index within event to be able to use short
          const int local_hit_index = track.hit_indices[i_hit] - event_hit_offset;
          tr.addHit(local_hit_index);
        }
        assert(tr.hitsNum < SciFi::Constants::max_track_size);

        if (*n_forward_tracks >= SciFi::Constants::max_tracks) printf("n_forward_tracks = %u \n", *n_forward_tracks);
        assert(*n_forward_tracks < SciFi::Constants::max_tracks);
#ifndef __CUDA_ARCH__
        outputTracks[(*n_forward_tracks)++] = tr;
#else
        uint n_tracks = atomicAdd(n_forward_tracks, 1);
        assert(n_tracks < SciFi::Constants::max_tracks);
        outputTracks[n_tracks] = tr;
#endif
      }
    }
  }
}

// Turn SciFi::Tracking::Track into a SciFi::Track
__host__ __device__ SciFi::TrackHits makeTrack(SciFi::Tracking::Track track)
{
  SciFi::TrackHits tr;
  tr.qop = track.qop;
  tr.quality = track.quality;

  // add state at zEndT
  const float z = SciFi::Constants::ZEndT;
  MiniState state(track.x(z), track.y(z), z, track.xSlope(z), track.ySlope(z));

  tr.state = state;

  return tr;
}

//=========================================================================
//  Create Full candidates out of xCandidates
//  Searching for stereo hits
//  Fit of all hits
//  save everything in track candidate folder
//=========================================================================
__host__ __device__ void selectFullCandidates(
  const SciFi::Hits& scifi_hits,
  const SciFi::HitCount& scifi_hit_count,
  SciFi::Tracking::Track* candidate_tracks,
  int& n_candidate_tracks,
  SciFi::Tracking::Track* selected_tracks,
  int& n_selected_tracks,
  const float xParams_seed[4],
  const float yParams_seed[4],
  MiniState velo_state,
  const float VeloUT_qOverP,
  SciFi::Tracking::HitSearchCuts& pars,
  const SciFi::Tracking::TMVA* tmva1,
  const SciFi::Tracking::TMVA* tmva2,
  const SciFi::Tracking::Arrays* constArrays,
  const bool secondLoop)
{

  PlaneCounter planeCounter;
  planeCounter.clear();
  float mlpInput[7] = {0};

  for (int i_track = 0; i_track < n_candidate_tracks; ++i_track) {
    SciFi::Tracking::Track* cand = candidate_tracks + i_track;

    pars.minStereoHits = 4;

    if (cand->hitsNum + pars.minStereoHits < SciFi::Tracking::minTotalHits) {
      pars.minStereoHits = SciFi::Tracking::minTotalHits - cand->hitsNum;
    }

    int stereoHits[SciFi::Tracking::max_stereo_hits];
    int n_stereoHits = 0;
    float stereoCoords[SciFi::Tracking::max_stereo_hits];
    collectStereoHits(
      scifi_hits, scifi_hit_count, *cand, velo_state, pars, constArrays, stereoCoords, stereoHits, n_stereoHits);

    if (n_stereoHits < pars.minStereoHits) continue;

    if (!selectStereoHits(
          scifi_hits, scifi_hit_count, *cand, constArrays, stereoCoords, stereoHits, n_stereoHits, velo_state, pars))
      continue;

    planeCounter.clear();
    for (int i_hit = 0; i_hit < cand->hitsNum; ++i_hit) {
      int hit = cand->hit_indices[i_hit];
      planeCounter.addHit(scifi_hits.planeCode(hit) / 2);
    }

    // make a fit of ALL hits using their x coordinate
    if (!quadraticFitX(scifi_hits, cand->trackParams, cand->hit_indices, cand->hitsNum, planeCounter, pars)) continue;

    // track has enough hits, calcualte quality and save if good enough
    if (planeCounter.nbDifferent >= SciFi::Tracking::minTotalHits) {

      const float qOverP = calcqOverP(cand->trackParams[1], constArrays, velo_state);
      // orig params before fitting , TODO faster if only calc once?? mem usage?
      const float xAtRef = cand->trackParams[0];
      float dSlope = (velo_state.x + (SciFi::Tracking::zReference - velo_state.z) * velo_state.tx - xAtRef) /
                     (SciFi::Tracking::zReference - constArrays->zMagnetParams[0]);
      const float zMagSlope =
        constArrays->zMagnetParams[2] * pow(velo_state.tx, 2) + constArrays->zMagnetParams[3] * pow(velo_state.ty, 2);
      const float zMag = constArrays->zMagnetParams[0] + constArrays->zMagnetParams[1] * dSlope * dSlope + zMagSlope;
      const float xMag = velo_state.x + (zMag - velo_state.z) * velo_state.tx;
      const float slopeT = (xAtRef - xMag) / (SciFi::Tracking::zReference - zMag);
      dSlope = slopeT - velo_state.tx;
      const float dyCoef = dSlope * dSlope * velo_state.ty;

      float bx = slopeT;
      float ay = velo_state.y + (SciFi::Tracking::zReference - velo_state.z) * velo_state.ty;
      float by = velo_state.ty + dyCoef * SciFi::Tracking::byParams;

      // ay,by,bx params
      const float ay1 = cand->trackParams[4];
      const float by1 = cand->trackParams[5];
      const float bx1 = cand->trackParams[1];

      mlpInput[0] = planeCounter.nbDifferent;
      mlpInput[1] = qOverP;
      mlpInput[2] = VeloUT_qOverP - qOverP;                // veloUT - scifi
      if (fabsf(VeloUT_qOverP) < 1e-9f) mlpInput[2] = 0.f; // no momentum estiamte
      mlpInput[3] = pow(velo_state.tx, 2) + pow(velo_state.ty, 2);
      mlpInput[4] = by - by1;
      mlpInput[5] = bx - bx1;
      mlpInput[6] = ay - ay1;

      float quality = 0.f;
      /// WARNING: if the NN classes straight out of TMVA are used, put a mutex here!
      if (pars.minXHits > 4)
        quality = GetMvaValue(mlpInput, tmva1); // 1st loop NN
      else
        quality = GetMvaValue(mlpInput, tmva2); // 2nd loop NN

      quality = 1.f - quality; // backward compability

      if (quality < SciFi::Tracking::maxQuality) {
        cand->quality = quality;
        cand->qop = qOverP;
        if (!secondLoop)
          assert(n_selected_tracks < SciFi::Tracking::max_selected_tracks);
        else if (secondLoop)
          assert(n_selected_tracks < SciFi::Tracking::max_tracks_second_loop);
        selected_tracks[n_selected_tracks++] = *cand;
        if (!secondLoop) {
          if (n_selected_tracks >= SciFi::Tracking::max_selected_tracks) break;
        }
        else if (secondLoop) {
          if (n_selected_tracks >= SciFi::Tracking::max_tracks_second_loop) break;
        }
      }
    }
  }
}
