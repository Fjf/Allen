#include "hip/hip_runtime.h"
#include "LinearFitting.cuh"

/**
   Functions related to fitting a straight line
 */

__host__ __device__ float getLineFitDistance(
  SciFi::Tracking::LineFitterPars &parameters,
  const SciFi::SciFiHits& scifi_hits,
  float coordX[SciFi::Tracking::max_x_hits],
  int allXHits[SciFi::Tracking::max_x_hits],
  int it )
{ 
  return coordX[it] - (parameters.m_c0 + (scifi_hits.z0[ allXHits[it] ] - parameters.m_z0) * parameters.m_tc);
}

__host__ __device__ float getLineFitChi2(
  SciFi::Tracking::LineFitterPars &parameters,
  const SciFi::SciFiHits& scifi_hits,
  float coordX[SciFi::Tracking::max_x_hits],
  int allXHits[SciFi::Tracking::max_x_hits],
  int it) {
  float d = getLineFitDistance( parameters, scifi_hits, coordX, allXHits, it ); 
  return d * d * coordX[it]; 
}

__host__ __device__ void solveLineFit(SciFi::Tracking::LineFitterPars &parameters)  {
  float den = (parameters.m_sz*parameters.m_sz-parameters.m_s0*parameters.m_sz2);
  parameters.m_c0  = (parameters.m_scz * parameters.m_sz - parameters.m_sc * parameters.m_sz2) / den;
  parameters.m_tc  = (parameters.m_sc *  parameters.m_sz - parameters.m_s0 * parameters.m_scz) / den;
}

__host__ __device__ void incrementLineFitParameters(
  SciFi::Tracking::LineFitterPars &parameters,
  const SciFi::SciFiHits& scifi_hits,
  const float coordX[SciFi::Tracking::max_x_hits],
  const int allXHits[SciFi::Tracking::max_x_hits],
  const int it)
{
    float c = coordX[it];
    const int hit = allXHits[it];
    float w = scifi_hits.w[hit];
    float z = scifi_hits.z0[hit] - parameters.m_z0;
    parameters.m_s0   += w;
    parameters.m_sz   += w * z;
    parameters.m_sz2  += w * z * z;
    parameters.m_sc   += w * c;
    parameters.m_scz  += w * c * z;
} 

__host__ __device__ void fitHitsFromSingleHitPlanes(
  const int it1,
  const int it2,
  const bool usedHits[SciFi::Tracking::max_x_hits],
  const SciFi::SciFiHits& scifi_hits,
  const int allXHits[SciFi::Tracking::max_x_hits],
  const int n_x_hits,
  const PlaneCounter planeCounter,
  SciFi::Tracking::LineFitterPars& lineFitParameters,
  const float coordX[SciFi::Tracking::max_x_hits],
  int otherHits[SciFi::Constants::n_layers][SciFi::Tracking::max_other_hits],
  int nOtherHits[SciFi::Constants::n_layers]) {

  for(auto itH = it1; it2 > itH; ++itH ){
    assert( itH < n_x_hits );
    if( usedHits[itH] ) continue;
    int planeCode = scifi_hits.planeCode[allXHits[itH]]/2;
    if( planeCounter.nbInPlane(planeCode) == 1 ){
      incrementLineFitParameters(lineFitParameters, scifi_hits, coordX, allXHits, itH);
    }else{
      if ( nOtherHits[planeCode] < SciFi::Tracking::max_other_hits ) {
        assert( nOtherHits[planeCode] < SciFi::Tracking::max_other_hits );
        otherHits[planeCode][ nOtherHits[planeCode]++ ] = itH;
      }
    }
  }
  solveLineFit(lineFitParameters);

}

__host__ __device__ void fastLinearFit(
  const SciFi::SciFiHits& scifi_hits,
  float trackParameters[SciFi::Tracking::nTrackParams], 
  int coordToFit[SciFi::Tracking::max_coordToFit],
  int& n_coordToFit,
  PlaneCounter planeCounter,
  SciFi::Tracking::HitSearchCuts& pars)
{
  bool fit = true;
  while (fit) {
    //== Fit a line
    float s0   = 0.;
    float sz   = 0.;
    float sz2  = 0.;
    float sd   = 0.;
    float sdz  = 0.;

    for ( int i_hit = 0; i_hit < n_coordToFit; ++i_hit ) {
      int hit = coordToFit[i_hit];
      const float parsX[4] = {trackParameters[0],
                              trackParameters[1],
                              trackParameters[2],
                              trackParameters[3]};
      const float zHit = scifi_hits.z0[hit];
      float track_x_at_zHit = straightLineExtend(parsX,zHit);
      const float d = scifi_hits.x0[hit] - track_x_at_zHit;
      const float w = scifi_hits.w[hit];
      const float z = zHit - SciFi::Tracking::zReference;
      s0   += w;
      sz   += w * z; 
      sz2  += w * z * z; 
      sd   += w * d; 
      sdz  += w * d * z; 
    }    
    float den = (sz*sz-s0*sz2);
    if( !(fabsf(den) > 1e-5))return;
    const float da  = (sdz * sz - sd * sz2) / den; 
    const float db  = (sd *  sz - s0 * sdz) / den; 
    trackParameters[0] += da;
    trackParameters[1] += db;
    fit = false;

    if ( n_coordToFit < pars.minXHits ) return;

    int worst = n_coordToFit;
    float maxChi2 = 0.f; 
    const bool notMultiple = planeCounter.nbDifferent == n_coordToFit;
    //TODO how many multiple hits do we normaly have?
    //how often do we do the right thing here?
    //delete two hits at same time?
    for ( int i_hit = 0; i_hit < n_coordToFit; ++i_hit ) {
      int hit = coordToFit[i_hit];
      // This could certainly be wrapped in some helper function with a lot
      // of passing around or copying etc... 
      const float parsX[4] = {trackParameters[0],
                              trackParameters[1],
                              trackParameters[2],
                              trackParameters[3]};
      const float chi2 = chi2XHit( parsX, scifi_hits, hit );
      if ( chi2 > maxChi2 && ( notMultiple || planeCounter.nbInPlane( scifi_hits.planeCode[hit]/2 ) > 1 ) ) {
        maxChi2 = chi2;
        worst   = i_hit; 
      }    
    }    
    if ( maxChi2 > SciFi::Tracking::maxChi2LinearFit || ( !notMultiple && maxChi2 > 4.f ) ) {
      removeOutlier( scifi_hits, planeCounter, coordToFit, n_coordToFit, coordToFit[worst] );
      fit = true;
    }
  }
}
 
