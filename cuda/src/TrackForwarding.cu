#include "hip/hip_runtime.h"
#include "SearchByTriplet.cuh"

/**
 * @brief Fits hits to tracks.
 * 
 * @details In case the tolerances constraints are met,
 *          returns the chi2 weight of the track. Otherwise,
 *          returns FLT_MAX.
 */
__device__ float fitHitToTrack(
  const Hit& h0,
  const Hit& h2,
  const float predx,
  const float predy,
  const float scatterDenom2
) {
  // tolerances
  const float x_prediction = h0.x + predx;
  const float dx = fabs(x_prediction - h2.x);
  const bool tolx_condition = dx < TOLERANCE;

  const float y_prediction = h0.y + predy;
  const float dy = fabs(y_prediction - h2.y);
  const bool toly_condition = dy < TOLERANCE;

  // Scatter
  const float scatterNum = (dx * dx) + (dy * dy);
  const float scatter = scatterNum * scatterDenom2;

  const bool scatter_condition = scatter < MAX_SCATTER_FORWARDING;
  const bool condition = tolx_condition && toly_condition && scatter_condition;

  return condition * scatter + !condition * FLT_MAX;
}

/**
 * @brief Performs the track forwarding of forming tracks
 */
__device__ void trackForwarding(
  const float* hit_Xs,
  const float* hit_Ys,
  const float* hit_Zs,
  bool* hit_used,
  unsigned int* tracks_insertPointer,
  unsigned int* ttf_insertPointer,
  unsigned int* weaktracks_insertPointer,
  const Module* module_data,
  const unsigned int diff_ttf,
  unsigned int* tracks_to_follow,
  unsigned int* weak_tracks,
  const unsigned int prev_ttf,
  Track* tracklets,
  Track* tracks,
  const unsigned int number_of_hits,
  const unsigned int first_module,
  const float* module_Zs,
  const unsigned int* module_hitStarts,
  const unsigned int* module_hitNums
) {
  // Assign a track to follow to each thread
  for (int i=0; i<(diff_ttf + blockDim.x - 1) / blockDim.x; ++i) {
    const unsigned int ttf_element = blockDim.x * i + threadIdx.x;
    if (ttf_element < diff_ttf) {
      const auto fulltrackno = tracks_to_follow[(prev_ttf + ttf_element) % TTF_MODULO];
      const bool track_flag = (fulltrackno & 0x80000000) == 0x80000000;
      const auto skipped_modules = (fulltrackno & 0x70000000) >> 28;
      auto trackno = fulltrackno & 0x0FFFFFFF;

      const Track* track_pointer = track_flag ? tracklets : tracks;
      
      ASSERT(track_pointer==tracklets ? trackno < number_of_hits : true)
      ASSERT(track_pointer==tracks ? trackno < MAX_TRACKS : true)
      auto t = track_pointer[trackno];

      // Load last two hits in h0, h1
      ASSERT(t.hitsNum < MAX_TRACK_SIZE)
      const auto h0_num = t.hits[t.hitsNum - 2];
      const auto h1_num = t.hits[t.hitsNum - 1];

      ASSERT(h0_num < number_of_hits)
      const Hit h0 {hit_Xs[h0_num], hit_Ys[h0_num]};
      const auto h0_z = hit_Zs[h0_num];

      ASSERT(h1_num < number_of_hits)
      const Hit h1 {hit_Xs[h1_num], hit_Ys[h1_num]};
      const auto h1_z = hit_Zs[h1_num];

      // Track forwarding over t, for all hits in the next module
      // Line calculations
      const auto td = 1.0f / (h1_z - h0_z);
      const auto txn = (h1.x - h0.x);
      const auto tyn = (h1.y - h0.y);
      const auto tx = txn * td;
      const auto ty = tyn * td;

      // Find the best candidate
      float best_fit = FLT_MAX;
      unsigned short best_h2;

      // Some constants of fitting
      const auto h2_z = module_data[2].z;
      const auto dz = h2_z - h0_z;
      const auto predx = tx * dz;
      const auto predy = ty * dz;
      const auto scatterDenom2 = 1.f / ((h2_z - h1_z) * (h2_z - h1_z));

      for (auto j=0; j<module_data[2].hitNums; ++j) {
        const auto h2_index = module_data[2].hitStart + j;
        const Hit h2 {hit_Xs[h2_index], hit_Ys[h2_index]};
        const auto fit = fitHitToTrack(
          h0,
          h2,
          predx,
          predy,
          scatterDenom2
        );
        const auto fit_is_better = fit < best_fit;
        best_fit = fit_is_better*fit + !fit_is_better*best_fit;
        best_h2 = fit_is_better*h2_index + !fit_is_better*best_h2;
      }

      // Condition for finding a h2
      if (best_fit != FLT_MAX) {
        // Mark h2 as used
        ASSERT(best_h2 < number_of_hits)
        hit_used[best_h2] = true;

        // Update the tracks to follow, we'll have to follow up
        // this track on the next iteration :)
        ASSERT(t.hitsNum < MAX_TRACK_SIZE)
        t.hits[t.hitsNum++] = best_h2;

        // Update the track in the bag
        if (t.hitsNum <= 4) {
          ASSERT(t.hits[0] < number_of_hits)
          ASSERT(t.hits[1] < number_of_hits)
          ASSERT(t.hits[2] < number_of_hits)

          // Also mark the first three as used
          hit_used[t.hits[0]] = true;
          hit_used[t.hits[1]] = true;
          hit_used[t.hits[2]] = true;

          // If it is a track made out of less than or equal than 4 hits,
          // we have to allocate it in the tracks pointer
          trackno = atomicAdd(tracks_insertPointer, 1);
        }

        // Copy the track into tracks
        ASSERT(trackno < number_of_hits)
        tracks[trackno] = t;

        // Add the tracks to the bag of tracks to_follow
        const auto ttfP = atomicAdd(ttf_insertPointer, 1) % TTF_MODULO;
        tracks_to_follow[ttfP] = trackno;
      }
      // A track just skipped a module
      // We keep it for another round
      else if (skipped_modules <= MAX_SKIPPED_MODULES) {
        // Form the new mask
        trackno = ((skipped_modules + 1) << 28) | (fulltrackno & 0x8FFFFFFF);

        // Add the tracks to the bag of tracks to_follow
        const auto ttfP = atomicAdd(ttf_insertPointer, 1) % TTF_MODULO;
        tracks_to_follow[ttfP] = trackno;
      }
      // If there are only three hits in this track,
      // mark it as "doubtful"
      else if (t.hitsNum == 3) {
        const auto weakP = atomicAdd(weaktracks_insertPointer, 1);
        ASSERT(weakP < number_of_hits)
        weak_tracks[weakP] = trackno;
      }
      // In the "else" case, we couldn't follow up the track,
      // so we won't be track following it anymore.
    }
  }
}
