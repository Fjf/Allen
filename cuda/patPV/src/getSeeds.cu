#include "hip/hip_runtime.h"
#include "getSeeds.cuh"
//simplficiations: no tracks2disable

// steering parameters for merging procedure
__constant__ double mcu_maxChi2Merge = 25.;
__constant__ double mcu_factorToIncreaseErrors = 15.;

//try parameters from RecoUpgradeTracking.py
__constant__ int    mcu_minClusterMult = 4;
__constant__ int    mcu_minCloseTracksInCluster = 3;


// steering parameters for final cluster selection
// int    m_minClusterMult = 3;
__constant__ double mcu_dzCloseTracksInCluster = 5.; // unit: mm
// int    m_minCloseTracksInCluster = 3;
__constant__ int    mcu_highMult = 10;
__constant__ double mcu_ratioSig2HighMult = 1.0;
__constant__ double mcu_ratioSig2LowMult = 0.9;

__constant__ int mcu_max_clusters = 200; // maximmum number of clusters

__constant__ double mcu_x0MS = 0.01;// X0 (tunable) of MS to add for extrapolation of
                                                       // track parameters to PV

//don't forget to actually calculate this!!
//double  m_scatCons = 0;     // calculated from m_x0MS
__constant__ double X0cu = 0.01;
//__constant__ double m_scatCons = (13.6*sqrt(X0)*(1.+0.038*log(X0)));
__constant__ double mcu_scatCons = 0.01;


__device__ double zCloseBeam( Velo::State track, const XYZPoint& beamspot) {

  XYZPoint tpoint(track.x, track.y, track.z);
  XYZPoint tdir(track.tx, track.ty, 1.);

  double wx = ( 1. + tdir.x * tdir.x ) / track.c00;
  double wy = ( 1. + tdir.y * tdir.y ) / track.c11;

  double x0 = tpoint.x - tpoint.z * tdir.x - beamspot.x;
  double y0 = tpoint.y - tpoint.z * tdir.y - beamspot.y;
  double den = wx * tdir.x * tdir.x + wy * tdir.y * tdir.y;
  double zAtBeam = - ( wx * x0 * tdir.x + wy * y0 * tdir.y ) / den ;

  double xb = tpoint.x + tdir.x * ( zAtBeam - tpoint.z ) - beamspot.x;
  double yb = tpoint.y + tdir.y * ( zAtBeam - tpoint.z ) - beamspot.y;
  double r2AtBeam = xb*xb + yb*yb ;

  return r2AtBeam < 0.5*0.5 ? zAtBeam : 10e8;
}



__device__ void errorForPVSeedFinding(double tx, double ty, double &sigz2)  {

    // the seeding results depend weakly on this eror parametrization

    double pMean = 3000.; // unit: MeV

    double tanTheta2 =  tx * tx + ty * ty;
    double sinTheta2 =  tanTheta2 / ( 1. + tanTheta2 );

    // assume that first hit in VD at 8 mm
    double distr        = 8.; // unit: mm
    double dist2        = distr*distr/sinTheta2;
    double sigma_ms2    = mcu_scatCons * mcu_scatCons * dist2 / (pMean*pMean);
    double fslope2      = 0.0005*0.0005;
    double sigma_slope2 = fslope2*dist2;

    sigz2 = (sigma_ms2 + sigma_slope2) / sinTheta2;
    if(sigz2 == 0) sigz2 = 100.;

}




 __global__ void getSeeds(
  uint* dev_kalmanvelo_states,
  int * dev_atomics_storage,
  uint* dev_velo_track_hit_number,
  XYZPoint * dev_seeds,
  uint * dev_number_seed) {

  XYZPoint beamspot;
  beamspot.x = 0;
  beamspot.y = 0;
  beamspot.z = 0;

  const uint number_of_events = gridDim.x;
  const uint event_number = blockIdx.x;

  const Velo::Consolidated::Tracks velo_tracks {(uint*) dev_atomics_storage, dev_velo_track_hit_number, event_number, number_of_events};
  const Velo::Consolidated::States velo_states {dev_kalmanvelo_states, velo_tracks.total_number_of_tracks};
  const uint number_of_tracks_event = velo_tracks.number_of_tracks(event_number);
  const uint event_tracks_offset = velo_tracks.tracks_offset(event_number);

  



  

    vtxCluster  vclusters[VeloTracking::max_tracks];



  int counter_number_of_clusters = 0;
  for (int i = 0; i < number_of_tracks_event; i++) {

    
    double sigsq;
    double zclu;
    Velo::State trk = velo_states.get(event_tracks_offset + i);


    zclu = zCloseBeam(trk,beamspot);
    errorForPVSeedFinding(trk.tx, trk.ty,sigsq);

    if ( fabs(zclu)>2000.) continue;
    vtxCluster clu;
    clu.z = zclu;
    clu.sigsq = sigsq;
    clu.sigsqmin = clu.sigsq;
    clu.ntracks = 1;
    vclusters[counter_number_of_clusters] = clu;
    
    counter_number_of_clusters++;

  }

  double  zseeds[VeloTracking::max_tracks];

  int number_final_clusters = findClusters(vclusters, zseeds, counter_number_of_clusters);

  for(int i = 0; i < number_final_clusters; i++) dev_seeds[event_number * PatPV::max_number_vertices + i] = XYZPoint{ beamspot.x, beamspot.y, zseeds[i]};
  

  dev_number_seed[event_number] = number_final_clusters;


 };

 __device__ int findClusters(vtxCluster * vclus, double * zclusters, int number_of_clusters)  {


  
  

  
  for(int i = 0; i < number_of_clusters; i++) {
    vclus[i].sigsq *= mcu_factorToIncreaseErrors*mcu_factorToIncreaseErrors; // blow up errors
    vclus[i].sigsqmin = vclus[i].sigsq;
  }


  //maybe sort in z before merging? -> does not seem to help

  bool no_merges = false;
  while(!no_merges) {
    //reset merged flags
    for (int j = 0; j < number_of_clusters; j++) vclus[j].merged = false;

    no_merges = true;
    for(int index_cluster = 0; index_cluster < number_of_clusters - 1; index_cluster++) {


      //skip cluster which have already been merged
      if(vclus[index_cluster].ntracks == 0) continue;

   //sorting by chi2dist seems to increase efficiency in nominal code


      
      for(int index_second_cluster = 0; index_second_cluster < number_of_clusters ; index_second_cluster++) {
        if(vclus[index_second_cluster].merged || vclus[index_cluster].merged) continue;
        //skip cluster which have already been merged
        if(vclus[index_second_cluster].ntracks == 0) continue;
        if(index_cluster == index_second_cluster) continue;
        double z1 = vclus[index_cluster].z;
        double z2 = vclus[index_second_cluster].z;
        double s1 = vclus[index_cluster].sigsq;
        double s2 = vclus[index_second_cluster].sigsq;
        double s1min = vclus[index_cluster].sigsqmin;
        double s2min = vclus[index_second_cluster].sigsqmin;
        double sigsqmin = s1min;
        if(s2min<s1min) sigsqmin = s2min;


        double zdist = z1 - z2;
        double chi2dist = zdist*zdist/(s1+s2);
        //merge if chi2dist is smaller than max
        if (chi2dist<mcu_maxChi2Merge ) {
          no_merges = false;
          double w_inv = (s1*s2/(s1+s2));
          double zmerge = w_inv*(z1/s1+z2/s2);

          vclus[index_cluster].z        = zmerge;
          vclus[index_cluster].sigsq    = w_inv;
          vclus[index_cluster].sigsqmin = sigsqmin;
          vclus[index_cluster].ntracks += vclus[index_second_cluster].ntracks;
          vclus[index_second_cluster].ntracks = 0;  // mark second cluster as used
          vclus[index_cluster].merged = true;
          vclus[index_second_cluster].merged = true;

          //break;
        } 
      }
    }
  }
 
  

  int return_number_of_clusters = 0;
  //count final number of clusters
  vtxCluster pvclus[VeloTracking::max_tracks];
  for(int i = 0; i < number_of_clusters; i++) {
    if(vclus[i].ntracks != 0)     {pvclus[return_number_of_clusters] = vclus[i]; return_number_of_clusters++;}
  } 


  //clean up clusters, do we gain much from this?

  // Select good clusters.

  int number_good_clusters = 0;

  for(int index = 0; index < return_number_of_clusters; index++) {

    int n_tracks_close = 0;
    for(int i = 0; i < number_of_clusters; i++) if(fabs(vclus[i].z - pvclus[index].z ) < mcu_dzCloseTracksInCluster ) n_tracks_close++;
  

    double dist_to_closest = 1000000.;
    if(return_number_of_clusters > 1) {
      for(int index2 = 0; index2 < return_number_of_clusters; index2++) {
        if( index!=index2 && ( fabs( pvclus[index2].z - pvclus[index].z) < dist_to_closest) )  dist_to_closest = fabs( pvclus[index2].z - pvclus[index].z);
      }
    }

    // ratio to remove clusters made of one low error track and many large error ones
    double rat = pvclus[index].sigsq/pvclus[index].sigsqmin;
    bool igood = false;
    int ntracks = pvclus[index].ntracks;
    if( ntracks >= mcu_minClusterMult ) {
      if( dist_to_closest>10. && rat<0.95) igood=true;
      if( ntracks >= mcu_highMult && rat < mcu_ratioSig2HighMult)  igood=true;
      if( ntracks <  mcu_highMult && rat < mcu_ratioSig2LowMult )  igood=true;
    }
    // veto
    if( n_tracks_close < mcu_minCloseTracksInCluster ) igood = false;
    if(igood) {zclusters[number_good_clusters] = pvclus[index].z; number_good_clusters++;}


  }



  //return return_number_of_clusters;
  return number_good_clusters;

}