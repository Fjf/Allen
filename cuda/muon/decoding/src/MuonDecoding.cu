#include "hip/hip_runtime.h"
#include "MuonDecoding.cuh"
#include <stdio.h>

using namespace Muon;

__global__ void muon_decoding(char* events, unsigned int* offsets, Muon::MuonRawToHits* muon_raw_to_hits,
    Muon::HitsSoA* unordered_muon_hits, Muon::HitsSoA* muon_hits) {
  __shared__ int currentHitIndex;
  size_t eventId = blockIdx.x;
  if (eventId != 1) {
    return;
  }
  //printf("blockIdx.x = %u\n", blockIdx.x);

  size_t station = threadIdx.x / (Muon::Constants::n_regions * Muon::Constants::n_quarters);
  size_t region = (threadIdx.x % (Muon::Constants::n_regions * Muon::Constants::n_quarters)) / Muon::Constants::n_regions;
  size_t quarter = threadIdx.x % Muon::Constants::n_quarters;
  //printf("threadIdx.x = %u\n, ", threadIdx.x);
  //printf("station = %u, ", station);
  //printf("region = %u, ", region);
  //printf("quarter = %u\n", quarter);
  __shared__ unsigned int storageTileId[Constants::max_numhits_per_event];
  __shared__ unsigned int storageTdcValue[Constants::max_numhits_per_event];
  __shared__ unsigned int sortedStorageTileId[Constants::max_numhits_per_event];
  __shared__ unsigned int sortedStorageTdcValue[Constants::max_numhits_per_event];
  __shared__ Digit digits[Constants::max_numhits_per_event];
  __shared__ int currentStorageIndex;
  __shared__ int storageStationRegionQuarterOccurrences[Muon::Constants::n_stations * Muon::Constants::n_regions * Muon::Constants::n_quarters];
  __shared__ int storageStationRegionQuarterOccurrencesOffset[Muon::Constants::n_stations * Muon::Constants::n_regions * Muon::Constants::n_quarters + 1];
  __shared__ int originalStorageStationRegionQuarterOccurrencesOffset[Muon::Constants::n_stations * Muon::Constants::n_regions * Muon::Constants::n_quarters + 1];
  __shared__ bool used[Constants::max_numhits_per_event];
  __shared__ int stationOccurrences[Muon::Constants::n_stations];
  __shared__ int stationOccurrencesOffset[Muon::Constants::n_stations + 1];

  if (threadIdx.x == 0) {
    currentHitIndex = 0;
    currentStorageIndex = 0;
    memset(storageStationRegionQuarterOccurrences, 0, sizeof(storageStationRegionQuarterOccurrences));
    storageStationRegionQuarterOccurrencesOffset[0] = 0;
    originalStorageStationRegionQuarterOccurrencesOffset[0] = 0;
    memset(used, false, sizeof(used));
    memset(stationOccurrences, 0, sizeof(stationOccurrences));
  }
  __syncthreads();

  if (region == 0 && quarter == 0) {
    MuonRawEvent rawEvent = MuonRawEvent((const char*) events + offsets[eventId]);
    for (uint32_t bank_index = 0; bank_index < rawEvent.number_of_raw_banks; bank_index++) {
      unsigned int tell1Number = rawEvent.getMuonBank(bank_index).sourceID;
      size_t stationByBankNumber = (tell1Number < 4 ? 0 : tell1Number < 6 ? 1 : tell1Number < 8 ? 2 : 3);
      if (stationByBankNumber != station) {
        continue;
      }
      MuonRawBank rawBank = rawEvent.getMuonBank(bank_index);
      uint16_t* p = rawBank.data;
      int preamble_size = 2 * ((* p + 3) / 2);
      p += preamble_size;
      for (size_t i = 0; i < 4; i++) {
        uint16_t frontValue = * p;
        for (size_t shift = 1; shift < 1 + frontValue; shift++) {
          unsigned int pp = * (p + shift);
          unsigned int add = (pp & 0x0FFF);
          unsigned int tdc_value = ((pp & 0xF000) >> 12);
          unsigned int tileId = muon_raw_to_hits->muonGeometry.getADDInTell1(tell1Number, add);
          //printf("tileId = %u\n", tileId);
          if (tileId != 0) {
            //TODO атомарное присвоение
            //int localCurrentStorageIndex = currentStorageIndex;
            //atomicAdd(&currentStorageIndex, 1);
            //printf("currentStorageIndex = %u\n", currentStorageIndex);
            int localCurrentStorageIndex = atomicAdd(&currentStorageIndex, 1);
            storageTileId[localCurrentStorageIndex] = tileId;
            storageTdcValue[localCurrentStorageIndex] = tdc_value;
          }
        }
        p += 1 + frontValue;
      }
    }
  }
  __syncthreads();
  if (threadIdx.x == 0) {
    printf("currentStorageIndex = %d\n", currentStorageIndex);
    for (size_t i = 0; i < currentStorageIndex; i++) {
      size_t stationRegionQuarter = Muon::MuonTileID::stationRegionQuarter(storageTileId[i]);
      storageStationRegionQuarterOccurrences[stationRegionQuarter]++;
    }
    for (size_t i = 0; i < Muon::Constants::n_stations * Muon::Constants::n_regions * Muon::Constants::n_quarters; i++) {
      storageStationRegionQuarterOccurrencesOffset[i + 1] =
          storageStationRegionQuarterOccurrencesOffset[i] + storageStationRegionQuarterOccurrences[i];
      originalStorageStationRegionQuarterOccurrencesOffset[i + 1] = storageStationRegionQuarterOccurrencesOffset[i + 1];
    }
    for (size_t i = 0; i < currentStorageIndex; i++) {
      size_t stationRegionQuarter = Muon::MuonTileID::stationRegionQuarter(storageTileId[i]);
      size_t index = storageStationRegionQuarterOccurrencesOffset[stationRegionQuarter];
      storageStationRegionQuarterOccurrencesOffset[stationRegionQuarter]++;
      sortedStorageTileId[index] = storageTileId[i];
      sortedStorageTdcValue[index] = storageTdcValue[i];
    }
  }
  __syncthreads();
  //TODO проверить, нормально ли всё посчиталось
  //printf("OFFSET = %d\n", storageStationRegionQuarterOccurrences[threadIdx.x]*10000000 + originalStorageStationRegionQuarterOccurrencesOffset[threadIdx.x] * 10000 + station*100 + region*10+quarter);
  //return;
  //originalStorageStationRegionQuarterOccurrencesOffset убедиться что они норм
  muon_raw_to_hits->addCoordsCrossingMap(
      sortedStorageTileId,
      sortedStorageTdcValue,
      used,
      originalStorageStationRegionQuarterOccurrencesOffset[threadIdx.x],
      originalStorageStationRegionQuarterOccurrencesOffset[threadIdx.x + 1],
      unordered_muon_hits,
      currentHitIndex
  );
  __syncthreads();
  //return;
  if (threadIdx.x == 0) {
    printf("currentHitIndex = %d\n", currentHitIndex);
    for (size_t i = 0; i < currentHitIndex; i++) {
      size_t currentStation = Muon::MuonTileID::station(unordered_muon_hits->tile[i]);
      stationOccurrences[currentStation]++;
    }
    for (size_t i = 0; i < Muon::Constants::n_stations; i++) {
      stationOccurrencesOffset[i + 1] = stationOccurrencesOffset[i] + stationOccurrences[i];
    }
    for (size_t i = 0; i < Muon::Constants::n_stations; i++) {
      muon_hits->station_offsets[i] = stationOccurrencesOffset[i];
      muon_hits->number_of_hits_per_station[i] = stationOccurrences[i];
    }

    //можно распаллелить по станциям
    for (size_t i = 0; i < currentStorageIndex; i++) {
      size_t currentStation = Muon::MuonTileID::station(unordered_muon_hits->tile[i]);
      size_t index = stationOccurrencesOffset[currentStation];
      stationOccurrencesOffset[currentStation]++;
      Muon::setAtIndex(
          muon_hits,
          index,
          unordered_muon_hits->tile[i],
          unordered_muon_hits->x[i],
          unordered_muon_hits->dx[i],
          unordered_muon_hits->y[i],
          unordered_muon_hits->dy[i],
          unordered_muon_hits->z[i],
          unordered_muon_hits->dz[i],
          unordered_muon_hits->uncrossed[i],
          unordered_muon_hits->time[i],
          unordered_muon_hits->delta_time[i],
          unordered_muon_hits->cluster_size[i],
          unordered_muon_hits->region_id[i]
      );
    }
  }
  __syncthreads();
}