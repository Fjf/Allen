#include "hip/hip_runtime.h"
#include "MuonSortByStation.cuh"

__global__ void muon_sort_by_station::muon_sort_by_station(muon_sort_by_station::Parameters parameters)
{
  const auto number_of_events = gridDim.x;
  const auto event_number = blockIdx.x;
  const auto number_of_hits = parameters.dev_atomics_muon[number_of_events + event_number];
  const auto station_ocurrences_offset =
    parameters.dev_station_ocurrences_offset + event_number * Muon::Constants::n_stations;
  const auto storage_tile_id = parameters.dev_storage_tile_id + event_number * Muon::Constants::max_numhits_per_event;
  const auto storage_tdc_value =
    parameters.dev_storage_tdc_value + event_number * Muon::Constants::max_numhits_per_event;
  const auto muon_compact_hit = parameters.dev_muon_compact_hit + event_number * Muon::Constants::max_numhits_per_event;
  auto permutation_station = parameters.dev_permutation_station.get() + event_number * Muon::Constants::max_numhits_per_event;
  auto event_muon_hits = parameters.dev_muon_hits.get() + event_number;

  // Populate number of hits per station and offsets
  // TODO: There should be no need to re-populate this
  for (uint i = threadIdx.x; i < Muon::Constants::n_stations; i += blockDim.x) {
    event_muon_hits->station_offsets[i] = station_ocurrences_offset[i];
    event_muon_hits->number_of_hits_per_station[i] = station_ocurrences_offset[i + 1] - station_ocurrences_offset[i];
  }

  // Create a permutation according to Muon::MuonTileID::stationRegionQuarter
  const auto get_station = [&muon_compact_hit](const uint a, const uint b) {
    const auto muon_compact_hit_a = muon_compact_hit[a] & 0xF;
    const auto muon_compact_hit_b = muon_compact_hit[b] & 0xF;

    return (muon_compact_hit_a > muon_compact_hit_b) - (muon_compact_hit_a < muon_compact_hit_b);
  };

  find_permutation(0, 0, number_of_hits, permutation_station, get_station);

  __syncthreads();

  // Do actual decoding
  for (uint i = threadIdx.x; i < number_of_hits; i += blockDim.x) {
    const uint64_t compact_hit = muon_compact_hit[permutation_station[i]];

    const uint8_t uncrossed = compact_hit >> 63;
    const uint digitsOneIndex_index = (compact_hit >> 48) & 0x7FFF;
    const uint digitsTwoIndex = (compact_hit >> 32) & 0xFFFF;
    const uint thisGridX = (compact_hit >> 18) & 0x3FFF;
    const uint otherGridY_condition = (compact_hit >> 4) & 0x3FFF;

    float x = 0.f;
    float dx = 0.f;
    float y = 0.f;
    float dy = 0.f;
    float z = 0.f;
    float dz = 0.f;
    int delta_time;
    int id;
    int region;

    if (!uncrossed) {
      Muon::MuonTileID padTile(storage_tile_id[digitsOneIndex_index]);
      padTile.setY(Muon::MuonTileID::nY(storage_tile_id[digitsTwoIndex]));
      padTile.setLayout(Muon::MuonLayout(thisGridX, otherGridY_condition));

      Muon::calcTilePos(parameters.dev_muon_raw_to_hits.get()->muonTables, padTile, x, dx, y, dy, z);
      region = padTile.region();
      id = padTile.id();
      delta_time = storage_tdc_value[digitsOneIndex_index] - storage_tdc_value[digitsTwoIndex];
    }
    else {
      const auto tile = Muon::MuonTileID(storage_tile_id[digitsOneIndex_index]);
      region = tile.region();
      if (otherGridY_condition == 0) {
        calcTilePos(parameters.dev_muon_raw_to_hits.get()->muonTables, tile, x, dx, y, dy, z);
      }
      else if (otherGridY_condition == 1) {
        calcStripXPos(parameters.dev_muon_raw_to_hits.get()->muonTables, tile, x, dx, y, dy, z);
      }
      else {
        calcStripYPos(parameters.dev_muon_raw_to_hits.get()->muonTables, tile, x, dx, y, dy, z);
      }
      id = tile.id();
      delta_time = storage_tdc_value[digitsOneIndex_index];
    }

    setAtIndex(
      event_muon_hits,
      i,
      id,
      x,
      dx,
      y,
      dy,
      z,
      dz,
      uncrossed,
      storage_tdc_value[digitsOneIndex_index],
      delta_time,
      0,
      region);
  }
}
