#include "hip/hip_runtime.h"
#include "MuonAddCoordsCrossingMaps.cuh"

__global__ void muon_add_coords_crossing_maps(
  uint* dev_storage_station_region_quarter_offsets,
  uint* dev_storage_tile_id,
  uint* dev_storage_tdc_value,
  uint* dev_atomics_muon,
  uint* dev_permutation_srq,
  Muon::MuonRawToHits* muon_raw_to_hits,
  Muon::HitsSoA* muon_hits)
{
  const auto number_of_events = gridDim.x;
  const auto event_number = blockIdx.x;

  auto event_muon_hits = muon_hits + event_number;
  auto storage_tile_id = dev_storage_tile_id + event_number * Muon::Constants::max_numhits_per_event;
  auto storage_tdc_value = dev_storage_tdc_value + event_number * Muon::Constants::max_numhits_per_event;
  auto current_hit_index = dev_atomics_muon + number_of_events + event_number;
  auto storage_station_region_quarter_offsets =
    dev_storage_station_region_quarter_offsets +
    event_number * Muon::Constants::n_stations * Muon::Constants::n_regions * Muon::Constants::n_quarters;
  const auto base_offset = storage_station_region_quarter_offsets[0];

  for (int i = threadIdx.x; i < Muon::Constants::n_stations * Muon::Constants::n_regions * Muon::Constants::n_quarters;
       i += blockDim.x) {

    const auto start_index = storage_station_region_quarter_offsets[i] - base_offset;
    const auto end_index = storage_station_region_quarter_offsets[i + 1] - base_offset;

    // TODO: We are fetching the first tile ID
    //       We should verify this logic holds (it does not atm)
    const auto tile = Muon::MuonTileID(storage_tile_id[start_index]);
    const auto station = tile.station();
    const auto region = tile.region();

    const auto x1 = getLayoutX(muon_raw_to_hits->muonTables, Muon::MuonTables::stripXTableNumber, station, region);
    const auto y1 = getLayoutY(muon_raw_to_hits->muonTables, Muon::MuonTables::stripXTableNumber, station, region);
    const auto x2 = getLayoutX(muon_raw_to_hits->muonTables, Muon::MuonTables::stripYTableNumber, station, region);
    const auto y2 = getLayoutY(muon_raw_to_hits->muonTables, Muon::MuonTables::stripYTableNumber, station, region);

    Muon::MuonLayout layout_one;
    if (x1 > x2) {
      layout_one = Muon::MuonLayout {x1, y1};
    }
    else {
      layout_one = Muon::MuonLayout {x2, y2};
    }

    for (int j = start_index; j < end_index; ++j) {
      float x = 0.f, dx = 0.f, y = 0.f, dy = 0.f, z = 0.f, dz = 0.f;
      const auto tile = Muon::MuonTileID(storage_tile_id[j]);
      const auto region = tile.region();
      if (tile.station() > (Muon::Constants::n_stations - 3) && region == 0) {
        calcTilePos(muon_raw_to_hits->muonTables, tile, x, dx, y, dy, z);
      }
      else {
        if (Muon::MuonTileID::layout(storage_tile_id[j]) == layout_one) {
          calcStripXPos(muon_raw_to_hits->muonTables, tile, x, dx, y, dy, z);
        }
        else {
          calcStripYPos(muon_raw_to_hits->muonTables, tile, x, dx, y, dy, z);
        }
      }
      const uint uncrossed = 1;
      const int clusterSize = 0;
      const auto localCurrentHitIndex = atomicAdd(current_hit_index, 1);

      setAtIndex(
        event_muon_hits,
        localCurrentHitIndex,
        tile.id(),
        x,
        dx,
        y,
        dy,
        z,
        dz,
        uncrossed,
        storage_tdc_value[j],
        storage_tdc_value[j],
        clusterSize,
        region);
    }
  }
}
