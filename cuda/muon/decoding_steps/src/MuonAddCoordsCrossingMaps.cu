#include "hip/hip_runtime.h"
#include "MuonAddCoordsCrossingMaps.cuh"

__global__ void muon_add_coords_crossing_maps(
  uint* dev_storage_station_region_quarter_offsets,
  uint* dev_storage_tile_id,
  uint* dev_storage_tdc_value,
  uint* dev_atomics_muon,
  uint* dev_permutation_srq,
  Muon::MuonRawToHits* muon_raw_to_hits,
  uint64_t* dev_muon_compact_hit,
  uint* dev_station_ocurrences_offset)
{
  const auto number_of_events = gridDim.x;
  const auto event_number = blockIdx.x;

  __shared__ bool used[Muon::Constants::max_numhits_per_event];
  for (int i = threadIdx.x; i < Muon::Constants::max_numhits_per_event; i += blockDim.x) {
    used[i] = false;
  }

  __syncthreads();

  auto muon_compact_hit = dev_muon_compact_hit + event_number * Muon::Constants::max_numhits_per_event;
  auto storage_tile_id = dev_storage_tile_id + event_number * Muon::Constants::max_numhits_per_event;
  auto storage_tdc_value = dev_storage_tdc_value + event_number * Muon::Constants::max_numhits_per_event;
  auto current_hit_index = dev_atomics_muon + number_of_events + event_number;
  auto storage_station_region_quarter_offsets =
    dev_storage_station_region_quarter_offsets +
    event_number * Muon::Constants::n_stations * Muon::Constants::n_regions * Muon::Constants::n_quarters;
  auto station_ocurrences_offset = dev_station_ocurrences_offset + event_number * Muon::Constants::n_stations;
  const auto base_offset = storage_station_region_quarter_offsets[0];

  for (int i = threadIdx.x; i < Muon::Constants::n_stations * Muon::Constants::n_regions * Muon::Constants::n_quarters;
       i += blockDim.x) {

    const auto start_index = storage_station_region_quarter_offsets[i] - base_offset;
    const auto end_index = storage_station_region_quarter_offsets[i + 1] - base_offset;

    if (start_index != end_index) {

      // muon_raw_to_hits->addCoordsCrossingMap(storage_tile_id, storage_tdc_value, used,
      //   start_index, end_index, event_muon_hits, *current_hit_index);

      // TODO: We are fetching the first tile ID
      //       We should verify this logic holds (it does not atm)
      const auto tile = Muon::MuonTileID(storage_tile_id[start_index]);
      const auto station = tile.station();
      const auto region = tile.region();

      const auto x1 = getLayoutX(muon_raw_to_hits->muonTables, Muon::MuonTables::stripXTableNumber, station, region);
      const auto y1 = getLayoutY(muon_raw_to_hits->muonTables, Muon::MuonTables::stripXTableNumber, station, region);
      const auto x2 = getLayoutX(muon_raw_to_hits->muonTables, Muon::MuonTables::stripYTableNumber, station, region);
      const auto y2 = getLayoutY(muon_raw_to_hits->muonTables, Muon::MuonTables::stripYTableNumber, station, region);

      Muon::MuonLayout layout_one;
      Muon::MuonLayout layout_two;
      if (x1 > x2) {
        layout_one = Muon::MuonLayout {x1, y1};
        layout_two = Muon::MuonLayout {x2, y2};
      }
      else {
        layout_one = Muon::MuonLayout {x2, y2};
        layout_two = Muon::MuonLayout {x1, y1};
      }

      uint mid_index = start_index;
      unsigned int tmp;
      for (uint j = start_index; j < end_index; ++j) {
        if (Muon::MuonTileID::layout(storage_tile_id[j]) == layout_one) {
          if (mid_index != j) {
            tmp = storage_tile_id[j];
            storage_tile_id[j] = storage_tile_id[mid_index];
            storage_tile_id[mid_index] = tmp;

            tmp = storage_tdc_value[j];
            storage_tdc_value[j] = storage_tdc_value[mid_index];
            storage_tdc_value[mid_index] = tmp;
          }
          mid_index++;
        }
      }

      const int thisGridX = layout_one.xGrid();
      const int thisGridY = layout_one.yGrid();
      const int otherGridX = layout_two.xGrid();
      const int otherGridY = layout_two.yGrid();
      for (uint digitsOneIndex = start_index; digitsOneIndex < mid_index; digitsOneIndex++) {
        const unsigned int keyX = Muon::MuonTileID::nX(storage_tile_id[digitsOneIndex]) * otherGridX / thisGridX;
        const unsigned int keyY = Muon::MuonTileID::nY(storage_tile_id[digitsOneIndex]);

        for (uint digitsTwoIndex = mid_index; digitsTwoIndex < end_index; digitsTwoIndex++) {
          const unsigned int candidateX = Muon::MuonTileID::nX(storage_tile_id[digitsTwoIndex]);
          const unsigned int candidateY =
            Muon::MuonTileID::nY(storage_tile_id[digitsTwoIndex]) * thisGridY / otherGridY;

          if (keyX == candidateX && keyY == candidateY) {
            Muon::MuonTileID padTile(storage_tile_id[digitsOneIndex]);
            // padTile.setY(Muon::MuonTileID::nY(storage_tile_id[digitsTwoIndex]));
            // padTile.setLayout(Muon::MuonLayout(thisGridX, otherGridY));
            // float x = 0., dx = 0., y = 0., dy = 0., z = 0., dz = 0.;
            // Muon::calcTilePos(muon_raw_to_hits->muonTables, padTile, x, dx, y, dy, z);
            // const int clusterSize = 0;
            // const int region = padTile.region();
            // setAtIndex(
            //   event_muon_hits,
            //   localCurrentHitIndex,
            //   padTile.id(),
            //   x,
            //   dx,
            //   y,
            //   dy,
            //   z,
            //   dz,
            //   uncrossed,
            //   storage_tdc_value[digitsOneIndex],
            //   storage_tdc_value[digitsOneIndex] - storage_tdc_value[digitsTwoIndex],
            //   clusterSize,
            //   region);
            const int localCurrentHitIndex = atomicAdd(current_hit_index, 1);

            uint64_t compact_hit =
              (((uint64_t)(digitsOneIndex & 0x7FFF)) << 48) | (((uint64_t)(digitsTwoIndex & 0xFFFF)) << 32) |
              ((thisGridX & 0x3FFF) << 18) | ((otherGridY & 0x3FFF) << 4) |
              (((padTile.id() & Muon::MuonBase::MaskStation) >> Muon::MuonBase::ShiftStation) & 0xF);

            muon_compact_hit[localCurrentHitIndex] = compact_hit;

            atomicAdd(station_ocurrences_offset + station, 1);

            used[digitsOneIndex] = used[digitsTwoIndex] = true;
          }
        }
      }

      for (int index = start_index; index < end_index; ++index) {
        if (!used[index]) {
          // float x = 0., dx = 0., y = 0., dy = 0., z = 0., dz = 0.;
          const auto tile = Muon::MuonTileID(storage_tile_id[index]);
          const int region = tile.region();

          int condition;
          if (tile.station() > (Muon::Constants::n_stations - 3) && region == 0) {
            condition = 0;
            // calcTilePos(muon_raw_to_hits->muonTables, tile, x, dx, y, dy, z);
          }
          else {
            if (index < mid_index) {
              condition = 1;
              // calcStripXPos(muon_raw_to_hits->muonTables, tile, x, dx, y, dy, z);
            }
            else {
              condition = 2;
              // calcStripYPos(muon_raw_to_hits->muonTables, tile, x, dx, y, dy, z);
            }
          }
          // const int clusterSize = 0;
          // setAtIndex(
          //   event_muon_hits,
          //   localCurrentHitIndex,
          //   tile.id(),
          //   x,
          //   dx,
          //   y,
          //   dy,
          //   z,
          //   dz,
          //   uncrossed,
          //   storage_tdc_value[index],
          //   storage_tdc_value[index],
          //   clusterSize,
          //   region);

          const int localCurrentHitIndex = atomicAdd(current_hit_index, 1);
          const unsigned int uncrossed = 1;

          uint64_t compact_hit = (((uint64_t)(uncrossed & 0x1)) << 63) | (((uint64_t)(index & 0x7FFF)) << 48) |
                                 (condition << 4) |
                                 (((tile.id() & Muon::MuonBase::MaskStation) >> Muon::MuonBase::ShiftStation) & 0xF);
          muon_compact_hit[localCurrentHitIndex] = compact_hit;

          atomicAdd(station_ocurrences_offset + station, 1);
        }
      }
    }
  }
}
