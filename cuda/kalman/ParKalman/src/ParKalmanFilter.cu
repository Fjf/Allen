#include "hip/hip_runtime.h"
#include "ParKalmanFilter.cuh"

namespace ParKalmanFilter {

  //----------------------------------------------------------------------
  // Create the output track.
  __device__ void MakeTrack(
    const uint n_velo_hits,
    const uint n_ut_layers,
    const uint n_scifi_layers,
    const Vector5& x,
    const SymMatrix5x5& C,
    const KalmanFloat& z,
    const trackInfo& tI,
    FittedTrack& track)
  {
    track.chi2 = tI.m_chi2;
    track.chi2V = tI.m_chi2V;
    track.chi2T = tI.m_chi2T;
    track.ndof = tI.m_ndof - 5;
    track.ndofV = tI.m_ndofV - 5;
    track.ndofT = tI.m_ndofT - 5;
    track.state = x;
    track.cov = C;
    track.z = z;
    track.first_qop = tI.m_FirstMomEst;
    track.best_qop = tI.m_BestMomEst;
    int n_hits = n_velo_hits + n_ut_layers + n_scifi_layers;
    track.nhits = n_hits;
  }

  //----------------------------------------------------------------------
  // Run the Kalman filter.
  __device__ void fit(
    const Velo::Consolidated::Hits& velo_hits,
    const uint n_velo_hits,
    const UT::Consolidated::Hits& ut_hits,
    const uint n_ut_hits,
    const SciFi::Consolidated::Hits& scifi_hits,
    const uint n_scifi_hits,
    const KalmanFloat init_qop,
    const KalmanParametrizations* kalman_params,
    FittedTrack& track)
  {

    // Fit information.
    trackInfo tI;
    tI.m_extr = kalman_params;
    tI.m_BestMomEst = init_qop;
    tI.m_FirstMomEst = init_qop;

    // Get UT hit indices.
    uint32_t n_ut_layers = n_ut_hits;
    for (uint i_ut = 0; i_ut < n_ut_hits; i_ut++) {
      uint8_t layer = ut_hits.plane_code[i_ut];
      tI.m_UTLayerIdxs[layer] = i_ut;
    }

    // Get SciFi hit indices.
    uint n_scifi_layers = n_scifi_hits;
    for (uint i_scifi = 0; i_scifi < n_scifi_hits; i_scifi++) {
      uint32_t layer = scifi_hits.planeCode(i_scifi);
      if (tI.m_SciFiLayerIdxs[layer / 2] >= 0) {
        n_scifi_layers--;
      }
      else
        tI.m_SciFiLayerIdxs[layer / 2] = i_scifi;
    }

    uint n_total_hits = n_velo_hits + n_ut_layers + n_scifi_layers;
    tI.m_NHits = n_total_hits;
    tI.m_NHitsV = n_velo_hits;
    tI.m_NHitsUT = n_ut_layers;
    tI.m_NHitsT = n_scifi_layers;
    tI.m_ndof = n_scifi_layers + n_ut_layers + 2 * n_velo_hits;
    tI.m_ndofT = n_scifi_layers;
    tI.m_ndofUT = n_ut_layers;
    tI.m_ndofV = 2 * n_velo_hits;

    // Initialize the reference propogation matrix.
    // tI.m_RefPropForwardTotal.SetElements(F_diag);

    // Run the fit.
    KalmanFloat lastz = -1.;
    Vector5 x;
    SymMatrix5x5 C;

    // Best state is closest to the beamline.
    KalmanFloat zBest = 0.;
    Vector5 xBest;
    SymMatrix5x5 CBest;

    // Do a forward iteration.
    CreateVeloSeedState(velo_hits, n_velo_hits, 0, x, C, lastz, tI);
    tI.m_chi2 = 0;
    tI.m_chi2V = 0;

    //------------------------------ Start forward fit.
    // Velo loop.
    UpdateStateV(velo_hits, 1, n_velo_hits - 1, x, C, tI);
    for (uint i_hit = 1; i_hit < n_velo_hits; i_hit++) {
      PredictStateV(velo_hits, n_velo_hits - 1 - i_hit, x, C, lastz, tI);
      UpdateStateV(velo_hits, 1, n_velo_hits - 1 - i_hit, x, C, tI);
    }
    __syncthreads();
    KalmanFloat endVeloZ = lastz;

    // Velo -> UT.
    PredictStateVUT(ut_hits, x, C, lastz, tI);
    tI.m_PrevUTLayer = 0;
    while (tI.m_PrevUTLayer < 3 && tI.m_UTLayerIdxs[tI.m_PrevUTLayer] < 0) {
      tI.m_PrevUTLayer++;
      PredictStateUT(ut_hits, tI.m_PrevUTLayer, x, C, lastz, tI);
    }
    __syncthreads();

    // UT loop.
    UpdateStateUT(ut_hits, tI.m_PrevUTLayer, x, C, lastz, tI);
    for (uint32_t i_hit = 1; i_hit < n_ut_layers; i_hit++) {
      tI.m_PrevUTLayer++;
      PredictStateUT(ut_hits, tI.m_PrevUTLayer, x, C, lastz, tI);
      while (tI.m_PrevUTLayer < 3 && tI.m_UTLayerIdxs[tI.m_PrevUTLayer] < 0) {
        tI.m_PrevUTLayer++;
        PredictStateUT(ut_hits, tI.m_PrevUTLayer, x, C, lastz, tI);
      }
      UpdateStateUT(ut_hits, tI.m_PrevUTLayer, x, C, lastz, tI);
    }
    __syncthreads();

    // UT -> SciFi.
    while (tI.m_PrevUTLayer < 3) {
      tI.m_PrevUTLayer++;
      PredictStateUT(ut_hits, tI.m_PrevUTLayer, x, C, lastz, tI);
    }
    PredictStateUTT(x, C, lastz, tI);
    tI.m_PrevSciFiLayer = 0;
    while (tI.m_SciFiLayerIdxs[tI.m_PrevSciFiLayer] < 0) {
      tI.m_PrevSciFiLayer++;
      PredictStateT(scifi_hits, tI.m_PrevSciFiLayer, x, C, lastz, tI);
    }
    __syncthreads();

    // SciFi loop.
    UpdateStateT(scifi_hits, tI.m_PrevSciFiLayer, x, C, lastz, tI);
    for (uint i_hit = 1; i_hit < n_scifi_layers; i_hit++) {
      tI.m_PrevSciFiLayer++;
      PredictStateT(scifi_hits, tI.m_PrevSciFiLayer, x, C, lastz, tI);
      while (tI.m_PrevSciFiLayer < 11 && tI.m_SciFiLayerIdxs[tI.m_PrevSciFiLayer] < 0) {
        tI.m_PrevSciFiLayer++;
        PredictStateT(scifi_hits, tI.m_PrevSciFiLayer, x, C, lastz, tI);
      }
      UpdateStateT(scifi_hits, tI.m_PrevSciFiLayer, x, C, lastz, tI);
    }
    __syncthreads();
    //------------------------------ End forward fit.

    // Set state and covariance for VELO-only backward fit
    tI.m_BestMomEst = x[4];
    tI.m_RefStateForwardV[4] = x[4];
    x = tI.m_RefStateForwardV;
    C = similarity_5_5(inverse(tI.m_RefPropForwardTotal), C);
    lastz = endVeloZ;

    //------------------------------ Start backward fit.
    // Velo loop.
    UpdateStateV(velo_hits, -1, 0, x, C, tI);
    for (int i_hit = n_velo_hits - 2; i_hit >= 0; i_hit--) {
      PredictStateV(velo_hits, n_velo_hits - 1 - i_hit, x, C, lastz, tI);
      UpdateStateV(velo_hits, -1, n_velo_hits - 1 - i_hit, x, C, tI);
    }
    __syncthreads();
    //------------------------------ End backward fit.

    xBest = x;
    CBest = C;
    zBest = lastz;

    // Straight line extrapolation to the closest point to the beamline.
    // NOTE: Don't do this for now. The track is extrapolated again
    // when calculating IP info anyway.
    // ExtrapolateToVertex(xBest, C, lastz);

    MakeTrack(n_velo_hits, n_ut_layers, n_scifi_layers, xBest, CBest, zBest, tI, track);
  }
} // End namespace ParKalmanFilter.

//----------------------------------------------------------------------
// Kalman filter kernel.
__global__ void kalman_filter(
  uint* dev_atomics_storage,
  uint* dev_velo_track_hit_number,
  char* dev_velo_track_hits,
  uint* dev_atomics_veloUT,
  uint* dev_ut_track_hit_number,
  char* dev_ut_consolidated_hits,
  float* dev_ut_qop,
  uint* dev_velo_indices,
  uint* dev_n_scifi_tracks,
  uint* dev_scifi_track_hit_number,
  char* dev_scifi_consolidated_hits,
  float* dev_scifi_qop,
  MiniState* dev_scifi_states,
  uint* dev_ut_indices,
  ParKalmanFilter::FittedTrack* dev_kf_tracks,
  const char* dev_scifi_geometry,
  const float* dev_inv_clus_res,
  const ParKalmanFilter::KalmanParametrizations* dev_kalman_params)
{

  const uint number_of_events = gridDim.x;
  const uint event_number = blockIdx.x;

  // Create velo tracks.
  const Velo::Consolidated::Tracks velo_tracks {
    (uint*) dev_atomics_storage, (uint*) dev_velo_track_hit_number, event_number, number_of_events};

  // Create UT tracks.
  const UT::Consolidated::Tracks ut_tracks {(uint*) dev_atomics_veloUT,
                                            (uint*) dev_ut_track_hit_number,
                                            (float*) dev_ut_qop,
                                            (uint*) dev_velo_indices,
                                            event_number,
                                            number_of_events};

  // Create SciFi tracks.
  const SciFi::Consolidated::Tracks scifi_tracks {(uint*) dev_n_scifi_tracks,
                                                  (uint*) dev_scifi_track_hit_number,
                                                  (float*) dev_scifi_qop,
                                                  (MiniState*) dev_scifi_states,
                                                  (uint*) dev_ut_indices,
                                                  event_number,
                                                  number_of_events};
  const SciFi::SciFiGeometry scifi_geometry {dev_scifi_geometry};

  // Loop over SciFi tracks and get associated UT and VELO tracks.
  const uint n_scifi_tracks = scifi_tracks.number_of_tracks(event_number);
  for (uint i_scifi_track = threadIdx.x; i_scifi_track < n_scifi_tracks; i_scifi_track += blockDim.x) {
    // Prepare fit input.
    const SciFi::Consolidated::Hits scifi_hits =
      scifi_tracks.get_hits(dev_scifi_consolidated_hits, i_scifi_track, &scifi_geometry, dev_inv_clus_res);
    const uint n_scifi_hits = scifi_tracks.number_of_hits(i_scifi_track);
    const int i_ut_track = scifi_tracks.ut_track[i_scifi_track];
    const UT::Consolidated::Hits ut_hits = ut_tracks.get_hits(dev_ut_consolidated_hits, i_ut_track);
    const uint n_ut_hits = ut_tracks.number_of_hits(i_ut_track);
    const int i_velo_track = ut_tracks.velo_track[i_ut_track];
    const Velo::Consolidated::Hits velo_hits = velo_tracks.get_hits((char*) dev_velo_track_hits, i_velo_track);
    const uint n_velo_hits = velo_tracks.number_of_hits(i_velo_track);
    const KalmanFloat init_qop = (KalmanFloat) scifi_tracks.qop[i_scifi_track];
    fit(
      velo_hits,
      n_velo_hits,
      ut_hits,
      n_ut_hits,
      scifi_hits,
      n_scifi_hits,
      init_qop,
      dev_kalman_params,
      dev_kf_tracks[scifi_tracks.tracks_offset(event_number) + i_scifi_track]);
  }
}
