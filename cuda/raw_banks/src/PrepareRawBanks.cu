#include "hip/hip_runtime.h"
#include "PrepareRawBanks.cuh"

__global__ void prepare_raw_banks::prepare_raw_banks(prepare_raw_banks::Parameters parameters)
{

  const uint number_of_events = gridDim.x;
  const uint event_number = blockIdx.x;

  // Tracks.
  const uint* event_tracks_offsets = parameters.dev_atomics_scifi + number_of_events;
  const bool* event_one_track_results = parameters.dev_one_track_results + event_tracks_offsets[event_number];
  const bool* event_single_muon_results = parameters.dev_single_muon_results + event_tracks_offsets[event_number];
  const int n_tracks_event = parameters.dev_atomics_scifi[event_number];

  // Vertices.
  const bool* event_two_track_results = parameters.dev_two_track_results + parameters.dev_sv_offsets[event_number];
  const bool* event_disp_dimuon_results = dev_disp_dimuon_results + parameters.parameters.dev_sv_offsets[event_number];
  const bool* event_high_mass_dimuon_results =
    parameters.dev_high_mass_dimuon_results + parameters.dev_sv_offsets[event_number];
  const int n_vertices_event = parameters.dev_sv_offsets[event_number + 1] - parameters.dev_sv_offsets[event_number];

  // Dec reports.
  const int n_hlt1_lines = Hlt1::Hlt1Lines::End;
  uint32_t* event_dec_reports = parameters.dev_dec_reports + (2 + n_hlt1_lines) * event_number;

  // Set track decisions.
  uint32_t dec_mask = HltDecReport::decReportMasks::decisionMask;
  for (int i_track = threadIdx.x; i_track < n_tracks_event; i_track += blockDim.x) {
    // One track.
    uint32_t dec = ((event_one_track_results[i_track] ? 1 : 0) & dec_mask);
    atomicOr(event_dec_reports + 2 + Hlt1::Hlt1Lines::OneTrackMVA, dec);
    // Single muon decision.
    dec = ((event_single_muon_results[i_track] ? 1 : 0) & dec_mask);
    atomicOr(event_dec_reports + 2 + Hlt1::Hlt1Lines::SingleMuon, dec);
  }
  __syncthreads();

  // Set vertex decisions.
  for (int i_sv = threadIdx.x; i_sv < n_vertices_event; i_sv += blockDim.x) {
    // Two track.
    uint32_t dec = ((event_two_track_results[i_sv] ? 1 : 0) & dec_mask);
    atomicOr(event_dec_reports + 2 + Hlt1::Hlt1Lines::TwoTrackMVA, dec);
    // Displaced dimuon.
    dec = ((event_disp_dimuon_results[i_sv] ? 1 : 0) & dec_mask);
    atomicOr(event_dec_reports + 2 + Hlt1::Hlt1Lines::DisplacedDiMuon, dec);
    // High mass dimuon.
    dec = ((event_high_mass_dimuon_results[i_sv] ? 1 : 0) & dec_mask);
    atomicOr(event_dec_reports + 2 + Hlt1::Hlt1Lines::HighMassDiMuon, dec);
  }
  __syncthreads();

  // If any line is passed, add to selected events and create the rest of the DecReport.
  if (threadIdx.x == 0) {

    // Return if event has not passed.
    bool pass = false;
    for (int i_line = 0; i_line < Hlt1::Hlt1Lines::End; i_line++) {
      pass = pass || ((event_dec_reports[2 + i_line] & dec_mask) == (1 & dec_mask));
      if (pass) {
        break;
      }
    }
    if (!pass) return;

    const uint n_pass = atomicAdd(parameters.dev_number_of_passing_events, 1);
    parameters.dev_passing_event_list[n_pass] = event_number;
    // Create the rest of the dec report.
    event_dec_reports[0] = Hlt1::TCK;
    event_dec_reports[1] = Hlt1::taskID;
    for (uint i_line = 0; i_line < Hlt1::Hlt1Lines::End; i_line++) {
      HltDecReport dec_report;
      dec_report.setDecision(false);
      // TODO: These are all placeholder values for now.
      dec_report.setErrorBits(0);
      dec_report.setNumberOfCandidates(1);
      dec_report.setIntDecisionID(i_line);
      dec_report.setExecutionStage(1);
      // Set the final dec report.
      event_dec_reports[2 + i_line] |= dec_report.getDecReport();
    }
  }
}
