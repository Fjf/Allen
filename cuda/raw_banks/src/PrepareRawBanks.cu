#include "hip/hip_runtime.h"
#include "PrepareRawBanks.cuh"

__global__ void prepare_raw_banks::prepare_raw_banks(prepare_raw_banks::Parameters parameters)
{
  const uint event_number = blockIdx.x;

  // Tracks.
  const int n_tracks_event =
    parameters.dev_offsets_forward_tracks[event_number + 1] - parameters.dev_offsets_forward_tracks[event_number];

  // Vertices.
  const uint n_vertices_event = parameters.dev_sv_offsets[event_number + 1] - parameters.dev_sv_offsets[event_number];

  // Results.
  const uint* dev_sel_results_offsets = parameters.dev_sel_results_atomics + Hlt1::Hlt1Lines::End;
  
  // Dec reports.
  const int n_hlt1_lines = Hlt1::Hlt1Lines::End;
  uint32_t* event_dec_reports = parameters.dev_dec_reports + (2 + n_hlt1_lines) * event_number;
  
  // Set track decisions.
  uint32_t dec_mask = HltDecReport::decReportMasks::decisionMask;
  for (uint i_line = Hlt1::startOneTrackLines; i_line < Hlt1::startTwoTrackLines; i_line++) {
    const bool* decisions = parameters.dev_sel_results +
      dev_sel_results_offsets[i_line] + parameters.dev_offsets_forward_tracks[event_number];
    for (int i_track = threadIdx.x; i_track < n_tracks_event; i_track += blockDim.x) {    
      // One track.
      uint32_t dec = ((decisions[i_track] ? 1 : 0) & dec_mask);
      atomicOr(event_dec_reports + 2 + i_line, dec);
    }
  }
  
  // Set vertex decisions.
  for (uint i_line = Hlt1::startTwoTrackLines; i_line < Hlt1::startThreeTrackLines; i_line++) {
    const bool* decisions = parameters.dev_sel_results +
      dev_sel_results_offsets[i_line] + parameters.dev_sv_offsets[event_number];
    for (int i_sv = threadIdx.x; i_sv < n_vertices_event; i_sv += blockDim.x) {
      // Two track.
      uint32_t dec = ((decisions[i_sv] ? 1 : 0) & dec_mask);
      atomicOr(event_dec_reports + 2 + i_line, dec);
    }
  }
  __syncthreads();

  // If any line is passed, add to selected events and create the rest of the DecReport.
  if (threadIdx.x == 0) {

    // Return if event has not passed.
    bool pass = false;
    for (int i_line = 0; i_line < Hlt1::Hlt1Lines::End; i_line++) {
      pass = pass || ((event_dec_reports[2 + i_line] & dec_mask) == (1 & dec_mask));
      if (pass) {
        break;
      }
    }
    if (!pass) return;

    const uint n_pass = atomicAdd(parameters.dev_number_of_passing_events.get(), 1);
    parameters.dev_passing_event_list[n_pass] = parameters.dev_event_list[event_number];
    // Create the rest of the dec report.
    event_dec_reports[0] = Hlt1::TCK;
    event_dec_reports[1] = Hlt1::taskID;
    for (uint i_line = 0; i_line < Hlt1::Hlt1Lines::End; i_line++) {
      HltDecReport dec_report;
      dec_report.setDecision(false);
      // TODO: These are all placeholder values for now.
      dec_report.setErrorBits(0);
      dec_report.setNumberOfCandidates(1);
      dec_report.setIntDecisionID(i_line);
      dec_report.setExecutionStage(1);
      // Set the final dec report.
      event_dec_reports[2 + i_line] |= dec_report.getDecReport();
    }
  }
}
