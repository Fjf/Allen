#include "hip/hip_runtime.h"
#include "PackageSelReports.cuh"

__global__ void package_sel_reports::package_sel_reports(
  package_sel_reports::Parameters parameters,
  const uint number_of_events,
  const uint selected_number_of_events,
  const uint event_start)
{
  for (auto selected_event_number = blockIdx.x * blockDim.x + threadIdx.x; selected_event_number < number_of_events; selected_event_number += blockDim.x * gridDim.x) {

    const uint event_number = parameters.dev_event_list[selected_event_number] - event_start;
    
    const uint event_sel_rb_stdinfo_offset = event_number * Hlt1::maxStdInfoEvent;
    uint32_t* event_sel_rb_stdinfo = parameters.dev_sel_rb_stdinfo + event_sel_rb_stdinfo_offset;
    const uint event_sel_rb_objtyp_offset = event_number * (Hlt1::nObjTyp + 1);
    uint32_t* event_sel_rb_objtyp = parameters.dev_sel_rb_objtyp + event_sel_rb_objtyp_offset;
    const uint event_sel_rb_substr_offset =
      event_number * Hlt1::subStrDefaultAllocationSize;
    uint32_t* event_sel_rb_substr = parameters.dev_sel_rb_substr + event_sel_rb_substr_offset;
    
    HltSelRepRBSubstr substr_bank;
    substr_bank.m_location = event_sel_rb_substr;
    substr_bank.saveSize();
    HltSelRepRBStdInfo stdinfo_bank;
    stdinfo_bank.m_location = event_sel_rb_stdinfo;
    stdinfo_bank.saveSize();
    HltSelRepRBObjTyp objtyp_bank;
    objtyp_bank.m_location = event_sel_rb_objtyp;
    objtyp_bank.saveSize();

    HltSelRepRawBank selrep_bank(
      parameters.dev_sel_rep_raw_banks + parameters.dev_sel_rep_offsets[event_number]);
    selrep_bank.push_back(
      HltSelRepRBEnums::kObjTypID,
      objtyp_bank.m_location,
      objtyp_bank.size());
    selrep_bank.push_back(
      HltSelRepRBEnums::kSubstrID,
      substr_bank.m_location,
      substr_bank.size());

    if (selected_event_number < selected_number_of_events) {
      const uint event_sel_rb_hits_offset =
        parameters.dev_offsets_forward_tracks[selected_event_number] * ParKalmanFilter::nMaxMeasurements +
        3 * selected_event_number;
      uint32_t* event_sel_rb_hits = parameters.dev_sel_rb_hits + event_sel_rb_hits_offset;

      HltSelRepRBHits hits_bank;
      hits_bank.m_location = event_sel_rb_hits;

      selrep_bank.push_back(
        HltSelRepRBEnums::kHitsID,
        hits_bank.m_location,
        hits_bank.size());
    }
    
    if (stdinfo_bank.sizeStored() < Hlt1::maxStdInfoEvent) {
      selrep_bank.push_back(
        HltSelRepRBEnums::kStdInfoID,
        stdinfo_bank.m_location,
        stdinfo_bank.size());
    }
  }
}