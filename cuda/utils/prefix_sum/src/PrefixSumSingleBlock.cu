#include "hip/hip_runtime.h"
#include "PrefixSum.cuh"
#include "Invoke.cuh"

void copy_and_prefix_sum_single_block_velo_t::invoke() {
  invoke_helper(handler);
}

void copy_velo_track_hit_number_t::invoke() {
  invoke_helper(handler);
}

void copy_and_prefix_sum_single_block_ut_t::invoke() {
  invoke_helper(handler);
}

void copy_ut_track_hit_number_t::invoke() {
  invoke_helper(handler);
}

void copy_and_prefix_sum_single_block_scifi_t::invoke() {
  invoke_helper(handler);
}

void copy_scifi_track_hit_number_t::invoke() {
  invoke_helper(handler);
}

void copy_and_prefix_sum_single_block_sv_t::invoke() {
  invoke_helper(handler);
}

/**
 * @brief Up-Sweep
 */
__device__ void up_sweep_2048(uint* data_block)
{
  uint starting_elem = 1;
  for (uint i = 2; i <= 2048; i <<= 1) {
    for (uint j = 0; j < (2047 + blockDim.x) / i; ++j) {
      const uint element = starting_elem + (j * blockDim.x + threadIdx.x) * i;
      if (element < 2048) {
        data_block[element] += data_block[element - (i >> 1)];
      }
    }
    starting_elem += i;
    __syncthreads();
  }
}

/**
 * @brief Down-sweep
 */
__device__ void down_sweep_2048(uint* data_block)
{
  for (uint i = 2048; i >= 2; i >>= 1) {
    for (uint j = 0; j < (2047 + blockDim.x) / i; ++j) {
      const auto element = 2047 - (j * blockDim.x + threadIdx.x) * i;
      if (element < 2048) {
        const auto other_element = element - (i >> 1);
        const auto value = data_block[other_element];
        data_block[other_element] = data_block[element];
        data_block[element] += value;
      }
    }
    __syncthreads();
  }
}

__device__ void
prefix_sum_single_block_implementation(uint* dev_total_sum, uint* dev_array, const uint array_size, uint* data_block)
{
  // Prefix sum of elements in dev_array
  // Using Blelloch scan https://www.youtube.com/watch?v=mmYv3Haj6uc

  // Let's do it in blocks of 2048 (2^11)
  unsigned prev_last_elem = 0;
  for (uint block = 0; block < (array_size >> 11); ++block) {
    const uint first_elem = block << 11;

    // Load elements into shared memory, add prev_last_elem
    data_block[2 * threadIdx.x] = dev_array[first_elem + 2 * threadIdx.x];
    data_block[2 * threadIdx.x + 1] = dev_array[first_elem + 2 * threadIdx.x + 1];

    __syncthreads();

    up_sweep_2048((uint*) &data_block[0]);

    const uint new_last_elem = data_block[2047];

    __syncthreads();
    data_block[2047] = 0;
    __syncthreads();

    down_sweep_2048((uint*) &data_block[0]);

    // Store back elements
    dev_array[first_elem + 2 * threadIdx.x] = data_block[2 * threadIdx.x] + prev_last_elem;
    dev_array[first_elem + 2 * threadIdx.x + 1] = data_block[2 * threadIdx.x + 1] + prev_last_elem;
    prev_last_elem += new_last_elem;

    __syncthreads();
  }

  // Last iteration is special because
  // it may contain an unspecified number of elements
  const auto elements_remaining = array_size & 0x7FF; // % 2048
  if (elements_remaining > 0) {
    const auto first_elem = array_size - elements_remaining;

    // Initialize all elements to zero
    data_block[2 * threadIdx.x] = 0;
    data_block[2 * threadIdx.x + 1] = 0;

    // Load elements
    const auto elem_index = first_elem + 2 * threadIdx.x;
    if (elem_index < array_size) {
      data_block[2 * threadIdx.x] = dev_array[elem_index];
    }
    if ((elem_index + 1) < array_size) {
      data_block[2 * threadIdx.x + 1] = dev_array[elem_index + 1];
    }

    __syncthreads();

    up_sweep_2048((uint*) &data_block[0]);

    // Store sum of all elements
    if (threadIdx.x == 0) {
      dev_total_sum[0] = prev_last_elem + data_block[2047];
    }

    __syncthreads();
    data_block[2047] = 0;
    __syncthreads();

    down_sweep_2048((uint*) &data_block[0]);

    // Store back elements
    if (elem_index < array_size) {
      dev_array[elem_index] = data_block[2 * threadIdx.x] + prev_last_elem;
    }
    if ((elem_index + 1) < array_size) {
      dev_array[elem_index + 1] = data_block[2 * threadIdx.x + 1] + prev_last_elem;
    }
  }
  else {
    // Special case where number of elements is binary
    if (threadIdx.x == 0) {
      dev_total_sum[0] = prev_last_elem;
    }
  }
}

__global__ void prefix_sum_single_block(uint* dev_total_sum, uint* dev_array, const uint array_size)
{
  __shared__ uint data_block[2048];

  prefix_sum_single_block_implementation(dev_total_sum, dev_array, array_size, data_block);
}

__global__ void copy_and_prefix_sum_single_block(
  uint* dev_total_sum,
  uint* dev_input_array,
  uint* dev_output_array,
  const uint array_size)
{
  __shared__ uint data_block[2048];

  // Copy the input array into the output array
  for (uint i = 0; i < (array_size + blockDim.x - 1) / blockDim.x; ++i) {
    const auto element = i * blockDim.x + threadIdx.x;
    if (element < array_size) {
      dev_output_array[element] = dev_input_array[element];
    }
  }

  __syncthreads();

  // Perform prefix_sum over output array
  prefix_sum_single_block_implementation(dev_total_sum, dev_output_array, array_size, data_block);
}

__global__ void copy_square_and_prefix_sum_single_block(
  uint* dev_total_sum,
  uint* dev_input_array,
  uint* dev_output_array,
  const uint array_size)
{
  __shared__ uint data_block[2048];

  // Copy N(N-1)/2 to the output location.
  for (uint i = 0; i < (array_size + blockDim.x - 1) / blockDim.x; ++i) {
    const auto element = i * blockDim.x + threadIdx.x;
    if (element < array_size) {
      dev_output_array[element] = (dev_input_array[element] * (dev_input_array[element] - 1)) >> 1;
    }
  }
  __syncthreads();

  // Perform the prefix sum over the output array.
  prefix_sum_single_block_implementation(dev_total_sum, dev_output_array, array_size, data_block);
}

/**
 * @brief Copies Velo track hit numbers on a consecutive container
 */
__global__ void copy_velo_track_hit_number(
  const Velo::TrackHits* dev_tracks,
  uint* dev_atomics_storage,
  uint* dev_velo_track_hit_number)
{
  const auto number_of_events = gridDim.x;
  const auto event_number = blockIdx.x;
  const auto* event_tracks = dev_tracks + event_number * Velo::Constants::max_tracks;
  const auto accumulated_tracks = dev_atomics_storage[number_of_events + event_number];
  const auto number_of_tracks = dev_atomics_storage[event_number];

  // Pointer to velo_track_hit_number of current event
  uint* velo_track_hit_number = dev_velo_track_hit_number + accumulated_tracks;

  for (uint element = threadIdx.x; element < number_of_tracks; ++element) {
    velo_track_hit_number[element] = event_tracks[element].hitsNum;
  }
}

/**
 * @brief Copies UT track hit numbers on a consecutive container.
 */
__global__ void copy_ut_track_hit_number(
  const UT::TrackHits* dev_veloUT_tracks,
  uint* dev_atomics_veloUT,
  uint* dev_ut_track_hit_number)
{
  const auto number_of_events = gridDim.x;
  const auto event_number = blockIdx.x;
  const auto* event_tracks = dev_veloUT_tracks + event_number * UT::Constants::max_num_tracks;
  const auto accumulated_tracks = dev_atomics_veloUT[number_of_events + event_number];
  const auto number_of_tracks = dev_atomics_veloUT[event_number];

  // Pointer to ut_track_hit_number of current event.
  uint* ut_track_hit_number = dev_ut_track_hit_number + accumulated_tracks;

  // Loop over tracks.
  for (uint element = threadIdx.x; element < number_of_tracks; ++element) {
    ut_track_hit_number[element] = event_tracks[element].hits_num;
  }
}

/**
 * @brief Copies SciFi track hit numbers to a consecutive container.
 */
__global__ void copy_scifi_track_hit_number(
  const uint* dev_atomics_ut,
  const SciFi::TrackHits* dev_scifi_tracks,
  uint* dev_n_scifi_tracks,
  uint* dev_scifi_track_hit_number)
{
  const auto number_of_events = gridDim.x;
  const auto event_number = blockIdx.x;
  const auto ut_event_tracks_offset = dev_atomics_ut[number_of_events + event_number];

  const auto* event_tracks =
    dev_scifi_tracks + ut_event_tracks_offset * SciFi::Constants::max_SciFi_tracks_per_UT_track;
  // const SciFi::TrackHits* event_tracks =
  //   dev_scifi_tracks + ut_event_tracks_offset *
  //   LookingForward::maximum_number_of_candidates_per_ut_track_after_x_filter;
  const auto accumulated_tracks = dev_n_scifi_tracks[number_of_events + event_number];
  const auto number_of_tracks = dev_n_scifi_tracks[event_number];

  // Pointer to scifi_track_hit_number of current event.
  uint* scifi_track_hit_number = dev_scifi_track_hit_number + accumulated_tracks;

  // Loop over tracks.
  for (uint element = threadIdx.x; element < number_of_tracks; ++element) {
    scifi_track_hit_number[element] = event_tracks[element].hitsNum;
  }
}
