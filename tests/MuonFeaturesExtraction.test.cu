#include "hip/hip_runtime.h"
/*
*   Tests for calculation muon catbost features
*   How to run it
*   ./cuda/muon/TestFeatures
*/
#include "catch.hpp"
#include "MuonFeaturesExtraction.test.cuh"

SCENARIO( "Check closest hit works in case there is no extrapolation" ) {

    dev_allocate_memory();

    GIVEN( 
        "Grid of hits\n" 
        "There is 9 hits on each station with coordinates x,y: \n"
        "\t (-1, 1) - ( 0, 1) - ( 1, 1) \n"
        "\t (-1, 0) - ( 0, 0) - ( 1, 0) \n"
        "\t (-1,-1) - ( 0,-1) - ( 1,-1) \n"
        "z = i_station + 1 \n"
        "Hits indices on first station: \n"
        "\t 6	-    7	  -    8 \n"
        "\t 3	-    4	  -    5 \n"
        "\t 0	-    1	  -    2 \n"
        "and so on \n"
        "dx = index, dy = 2 * index, dz = 0 \n"
    ) {

        std::vector<Muon::HitsSoA> muon_hits_events;
        Muon::HitsSoA muon_hits = ConstructMockMuonHit();

        // One event
        muon_hits_events.push_back(muon_hits);
        
        Muon::HitsSoA *dev_muon_hits;
        hipMalloc(&dev_muon_hits, muon_hits_events.size() * sizeof(Muon::HitsSoA));
        hipMemcpy(dev_muon_hits, muon_hits_events.data(), muon_hits_events.size() * sizeof(Muon::HitsSoA), hipMemcpyHostToDevice);

        float *host_features = (float*)malloc(1 * n_features * sizeof(float));

        WHEN( "Track inside grid of hits and parallel to the axis OZ (x=0.9, y=0.9, z=0, dx=0, dy=0)" ) {

            // Track initialization
            MiniState track = MiniState(0.9, 0.9, 0, 0, 0);
            hipMemcpy(dev_track, &track, 1 * sizeof(MiniState), hipMemcpyHostToDevice);

            muon_catboost_features_extraction<<<dim3(1, 4), 1>>>(
                dev_atomics_scifi,
                dev_scifi_track_hit_number,
                dev_qop,
                dev_track,
                dev_scifi_track_ut_indices,
                dev_muon_hits,
                dev_features
            );

            hipMemcpy(host_features, dev_features, n_features * sizeof(float), hipMemcpyDeviceToHost);

            THEN(
                "Extrapolation of track:   \n"
                "\t station 0 - (0.9, 0.9) \n"
                "\t station 1 - (0.9, 0.9) \n"
                "\t station 2 - (0.9, 0.9) \n"
                "\t station 3 - (0.9, 0.9) \n"
                "Closest hits: \n"
                "\t station 0 - ( 1, 1), index = 8  \n"
                "\t station 1 - ( 1, 1), index = 17 \n"
                "\t station 2 - ( 1, 1), index = 26 \n"
                "\t station 3 - ( 1, 1), index = 35 \n"
                "Traveled distance: \n"
                "\t station 0 - 0   \n"
                "\t station 1 - 1   \n"
                "\t station 2 - 2   \n"
                "\t station 3 - 3   \n"
            ) {
                const std::vector<int> closest_hits = {8, 17, 26, 35};
                const std::vector<float> trav_dist = {0, 1, 2, 3};
                const std::vector<float> extrapolation_x = {0.9, 0.9, 0.9, 0.9};
                const std::vector<float> extrapolation_y = {0.9, 0.9, 0.9, 0.9};
                for (int i_station = 0; i_station < Muon::Constants::n_stations; i_station++) {
                    const int closest_idx = closest_hits[i_station];
                    const float multiple_scattering_error = COMMON_FACTOR * trav_dist[i_station] * sqrt(trav_dist[i_station]);
                    CHECK(host_features[offset::DTS + i_station] == muon_hits_events[0].delta_time[closest_idx]);
                    CHECK(host_features[offset::TIMES + i_station] == muon_hits_events[0].time[closest_idx]);
                    CHECK(host_features[offset::CROSS + i_station] + muon_hits_events[0].uncrossed[closest_idx] == 2);
                    CHECK_THAT(host_features[offset::RES_X + i_station], 
                        Catch::Matchers::WithinAbs((extrapolation_x[i_station] - muon_hits_events[0].x[closest_idx]) / 
                        sqrt(closest_idx * closest_idx * INVSQRT3 * INVSQRT3 + multiple_scattering_error * multiple_scattering_error), eps)
                    );
                    CHECK_THAT(host_features[offset::RES_Y + i_station], 
                        Catch::Matchers::WithinAbs((extrapolation_y[i_station] - muon_hits_events[0].y[closest_idx]) / 
                        sqrt(4 * closest_idx * closest_idx * INVSQRT3 * INVSQRT3 + multiple_scattering_error * multiple_scattering_error), eps)
                    );
                }
            }
        }
        
        WHEN( "Track equidistant from 4 hits and parallel to the axis OZ (x=0.5, y=0.5, z=0, dx=0, dy=0)" ) {

            // Track initialization
            MiniState track = MiniState(0.5, 0.5, 0, 0, 0);
            hipMemcpy(dev_track, &track, 1 * sizeof(MiniState), hipMemcpyHostToDevice);

            muon_catboost_features_extraction<<<dim3(1, 4), 1>>>(
                dev_atomics_scifi,
                dev_scifi_track_hit_number,
                dev_qop,
                dev_track,
                dev_scifi_track_ut_indices,
                dev_muon_hits,
                dev_features
            );

            hipMemcpy(host_features, dev_features, n_features * sizeof(float), hipMemcpyDeviceToHost);

            THEN(
                "Extrapolation of track:   \n"
                "\t station 0 - (0.5, 0.5) \n"
                "\t station 1 - (0.5, 0.5) \n"
                "\t station 2 - (0.5, 0.5) \n"
                "\t station 3 - (0.5, 0.5) \n"
                "Closest hits: \n"
                "\t station 0 - ( 0, 0) or ( 0, 1) or ( 1, 0) or ( 1, 1), index =  4 or  7 or  5 or 8  \n"
                "\t station 1 - ( 0, 0) or ( 0, 1) or ( 1, 0) or ( 1, 1), index = 13 or 16 or 14 or 17 \n"
                "\t station 2 - ( 0, 0) or ( 0, 1) or ( 1, 0) or ( 1, 1), index = 22 or 25 or 23 or 26 \n"
                "\t station 3 - ( 0, 0) or ( 0, 1) or ( 1, 0) or ( 1, 1), index = 31 or 34 or 32 or 35 \n"
                "Traveled distance: \n"
                "\t station 0 - 0   \n"
                "\t station 1 - 1   \n"
                "\t station 2 - 2   \n"
                "\t station 3 - 3   \n"
            ) {
                const std::vector<std::vector<int>> closest_hits = {
                    { 4,  7,  5,  8}, 
                    {13, 16, 14, 17}, 
                    {22, 25, 23, 26}, 
                    {31, 34, 32, 35}
                };
                const std::vector<float> trav_dist = {0, 1, 2, 3};
                const std::vector<float> extrapolation_x = {0.5, 0.5, 0.5, 0.5};
                const std::vector<float> extrapolation_y = {0.5, 0.5, 0.5, 0.5};
                for (int i_station = 0; i_station < Muon::Constants::n_stations; i_station++) {
                    const float multiple_scattering_error = COMMON_FACTOR * trav_dist[i_station] * sqrt(trav_dist[i_station]);
                    CHECK(
                        any_of(
                            closest_hits[i_station], 
                            host_features[offset::DTS + i_station], 
                            muon_hits_events[0].delta_time) == true
                    );
                    CHECK(
                        any_of(
                            closest_hits[i_station], 
                            host_features[offset::TIMES + i_station], 
                            (int*) muon_hits_events[0].time) == true
                    );
                    CHECK(
                        any_of(
                            closest_hits[i_station], 
                            2 - host_features[offset::CROSS + i_station], 
                            muon_hits_events[0].uncrossed) == true
                    );
                    const std::vector<float> true_res_x = calculate_res(
                        closest_hits[i_station],
                        extrapolation_x[i_station],
                        muon_hits_events[0].x,
                        muon_hits_events[0].dx,
                        multiple_scattering_error
                    );
                    CHECK_THAT(host_features[offset::RES_X + i_station], 
                        Catch::Matchers::WithinAbs(true_res_x[0], eps) ||
                        Catch::Matchers::WithinAbs(true_res_x[1], eps) ||
                        Catch::Matchers::WithinAbs(true_res_x[2], eps) ||
                        Catch::Matchers::WithinAbs(true_res_x[3], eps)
                    );
                    const std::vector<float> true_res_y = calculate_res(
                        closest_hits[i_station],
                        extrapolation_y[i_station],
                        muon_hits_events[0].y,
                        muon_hits_events[0].dy,
                        multiple_scattering_error
                    );
                    CHECK_THAT(host_features[offset::RES_Y + i_station], 
                        Catch::Matchers::WithinAbs(true_res_y[0], eps) ||
                        Catch::Matchers::WithinAbs(true_res_y[1], eps) ||
                        Catch::Matchers::WithinAbs(true_res_y[2], eps) ||
                        Catch::Matchers::WithinAbs(true_res_y[3], eps)
                    );
                }
            }
        }

        WHEN( "Track mathes hit and parallel to the axis OZ (x=1, y=-1, z=0, dx=0, dy=0)" ) {

            // Track initialization
            MiniState track = MiniState(1, -1, 0, 0, 0);
            hipMemcpy(dev_track, &track, 1 * sizeof(MiniState), hipMemcpyHostToDevice);

            muon_catboost_features_extraction<<<dim3(1, 4), 1>>>(
                dev_atomics_scifi,
                dev_scifi_track_hit_number,
                dev_qop,
                dev_track,
                dev_scifi_track_ut_indices,
                dev_muon_hits,
                dev_features
            );

            hipMemcpy(host_features, dev_features, n_features * sizeof(float), hipMemcpyDeviceToHost);

            THEN(
                "Extrapolation of track:\n"
                "\t station 0 - ( 1,-1) \n"
                "\t station 1 - ( 1,-1) \n"
                "\t station 2 - ( 1,-1) \n"
                "\t station 3 - ( 1,-1) \n"
                "Closest hits: \n"
                "\t station 0 - ( 1,-1), index = 2  \n"
                "\t station 1 - ( 1,-1), index = 11 \n"
                "\t station 2 - ( 1,-1), index = 20 \n"
                "\t station 3 - ( 1,-1), index = 29 \n"
                "Traveled distance: \n"
                "\t station 0 - 0   \n"
                "\t station 1 - 1   \n"
                "\t station 2 - 2   \n"
                "\t station 3 - 3   \n"
            ) {
                const std::vector<int> closest_hits = {2, 11, 20, 29};
                const std::vector<float> trav_dist = {0, 1, 2, 3};
                const std::vector<float> extrapolation_x = {1, 1, 1, 1};
                const std::vector<float> extrapolation_y = {-1, -1, -1, -1};
                for (int i_station = 0; i_station < Muon::Constants::n_stations; i_station++) {
                    const int closest_idx = closest_hits[i_station];
                    const float multiple_scattering_error = COMMON_FACTOR * trav_dist[i_station] * sqrt(trav_dist[i_station]);
                    CHECK(host_features[offset::DTS + i_station] == muon_hits_events[0].delta_time[closest_idx]);
                    CHECK(host_features[offset::TIMES + i_station] == muon_hits_events[0].time[closest_idx]);
                    CHECK(host_features[offset::CROSS + i_station] + muon_hits_events[0].uncrossed[closest_idx] == 2);
                    CHECK_THAT(host_features[offset::RES_X + i_station], 
                        Catch::Matchers::WithinAbs((extrapolation_x[i_station] - muon_hits_events[0].x[closest_idx]) / 
                        sqrt(closest_idx * closest_idx * INVSQRT3 * INVSQRT3 + multiple_scattering_error * multiple_scattering_error), eps)
                    );
                    CHECK_THAT(host_features[offset::RES_Y + i_station], 
                        Catch::Matchers::WithinAbs((extrapolation_y[i_station] - muon_hits_events[0].y[closest_idx]) / 
                        sqrt(4 * closest_idx * closest_idx * INVSQRT3 * INVSQRT3 + multiple_scattering_error * multiple_scattering_error), eps)
                    );
                }
            }
        }

        WHEN( "Track is far away from all hits and parallel to the axis OZ (x=1000, y=0.4, z=0, dx=0, dy=0)" ) {

            // Track initialization
            MiniState track = MiniState(1000, 0.4, 0, 0, 0);
            hipMemcpy(dev_track, &track, 1 * sizeof(MiniState), hipMemcpyHostToDevice);

            muon_catboost_features_extraction<<<dim3(1, 4), 1>>>(
                dev_atomics_scifi,
                dev_scifi_track_hit_number,
                dev_qop,
                dev_track,
                dev_scifi_track_ut_indices,
                dev_muon_hits,
                dev_features
            );

            hipMemcpy(host_features, dev_features, n_features * sizeof(float), hipMemcpyDeviceToHost);
            THEN(
                "Extrapolation of track:\n"
                "\t station 0 - (1000, 0.4) \n"
                "\t station 1 - (1000, 0.4) \n"
                "\t station 2 - (1000, 0.4) \n"
                "\t station 3 - (1000, 0.4) \n"
                "Closest hits: \n"
                "\t station 0 - ( 1, 0), index = 5  \n"
                "\t station 1 - ( 1, 0), index = 14 \n"
                "\t station 2 - ( 1, 0), index = 23 \n"
                "\t station 3 - ( 1, 0), index = 32 \n"
                "Traveled distance: \n"
                "\t station 0 - 0   \n"
                "\t station 1 - 1   \n"
                "\t station 2 - 2   \n"
                "\t station 3 - 3   \n"
            ) {
                const std::vector<int> closest_hits = {5, 14, 23, 32};
                const std::vector<float> trav_dist = {0, 1, 2, 3};
                const std::vector<float> extrapolation_x = {1000, 1000, 1000, 1000};
                const std::vector<float> extrapolation_y = {0.4, 0.4, 0.4, 0.4};
                for (int i_station = 0; i_station < Muon::Constants::n_stations; i_station++) {
                    const int closest_idx = closest_hits[i_station];
                    const float multiple_scattering_error = COMMON_FACTOR * trav_dist[i_station] * sqrt(trav_dist[i_station]);
                    CHECK(host_features[offset::DTS + i_station] == muon_hits_events[0].delta_time[closest_idx]);
                    CHECK(host_features[offset::TIMES + i_station] == muon_hits_events[0].time[closest_idx]);
                    CHECK(host_features[offset::CROSS + i_station] + muon_hits_events[0].uncrossed[closest_idx] == 2);
                    CHECK_THAT(host_features[offset::RES_X + i_station], 
                        Catch::Matchers::WithinAbs((extrapolation_x[i_station] - muon_hits_events[0].x[closest_idx]) / 
                        sqrt(closest_idx * closest_idx * INVSQRT3 * INVSQRT3 + multiple_scattering_error * multiple_scattering_error), eps)
                    );
                    CHECK_THAT(host_features[offset::RES_Y + i_station],
                        Catch::Matchers::WithinAbs((extrapolation_y[i_station] - muon_hits_events[0].y[closest_idx]) / 
                        sqrt(4 * closest_idx * closest_idx * INVSQRT3 * INVSQRT3 + multiple_scattering_error * multiple_scattering_error), eps)
                    );
                }
            }
        }
        free(host_features);
        hipFree(dev_muon_hits);
    }
    dev_free_memory();
}

SCENARIO( "Check closest hit works in general case" ) {

    dev_allocate_memory();

    GIVEN( 
        "Grid of hits\n" 
        "There is 9 hits on each station with coordinates x,y: \n"
        "\t (-1, 1) - ( 0, 1) - ( 1, 1) \n"
        "\t (-1, 0) - ( 0, 0) - ( 1, 0) \n"
        "\t (-1,-1) - ( 0,-1) - ( 1,-1) \n"
        "z = i_station + 1 \n"
        "Hits indices on first station: \n"
        "\t 6	-    7	  -    8 \n"
        "\t 3	-    4	  -    5 \n"
        "\t 0	-    1	  -    2 \n"
        "and so on \n"
        "dx = index, dy = 2 * index, dz = 0 \n"
    ) {

        std::vector<Muon::HitsSoA> muon_hits_events;
        Muon::HitsSoA muon_hits = ConstructMockMuonHit();

        // One event
        muon_hits_events.push_back(muon_hits);
        
        Muon::HitsSoA *dev_muon_hits;
        hipMalloc(&dev_muon_hits, muon_hits_events.size() * sizeof(Muon::HitsSoA));
        hipMemcpy(dev_muon_hits, muon_hits_events.data(), muon_hits_events.size() * sizeof(Muon::HitsSoA), hipMemcpyHostToDevice);

        float *host_features = (float*)malloc(1 * n_features * sizeof(float));

        WHEN( "Track inside grid of hits (x=-2.7, y=-2.7, z=0, dx=1, dy=1)" ) {

            // Track initialization
            MiniState track = MiniState(-2.7, -2.7, 0, 1, 1);
            hipMemcpy(dev_track, &track, 1 * sizeof(MiniState), hipMemcpyHostToDevice);

            muon_catboost_features_extraction<<<dim3(1, 4), 1>>>(
                dev_atomics_scifi,
                dev_scifi_track_hit_number,
                dev_qop,
                dev_track,
                dev_scifi_track_ut_indices,
                dev_muon_hits,
                dev_features
            );

            hipMemcpy(host_features, dev_features, n_features * sizeof(float), hipMemcpyDeviceToHost);

            THEN(
                "Extrapolation of track: \n"
                "\t station 1 - (-1.7,-1.7) \n"
                "\t station 2 - (-0.7,-0.7) \n"
                "\t station 3 - ( 0.3, 0.3) \n"
                "\t station 4 - ( 1.3, 1.3) \n"
                "Closest hits: \n"
                "\t station 1 - (-1,-1), index = 0 \n"
                "\t station 2 - (-1,-1), index = 9 \n"
                "\t station 3 - ( 0, 0), index = 22 \n"
                "\t station 4 - ( 1, 1), index = 35 \n"
                "Traveled distance: \n"
                "\t station 1 - 0 \n"
                "\t station 2 - sqrt(3) \n"
                "\t station 3 - sqrt(12) \n"
                "\t station 4 - sqrt(27) \n"
            ) {
                const std::vector<int> closest_hits = {0, 9, 22, 35};
                const std::vector<float> extrapolation_x = {-1.7, -0.7, 0.3, 1.3};
                const std::vector<float> extrapolation_y = {-1.7, -0.7, 0.3, 1.3};
                const std::vector<float> trav_dist = {0, sqrt(3.0f), sqrt(12.0f), sqrt(27.0f)};
                for (int i_station = 0; i_station < Muon::Constants::n_stations; i_station++) {
                    const int closest_idx = closest_hits[i_station];
                    const float multiple_scattering_error = COMMON_FACTOR * trav_dist[i_station] * sqrt(trav_dist[i_station]);
                    CHECK(host_features[offset::DTS + i_station] == muon_hits_events[0].delta_time[closest_idx]);
                    CHECK(host_features[offset::TIMES + i_station] == muon_hits_events[0].time[closest_idx]);
                    CHECK(host_features[offset::CROSS + i_station] + muon_hits_events[0].uncrossed[closest_idx] == 2);
                    CHECK_THAT(
                        host_features[offset::RES_X + i_station], Catch::Matchers::WithinAbs((extrapolation_x[i_station] - muon_hits_events[0].x[closest_idx]) / 
                        sqrt(closest_idx * closest_idx * INVSQRT3 * INVSQRT3 + multiple_scattering_error * multiple_scattering_error), eps)
                    );
                    CHECK_THAT(
                        host_features[offset::RES_Y + i_station], Catch::Matchers::WithinAbs((extrapolation_y[i_station] - muon_hits_events[0].y[closest_idx]) / 
                        sqrt(4 * closest_idx * closest_idx * INVSQRT3 * INVSQRT3 + multiple_scattering_error * multiple_scattering_error), eps)
                    );
                }
            }
        }

        WHEN( "Track inside grid of hits (x=-2.2, y=2.1, z=0, dx=1, dy=-0.5)" ) {

            // Track initialization
            MiniState track = MiniState(-2.2, 2.1, 0, 1, -0.5);
            hipMemcpy(dev_track, &track, 1 * sizeof(MiniState), hipMemcpyHostToDevice);

            muon_catboost_features_extraction<<<dim3(1, 4), 1>>>(
                dev_atomics_scifi,
                dev_scifi_track_hit_number,
                dev_qop,
                dev_track,
                dev_scifi_track_ut_indices,
                dev_muon_hits,
                dev_features
            );

            hipMemcpy(host_features, dev_features, n_features * sizeof(float), hipMemcpyDeviceToHost);

            THEN(
                "Extrapolation of track:    \n"
                "\t station 1 - (-1.2, 1.6) \n"
                "\t station 2 - (-0.2, 1.1) \n"
                "\t station 3 - ( 0.8, 0.6) \n"
                "\t station 4 - ( 1.8, 0.1) \n"
                "Closest hits: \n"
                "\t station 1 - (-1,-1), index = 6  \n"
                "\t station 2 - ( 0, 1), index = 16 \n"
                "\t station 3 - ( 1, 1), index = 26 \n"
                "\t station 4 - ( 1, 0), index = 32 \n"
                "Traveled distance: \n"
                "\t station 1 - 0   \n"
                "\t station 2 - 1.5 \n"
                "\t station 3 - 3   \n"
                "\t station 4 - sqrt(20.25) \n"
            ) {
                const std::vector<int> closest_hits = {6, 16, 26, 32};
                const std::vector<float> extrapolation_x = {-1.2, -0.2, 0.8, 1.8};
                const std::vector<float> extrapolation_y = { 1.6,  1.1, 0.6, 0.1};
                const std::vector<float> trav_dist = {0, 1.5, 3, sqrt(20.25f)};
                for (int i_station = 0; i_station < Muon::Constants::n_stations; i_station++) {
                    const int closest_idx = closest_hits[i_station];
                    const float multiple_scattering_error = COMMON_FACTOR * trav_dist[i_station] * sqrt(trav_dist[i_station]);
                    CHECK(host_features[offset::DTS + i_station] == muon_hits_events[0].delta_time[closest_idx]);
                    CHECK(host_features[offset::TIMES + i_station] == muon_hits_events[0].time[closest_idx]);
                    CHECK(host_features[offset::CROSS + i_station] + muon_hits_events[0].uncrossed[closest_idx] == 2);
                    CHECK_THAT(
                        host_features[offset::RES_X + i_station], Catch::Matchers::WithinAbs((extrapolation_x[i_station] - muon_hits_events[0].x[closest_idx]) / 
                        sqrt(closest_idx * closest_idx * INVSQRT3 * INVSQRT3 + multiple_scattering_error * multiple_scattering_error), eps)
                    );
                    CHECK_THAT(
                        host_features[offset::RES_Y + i_station], Catch::Matchers::WithinAbs((extrapolation_y[i_station] - muon_hits_events[0].y[closest_idx]) / 
                        sqrt(4 * closest_idx * closest_idx * INVSQRT3 * INVSQRT3 + multiple_scattering_error * multiple_scattering_error), eps)
                    );
                }
            }
        }

        WHEN( "Track is far away from hits (x=999, y=-2.7, z=0, dx=1, dy=1)" ) {

            // Track initialization
            MiniState track = MiniState(999, -2.7, 0, 1, 1);
            hipMemcpy(dev_track, &track, 1 * sizeof(MiniState), hipMemcpyHostToDevice);

            muon_catboost_features_extraction<<<dim3(1, 4), 1>>>(
                dev_atomics_scifi,
                dev_scifi_track_hit_number,
                dev_qop,
                dev_track,
                dev_scifi_track_ut_indices,
                dev_muon_hits,
                dev_features
            );

            hipMemcpy(host_features, dev_features, n_features * sizeof(float), hipMemcpyDeviceToHost);

            THEN(
                "Extrapolation of track:    \n"
                "\t station 1 - (1000,-1.7) \n"
                "\t station 2 - (1001,-0.7) \n"
                "\t station 3 - (1002, 0.3) \n"
                "\t station 4 - (1003, 1.3) \n"
                "Closest hits: \n"
                "\t station 1 - ( 1,-1), index = 2  \n"
                "\t station 2 - ( 1,-1), index = 11 \n"
                "\t station 3 - ( 1, 0), index = 23 \n"
                "\t station 4 - ( 1, 1), index = 35 \n"
                "Traveled distance:      \n"
                "\t station 1 - 0        \n"
                "\t station 2 - sqrt(3)  \n"
                "\t station 3 - sqrt(12) \n"
                "\t station 4 - sqrt(27) \n"
            ) {
                const std::vector<int> closest_hits = {2, 11, 23, 35};
                const std::vector<float> extrapolation_x = {1000, 1001, 1002, 1003};
                const std::vector<float> extrapolation_y = {-1.7, -0.7, 0.3, 1.3};
                const std::vector<float> trav_dist = {0, sqrt(3.0f), sqrt(12.0f), sqrt(27.0f)};
                for (int i_station = 0; i_station < Muon::Constants::n_stations; i_station++) {
                    const int closest_idx = closest_hits[i_station];
                    const float multiple_scattering_error = COMMON_FACTOR * trav_dist[i_station] * sqrt(trav_dist[i_station]);
                    CHECK(host_features[offset::DTS + i_station] == muon_hits_events[0].delta_time[closest_idx]);
                    CHECK(host_features[offset::TIMES + i_station] == muon_hits_events[0].time[closest_idx]);
                    CHECK(host_features[offset::CROSS + i_station] + muon_hits_events[0].uncrossed[closest_idx] == 2);
                    CHECK_THAT(
                        host_features[offset::RES_X + i_station], Catch::Matchers::WithinAbs((extrapolation_x[i_station] - muon_hits_events[0].x[closest_idx]) / 
                        sqrt(closest_idx * closest_idx * INVSQRT3 * INVSQRT3 + multiple_scattering_error * multiple_scattering_error), eps)
                    );
                    CHECK_THAT(
                        host_features[offset::RES_Y + i_station], Catch::Matchers::WithinAbs((extrapolation_y[i_station] - muon_hits_events[0].y[closest_idx]) / 
                        sqrt(4 * closest_idx * closest_idx * INVSQRT3 * INVSQRT3 + multiple_scattering_error * multiple_scattering_error), eps)
                    );
                }
            }
        }
        free(host_features);
        hipFree(dev_muon_hits);
    }
    dev_free_memory();
}