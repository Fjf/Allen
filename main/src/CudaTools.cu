#include "Tools.h"
#include "CudaCommon.h"

void reserve_pinned(void** buffer, size_t size) { cudaCheck(hipHostMalloc(buffer, size)); }

#ifdef CPU

#include <fstream>
#include <regex>
#include <ext/stdio_filebuf.h>

void reset() {}
void print_gpu_memory_consumption() {}

std::tuple<bool, std::string> set_device(int, size_t)
{
  // Assume a linux system and try to get the CPU type
  FILE* cmd =
    popen("cat /proc/cpuinfo | grep 'model name' | head -n1 | awk '{ print substr($0, index($0,$4)) }'", "r");
  if (cmd == NULL) return {true, "CPU"};

  // Get a string that identifies the CPU
  const int fd = fileno(cmd);
  __gnu_cxx::stdio_filebuf<char> filebuf {fd, std::ios::in};
  std::istream cmd_ifstream {&filebuf};
  std::string processor_name {(std::istreambuf_iterator<char>(cmd_ifstream)), (std::istreambuf_iterator<char>())};

  // Clean the string
  const std::regex regex_to_remove {"(\\(R\\))|(CPU )|( @.*)"};
  processor_name = std::regex_replace(processor_name, regex_to_remove, std::string{});

  return {true, processor_name};
}

#else

void reset() { cudaCheck(hipDeviceReset()); }

/**
 * @brief Prints the memory consumption of the device.
 */
void print_gpu_memory_consumption()
{
  size_t free_byte;
  size_t total_byte;
  cudaCheck(hipMemGetInfo(&free_byte, &total_byte));
  float free_percent = (float) free_byte / total_byte * 100;
  float used_percent = (float) (total_byte - free_byte) / total_byte * 100;
  verbose_cout << "GPU memory: " << free_percent << " percent free, " << used_percent << " percent used " << std::endl;
}

std::tuple<bool, std::string> set_device(int cuda_device, size_t stream_id)
{
  int n_devices = 0;
  hipDeviceProp_t device_properties;

  try {
    cudaCheck(hipGetDeviceCount(&n_devices));

    debug_cout << "There are " << n_devices << " CUDA devices available\n";
    for (int cd = 0; cd < n_devices; ++cd) {
      hipDeviceProp_t device_properties;
      cudaCheck(hipGetDeviceProperties(&device_properties, cd));
      debug_cout << std::setw(3) << cd << " " << device_properties.name << "\n";
    }

    if (cuda_device >= n_devices) {
      error_cout << "Chosen device (" << cuda_device << ") is not available.\n";
      return {false, ""};
    }
    debug_cout << "\n";

    cudaCheck(hipSetDevice(cuda_device));
    cudaCheck(hipGetDeviceProperties(&device_properties, cuda_device));

    if (n_devices == 0) {
      error_cout << "Failed to select device " << cuda_device << "\n";
      return {false, ""};
    }
    else {
      debug_cout << "Stream " << stream_id << " selected cuda device " << cuda_device << ": " << device_properties.name
                 << "\n\n";
    }
  } catch (const std::invalid_argument& e) {
    error_cout << e.what() << std::endl;
    error_cout << "Stream " << stream_id << " failed to select cuda device " << cuda_device << "\n";
    return {false, ""};
  }

  return {true, device_properties.name};
}

#endif
