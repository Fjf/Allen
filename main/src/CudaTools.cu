#include "Tools.h"
#include "CudaCommon.h"

void reset() {
  cudaCheck(hipDeviceReset());
}

void reserve_pinned(void** buffer, size_t size) {
  cudaCheck(hipHostMalloc(buffer, size));
}
