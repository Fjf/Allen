#include "Tools.h"
#include "CudaCommon.h"

void reset() { cudaCheck(hipDeviceReset()); }

void reserve_pinned(void** buffer, size_t size) { cudaCheck(hipHostMalloc(buffer, size)); }

/**
 * @brief Prints the memory consumption of the device.
 */
void print_gpu_memory_consumption()
{
  size_t free_byte;
  size_t total_byte;
  cudaCheck(hipMemGetInfo(&free_byte, &total_byte));
  float free_percent = (float) free_byte / total_byte * 100;
  float used_percent = (float) (total_byte - free_byte) / total_byte * 100;
  verbose_cout << "GPU memory: " << free_percent << " percent free, " << used_percent << " percent used " << std::endl;
}

std::pair<size_t, std::string> set_device(int cuda_device)
{
  int n_devices = 0;
  hipDeviceProp_t device_properties;
  cudaCheck(hipGetDeviceCount(&n_devices));

  debug_cout << "There are " << n_devices << " CUDA devices available" << std::endl;
  for (int cd = 0; cd < n_devices; ++cd) {
    hipDeviceProp_t device_properties;
    cudaCheck(hipGetDeviceProperties(&device_properties, cd));
    debug_cout << std::setw(3) << cd << " " << device_properties.name << std::endl;
  }

  if (cuda_device >= n_devices) {
    error_cout << "Chosen device (" << cuda_device << ") is not available." << std::endl;
    return {0, ""};
  }
  debug_cout << std::endl;

  cudaCheck(hipSetDevice(cuda_device));
  cudaCheck(hipGetDeviceProperties(&device_properties, cuda_device));
  return {n_devices, device_properties.name};
}
