#include "hip/hip_runtime.h"
#include "CudaCommon.h"

#ifdef CPU

#include <cstring>
#include "stdlib.h"

thread_local GridDimensions gridDim;
thread_local BlockIndices blockIdx;

dim3::dim3(const unsigned int& x) : x(x) {}
dim3::dim3(const unsigned int& x, const unsigned int& y) : x(x), y(y) {}
dim3::dim3(const unsigned int& x, const unsigned int& y, const unsigned int& z) : x(x), y(y), z(z) {}

hipError_t hipMalloc(void** devPtr, size_t size)
{
  posix_memalign(devPtr, 64, size);
  return 0;
}

hipError_t hipHostMalloc(void** ptr, size_t size)
{
  posix_memalign(ptr, 64, size);
  return 0;
}

hipError_t hipMemcpy(void* dst, const void* src, size_t count, enum hipMemcpyKind)
{
  std::memcpy(dst, src, count);
  return 0;
}

hipError_t hipMemcpyAsync(void* dst, const void* src, size_t count, enum hipMemcpyKind, hipStream_t)
{
  std::memcpy(dst, src, count);
  return 0;
}

hipError_t hipMemset(void* devPtr, int value, size_t count)
{
  std::memset(devPtr, value, count);
  return 0;
}

hipError_t hipMemsetAsync(void* devPtr, int value, size_t count, hipStream_t)
{
  std::memset(devPtr, value, count);
  return 0;
}

hipError_t hipPeekAtLastError() { return 0; }

hipError_t hipEventCreate(hipEvent_t*) { return 0; }

hipError_t hipEventCreateWithFlags(hipEvent_t*, int) { return 0; }

hipError_t hipEventSynchronize(hipEvent_t) { return 0; }

hipError_t hipEventRecord(hipEvent_t, hipStream_t) { return 0; }

hipError_t hipHostFree(void* ptr)
{
  free(ptr);
  return 0;
}

hipError_t hipFree(void* ptr)
{
  free(ptr);
  return 0;
}

hipError_t hipDeviceReset() { return 0; }

hipError_t hipStreamCreate(hipStream_t*) { return 0; }

hipError_t hipMemcpyToSymbol(HIP_SYMBOL(void* symbol), const void* src, size_t count, size_t offset, enum hipMemcpyKind)
{
  std::memcpy(symbol, reinterpret_cast<const char*>(src) + offset, count);
  return 0;
}

unsigned int atomicInc(unsigned int* address, unsigned int val)
{
  unsigned int old = *address;
  *address = ((old >= val) ? 0 : (old + 1));
  return old;
}

namespace Configuration {
  uint verbosity_level;
}

#endif

#if defined(CPU) || defined(HIP)

__device__ __host__ int32_t intbits(const float f)
{
  int32_t i;
  std::memcpy(&i, &f, sizeof(float));
  return i;
}

__device__ __host__ float floatbits(const int32_t i)
{
  float f;
  std::memcpy(&f, &i, sizeof(float));
  return f;
}

half_t::half_t(const float f) {
  // via Fabian "ryg" Giesen.
  // https://gist.github.com/2156668
  uint32_t sign_mask = 0x80000000u;
  int32_t o;

  int32_t fint = intbits(f);
  int32_t sign = fint & sign_mask;
  fint ^= sign;

  // NOTE all the integer compares in this function can be safely
  // compiled into signed compares since all operands are below
  // 0x80000000. Important if you want fast straight SSE2 code (since
  // there's no unsigned PCMPGTD).

  // Inf or NaN (all exponent bits set)
  // NaN->qNaN and Inf->Inf
  // unconditional assignment here, will override with right value for
  // the regular case below.
  int32_t f32infty = 255ul << 23;
  o = (fint > f32infty) ? 0x7e00u : 0x7c00u;

  // (De)normalized number or zero
  // update fint unconditionally to save the blending; we don't need it
  // anymore for the Inf/NaN case anyway.

  // const uint32_t round_mask = ~0xffful;
  const uint32_t round_mask = ~0xfffu;
  const int32_t magic = 15ul << 23;
  const int32_t f16infty = 31ul << 23;

  int32_t fint2 = intbits(floatbits(fint & round_mask) * floatbits(magic)) - round_mask;
  fint2 = (fint2 > f16infty) ? f16infty : fint2; // Clamp to signed infinity if overflowed

  if (fint < f32infty) o = fint2 >> 13; // Take the bits!

  m_value = (o | (sign >> 16));
}

half_t::operator float() const {
  constexpr uint32_t shifted_exp = 0x7c00 << 13; // exponent mask after shift

  int32_t o = ((int32_t)(m_value & 0x7fff)) << 13; // exponent/mantissa bits
  uint32_t exp = shifted_exp & o;            // just the exponent
  o += (127 - 15) << 23;                     // exponent adjust

  // handle exponent special cases
  if (exp == shifted_exp)                                   // Inf/NaN?
    o += (128 - 16) << 23;                                  // extra exp adjust
  else if (exp == 0) {                                      // Zero/Denormal?
    o += 1 << 23;                                           // extra exp adjust
    o = intbits(floatbits(o) - floatbits(113 << 23)); // renormalize
  }

  o |= ((int32_t)(m_value & 0x8000)) << 16; // sign bit
  return floatbits(o);
}

int16_t half_t::get() const { return m_value; }

#endif
